#include "hip/hip_runtime.h"

// Version 0.51

// Been over some attempts to ameliorate local accesses -- not v successful basically.
// Correction in "Get Lap phi" routine.

// Version 0.52:

// Change Lap A, Grad A routines to load CHAR4 p_tri_per_neigh instead of loading data
// to interrogate neighbour periodic status.

// Change major area calc in the INNERMOST/OUTERMOST case.

// Note that central area calc does not look right.

// Version 0.53:

// Made changes to Reladvect_nT because it was taking wrong connection for OUTERMOST.
// Changed flag tests & treatment of Inner verts in preceding routines.

// Version 0.54:

// Adjusted area calculations as written in spec. 
// We set ins crossing tri minor area = 0, centroid on ins;
// frill area = 0, centroid on boundary.

// Version 0.6:

// Debugging and making corrections.



// PLAN: 
// Allow that on GPU we can move outside domain and it's fine, we do not change PB data.
// PB data will be only changed on CPU.
// Nonetheless we kept PBCTri lists which can be updated, unlike has_periodic alone, in case
// of moving something to its image within the domain.



// NOTES:

		// Ensure that outside the domain, n_major is recorded as 0

		// Ensure that outside the domain, resistive_heat is recorded as 0

// Notes as of 11/04/17:

// Getting two problems basically according to ptxas --v.

// Both Lap_A and Midpt have got (1252, 2500); (1000,2332) spill (stores,loads).
// Both of them will try to run 512 threads/SM which is what you get with 63 registers used.
// The registers represent ~~ 128KB / SM or 32 doubles/thread.
// The extra for L1 would be, for midpt, 48KB => 12 doubles.
// For Lap_A we do not even get that.
// But we have spill stores of e.g. 1024 = 128 doubles extra.

// We basically want to halve the amount of stored data for midpt.
// We went to some lengths to AVOID global random read/writes and now
// we are looking at reading 256 DOUBLES PER KERNEL RUN.
// PROCESS THAT !! So what can be done???

// For midpt there is scope to use more shared memory but it will clearly
// only represent 4 doubles extra per thread in total.

// We could reduce to 256 threads running at once and this might well be faster.
// Then we get L1 with 24 doubles for midpt. -- more likely there to be worth having.

// For Lap_A routine some kind of drastic action is called for.

// Notes as of 11/04/17.


#include <math.h>
#include <time.h>
#include <stdio.h>

#include "flags.h"

#define OUTPUT 1

// Note that this file has to first be compiled with nvcc
// Then with -dlink, apply nvcc to the obj file to produce another obj file;
// Include both obj files and cudart.lib in the main project.

// -dlink command line:

// E:\focusfusion\FFxtubes\cudaproj\x64\Release>
// nvcc -dlink -gencode=arch=compute_20,code=\"sm_20,compute_20\" --machine 64 -ccbin "E:\Program Files (x86)\Microsoft Visual Studio 9.0\VC\bin" newwrapper2.cu.obj -o newwrapper2.obj

// E:\focusfusion\FFxtubes\cudaproj\x64\Debug>
//nvcc -dlink -gencode=arch=compute_20,code=\"sm_20,compute_20\" --machine 64 -Xcompiler "/EHsc /W3 /nologo /O2 /Zi /MTd "  -ccbin "E:\Program Files (x86)\Microsoft Visual Studio 9.0\VC\bin" newkernel2.cu.obj -o newkernel2.obj

#define SIXTH 0.166666666666667
#define TWELTH 0.083333333333333
#define FIVETWELTHS 0.416666666666667
#define REPORT 69500
#define DEVICE_INSULATOR_OUTER_RADIUS 3.44

#include "cuda_struct.h"

#define CallMAC(cudaStatus) Call(cudaStatus, #cudaStatus )   
						// { Call(cudaStatus, "cudaStatus") } ?
extern real FRILL_CENTROID_OUTER_RADIUS, FRILL_CENTROID_INNER_RADIUS;

//long BLOCK_START_OF_FRILL_SEARCH = 256;
// Do 288-256 = 32 blocks -- assuming 256*288 tris
// This won't get the inner frills!!!!


// Global host helper:
__host__ bool Call(hipError_t cudaStatus,char str[]);

#include "systdata.cpp"

char * report_time_(int action)
{
	/* action = 0: reset ; action = 1: report */
	/* requires timebuffy to be defined as char[255] globally */

	static char timebuffer[255];
	static clock_t start;
	double timenow;
	long ops;

	if (action == 0) 
		{
		start = clock();
		}
	else
		{
		timenow = ((double)(clock()-start)/(double)CLOCKS_PER_SEC);
		ops = (long)(clock()-start);
		/* create a null-terminated string */
		sprintf (timebuffer, "%6.4f sec.",timenow);
		};
	return &(timebuffer[0]);	
};


// Global helper functions for kernels:

__device__ __forceinline__ f64 Get_lnLambda_ion_d(f64 n_ion,f64 T_ion);
__device__ __forceinline__ f64 Get_lnLambda_d(real n_e,real T_e);
__device__ __forceinline__ f64_vec2 Anticlock_rotate2(const f64_vec2 arg);
__device__ __forceinline__ f64_vec2 Clockwise_rotate2(const f64_vec2 arg);
__device__ __forceinline__ f64_vec3 Anticlock_rotate3(const f64_vec3 arg);
__device__ __forceinline__ f64 Estimate_Neutral_MT_Cross_section(f64 T);
__device__ __forceinline__ f64 Estimate_Neutral_Neutral_Viscosity_Cross_section(f64 T) ;
__device__ __forceinline__ f64 Estimate_Ion_Neutral_Viscosity_Cross_section(f64 T);
__device__ __forceinline__ f64 Calculate_Kappa_Neutral(f64 n_i, f64 T_i, f64 n_n, f64 T_n);


#ifdef __HIPCC__
__device__ __forceinline__ f64 GetEzShape(f64 r) {
	return 1.0-1.0/(1.0+exp(-16.0*(r-4.2))); // At 4.0cm it is 96% as strong as at tooth. At 4.4 it is 4%.
}
#else
f64 inline GetEzShape_(f64 r) {
	return 1.0-1.0/(1.0+exp(-16.0*(r-4.2))); // At 4.0cm it is 96% as strong as at tooth. At 4.4 it is 4%.
}
#endif



// Device-accessible constants not known at compile time:
__constant__ long nBlocks, Nverts, uDataLen_d; // Nverts == numVertices

__constant__ f64_tens2 Anticlockwise2, Clockwise2; // use this to do rotation.

// Set from host constant definitions:
__constant__ f64 sC, kB, c,Z, e,q,m_e, m_ion, m_n,
				 eoverm, qoverM, moverM, eovermc, qoverMc, 
				 FOURPI_Q_OVER_C, FOURPI_Q, FOURPI_OVER_C,
				 NU_EI_FACTOR, // Note: NU_EI_FACTOR goes with T in eV -- !!
				 nu_eiBarconst, csq, m_s, 
				  // New:
				 FOUR_PI;
//__constant__ long BLOCK_START_OF_FRILL_SEARCH_d;

__constant__ f64 cross_s_vals_viscosity_ni_d[10], cross_s_vals_viscosity_nn_d[10],
				 cross_T_vals_d[10], cross_s_vals_MT_ni_d[10];

// Set from calculations in host routine:
__constant__ f64 Nu_ii_Factor, kB_to_3halves, 
				 one_over_kB, one_over_kB_cubed,
				over_sqrt_m_ion,over_sqrt_m_e,over_sqrt_m_neutral;

// Other:
__constant__ f64 T_ion_avg_sq_d, T_neut_avg_sq_d, T_elec_avg_sq_d,
				v_ion_avg_sq_d,v_neut_avg_sq_d,
				MAXERRPPNSQ_d, AVGFAC_d, ABSTHRESHFLUX_SQ_d,ENDPT_MAXERRPPN_SQ_d,
				avgTe,avgTi,avgTn;

//__constant__ long ReverseJzIndexStart, ReverseJzIndexEnd; // MaxNeigh_d
// use #define MAXNEIGH_d but we will have to allow that there is a different
// maximum used for arrays loaded-in than for actual max in list.
// Could be 10 vs 20.

__constant__ f64 four_pi_over_c_ReverseJz;

__constant__ f64 FRILL_CENTROID_OUTER_RADIUS_d, 
				FRILL_CENTROID_INNER_RADIUS_d;

__device__ real * p_summands, * p_Iz0_summands, * p_Iz0_initial,
				* p_scratch_d, 
				* p_resistive_heat_neut_minor,
				* p_resistive_heat_ion_minor,
				* p_resistive_heat_elec_minor,
				* p_Lapphi;
__device__ f64_vec2 * p_grad_phidot;
__device__ f64_vec3 * p_MAR_neut, * p_MAR_ion, * p_MAR_elec;
__device__ nn *p_nn_ionrec_minor;

#include "E:/focusfusion/FFxtubes/helpers.cu"

#define Set_f64_constant(dest, src) { \
		Call(hipGetSymbolAddress((void **)(&f64address), dest ), \
			"hipGetSymbolAddress((void **)(&f64address), dest )");\
		Call(hipMemcpy( f64address, &src, sizeof(f64),hipMemcpyHostToDevice),\
			"hipMemcpy( f64address, &src, sizeof(f64),hipMemcpyHostToDevice) src dest");\
						}

Systdata Syst1, Systhalf, Syst2, SystAdv;

__host__ bool Call(hipError_t cudaStatus,char str[])
{
	if (cudaStatus == hipSuccess) return false;	
	printf("Error: %s\nReturned %d : %s\n",
		str, cudaStatus,hipGetErrorString(cudaStatus));
	printf("Anykey.\n");	getch();
	return true;
}

real GetIzPrescribed(real const t)
{
	real Iz = -PEAKCURRENT_STATCOULOMB * sin ((t + ZCURRENTBASETIME) * PIOVERPEAKTIME );
	//printf("\nGetIzPrescribed : t + ZCURRENTBASETIME = %1.5E : %1.12E\n", t + ZCURRENTBASETIME, Iz);
	return Iz;
}

// Do we want to create a 1:1 link between major tiles and minor tiles, or do we want
// a border on each. ...
/*__device__ void __forceinline__ atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                                          (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, 
                        __double_as_longlong(val + 
                        __longlong_as_double(assumed)));
    } while (assumed != old);
    // return __longlong_as_double(old);
}*/

#include "newkernel2.cu"


nn * p_nn_host;
f64_vec3 * p_MAR_ion_host, * p_MAR_neut_host, * p_MAR_elec_host;

void Systdata::AsciiOutput (const char filename[]) const 
{
	FILE * file = fopen(filename,"w");
	if (file == 0) {
		printf("could not open %s",filename);
		getch();
		getch();
		return;
	} ;
	printf("%s opened",filename);
	
	fprintf(file,"index flag | n_neut T_neut n_ion T_ion n_elec T_elec | ionise recombine | "
		"Bx By Bz | vnx vny vnz vix viy viz vex vey vez | "
		"gradphi_x gradphi_y Lap_A_x Lap_A_y Lap_A_z Az Adot_x Adot_y Adot_z | X1_Adot_z | "
		"MAR_neutx MAR_neuty MAR_neutz MAR_ionx MAR_iony MAR_ionz MAR_elecx MAR_elecy MAR_elecz | "
		"GradTe_x GradTe_y phi \n");
	
	for (int iMinor = 0; iMinor < this->Nminor; iMinor++)
	{
		f64 temp1;
	//	hipMemcpy(&temp1, pX1->p_Adot+iMinor, 
	//	sizeof(f64_vec3),			hipMemcpyDeviceToHost);
		
		if (iMinor % 500 == 0) printf("%d ",iMinor);
		
		int flag;
		if (iMinor < BEGINNING_OF_CENTRAL) {
			flag = this->p_tri_perinfo[iMinor].flag;
		} else {
			flag = this->p_info[iMinor-BEGINNING_OF_CENTRAL].flag;
		};

		fprintf(file,"%d %d | %1.14E %1.14E %1.14E %1.14E %1.14E %1.14E |  %1.14E %1.14E | "
			" %1.14E %1.14E %1.14E | %1.14E %1.14E %1.14E %1.14E %1.14E %1.14E %1.14E %1.14E %1.14E | ",
			iMinor, flag,
			this->p_nT_neut_minor[iMinor].n,this->p_nT_neut_minor[iMinor].T,
			this->p_nT_ion_minor[iMinor].n,this->p_nT_ion_minor[iMinor].T,
			this->p_nT_elec_minor[iMinor].n,this->p_nT_elec_minor[iMinor].T,
			p_nn_host[iMinor].n_ionise, p_nn_host[iMinor].n_recombine,
			this->p_B[iMinor].x,this->p_B[iMinor].y,this->p_B[iMinor].z,
			this->p_v_neut[iMinor].x,this->p_v_neut[iMinor].y,this->p_v_neut[iMinor].z,
			this->p_v_ion[iMinor].x,this->p_v_ion[iMinor].y,this->p_v_ion[iMinor].z,
			this->p_v_elec[iMinor].x,this->p_v_elec[iMinor].y,this->p_v_elec[iMinor].z
			);
			
		fprintf(file,	
			" %1.14E %1.14E %1.14E %1.14E %1.14E %1.14E %1.14E %1.14E %1.14E | %1.14E | ",
			this->p_grad_phi[iMinor].x,this->p_grad_phi[iMinor].y,
			this->p_Lap_A[iMinor].x,this->p_Lap_A[iMinor].y,this->p_Lap_A[iMinor].z,
			this->p_A[iMinor].z,
			this->p_Adot[iMinor].x,this->p_Adot[iMinor].y,this->p_Adot[iMinor].z,
			0.0);
			//temp1);
		
		fprintf(file,	
			" %1.14E %1.14E %1.14E ",
			p_MAR_neut_host[iMinor].x,p_MAR_neut_host[iMinor].y,p_MAR_neut_host[iMinor].z);
		
		fprintf(file,	
			" %1.14E %1.14E %1.14E ",
			p_MAR_ion_host[iMinor].x,p_MAR_ion_host[iMinor].y,p_MAR_ion_host[iMinor].z);
		
		fprintf(file,	
			" %1.14E %1.14E %1.14E ",
			p_MAR_elec_host[iMinor].x,p_MAR_elec_host[iMinor].y,p_MAR_elec_host[iMinor].z);
		
		fprintf(file,		" %1.14E %1.14E ",
			this->p_GradTe[iMinor].x,this->p_GradTe[iMinor].y);
		
		if (iMinor < BEGINNING_OF_CENTRAL) {
			fprintf(file," %1.10E %1.10E ",this->p_tri_centroid[iMinor].x,this->p_tri_centroid[iMinor].y);
		} else {
			fprintf(file," %1.10E | %1.10E %1.10E ",
				this->p_phi[iMinor-BEGINNING_OF_CENTRAL],
				this->p_info[iMinor-BEGINNING_OF_CENTRAL].pos.x,
				this->p_info[iMinor-BEGINNING_OF_CENTRAL].pos.y);
		};
		fprintf(file,"\n");
	};
	fclose(file);
	
}


void SendToHost(const Systdata * pX_nvT, const Systdata * pXhalf, const Systdata * pX_host)
{
	hipMemcpy(pX_host->p_phi,					pXhalf->p_phi,
		sizeof(f64)*pX_host->Nverts,			hipMemcpyDeviceToHost);
	hipMemcpy(pX_host->p_nT_neut_minor,		pX_nvT->p_nT_neut_minor,
		sizeof(nT)*pX_host->Nminor,			hipMemcpyDeviceToHost);
	hipMemcpy(pX_host->p_nT_ion_minor,			pX_nvT->p_nT_ion_minor,
		sizeof(nT)*pX_host->Nminor,			hipMemcpyDeviceToHost);
	hipMemcpy(pX_host->p_nT_elec_minor,		pX_nvT->p_nT_elec_minor,
		sizeof(nT)*pX_host->Nminor,			hipMemcpyDeviceToHost);

	hipMemcpy(pX_host->p_tri_centroid,		pXhalf->p_tri_centroid,
		sizeof(f64_vec2)*pX_host->Ntris,			hipMemcpyDeviceToHost);
	hipMemcpy(pX_host->p_B,				pXhalf->p_B,
		sizeof(f64_vec3)*pX_host->Nminor,			hipMemcpyDeviceToHost);

	hipMemcpy(pX_host->p_v_neut,			pX_nvT->p_v_neut,
		sizeof(f64_vec3)*pX_host->Nminor,			hipMemcpyDeviceToHost);
	hipMemcpy(pX_host->p_v_ion,			pX_nvT->p_v_ion,
		sizeof(f64_vec3)*pX_host->Nminor,			hipMemcpyDeviceToHost);
	hipMemcpy(pX_host->p_v_elec,			pX_nvT->p_v_elec,
		sizeof(f64_vec3)*pX_host->Nminor,			hipMemcpyDeviceToHost);

	hipMemcpy(pX_host->p_grad_phi,			pXhalf->p_grad_phi,
		sizeof(f64_vec2)*pX_host->Nminor,			hipMemcpyDeviceToHost);
	hipMemcpy(pX_host->p_Lap_A,			pXhalf->p_Lap_A,
		sizeof(f64_vec3)*pX_host->Nminor,			hipMemcpyDeviceToHost);
	hipMemcpy(pX_host->p_Adot,				pXhalf->p_Adot,
		sizeof(f64_vec3)*pX_host->Nminor,			hipMemcpyDeviceToHost);

	hipMemcpy(pX_host->p_GradTe,			pX_nvT->p_GradTe,
		sizeof(f64_vec2)*pX_host->Nminor,			hipMemcpyDeviceToHost);

	hipMemcpy(p_nn_host,					p_nn_ionrec_minor,
		sizeof(nn)*pX_host->Nminor,					hipMemcpyDeviceToHost);
	hipMemcpy(p_MAR_neut_host,				p_MAR_neut,
		sizeof(f64_vec3)*pX_host->Nminor,			hipMemcpyDeviceToHost);
	hipMemcpy(p_MAR_ion_host,				p_MAR_ion,
		sizeof(f64_vec3)*pX_host->Nminor,			hipMemcpyDeviceToHost);
	hipMemcpy(p_MAR_elec_host,				p_MAR_elec,
		sizeof(f64_vec3)*pX_host->Nminor,			hipMemcpyDeviceToHost);
	
	Call(hipDeviceSynchronize(),"hipDeviceSynchronize memcpies S2H");

}

void PerformCUDA_Advance_2 (
		const Systdata * pX_host, // populate in CPU MSVC routine...
		long numVerts,
		const real hsub, 
		const int numSubsteps,
		const Systdata * pX_host_target
		)
{
	// Preliminaries:
	
	char buffer[256];
	FILE * fpdebug;
	
	long iVertex;
	Systdata * pX1,*pX2,*pXhalf,*pXusable;
	real * p_summands_host, * p_Iz0_summands_host, *p_scratch, *p_Iz0_initial_host, * p_scratch_host;
	structural * p_scratch_info;
	int iSubsubstep;
	size_t uFree, uTotal;	
	long numVertices = numVerts;
	real const hstep = hsub/(real)numSubsteps;
	
	real evaltime = pX_host->evaltime;
	real t = evaltime;

	printf("pXhost->p_Adot[20000 + BEGINNING_OF_CENTRAL].z %1.10E\n",pX_host->p_Adot[20000 + BEGINNING_OF_CENTRAL].z);

	printf("sizeof(CHAR4): %d \n"
		"sizeof(structural): %d \n"
		"sizeof(LONG3): %d \n"
		"sizeof(nn): %d \n",
		   sizeof(CHAR4),sizeof(structural),sizeof(LONG3),sizeof(nn));
	getch();


	if (Syst1.bInvoked == false) {
		
		Call(hipMemGetInfo (&uFree,&uTotal),"hipMemGetInfo (&uFree,&uTotal)");
		printf("Before Invokes: uFree %d uTotal %d\n",uFree,uTotal);
		
		Syst1.Invoke(numVertices);
		Systhalf.Invoke(numVertices);
		Syst2.Invoke(numVertices);
		SystAdv.Invoke(numVertices);
		
		Call(hipMemGetInfo (&uFree,&uTotal),"hipMemGetInfo (&uFree,&uTotal)");
		printf("After Invokes: uFree %d uTotal %d\n",uFree,uTotal);
	}
	
	// -----  What needs to change in all this stuff?  -----
	
	// Populate video constant memory:
	// ________________________________
	
	long * address;
	f64 * f64address;
	
	// not used? :

//	Call(hipGetSymbolAddress((void **)(&address),Nverts), 
//		"hipGetSymbolAddress((void **)(&address),Nverts)");
//	Call(hipMemcpy( address, &numVertices, sizeof(long),hipMemcpyHostToDevice),
//		"hipMemcpy( address, &numVertices, sizeof(long),hipMemcpyHostToDevice) 2 ");
// good
	
	//memcpy(IndexNeigh,pIndexNeigh + long_neighs_stride*index,long_neighs_stride);
	//memcpy(PBCNeigh,pPBCneigh + char_neighs_stride*index,char_neighs_stride);
	
	// Eventually change this :
	//long MaxNeigh = MAXNEIGH_d;
	//CallMAC(hipGetSymbolAddress((void **)(&address),MAXNEIGH_d));
	//CallMAC(hipMemcpy( address, &MaxNeigh, sizeof(long),hipMemcpyHostToDevice));
	
	// not used:
	//Call(hipGetSymbolAddress((void **)(&address),uDataLen_d), 
	//	"hipGetSymbolAddress((void **)(&address),uDataLen_d)");
	//Call(hipMemcpy( address, &numDataLen, sizeof(long),hipMemcpyHostToDevice),
	//	"hipMemcpy( address, &numDataLen, sizeof(long),hipMemcpyHostToDevice) 3 ");
	
	//Tensor2 const Anticlockwise(cos(FULLANGLE),-sin(FULLANGLE),sin(FULLANGLE),cos(FULLANGLE));
	f64_tens2 anticlock2;
	// Note that objects appearing in constant memory must have empty constructor & destructor.
	anticlock2.xx = cos(FULLANGLE);
	anticlock2.xy = -sin(FULLANGLE);
	anticlock2.yx = sin(FULLANGLE);
	anticlock2.yy = cos(FULLANGLE);
	
	Tensor2 * T2address;
	Call(hipGetSymbolAddress((void **)(&T2address),Anticlockwise2), 
		"hipGetSymbolAddress((void **)(&T2address),Anticlockwise2)");
	Call(hipMemcpy( T2address, &anticlock2, sizeof(f64_tens2),hipMemcpyHostToDevice),
		"hipMemcpy( T2address, &anticlock2, sizeof(f64_tens2),hipMemcpyHostToDevice) U");
	
	f64_tens2 clock2;
	// Note that objects appearing in constant memory must have empty constructor & destructor.
	clock2.xx = cos(FULLANGLE);
	clock2.xy = sin(FULLANGLE);
	clock2.yx = -sin(FULLANGLE);
	clock2.yy = cos(FULLANGLE);
	
	Call(hipGetSymbolAddress((void **)(&T2address),Clockwise2), 
		"hipGetSymbolAddress((void **)(&T2address),Clockwise2)");
	Call(hipMemcpy( T2address, &clock2, sizeof(f64_tens2),hipMemcpyHostToDevice),
		"hipMemcpy( T2address, &clock2, sizeof(f64_tens2),hipMemcpyHostToDevice) U");
	
	//CallMAC(hipGetSymbolAddress((void **)(&address),ReverseJzIndexStart));
	//CallMAC(hipMemcpy( address, &numStartZCurrentRow, sizeof(long),hipMemcpyHostToDevice));
	//long past_end = numEndZCurrentRow+1;
	//CallMAC(hipGetSymbolAddress((void **)(&address),ReverseJzIndexEnd));
	//CallMAC(hipMemcpy( address, &past_end, sizeof(long),hipMemcpyHostToDevice));
	

	//// numEndZCurrentRow = numVertices-1; // the previous one.
	//// numStartZCurrentRow = numVertices-numRow[numRow1];
	
	// For floating point constants you have two choices:
	// 1. #define MAY be faster, but can only be used if no danger of
	// false match.
	// 2. __constant__. 
	// global const is not even supposed to work for integers.
	
	Set_f64_constant(FRILL_CENTROID_OUTER_RADIUS_d,pX_host->OutermostFrillCentroidRadius);
	Set_f64_constant(FRILL_CENTROID_INNER_RADIUS_d,pX_host->InnermostFrillCentroidRadius);
	Set_f64_constant(sC,sC_); // ever used?
	Set_f64_constant(kB,kB_);
	Set_f64_constant(c,c_); // ever used? likely not
	Set_f64_constant(Z,Z_);
	Set_f64_constant(e,e_);
	Set_f64_constant(q,q_);
	Set_f64_constant(m_e,m_e_);
	Set_f64_constant(m_ion,m_ion_);
	Set_f64_constant(m_n,m_n_);
	Set_f64_constant(eoverm, eoverm_);
	Set_f64_constant(qoverM, qoverM_);
	Set_f64_constant(moverM, moverM_);
	Set_f64_constant(eovermc, eovermc_);
	Set_f64_constant(qoverMc, qoverMc_);
	Set_f64_constant(FOURPI_Q_OVER_C, FOUR_PI_Q_OVER_C_);
	Set_f64_constant(FOURPI_Q, FOUR_PI_Q_);
	Set_f64_constant(FOURPI_OVER_C, FOURPI_OVER_C_);
	Set_f64_constant(NU_EI_FACTOR, NU_EI_FACTOR_);
	Set_f64_constant(nu_eiBarconst, nu_eiBarconst_);
	// Supposedly things will now be easier since device constants have the 
	// easiest labels.
	// Granted some of these could safely be #define.

	f64 temp;
	temp = 1.0/(sqrt(2.0)*2.09e7);
	Set_f64_constant(Nu_ii_Factor,temp);
	temp = sqrt(kB_*kB_*kB_);
	Set_f64_constant(kB_to_3halves, temp);
	temp = 1.0/kB_;
	Set_f64_constant(one_over_kB,temp);
	temp = temp*temp*temp;
	Set_f64_constant(one_over_kB_cubed,temp);
	temp = 1.0/sqrt(m_ion_);
	Set_f64_constant(over_sqrt_m_ion,temp);
	temp = 1.0/sqrt(m_e_);
	Set_f64_constant(over_sqrt_m_e,temp);
	temp = 1.0/sqrt(m_n_);
	Set_f64_constant(over_sqrt_m_neutral,temp);
	temp = c_*c_;
	Set_f64_constant(csq,temp);
	
	//Call(hipGetSymbolAddress((void **)(&address), BLOCK_START_OF_FRILL_SEARCH_d),
	//		"hipGetSymbolAddress((void **)(&address), dest )");
	//Call(hipMemcpy( address, &BLOCK_START_OF_FRILL_SEARCH, sizeof(long),hipMemcpyHostToDevice),
	//		"hipMemcpy( address, &BLOCK_START_OF_FRILL_SEARCH, sizeof(long),hipMemcpyHostToDevice) src dest");
		
//	Set_f64_constant(MAXERRPPNSQ_d, MAXERRPPNSQ);
//	Set_f64_constant(AVGFAC_d,AVGFAC);
//	Set_f64_constant(ABSTHRESHFLUX_SQ_d,ABSTHRESHFLUX_SQ);
//	Set_f64_constant(ENDPT_MAXERRPPN_SQ_d,ENDPT_MAXERRPPN_SQ);

	// These have to be set if doing that type of controlling the flux change.

	Call(hipMemcpyToSymbol(HIP_SYMBOL(cross_T_vals_d),cross_T_vals, 10*sizeof(f64)),
		"hipMemcpyToSymbol(HIP_SYMBOL(cross_T_vals_d),cross_T_vals, 10*sizeof(f64))");
	Call(hipMemcpyToSymbol(HIP_SYMBOL(cross_s_vals_viscosity_ni_d),cross_s_vals_viscosity_ni,
								10*sizeof(f64)),
		"hipMemcpyToSymbol(HIP_SYMBOL(cross_s_vals_viscosity_ni_d),cross_s_vals_viscosity_ni, \
		10*sizeof(f64))");
	Call(hipMemcpyToSymbol(HIP_SYMBOL(cross_s_vals_viscosity_nn_d),cross_s_vals_viscosity_nn,
								10*sizeof(f64)),
		"hipMemcpyToSymbol(HIP_SYMBOL(cross_s_vals_viscosity_nn_d),cross_s_vals_viscosity_nn, \
		10*sizeof(f64))");
	Call(hipMemcpyToSymbol(HIP_SYMBOL(cross_s_vals_MT_ni_d),cross_s_vals_momtrans_ni,
								10*sizeof(f64)),
		"hipMemcpyToSymbol(HIP_SYMBOL(cross_s_vals_MT_ni_d),cross_s_vals_momtrans_ni, \
		10*sizeof(f64))");
	
	// 1. More cudaMallocs for d/dt arrays and main data:
	// and aggregation arrays...
	
	CallMAC(hipMalloc((void **)&p_summands,numTilesMinor*sizeof(f64)));
	CallMAC(hipMalloc((void **)&p_Iz0_summands,numTilesMinor*sizeof(f64)));
	CallMAC(hipMalloc((void **)&p_Iz0_initial,numTilesMinor*sizeof(f64)));
	
	CallMAC(hipMalloc((void **)&p_scratch_d,numVertices*sizeof(f64)));
		// used for?
	CallMAC(hipMalloc((void **)&p_nn_ionrec_minor,Syst1.Nminor*sizeof(nn)));
	
	CallMAC(hipMalloc((void **)&p_resistive_heat_neut_minor,Syst1.Nminor*sizeof(f64)));
	CallMAC(hipMalloc((void **)&p_resistive_heat_ion_minor,Syst1.Nminor*sizeof(f64)));
	CallMAC(hipMalloc((void **)&p_resistive_heat_elec_minor,Syst1.Nminor*sizeof(f64)));
	
	CallMAC(hipMalloc((void **)&p_MAR_neut,Syst1.Nminor*sizeof(f64_vec3)));
	CallMAC(hipMalloc((void **)&p_MAR_ion,Syst1.Nminor*sizeof(f64_vec3)));
	CallMAC(hipMalloc((void **)&p_MAR_elec,Syst1.Nminor*sizeof(f64_vec3)));
	
	CallMAC(hipMalloc((void **)&p_Lapphi,Syst1.Nminor*sizeof(f64)));
	CallMAC(hipMalloc((void **)&p_grad_phidot,Syst1.Nminor*sizeof(f64_vec2)));
	
	p_summands_host = (f64 *)malloc(numTilesMinor*sizeof(f64));
	p_Iz0_summands_host = (f64 *)malloc(numTilesMinor*sizeof(f64));
	p_Iz0_initial_host = (f64 *)malloc(numTilesMinor*sizeof(f64));
	p_scratch = (f64 *)malloc((numVertices+1000)*sizeof(f64));
	p_scratch_info = (structural *)malloc((numVertices+1000)*sizeof(structural));
	p_scratch_host = (f64 *)malloc((pX_host->Nminor+1000)*sizeof(f64));

	p_nn_host = (nn *)malloc(pX_host->Nminor*sizeof(nn));
	p_MAR_neut_host = (f64_vec3 *)malloc(pX_host->Nminor*sizeof(f64_vec3));
	p_MAR_ion_host = (f64_vec3 *)malloc(pX_host->Nminor*sizeof(f64_vec3));
	p_MAR_elec_host = (f64_vec3 *)malloc(pX_host->Nminor*sizeof(f64_vec3));
	
	// 2. hipMemcpy system state from host: this happens always
	// __________________________________________________________
	
 	CallMAC(hipMemcpy(Syst1.p_phi, pX_host->p_phi, numVertices*sizeof(f64), hipMemcpyHostToDevice));
	CallMAC(hipMemcpy(Syst1.p_phidot, pX_host->p_phidot, numVertices*sizeof(f64), hipMemcpyHostToDevice));
	CallMAC(hipMemcpy(Syst1.p_A, pX_host->p_A, Syst1.Nminor*sizeof(f64_vec3), hipMemcpyHostToDevice));
	CallMAC(hipMemcpy(Syst1.p_Adot, pX_host->p_Adot, Syst1.Nminor*sizeof(f64_vec3), hipMemcpyHostToDevice));
	// Transfer initial B field!
	CallMAC(hipMemcpy(Syst1.p_B, pX_host->p_B, Syst1.Nminor*sizeof(f64), hipMemcpyHostToDevice));
	CallMAC(hipMemcpy(Syst1.p_area, pX_host->p_area, numVertices*sizeof(f64), hipMemcpyHostToDevice));
	CallMAC(hipMemcpy(Syst1.p_area_minor, pX_host->p_area_minor, Syst1.Nminor*sizeof(f64), hipMemcpyHostToDevice));
		
	Syst1.InnermostFrillCentroidRadius = pX_host->InnermostFrillCentroidRadius;
	Syst1.OutermostFrillCentroidRadius = pX_host->OutermostFrillCentroidRadius;
	Systhalf.InnermostFrillCentroidRadius = Syst1.InnermostFrillCentroidRadius;
	Systhalf.OutermostFrillCentroidRadius = Syst1.OutermostFrillCentroidRadius;
	Syst2.InnermostFrillCentroidRadius = Syst1.InnermostFrillCentroidRadius;
	Syst2.OutermostFrillCentroidRadius = Syst1.OutermostFrillCentroidRadius;
	SystAdv.InnermostFrillCentroidRadius = Syst1.InnermostFrillCentroidRadius;
	SystAdv.OutermostFrillCentroidRadius = Syst1.OutermostFrillCentroidRadius;

	Syst1.EzTuning = pX_host->EzTuning; // fail?
		
	printf("Syst1.Ez %1.9E pX_host Ez %1.9E \n",
		Syst1.EzTuning,pX_host->EzTuning);
	getch();

	CallMAC(hipMemcpy(Syst1.p_nT_neut_minor, pX_host->p_nT_neut_minor, Syst1.Nminor*sizeof(nT), hipMemcpyHostToDevice));
	CallMAC(hipMemcpy(Syst1.p_nT_ion_minor, pX_host->p_nT_ion_minor, Syst1.Nminor*sizeof(nT), hipMemcpyHostToDevice));
	CallMAC(hipMemcpy(Syst1.p_nT_elec_minor, pX_host->p_nT_elec_minor, Syst1.Nminor*sizeof(nT), hipMemcpyHostToDevice));
	
	CallMAC(hipMemcpy(Syst1.p_v_neut, pX_host->p_v_neut, Syst1.Nminor*sizeof(f64_vec3), hipMemcpyHostToDevice));
	CallMAC(hipMemcpy(Syst1.p_v_ion, pX_host->p_v_ion, Syst1.Nminor*sizeof(f64_vec3), hipMemcpyHostToDevice));
	CallMAC(hipMemcpy(Syst1.p_v_elec, pX_host->p_v_elec, Syst1.Nminor*sizeof(f64_vec3), hipMemcpyHostToDevice));
	
	CallMAC(hipMemcpy(Syst1.p_info, pX_host->p_info, numVertices*sizeof(structural), hipMemcpyHostToDevice));
	CallMAC(hipMemcpy(Syst1.p_tri_perinfo, pX_host->p_tri_perinfo,  Syst1.Nminor*sizeof(CHAR4), hipMemcpyHostToDevice));
	CallMAC(hipMemcpy(Syst1.p_tri_corner_index, pX_host->p_tri_corner_index, Syst1.Ntris*sizeof(LONG3), hipMemcpyHostToDevice));
	CallMAC(hipMemcpy(Syst1.p_tri_per_neigh, pX_host->p_tri_per_neigh, Syst1.Ntris*sizeof(CHAR4), hipMemcpyHostToDevice));
	
	CallMAC(hipMemcpy(Syst1.pIndexNeigh, pX_host->pIndexNeigh, numVertices*MAXNEIGH_d*sizeof(long),hipMemcpyHostToDevice));
	CallMAC(hipMemcpy(Syst1.pPBCneigh, pX_host->pPBCneigh, numVertices*MAXNEIGH_d*sizeof(char),hipMemcpyHostToDevice));
	
	CallMAC(hipMemcpy(Syst1.pIndexTri, pX_host->pIndexTri, numVertices*MAXNEIGH_d*sizeof(long), hipMemcpyHostToDevice));
	CallMAC(hipMemcpy(Syst1.pPBCtri, pX_host->pPBCtri, numVertices*MAXNEIGH_d*sizeof(char), hipMemcpyHostToDevice));
		
	CallMAC(hipMemcpy(Syst1.p_neigh_tri_index, pX_host->p_neigh_tri_index, Syst1.Ntris*sizeof(LONG3), hipMemcpyHostToDevice));
	
	// Now copy across to the other systems we initialized.

	CallMAC(hipMemcpy(Systhalf.p_info, Syst1.p_info, numVertices*sizeof(structural), hipMemcpyDeviceToDevice));
	CallMAC(hipMemcpy(Systhalf.p_tri_perinfo, Syst1.p_tri_perinfo, Syst1.Ntris*sizeof(CHAR4), hipMemcpyDeviceToDevice));
	CallMAC(hipMemcpy(Systhalf.p_tri_corner_index, Syst1.p_tri_corner_index, Syst1.Ntris*sizeof(LONG3), hipMemcpyDeviceToDevice));
	CallMAC(hipMemcpy(Systhalf.p_tri_per_neigh, Syst1.p_tri_per_neigh, Syst1.Ntris*sizeof(CHAR4), hipMemcpyDeviceToDevice));
	CallMAC(hipMemcpy(Systhalf.p_neigh_tri_index, Syst1.p_neigh_tri_index, Syst1.Ntris*sizeof(LONG3), hipMemcpyDeviceToDevice));
	
	CallMAC(hipMemcpy(Systhalf.pIndexTri, Syst1.pIndexTri, numVertices*MAXNEIGH_d*sizeof(long), hipMemcpyDeviceToDevice));
	CallMAC(hipMemcpy(Systhalf.pPBCtri, Syst1.pPBCtri, numVertices*MAXNEIGH_d*sizeof(char), hipMemcpyDeviceToDevice));
	CallMAC(hipMemcpy(Systhalf.pIndexNeigh, Syst1.pIndexNeigh,numVertices*MAXNEIGH_d*sizeof(long),hipMemcpyDeviceToDevice));
	CallMAC(hipMemcpy(Systhalf.pPBCneigh, Syst1.pPBCneigh,numVertices*MAXNEIGH_d*sizeof(char),hipMemcpyDeviceToDevice));
	
	// Of course, this is duplicated information for the whole cycle, which makes it clear
	// that we should just have 1 copy of this really.

	CallMAC(hipMemcpy(Syst2.p_info, Syst1.p_info, numVertices*sizeof(structural), hipMemcpyDeviceToDevice));
	CallMAC(hipMemcpy(Syst2.p_tri_perinfo, Syst1.p_tri_perinfo, Syst1.Ntris*sizeof(CHAR4), hipMemcpyDeviceToDevice));
	CallMAC(hipMemcpy(Syst2.p_tri_corner_index, Syst1.p_tri_corner_index, Syst1.Ntris*sizeof(LONG3), hipMemcpyDeviceToDevice));
	CallMAC(hipMemcpy(Syst2.p_tri_per_neigh, Syst1.p_tri_per_neigh, Syst1.Ntris*sizeof(CHAR4), hipMemcpyDeviceToDevice));
	CallMAC(hipMemcpy(Syst2.p_neigh_tri_index, Syst1.p_neigh_tri_index, Syst1.Ntris*sizeof(LONG3), hipMemcpyDeviceToDevice));
	
	CallMAC(hipMemcpy(Syst2.pIndexTri, Syst1.pIndexTri, numVertices*MAXNEIGH_d*sizeof(long), hipMemcpyDeviceToDevice));
	CallMAC(hipMemcpy(Syst2.pPBCtri, Syst1.pPBCtri, numVertices*MAXNEIGH_d*sizeof(char), hipMemcpyDeviceToDevice));
	CallMAC(hipMemcpy(Syst2.pIndexNeigh, Syst1.pIndexNeigh,numVertices*MAXNEIGH_d*sizeof(long),hipMemcpyDeviceToDevice));
	CallMAC(hipMemcpy(Syst2.pPBCneigh, Syst1.pPBCneigh,numVertices*MAXNEIGH_d*sizeof(char),hipMemcpyDeviceToDevice));
	
	CallMAC(hipMemcpy(SystAdv.p_info, Syst1.p_info, numVertices*sizeof(structural), hipMemcpyDeviceToDevice));
	CallMAC(hipMemcpy(SystAdv.p_tri_perinfo, Syst1.p_tri_perinfo, Syst1.Ntris*sizeof(CHAR4), hipMemcpyDeviceToDevice));
	CallMAC(hipMemcpy(SystAdv.p_tri_corner_index, Syst1.p_tri_corner_index, Syst1.Ntris*sizeof(LONG3), hipMemcpyDeviceToDevice));
	CallMAC(hipMemcpy(SystAdv.p_tri_per_neigh, Syst1.p_tri_per_neigh, Syst1.Ntris*sizeof(CHAR4), hipMemcpyDeviceToDevice));
	CallMAC(hipMemcpy(SystAdv.p_neigh_tri_index, Syst1.p_neigh_tri_index, Syst1.Ntris*sizeof(LONG3), hipMemcpyDeviceToDevice));
	
	CallMAC(hipMemcpy(SystAdv.pIndexTri, Syst1.pIndexTri, numVertices*MAXNEIGH_d*sizeof(long), hipMemcpyDeviceToDevice));
	CallMAC(hipMemcpy(SystAdv.pPBCtri, Syst1.pPBCtri, numVertices*MAXNEIGH_d*sizeof(char), hipMemcpyDeviceToDevice));
	CallMAC(hipMemcpy(SystAdv.pIndexNeigh, Syst1.pIndexNeigh,numVertices*MAXNEIGH_d*sizeof(long),hipMemcpyDeviceToDevice));
	CallMAC(hipMemcpy(SystAdv.pPBCneigh, Syst1.pPBCneigh,numVertices*MAXNEIGH_d*sizeof(char),hipMemcpyDeviceToDevice));
	
	// None of these are being modified during a CUDA run cycle.
	// So what we want is another class called something like "system_structure" with only 1 object existing.
	
	printf("Done main hipMemcpy to video memory.\n");
	
	// Let's test what we've been given:
	//FILE * fp = fopen("contribsIz.txt","w");
	f64 Iz0 = 0.0, Ne=0.0;
	for (long iVertex = 0; iVertex < numVertices; iVertex++)
	{
		if (pX_host->p_info[iVertex].flag == DOMAIN_VERTEX) 
		{
			Iz0 += q_*(pX_host->p_nT_ion_minor[iVertex + BEGINNING_OF_CENTRAL].n*pX_host->p_v_ion[iVertex + BEGINNING_OF_CENTRAL].z
			        - pX_host->p_nT_elec_minor[iVertex + BEGINNING_OF_CENTRAL].n*pX_host->p_v_elec[iVertex + BEGINNING_OF_CENTRAL].z)*
					pX_host->p_area[iVertex];
			
			Ne += pX_host->p_nT_elec_minor[iVertex + BEGINNING_OF_CENTRAL].n * pX_host->p_area[iVertex];
		};
	
		// save off the contribs:
		//	fprintf(fp,"%d %1.10E \n",iVertex,
		//		q_*(pX_host->p_nT_ion[iVertex].n*pX_host->p_v_ion[iVertex].z
		//	        - pX_host->p_nT_elec[iVertex].n*pX_host->p_v_elec[iVertex].z)*
		//			pX_host->p_area[iVertex]);
	};
	printf("pX_host Iz0 %1.12E Ne %1.8E IzPresc %1.12E \n",Iz0, Ne, GetIzPrescribed(t)); 
	//fclose(fp);
	
	pX1 = &Syst1;
	pXhalf = &Systhalf;
	pX2 = &Syst2;
	pXusable = &SystAdv;
	
	hipDeviceSetCacheConfig(hipFuncCachePreferShared);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(::Kernel_Ionisation),
							hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(::Kernel_Midpoint_v_and_Adot),
							hipFuncCachePreferL1);
	
	// For anything else that uses < 16kB shared, it always might help
	// to set to prefer L1.
	
	// Not sure if this will help speed or just prevent 32-bit allocation:
	//hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
				
	CallMAC(hipMemset(p_summands,0,sizeof(f64)*numTilesMinor));
	Kernel_GetZCurrent<<<numTilesMinor,threadsPerTileMinor>>>(
		pX1->p_tri_perinfo,
		pX1->p_nT_ion_minor,
		pX1->p_nT_elec_minor,
		pX1->p_v_ion,
		pX1->p_v_elec, // Not clear if this should be nv or {n,v} yet - think.
		pX1->p_area_minor,
		p_summands
		);
	Call(hipDeviceSynchronize(),"hipDeviceSynchronize GetZCurrent 1.");
	
	CallMAC(hipMemcpy(p_summands_host,p_summands,sizeof(f64)*numTilesMinor,hipMemcpyDeviceToHost));
	Iz0 = 0.0;
	for (int ii = 0; ii < numTilesMinor; ii++)
	{
		Iz0 += p_summands_host[ii];
	};	
	printf("Iz X1 before area calc %1.12E \n",Iz0); 
	
	// 3. Advance:	
	// For now do a version where the mesh motion is done every innermost step. 
	hipEvent_t start, stop;
	float elapsedTime;
	
	// Bring areas back to host, spit them out alongside previous:
	//CallMAC(hipMemcpy(p_scratch,pX1->p_area,sizeof(f64)*numVertices,hipMemcpyDeviceToHost));
	//FILE * fp = fopen("area_compare.txt","w");
	//for (iVertex = 0; iVertex < numVertices; iVertex++)
	//{
	//	fprintf(fp,"%d %d %1.14E %1.14E \n",
	//		iVertex, pX_host->p_info[iVertex].flag,
	//		pX_host->p_area[iVertex], p_scratch[iVertex]);
	//}
	//fclose(fp);
	//printf("Compared areas output...\n");
	
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
			
// _____________________________________________________________________________

	// Kernel calling code:

	// k1 ln 2.8 + k2 = -V
	// k1 ln 4.6 + k2 = V
	// 2V/(ln 4.6-ln 2.8) = k1
	
	f64 V = pX1->EzTuning*3.5; // 7cm assumed effective distance
	// EzTuning is what goes into E since EzShape ~= 1 near tooth. Check ?
	f64 k1 = 2.0*V/(log(4.6)-log(2.8));
	f64 k2 = V - k1*log(4.6);
	Kernel_InitialisePhi<<<numTilesMajor, threadsPerTileMajor>>>
		(
			pX1->p_info,
			k1,k2,
			V,
			pX1->p_phi
		);
	Call(hipDeviceSynchronize(),"hipDeviceSynchronize InitialisePhi.");
	
	//f64 tempf64;
	//hipMemcpy(&tempf64,pX1->p_phi+10000,sizeof(f64),hipMemcpyDeviceToHost);
	//printf("pX1->p_phi[10000] %1.9E k1 %1.5E k2 %1.5E\n"
	//	"==============================================\n",
	//	tempf64,k1,k2);
	// First thing is to see why this is zero, then see why #IND in Xhalf.
	
	
	Kernel_CalculateTriMinorAreas_AndCentroids<<<numTriTiles, threadsPerTileMinor>>>		
		(
			pX1->p_info,
			pX1->p_tri_corner_index,
			pX1->p_tri_perinfo,
		 // Output:
			pX1->p_area_minor,
			pX1->p_tri_centroid
		);
		//// Minor tiles need to carry information about what # of major values are used.
		//// And vice versa.
	Call(hipDeviceSynchronize(),"hipDeviceSynchronize CalculateTriMinorAreas 1.");
	
	if (hipPeekAtLastError() != hipSuccess) {
		printf("cudaPALEarea %s\n",hipGetErrorString(hipGetLastError()));
		getch();
	} else {
		printf("Kernel_CalculateTriMinorAreas_AndCentroids No error found,\n");
	}
	
	Kernel_CalculateCentralMinorAreas<<<numTilesMajor, threadsPerTileMajor>>>(
		 pX1->p_info,
		 pX1->pIndexTri, // lists of length 12
		 pX1->p_area_minor,
		 // Output:
		 pX1->p_area_minor + BEGINNING_OF_CENTRAL);
	Call(hipDeviceSynchronize(),"hipDeviceSynchronize CalculateCentralMinorAreas 1.");
	
	if (hipPeekAtLastError() != hipSuccess) {
		printf("cudaPALEarea %s\n",hipGetErrorString(hipGetLastError()));
		getch();
	} else {
		printf("Kernel_CalculateCentralMinorAreas No error found,\n");
	}
	
	//CallMAC(hipMemcpy(p_scratch_host,pX1->p_area_minor,sizeof(f64)*pX1->Nminor,hipMemcpyDeviceToHost));
	//f64 areasum = 0.0;
	//int iTest;
	//for (iTest = 0; iTest < pX1->Ntris; iTest++)
	//{
	//	if (p_scratch_host[iTest] < 0.0) {
	//		printf("iTest %d %1.5E \n",iTest,p_scratch_host[iTest]);
	//	};
	//	areasum += p_scratch_host[iTest];
	//}
	//printf("Areasum tris only %1.12E \n",areasum); // -2500.
	//for (; iTest < pX1->Nminor; iTest++)
	//{
	//	areasum += p_scratch_host[iTest];
	//}
	//printf("Areasum %1.12E \n",areasum);

	//CallMAC(hipMemcpy(p_scratch_host,pX1->p_area,sizeof(f64)*pX1->Nverts,hipMemcpyDeviceToHost));
	//areasum = 0.0;
	//FILE * fp = fopen("oldareas.txt","w");
	//for (int iTest = 0; iTest < pX1->Nverts; iTest++)
	//{
	//	areasum += p_scratch_host[iTest];
	//	fprintf(fp,"%d %1.15E\n",iTest,p_scratch_host[iTest]);
	//}
	//fclose(fp);
	//printf("Areasum_major old %1.12E \n",areasum);

	Kernel_CalculateMajorAreas<<<numTilesMajor,threadsPerTileMajor>>>(
			pX1->p_info,
			pX1->p_tri_centroid,
			pX1->pIndexTri,
			pX1->pPBCtri,
			pX1->p_area
			);
	Call(hipDeviceSynchronize(),"hipDeviceSynchronize CalculateMajorAreas");
	
	FILE * fp = fopen("newareas.txt","w");
	CallMAC(hipMemcpy(p_scratch_host,pX1->p_area,sizeof(f64)*pX1->Nverts,hipMemcpyDeviceToHost));
	f64 areasum = 0.0;
	for (int iTest = 0; iTest < pX1->Nverts; iTest++)
	{
		areasum += p_scratch_host[iTest];
		fprintf(fp,"%d %1.15E\n",iTest,p_scratch_host[iTest]);
	}
	fclose(fp);
	printf("Areasum_major II %1.12E \n",areasum);
	
	
	// The number of triangles will not be exactly numTriTiles*threadsPerTileMinor.
	// THAT WOULD BE VERY BAD NEWS: It means that the array has a hole in it!!
		
	// NEED geometry to give exact # tris and ideally # vertices also.
	// However, #tris = #firstrow + #lastrow + 2* sum of other #in_row
	// On that it looks unlikely that we'd happen to achieve a multiple of 128 if #tris % 256=0;
	// Bear in mind.
	// Alternative would be to put tris hanging off first row and outside last row.
	// This doesn't serve an obvious purpose and causes aggro: what is corner index in meaningless tri.
	// But it might actually make getting a #tris easier.
	
	// Central cells use a bigger shared memory footprint by looking at tri data with indices.
	// So we probably don't want larger blocks for them, as large as tri blocks.
	// Makes more sense to put tris in blocks
	
	
	// ***  Document design decisions and reasons. --- Weds
	
	// Here's a problem: we have said that we need to start minor block at 2* start index
	// Otherwise ... maybe it would still work if we loaded a start point into shared data,
	// but we do not know.
	// We can only load consecutive values anyway, basically. We could have an edge fetch
	// I suppose. Which would only complicated matters.
	// But it probably won't recognise contiguous access if there is a variable in the index???
	// It might or might not. I'm not sure.

	// However for GetZCurrent we can use large blocks --- this may overrun the total array
	// of minors, just put a test in there to stop it.
			
	CallMAC(hipMemset(p_summands,0,sizeof(f64)*numTilesMinor));
	
	Kernel_GetZCurrent<<<numTilesMinor,threadsPerTileMinor>>>(
		pX1->p_tri_perinfo,
		pX1->p_nT_ion_minor,
		pX1->p_nT_elec_minor,
		pX1->p_v_ion,
		pX1->p_v_elec, // Not clear if this should be nv or {n,v} yet - think.
		pX1->p_area_minor,
		p_summands
		);
	Call(hipDeviceSynchronize(),"hipDeviceSynchronize GetZCurrent 1.");
	// To get current, just take n v area in each minor cell and central cell.
	// No reason we would not know areas already but nvm that.
	
	// HMMMM
	// How to handle nT data?
	
	CallMAC(hipMemcpy(p_summands_host,p_summands,sizeof(f64)*numTilesMinor,hipMemcpyDeviceToHost));
	Iz0 = 0.0;
	int ii;
	for (ii = 0; ii < numTriTiles; ii++)
	{
		Iz0 += p_summands_host[ii];
	};	
	printf("Iz after tri areas %1.12E \n",Iz0);
	for (; ii < numTilesMinor; ii++)
	{
		Iz0 += p_summands_host[ii];
	};	
	printf("Iz after areas %1.12E \n",Iz0);
	getch();


	// DEBUG  :
	// ========

	::Kernel_Populate_A_frill<<<numTriTiles, threadsPerTileMinor>>>
		(
			pX1->p_tri_perinfo,
			pX1->p_A, // update own, read others
			pX1->p_tri_centroid,
			pX1->p_neigh_tri_index
		);

	::Kernel_Compute_Lap_A_and_Grad_A_to_get_B_on_all_minor<<<numTriTiles, threadsPerTileMinor>>>
		(
			pX1->p_A,
			pX1->p_A + BEGINNING_OF_CENTRAL,
			pX1->p_info, // does this make it work ?
			pX1->p_tri_centroid,
			pX1->p_tri_perinfo,
			pX1->p_tri_per_neigh,
			pX1->p_tri_corner_index,
			pX1->p_neigh_tri_index,
			pX1->pIndexTri,
			pX1->p_Lap_A,
			pX1->p_Lap_A + BEGINNING_OF_CENTRAL,
			pX1->p_B,
			pX1->p_B + BEGINNING_OF_CENTRAL
		);
	Call(hipDeviceSynchronize(),"hipDeviceSynchronize Compute Lap A I");

	::Kernel_Compute_grad_phi_Te_tris<<<numTriTiles, threadsPerTileMinor>>>
		(
		pX1->p_info,
		pX1->p_phi,     // on majors
		pX1->p_nT_elec_minor + BEGINNING_OF_CENTRAL, // on majors
		pX1->p_tri_corner_index,
		pX1->p_tri_perinfo,
		pX1->p_grad_phi,
		pX1->p_GradTe
		);
	Call(hipDeviceSynchronize(),"hipDeviceSynchronize Compute grad phi tri I");
	
	::Kernel_Compute_grad_phi_Te_centrals<<<numTilesMajor, threadsPerTileMajor>>>
		(
		pX1->p_info,
		pX1->p_phi,
		pX1->p_nT_elec_minor + BEGINNING_OF_CENTRAL,
		pX1->pIndexNeigh,
		// output:
		pX1->p_grad_phi + BEGINNING_OF_CENTRAL,
		pX1->p_GradTe + BEGINNING_OF_CENTRAL
		);
	Call(hipDeviceSynchronize(),"hipDeviceSynchronize Compute grad phi central I");
	// CHECK PARAMETERS <<< >>> 
			
	// Get thermal pressure on each accelerating region...
	// Better off probably to do the ionisation stage first, it will give a better idea
	// of the half-time thermal pressure we are ultimately aiming for.
	::Kernel_GetThermalPressureTris<<<numTriTiles,threadsPerTileMinor>>>
		( 
		pX1->p_info,			
		pX1->p_nT_neut_minor + BEGINNING_OF_CENTRAL,
		pX1->p_nT_ion_minor + BEGINNING_OF_CENTRAL,
		pX1->p_nT_elec_minor + BEGINNING_OF_CENTRAL,
		pX1->p_tri_corner_index,
		pX1->p_tri_perinfo,
		p_MAR_neut,
		p_MAR_ion,
		p_MAR_elec
		);
	// So far it only works on DOMAIN_TRIANGLE, CROSSING_INS gets 0.
	Call(hipDeviceSynchronize(),"hipDeviceSynchronize Thermal pressure tris");
	
	Kernel_GetThermalPressureCentrals<<<numTilesMajor,threadsPerTileMajor>>>
		(
		pX1->p_info,
		pX1->p_nT_neut_minor + BEGINNING_OF_CENTRAL,
		pX1->p_nT_ion_minor + BEGINNING_OF_CENTRAL,
		pX1->p_nT_elec_minor + BEGINNING_OF_CENTRAL,
		pX1->pIndexNeigh,
		p_MAR_neut + BEGINNING_OF_CENTRAL,
		p_MAR_ion + BEGINNING_OF_CENTRAL,
		p_MAR_elec + BEGINNING_OF_CENTRAL
		); // works on DOMAIN_VERTEX only
	Call(hipDeviceSynchronize(),"hipDeviceSynchronize Thermal pressure");
	
	printf("done GTPC\n");
		
	// End debug
	
	SendToHost(pX1, pX1, pX_host);
	pX_host->AsciiOutput("inputs_pX1.txt");
	printf("done ascii output of pX1\n\n");
	getch();

	int iSubstep;
	for (iSubstep = 0; iSubstep < numSubsteps; iSubstep++)
	{
		printf("Step %d / %d : ",iSubstep,numSubsteps);

		// First set up Iz_presc_1/2 etc:
	
		f64 thalf = t + hstep*0.5;
		// Set ReverseJz before each call to Advance Potentials.
		f64 Iz_prescribed = GetIzPrescribed(thalf);
		f64 fourpioverc_reverse_Jz = -FOUR_PI_OVER_C_*Iz_prescribed/(f64)(Syst1.numReverseJzTris);
		Set_f64_constant(four_pi_over_c_ReverseJz,fourpioverc_reverse_Jz);
		// thalf because we use it to advance dA/dt from k to k+1, via ReverseJz.
		// HOWEVER, we also aim for Iz_prescribed
		// So we need to set up 2 different variables.
		Iz_prescribed = GetIzPrescribed(t+hstep);
		
		::Kernel_Average_nT_to_tri_minors<<<numTriTiles,threadsPerTileMinor>>>(
			//pX1->p_info,
			pX1->p_tri_corner_index,
			pX1->p_tri_perinfo,
			pX1->p_nT_neut_minor + BEGINNING_OF_CENTRAL, 
			pX1->p_nT_ion_minor + BEGINNING_OF_CENTRAL, 
			pX1->p_nT_elec_minor + BEGINNING_OF_CENTRAL, 
			pX1->p_nT_neut_minor, pX1->p_nT_ion_minor,	pX1->p_nT_elec_minor			
			);
		// If one of the corners is an outermost then outermost n should be pop'd with benign value.
		// At insulator-crossing tri we require having set n = 0 inside insulator.
		
		Kernel_GetZCurrent<<<numTilesMinor,threadsPerTileMinor>>>(
			pX1->p_tri_perinfo,
			pX1->p_nT_ion_minor,
			pX1->p_nT_elec_minor,
			pX1->p_v_ion,
			pX1->p_v_elec, // Not clear if this should be nv or {n,v} ? {n,v}
			pX1->p_area_minor,
			p_summands	);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize GetZCurrent k.");
		CallMAC(hipMemcpy(p_summands_host,p_summands,sizeof(f64)*numTilesMinor,hipMemcpyDeviceToHost));
		Iz0 = 0.0;
		for (int ii = 0; ii < numTilesMinor; ii++)
		{
			Iz0 += p_summands_host[ii];
		};	
		printf("Iz t_k: %1.12E \n",Iz0);
		
		// We are going to actually try using nv, nT.
		
		// Mesh and fluid advection, h/2:
		// ==============================
		
		// 1. Create v_overall, on major cells (use central) and
		
		Kernel_Create_v_overall_and_newpos<<<numTilesMajor,threadsPerTileMajor>>>(
			pX1->p_info,
			hstep*0.5,
			
			pX1->p_nT_neut_minor + BEGINNING_OF_CENTRAL, 
			pX1->p_nT_ion_minor + BEGINNING_OF_CENTRAL, 
			pX1->p_nT_elec_minor + BEGINNING_OF_CENTRAL, 
			
			pX1->p_v_neut + BEGINNING_OF_CENTRAL,
			pX1->p_v_ion + BEGINNING_OF_CENTRAL,
			pX1->p_v_elec + BEGINNING_OF_CENTRAL, // central v
			
			pXhalf->p_info,
			pX1->p_v_overall + BEGINNING_OF_CENTRAL // make it for everything		
			);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Kernel_Create_v_overall.");
		
		// 2. transmit to minor : this is the mesh move v.
		//   ^^^ tri, function of central
		::Kernel_Average_v_overall_to_tris<<<numTriTiles,threadsPerTileMinor>>>(
			pX1->p_tri_corner_index,
			pX1->p_tri_perinfo,	
			pX1->p_v_overall + BEGINNING_OF_CENTRAL, // major v_overall
			pX1->p_tri_centroid,
			pX1->p_v_overall
			); // so motion will take place relative to this velocity.
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Kernel_Create_v_overall.");
		// Tri centroid was set up when?
		// Needs to be done for 1st system and thereafter.
		
		// Extra steps because fields unmoving :
		// 3. anti-advect phi, phidot [Lap phi advances dphi/dt also -- inconsistent with rest of scheme]
		//    + advance phi to half-time using updated phidot
		//   ^^^ major, function of major  but Lap phi uses edges from tri centroid
		
		// What we are supposed to do:
		// * phidot requires Lap_phi_k and rho_k to advance.
		// * phi uses the resulting phidot to advance to phi_half
		//  * We need grad phi and grad phidot, on major, to do anti-advect
		
		// * A uses dA/dt_k to advance to A_half
		//  * We need grad A and grad Adot, on all, to do anti-advect.
				
		Get_Lap_phi_on_major<<<numTilesMajor,threadsPerTileMajor>>>
			(
			pX1->p_phi,
			pX1->p_info,
			pX1->pIndexNeigh, // neighbours of vertices
			pX1->pPBCneigh, // rel periodic orientation of vertex neighbours
			p_Lapphi
			); // == 0 for INNERMOST & OUTERMOST
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Kernel_Get_Lap_phi_on_major.");
		
		::Kernel_Compute_grad_phi_Te_centrals<<<numTilesMajor,threadsPerTileMajor>>>(
			pX1->p_info,
			pX1->p_phidot,   // phidot is always for major
			pX1->p_nT_elec_minor + BEGINNING_OF_CENTRAL,
			pX1->pIndexNeigh,
			p_grad_phidot + BEGINNING_OF_CENTRAL,
			pX1->p_GradTe +  BEGINNING_OF_CENTRAL
			); 
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Kernel_Compute_grad_phi_centrals.");
		// sets == 0 outside of DOMAIN_VERTEX
		
		// Having created grad phi, we then want to use it to do the anti-move of phi_vertex.
		Kernel_Advance_Antiadvect_phidot<<<numTilesMajor,threadsPerTileMajor>>>(
				pX1->p_phidot,	
				pX1->p_v_overall + BEGINNING_OF_CENTRAL, // !!! NOTE BENE
				hstep*0.5,
				p_grad_phidot + BEGINNING_OF_CENTRAL, // on majors please
				p_Lapphi,
				pX1->p_nT_ion_minor + BEGINNING_OF_CENTRAL, 
				pX1->p_nT_elec_minor + BEGINNING_OF_CENTRAL, // --> rho _k
				pXhalf->p_phidot
				// This is just a ton of loading and a simple formula --
				// we should prefer to combine with Get_Lap_phi routine.
			);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Kernel_Advance_Antiadvect_phidot.");
		//
		//::Kernel_Compute_grad_phi_Te_tris<<<numTriTiles,threadsPerTileMinor>>>(
		//	pX1->p_info,
		//	pX1->p_phi, // NOTE BENE
		//	pX1->p_nT_elec_minor + BEGINNING_OF_CENTRAL,
		//	pX1->p_tri_corner_index,
		//	pX1->p_tri_perinfo,
		//	pX1->p_grad_phi // NOTE BENE --- this is for minors
		//	);
		//Call(hipDeviceSynchronize(),"hipDeviceSynchronize Compute_grad_phi_on_tris.");
		//
		Kernel_Compute_grad_phi_Te_centrals<<<numTilesMajor,threadsPerTileMajor>>>(
			pX1->p_info,
			pX1->p_phi,
			pX1->p_nT_elec_minor + BEGINNING_OF_CENTRAL,
			pX1->pIndexNeigh,
			pX1->p_grad_phi + BEGINNING_OF_CENTRAL, // NOTE BENE	
			pX1->p_GradTe + BEGINNING_OF_CENTRAL			
			);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Kernel_Compute_grad_phi_centrals.");
		// sets == 0 outside of DOMAIN_VERTEX
		
		f64 Vhalf = pX1->EzTuning*3.5*(GetIzPrescribed(thalf)/GetIzPrescribed(t));
		printf("EzTuning = %1.5E , V = %1.15E Vhalf = %1.15E \n",pX1->EzTuning,V,Vhalf);	// guesstimate 

		getch();

		Kernel_Advance_Antiadvect_phi<<<numTilesMajor,threadsPerTileMajor>>>
			(
				pX1->p_info, // for innermost & outermost, set = +-V:,
				Vhalf,
				pX1->p_phi,
				pX1->p_v_overall + BEGINNING_OF_CENTRAL, // SHOULD THIS NEED + numTris? Most obvious way is yes.
				hstep*0.5,
				pX1->p_grad_phi + BEGINNING_OF_CENTRAL,
				pXhalf->p_phidot	, // Think I'm correct to say, we use updated phidot here.
				pXhalf->p_phi
			);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Kernel_Antiadvect_phi.");
		
		::Kernel_Populate_A_frill<<<numTriTiles, threadsPerTileMinor>>>
			(
				pX1->p_tri_perinfo,
				pX1->p_A, // update own, read others
				pX1->p_tri_centroid,
				pX1->p_neigh_tri_index
			);
		// Note that if we are not sharing tri info for major tile, we can simply run with bigger blocks for major.
		// Only if we concatenate several neighbouring "tiles" that we organised.
		
		// 4. anti-advect A, dA/dt    [Advance A to half-time also]
		
		// _ Kernel_Compute_Lap_A_and_Grad_A_to_get_B_on_all_minor
		
		// Simpler(?) way: 2 separate kernels, large reload of grad A.
		// Better way: have a switch to apply results during same kernel.
		
		// Adot:
		Kernel_Compute_Grad_A_minor_antiadvect<<<numTriTiles,threadsPerTileMinor>>>(
			pX1->p_Adot,        // for creating grad
			pX1->p_Adot + BEGINNING_OF_CENTRAL,
			hstep*0.5,
			pX1->p_v_overall,    // hv = amt to anti-advect
			pX1->p_info,       // 
			pX1->p_tri_centroid, // 
			pX1->p_tri_perinfo,     // 
			pX1->p_tri_per_neigh,
			pX1->p_tri_corner_index,    // 
			pX1->p_neigh_tri_index, // 
			pX1->pIndexTri,         // we said carry on using this for now.
			false,
			0,
			// output:
			pXhalf->p_Adot// fill in for both tri and vert...			
			);
		
		if (hipPeekAtLastError() != hipSuccess) {
			printf("Kernel_Compute_Grad_A_minor_antiadvect %s\n",hipGetErrorString(hipGetLastError()));
		} else {
			printf("Kernel_Compute_Grad_A_minor_antiadvect No error found at invoc,\n");
		}

		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Antiadvect Adot.");
		if (hipPeekAtLastError() != hipSuccess) {
			printf("Kernel_Compute_Grad_A_minor_antiadvect ___ %s\n",hipGetErrorString(hipGetLastError()));
		} else {
			printf("Kernel_Compute_Grad_A_minor_antiadvect No error found at synchr,\n");
		}


		Kernel_Compute_Grad_A_minor_antiadvect<<<numTriTiles,threadsPerTileMinor>>>(
			pX1->p_A,        // for creating grad
			pX1->p_A + BEGINNING_OF_CENTRAL,
			hstep*0.5,
			pX1->p_v_overall,    // hv = amt to anti-advect
			pX1->p_info,       // 
			pX1->p_tri_centroid, // 
			pX1->p_tri_perinfo,     //
			pX1->p_tri_per_neigh,
			pX1->p_tri_corner_index,    // 
			pX1->p_neigh_tri_index, // 
			pX1->pIndexTri,         // we said carry on using this for now.
			
			true,
			pX1->p_Adot, // add h*0.5*Adot to A ...
			// output:
			pXhalf->p_A// fill in for both tri and vert...			
			);
		
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Antiadvect A etc.");
		
		// Get this right: Why we like to save Lap A half -- because it
		// contributes to advancing Adot during the main step.
		// So we have to be careful here and call for Lap A again AFTER
		// we advance A with Adot to halfway.
		// IS THAT NOT an equally good occasion to do the antiadvect of A?
		// i.e. sequence:
		
		// 1. Add h/2 * Adot to give A_half [at old positions] in X1
		
		// 2. Take grad Adot_k[old pos] -> Antiadvect Adot to
		//		pXhalf->Adot_k[new pos] = still the time k value but in new positions
		
		// 3. Take Lap A_half[old pos], grad A_half to get Ahalf[new positions]
		
		// A bit messed up:		
		// . We are taking Lap A at the old positions and want to apply it at new.
		
		// So let's do it the simple way first that involves 3 calls.
		// Then we can come back and experiment with this.
		


		// Clarity here : A needs to advance with A-dot_k.
		//  ( Adot does not need to advance until we do the midpt step )
		
		// That is the end of field evolution & mesh & species advection, 1st part.
		
		// =========================================================================
		
		
		// Old chat:
		
		// Well think again as regards nT. It's easily inferred from having n on same place.
		// It's the case that having nv creates certain amt of heat, but we can deal with heat via T given n.
		
		// But hang on a minute.
		// If we are given ns*vs and we need to know ns*(vs-v) then how do we get that?
		
		// Feels like breaking down and using {n,v} here is better after all.		
		// We need to be jolly careful about this.
		// How we estimate n on minor:
		// ===========================
		// 
		// We want n for central = n for major say. Apportion the rest of the mass so
		//   n_tri_cell = Sum (Area_intersection * n_major )/(area_tri_cell)
		// This means that they add back up -- correct?
		// We'll get mass_tri_cell = sum (area_intersection * n_major)
		// mass total for each major cell then = n_major * area_major
		
		// OK so that is a very simple way.
		
		// Let's stick with the {n,v,T} so popular up til now.
		// BUT,
		// what then do we do to ensure conservation of say Nv when we do advection of nv?
		
		// We know how much "Nv" is in the cell at the start .. ?
		// Simple way:
		// v is constant in edge cell;
		// therefore Nv = n.v.area
		
		// How do we know how much we have to finish off with?
		// Do N,NT advection first.
		// Same formula: and n_new = lc : Sum (Area_intersection * n_major )/(area_tri_cell)
		// We should allow that a certain amount of momentum has flowed in.
		// Then we choose v = (arrived mom)/(k+1 mass) to give the req amt of momentum.
		// So really there isn't a powerful reason to use nv ... even with use in corrector method it really makes no difference.
				
		// 5. Calculate half-time minor areas and estimated densities.

		// GOT TO CREATE TRI CENTROIDS BEFORE WE CREATE AREAS:

		::Kernel_CalculateTriMinorAreas_AndCentroids<<<numTriTiles,threadsPerTileMinor>>>
			(
			pXhalf->p_info,
			pXhalf->p_tri_corner_index,
			pXhalf->p_tri_perinfo,
			pXhalf->p_area_minor,
			pXhalf->p_tri_centroid
			);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize CalcMinorAreas+Centroids.");
		
		Kernel_CalculateMajorAreas<<<numTilesMajor,threadsPerTileMajor>>>(
			pXhalf->p_info,
			pXhalf->p_tri_centroid,
			pXhalf->pIndexTri,
			pXhalf->pPBCtri,
			pXhalf->p_area
			);
		// Call(hipDeviceSynchronize(),"hipDeviceSynchronize Kernel_Advance_A_with_Adot.");
		
		::Kernel_CalculateCentralMinorAreas<<<numTilesMajor,threadsPerTileMajor>>>( // central areas
			pXhalf->p_info, // used how?
			pXhalf->pIndexTri,
			pXhalf->p_area_minor,
			pXhalf->p_area_minor + BEGINNING_OF_CENTRAL
			);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize CalcCentralMinorAreas, MajorAreas.");
		
		// 6. Rel advection for each species:
		// 6a. Density & heat advection based on edge cells nv vs the move rate of the wall.
		//    ^^^ major cells sharing both major cell N,NT,area
		//        and tri cell nv, centroid --- correct?
		
		Kernel_RelAdvect_nT<<<numTilesMajor,threadsPerTileMajor>>>(
			hstep*0.5,
			pX1->p_info, 
			pX1->pIndexTri,
			//pX1->pPBCtri,
			pX1->p_tri_centroid,
			pX1->p_nT_neut_minor + BEGINNING_OF_CENTRAL,
			pX1->p_nT_ion_minor + BEGINNING_OF_CENTRAL,
			pX1->p_nT_elec_minor + BEGINNING_OF_CENTRAL,
			pX1->p_nT_neut_minor,
			pX1->p_nT_ion_minor,    
			pX1->p_nT_elec_minor,
			pX1->p_v_neut,  // should always be minor...
			pX1->p_v_ion,
			pX1->p_v_elec,
			pX1->p_v_overall, 
			pX1->p_area,
			pXhalf->p_area,
			// dest:
			pXhalf->p_nT_neut_minor + BEGINNING_OF_CENTRAL, 
			pXhalf->p_nT_ion_minor + BEGINNING_OF_CENTRAL, 
			pXhalf->p_nT_elec_minor + BEGINNING_OF_CENTRAL
			// Consider: {n,v,T} = 5 vars. One more is the magic number.
			// It is probably not the end of the world if we split into 2's and 3's, nT vs v.
			);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Reladvect nT pXhalf");

		::Kernel_Average_nT_to_tri_minors<<<numTriTiles,threadsPerTileMinor>>>(
										pXhalf->p_tri_corner_index,
										pXhalf->p_tri_perinfo, 
										pXhalf->p_nT_neut_minor + BEGINNING_OF_CENTRAL,
										pXhalf->p_nT_ion_minor + BEGINNING_OF_CENTRAL,
										pXhalf->p_nT_elec_minor + BEGINNING_OF_CENTRAL,
										pXhalf->p_nT_neut_minor,
										pXhalf->p_nT_ion_minor,
										pXhalf->p_nT_elec_minor);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize avg nT pXhalf");

			
		// 6b. Momentum advection for minor cells..
		//  2 kinds of walls: central-to-tri and tri-to-tri.
		
		::Kernel_Rel_advect_v_tris<<<numTriTiles,threadsPerTileMinor>>>(
			hstep*0.5,
			pX1->p_info,
			pX1->p_nT_neut_minor,   // -> momentum input
			pXhalf->p_nT_neut_minor, // destination n needed to divide Nv
			pX1->p_v_overall,
			pX1->p_v_neut,         // -> momentum input
			pX1->p_tri_centroid,
			pX1->p_tri_corner_index,
			pX1->p_neigh_tri_index,
			pX1->p_tri_perinfo,
			pX1->p_tri_per_neigh,  // ? does it need to exist?
			pX1->p_area_minor,
			pXhalf->p_area_minor,
			pXhalf->p_v_neut      // output
			);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Reladvect v tri neut");
		
		::Kernel_Rel_advect_v_central<<<numTilesMajor,threadsPerTileMajor>>>(
			hstep*0.5,
			pX1->p_info,
			pX1->p_tri_centroid,
			pX1->p_nT_neut_minor + BEGINNING_OF_CENTRAL,
			pX1->p_nT_neut_minor,
			pXhalf->p_nT_neut_minor + BEGINNING_OF_CENTRAL,
			pX1->p_v_neut,
			pX1->p_v_overall,
			pX1->pIndexTri,
			pX1->pPBCtri,
			pX1->p_area,
			pXhalf->p_area,
			pXhalf->p_v_neut + BEGINNING_OF_CENTRAL 
			);		

		if (hipPeekAtLastError() != hipSuccess) {
			printf("Kernel_Rel_advect_v_central %s\n",hipGetErrorString(hipGetLastError()));
		} else {
			printf("Kernel_Rel_advect_v_central No error found at invoc,\n");
		}

		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Reladvect v cent neut");
		
		::Kernel_Rel_advect_v_tris<<<numTriTiles,threadsPerTileMinor>>>(
			hstep*0.5,
			pX1->p_info,
			pX1->p_nT_ion_minor,
			pXhalf->p_nT_ion_minor,
			pX1->p_v_overall,
			pX1->p_v_ion,
			pX1->p_tri_centroid,
			pX1->p_tri_corner_index,
			pX1->p_neigh_tri_index,
			pX1->p_tri_perinfo,
			pX1->p_tri_per_neigh,
			pX1->p_area_minor,
			pXhalf->p_area_minor,
			pXhalf->p_v_ion
			);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Reladvect v tri ion");
		
		::Kernel_Rel_advect_v_central<<<numTilesMajor,threadsPerTileMajor>>>(
			hstep*0.5,
			pX1->p_info,
			pX1->p_tri_centroid,
			pX1->p_nT_ion_minor + BEGINNING_OF_CENTRAL,
			pX1->p_nT_ion_minor,
			pXhalf->p_nT_ion_minor + BEGINNING_OF_CENTRAL,
			pX1->p_v_ion,
			pX1->p_v_overall,
			pX1->pIndexTri,
			pX1->pPBCtri,
			pX1->p_area,
			pXhalf->p_area,
			pXhalf->p_v_ion + BEGINNING_OF_CENTRAL 
			);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Reladvect v cent ion");
		
		::Kernel_Rel_advect_v_tris<<<numTriTiles,threadsPerTileMinor>>>(
			hstep*0.5,
			pX1->p_info,
			pX1->p_nT_elec_minor,
			pXhalf->p_nT_elec_minor,
			pX1->p_v_overall,
			pX1->p_v_elec,
			pX1->p_tri_centroid,
			pX1->p_tri_corner_index,
			pX1->p_neigh_tri_index,
			pX1->p_tri_perinfo,
			pX1->p_tri_per_neigh,
			pX1->p_area_minor,
			pXhalf->p_area_minor,
			pXhalf->p_v_elec
			);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Reladvect v tri elec");
		
		::Kernel_Rel_advect_v_central<<<numTilesMajor,threadsPerTileMajor>>>(
			hstep*0.5,
			pX1->p_info,
			pX1->p_tri_centroid,
			pX1->p_nT_elec_minor + BEGINNING_OF_CENTRAL,
			pX1->p_nT_elec_minor,
			pXhalf->p_nT_elec_minor + BEGINNING_OF_CENTRAL,
			pX1->p_v_elec,
			pX1->p_v_overall,
			pX1->pIndexTri,
			pX1->pPBCtri,
			pX1->p_area,
			pXhalf->p_area,
			pXhalf->p_v_elec + BEGINNING_OF_CENTRAL 
			);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Reladvect v cent elec");
		/*
		hipMemcpy(pX_host->p_nT_ion_minor,			pX1->p_nT_ion_minor,
			sizeof(nT)*Syst1.Nminor,
			hipMemcpyDeviceToHost);
		hipMemcpy(pX_host->p_nT_elec_minor,			pXhalf->p_nT_ion_minor,
			sizeof(nT)*Syst1.Nminor,
			hipMemcpyDeviceToHost);
		hipMemcpy(pX_host->p_v_ion,			pX1->p_v_ion,
			sizeof(f64_vec3)*Syst1.Nminor,
			hipMemcpyDeviceToHost);
		hipMemcpy(pX_host->p_v_elec,			pXhalf->p_v_ion,
			sizeof(f64_vec3)*Syst1.Nminor,
			hipMemcpyDeviceToHost);
		hipMemcpy(pX_host->p_area_minor,			pXhalf->p_area_minor,
			sizeof(f64)*Syst1.Nminor,
			hipMemcpyDeviceToHost);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize memcpies");

		printf("43654: n_ion %1.10E  %d : %1.10E \n",pX_host->p_nT_elec_minor[43654].n,
			BEGINNING_OF_CENTRAL + 20000, pX_host->p_nT_elec_minor[BEGINNING_OF_CENTRAL + 20000].n);
		

		FILE * file = fopen("inputs0.txt","w");
		fprintf(file,"index | pX1_v_ion_x y z | pXhalf_v_ion_x y z | 1n 1T n T \n");
		
		for (int iMinor = 0; iMinor < Syst1.Nminor; iMinor++)
		{
			fprintf(file,"%d %d | %1.10E %1.10E %1.10E | %1.10E %1.10E %1.10E | "
				"%1.10E %1.10E %1.10E %1.10E | %1.10E \n",
				iMinor,pX_host->p_tri_perinfo[iMinor].flag,
				pX_host->p_v_ion[iMinor].x,pX_host->p_v_ion[iMinor].y,pX_host->p_v_ion[iMinor].z,
				pX_host->p_v_elec[iMinor].x,pX_host->p_v_elec[iMinor].y,pX_host->p_v_elec[iMinor].z,
				pX_host->p_nT_ion_minor[iMinor].n,pX_host->p_nT_ion_minor[iMinor].T,
				pX_host->p_nT_elec_minor[iMinor].n,pX_host->p_nT_elec_minor[iMinor].T,
				pX_host->p_area_minor[iMinor]);
		};
		hipMemcpy(pX_host->p_area_minor,			pX1->p_area_minor,
			sizeof(f64)*Syst1.Nminor,
			hipMemcpyDeviceToHost);
		for (int iMinor = 0; iMinor < Syst1.Nminor; iMinor++)
		{
			fprintf(file,"%d %1.10E \n",iMinor,pX_host->p_area_minor[iMinor]);
		};
		fclose(file);*/
		
		// RESULT SO FAR: n, T look normal. v_ion is OK in X1, becomes IND/INF in Xhalf.
		// From 85392 onwards, it's 0,0,large . To 73532 : IND=xy, INF=z.
		
		// ============================================================================
		// Now do estimates for half-time system ready for midpoint calls:
		
		// Get Grad phi _half, etc, for each minor cell:
		
		hipMemset(p_MAR_neut,0,sizeof(f64_vec3)*pX1->Nminor);
		hipMemset(p_MAR_ion,0,sizeof(f64_vec3)*pX1->Nminor);
		hipMemset(p_MAR_elec,0,sizeof(f64_vec3)*pX1->Nminor);

		//FILE * fp = fopen("tri_data.txt","w");
		//for (int iii = 0; iii < Syst1.Ntris; iii++)
		//{
		//	fprintf(fp,"%d %d %d %d\n",iii,pX_host->p_tri_corner_index[iii].i1,
		//		pX_host->p_tri_corner_index[iii].i2,
		//		pX_host->p_tri_corner_index[iii].i3);
		//}
		//fclose(fp);
		//hipMemcpy(pX_host->p_tri_corner_index,pX1->p_tri_corner_index,
		//	sizeof(LONG3)*Syst1.Ntris,
		//	hipMemcpyDeviceToHost
		//	);
		//fp = fopen("tri_data2.txt","w");
		//for (int iii = 0; iii < Syst1.Ntris; iii++)
		//{
		//	fprintf(fp,"%d %d %d %d\n",iii,pX_host->p_tri_corner_index[iii].i1,
		//		pX_host->p_tri_corner_index[iii].i2,
		//		pX_host->p_tri_corner_index[iii].i3);
		//}
		//fclose(fp);

		::Kernel_Populate_A_frill<<<numTriTiles, threadsPerTileMinor>>>
			(
				pXhalf->p_tri_perinfo,
				pXhalf->p_A, // update own, read others
				pXhalf->p_tri_centroid,
				pXhalf->p_neigh_tri_index
			);

		::Kernel_Compute_Lap_A_and_Grad_A_to_get_B_on_all_minor<<<numTriTiles, threadsPerTileMinor>>>
			(
				pXhalf->p_A,
				pXhalf->p_A + BEGINNING_OF_CENTRAL,
				pXhalf->p_info, // does this make it work ?
				pXhalf->p_tri_centroid,
				pXhalf->p_tri_perinfo,
				pXhalf->p_tri_per_neigh,
				pX1->p_tri_corner_index,
				pXhalf->p_neigh_tri_index,
				pXhalf->pIndexTri,
				pXhalf->p_Lap_A,
				pXhalf->p_Lap_A + BEGINNING_OF_CENTRAL,
				pXhalf->p_B,
				pXhalf->p_B + BEGINNING_OF_CENTRAL
			);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Compute Lap A I");

		::Kernel_Compute_grad_phi_Te_tris<<<numTriTiles, threadsPerTileMinor>>>
			(
			pXhalf->p_info,
			pXhalf->p_phi,     // on majors
			pXhalf->p_nT_elec_minor + BEGINNING_OF_CENTRAL, // on majors
			pXhalf->p_tri_corner_index,
			pXhalf->p_tri_perinfo,
			pXhalf->p_grad_phi,
			pXhalf->p_GradTe
			);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Compute grad phi tri I");
		
		::Kernel_Compute_grad_phi_Te_centrals<<<numTilesMajor, threadsPerTileMajor>>>
			(
			pXhalf->p_info,
			pXhalf->p_phi,
			pXhalf->p_nT_elec_minor + BEGINNING_OF_CENTRAL,
			pXhalf->pIndexNeigh,
			// output:
			pXhalf->p_grad_phi + BEGINNING_OF_CENTRAL,
			pXhalf->p_GradTe + BEGINNING_OF_CENTRAL
			);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Compute grad phi central I");
		// CHECK PARAMETERS <<< >>> 
				
		// Get thermal pressure on each accelerating region...
		// Better off probably to do the ionisation stage first, it will give a better idea
		// of the half-time thermal pressure we are ultimately aiming for.
		::Kernel_GetThermalPressureTris<<<numTriTiles,threadsPerTileMinor>>>
			( 
			pXhalf->p_info,			
			pXhalf->p_nT_neut_minor + BEGINNING_OF_CENTRAL,
			pXhalf->p_nT_ion_minor + BEGINNING_OF_CENTRAL,
			pXhalf->p_nT_elec_minor + BEGINNING_OF_CENTRAL,
			pXhalf->p_tri_corner_index,
			pXhalf->p_tri_perinfo,
			p_MAR_neut,
			p_MAR_ion,
			p_MAR_elec
			);
		// So far it only works on DOMAIN_TRIANGLE, CROSSING_INS gets 0.
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Thermal pressure tris");
		
		Kernel_GetThermalPressureCentrals<<<numTilesMajor,threadsPerTileMajor>>>
			(
			pXhalf->p_info,
			pXhalf->p_nT_neut_minor + BEGINNING_OF_CENTRAL,
			pXhalf->p_nT_ion_minor + BEGINNING_OF_CENTRAL,
			pXhalf->p_nT_elec_minor + BEGINNING_OF_CENTRAL,
			pXhalf->pIndexNeigh,
			p_MAR_neut + BEGINNING_OF_CENTRAL,
			p_MAR_ion + BEGINNING_OF_CENTRAL,
			p_MAR_elec + BEGINNING_OF_CENTRAL
			); // works on DOMAIN_VERTEX only
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Thermal pressure");
		
		printf("done GTPC\n");
		
		
		// %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
		// It might be reasonable to instead be using TRIANGLE nT in getting
		// thermal pressure on centrals? You'd think so.
		// %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
		
		// This should be pretty similar to grad phi, and like grad phi needs to
		// apply to every minor cell.
		
		// nv:
		// Would we also wish to switch to nT? Because this reflects the actual
		// amount of heating that is happening, and is then conserved.
		// Again, to get grad T, we load T into shared memory by dividing.
		
		// We choose v because we need to be taking grad v ?
		// Or we'd need estimated n to be picked up at the same time. But that would be OK.
		// No - not as good: it involves divisions...
		
		// -----------------------------------------------------
		// Now ionisation , accel & heating
		
		Kernel_Ionisation<<<numTilesMajor,threadsPerTileMajor>>>(
			hstep,
			pXhalf->p_info,
			pXhalf->p_area, // correct input? used?
			pXhalf->p_nT_neut_minor + BEGINNING_OF_CENTRAL, // src 
			pXhalf->p_nT_ion_minor + BEGINNING_OF_CENTRAL,
			pXhalf->p_nT_elec_minor + BEGINNING_OF_CENTRAL,		
			0,0,0,
	
			// No output, I think now, except nn_ionrec.			
			p_nn_ionrec_minor + BEGINNING_OF_CENTRAL, 
			0 // b2ndpass  --  ??
			);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Ionisation 1");

		// What do we do with information about momentum changing? Or do we only need nnionrec?

		Kernel_Average_nnionrec_to_tris<<<numTriTiles,threadsPerTileMinor>>>
			(
			pXhalf->p_tri_perinfo,
			pXhalf->p_tri_corner_index,
			p_nn_ionrec_minor + BEGINNING_OF_CENTRAL,
			p_nn_ionrec_minor
			);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize splitout nn");
		
		// OKAY what we see is that nn_ionrec is needed in centrals and we want to set it to the
		// major value.
		// Therefore nn_ionrec logically is organised with major/central values at the end.
		
		// Now run midpoint v step on minor cells.
		
		printf("about to do midpoint.\n");
		getch();
		
		// I think for debugging it would be good here to dump all the inputs to host
		// and spit it out to a spreadsheet.
		
		f64 temp1, temp2;
		nT nTtemp3, nTtemp4;

		//hipMemcpy(&temp1,&(pX1->p_phi[10000]),sizeof(f64),hipMemcpyDeviceToHost);
		//hipMemcpy(&temp2,&(pXhalf->p_phi[10000]),sizeof(f64),hipMemcpyDeviceToHost);
		//hipMemcpy(&nTtemp3,&(pX1->p_nT_elec_minor[89000]),sizeof(nT),hipMemcpyDeviceToHost);
		//hipMemcpy(&nTtemp4,&(pXhalf->p_nT_elec_minor[89000]),sizeof(nT),hipMemcpyDeviceToHost);
		//
		//printf("phi[10000] %1.9E %1.9E \nTe[89000] %1.5E %1.5E\n",
		//	temp1,temp2,nTtemp3.T,nTtemp4.T);
		//getch();
		
		SendToHost(pXhalf, pXhalf, pX_host);		
		pX_host->AsciiOutput("Inputs_half.txt");
		
		printf("start midpt step:\n");
		pXhalf->evaltime = pX1->evaltime + 0.5*hstep;
		Kernel_Midpoint_v_and_Adot<<<numTilesMinor,threadsPerTileMinor>>>
			(
				hstep,
				pXhalf->p_tri_perinfo,
				pXhalf->p_nT_neut_minor, // src
				pXhalf->p_nT_ion_minor, 
				pXhalf->p_nT_elec_minor, 
				// Both n_k and n_k+1 appear in the midpt formula, so we need n_k.
				
				//pXhalf->p_nT_neut, // k or k+1/2
				//pXhalf->p_nT_ion, // k or k+1/2
				//pXhalf->p_nT_elec, // k or k+1/2 --- for forming nu etc...
				0,0,0, // on b2ndPass == 0 we'll try not loading.
				
				p_nn_ionrec_minor, // Have to load 2 additional doubles due to doing ionisation outside.
							
				pXhalf->p_tri_centroid, // Defined ever?
				pXhalf->p_info,   // were these positions actually created?
				
				pXhalf->p_B,
				pXhalf->p_v_neut, // Do not update: we need v_k again.
				pXhalf->p_v_ion,
				pXhalf->p_v_elec,
				// Thing is, we have to create 0.5*(v_k+v_k+1) on 1st pass.
				// We want to leave v_k[advected] intact, so no update here.
				// We want to go again from v_k on 2nd and 3rd pass.
					
				pXhalf->p_area_minor,	// It's assumed to be area_k+1 but I guess it's area_k+1/2 ... too bad?
								// THIS MATTERS !!
								// I think area_1/2 is relevant both times in midpt but need to check that.
				pXhalf->p_grad_phi,
				pXhalf->p_Lap_A, // check this input?
				pXhalf->p_Adot,  // anti-advected Adot_k ...
				p_MAR_neut,
				p_MAR_ion,
				p_MAR_elec, // assume take integral(-grad(nT))/m_s
				pXhalf->p_GradTe,
						
				// output: ( Here interim values of v )
				pXusable->p_v_neut,
				pXusable->p_v_ion,
				pXusable->p_v_elec,
				p_resistive_heat_neut_minor, 
				p_resistive_heat_ion_minor,
				p_resistive_heat_elec_minor,
				pXusable->p_Adot,
				false, // 1st pass
				pX1->EzTuning, // put what here?
 				p_Iz0_summands,
				p_summands
			);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize MidptAccel 1");
		
		printf("midpt 1 done.");
		getch();

		// The amount of resistive heating depends on Ez of course...
		// but we don't want to have to run twice at this juncture.
		// Therefore?		
	
		// . The heating routine also cements the effects of ionisation on n.
		// . Assume central ionisation == vertcell ionisation.
		Kernel_Heating_routine<<<numTilesMajor,threadsPerTileMajor>>>(
			hstep,
			pXhalf->p_info,
			pXhalf->pIndexTri, // fetch htg amounts from minor cells...
			
			pXhalf->p_nT_neut_minor + BEGINNING_OF_CENTRAL, 
			pXhalf->p_nT_ion_minor + BEGINNING_OF_CENTRAL,  
			pXhalf->p_nT_elec_minor + BEGINNING_OF_CENTRAL,
			p_nn_ionrec_minor + BEGINNING_OF_CENTRAL,
			
			pXhalf->p_B + BEGINNING_OF_CENTRAL,
			
		//	p_visccond_heatrate_neut, // from conduction routine...
		//	p_visccond_heatrate_ion,  // applies to major cells
		//  p_visccond_heatrate_elec,
			
			p_resistive_heat_neut_minor, // from midpoint v acceleration routine
			p_resistive_heat_ion_minor,
			p_resistive_heat_elec_minor,
			
			pXhalf->p_area, // major areas
			
			// output:
			pXusable->p_nT_neut_minor + BEGINNING_OF_CENTRAL,
			pXusable->p_nT_ion_minor + BEGINNING_OF_CENTRAL,
			pXusable->p_nT_elec_minor + BEGINNING_OF_CENTRAL,
			false // not 2nd pass
			);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Heating 1");
		
		::Kernel_Average_nT_to_tri_minors<<<numTriTiles,threadsPerTileMinor>>>(
										pX1->p_tri_corner_index,
										pX1->p_tri_perinfo, 
										pXusable->p_nT_neut_minor + BEGINNING_OF_CENTRAL,
										pXusable->p_nT_ion_minor + BEGINNING_OF_CENTRAL,
										pXusable->p_nT_elec_minor + BEGINNING_OF_CENTRAL,
										pXusable->p_nT_neut_minor,
										pXusable->p_nT_ion_minor,
										pXusable->p_nT_elec_minor);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize avg nT pXusable");
		
		printf("Heating 1 done\n");
		
		//hipMemcpy(p_MAR_ion_host,				p_MAR_ion,
	//		sizeof(f64_vec3)*Syst1.Nminor,			hipMemcpyDeviceToHost);
	//	Call(hipDeviceSynchronize(),"hipDeviceSynchronize memcpies");
		
		/*file = fopen("ionMAR.txt","w");
		if (file == 0) {
			printf("could not open ionMAR.txt");
			while (1) getch();
		} else {
			printf("ionMAR.txt opened");
			getch();
		};*/


		// OKay let's think about visc htg and conductive htg.
		// Conduction has to be done on major cells, using the B field etc from the minor cells.
		// Whereas viscous heating? Each minor wall generates some heating. Look at the two 
		// effects 2T dT/dt|wall and share the sum to both major cells.
		// Viscous heating at edge of central cell: share the sum to the major cell it is within.
		// OK that seems to work out well.
		
		// -------------------------------------------------------------------------------------------
		// Aside:
		// Is it easier just to do heating on minors and then average back to major? NO
		// There would be nothing wrong with putting ionisation in minors as well ... but it's blurring
		// something that does not need to be, since T is averaged before we do it.
		// -----------------------------------------------------------------------------------
		
		
		// Next thing:
		
		// . Calculate ionisation again with half-time heat
		// . Two runs of midpt - first one establishes Ohm relationship.
				
		hipMemset(p_MAR_neut,0,sizeof(f64_vec3)*pX1->Nminor);
		hipMemset(p_MAR_ion,0,sizeof(f64_vec3)*pX1->Nminor);
		hipMemset(p_MAR_elec,0,sizeof(f64_vec3)*pX1->Nminor);
	
		// We have not changed A - it's still the same so no need to go again for Lap A.
		// We have not changed phi. But we have changed Te so re-estimate its gradient.
		::Kernel_Compute_grad_phi_Te_tris<<<numTriTiles, threadsPerTileMinor>>>
			(
			pXhalf->p_info,
			pXhalf->p_phi,     // on majors
			pXusable->p_nT_elec_minor + BEGINNING_OF_CENTRAL, // on majors
			pXhalf->p_tri_corner_index,
			pXhalf->p_tri_perinfo,
			pXhalf->p_grad_phi,
			pXusable->p_GradTe
			);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Compute grad phi tri I");
		
		::Kernel_Compute_grad_phi_Te_centrals<<<numTilesMajor, threadsPerTileMajor>>>
			(
			pXhalf->p_info,
			pXhalf->p_phi,
			pXusable->p_nT_elec_minor + BEGINNING_OF_CENTRAL,
			pXhalf->pIndexNeigh,
			// output:
			pXhalf->p_grad_phi + BEGINNING_OF_CENTRAL,
			pXusable->p_GradTe + BEGINNING_OF_CENTRAL
			);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Compute grad phi central I");
		// CHECK PARAMETERS <<< >>> 
		
		// Get thermal pressure on each accelerating region...
		// Better off probably to do the ionisation stage first, it will give a better idea
		// of the half-time thermal pressure we are ultimately aiming for.
		// Not so much because it doesn't include recombination heating update -- I don't think so anyway.
		::Kernel_GetThermalPressureTris<<<numTriTiles,threadsPerTileMinor>>>
			( 
			pXhalf->p_info,			
			pXusable->p_nT_neut_minor + BEGINNING_OF_CENTRAL,
			pXusable->p_nT_ion_minor + BEGINNING_OF_CENTRAL,
			pXusable->p_nT_elec_minor + BEGINNING_OF_CENTRAL,
			pXhalf->p_tri_corner_index,
			pXhalf->p_tri_perinfo,
			p_MAR_neut,
			p_MAR_ion,
			p_MAR_elec // overwrite
			);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Thermal pressure tris");
		
		Kernel_GetThermalPressureCentrals<<<numTilesMajor,threadsPerTileMajor>>>
			(
			pXhalf->p_info,
			pXusable->p_nT_neut_minor + BEGINNING_OF_CENTRAL,
			pXusable->p_nT_ion_minor + BEGINNING_OF_CENTRAL,
			pXusable->p_nT_elec_minor + BEGINNING_OF_CENTRAL,
			pXhalf->pIndexNeigh,
			p_MAR_neut + BEGINNING_OF_CENTRAL,
			p_MAR_ion + BEGINNING_OF_CENTRAL,
			p_MAR_elec + BEGINNING_OF_CENTRAL
			);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Thermal pressure");
		
		// %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
		// It might be reasonable to instead be using TRIANGLE nT in getting
		// thermal pressure on centrals? You'd think so.
		// %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
		
		// Set Ez given initially estimated relationship -- 
		// is there actually a point in this? Does it affect what we now find?
		// Really not. ?

		// 2nd call to ionisation calc:
		Kernel_Ionisation<<<numTilesMajor,threadsPerTileMajor>>>(
			hstep,
			pXhalf->p_info,
			pXhalf->p_area, // correct input? used?
			pXhalf->p_nT_neut_minor + BEGINNING_OF_CENTRAL, // src 
			pXhalf->p_nT_ion_minor + BEGINNING_OF_CENTRAL,
			pXhalf->p_nT_elec_minor + BEGINNING_OF_CENTRAL,		
			pXusable->p_nT_neut_minor + BEGINNING_OF_CENTRAL, // src 
			pXusable->p_nT_ion_minor + BEGINNING_OF_CENTRAL,
			pXusable->p_nT_elec_minor + BEGINNING_OF_CENTRAL,		
			// No output, I think now, except nn_ionrec.			
			p_nn_ionrec_minor + BEGINNING_OF_CENTRAL, 
			0 // b2ndpass  --  ??
			);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Ionisation 2");
				
		Kernel_Average_nnionrec_to_tris<<<numTriTiles,threadsPerTileMinor>>>
			(
			pXhalf->p_tri_perinfo,
			pXhalf->p_tri_corner_index,
			p_nn_ionrec_minor + BEGINNING_OF_CENTRAL,
			p_nn_ionrec_minor
			);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize splitout nn");

		SendToHost(pXusable,pXhalf,pX_host);

		pX_host->AsciiOutput("inputs_3__.txt");
		// v and gradphi come back as IND / viz,vez INF.

		// Establish Ohmic relationship:
		printf("ready to do midpt again\n");
		getch();

		pXusable->evaltime = pXhalf->evaltime;
		Kernel_Midpoint_v_and_Adot<<<numTilesMinor,threadsPerTileMinor>>>
		(
			hstep,
			pXhalf->p_tri_perinfo,
			pXhalf->p_nT_neut_minor, // src
			pXhalf->p_nT_ion_minor, 
			pXhalf->p_nT_elec_minor, 
			// Both n_k and n_k+1 appear in the midpt formula, so we need n_k.
			
			pXusable->p_nT_neut_minor, // use
			pXusable->p_nT_ion_minor, 
			pXusable->p_nT_elec_minor, 
				
			p_nn_ionrec_minor, // Have to load 2 additional doubles due to doing ionisation outside.
							
			pXhalf->p_tri_centroid, // Defined ever?
			pXhalf->p_info,   // were these positions actually created?
				
			pXhalf->p_B,
			pXhalf->p_v_neut, // src
			pXhalf->p_v_ion,
			pXhalf->p_v_elec,
				// Thing is, we have to create 0.5*(v_k+v_k+1) on 1st pass.
				// We want to leave v_k[advected] intact, so no update here.
				// We want to go again from v_k on 2nd and 3rd pass.
					
			pXhalf->p_area_minor,	// It's assumed to be area_k+1 but I guess it's area_k+1/2 ... too bad?
								// THIS MATTERS !!
								// I think area_1/2 is relevant both times in midpt but need to check that.
			pXhalf->p_grad_phi,
			pXhalf->p_Lap_A, // check this input?
			pXhalf->p_Adot,  // anti-advected Adot_k ...
			p_MAR_neut,
			p_MAR_ion,
			p_MAR_elec, // assume take integral(-grad(nT))/m_s
			pXusable->p_GradTe,

			// output: not used, of course
			pXusable->p_v_neut,
			pXusable->p_v_ion,
			pXusable->p_v_elec,
				// Why are pXusable->v even useful? I don't see anywhere they are used.
				// Therefore do not store them on 1st pass!
			
			p_resistive_heat_neut_minor, 
			p_resistive_heat_ion_minor,
			p_resistive_heat_elec_minor,
			pXusable->p_Adot,
			1, // 2nd pass    --- send an integer here
			pX1->EzTuning, 
 			p_Iz0_summands, // here is what we want
			p_summands
		);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize MidptAccel 2");
		
		// Establish Ohmic relationship:
		CallMAC(hipMemcpy(p_Iz0_summands_host,p_Iz0_summands,sizeof(f64)*numTilesMinor, hipMemcpyDeviceToHost));
		CallMAC(hipMemcpy(p_summands_host,p_summands,sizeof(f64)*numTilesMinor, hipMemcpyDeviceToHost));
		f64 Iz0 = 0.0, IzPerEzTuning = 0.0;
		for (int ii = 0; ii < numTilesMinor; ii++)
		{
			Iz0 += p_Iz0_summands_host[ii];
			IzPerEzTuning += p_summands_host[ii];
		};

		// Set pXhalf->EzTuning:
		pXhalf->EzTuning = pX1->EzTuning + (Iz_prescribed-Iz0)/IzPerEzTuning;
		
		printf("pX1->EzTuning %1.8E Iz_prescribed %1.8E \n"
			"Iz0 %1.8E IzPerEzTuning %1.8E \n"
			"pXhalf->EzTuning %1.8E \n",
			pX1->EzTuning,Iz_prescribed, Iz0, IzPerEzTuning,
			pXhalf->EzTuning);
		getch();

		// Call with same parameters over again:
		Kernel_Midpoint_v_and_Adot<<<numTilesMinor,threadsPerTileMinor>>>
		(
			hstep,
			pXhalf->p_tri_perinfo,
			pXhalf->p_nT_neut_minor, // src
			pXhalf->p_nT_ion_minor, 
			pXhalf->p_nT_elec_minor, 
			// Both n_k and n_k+1 appear in the midpt formula, so we need n_k.
			pXusable->p_nT_neut_minor, // use
			pXusable->p_nT_ion_minor, 
			pXusable->p_nT_elec_minor, 
			p_nn_ionrec_minor, // Have to load 2 additional doubles due to doing ionisation outside.
			pXhalf->p_tri_centroid, // Defined ever?
			pXhalf->p_info,   // were these positions actually created?
			pXhalf->p_B,
			pXhalf->p_v_neut, // src
			pXhalf->p_v_ion,
			pXhalf->p_v_elec,
			pXhalf->p_area_minor,	// It's assumed to be area_k+1 but I guess it's area_k+1/2 ... too bad?
			pXhalf->p_grad_phi,
			pXhalf->p_Lap_A, 
			pXhalf->p_Adot,  
			p_MAR_neut, p_MAR_ion, p_MAR_elec,
			pXusable->p_GradTe,

			// output: 
			pXusable->p_v_neut,
			pXusable->p_v_ion,
			pXusable->p_v_elec,
			p_resistive_heat_neut_minor, // save off resistive.
			p_resistive_heat_ion_minor,  // Does it need to be zeroed beforehand?
			p_resistive_heat_elec_minor,
			pXusable->p_Adot,
			2, // 3rd pass    --- send an integer here
			pXhalf->EzTuning, 
 			p_Iz0_summands, // here is what we want
			p_summands
		);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize MidptAccel 3");
		
		CallMAC(hipMemcpy(p_Iz0_summands_host,p_Iz0_summands,sizeof(f64)*numTilesMinor, hipMemcpyDeviceToHost));
		Iz0 = 0.0;
		for (int ii = 0; ii < numTilesMinor; ii++)
		{
			Iz0 += p_Iz0_summands_host[ii];
		};
		printf("Iz attained %1.8E Presc %1.8E Diff %1.4E\n",
			Iz0,Iz_prescribed,Iz0-Iz_prescribed);
		getch();
		// Can double-check here that Iz is being achieved:
		
		Kernel_Heating_routine<<<numTilesMajor,threadsPerTileMajor>>>(
			hstep,
			pXhalf->p_info,
			pXhalf->pIndexTri, // fetch htg amounts from minor cells...
			
			pXhalf->p_nT_neut_minor + BEGINNING_OF_CENTRAL, 
			pXhalf->p_nT_ion_minor + BEGINNING_OF_CENTRAL,  
			pXhalf->p_nT_elec_minor + BEGINNING_OF_CENTRAL,
			p_nn_ionrec_minor + BEGINNING_OF_CENTRAL,
			// On 2nd pass, let us use the output as the "used" nT, and update it.
			
			pXhalf->p_B + BEGINNING_OF_CENTRAL,
			
		//	p_visccond_heatrate_neut, // from conduction routine...
		//	p_visccond_heatrate_ion,  // applies to major cells
		//    p_visccond_heatrate_elec,
			
			p_resistive_heat_neut_minor, // from midpoint v acceleration routine
			p_resistive_heat_ion_minor,
			p_resistive_heat_elec_minor,
			
			pXhalf->p_area,
			
			// output:
			pXusable->p_nT_neut_minor + BEGINNING_OF_CENTRAL,
			pXusable->p_nT_ion_minor + BEGINNING_OF_CENTRAL,
			pXusable->p_nT_elec_minor + BEGINNING_OF_CENTRAL,
			true // 2nd pass
			);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Heating 2");
		printf("heating done\n");
		
		::Kernel_Average_nT_to_tri_minors<<<numTriTiles,threadsPerTileMinor>>>(
										pX1->p_tri_corner_index,
										pX1->p_tri_perinfo, 
										pXusable->p_nT_neut_minor + BEGINNING_OF_CENTRAL,
										pXusable->p_nT_ion_minor + BEGINNING_OF_CENTRAL,
										pXusable->p_nT_elec_minor + BEGINNING_OF_CENTRAL,
										pXusable->p_nT_neut_minor,
										pXusable->p_nT_ion_minor,
										pXusable->p_nT_elec_minor);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize avg nT pXusable");
		
		printf("end");
		while(1) getch();


		// We now created pXusable -> n,v,T and Adot. pXhalf->EzTuning.
		
		// Now the rest of the move.
		// Finish updating Adot:
		
		// Do advection, 2nd half. Going from pXusable to pX2.

		// v_overall will come from t_half.
		// If we want, we can extrapolate compared to t_k v_overall.
		Kernel_Create_v_overall_and_newpos<<<numTilesMajor,threadsPerTileMajor>>>(
			pXhalf->p_info,
			hstep*0.5,
			pXusable->p_nT_neut_minor + BEGINNING_OF_CENTRAL, 
			pXusable->p_nT_ion_minor + BEGINNING_OF_CENTRAL, 
			pXusable->p_nT_elec_minor + BEGINNING_OF_CENTRAL, 
			
			pXusable->p_v_neut + BEGINNING_OF_CENTRAL,
			pXusable->p_v_ion + BEGINNING_OF_CENTRAL,
			pXusable->p_v_elec + BEGINNING_OF_CENTRAL, // central v
			
			pXusable->p_info,
			pXusable->p_v_overall + BEGINNING_OF_CENTRAL // make it for everything		
			);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Kernel_Create_v_overall pXusable");
		::Kernel_Average_v_overall_to_tris<<<numTriTiles,threadsPerTileMinor>>>(
			pXusable->p_tri_corner_index,
			pXusable->p_tri_perinfo,	
			pXusable->p_v_overall + BEGINNING_OF_CENTRAL, // major v_overall
			pXusable->p_tri_centroid, 

			// HAS THIS BEEN POPULATED ??

			pXusable->p_v_overall
			); // so motion will take place relative to this velocity.
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Kernel_Create_v_overall pXusable");
				
		
		Kernel_Compute_Grad_A_minor_antiadvect<<<numTriTiles,threadsPerTileMinor>>>(
			pXhalf->p_A,        // for creating grad
			pXhalf->p_A + BEGINNING_OF_CENTRAL,  // ?
			hstep*0.5, 
			pXusable->p_v_overall,    // hv = amt to anti-advect
			
			// Take geometry from pXhalf which was advected.
			pXhalf->p_info,       // 
			pXhalf->p_tri_centroid, //  Defined?
			pXhalf->p_tri_perinfo,     // 
			pXhalf->p_tri_per_neigh,
			pXhalf->p_tri_corner_index,    // 
			pXhalf->p_neigh_tri_index, // 
			pXhalf->pIndexTri,         // we said carry on using this for now.
			
			true,
			pXusable->p_Adot, // add h*0.5*Adot_k+1 to A ...
			// output:
			pX2->p_A // fill in for both tri and vert...						
			);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Antiadvect A etc II.");
			
		// Adot:
		Kernel_Compute_Grad_A_minor_antiadvect<<<numTriTiles,threadsPerTileMinor>>>(
			pXusable->p_Adot,        // for creating grad
			pXusable->p_Adot + BEGINNING_OF_CENTRAL,
			
			hstep*0.5,
			pXusable->p_v_overall,    // hv = amt to anti-advect
			
			pXhalf->p_info,       // 
			pXhalf->p_tri_centroid, // 
			pXhalf->p_tri_perinfo,     // 
			pXhalf->p_tri_per_neigh,
			pXhalf->p_tri_corner_index,    // 
			pXhalf->p_neigh_tri_index, // 
			pXhalf->pIndexTri,         // we said carry on using this for now.
			false,
			0,
			// output:
			pX2->p_Adot// fill in for both tri and vert...			
			);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Antiadvect Adot II");
		
		// Note pXhalf->grad_phi is already populated.
				
		Vhalf = pXhalf->EzTuning*3.5;
		Kernel_Advance_Antiadvect_phi<<<numTilesMajor,threadsPerTileMajor>>>
			(
				pXhalf->p_info,
				Vhalf,
				pXhalf->p_phi,
				pXusable->p_v_overall + BEGINNING_OF_CENTRAL, // SHOULD THIS NEED + numTris? Most obvious way is yes.
				hstep*0.5,
				pXhalf->p_grad_phi + BEGINNING_OF_CENTRAL,
				pXhalf->p_phidot	, // still using half-time value
				pX2->p_phi
			);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Kernel_Antiadvect_phi II.");
		
		// Now get Lap phi_k+1
		// ...
		// If we were smart we'd avoid duplicating that effort.		
		// Now rel advect to produce pX2->nvT for doing rho_k+1 since we used unadvected for rho_k.
		
		::Kernel_CalculateTriMinorAreas_AndCentroids<<<numTriTiles,threadsPerTileMinor>>>
			(
			pX2->p_info,
			pX2->p_tri_corner_index,
			pX2->p_tri_perinfo,
			pX2->p_area_minor,
			pX2->p_tri_centroid
			);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize CalcMinorAreas+Centroids.pX2");
		
		Kernel_CalculateMajorAreas<<<numTilesMajor,threadsPerTileMajor>>>(
			pX2->p_info,
			pX2->p_tri_centroid,
			pX2->pIndexTri,
			pX2->pPBCtri,
			pX2->p_area
			);
		// Call(hipDeviceSynchronize(),"hipDeviceSynchronize Kernel_Advance_A_with_Adot.");
		
		::Kernel_CalculateCentralMinorAreas<<<numTilesMajor,threadsPerTileMajor>>>( // central areas
			pX2->p_info, // used how?
			pX2->pIndexTri,
			pX2->p_area_minor,
			pX2->p_area_minor + BEGINNING_OF_CENTRAL
			);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize CalcCentralMinorAreas, MajorAreas.pX2");
		
		
		Kernel_RelAdvect_nT<<<numTilesMajor,threadsPerTileMajor>>>(
			hstep*0.5,
			pXhalf->p_info, 
			pXhalf->pIndexTri,
	//		pXhalf->pPBCtri,			
			pXhalf->p_tri_centroid, 

			pXusable->p_nT_neut_minor + BEGINNING_OF_CENTRAL,
			pXusable->p_nT_ion_minor + BEGINNING_OF_CENTRAL,
			pXusable->p_nT_elec_minor + BEGINNING_OF_CENTRAL,
			pXusable->p_nT_neut_minor,
			pXusable->p_nT_ion_minor,    
			pXusable->p_nT_elec_minor,
			pXusable->p_v_neut,  // should always be minor...
			pXusable->p_v_ion,
			pXusable->p_v_elec,
			pXusable->p_v_overall, 

			pXhalf->p_area,
			pX2->p_area,
			// dest:
			pX2->p_nT_neut_minor + BEGINNING_OF_CENTRAL, 
			pX2->p_nT_ion_minor + BEGINNING_OF_CENTRAL, 
			pX2->p_nT_elec_minor + BEGINNING_OF_CENTRAL
			// Consider: {n,v,T} = 5 vars. One more is the magic number.
			// It is probably not the end of the world if we split into 2's and 3's, nT vs v.
			);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Reladvect nT pX2");
				
		::Kernel_Rel_advect_v_tris<<<numTriTiles,threadsPerTileMinor>>>(
			hstep*0.5,
			pXhalf->p_info,
			pXusable->p_nT_neut_minor,   // -> momentum input
			pX2->p_nT_neut_minor, // destination n needed to divide Nv
			pXusable->p_v_overall,
			pXusable->p_v_neut,          // -> momentum input
			
			pXhalf->p_tri_centroid,
			pXhalf->p_tri_corner_index,
			pXhalf->p_neigh_tri_index,
			pXhalf->p_tri_perinfo,
			pXhalf->p_tri_per_neigh,  // ? does it need to exist?
			pXhalf->p_area_minor,
			pX2->p_area_minor,
			pX2->p_v_neut      // output
			);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Reladvect v tri neut pX2");
		::Kernel_Rel_advect_v_central<<<numTilesMajor,threadsPerTileMajor>>>(
			hstep*0.5,
			pXhalf->p_info,
			pXhalf->p_tri_centroid,
			pXusable->p_nT_neut_minor + BEGINNING_OF_CENTRAL,
			pXusable->p_nT_neut_minor,
			pX2->p_nT_neut_minor + BEGINNING_OF_CENTRAL,
			pXusable->p_v_neut + BEGINNING_OF_CENTRAL,
			pXusable->p_v_overall,
			pXhalf->pIndexTri,
			pXhalf->pPBCtri,
			pXhalf->p_area,
			pX2->p_area,
			pX2->p_v_neut + BEGINNING_OF_CENTRAL // ?
			);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Reladvect v cent neut pX2");
		
		::Kernel_Rel_advect_v_tris<<<numTriTiles,threadsPerTileMinor>>>(
			hstep*0.5,
			pXhalf->p_info,
			pXusable->p_nT_ion_minor,
			pX2->p_nT_ion_minor,
			pXusable->p_v_overall,
			pXusable->p_v_ion,

			pXhalf->p_tri_centroid,
			pXhalf->p_tri_corner_index,
			pXhalf->p_neigh_tri_index,
			pXhalf->p_tri_perinfo,
			pXhalf->p_tri_per_neigh,
			
			pXhalf->p_area_minor,
			pX2->p_area_minor,
			pX2->p_v_ion
			);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Reladvect v tri ion");
		
		::Kernel_Rel_advect_v_central<<<numTilesMajor,threadsPerTileMajor>>>(
			hstep*0.5,
			pXhalf->p_info,
			pXhalf->p_tri_centroid,
			pXusable->p_nT_ion_minor + BEGINNING_OF_CENTRAL,
			pXusable->p_nT_ion_minor,
			pX2->p_nT_ion_minor + BEGINNING_OF_CENTRAL,
			pXusable->p_v_ion + BEGINNING_OF_CENTRAL,
			pXusable->p_v_overall,
			pXhalf->pIndexTri,
			pXhalf->pPBCtri,
			pXhalf->p_area,
			pX2->p_area,
			pX2->p_v_ion + BEGINNING_OF_CENTRAL // ? Check USAGE
			);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Reladvect v cent neut");
		

		::Kernel_Rel_advect_v_tris<<<numTriTiles,threadsPerTileMinor>>>(
			hstep*0.5,
			pXhalf->p_info,
			pXusable->p_nT_elec_minor,
			pX2->p_nT_elec_minor,
			pXusable->p_v_overall,
			pXusable->p_v_elec,

			pXhalf->p_tri_centroid,
			pXhalf->p_tri_corner_index,
			pXhalf->p_neigh_tri_index,
			pXhalf->p_tri_perinfo,
			pXhalf->p_tri_per_neigh,
			
			pXhalf->p_area_minor,
			pX2->p_area_minor,
			pX2->p_v_elec
			);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Reladvect v tri ion");
		
		::Kernel_Rel_advect_v_central<<<numTilesMajor,threadsPerTileMajor>>>(
			hstep*0.5,
			pXhalf->p_info,
			pXhalf->p_tri_centroid,
			pXusable->p_nT_elec_minor + BEGINNING_OF_CENTRAL,
			pXusable->p_nT_elec_minor,
			pX2->p_nT_elec_minor + BEGINNING_OF_CENTRAL,
			pXusable->p_v_elec + BEGINNING_OF_CENTRAL,
			pXusable->p_v_overall,
			pXhalf->pIndexTri,
			pXhalf->pPBCtri,
			pXhalf->p_area,
			pX2->p_area,
			pX2->p_v_elec + BEGINNING_OF_CENTRAL // ?
			);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Reladvect v cent neut");
		
		// =========================================================================
				
		Get_Lap_phi_on_major<<<numTilesMajor,threadsPerTileMajor>>>
			(
			pX2->p_phi,
			pX2->p_info,
			pX2->pIndexNeigh, // neighbours of vertices
			pX2->pPBCneigh, // rel periodic orientation of vertex neighbours
			p_Lapphi
			);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Kernel_Get_Lap_phi_on_major II.");
		
		// Get grad_phidot...		
		::Kernel_Compute_grad_phi_Te_centrals<<<numTilesMajor,threadsPerTileMajor>>>(
			pXhalf->p_info,
			pXhalf->p_phidot,   // phidot is always for major
			pXusable->p_nT_elec_minor + BEGINNING_OF_CENTRAL, // not important
			pXhalf->pIndexNeigh,
			p_grad_phidot + BEGINNING_OF_CENTRAL,
			pXhalf->p_GradTe + BEGINNING_OF_CENTRAL // nvm
			);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Kernel_Compute_grad_phi_centrals.");
		
		// Is that call avoidable from what is above?


		Kernel_Advance_Antiadvect_phidot<<<numTilesMajor,threadsPerTileMajor>>>(
				pXhalf->p_phidot,	
				pXusable->p_v_overall + BEGINNING_OF_CENTRAL, // !!! NOTE BENE
				hstep*0.5,
				p_grad_phidot + BEGINNING_OF_CENTRAL, // on majors please
				
				p_Lapphi,
				pX2->p_nT_ion_minor + BEGINNING_OF_CENTRAL, 
				pX2->p_nT_elec_minor + BEGINNING_OF_CENTRAL, // --> rho _k
				pX2->p_phidot
				// This is just a ton of loading and a simple formula --
				// we should prefer to combine with Get_Lap_phi routine.
			);
		Call(hipDeviceSynchronize(),"hipDeviceSynchronize Kernel_Advance_Antiadvect_phidot II");
			
		pX2->evaltime = pXusable->evaltime + hstep*0.5;
		
		
		// Document sequence with inputs labelled fully and showing where calc'd.
		// Then check that the calcs are as claimed in each routine.
		t += hstep;
	};
	
	
	// We intermittently return to CPU to do re-Delaunerization - to begin with.
	// Otherwise only send data back, every 2.5e-11 s, for graphing. -> 20fps gives 2s/ns. 50s/25ns
	
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&elapsedTime, start,stop);
	printf("Elapsed time : %f ms\n" ,elapsedTime);

	//printf("Time elapsed: %s",report_time_(1));
	
	
	// 4. hipMemcpy from device to host
	
	CallMAC(hipMemcpy(pX_host_target->p_phi, pX1->p_phi, numVertices*sizeof(f64), hipMemcpyDeviceToHost));
	CallMAC(hipMemcpy(pX_host_target->p_phidot, pX1->p_phidot, numVertices*sizeof(f64), hipMemcpyDeviceToHost));
	CallMAC(hipMemcpy(pX_host_target->p_A, pX1->p_A, numVertices*sizeof(f64_vec3), hipMemcpyDeviceToHost));
	CallMAC(hipMemcpy(pX_host_target->p_Adot, pX1->p_Adot, numVertices*sizeof(f64_vec3), hipMemcpyDeviceToHost));
	
	CallMAC(hipMemcpy(pX_host_target->p_nT_neut_minor, pX1->p_nT_neut_minor, numVertices*sizeof(nT), hipMemcpyDeviceToHost));
	CallMAC(hipMemcpy(pX_host_target->p_nT_ion_minor, pX1->p_nT_ion_minor, numVertices*sizeof(nT), hipMemcpyDeviceToHost));
	CallMAC(hipMemcpy(pX_host_target->p_nT_elec_minor, pX1->p_nT_elec_minor, numVertices*sizeof(nT), hipMemcpyDeviceToHost));
	CallMAC(hipMemcpy(pX_host_target->p_v_neut, pX1->p_v_neut, numVertices*sizeof(f64_vec3), hipMemcpyDeviceToHost));
	CallMAC(hipMemcpy(pX_host_target->p_v_ion, pX1->p_v_ion, numVertices*sizeof(f64_vec3), hipMemcpyDeviceToHost));
	CallMAC(hipMemcpy(pX_host_target->p_v_elec, pX1->p_v_elec, numVertices*sizeof(f64_vec3), hipMemcpyDeviceToHost));
		
	CallMAC(hipMemcpy(pX_host_target->p_info, pX1->p_info, numVertices*sizeof(structural), hipMemcpyDeviceToHost));
	CallMAC(hipMemcpy(pX_host_target->pIndexNeigh, pX1->pIndexNeigh, numVertices*MAXNEIGH_d*sizeof(long),hipMemcpyDeviceToHost));
	CallMAC(hipMemcpy(pX_host_target->pPBCneigh, pX1->pPBCneigh, numVertices*MAXNEIGH_d*sizeof(char),hipMemcpyDeviceToHost));
	CallMAC(hipMemcpy(pX_host_target->p_area, pX1->p_area, numVertices*sizeof(f64),hipMemcpyDeviceToHost));
	// Do we need to copy back neighbour arrays? Will they actually ever change?
	// pX->host_target has to point to an object already invoked ie with dimensioned arrays.
	
	
	hipFree(p_summands);
	hipFree(p_Iz0_summands);
	hipFree( p_resistive_heat_neut_minor);
	hipFree( p_resistive_heat_ion_minor);
	hipFree( p_resistive_heat_elec_minor);
	hipFree(p_scratch_d);
	hipFree(p_Iz0_initial);

	hipFree(p_nn_ionrec_minor);
	hipFree(p_MAR_neut);
	hipFree(p_MAR_ion);
	hipFree(p_MAR_elec);

	hipFree(p_Lapphi);
	hipFree(p_grad_phidot);

	free(p_summands_host);
	free(p_Iz0_summands_host);
	free(p_scratch);
	free(p_scratch_info);
	free(p_Iz0_initial_host);
	free(p_scratch_host);
	free(p_nn_host);
	free(p_MAR_neut_host);
	free(p_MAR_ion_host);
	free(p_MAR_elec_host);

	printf("Transferred back.\n");
	
	Call(hipMemGetInfo (&uFree,&uTotal),"hipMemGetInfo (&uFree,&uTotal)");
	printf("uFree %d uTotal %d\n",uFree,uTotal);
	
	printf("END OF CUDA");

	//" do not call cudaResetDevice(): save invoked stuff for next step."

}