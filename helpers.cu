#include "hip/hip_runtime.h"

// Device routines that can be #included by the kernels file.
#include "cuda_struct.h"
#include "kernel.h"
 

#ifdef __HIPCC__
__device__ __forceinline__ f64 GetEzShape(f64 r) {
	return 1.0 - 1.0 / (1.0 + exp(-24.0*(r - 4.32)));
	// return 1.0 - 1.0 / (1.0 + exp(-16.0*(r - 4.2))); // At 4.0cm it is 96% as strong as at tooth. At 4.4 it is 4%.
}
#else
f64 inline GetEzShape_(f64 r) {
	return 1.0 - 1.0 / (1.0 + exp(-16.0*(r - 4.2))); // At 4.0cm it is 96% as strong as at tooth. 4.2 50%. At 4.4 it is 4%.
}
#endif

__device__ __forceinline__ f64 Get_lnLambda_ion_d(f64 n_ion,f64 T_ion)
{
	// Assume static f64 const is no good in kernel.

	f64 factor, lnLambda_sq;
	f64 Tion_eV3 = T_ion*T_ion*T_ion*one_over_kB_cubed;
	f64 lnLambda = 23.0 - 0.5*log(n_ion/Tion_eV3); 

	// floor at 2.0:
	lnLambda_sq = lnLambda*lnLambda;
	factor = 1.0+0.5*lnLambda+0.25*lnLambda_sq+0.125*lnLambda*lnLambda_sq + 0.0625*lnLambda_sq*lnLambda_sq;
	lnLambda += 2.0/factor;

	return lnLambda;
}		

__device__ __forceinline__ f64 Get_lnLambda_d(real n_e,real T_e)
{
	real lnLambda, factor, lnLambda_sq, lnLambda1, lnLambda2;

	real Te_eV = T_e*one_over_kB;
	real Te_eV2 = Te_eV*Te_eV;
	real Te_eV3 = Te_eV*Te_eV2;

	if (n_e*Te_eV3 > 0.0) {
		
		lnLambda1 = 23.0 - 0.5*log(n_e/Te_eV3);
		lnLambda2 = 24.0 - 0.5*log(n_e/Te_eV2);
		// smooth between the two:
		factor = 2.0*fabs(Te_eV-10.0)*(Te_eV-10.0)/(1.0+4.0*(Te_eV-10.0)*(Te_eV-10.0));
		lnLambda = lnLambda1*(0.5-factor)+lnLambda2*(0.5+factor);
		
		// floor at 2 just in case, but it should not get near:
		lnLambda_sq = lnLambda*lnLambda;
		factor = 1.0+0.5*lnLambda+0.25*lnLambda_sq+0.125*lnLambda*lnLambda_sq + 0.0625*lnLambda_sq*lnLambda_sq;
		lnLambda += 2.0/factor;

		// Golant p.40 warns that it becomes invalid when an electron gyroradius is less than a Debye radius.
		// That is something to worry about if  B/400 > n^1/2 , so looks not a big concern.

		// There is also a quantum ceiling. It will not be anywhere near. At n=1e20, 0.5eV, the ceiling is only down to 29; it requires cold dense conditions to apply.

		if (lnLambda < 2.0) lnLambda = 2.0; // deal with negative inputs

	} else {
		lnLambda = 20.0;
	};
	return lnLambda;
}		


__device__ __forceinline__ f64_vec2 Anticlock_rotate2(const f64_vec2 arg)
{
	f64_vec2 result;
	result.x = Anticlockwise_d.xx*arg.x+Anticlockwise_d.xy*arg.y;
	result.y = Anticlockwise_d.yx*arg.x+Anticlockwise_d.yy*arg.y;
	return result;
}
__device__ __forceinline__ f64_vec2 Clockwise_rotate2(const f64_vec2 arg)
{
	f64_vec2 result;
	result.x = Clockwise_d.xx*arg.x+Clockwise_d.xy*arg.y;
	result.y = Clockwise_d.yx*arg.x+Clockwise_d.yy*arg.y;
	return result;
}

__device__ __forceinline__ f64_vec3 Anticlock_rotate3(const f64_vec3 arg)
{
	f64_vec3 result;
	result.x = Anticlockwise_d.xx*arg.x+Anticlockwise_d.xy*arg.y;
	result.y = Anticlockwise_d.yx*arg.x+Anticlockwise_d.yy*arg.y;
	result.z = arg.z;
	return result;
}
__device__ __forceinline__ f64_vec3 Clockwise_rotate3(const f64_vec3 arg)
{
	f64_vec3 result;
	result.x = Clockwise_d.xx*arg.x+Clockwise_d.xy*arg.y;
	result.y = Clockwise_d.yx*arg.x+Clockwise_d.yy*arg.y;
	result.z = arg.z;
	return result;
}

__device__  __forceinline__ void Estimate_Ion_Neutral_Cross_sections_d(real T, // call with T in electronVolts
	real * p_sigma_in_MT,
	real * p_sigma_in_visc)
{
	if (T > cross_T_vals_d[9]) {
		*p_sigma_in_MT = cross_s_vals_MT_ni_d[9];
		*p_sigma_in_visc = cross_s_vals_viscosity_ni_d[9];
		return;
	}
	if (T < cross_T_vals_d[0]) {
		*p_sigma_in_MT = cross_s_vals_MT_ni_d[0];
		*p_sigma_in_visc = cross_s_vals_viscosity_ni_d[0];
		return;
	}
	int i = 1;
	//while (T > cross_T_vals_d[i]) i++;

	if (T > cross_T_vals_d[5]) {
		if (T > cross_T_vals_d[7]) {
			if (T > cross_T_vals_d[8])
			{
				i = 9; // top of interval
			}
			else {
				i = 8;
			};
		}
		else {
			if (T > cross_T_vals_d[6]) {
				i = 7;
			}
			else {
				i = 6;
			};
		};
	}
	else {
		if (T > cross_T_vals_d[3]) {
			if (T > cross_T_vals_d[4]) {
				i = 5;
			}
			else {
				i = 4;
			};
		}
		else {
			if (T > cross_T_vals_d[2]) {
				i = 3;
			}
			else {
				if (T > cross_T_vals_d[1]) {
					i = 2;
				}
				else {
					i = 1;
				};
			};
		};
	};
	// T lies between i-1,i
	real ppn = (T - cross_T_vals_d[i - 1]) / (cross_T_vals_d[i] - cross_T_vals_d[i - 1]);

	*p_sigma_in_MT = ppn * cross_s_vals_MT_ni_d[i] + (1.0 - ppn)*cross_s_vals_MT_ni_d[i - 1];
	*p_sigma_in_visc = ppn * cross_s_vals_viscosity_ni_d[i] + (1.0 - ppn)*cross_s_vals_viscosity_ni_d[i - 1];
	return;
}

__device__ __forceinline__ f64 Estimate_Neutral_MT_Cross_section_d(f64 T)
{
	// CALL WITH T IN eV

	if (T > cross_T_vals_d[9]) return cross_s_vals_MT_ni_d[9];		
	if (T < cross_T_vals_d[0]) return cross_s_vals_MT_ni_d[0];
	
	int i = 1;
	//while (T > cross_T_vals_d[i]) i++;
	
	if (T > cross_T_vals_d[5]) {
		if (T > cross_T_vals_d[7]) {
			if (T > cross_T_vals_d[8])
			{
				i = 9; // top of interval
			} else {
				i = 8;
			};
		} else {
			if (T > cross_T_vals_d[6]) {
				i = 7;
			} else {
				i = 6;
			};
		};
	} else {
		if (T > cross_T_vals_d[3]) {
			if (T > cross_T_vals_d[4]) {
				i = 5;
			} else {
				i = 4;
			};
		} else {
			if (T > cross_T_vals_d[2]) {
				i = 3;
			} else {
				if (T > cross_T_vals_d[1]) {
					i = 2;
				} else {
					i = 1;
				};
			};
		};
	}; 
	
	// T lies between i-1,i
	real ppn = (T-cross_T_vals_d[i-1])/(cross_T_vals_d[i]-cross_T_vals_d[i-1]);
	return ppn*cross_s_vals_MT_ni_d[i] + (1.0-ppn)*cross_s_vals_MT_ni_d[i-1];

}

__device__ __forceinline__ f64 Estimate_Neutral_Neutral_Viscosity_Cross_section_d(f64 T) 
{
	// call with T in electronVolts
	
	if (T > cross_T_vals_d[9]) return cross_s_vals_viscosity_nn_d[9];
	if (T < cross_T_vals_d[0]) return cross_s_vals_viscosity_nn_d[0];

	int i = 1;
	//while (T > cross_T_vals_d[i]) i++;
	
	if (T > cross_T_vals_d[5]) {
		if (T > cross_T_vals_d[7]) {
			if (T > cross_T_vals_d[8])
			{
				i = 9; // top of interval
			} else {
				i = 8;
			};
		} else {
			if (T > cross_T_vals_d[6]) {
				i = 7;
			} else {
				i = 6;
			};
		};
	} else {
		if (T > cross_T_vals_d[3]) {
			if (T > cross_T_vals_d[4]) {
				i = 5;
			} else {
				i = 4;
			};
		} else {
			if (T > cross_T_vals_d[2]) {
				i = 3;
			} else {
				if (T > cross_T_vals_d[1]) {
					i = 2;
				} else {
					i = 1;
				};
			};
		};
	}; 

	// T lies between i-1,i
	real ppn = (T-cross_T_vals_d[i-1])/(cross_T_vals_d[i]-cross_T_vals_d[i-1]);
	return ppn*cross_s_vals_viscosity_nn_d[i] + (1.0-ppn)*cross_s_vals_viscosity_nn_d[i-1];
}

__device__ __forceinline__ f64 Estimate_Ion_Neutral_Viscosity_Cross_section(f64 T)
{
	if (T > cross_T_vals_d[9]) return cross_s_vals_viscosity_ni_d[9];		
	if (T < cross_T_vals_d[0]) return cross_s_vals_viscosity_ni_d[0];
	
	int i = 1;
	//while (T > cross_T_vals_d[i]) i++;
	
	if (T > cross_T_vals_d[5]) {
		if (T > cross_T_vals_d[7]) {
			if (T > cross_T_vals_d[8])
			{
				i = 9; // top of interval
			} else {
				i = 8;
			};
		} else {
			if (T > cross_T_vals_d[6]) {
				i = 7;
			} else {
				i = 6;
			};
		};
	} else {
		if (T > cross_T_vals_d[3]) {
			if (T > cross_T_vals_d[4]) {
				i = 5;
			} else {
				i = 4;
			};
		} else {
			if (T > cross_T_vals_d[2]) {
				i = 3;
			} else {
				if (T > cross_T_vals_d[1]) {
					i = 2;
				} else {
					i = 1;
				};
			};
		};
	}; 
	
	// T lies between i-1,i
	real ppn = (T-cross_T_vals_d[i-1])/(cross_T_vals_d[i]-cross_T_vals_d[i-1]);
	return ppn*cross_s_vals_viscosity_ni_d[i] + (1.0-ppn)*cross_s_vals_viscosity_ni_d[i-1];
}


__device__ __forceinline__ f64 Calculate_Kappa_Neutral(f64 n_i, f64 T_i, f64 n_n, f64 T_n)
{
	// NOTE:
	// It involves sqrt and we could easily find a way to calculate only once.
		
	if (n_n == 0.0) return 0.0;

	f64 s_in_visc, s_nn_visc;

	s_in_visc = Estimate_Ion_Neutral_Viscosity_Cross_section(T_i*one_over_kB);
	s_nn_visc = Estimate_Neutral_Neutral_Viscosity_Cross_section(T_n*one_over_kB);

	// Oh. So there's another two we have to port.
	// Yet for ion eta it's so different, apparently.
	
	f64 ionneut_thermal = sqrt(T_i/m_ion+T_n/m_n);
	f64	nu_ni_visc = n_i*s_in_visc*ionneut_thermal;
	f64	nu_nn_visc = n_n*s_nn_visc*sqrt(T_n/m_n);
	f64	nu_nheart = 0.75*nu_ni_visc + 0.25*nu_nn_visc;
	f64 kappa_n = NEUTRAL_KAPPA_FACTOR*n_n*T_n/(m_n*nu_nheart);
	// NEUTRAL_KAPPA_FACTOR should be in constant.h
	// e-n does not feature.
	return kappa_n;
}


__device__ __forceinline__ void Get_kappa_parallels_and_nu_hearts
				(real n_n,real T_n,real n_i,real T_i,real n_e,real T_e,
				f64 * pkappa_neut, f64 * pnu_nheart, 
				f64 * pkappa_ion_par, f64 * pnu_iheart,
				f64 * pkappa_e_par, f64 * pnu_eheart,
				f64 * pratio)
{
	f64 s_in_visc, s_nn_visc, s_en_visc;

	f64 ionneut_thermal, 
		nu_ni_visc, nu_nn_visc, nu_nheart,
		nu_in_visc, nu_en_visc, nu_ii, nu_iheart, nu_eheart,
		sqrt_Te, electron_thermal, nu_eiBar;
	 
	f64 lnLambda = Get_lnLambda_ion_d(n_i,T_i);

	ionneut_thermal = sqrt(T_i/m_ion+T_n/m_n);
	sqrt_Te = sqrt(T_e);
	
	s_in_visc = Estimate_Ion_Neutral_Viscosity_Cross_section(T_i*one_over_kB);
	s_nn_visc = Estimate_Neutral_Neutral_Viscosity_Cross_section(T_n*one_over_kB);
	
	nu_in_visc = n_n*s_in_visc*ionneut_thermal;
	nu_nn_visc = n_n*s_nn_visc*sqrt(T_n/m_n);
	nu_ni_visc = n_i*s_in_visc*ionneut_thermal;
	
	nu_ii = Nu_ii_Factor*kB_to_3halves*n_i*lnLambda/(T_i*sqrt(T_i));

	nu_iheart = 0.75*nu_in_visc
			+ 0.8*nu_ii-0.25*nu_in_visc*nu_ni_visc/(3.0*nu_ni_visc+nu_nn_visc);
	*pkappa_ion_par = 2.5*n_i*T_i/(m_ion*(nu_iheart));
	*pnu_iheart = nu_iheart;

	s_en_visc = Estimate_Ion_Neutral_Viscosity_Cross_section(T_e*one_over_kB);
	electron_thermal = (sqrt_Te*over_sqrt_m_e);
	
	lnLambda = Get_lnLambda_d(n_e,T_e);
	
	nu_eiBar = nu_eiBarconst*kB_to_3halves*n_i*lnLambda/(T_e*sqrt_Te);
	nu_en_visc = n_n*s_en_visc*electron_thermal;
	nu_eheart = 1.87*nu_eiBar + nu_en_visc;
	*pnu_eheart = nu_eheart;
	*pkappa_e_par =  2.5*n_e*T_e/(m_e*nu_eheart);

	// Store ratio for thermoelectric use:
	*pratio = nu_eiBar/nu_eheart;


	if (n_n == 0.0){
		*pkappa_neut = 0.0;
	} else {

		nu_nheart = 0.75*nu_ni_visc + 0.25*nu_nn_visc;
		*pkappa_neut = NEUTRAL_KAPPA_FACTOR*n_n*T_n/(m_n*nu_nheart);
		*pnu_nheart = nu_nheart;
		// NEUTRAL_KAPPA_FACTOR should be in constant.h
		// e-n does not feature.
	};
	 
}
__device__ __forceinline__ void RotateClockwise(f64_vec3 & v)
{
	f64 temp = Clockwise_d.xx*v.x + Clockwise_d.xy*v.y;
	v.y = Clockwise_d.yx*v.x + Clockwise_d.yy*v.y;
	v.x = temp;
}
__device__ __forceinline__ void RotateAnticlockwise(f64_vec3 & v)
{
	f64 temp = Anticlockwise_d.xx*v.x + Anticlockwise_d.xy*v.y;
	v.y = Anticlockwise_d.yx*v.x + Anticlockwise_d.yy*v.y;
	v.x = temp;
}

__device__ __forceinline__ f64_vec2 GetRadiusIntercept(f64_vec2 x1,f64_vec2 x2,f64 r)
{
	// where we meet radius r on the line passing through u0 and u1?
	f64_vec2 result;
	
	f64 den = (x2.x-x1.x)*(x2.x-x1.x) + (x2.y - x1.y)*(x2.y - x1.y) ;
	f64 a = (x1.x * (x2.x-x1.x) + x1.y * (x2.y-x1.y) ) / den;
	// (t + a)^2 - a^2 = (  c^2 - x1.x^2 - x1.y^2  )/den
	f64 root = sqrt( (r*r- x1.x*x1.x - x1.y*x1.y)/den + a*a ) ;
	f64 t1 = root - a;
	f64 t2 = -root - a;
	
	// since this is a sufficient condition to satisfy the circle, this probably means that
	// the other solution is on the other side of the circle.
	// Which root is within x1, x2 ? Remember x2 would be t = 1.

	if (t1 > 1.0) 
	{
		if ((t2 < 0.0) || (t2 > 1.0))
		{	
			// This usually means one of the points actually is on the curve.
			f64 dist1 = min(fabs(t1-1.0),fabs(t1));
			f64 dist2 = min(fabs(t2-1.0),fabs(t2));
			if (dist1 < dist2)
			{
				// use t1				
				result.x = x1.x + t1*(x2.x-x1.x);
				result.y = x1.y + t1*(x2.y-x1.y);
		//		printf("t1@@");
			} else {
				// use t2				
				result.x = x1.x + t2*(x2.x-x1.x);
				result.y = x1.y + t2*(x2.y-x1.y);
		//		printf("t2@@");
			};
		} else {		
			// use t2:		
			result.x = x1.x + t2*(x2.x-x1.x);
			result.y = x1.y + t2*(x2.y-x1.y);
		//	printf("t2~");
		};
	} else {
		result.x = x1.x + t1*(x2.x-x1.x);
		result.y = x1.y + t1*(x2.y-x1.y);	
		//printf("t1~");
	};

	// For some reason this is only hitting the radius to single precision.

	// printf to compare difference between achieved radius and r.
	
	//if ((result.x < -0.145) && (result.x > -0.155))
	//{
	//	f64 achieve = result.modulus();
	//	printf("ach %1.12E r %1.2f t1 %1.10E \nx %1.12E y %1.12E\n",achieve,r,t1,result.x,result.y);
	//}

	// So what do we do?

	// We could boost back but there seem to be bigger problems thereafter.

	// Ideally we'd go through and compare and see, is it t1 that is a bit wrong here?
	// 

	return result;
}

