#include "hip/hip_runtime.h"

#ifndef surfacegraph2_cpp
#define surfacegraph2_cpp

#include <time.h>

#include "headers.h"
#include "d3d.h"
#include "surfacegraph_tri.h"
#include "FFxtubes.h"
#include "globals.h"

#include "mesh.h"
#include "mesh.cu"
#include "meshutil.cu"

#define VERBOSEGRAPHICS 0 // also in meshutil

//extern FixedMesh Fixed;
extern int GlobalWhichLabels;
extern real GlobalRescaling;

#define PI32bit  3.14159268f

extern unsigned int cw;

extern float Historic_max[512][HISTORY]; // if max is falling, use historic maximum for graph.
extern float Historic_min[512][HISTORY];
extern int Historic_powermax[512]; // if max is falling, use historic maximum for graph.
extern int Historic_powermin[512];
extern bool boolGlobalHistory;
extern bool bCullNone;

long numVerticesKey;
D3D Direct3D;

void strip_0(char * buffer)
{
	char * p = buffer;
	while (*p != 'E') ++p;
	++p;
//	if (*p == '+')
//	{
//		memmove(p,p+1,strlen(p));
//	} else {
		++p;
//	}
	if (*p == '0') memmove(p,p+1,strlen(p));
}


DrawableTex2D::DrawableTex2D(UINT width, UINT height, UINT mipLevels,
		D3DFORMAT texFormat, bool useDepthBuffer,
		D3DFORMAT depthFormat, D3DVIEWPORT9& viewport,  bool autoGenMips)
: mTex(0), mRTS(0), mTopSurf(0), mWidth(width), mHeight(height), 
  mMipLevels(mipLevels), mTexFormat(texFormat), mUseDepthBuffer(useDepthBuffer),
  mDepthFormat(depthFormat), mViewPort(viewport), mAutoGenMips(autoGenMips)
{
}

DrawableTex2D::~DrawableTex2D()
{
	onLostDevice();
}

IDirect3DTexture9* DrawableTex2D::d3dTex()
{
	return mTex;
}

void DrawableTex2D::onLostDevice()
{
	SAFE_RELEASE2(mTex);
	SAFE_RELEASE2(mRTS);
	SAFE_RELEASE2(mTopSurf);
}

void DrawableTex2D::onResetDevice()
{
	UINT usage = D3DUSAGE_RENDERTARGET;
	if(mAutoGenMips)
		usage |= D3DUSAGE_AUTOGENMIPMAP;

	DXChk(D3DXCreateTexture(Direct3D.pd3dDevice, mWidth, mHeight, mMipLevels, usage, mTexFormat, D3DPOOL_DEFAULT, &mTex));
	DXChk(D3DXCreateRenderToSurface(Direct3D.pd3dDevice, mWidth, mHeight, mTexFormat, mUseDepthBuffer, mDepthFormat, &mRTS));
	DXChk(mTex->GetSurfaceLevel(0, &mTopSurf));
}

void DrawableTex2D::beginScene()
{
	mRTS->BeginScene(mTopSurf, &mViewPort);
}

void DrawableTex2D::endScene()
{
	mRTS->EndScene(D3DX_FILTER_NONE);
}

// vertex.cpp copy paste:

IDirect3DVertexDeclaration9* VertexPos::Decl = 0;
IDirect3DVertexDeclaration9* VertexCol::Decl = 0;
IDirect3DVertexDeclaration9* VertexPN::Decl  = 0;
IDirect3DVertexDeclaration9* VertexPNT::Decl = 0;
IDirect3DVertexDeclaration9* VertexPNT3::Decl = 0;
IDirect3DVertexDeclaration9* VertexPNf::Decl = 0;

void D3D::InitAllVertexDeclarations()
{
	//===============================================================
	// VertexPos

	D3DVERTEXELEMENT9 VertexPosElements[] = 
	{
		{0, 0,  D3DDECLTYPE_FLOAT3, D3DDECLMETHOD_DEFAULT, D3DDECLUSAGE_POSITION, 0},
		D3DDECL_END()
	};	
	DXChk(pd3dDevice->CreateVertexDeclaration(VertexPosElements, &VertexPos::Decl));

	//===============================================================
	// VertexCol

	D3DVERTEXELEMENT9 VertexColElements[] = 
	{
		{0, 0,  D3DDECLTYPE_FLOAT3, D3DDECLMETHOD_DEFAULT, D3DDECLUSAGE_POSITION, 0},
		{0, 12, D3DDECLTYPE_D3DCOLOR, D3DDECLMETHOD_DEFAULT, D3DDECLUSAGE_COLOR, 0},
		D3DDECL_END()
	};	
	DXChk(pd3dDevice->CreateVertexDeclaration(VertexColElements, &VertexCol::Decl));

	//===============================================================
	// VertexPN

	D3DVERTEXELEMENT9 VertexPNElements[] = 
	{
		{0, 0,  D3DDECLTYPE_FLOAT3, D3DDECLMETHOD_DEFAULT, D3DDECLUSAGE_POSITION, 0},
		{0, 12, D3DDECLTYPE_FLOAT3, D3DDECLMETHOD_DEFAULT, D3DDECLUSAGE_NORMAL, 0},
		D3DDECL_END()
	};	
	DXChk(pd3dDevice->CreateVertexDeclaration(VertexPNElements, &VertexPN::Decl));

	//===============================================================
	// VertexPNT

	D3DVERTEXELEMENT9 VertexPNTElements[] = 
	{
		{0, 0,  D3DDECLTYPE_FLOAT3, D3DDECLMETHOD_DEFAULT, D3DDECLUSAGE_POSITION, 0},
		{0, 12, D3DDECLTYPE_FLOAT3, D3DDECLMETHOD_DEFAULT, D3DDECLUSAGE_NORMAL, 0},
		{0, 24, D3DDECLTYPE_FLOAT2, D3DDECLMETHOD_DEFAULT, D3DDECLUSAGE_TEXCOORD, 0},
		D3DDECL_END()
	};	
	DXChk(pd3dDevice->CreateVertexDeclaration(VertexPNTElements, &VertexPNT::Decl));

	D3DVERTEXELEMENT9 VertexPNT3Elements[] = 
	{
		{0, 0,  D3DDECLTYPE_FLOAT3, D3DDECLMETHOD_DEFAULT, D3DDECLUSAGE_POSITION, 0},
		{0, 12, D3DDECLTYPE_FLOAT3, D3DDECLMETHOD_DEFAULT, D3DDECLUSAGE_NORMAL, 0},
		{0, 24, D3DDECLTYPE_FLOAT3, D3DDECLMETHOD_DEFAULT, D3DDECLUSAGE_TEXCOORD, 0},
		D3DDECL_END()
	};	
	DXChk(pd3dDevice->CreateVertexDeclaration(VertexPNT3Elements, &VertexPNT3::Decl));

	
	
	D3DVERTEXELEMENT9 VertexPNfElements[] = 
	{
		{0, 0,  D3DDECLTYPE_FLOAT3, D3DDECLMETHOD_DEFAULT, D3DDECLUSAGE_POSITION, 0},
		{0, 12, D3DDECLTYPE_FLOAT3, D3DDECLMETHOD_DEFAULT, D3DDECLUSAGE_NORMAL, 0},
		{0, 24, D3DDECLTYPE_FLOAT1, D3DDECLMETHOD_DEFAULT, D3DDECLUSAGE_TEXCOORD, 0},
		D3DDECL_END()
	};	
	DXChk(pd3dDevice->CreateVertexDeclaration(VertexPNfElements, &VertexPNf::Decl));
}

void D3D::DestroyAllVertexDeclarations()
{
	SAFE_RELEASE2(VertexPos::Decl);
	SAFE_RELEASE2(VertexCol::Decl);
	SAFE_RELEASE2(VertexPN::Decl);
	SAFE_RELEASE2(VertexPNT::Decl);
	SAFE_RELEASE2(VertexPNT3::Decl);
	SAFE_RELEASE2(VertexPNf::Decl);
}




HRESULT surfacegraph::InitialiseWithoutBuffers(int vpleft, int vptop, int vpwidth, int vpheight,
											   D3DXVECTOR3 in_Eye, D3DXVECTOR3 in_Lookat,
	bool bOrtho)
{
	
	_controlfp_s(&cw, _EM_INEXACT | _EM_UNDERFLOW | _EM_ZERODIVIDE , _MCW_EM);


	shadow_vp.X = 0;
	shadow_vp.Y = 0;
	shadow_vp.Width = SHADOWMAPRESOLUTION;
	shadow_vp.Height = SHADOWMAPRESOLUTION;
	shadow_vp.MinZ = 0.0f;
	shadow_vp.MaxZ = 1.0f;
	
	mShadowMap = new DrawableTex2D(SHADOWMAPRESOLUTION, SHADOWMAPRESOLUTION, 1, D3DFMT_R32F, true, D3DFMT_D24X8, shadow_vp, false);
	
	//mShadowMap->onResetDevice(); // do any good?

	vp.X = vpleft;
	vp.Y = vptop;
	vp.Width = vpwidth;
	vp.Height = vpheight;
	vp.MinZ = 0.0f;
	vp.MaxZ = 1.0f; // WHY 0 ?
	
	float AspectRatio = ((float)vp.Width)/(float)vp.Height;
	
	
	// Create the FX from a .fx file.
	ID3DXBuffer* errors = 0;
	DXChk(D3DXCreateEffectFromFile(Direct3D.pd3dDevice, "shadow5.fx", //"surf.fx", 
		0, 0, D3DXSHADER_DEBUG | D3DXFX_LARGEADDRESSAWARE, 0, &mFX, &errors));
	// note debug
	
	if( errors ) 
	{
		MessageBox(0, (char*)errors->GetBufferPointer(), 0, 0);	
		PostQuitMessage(3000000);
	};
	if (mFX == 0) {
		MessageBox(0, "oh","mFX==0",MB_OK);
		PostQuitMessage(1000000);
	};
		
	// Obtain handles.
	
	meshTech = mFX->GetTechniqueByName("MeshTech");

	// This will be overwritten the first time RefreshGraphs is called.

	//if (flag == FLAG_SEGUE)
	//{
		mhTech = mFX->GetTechniqueByName("SegueTech");   // Yep
	//} else {
	//	if (flag == FLAG_VELOCITY)
	//	{
	//		mhTech = mFX->GetTechniqueByName("VelociTech");
	//	} else {
	//		if (flag == FLAG_MESH)
	//		{
	//			mhTech = mFX->GetTechniqueByName("MeshTech");
	//		} else {
	//			if (flag == FLAG_CURRENT)
	//			{
	//				mhTech = mFX->GetTechniqueByName("XYZTech");
	//			} else {
	//			//	if (flag == )
	//			//	{
	//			//	}
	//				MessageBox(NULL,"strewth matey","flag not set",MB_OK);
	//			};
	//		};
	//	};
	//};

	mhWVP  = mFX->GetParameterByName(0, "gWVP");
	//mhTex = mFX->GetParameterByName(0, "WarwickTexture");
	
	mhLightVector = mFX->GetParameterByName(0, "LightVecW");

	mhLightDiffuseColour = mFX->GetParameterByName(0, "LightDiffuseColor");
	mhDiffuseMtrl = mFX->GetParameterByName(0, "DiffuseMtrl");

	mhLightAmbientColour = mFX->GetParameterByName(0, "LightAmbientColor");
	mhAmbientMtrl = mFX->GetParameterByName(0, "AmbientMtrl");

	mhLightSpecularColour = mFX->GetParameterByName(0, "LightSpecularColor");
	mhSpecularMtrl = mFX->GetParameterByName(0, "SpecularMtrl");
	mhSpecularPower = mFX->GetParameterByName(0, "SpecularPower");
	mhEyePos = mFX->GetParameterByName(0, "gEyePosW");
	
	mhBuildShadowMapTech = mFX->GetTechniqueByName("ShadowMapTech");
	mhLightWVP           = mFX->GetParameterByName(0, "gLightWVP");
	mhShadowMap          = mFX->GetParameterByName(0, "gShadowMap");

	mhSwitch             = mFX->GetParameterByName(0, "iswitch");

	mhColourMax            = mFX->GetParameterByName(0, "Maxv");

	mhbCullNone = mFX->GetParameterByName(0,"bCullNone");

	mhbTransparency = mFX->GetParameterByName(0,"bTransparent");
	mhfTransparentAlpha = mFX->GetParameterByName(0,"fTransparentAlpha");

	mShadowMap->onResetDevice();

	
	//static D3DXVECTOR3 vLookatPt( 0.0f, 1.2f, 0.0f );
	static D3DXVECTOR3 vUpVec( 0.0f, 1.0f, 0.0f );
    
	Eye = in_Eye;
	Lookat = in_Lookat;

	//Eyesph.x = 8.4f;				// r
	//Eyesph.y = 3.1415927f*0.34f;	// phi
	//Eyesph.z = -3.1415927f*0.3f;	// theta 
	//Eye.x = Eyesph.x * (sin(Eyesph.z));
	//Eye.y = Eyesph.x * (sin(Eyesph.y)*cos(Eyesph.z));
	//Eye.z = -Eyesph.x * (cos(Eyesph.y)*cos(Eyesph.z));
	
	// convert from spherical coordinates to Cartesian:
	// x means r, y means phi (affects y coord most), z means theta (affects z coord most)
// theta is angle around circle that would be made in the x-y plane
// phi is angle around circle in the y-z plane
// theta does not affect z but it does affect how phi affects z
// phi does not affect x
	// note: this is a rel pos: if we translate vLookatPt then we should translate Eye
	
	//D3DXMatrixScaling( &matWorld,1.0f,1.0f,1.0f );
	// changed from identity
	//pd3dDevice->SetTransform( D3DTS_WORLD, &matWorld );
	
	D3DXMatrixLookAtLH( &matView, &Eye, &Lookat, &vUpVec );
	
	// For the projection matrix, we set up a perspective transform (which
	// transforms geometry from 3D view space to 2D viewport space, with
	// a perspective divide making objects smaller in the distance). To build
	// a perpsective transform, we need the field of view (1/4 pi is common),
	// the aspect ratio, and the near and far clipping planes (which define at
	// what distances geometry should be no longer be rendered).
	
	// for some reason this throws "dividing by zero" exception.

	D3DXMatrixPerspectiveFovLH( &matProj, D3DX_PI / 6.0f, AspectRatio, NEAR_CLIPPING_PLANE, FAR_CLIPPING_PLANE );
	// apparently in rel coords - DO NOT set near clipping plane value to be negative

	if (bOrtho) D3DXMatrixOrthoLH(&matProj, 25.0f, 25.0f, NEAR_CLIPPING_PLANE, FAR_CLIPPING_PLANE);
	
	// So here we have set:
	// matWorld, matView, matProj
	
	// set up mLight
	mLight.ambient = D3DXCOLOR(0.5f, 0.5f, 0.5f, 1.0f);
	mLight.diffuse = D3DXCOLOR(0.5f, 0.5f, 0.5f, 1.0f);
	mLight.spec    = D3DXCOLOR(0.6f, 0.6f, 0.6f, 1.0f);
	
	// setting diffuse intensity > 1 does perform as expected
	
	// set up mWhiteMtrl
	mWhiteMtrl.ambient = WHITE*1.0f;
	mWhiteMtrl.diffuse = WHITE*1.0f;
	mWhiteMtrl.spec    = WHITE*1.0f;
	mWhiteMtrl.specPower = 3.2f;
	
	// Shadow mapping:
	
	D3DXMATRIX lightView;
	D3DXVECTOR3 lightPosW(20.0f, 14.0f, -10.0f+DEVICE_RADIUS_INITIAL_FILAMENT_CENTRE*xzscale);
	D3DXVECTOR3 lightTargetW(0.0f, 0.0f, DEVICE_RADIUS_INITIAL_FILAMENT_CENTRE*xzscale);//e.g.
//	D3DXVECTOR3 lightUpW(1.0f, 0.0f, 0.0f);   // um why is up being set to 1,0,0 ? Try 0,1,0
	D3DXVECTOR3 lightUpW(0.0f, 1.0f, 0.0f);	

	D3DXMatrixLookAtLH(&lightView, &lightPosW, &lightTargetW, &lightUpW);

	D3DXMATRIX lightLens;
	float lightFOV = D3DX_PI*0.25f; // see if this changes anything
	D3DXMatrixPerspectiveFovLH(&lightLens, lightFOV, AspectRatio, NEAR_CLIPPING_PLANE, FAR_CLIPPING_PLANE);
	if (bOrtho) D3DXMatrixOrthoLH(&lightLens,25.0f,25.0f,NEAR_CLIPPING_PLANE,FAR_CLIPPING_PLANE);
	// 10,10 ?
	mLightVP = lightView*lightLens;
	mLight.dirW = lightTargetW-lightPosW;
	D3DXVec3Normalize(&mLight.dirW, &mLight.dirW);
	
	_controlfp_s(0,cw, _MCW_EM);

	return S_OK;
}

HRESULT surfacegraph::SetEyePlan(const D3DXVECTOR3 & newEye)
{
	_controlfp_s(&cw, _EM_INEXACT | _EM_UNDERFLOW | _EM_ZERODIVIDE , _MCW_EM);

	D3DXVECTOR3 vUpVec( 0.0f, 0.0f, 1.0f );   // DIFFERENT

	Eye = newEye;
	
	Lookat.x = Eye.x;
	Lookat.y = 0.0;
	Lookat.z = Eye.z*1.000001; // quite possible that where they are equal, this is making us spin?

	D3DXMatrixLookAtLH( &matView, &Eye, &Lookat, &vUpVec );
	
	_controlfp_s(0,cw , _MCW_EM);

	return S_OK;
}

HRESULT surfacegraph::SetEyeAndLookat(const D3DXVECTOR3 & newEye,
							 const D3DXVECTOR3 & newLookat)
{
	_controlfp_s(&cw, _EM_INEXACT | _EM_UNDERFLOW | _EM_ZERODIVIDE , _MCW_EM);

	D3DXVECTOR3 vUpVec( 0.0f, 1.0f, 0.0f );  

	Eye = newEye;
	Lookat = newLookat;
	
	D3DXMatrixLookAtLH( &matView, &Eye, &Lookat, &vUpVec );
	
	_controlfp_s(0,cw , _MCW_EM);

	return S_OK;
}

HRESULT surfacegraph::SetEye_NotPlan(const D3DXVECTOR3 & newEye)
{
	_controlfp_s(&cw, _EM_INEXACT | _EM_UNDERFLOW | _EM_ZERODIVIDE , _MCW_EM);

	D3DXVECTOR3 vUpVec( 0.0f, 1.0f, 0.0f );  

	Eye = newEye;
	
	D3DXMatrixLookAtLH( &matView, &Eye, &Lookat, &vUpVec );
	
	_controlfp_s(0,cw , _MCW_EM);

	return S_OK;
}

HRESULT surfacegraph::SetLookat_NotPlan(const D3DXVECTOR3 & newLookat)
{
	_controlfp_s(&cw, _EM_INEXACT | _EM_UNDERFLOW | _EM_ZERODIVIDE , _MCW_EM);

	D3DXVECTOR3 vUpVec( 0.0f, 1.0f, 0.0f );  

	Lookat = newLookat;
	
	D3DXMatrixLookAtLH( &matView, &Eye, &Lookat, &vUpVec );
	_controlfp_s(0, cw, _MCW_EM);

	return S_OK;
}


HRESULT surfacegraph::InitialiseBuffers(const TriMesh & X)
{
	int j,failvertex,failindex;

	// Plan to deal with too large number of vertices for a buffer:
	
	// Pick a radius, split by radius, (get acceptable number in band),
	// include those that are neighbours outside the radius;
	// use triangles that apply to only these vertices.

	// Cannot assume the vertices are in any particular order.

	// For now we stuff all into one buffer.
	long numTrianglesKey;

	numVerticesKey = X.GetNumKeyVerticesGraphics(&numTrianglesKey);

	// Dimension buffer[0] for key. 

	if ((numVerticesKey != numVertices[0]) || (numTrianglesKey != numTriangles[0]))
	{
		SAFE_RELEASE2(VertexBuffer[0]);

		failvertex = DXChk(
						Direct3D.pd3dDevice->CreateVertexBuffer(
						numVerticesKey*sizeof(VertexPNT3),
						D3DUSAGE_WRITEONLY,
						0,	
						D3DPOOL_MANAGED,
						&VertexBuffer[0],
						0),1);

		SAFE_RELEASE2(IndexBuffer[0]);

		failindex = DXChk( 
					Direct3D.pd3dDevice->CreateIndexBuffer(
						numTrianglesKey*3*sizeof(DWORD),
						D3DUSAGE_WRITEONLY,
						D3DFMT_INDEX32,
						D3DPOOL_MANAGED,
						&IndexBuffer[0],
						0),2);

		if (failvertex || failindex)
		{
			MessageBox(NULL,"serious probs","graphics mem not allocated 0",MB_OK);
			return E_FAIL;
		};
		
		numVertices[0] = numVerticesKey;
		numTriangles[0] = numTrianglesKey; // successfully dimmed this amt.

	};

	// Other times we might do something else with the key buffer.

	// Start graph array at buffer 1.

	// This quick test for now; when we use more arrays, we need to store total in another variable...

	if (1) { // this->boolDisplayInnerMesh) {

		// New policy: for sanity we need to always output all vertices. Otherwise we'd have to track and map
		// what drawn vertex corresponds to what used one, if we cannot guarantee that domain vertices are
		// indexed after insulator ones.

		numVerticesTotal = X.numVertices;
		numTrianglesTotal = X.numTriangles;
		const Triangle * pTri = &(X.T[0]);
		for (long iTri = 0; iTri < X.numTriangles; iTri++)
		{
			if (pTri->periodic) numVerticesTotal+=2; // add some for periodic
			++pTri;
		};
	} else {
		numVerticesTotal = X.numDomainVertices;
		// count triangles that do not involve inner vertices.

		// At a future time to be revealed, we shall try to include tri 'centres' on the insulator
		// in the graph. But not right now.

		// count through triangles to see which ones are in domain.
		numTrianglesTotal = 0;
		const Triangle * pTri = &(X.T[0]);
		for (long iTri = 0; iTri < X.numTriangles; iTri++)
		{
			if (pTri->u8domain_flag == DOMAIN_TRIANGLE){
				numTrianglesTotal++;
				if (pTri->periodic) numVerticesTotal+=2; // add some for periodic
			};
			++pTri;
		};
	};
	if (numVerticesTotal > VERTICES_PER_ARRAY) 
	{
		printf("Warning -- too many vertices for array? Code needs adding.\n");
		getch();
	};
	{
		if( (numVerticesTotal != numVertices[1]) || (numTrianglesTotal != numTriangles[1]))
		{
			SAFE_RELEASE2(VertexBuffer[1]);
			failvertex = DXChk(
							Direct3D.pd3dDevice->CreateVertexBuffer(
							numVerticesTotal*sizeof(VertexPNT3),
							D3DUSAGE_WRITEONLY,
							0,	
							D3DPOOL_MANAGED,
							&VertexBuffer[1],
							0),1);

			SAFE_RELEASE2(IndexBuffer[1]);
			failindex = DXChk( 
						Direct3D.pd3dDevice->CreateIndexBuffer(
							numTrianglesTotal*3*sizeof(DWORD),
							D3DUSAGE_WRITEONLY,
							D3DFMT_INDEX32,
							D3DPOOL_MANAGED,
							&IndexBuffer[1],
							0),2);
		

			if (failvertex || failindex)
			{
				MessageBox(NULL,"serious probs","graphics mem not allocated",MB_OK);
				return E_FAIL;
			};
			
			numVertices[1] = numVerticesTotal;
			numTriangles[1] = numTrianglesTotal; // successfully dimmed this amt.


			// where is texture created for shadow map?


			// For transparent triangles:
			// take 1/3 of size.

			SAFE_RELEASE2(IndexBuffer[2]);
			failindex = DXChk( 
						Direct3D.pd3dDevice->CreateIndexBuffer(
							numTrianglesTotal*3*sizeof(DWORD),
							D3DUSAGE_WRITEONLY,
							D3DFMT_INDEX32,
							D3DPOOL_MANAGED,
							&IndexBuffer[2],
							0),2);

			// dummy: allocate 1 vertex only
			SAFE_RELEASE2(VertexBuffer[2]);
			failvertex = DXChk(
							Direct3D.pd3dDevice->CreateVertexBuffer(
							1*sizeof(VertexPNT3),
							D3DUSAGE_WRITEONLY,
							0,	
							D3DPOOL_MANAGED,
							&VertexBuffer[2],
							0),1);

			if (failvertex || failindex)
			{
				MessageBox(NULL,"serious probs","graphics mem not allocated",MB_OK);
				return E_FAIL;
			};

			numTriangles[2] = numTrianglesTotal; // successfully dimmed this amt.
			

		// er, what is the following used for?
		//????????????????????????????????????????

		//D3DSURFACE_DESC textureDesc;
		//texture_grid->GetLevelDesc(0, &textureDesc);
		//if ( textureDesc.Format != D3DFMT_X8R8G8B8 ) return E_FAIL;
		
		};
	};

	return S_OK;
}

HRESULT surfacegraph::InitialiseBuffersAux(TriMesh & X, int iLevel,
										   int NTris)
{
	int j,failvertex,failindex;// ,N

//	numVertices *= 2;
//	numTriangles *= 2;    // this is the number to dimension

	// set Number_Rows_Vertex_Array[] -- we might as well do here, it's not expensive
//	j = NUMBER_BLOCKS_Y;
//	for (N = 0; N < NUMBER_VERTEX_ARRAYS; N++)
//	{
//		if (j >= BLOCK_ROWS_PER_VERTEX_ARRAY)
//		{
//			j -= BLOCK_ROWS_PER_VERTEX_ARRAY;
//			Number_Rows_Vertex_Array[N] = BLOCK_ROWS_PER_VERTEX_ARRAY;
//		} else {
//			Number_Rows_Vertex_Array[N] = j;
//			j = 0;
//		};
//	};
			
	if (bScrewPinch) {
		numVerticesTotal = X.numAuxVertices[iLevel];
	} else {
		// allow for periodic extras:
		numVerticesTotal = X.GetNumVerticesGraphicsAux(iLevel);
	};
	numTrianglesTotal = X.numAuxTriangles[iLevel]; // only display the actual ones, once.
		
	numVerticesUsed[0] = 0;
	numTrianglesUsed[0] = 0;
	numVerticesUsed[1] = 0;
	numTrianglesUsed[1] = 0;

	if( (numVerticesTotal != numVertices[1]) || (numTrianglesTotal != numTriangles[1]))
	{

		SAFE_RELEASE2(VertexBuffer[1]);

		failvertex = DXChk(
						Direct3D.pd3dDevice->CreateVertexBuffer(
						numVerticesTotal*sizeof(VertexPNT3),
						D3DUSAGE_WRITEONLY,
						0,	
						D3DPOOL_MANAGED,
						&VertexBuffer[1],
						0),1);

		SAFE_RELEASE2(IndexBuffer[1]);

		failindex = DXChk( 
					Direct3D.pd3dDevice->CreateIndexBuffer(
						numTrianglesTotal*3*sizeof(DWORD),
						D3DUSAGE_WRITEONLY,
						D3DFMT_INDEX32,
						D3DPOOL_MANAGED,
						&IndexBuffer[1],
						0),2);

		if (failvertex || failindex)
		{
			MessageBox(NULL,"serious probs","graphics mem not allocated, aux",MB_OK);
			return E_FAIL;
		};
		
		numVertices[1] = numVerticesTotal;
		numTriangles[1] = numTrianglesTotal; // successfully dimmed this amt.
		if (NTris > 0) numTriangles[1] = NTris;//+50
		//if (numTriangles[1] > numTrianglesTotal)
		//		numTriangles[1] = numTrianglesTotal; // 
	};

	return S_OK;
}
/*
HRESULT surfacegraph::Initialise(int nperRow, // data points per row if square grid data to be supplied
								 int left, int top, int width, int height, float fscale, D3DXVECTOR3 in_Eye,
								 bool Polar = 0, 
								 int number_in_outermost = 0, // data points in outermost circle if radially distributed data
								 int number_circles = 0,    // number of circles              if radially distributed data
								 float radius_outermost = 0.0f    //  max radius              if radially distributed data
								 )
{
	float distance;
	int i,j;
	float radius;

	shadow_vp.X = 0;
	shadow_vp.Y = 0;
	shadow_vp.Width = SHADOWMAPRESOLUTION;
	shadow_vp.Height = SHADOWMAPRESOLUTION;
	shadow_vp.MinZ = 0.0f;
	shadow_vp.MaxZ = 1.0f;
	
	mShadowMap = new DrawableTex2D(SHADOWMAPRESOLUTION, SHADOWMAPRESOLUTION, 1, D3DFMT_R32F, true, D3DFMT_D24X8, shadow_vp, false);
	
	//mShadowMap->onResetDevice(); // do any good?

	vp.X = left;
	vp.Y = top;
	vp.Width = width;
	vp.Height = height;
	vp.MinZ = 0.0f;
	vp.MaxZ = 0.0f;

	float GlobalAspectRatio = ((float)vp.Width)/(float)vp.Height;
	

	// Create the FX from a .fx file.
	ID3DXBuffer* errors = 0;
	D3DXCreateEffectFromFile(Direct3D.pd3dDevice, "test1.fx", 
		0, 0, D3DXSHADER_DEBUG, 0, &mFX, &errors);
	// note debug

	if( errors ) 
		MessageBox(0, (char*)errors->GetBufferPointer(), 0, 0);	
	
	// Obtain handles.
	
	mhTech = mFX->GetTechniqueByName("WarwickTech");
	mhWVP  = mFX->GetParameterByName(0, "gWVP");
	mhTex = mFX->GetParameterByName(0, "WarwickTexture");

	mhLightVector = mFX->GetParameterByName(0, "LightVecW");

	mhLightDiffuseColour = mFX->GetParameterByName(0, "LightDiffuseColor");
	mhDiffuseMtrl = mFX->GetParameterByName(0, "DiffuseMtrl");

	mhLightAmbientColour = mFX->GetParameterByName(0, "LightAmbientColor");
	mhAmbientMtrl = mFX->GetParameterByName(0, "AmbientMtrl");

	mhLightSpecularColour = mFX->GetParameterByName(0, "LightSpecularColor");
	mhSpecularMtrl = mFX->GetParameterByName(0, "SpecularMtrl");
	mhSpecularPower = mFX->GetParameterByName(0, "SpecularPower");
	mhEyePos = mFX->GetParameterByName(0, "gEyePosW");
	
	mhBuildShadowMapTech = mFX->GetTechniqueByName("BuildShadowMapTech");
	mhLightWVP           = mFX->GetParameterByName(0, "gLightWVP");
	mhShadowMap          = mFX->GetParameterByName(0, "gShadowMap");

	mhSwitch             = mFX->GetParameterByName(0, "iswitch");



	mShadowMap->onResetDevice();

	
	static D3DXVECTOR3 vLookatPt( 0.0f, 1.2f, 0.0f );
	static D3DXVECTOR3 vUpVec( 0.0f, 1.0f, 0.0f );
    
	Eye = in_Eye;

	//Eyesph.x = 8.4f;				// r
	//Eyesph.y = 3.1415927f*0.34f;	// phi
	//Eyesph.z = -3.1415927f*0.3f;	// theta 
	//Eye.x = Eyesph.x * (sin(Eyesph.z));
	//Eye.y = Eyesph.x * (sin(Eyesph.y)*cos(Eyesph.z));
	//Eye.z = -Eyesph.x * (cos(Eyesph.y)*cos(Eyesph.z));
	

	// convert from spherical coordinates to Cartesian:
	// x means r, y means phi (affects y coord most), z means theta (affects z coord most)
// theta is angle around circle that would be made in the x-y plane
// phi is angle around circle in the y-z plane
// theta does not affect z but it does affect how phi affects z
// phi does not affect x
	// note: this is a rel pos: if we translate vLookatPt then we should translate Eye
	
	//D3DXMatrixScaling( &matWorld,1.0f,1.0f,1.0f );
	// changed from identity
	//pd3dDevice->SetTransform( D3DTS_WORLD, &matWorld );
	
	D3DXMatrixLookAtLH( &matView, &Eye, &vLookatPt, &vUpVec );
	
	// For the projection matrix, we set up a perspective transform (which
	// transforms geometry from 3D view space to 2D viewport space, with
	// a perspective divide making objects smaller in the distance). To build
	// a perpsective transform, we need the field of view (1/4 pi is common),
	// the aspect ratio, and the near and far clipping planes (which define at
	// what distances geometry should be no longer be rendered).
	D3DXMatrixPerspectiveFovLH( &matProj, D3DX_PI / 4.0f, GlobalAspectRatio, 1.0f, 12.0f );
	// apparently in rel coords - DO NOT set near clipping plane value to be negative
	


	// So here we have set:
	// matWorld, matView, matProj
	
	// set up mLight
	mLight.dirW    = D3DXVECTOR3(-1.0f, -0.5f, 0.5f);// remember y is height
	D3DXVec3Normalize(&mLight.dirW, &mLight.dirW);
	mLight.ambient = D3DXCOLOR(0.4f, 0.4f, 0.4f, 1.0f);
	mLight.diffuse = D3DXCOLOR(0.6f, 0.6f, 0.6f, 1.0f);
	mLight.spec    = D3DXCOLOR(0.5f, 0.5f, 0.5f, 1.0f);
	
	// setting diffuse intensity > 1 does perform as expected
	
	// set up mWhiteMtrl
	mWhiteMtrl.ambient = WHITE*1.0f;
	mWhiteMtrl.diffuse = WHITE*1.0f;
	mWhiteMtrl.spec    = WHITE*1.0f;
	mWhiteMtrl.specPower = 8.0f;
	
	// Shadow mapping:
	
	D3DXMATRIX lightView;
	D3DXVECTOR3 lightPosW(20.0f, 10.0f, -10.0f);
	D3DXVECTOR3 lightTargetW(0.0f, 0.0f, 0.0f);
//	D3DXVECTOR3 lightUpW(1.0f, 0.0f, 0.0f);   // um why is up being set to 1,0,0 ? Try 0,1,0
	D3DXVECTOR3 lightUpW(0.0f, 1.0f, 0.0f);	

	D3DXMatrixLookAtLH(&lightView, &lightPosW, &lightTargetW, &lightUpW);
		
	D3DXMATRIX lightLens;
	float lightFOV = D3DX_PI*0.25f;
	D3DXMatrixPerspectiveFovLH(&lightLens, lightFOV, 1.0f, 1.0f, 100.0f);
	
	mLightVP = lightView*lightLens;
	

		
	// From here, create different functions for Cartesian and polar

	// just have a split

	if (Polar == 0)
	{

		perRow = nperRow;

		numVertices = perRow*perRow;
		numTriangles = 2*(perRow-1)*(perRow-1);
	
		texWidth = perRow;
		texHeight = perRow;
		
		definedindices = 0;

	} else {
		perRow = nperRow;

		Ncircles = number_circles;
		// infer how many are in each interior circle accordingly
		
		distance = PI32bit*radius_outermost/(float)number_in_outermost;
				
		numVertices = 0;
		numTriangles = 0;
		
		number_in_circle = new int[Ncircles];

		for (j = number_circles-1; j > 0; j--)
		{
			radius = radius_outermost*((float)j/(float)(number_circles-1));
			
			number_in_circle[j] = (int)(PI32bit*radius/(float)distance)+1;
			if (j == number_circles-1) number_in_circle[j] = number_in_outermost;
			
			numVertices += number_in_circle[j];
			// number of triangles: there will be one on the interior side of this circle
			// for every edge in this circle. // so number-1 ;
			// There will also be one facing every point here
			
		}
		
		// circle 0 just contains the point at the centre:
		numVertices += 1;
		number_in_circle[0] = 1;
		

		for (j = number_circles-1; j > 0; j--)
		{
			numTriangles += number_in_circle[j]+number_in_circle[j-1];

		};
		numTriangles--; // circle of 1 leads to 0 triangles

		// how to handle texture?
		// can we give texture coords that are something strange?
		// Should still be Cartesian:

		texWidth = perRow;
		texHeight = perRow;
		
		indexinner = new int[numVertices];  // choose the nearest point Clockwise from this one, on the inner circle
		indexclockwise = new int[numVertices]; // the nearest point Clockwise on this same circle
		definedindices = 1;

	};
	
	if (
	
	DXChk(
	Direct3D.pd3dDevice->
		CreateVertexBuffer(
			numVertices*sizeof(VertexPNT),
			D3DUSAGE_WRITEONLY,
			0,
			D3DPOOL_MANAGED,
			&TerrainVertexBuffer,
			0)
								,1)
		+ 
	DXChk( 
	Direct3D.pd3dDevice->
		CreateIndexBuffer(
			numTriangles*3*sizeof(DWORD),
			D3DUSAGE_WRITEONLY,
			D3DFMT_INDEX32,
			D3DPOOL_MANAGED,
			&TerrainIndexBuffer,
			0)
								,2)
		+
		
	DXChk(
		D3DXCreateTexture(
		   Direct3D.pd3dDevice, 
		   texWidth,texHeight, 0,0,D3DFMT_X8R8G8B8,D3DPOOL_MANAGED,
		   &texture_grid)
							    ,3)			
		)
		return E_FAIL;
	
	D3DSURFACE_DESC textureDesc;
	texture_grid->GetLevelDesc(0, &textureDesc);
	if ( textureDesc.Format != D3DFMT_X8R8G8B8 ) return E_FAIL;
	
	nscale = fscale;

	return S_OK;

}
*/

 
HRESULT surfacegraph::SetDataWithColour(const TriMesh & X, 
										int colourflag, int heightflag, 
										int offset_data, int offset_vcolour,
										//int offset_data_slim, int offset_vcolour_slim, 
										int code)
										// need to add code so that we can check historic data max and for absolute per-variable colour scales
{

	DWORD * indices[NUMBER_VERTEX_ARRAYS];
	VertexPNT3 * vertices[NUMBER_VERTEX_ARRAYS];
	//int numIndicesUsed[NUMBER_VERTEX_ARRAYS]; // not used
	//int numVerticesUsed[NUMBER_VERTEX_ARRAYS]; // class member

	// Moved here from CreateSurfaceGraphs
	// because it depends on boolDisplayInnerMesh being set correctly:
	this->InitialiseBuffers(X);

	int i,N;
	real maximum, minimum, max2, min2;

	if (heightflag != FLAG_FLAT_MESH)
	{
		if (heightflag == FLAG_VELOCITY_HEIGHT)
		{
			maximum = X.ReturnL4_Velocity(offset_data,
				this->boolDisplayInnerMesh);		
			store_max = X.ReturnMaximumVelocity(offset_data,
				this->boolDisplayInnerMesh);
			
			minimum = 0.0;
			this->store_min = 0.0f;
		} else {
			if (heightflag == FLAG_VEC3_HEIGHT) 
			{ 
			//	maximum = X.ReturnMaximum3DMagnitude(offset_data,
			//		this->boolDisplayInnerMesh);
				maximum = X.ReturnL4_3DMagnitude(offset_data, 
					this->boolDisplayInnerMesh);
				store_max = X.ReturnMaximum3DMagnitude(offset_data, 
					this->boolDisplayInnerMesh);
				
				minimum = 0.0;
				this->store_min = 0.0f;
			} else {
				X.ReturnMaxMinData(offset_data, &store_max, &store_min,
					this->boolDisplayInnerMesh);
			
				X.Return3rdmaxData(offset_data, &maximum, &minimum,
					this->boolDisplayInnerMesh);

	//			printf("*****+++++++\ncode %d minimum %1.3E maximum %1.3E store_max %1.3E store_min %1.4E \n",
	//				code, minimum, maximum, store_max, store_min);
			};
		} 
	} else {

		//  plan view flat mesh case:
		// ---------------------------
		zeroplane = 0.0f;
		yscale = 1.0f;
		
		if (colourflag == FLAG_VELOCITY_COLOUR)
		{
			maximum = X.ReturnL4_Velocity(offset_data,
				this->boolDisplayInnerMesh);		
			store_max = X.ReturnMaximumVelocity(offset_data,
				this->boolDisplayInnerMesh);
			
			minimum = 0.0;
			this->store_min = 0.0f;
		} else {
			if (colourflag == FLAG_CURRENT_COLOUR) 
			{ 
				maximum = X.ReturnL4_3DMagnitude(offset_data, 
					this->boolDisplayInnerMesh);
				store_max = X.ReturnMaximum3DMagnitude(offset_data, 
					this->boolDisplayInnerMesh);
				minimum = 0.0;
				this->store_min = 0.0f;
			} else {
				X.ReturnMaxMinData(offset_data, &store_max, &store_min,
					this->boolDisplayInnerMesh);
				X.ReturnL5Data(offset_data, &maximum, &minimum,
					this->boolDisplayInnerMesh);
				if (VERBOSEGRAPHICS) printf("SDWC colourflag %d code %d minimum %1.3E maximum %1.3E store_max %1.3E store_min %1.4E \n",
					colourflag, code, minimum, maximum, store_max, store_min);

			};
		};
	};

	if (_isnan(maximum) || (!_finite(maximum)) || _isnan(minimum) || (!_finite(minimum)) )
	{
		printf("maximum %1.5E minimum %1.5E offset %d ",maximum, minimum, offset_data);
		getch();
	};	
	
	// Now decide on the actual max to use: set zeroplane, yscale.
	int powermax, powermin;
	if (maximum > 0.0) {
		real logmaxbase_ours = log(maximum)/log(GRAPH_SCALE_GEOMETRIC_INCREMENT);
		powermax = (int)logmaxbase_ours+1;

		if ((boolGlobalHistory) && (powermax == Historic_powermax[code]-1))
			powermax++;

		maximum = pow(GRAPH_SCALE_GEOMETRIC_INCREMENT,powermax);
	} else {
		powermax = 0;
	};
	if (minimum < 0.0) {
		real logminbase_ours = log(-minimum)/log(GRAPH_SCALE_GEOMETRIC_INCREMENT);
		powermin = (int)logminbase_ours+1;

		if ((boolGlobalHistory) && (powermin == Historic_powermin[code]-1))
			powermin++;
		
		minimum = -pow(GRAPH_SCALE_GEOMETRIC_INCREMENT,powermin);
	} else {
		powermin = 0;
	};
	Historic_powermax[code] = powermax;
	Historic_powermin[code] = powermin;

	// Aim to set graphic y value = zeroplane y + factor * value:
	if (maximum > 1.0e40) maximum = 1.0e40;
	if (minimum > 1.0e40) minimum = 1.0e40;
	this->ymax = max(0.0,maximum);
	this->ymin = min(0.0,minimum);
	if (this->ymax > this->ymin) {
		this->yscale = (GRAPHIC_MAX_Y - GRAPHIC_MIN_Y)/(this->ymax - this->ymin);
		if (this->ymax > 0.0)
		{
			if (this->ymin < 0.0)
			{
				zeroplane = GRAPHIC_MIN_Y + 
					(GRAPHIC_MAX_Y - GRAPHIC_MIN_Y)*(-this->ymin/(this->ymax-this->ymin));
			} else {
				zeroplane = GRAPHIC_MIN_Y;
			}
		} else {
			zeroplane = GRAPHIC_MAX_Y;
		};
	} else {
		// all zero
		zeroplane = 0.0f;
		this->yscale = 1.0f;
	};

	

	//printf("*****+++++++\ncode %d zeroplane %f yscale %f ymax %f ymin %f minimum %1.3E maximum %1.3E \n",
	//	code, zeroplane, this->yscale, ymax, ymin, minimum, maximum);

	// We set colourmax as the value we pass to shader -- 
	// and notice that sometimes we have DATA_HEIGHT but _VELOCITY_COLOUR.
	switch(colourflag)
	{
		case FLAG_VELOCITY_COLOUR:
	
		if (heightflag == FLAG_VELOCITY_HEIGHT) {
			// make max for colours match max for height:
				colourmax = maximum; 
			} else {
				colourmax = X.ReturnL4_Velocity(offset_vcolour,this->boolDisplayInnerMesh); // == 0 initially
			};
			if (_isnan(colourmax) || (!_finite(colourmax)))
			{
				printf("colourmax maximum %1.5E minimum %1.5E offset %d ",maximum, minimum, offset_vcolour);
				getch();
			};	
		break;
		case FLAG_SEGUE_COLOUR:
			colourmax = 1.0; 

			break;
		case FLAG_IONISE_COLOUR:
		case FLAG_PPN_COLOUR:
			// For temperature want the following absolute scale;
			colourmax = 1.0;
			// use code to determine if this is temperature or what.
			break;
		case FLAG_AZSEGUE_COLOUR:

			X.Return3rdmaxData(offset_vcolour, &maximum, &minimum,
				this->boolDisplayInnerMesh);
			colourmax = max(maximum, fabs(minimum)); // scale both + and - according to this.

			break;
		case FLAG_CURRENT_COLOUR:
			colourmax = X.ReturnL4_3DMagnitude(offset_vcolour,this->boolDisplayInnerMesh
				); // max modulus of J
			
		//	if (_isnan(colourmax) || (!_finite(colourmax)))	
			{
				printf("FLAG_CURRENT_COLOUR maximum %1.5E minimum %1.5E offset %d flag %d ",maximum, minimum, offset_vcolour, code);
		//	getch();
			};	
			break;
	};
	if (VERBOSEGRAPHICS) printf("Lock vertex buffers :\n");
	for (N = 0; N < NUMBER_VERTEX_ARRAYS; N++)
	{
		
		if ( VertexBuffer[N] != NULL)
		{
			if (DXChk(VertexBuffer[N]->Lock(0,0,(void **)&vertices[N],0)) ||
				DXChk( IndexBuffer[N]->Lock(0,0, (void **)&indices[N],0)) )
					MessageBox(NULL,"oh dear","lock failed",MB_OK);
		};
	};
	if (this->boolDisplayMainMesh) // should be always on!
	{
		if (VERBOSEGRAPHICS) printf("Call SetVerticesAndIndices:\n");
		// At the moment this now just pours information into vertices[2], indices[2] :
		X.SetVerticesAndIndices(vertices, indices,       // better to do in the other class...
							numVertices, numTriangles, // pass it the integer counts so that it can test for overrun & redim
		                    numVerticesUsed, numTrianglesUsed,
							colourflag,heightflag, offset_data, offset_vcolour, zeroplane, yscale, this->boolDisplayInnerMesh);
	} else {
		numVerticesUsed[1] = 0;
		numTrianglesUsed[1] = 0;
	};
	
	if ((this->boolDisplayKeyButton) && (bScrewPinch == 0))
	{
		if (VERBOSEGRAPHICS) printf("Call SetVerticesKeyButton: %d", numVerticesKey);
		X.SetVerticesKeyButton(vertices[0],indices[0],colourmax,colourflag);
		numVerticesUsed[0] = X.GetNumKeyVerticesGraphics(&(numTrianglesUsed[0]));
	} else {
		numVerticesUsed[0] = 0;
		numTrianglesUsed[0] = 0;
	};
	
	//if (this->boolDisplayInnerMesh)
	//{
	//	X.SetVerticesAndIndicesInner(vertices[1],indices[1],
	//								numVertices[1],
	//								colourflag,heightflag,offset_data_slim,offset_vcolour_slim,
	//								zeroplane, yscale);
	//	numVerticesUsed[1] = X.numInnerVertices;
	//	numTrianglesUsed[1] = X.numInnerTriangles; 
	//} else {
	//	numVerticesUsed[1] = 0;
	//	numTrianglesUsed[1] = 0;
	//};
	
	//for (N = 0; N < NUMBER_VERTEX_ARRAYS; N++)
	//	numTrianglesUsed[N] = numIndicesUsed[N]/3;
	
	for (N = 0; N < NUMBER_VERTEX_ARRAYS; N++)
		for (i = 0; i < numVerticesUsed[N]; i++)
			vertices[N][i].normal = -vertices[N][i].normal;
	
	for (N = 0; N < NUMBER_VERTEX_ARRAYS; N++)
	{
		if ( VertexBuffer[N] != NULL)
		{
			VertexBuffer[N]->Unlock();
			IndexBuffer[N]->Unlock();
		};
	};
		
	if (VERBOSEGRAPHICS) printf("end SDWC\n");

	return S_OK;
}
/*
HRESULT surfacegraph::SetDataWithColourAux(TriMesh & X, int iLevel, int colourflag, 
										   int heightflag, int offset_data, int offset_vcolour,
										   int NTris // introduced for debugging
										   )
{
		
	DWORD * indices[NUMBER_VERTEX_ARRAYS];
	VertexPNT3 * vertices[NUMBER_VERTEX_ARRAYS];
	real maximum, minimum;
	int i,N;

	if (heightflag != FLAG_FLAT_MESH)
	{

		X.ReturnMaxMinDataAux(iLevel,offset_data, &maximum, &minimum);

		if (_isnan(maximum) || (!_finite(maximum)) || _isnan(minimum) || (!_finite(minimum)) )
		{
			printf("maximum %1.5E minimum %1.5E offset %d ",maximum, minimum, offset_data);
			getch();
		};	

		this->ymax = max(0.0,maximum);
		this->ymin = min(0.0,minimum);
		if (this->ymax > this->ymin) {
			this->yscale = (GRAPHIC_MAX_Y - GRAPHIC_MIN_Y)/(this->ymax - this->ymin);
			if (this->ymax > 0.0)
			{
				if (this->ymin < 0.0)
				{
					zeroplane = GRAPHIC_MIN_Y + 
						(GRAPHIC_MAX_Y - GRAPHIC_MIN_Y)*(-this->ymin/(this->ymax-this->ymin));
				} else {
					zeroplane = GRAPHIC_MIN_Y;
				}
			} else {
				zeroplane = GRAPHIC_MAX_Y;
			};
		} else {
			// all zero
			zeroplane = 0.0f;
			this->yscale = 1.0f;
		};
		this->colourmax = max(maximum, fabs(minimum)); 

	} else {
		//  plan view
		zeroplane = 0.0f;
		yscale = 1.0f;
		this->colourmax = 1.0; 
	};

	// DEBUG:
	// if (iLevel < 2) colourmax = 1.0;
	// purpose?

	for (N = 0; N < NUMBER_VERTEX_ARRAYS; N++)
	{
		if ( VertexBuffer[N] != NULL)
		{
			if (DXChk(VertexBuffer[N]->Lock(0,0,(void **)&vertices[N],0)) ||
				DXChk(IndexBuffer[N]->Lock(0,0,(void **)&indices[N],0)) )
					MessageBox(NULL,"oh dear","lock failed",MB_OK);
		};
	};
	
	X.SetVerticesAndIndicesAux(iLevel, vertices[1], indices[1],
			numVertices[1], numTriangles[1], // pass it the dimmed counts so that it can test for memory overrun 
		    colourflag,heightflag, offset_data, offset_vcolour, 
			zeroplane, yscale,
			NTris); 
	numVerticesUsed[1] = X.numAuxVertices[iLevel];
	numTrianglesUsed[1] = X.numAuxTriangles[iLevel];
	//printf("numTrianglesUs1ed[1] %d \n",numTrianglesUsed[1] );

	for (N = 0; N < NUMBER_VERTEX_ARRAYS; N++)
		for (i = 0; i < numVerticesUsed[N]; i++)
			vertices[N][i].normal = -vertices[N][i].normal;

	for (N = 0; N < NUMBER_VERTEX_ARRAYS; N++)
	{
		if ( VertexBuffer[N] != NULL)
		{
			VertexBuffer[N]->Unlock();
			IndexBuffer[N]->Unlock();
		};
	};
		
	return S_OK;
}

VOID surfacegraph::RenderAux(const char * szTitle, int const iLabels, const TriMesh * pX, int iLevel)
{
	// Simplified version for auxiliary meshes.

	char buffer[128];
	D3DXMATRIXA16 matWorld;

	Direct3D.pd3dDevice->SetViewport(&vp); 

	mShadowMap->beginScene();
	Direct3D.pd3dDevice->Clear( 0, 0, D3DCLEAR_TARGET | D3DCLEAR_ZBUFFER,
		//0x00000000, 1.0f,0);			    
		D3DCOLOR_XRGB( 255, 255, 255 ), 1.0f, 0 );  

	DXChk(mFX->SetTechnique(mhBuildShadowMapTech));
	
	UINT numPasses = 1;
	DXChk(mFX->Begin(&numPasses, 0));	
	DXChk(mFX->SetMatrix(mhLightWVP, &(mLightVP)));
	DXChk(mFX->SetTexture(mhShadowMap, mShadowMap->d3dTex()));
	Direct3D.pd3dDevice->SetVertexDeclaration(VertexPNT3::Decl);
	
	DXChk(mFX->CommitChanges());
	for (UINT ii = 0; ii < numPasses; ++ii)
	{
		DXChk(mFX->BeginPass(ii));
		int N = 1;
		{
			DXChk(Direct3D.pd3dDevice->SetStreamSource(0, VertexBuffer[N], 0, sizeof(VertexPNT3)));
			DXChk(Direct3D.pd3dDevice->SetIndices(IndexBuffer[N]));
			DXChk(Direct3D.pd3dDevice->DrawIndexedPrimitive(
				D3DPT_TRIANGLELIST,
				0,
				0,
				numVerticesUsed[N],
				0,
				numTrianglesUsed[N]));
		};
		
		DXChk(mFX->EndPass());
	};
	DXChk(mFX->End());
	mShadowMap->endScene();
	
	// ==================================================

	D3DXMatrixIdentity(&matWorld);
	Direct3D.pd3dDevice->SetTransform( D3DTS_WORLD, &matWorld );
	Direct3D.pd3dDevice->SetTransform( D3DTS_VIEW, &matView );
	Direct3D.pd3dDevice->SetTransform( D3DTS_PROJECTION, &matProj );
	Direct3D.pd3dDevice->SetRenderState(D3DRS_ALPHABLENDENABLE , false);
	if ((GlobalCutaway) || (bCullNone)) {
		Direct3D.pd3dDevice->SetRenderState( D3DRS_CULLMODE, D3DCULL_NONE );	
	} else {
		Direct3D.pd3dDevice->SetRenderState( D3DRS_CULLMODE, D3DCULL_CCW );	
	};

	if( SUCCEEDED( Direct3D.pd3dDevice->BeginScene() ) )
    {
		
		DXChk(Direct3D.pd3dDevice->Clear( 0, NULL, D3DCLEAR_TARGET | D3DCLEAR_ZBUFFER,
						 D3DCOLOR_XRGB( 240, 255, 250 ), 1.0f, 0 ), 233);
		
		// The following parameters only do anything if the shader picks them up.

		DXChk(mFX->SetValue(mhLightVector, &mLight.dirW, sizeof(D3DXVECTOR3)));
		DXChk(mFX->SetValue(mhLightDiffuseColour, &mLight.diffuse, sizeof(D3DXCOLOR)));
		DXChk(mFX->SetValue(mhDiffuseMtrl, &mWhiteMtrl.diffuse, sizeof(D3DXCOLOR)));
		DXChk(mFX->SetValue(mhLightAmbientColour, &mLight.ambient, sizeof(D3DXCOLOR)));
		DXChk(mFX->SetValue(mhAmbientMtrl, &mWhiteMtrl.ambient, sizeof(D3DXCOLOR)));
		DXChk(mFX->SetValue(mhLightSpecularColour, &mLight.spec, sizeof(D3DXCOLOR)));
		DXChk(mFX->SetValue(mhSpecularPower, &mWhiteMtrl.specPower, sizeof(float)));
		DXChk(mFX->SetValue(mhSpecularMtrl, &mWhiteMtrl.spec, sizeof(D3DXCOLOR)));
		DXChk(mFX->SetValue(mhEyePos, &Eye, sizeof(D3DXVECTOR3)));
		DXChk(mFX->SetFloat(mhColourMax,colourmax));
		
		DXChk(mFX->SetBool(mhbTransparency, false));
		Direct3D.pd3dDevice->SetRenderState(D3DRS_ALPHABLENDENABLE , false);	
		DXChk(mFX->SetMatrix(mhWVP, &(matView*matProj)));
		DXChk(mFX->SetMatrix(mhLightWVP, &(mLightVP)));
		DXChk(mFX->SetTexture(mhShadowMap, mShadowMap->d3dTex()));
		DXChk(mFX->SetTechnique(mhTech));
		
		DXChk(mFX->CommitChanges());
				
		Direct3D.pd3dDevice->SetVertexDeclaration(VertexPNT3::Decl);

		mFX->Begin(&numPasses,0); // begins technique
		mFX->BeginPass(0);
		
		int N = 1;
		//for (int N = 0; N < 2; N++)
		{
			Direct3D.pd3dDevice->SetStreamSource(0, VertexBuffer[N], 0, sizeof(VertexPNT3));		
			Direct3D.pd3dDevice->SetIndices(IndexBuffer[N]);
		    Direct3D.pd3dDevice->DrawIndexedPrimitive(
				D3DPT_TRIANGLELIST,
				0,
				0,
				numVerticesUsed[N],
				0,
				numTrianglesUsed[N]);	
		};
		printf("verts rendered %d tris %d \n",numVerticesUsed[N],numTrianglesUsed[N]);

		mFX->EndPass();
		mFX->End();

		D3DXVECTOR3 norm(0.0,0.0,1.0);
		D3DXVECTOR3 norm2(0.0,0.0,1.0); // second tri becomes lit - weird

		if (boolDisplayMeshWireframe)
		{		
			if (boolClearZBufferBeforeWireframe) {
				Direct3D.pd3dDevice->Clear( 0, NULL, D3DCLEAR_ZBUFFER, (D3DCOLOR)0, 1.0f, 0 );
			};
			numPasses = 1;
			DXChk(mFX->SetTechnique(meshTech));
			DXChk(mFX->CommitChanges());
			mFX->Begin(&numPasses,0); // begins technique
			mFX->BeginPass(0);

			Direct3D.pd3dDevice->SetStreamSource(0, VertexBuffer[1], 0, sizeof(VertexPNT3));
			Direct3D.pd3dDevice->SetIndices(IndexBuffer[1]);
			Direct3D.pd3dDevice->DrawIndexedPrimitive(
				D3DPT_TRIANGLELIST,
				0,
				0,
				numVerticesUsed[1],
				0,
				numTrianglesUsed[1]);
			printf("verts rendered %d tris %d \n",numVerticesUsed[1],numTrianglesUsed[1]);

			mFX->EndPass();
			mFX->End();			
		};	

		if (iLabels == 1) {

			Vertex * pVertex = pX->AuxX[iLevel];
			for (long iVertex = 0; iVertex < pX->numAuxVertices[iLevel]; iVertex++)
			{
				sprintf(buffer,"%d",pVertex->iVolley);
				
				RenderLabel2(buffer,  // text
						pVertex->pos.x*xzscale,
						0.00001f+zeroplane,
						pVertex->pos.y*xzscale,0); 
				++pVertex;
			};
		};

		Direct3D.pd3dDevice->EndScene();
	};
}*/

VOID surfacegraph::Render(const char * szTitle, bool RenderTriLabels, 
						  const TriMesh * pX, // = 0 by default
						  char * szLinebelow) // = 0 by default
{
	if (VERBOSEGRAPHICS) printf("Render: szTitle %s \n", szTitle);

	long tri_len, izTri[128];

	//static DWORD time = timeGetTime();
	//DWORD oldtime;
	//float timestep, temporary;
	//int i;
	int iSwitch = 0;
	D3DXMATRIXA16 matWorld;
	//RECT rect;

	vertex1 linedata[10000];
	vertex1 linedata2[12];

		float x,y,z;
		char buffer[256];
		int i;
	real tempval;
	   
	//float values[11];
	//char buffer[2048];
	//D3DRECT dr;
	
	// DRAW SHADOW MAP:
	
	Direct3D.pd3dDevice->SetViewport(&vp); 
	// without this, 5 graphs, all except the last rendered, appear as small shadows on white

	mShadowMap->beginScene();
	
	Direct3D.pd3dDevice->Clear( 0, 0, D3DCLEAR_TARGET | D3DCLEAR_ZBUFFER,
		//0x00000000, 1.0f,0);			    
		D3DCOLOR_XRGB( 255, 255, 255 ), 1.0f, 0 );   
	// this clearly does affect shadow map successfully...         
		//0x00000000, 1.0f, 0 ); // Luna's code		// because *this* leads to everything being in shadow

	// at the moment, whatever value we put here, is staying the same.
	// in other words, even a shader with "return 0" is having no effect.
	
	DXChk(mFX->SetTechnique(mhBuildShadowMapTech));	
	UINT numPasses = 1;
	DXChk(mFX->Begin(&numPasses, 0));
	DXChk(mFX->SetMatrix(mhLightWVP, &(mLightVP)));
	DXChk(mFX->SetTexture(mhShadowMap, mShadowMap->d3dTex()));
	
	//if (flag == FLAG_SEGUE)
	//{
	//	Direct3D.pd3dDevice->SetVertexDeclaration(VertexPNf::Decl);
	//} else {
	//	if (flag == FLAG_VELOCITY)
	//	{
	Direct3D.pd3dDevice->SetVertexDeclaration(VertexPNT3::Decl);
	//	} else {
	//		MessageBox(NULL,"A flag was not set.","bad news",MB_OK);
	//	};
	//};
	
	DXChk(mFX->CommitChanges());
	
	if (boolDisplayShadow == false) numPasses = 0; // try to get rid of shadows?

	//printf ("Shadow\n");
	for (UINT ii = 0; ii < numPasses; ++ii)
	{
		DXChk(mFX->BeginPass(ii));
		
		// Key button etc = array 0 which casts no shadow.
		// Inner mesh = array 1
		// We do not always have it populated. Easy way for now: inner mesh casts no shadow.

	//	for (int N = 1; N <= NUMBER_VERTEX_ARRAYS; N++) 
		int N = 1;
		{
			DXChk(Direct3D.pd3dDevice->SetStreamSource(0, VertexBuffer[N], 0, sizeof(VertexPNT3)));
			DXChk(Direct3D.pd3dDevice->SetIndices(IndexBuffer[N]));
			DXChk(Direct3D.pd3dDevice->DrawIndexedPrimitive(
				D3DPT_TRIANGLELIST,
				0,
				0,
				numVerticesUsed[N],
				0,
				numTrianglesUsed[N]));
	//		printf("%d size %d -- ",N,numVerticesUsed[N]);
		};
		
		DXChk(mFX->EndPass());
	};
	
	DXChk(mFX->End());
	mShadowMap->endScene();
	
	//SAFE_RELEASE2(mCylinder);
	
	// for axis drawing in fixed function pipeline:
	//Direct3D.pd3dDevice->SetRenderTarget(0,0); // invalid call
	Direct3D.pd3dDevice->SetViewport(&vp);
	
	D3DXMatrixIdentity(&matWorld);
	Direct3D.pd3dDevice->SetTransform( D3DTS_WORLD, &matWorld );
	Direct3D.pd3dDevice->SetTransform( D3DTS_VIEW, &matView );
	Direct3D.pd3dDevice->SetTransform( D3DTS_PROJECTION, &matProj );
	
	Direct3D.pd3dDevice->Clear( 0, NULL, D3DCLEAR_TARGET | D3DCLEAR_ZBUFFER,
						 D3DCOLOR_XRGB( 240, 255, 250 ), 1.0f, 0 );
		
	if ((GlobalCutaway) || (bCullNone)) {
		Direct3D.pd3dDevice->SetRenderState( D3DRS_CULLMODE, D3DCULL_NONE );	
	} else {
		Direct3D.pd3dDevice->SetRenderState( D3DRS_CULLMODE, D3DCULL_CCW );	
	};
		
	if( SUCCEEDED( Direct3D.pd3dDevice->BeginScene() ) )
    {
		if (bScrewPinch == false) 
		{
			// drawing lines around base:
			
			x = (float)(-sin(HALFANGLE)*DEVICE_RADIUS_INSULATOR_OUTER)*xzscale;
			z = (float)(cos(HALFANGLE)*DEVICE_RADIUS_INSULATOR_OUTER)*xzscale;
			y = zeroplane;
			linedata[0].x = x; linedata[0].y = y; linedata[0].z = z;
			x = (float)(-sin(HALFANGLE)*DOMAIN_OUTER_RADIUS)*xzscale;
			z = (float)(cos(HALFANGLE)*DOMAIN_OUTER_RADIUS)*xzscale;
			linedata[1].x = x; linedata[1].y = y; linedata[1].z = z;
			for (i = 0; i < 12; i++)
				linedata[i].colour = 0xff000000;
			Direct3D.pd3dDevice->SetFVF(point_fvf);
			Direct3D.pd3dDevice->DrawPrimitiveUP(D3DPT_LINESTRIP,1,linedata,sizeof(vertex1));
			
			x = (float)(sin(HALFANGLE)*DEVICE_RADIUS_INSULATOR_OUTER)*xzscale;
			z = (float)(cos(HALFANGLE)*DEVICE_RADIUS_INSULATOR_OUTER)*xzscale;
			y = zeroplane;
			linedata[0].x = x; linedata[0].y = y; linedata[0].z = z;
			x = (float)(sin(HALFANGLE)*DOMAIN_OUTER_RADIUS)*xzscale;
			z = (float)(cos(HALFANGLE)*DOMAIN_OUTER_RADIUS)*xzscale;
			linedata[1].x = x; linedata[1].y = y; linedata[1].z = z;
			for (i = 0; i < 12; i++)
				linedata[i].colour = 0;
			Direct3D.pd3dDevice->SetFVF(point_fvf);
			Direct3D.pd3dDevice->DrawPrimitiveUP(D3DPT_LINESTRIP,1,linedata,sizeof(vertex1));
			
			real theta = -HALFANGLE;
			real r = 3.44;
			for (int asdf = 0; asdf < 10000; asdf++)
			{
				theta += FULLANGLE/10000.0; 
				linedata[asdf].x = -r*sin(theta)*xzscale;
				linedata[asdf].y = this->zeroplane;
				linedata[asdf].z = r*cos(theta)*xzscale;
				linedata[asdf].colour = 0;
			};
				
			Direct3D.pd3dDevice->SetFVF(point_fvf);
			Direct3D.pd3dDevice->DrawPrimitiveUP(D3DPT_LINESTRIP,9999,linedata,sizeof(vertex1));
						
			// Now do 3.6, 3.75, 3.9, 4.05, 4.2
			r = 3.45;
			for (i = 0; i < 12; i++) {
				theta = -HALFANGLE*1.1;
				for (int asdf = 0; asdf < 10000; asdf++)
				{
					theta += FULLANGLE*1.1/10000.0; 
					linedata[asdf].x = r*sin(theta)*xzscale; // note: sin (negative) < 0
					linedata[asdf].z = r*cos(theta)*xzscale;
					
					if (theta < -0.5*HALFANGLE) {
						linedata[asdf].colour = 0xff000000;
					} else {
						linedata[asdf].colour = 0xff444444;
					};
				};					
				Direct3D.pd3dDevice->DrawPrimitiveUP(D3DPT_LINESTRIP,9999,linedata,sizeof(vertex1));

				sprintf(buffer,"%1.2f",r);
				RenderLabel(buffer, linedata[4100].x,zeroplane,linedata[4100].z,true);
				if (i == 0) r = 3.45;
				r += 0.09;
				if (i >= 7) r += 0.11; // last 4
				if (i >= 9) r = 5.52;
				if (i >= 10) r = 6.5;
				if (i > 10) r += 1.0*(double)(i - 10);
			};

			// Vertical lines:
			for (int iSide = 0; iSide <= 1; iSide++)
			for (int iWhich = 0; iWhich <= 2; iWhich++)
			{
				if (iWhich == 0) {
					x = (float)(-sin(HALFANGLE)*DEVICE_RADIUS_INSULATOR_OUTER)*xzscale;
					z = (float)(cos(HALFANGLE)*DEVICE_RADIUS_INSULATOR_OUTER)*xzscale;
				} else {
					if (iWhich == 1) {
						x = (float)(-sin(HALFANGLE)*DOMAIN_OUTER_RADIUS)*xzscale;
						z = (float)(cos(HALFANGLE)*DOMAIN_OUTER_RADIUS)*xzscale;
					} else {
						x = (float)(-sin(HALFANGLE)*0.5*DOMAIN_OUTER_RADIUS)*xzscale;
						z = (float)(cos(HALFANGLE)*0.5*DOMAIN_OUTER_RADIUS)*xzscale;
					}
				};
				
				if (iSide == 1) x = -x;

				linedata[0].x = x; linedata[0].z = z;
				linedata[1].x = x; linedata[1].z = z;
				linedata[0].colour = 0xff000000;
				linedata[1].colour = 0xff000000;
				
				linedata[0].y = GRAPHIC_MIN_Y-1.0f;  linedata[1].y = GRAPHIC_MAX_Y+2.5f;

				Direct3D.pd3dDevice->DrawPrimitiveUP(D3DPT_LINESTRIP,1,linedata,sizeof(vertex1));
			};
		
		} else {
			// screw pinch: circle and semi-circle
			// I think we need a separate "pass" AND CLEAR z buffer
			// if we want stuff to appear on top.

			real theta = 0.0;
			real r = pX->OuterRadiusAttained;
			for (int asdf = 0; asdf < 10000; asdf++)
			{
				theta += 2.0*PI/10000.0; 
				linedata[asdf].x = -r*sin(theta)*xzscale;//(-TRAP_HALFWIDTH+DELTA_0*((real)asdf))*block::xzscale;
				linedata[asdf].y = this->zeroplane;
				linedata[asdf].z = (r*cos(theta)+SP_CENTRE_Y)*xzscale;
				linedata[asdf].colour = 0;
			};
			Direct3D.pd3dDevice->SetFVF(point_fvf);
			Direct3D.pd3dDevice->DrawPrimitiveUP(D3DPT_LINESTRIP,9999,linedata,sizeof(vertex1));

			for (int asdf = 0; asdf < 10000; asdf++)
			{
				theta += PI/10000.0; 
				linedata[asdf].x = -r*cos(theta)*xzscale;
				linedata[asdf].y = GRAPHIC_MAX_Y;
				linedata[asdf].z = (r*sin(theta)+SP_CENTRE_Y)*xzscale;
				linedata[asdf].colour = 0;
			};
			Direct3D.pd3dDevice->SetFVF(point_fvf);
			Direct3D.pd3dDevice->DrawPrimitiveUP(D3DPT_LINESTRIP,9999,linedata,sizeof(vertex1));

				// Vertical lines:
			for (int iSide = 0; iSide <= 1; iSide++)
			for (int iWhich = 0; iWhich <= 1; iWhich++)
			{
				
					// only really want to appear in 2 places ??
				x = (float)(-sin(PI/5.0)*pX->OuterRadiusAttained)*xzscale;
				z = (float)(cos(PI/5.0)*pX->OuterRadiusAttained+SP_CENTRE_Y)*xzscale; // just let it draw twice over
				
				if (iSide == 1) x = -x;

				linedata[0].x = x; linedata[0].z = z;
				linedata[1].x = x; linedata[1].z = z;
				linedata[0].colour = 0xff000000;
				linedata[1].colour = 0xff000000;
				
				linedata[0].y = GRAPHIC_MIN_Y-1.0f;  linedata[1].y = GRAPHIC_MAX_Y+2.5f;

				Direct3D.pd3dDevice->DrawPrimitiveUP(D3DPT_LINESTRIP,1,linedata,sizeof(vertex1));
			};
		};
 	
		// The following parameters only do anything if the shader picks them up.

		DXChk(mFX->SetValue(mhLightVector, &mLight.dirW, sizeof(D3DXVECTOR3)));
		DXChk(mFX->SetValue(mhLightDiffuseColour, &mLight.diffuse, sizeof(D3DXCOLOR)));
		DXChk(mFX->SetValue(mhDiffuseMtrl, &mWhiteMtrl.diffuse, sizeof(D3DXCOLOR)));
		DXChk(mFX->SetValue(mhLightAmbientColour, &mLight.ambient, sizeof(D3DXCOLOR)));
		DXChk(mFX->SetValue(mhAmbientMtrl, &mWhiteMtrl.ambient, sizeof(D3DXCOLOR)));
		DXChk(mFX->SetValue(mhLightSpecularColour, &mLight.spec, sizeof(D3DXCOLOR)));
		DXChk(mFX->SetValue(mhSpecularPower, &mWhiteMtrl.specPower, sizeof(float)));
		DXChk(mFX->SetValue(mhSpecularMtrl, &mWhiteMtrl.spec, sizeof(D3DXCOLOR)));
		DXChk(mFX->SetValue(mhEyePos, &Eye, sizeof(D3DXVECTOR3)));
		DXChk(mFX->SetFloat(mhColourMax,colourmax));
		
		if (VERBOSEGRAPHICS) printf("Render: Setted colourmax == %f \n", colourmax);

		DXChk(mFX->SetBool(mhbTransparency, false));
		Direct3D.pd3dDevice->SetRenderState(D3DRS_ALPHABLENDENABLE , false);	
		
		mFX->SetTechnique(mhTech);
		mFX->SetMatrix(mhWVP, &(matView*matProj));
		
		//mFX->SetTexture(mhTex, texture_grid);      
		
		// shadow map:
		DXChk(mFX->SetMatrix(mhLightWVP, &(mLightVP)));
		DXChk(mFX->SetTexture(mhShadowMap, mShadowMap->d3dTex()));
		
		//if (flag == FLAG_SEGUE)
		//{
		//	Direct3D.pd3dDevice->SetVertexDeclaration(VertexPNf::Decl);
		Direct3D.pd3dDevice->SetVertexDeclaration(VertexPNT3::Decl);
		
		// Looks like we need to be careful how to apply different shaders.
		// Can we do it during a {Begin ... End} block ?
		// We can do it even during BeginPass if we do CommitChanges, supposedly.
		
		DXChk(mFX->SetBool(mhbTransparency, false)); // telling shader to give everything alpha = 1 until further notice.
		
		// Cycle the transparency every 2 ns:
		real timeover = evaltime;
		while (timeover > 2.0e-9) timeover -= 2.0e-9;
		timeover = fabs(timeover-1.0e-9)/1.0e-9; 		
		DXChk(mFX->SetFloat(mhfTransparentAlpha, 0.2f));
			//0.08f + 0.18f*(float)timeover));
		DXChk(mFX->CommitChanges());
		Direct3D.pd3dDevice->SetRenderState(D3DRS_ALPHABLENDENABLE , true);	
		Direct3D.pd3dDevice->SetRenderState(D3DRS_SRCBLEND, D3DBLEND_SRCALPHA);
		Direct3D.pd3dDevice->SetRenderState(D3DRS_DESTBLEND, D3DBLEND_INVSRCALPHA); // try D3DBLEND_ONE
		
		DXChk(mFX->CommitChanges());
							
		numPasses = 1;
		mFX->Begin(&numPasses,0); // begins technique
		mFX->BeginPass(0);
		
		int N;
		for (int N = 0; N < 2; N++)
		{
			Direct3D.pd3dDevice->SetStreamSource(0, VertexBuffer[N], 0, sizeof(VertexPNT3));		
			Direct3D.pd3dDevice->SetIndices(IndexBuffer[N]);
		    Direct3D.pd3dDevice->DrawIndexedPrimitive(
				D3DPT_TRIANGLELIST,
				0,
				0,
				numVerticesUsed[N],
				0,
				numTrianglesUsed[N]);	
		};
		

		if (GlobalCutaway) {
			DXChk(mFX->SetBool(mhbTransparency, true)); // render the rest with low alpha
			DXChk(mFX->CommitChanges());

			Direct3D.pd3dDevice->SetStreamSource(0, VertexBuffer[1], 0, sizeof(VertexPNT3));		
			Direct3D.pd3dDevice->SetIndices(IndexBuffer[2]);
			Direct3D.pd3dDevice->DrawIndexedPrimitive(
				D3DPT_TRIANGLELIST,
				0,
				0,
				numVerticesUsed[1],
				0,
				numTrianglesUsed[2]);
		// Maybe though, this method does cause some ruckus where there are no triangles placed.
		// Try creating separate vertex array...
		};

		mFX->EndPass();

		//DXChk(mFX->SetBool(mhbTransparency, true));
		//DXChk(mFX->SetFloat(mhfTransparentAlpha, 0.08f));
		//DXChk(mFX->CommitChanges());
		

		//Direct3D.pd3dDevice->SetRenderState(D3DRS_ALPHABLENDENABLE , true);	
		//Direct3D.pd3dDevice->SetRenderState(D3DRS_SRCBLEND, D3DBLEND_SRCALPHA);
		//Direct3D.pd3dDevice->SetRenderState(D3DRS_DESTBLEND, D3DBLEND_INVSRCALPHA); // try D3DBLEND_ONE
		
		// If we do it within Begin(,) ... End()
		// then the render state doesn't take.
		// If we do it outside, then what is there from the 1st pass gets overwritten for some reason.

		// The reason we can't do it a sensible way with 1 pair of buffers and 2 passes in shader is that it sees z-fighting
		// of a copy with alpha = 0(I think) of the rest of the surface -- which I cannot understand.

		//numPasses = 1;
		//mFX->BeginPass(1);

		//Direct3D.pd3dDevice->SetStreamSource(0, VertexBuffer[1], 0, sizeof(VertexPNT3));		
		//Direct3D.pd3dDevice->SetIndices(IndexBuffer[2]);
		//Direct3D.pd3dDevice->DrawIndexedPrimitive(
		//		D3DPT_TRIANGLELIST,
		//		0,
		//		0,
		//		numVerticesUsed[1],
		//		0,
		//		numTrianglesUsed[2]);

		//mFX->EndPass();
		
		mFX->End();

		DXChk(mFX->SetBool(mhbTransparency, false));
		DXChk(mFX->CommitChanges());
		Direct3D.pd3dDevice->SetRenderState(D3DRS_ALPHABLENDENABLE , false);	

		D3DXVECTOR3 norm(0.0,0.0,1.0);
		D3DXVECTOR3 norm2(0.0,0.0,1.0); // second tri becomes lit - weird

		// Now try to draw the scales:
		if (this->boolDisplayScales) {

			for (int iSide = 0; iSide <= 1; iSide++)
			for (int iWhich = 0; iWhich <= 2; iWhich++)
			{
				if (bScrewPinch == false) {
					if (iWhich == 0) {
						x = (float)(-sin(HALFANGLE)*DEVICE_RADIUS_INSULATOR_OUTER)*xzscale;
						z = (float)(cos(HALFANGLE)*DEVICE_RADIUS_INSULATOR_OUTER)*xzscale;
					} else {
						if (iWhich == 1) {
							x = (float)(-sin(HALFANGLE)*DOMAIN_OUTER_RADIUS)*xzscale;
							z = (float)(cos(HALFANGLE)*DOMAIN_OUTER_RADIUS)*xzscale;
						} else {
							x = (float)(-sin(HALFANGLE)*0.5*DOMAIN_OUTER_RADIUS)*xzscale;
							z = (float)(cos(HALFANGLE)*0.5*DOMAIN_OUTER_RADIUS)*xzscale;
						}
					};
				} else {
					// only really want to appear in 2 places ??
					x = (float)(-sin(PI/5.0)*pX->OuterRadiusAttained)*xzscale;
					z = (float)(cos(PI/5.0)*pX->OuterRadiusAttained+SP_CENTRE_Y)*xzscale; // just let it draw twice over
				}
				if (iSide == 1) x = -x;

				// New way: 
				real const lead[12] = {0.5,0.675,0.9,1.2,1.6,2.14,2.85,3.8,5.0,6.75,9.0,12.0 };
				int log_base_10, leadindex;
				real useval, leadval, leadval2, leadvalneg, leadval2neg, temp, value[5];

				real scalemax, scalemin;
				if (label_insist_max) {
					scalemax = store_max;
					scalemin = store_min;
				} else {
					scalemax = (real)ymax;
					scalemin = (real)ymin;
				}

				if (scalemax > 0.0) {
					log_base_10 = (int)(log10(scalemax));
					// careful: what if negative?
					if (scalemax < 1.0) log_base_10--; 
					// if 0.8 then we want that to say 8.00e-1 
					// keep all markers in same e-1 however.
					useval = scalemax/pow(10.0,log_base_10); // maybe to create 10^b would be often faster.
					leadindex = 2; while (useval*0.9 > lead[leadindex+1]) leadindex++;
					leadval = lead[leadindex]*pow(10.0,log_base_10);
					leadval2 = lead[leadindex-1]*pow(10.0,log_base_10);
				};
				if (scalemin < 0.0) {
					temp = -scalemin;
					log_base_10 = (int)(log10(temp));
					if (temp < 1.0) log_base_10--;
					useval = temp/pow(10.0,log_base_10);
					leadindex = 2; while (useval*0.9 > lead[leadindex+1]) leadindex++;
					leadvalneg = -lead[leadindex]*pow(10.0,log_base_10);
					leadval2neg = -lead[leadindex-1]*pow(10.0,log_base_10);
				};

				if (scalemin >= 0.0) {
					if (scalemax > 0.0) {
						value[0] = scalemax;
						value[1] = leadval;
						value[2] = leadval2;
						value[3] = 0.5*value[2];
						value[4] = 0.0;
					} else {
						// both == 0
						value[0] = 0.0; value[1] = 0.0; value[2] = 0.0; 
						value[3] = 0.0; value[4] = 0.0;
					};
				} else {
					if (scalemax > 0.0) {
						if (fabs(scalemin) > 2.0*scalemax) {
							value[0] = scalemax;
							value[1] = 0.0;
							value[2] = 0.5*leadval2neg;
							value[3] = leadval2neg;
							value[4] = scalemin;
						} else {
							if (fabs(scalemin) < 0.5*scalemax) {
								value[0] = scalemax;
								value[1] = leadval2;
								value[2] = 0.5*leadval2;
								value[3] = 0.0;
								value[4] = scalemin;
							} else {
								value[0] = scalemax;
								value[1] = leadval2;
								value[2] = 0.0;
								value[3] = leadval2neg;
								value[4] = scalemin;
							};
						};
					} else {
						value[0] = 0.0;
						value[1] = 0.5*leadval2neg;
						value[2] = leadval2neg;
						value[3] = leadvalneg;
						value[4] = scalemin;
					};
				};
				
				for (int i = 0; i < 5; i++)
				{
					sprintf(buffer,"%1.2E",value[i]*this->TickRescaling);
					RenderLabel(buffer, x, zeroplane + yscale*value[i], z); // 3D position for top-right of text
					
			//	printf("szTitle = %s ",szTitle);
			//	printf("buffer %s x %f y %f z %f\n",
			//		buffer,x,zeroplane+yscale*value[i],z);
				};
				if (scalemax*1.06 < store_max) {
					sprintf(buffer, "%1.3E", store_max*this->TickRescaling);
					RenderLabel(buffer, x, zeroplane + yscale*store_max, z, 0, 0, true);
				};

			};
		};


		// Mesh wireframe:

		if (boolDisplayMeshWireframe)
		{
			if (boolClearZBufferBeforeWireframe) {
				Direct3D.pd3dDevice->Clear( 0, NULL, 
					D3DCLEAR_ZBUFFER, (D3DCOLOR)0, 1.0f, 0 );
			};

			numPasses = 1;
			DXChk(mFX->SetTechnique(meshTech));
			DXChk(mFX->CommitChanges());

			mFX->Begin(&numPasses,0); // begins technique
//			 The default value 0 specifies that ID3DXEffect::Begin and ID3DXEffect::End will save and restore all state modified by the effect (including pixel and vertex shader constants). Valid flags can be seen at Effect State Save and Restore Flags.

			mFX->BeginPass(0);

			//if (boolDisplayInnerMesh)
			//{
			//	Direct3D.pd3dDevice->SetStreamSource(0, VertexBuffer[1], 0, sizeof(VertexPNT3));
			//	Direct3D.pd3dDevice->SetIndices(IndexBuffer[1]);
			//	Direct3D.pd3dDevice->DrawIndexedPrimitive(
			//		D3DPT_TRIANGLELIST,
			//		0,
			//		0,
			//		numVerticesUsed[1],
			//		0,
			//		numTrianglesUsed[1]);
			//};

			Direct3D.pd3dDevice->SetStreamSource(0, VertexBuffer[1], 0, sizeof(VertexPNT3));
			Direct3D.pd3dDevice->SetIndices(IndexBuffer[1]);
			Direct3D.pd3dDevice->DrawIndexedPrimitive(
				D3DPT_TRIANGLELIST,
				0,
				0,
				numVerticesUsed[1],
				0,
				numTrianglesUsed[1]);

			mFX->EndPass();
			mFX->End();
		};		
		
		Direct3D.pd3dDevice->EndScene();
		
		if( SUCCEEDED( Direct3D.pd3dDevice->BeginScene() ) )
		{ // seems to make no difference
			
			// This makes the difference:
			Direct3D.pd3dDevice->Clear( 0, NULL,D3DCLEAR_ZBUFFER,
						 (D3DCOLOR)0, 1.0f, 0 );
			// prevents z-fighting: overwrite graphics from here

			// draw black line graph at cutaway:
			if ((GlobalCutaway)) {

				// Skip it for now, build up to it.

				real * radiusArray8000;
				long * VertexIndexArray8000;
			//	// Render some data height labels along the line of the cutaway.

				VertexPNT3 * pPNT;
				VertexPNT3 * vertices_buffer;
				VertexIndexArray8000 = new long[8000];
				radiusArray8000 = new real[8000];

				if (radiusArray8000 == 0) {
					printf("\n\n?@#>@>#?\n\n");
					getch();
				};		
		
				long numVertsCutawayUse = pX->GetVertsRightOfCutawayLine_Sorted(VertexIndexArray8000, radiusArray8000, this->boolDisplayInnerMesh);
				// does not involve test for rr > 3.44*3.44 but so easily could do.


				// Render a line along the cutaway? Ambitious....
				// quite a dirty way for now: exploit that there is only 1 array of vertex positions in graphic space
				// In general, better to use TriMesh object
				// and create a function to return graphic positions.
				
				//long diff = pX->Xdomain-pX->X;
				//if (this->boolDisplayInnerMesh) diff = 0 ;
				long diff = 0;

				for (int iSubpass = 1; iSubpass < 2; iSubpass++ )
				{
					if (iSubpass == 0) {
						DXChk(VertexBuffer[1]->Lock(0,0,(void **)&vertices_buffer,D3DLOCK_READONLY));
					} else {
						DXChk(VertexBuffer[1]->Lock(0,0,(void **)&vertices_buffer,0));

						bool has_more, has_less, has_grad;
						const Vertex * pVert2, *pVertex;
						int iWhich, iCorner;
						D3DXVECTOR3 newpos;
						float wt0, wt1, wt2, wttotal, dist0, dist1, dist2;
						VertexPNT3 * pPNT0, * pPNT1, * pPNT2;
						const Triangle * pTri;
						
						// add here code to shift stuff about .....
						// interpolate using pos.y but having found the relevant index
						// from the TriMesh object
						for (int asdf = 0; asdf < numVertsCutawayUse; asdf++)
						{
							if (VertexIndexArray8000[asdf] >= 0) // VertexIndexArray8000[asdf]-diff >=0
							{
								pVertex = pX->X + VertexIndexArray8000[asdf];

								// We want the tri directly to the left of it, through which (-1,0) passes.
								// 1.Get these vertex indices
								// which tri contains a point which is further and a point less far?

								real rr = pVertex->pos.x*pVertex->pos.x+pVertex->pos.y*pVertex->pos.y;
								iWhich = -1;
								tri_len = pVertex->GetTriIndexArray(izTri);
								for (i = 0; i < tri_len; i++)
								{
									pTri = pX->T+izTri[i];
									has_more = false; has_less = false; has_grad = false;
									for (iCorner =0 ; iCorner < 3; iCorner++)
									{
										pVert2 = pTri->cornerptr[iCorner];
										if (pVert2 != pVertex) 
										{
											if (pVert2->pos.x*pVert2->pos.x+pVert2->pos.y*pVert2->pos.y > rr)
											{
												has_more = true;
											} else {
												has_less = true;
											};
										};
										if (pVert2->pos.x/pVert2->pos.y < pVertex->pos.x/pVertex->pos.y)
											has_grad = true;
									};
									
									if (has_more && has_less && has_grad)
									{
										iWhich = i;										
									}									
								};
									
								if ((iWhich == -1) || (
									(this->boolDisplayInnerMesh == false) 
									 && ((pX->T + izTri[iWhich])->u8domain_flag != DOMAIN_TRIANGLE)
									 && ((pX->T + izTri[iWhich])->u8domain_flag != CROSSING_CATH))
									) {
									// give up, do nothing
									// but how to set position?
									// use own point:

									pPNT = &(vertices_buffer[VertexIndexArray8000[asdf]]); // - diff
									pPNT->pos = vertices_buffer[(pVertex - pX->X)].pos;	// - diff

									// In cathode rod, so we want to project to cutaway:
									pPNT->pos.x = (float)(((double)pPNT->pos.z)*(CUTAWAYANGLE));
									
									if (rr > 3.44*3.44) pPNT->pos.y = zeroplane;

									// Does this ever happen except in rod?

								} else {
									pTri = pX->T + izTri[iWhich];
									//if (this->boolDisplayInnerMesh == false) && 
									//{
									//	while (pTri->u8domain_flag != DOMAIN_TRIANGLE) {
									//		iWhich--;
									//		if (iWhich == -1) {
									//			printf("give up! iVertex %d asdf %d", pVertex-X, asdf);

									//			// Looks like it tries to move clockwise to attain a domain triangle rather than crossing_ins

									//			getch();
									//			return;
									//		};
									//		pTri = pX->T + izTri[iWhich];
									//	}
									//};
									// 2. shift this pos.xyz to be on the line and y-interpolated.
									
									// is our origin also the pos origin?
									pPNT = &(vertices_buffer[VertexIndexArray8000[asdf]]); // -diff
									
									newpos.z = pPNT->pos.z;
									newpos.x = (float)(((double)pPNT->pos.z)*(CUTAWAYANGLE));
									
									pPNT0 = &(vertices_buffer[(pTri->cornerptr[0]-pX->X)]);  // -diff
									pPNT1 = &(vertices_buffer[(pTri->cornerptr[1]-pX->X)]);
									pPNT2 = &(vertices_buffer[(pTri->cornerptr[2]-pX->X)]);
																		
									dist0 = sqrt((pPNT0->pos.x-newpos.x)*(pPNT0->pos.x-newpos.x)
												+ (pPNT0->pos.z-newpos.z)*(pPNT0->pos.z-newpos.z));
									dist1 = sqrt((pPNT1->pos.x-newpos.x)*(pPNT1->pos.x-newpos.x)
												+ (pPNT1->pos.z-newpos.z)*(pPNT1->pos.z-newpos.z));
									dist2 = sqrt((pPNT2->pos.x-newpos.x)*(pPNT2->pos.x-newpos.x)
												+ (pPNT2->pos.z-newpos.z)*(pPNT2->pos.z-newpos.z));
									
									if (dist0 == 0.0) {
										wt0 = 1.0; wt1 = 0.0; wt2 = 0.0;
									}
									else {
										if (dist1 == 0.0) {
											wt0 = 0.0; wt1 = 1.0; wt2 = 0.0;
										}
										else {
											if (dist2 == 0.0) {
												wt0 = 0.0; wt1 = 0.0; wt2 = 1.0;
											}
											else {

												wt0 = 1.0f / dist0;
												wt1 = 1.0f / dist1;
												wt2 = 1.0f / dist2;
												wttotal = wt0 + wt1 + wt2;
												wt0 /= wttotal;
												wt1 /= wttotal;
												wt2 /= wttotal;
											}
										}
									}
									if ((rr > (CATHODE_ROD_R_POSITION-CATHODE_ROD_RADIUS)*(CATHODE_ROD_R_POSITION-CATHODE_ROD_RADIUS))
										&& (rr < (CATHODE_ROD_R_POSITION + CATHODE_ROD_RADIUS)*(CATHODE_ROD_R_POSITION + CATHODE_ROD_RADIUS))
										)
									{
										newpos.y = zeroplane;
									} else {
										newpos.y = wt0*pPNT0->pos.y + wt1*pPNT1->pos.y + wt2*pPNT2->pos.y;
									};
									pPNT->pos = newpos;	
								};
							// We could even try a cheat: do that before we do the surface graph. Does it stretch across?
							};
						}; // asdf
					}; // if iSubpass == 0

					for (int asdf = 0; asdf < numVertsCutawayUse; asdf++)
					{
						pPNT = &(vertices_buffer[VertexIndexArray8000[asdf]]);
						linedata[asdf].x = pPNT->pos.x;
						linedata[asdf].y = pPNT->pos.y; 
						linedata[asdf].z = pPNT->pos.z;
						linedata[asdf].colour = 0;
					};
					
					Direct3D.pd3dDevice->SetFVF(point_fvf);
					Direct3D.pd3dDevice->DrawPrimitiveUP(D3DPT_LINESTRIP,numVertsCutawayUse-1,linedata,sizeof(vertex1));

					int asdf = 0;			
					real r = 3.439999999;
					for (i = 0; i < 14; i++) {

						while ((asdf < 8000) && (radiusArray8000[asdf] < r)) asdf++;	
						if (asdf == 8000) {
							printf("asdf==8000 r %1.10E\nradiusArray[0] %1.10E [100] %1.10E [1000] %1.10E \n",
								r, radiusArray8000[0], radiusArray8000[100], radiusArray8000[1000]);
							while (1) getch(); // on debug we get here & it's unpopulated.
						}

						pPNT = &(vertices_buffer[VertexIndexArray8000[asdf]]);

						x = pPNT->pos.x;
						y = zeroplane;
						z = pPNT->pos.z;
						linedata[0].x = x; linedata[0].y = y; linedata[0].z = z;
						y = pPNT->pos.y;
						linedata[1].x = x; linedata[1].y = y; linedata[1].z = z;
						Direct3D.pd3dDevice->DrawPrimitiveUP(D3DPT_LINESTRIP,1,linedata,sizeof(vertex1));

						tempval = (pPNT->pos.y - zeroplane)/yscale;
						sprintf(buffer,"%1.2E",tempval);
						
						// DEBUG:
					//	if (tempval == 0.0) printf("r = %f pPNT->pos.y %1.9E zeroplane %1.9E index %d \n",
					//		r, pPNT->pos.y, zeroplane, VertexIndexArray8000[asdf]);

						strip_0(buffer);
						if (tempval < 0.0) {
							RenderLabel(buffer, CUTAWAYANGLE*pPNT->pos.z, zeroplane, pPNT->pos.z, false, true);
						} else {
							RenderLabel(buffer, CUTAWAYANGLE*pPNT->pos.z, zeroplane, pPNT->pos.z);
						};
						if (i == 0) r = 3.45;
						r += 0.09;
						if (i >= 7) r += 0.11; // last 4
						if (i >= 9) r = 5.52;
						if (i >= 10) r = 6.5;
						if (i > 10) r += 1.0*(double)(i - 10);
					}; 
					// line underneath:
					linedata[0].x = sin(CUTAWAYANGLE)*DEVICE_RADIUS_INSULATOR_OUTER*xzscale;
					linedata[0].y = zeroplane;
					linedata[0].z = cos(CUTAWAYANGLE)*DEVICE_RADIUS_INSULATOR_OUTER*xzscale;
					linedata[1].x = sin(CUTAWAYANGLE)*DOMAIN_OUTER_RADIUS*xzscale;
					linedata[1].y = zeroplane;
					linedata[1].z = cos(CUTAWAYANGLE)*DOMAIN_OUTER_RADIUS*xzscale;
					Direct3D.pd3dDevice->DrawPrimitiveUP(D3DPT_LINESTRIP,1,linedata,sizeof(vertex1));
					

					// Not sure we had to lock to read anyway?
					VertexBuffer[1]->Unlock();
				}; // iSubpass
			//	
			//	// label: seek for where r > 3.61

			//	i = 0; 
			//	while (radiusArray8000[i] < 3.61) i++;

			//	pPNT = VertexBuffer[2][VertexIndexArray8000[i]].pPNT;
			//	sprintf(buffer,"%1.4E",pPNT.y*some_kind_of_y_scale_back);
			//	RenderLabel(buffer, pPNT->x,pPNT->y,pPNT->z);
			//
				delete[] VertexIndexArray8000;
				delete[] radiusArray8000;
			};

			// Now try to draw around the edge of the viewport:

			// x and y in screen coordinates, z coordinate ignored...

			linedata2[0].x = vp.X;			  linedata2[0].y = vp.Y+vp.Height-1;
			linedata2[1].x = vp.X+vp.Width-1; linedata2[1].y = vp.Y+vp.Height-1;
			linedata2[2].x = vp.X+vp.Width-1; linedata2[2].y = vp.Y;
			linedata2[3].x = vp.X;			  linedata2[3].y = vp.Y;
			linedata2[4].x = vp.X;			  linedata2[4].y = vp.Y+vp.Height-1;
			
			linedata2[0].z = 0; linedata2[1].z = 0; linedata2[2].z = 0;
			linedata2[3].z = 0; linedata2[4].z = 0;

			linedata2[0].colour = 0;
			linedata2[1].colour = 0;
			linedata2[2].colour = 0;
			linedata2[3].colour = 0;
			linedata2[4].colour = 0;

			Direct3D.pd3dDevice->SetFVF(D3DFVF_XYZRHW | D3DFVF_DIFFUSE);
			Direct3D.pd3dDevice->DrawPrimitiveUP(D3DPT_LINESTRIP,4,linedata2,sizeof(vertex1));

			RenderText(szTitle,0);
			if (szLinebelow != 0) RenderText(szLinebelow,1);

			if (RenderTriLabels)
			{
				// Let's render some on vertices instead.

				const Vertex * pVertex = pX->Xdomain;
			//	for (long iVertex = 0; iVertex < pX->numVertices; iVertex++)
				for (long iVertex = pX->Xdomain-pX->X; iVertex < NUMVERTICES; iVertex++)
				{
					if (GlobalWhichLabels == 0) {
						sprintf(buffer,"%d",iVertex);
						
						RenderLabel2(buffer,  // text
							pVertex->pos.x*xzscale,
							0.00001f+zeroplane,
							pVertex->pos.y*xzscale,0); 
					}
					if (GlobalWhichLabels == 1) {
						
						sprintf(buffer,"%1.2E", pX->pData[iVertex + BEGINNING_OF_CENTRAL].n);
						RenderLabel2(buffer,  // text
							pVertex->pos.x*xzscale,
							0.00001f+zeroplane,
							pVertex->pos.y*xzscale,0); 
					};
					if (GlobalWhichLabels == 2) {
						sprintf(buffer,"%1.1E", pX->pData[iVertex + BEGINNING_OF_CENTRAL].vxy.x);
						RenderLabel2(buffer,  // text
							pVertex->pos.x*xzscale,
							0.00001f+zeroplane,
							pVertex->pos.y*xzscale,0); 
						sprintf(buffer,"%1.1E",pX->pData[iVertex+ BEGINNING_OF_CENTRAL].vxy.y);
						strip_0(buffer);
						RenderLabel2(buffer,  // text
							pVertex->pos.x*xzscale,
							0.00001f+zeroplane,
							pVertex->pos.y*xzscale,1); 
					};
					
					if (GlobalWhichLabels == 3) {
						sprintf(buffer,"%1.1E", pX->pData[iVertex + BEGINNING_OF_CENTRAL].pos.x);
						strip_0(buffer);
						RenderLabel2(buffer,  // text
							pVertex->pos.x*xzscale,
							0.00001f+zeroplane,
							pVertex->pos.y*xzscale,0); 
						sprintf(buffer,"%1.1E", pX->pData[iVertex + BEGINNING_OF_CENTRAL].pos.y);
						strip_0(buffer);
						RenderLabel2(buffer,  // text
							pVertex->pos.x*xzscale,
							0.00001f+zeroplane,
							pVertex->pos.y*xzscale,1); 
					};
					
					if (GlobalWhichLabels == 4) {
						sprintf(buffer,"%d",iVertex);
						RenderLabel2(buffer,  // text
							pVertex->pos.x*xzscale,
							0.00001f+zeroplane,
							pVertex->pos.y*xzscale,0); 
						sprintf(buffer, "%1.1E", pX->pData[iVertex + BEGINNING_OF_CENTRAL].Te); // pVertex->phi);
						strip_0(buffer);
						RenderLabel2(buffer,  // text
							pVertex->pos.x*xzscale,
							0.00001f+zeroplane,
							pVertex->pos.y*xzscale,1); 
					};
					
					if (GlobalWhichLabels == 5) {
						sprintf(buffer,"%d",iVertex);
						RenderLabel2(buffer,  // text
							pVertex->pos.x*xzscale,
							0.00001f+zeroplane,
							pVertex->pos.y*xzscale,0); 
						sprintf(buffer, "%1.1E", 500.0); // pVertex->Temp.x);
						strip_0(buffer);
						RenderLabel2(buffer,  // text
							pVertex->pos.x*xzscale,
							0.00001f+zeroplane,
							pVertex->pos.y*xzscale,1); 
					};
					// How often are these n,T,v going to be maintained?
					
					++pVertex;
				};
			};
			Direct3D.pd3dDevice->EndScene();
		} // if SUCCEEDED BeginScene
		else {
			printf("BeginScene (2) failed!\n\n");
			getch();
		}
	} else { // if SUCCEEDED BeginScene
		printf("BeginScene (1) failed !!\n\n");
		getch();
	};

	if (VERBOSEGRAPHICS) printf("Render done. %s \n", szTitle);
}


void inline surfacegraph::RenderText (const char * text, int lines_down)
{
	RECT rect;
	rect.top = vp.Y +20+30*lines_down;
	rect.right = vp.X + vp.Width-15;

	rect.bottom=rect.top+30;
	rect.left = rect.right-300;
	
	rect.top-=2;
	rect.right+=2;
	rect.left+=2;
	rect.bottom-=2;

	Direct3D.g_pFont2->DrawText(NULL,text,strlen(text),&rect,DT_RIGHT|DT_VCENTER,0xff000000);

	rect.top+=2;
	rect.right-=2;
	rect.left-=2;
	rect.bottom+=2;

	Direct3D.g_pFont2->DrawText(NULL,text,strlen(text),&rect,DT_RIGHT|DT_VCENTER,0xff99ffff);

	rect.top-=4;
	rect.bottom-=4;

	Direct3D.g_pFont2->DrawText(NULL,text,strlen(text),&rect,DT_RIGHT|DT_VCENTER,0xff99ffff);

	rect.right+=4;
	rect.left+=4;

	Direct3D.g_pFont2->DrawText(NULL,text,strlen(text),&rect,DT_RIGHT|DT_VCENTER,0xff99ffff);
	
	rect.top+=4;
	rect.bottom+=4;

	Direct3D.g_pFont2->DrawText(NULL,text,strlen(text),&rect,DT_RIGHT|DT_VCENTER,0xffffffff);

	rect.top-=2;
	rect.right-=2;
	rect.left-=2;
	rect.bottom-=2;

	Direct3D.g_pFont2->DrawText(NULL,text,strlen(text),&rect,DT_RIGHT|DT_VCENTER,0xff000000);
	// Cyan came out on top. Supposed to do what about it?
}

void inline surfacegraph::RenderLabel (char * text, float x, float y, float z, 
									   bool extrainfo, bool botleft, bool bColoured)
	{
		RECT rect;
		D3DXVECTOR3 transformed;
		// The following was static const. That caused badness! vp needs to change.
		D3DXMATRIXA16 screenmat(((float)vp.Width)*0.5f,0.0f,0.0f,0.0f,
			                    0.0f,  -((float)vp.Height)*0.5f,0.0f,0.0f,
								0.0f,0.0f,((float)vp.MaxZ)-((float)vp.MinZ),0.0f,
							    ((float)vp.X)+((float)vp.Width)*0.5f,((float)vp.Y)+((float)vp.Height)*0.5f,((float)vp.MinZ),1.0f);

		D3DXVECTOR3 position(x,y,z);
		D3DXVECTOR3 screencoord;
	//	printf("position %1.9E %1.9E %1.9E\n", position.x, position.y, position.z);
	//	printf("matView %1.4E %1.4E %1.4E ~ %1.4E %1.4E %1.4E ~ %1.4E %1.4E %1.4E \n",
	//		matView._11, matView._12, matView._13, matView._21, matView._22, matView._23, matView._31, matView._32, matView._33);
		
		D3DXVec3TransformCoord(&transformed, &position, &(matView*matProj));

	//	printf("transformed %1.9E %1.9E %1.9E \n", transformed.x, transformed.y, transformed.z);
		
		D3DXVec3TransformCoord(&screencoord, &transformed, &screenmat);

	//	printf("done D3DXVec3Xform \n");
		DWORD format = DT_TOP|DT_RIGHT;

		rect.top = (int)screencoord.y;
		rect.right = (int)screencoord.x;

		rect.bottom = rect.top + 30;
		rect.left = rect.right - 200;

		if (botleft) {
			format = DT_BOTTOM | DT_LEFT;

			rect.bottom = (int)screencoord.y;
			rect.left = (int)screencoord.x;

			rect.top = rect.bottom - 30;
			rect.right = rect.left + 200;
		};

		D3DCOLOR textcolor = 0xff000000;
		if (bColoured) textcolor = 0xff2200bb;

		if (extrainfo) {
			format = DT_CENTER | DT_VCENTER; // also changing rect, below.
			textcolor = 0xff700022;
		
		// see http://msdn.microsoft.com/en-us/library/windows/desktop/bb206341(v=vs.85).aspx
		
			rect.bottom -= 15;
			rect.top -= 15;
			rect.left += 100;
			rect.right += 100;
		};
		
		Direct3D.g_pFont->DrawText(NULL,text,strlen(text),&rect,format,textcolor);
		
		rect.top += 1;
		rect.bottom += 1;
		rect.left += 1;
		rect.right += 1;

		Direct3D.g_pFont->DrawText(NULL,text,strlen(text),&rect,format,0xffffffff);

		rect.top -= 2;
		rect.bottom -= 2;

		Direct3D.g_pFont->DrawText(NULL,text,strlen(text),&rect,format,0xffffffff);
		
		rect.left -= 2;
		rect.right -= 2;

		Direct3D.g_pFont->DrawText(NULL,text,strlen(text),&rect,format,0xffffffff);

		rect.top += 2;
		rect.bottom += 2;

		Direct3D.g_pFont->DrawText(NULL,text,strlen(text),&rect,format,0xffffffff);

		rect.top -= 1;
		rect.bottom -= 1;
		rect.left += 1;
		rect.right += 1;
		

		// even more white:
		rect.left -= 2;
		rect.right -=2;
		Direct3D.g_pFont->DrawText(NULL,text,strlen(text),&rect,format,0xffffffff);
		rect.left += 4;
		rect.right += 4;
		Direct3D.g_pFont->DrawText(NULL,text,strlen(text),&rect,format,0xffffffff);

		rect.top += 1;
		rect.bottom += 1;
		Direct3D.g_pFont->DrawText(NULL, text, strlen(text), &rect, format, 0xffffffff);
		rect.left -= 4;
		rect.right -= 4;
		Direct3D.g_pFont->DrawText(NULL, text, strlen(text), &rect, format, 0xffffffff);

		rect.top -= 1;
		rect.bottom -= 1;

		rect.left += 2;
		rect.right += 2;

		Direct3D.g_pFont->DrawText(NULL,text,strlen(text),&rect,format,textcolor);
		
	}
	

void inline surfacegraph::RenderLabel2 (char * text, float x, float y, float z, int whichline, unsigned int color, bool bLong)
{
	RECT rect;
	D3DXVECTOR3 position(x,y,z);
	D3DXVECTOR3 transformed;

	// see http://msdn.microsoft.com/en-us/library/windows/desktop/bb206341(v=vs.85).aspx

	D3DXMATRIXA16 screenmat(((float)vp.Width)*0.5f,0.0f,0.0f,0.0f,
			                0.0f,  -((float)vp.Height)*0.5f,0.0f,0.0f,
							0.0f,0.0f,((float)vp.MaxZ)-((float)vp.MinZ),0.0f,
							((float)vp.X)+((float)vp.Width)*0.5f,((float)vp.Y)+((float)vp.Height)*0.5f,((float)vp.MinZ),1.0f);

	D3DXVECTOR3 screencoord;

	if (position.x == 0.0f) position.x = 1.0e-7f;
	if (position.y == 0.0f) position.y = 1.0e-7f;
	if (position.z == 0.0f) position.z = 1.0e-7f;

//	printf("Dbg position %1.9E %1.9E %1.9E matView 112233 %1.9E %1.9E %1.9E matProj %1.9E %1.9E %1.9E \n", position.x, position.y, position.z,
//		matView._11, matView._22, matView._33, matProj._11, matProj._22, matProj._33);

	D3DXVec3TransformCoord(&transformed, &position,&(matView*matProj));

	// it looks like FP trap is found above not here. smth bad about transformed here
	// or, smth bad about the input to the line above.

	// DEBUG:
//	printf("Dbg transformed %1.10E %1.10E %1.10E \n", transformed.x, transformed.y, transformed.z);
//	printf("Screenmat diag %1.10E %1.10E %1.10E \n", screenmat._11, screenmat._22, screenmat._33);

	D3DXVec3TransformCoord(&screencoord, &transformed, &screenmat);

	if (bLong == false) {

		rect.top = (int)screencoord.y - 15;
		rect.right = (int)screencoord.x + 100;

		if ((screencoord.x > vp.X) && (screencoord.x < vp.X + vp.Width)) {

			if (whichline == 0) {
				rect.top -= 8;
			}
			else {
				rect.top += 7;
			}

			rect.bottom = rect.top + 30;
			rect.left = rect.right - 200;

			Direct3D.g_pFontsmall->DrawText(NULL, text, strlen(text), &rect, DT_CENTER | DT_VCENTER, color);

			rect.top += 1;
			rect.bottom += 1;
			rect.left += 1;
			rect.right += 1;

			Direct3D.g_pFontsmall->DrawText(NULL, text, strlen(text), &rect, DT_CENTER | DT_VCENTER, 0xffffffff);

			rect.top -= 2;
			rect.bottom -= 2;
			Direct3D.g_pFontsmall->DrawText(NULL, text, strlen(text), &rect, DT_CENTER | DT_VCENTER, 0xffffffff);

			rect.left -= 2;
			rect.right -= 2;

			Direct3D.g_pFontsmall->DrawText(NULL, text, strlen(text), &rect, DT_CENTER | DT_VCENTER, 0xffffffff);

			rect.top += 2;
			rect.bottom += 2;
			Direct3D.g_pFontsmall->DrawText(NULL, text, strlen(text), &rect, DT_CENTER | DT_VCENTER, 0xffffffff);

			rect.top -= 3;
			rect.bottom -= 3;
			rect.left -= 1;
			rect.right -= 1;
			Direct3D.g_pFontsmall->DrawText(NULL, text, strlen(text), &rect, DT_CENTER | DT_VCENTER, 0xffffff55);
			rect.left += 4;
			rect.right += 4;
			Direct3D.g_pFontsmall->DrawText(NULL, text, strlen(text), &rect, DT_CENTER | DT_VCENTER, 0xffffff55);

			rect.top += 2;
			rect.bottom += 2;
			rect.left -= 2;
			rect.right -= 2;
			Direct3D.g_pFontsmall->DrawText(NULL, text, strlen(text), &rect, DT_CENTER | DT_VCENTER, 0xff000000);
			// try rendering black twice and hope it ends up on top.
		}
	}
	else {

		rect.top = (int)screencoord.y - 15;
		rect.left = (int)screencoord.x;

		if ((screencoord.x > vp.X) && (screencoord.x < vp.X + vp.Width)) {

			if (whichline == 0) {
				rect.top -= 8;
			}
			else {
				rect.top += 7;
			}

			rect.bottom = rect.top + 30;
			rect.right = rect.left + 500;

			Direct3D.g_pFontsmall->DrawText(NULL, text, strlen(text), &rect, DT_LEFT | DT_VCENTER, color);

			rect.top += 1;
			rect.bottom += 1;
			rect.left += 1;
			rect.right += 1;

			Direct3D.g_pFontsmall->DrawText(NULL, text, strlen(text), &rect, DT_LEFT | DT_VCENTER, 0xffffffff);

			rect.top -= 2;
			rect.bottom -= 2;
			Direct3D.g_pFontsmall->DrawText(NULL, text, strlen(text), &rect, DT_LEFT | DT_VCENTER, 0xffffffff);

			rect.left -= 2;
			rect.right -= 2;

			Direct3D.g_pFontsmall->DrawText(NULL, text, strlen(text), &rect, DT_LEFT | DT_VCENTER, 0xffffffff);

			rect.top += 2;
			rect.bottom += 2;
			Direct3D.g_pFontsmall->DrawText(NULL, text, strlen(text), &rect, DT_LEFT | DT_VCENTER, 0xffffffff);

			rect.top -= 3;
			rect.bottom -= 3;
			rect.left -= 1;
			rect.right -= 1;
			Direct3D.g_pFontsmall->DrawText(NULL, text, strlen(text), &rect, DT_LEFT | DT_VCENTER, 0xffffff55);
			rect.left += 4;
			rect.right += 4;
			Direct3D.g_pFontsmall->DrawText(NULL, text, strlen(text), &rect, DT_LEFT | DT_VCENTER, 0xffffff55);

			rect.top += 2;
			rect.bottom += 2;
			rect.left -= 2;
			rect.right -= 2;
			Direct3D.g_pFontsmall->DrawText(NULL, text, strlen(text), &rect, DT_LEFT | DT_VCENTER, color);
			// try rendering black twice and hope it ends up on top.

		}
	}
}

void inline surfacegraph::RenderLabel3(char * text, float x, float y, float z, int whichline, unsigned int color)
{
	// version for long text on 1D legend.

	RECT rect;
	D3DXVECTOR3 position(x, y, z);
	D3DXVECTOR3 transformed;

	// see http://msdn.microsoft.com/en-us/library/windows/desktop/bb206341(v=vs.85).aspx

	D3DXMATRIXA16 screenmat(((float)vp.Width)*0.5f, 0.0f, 0.0f, 0.0f,
		0.0f, -((float)vp.Height)*0.5f, 0.0f, 0.0f,
		0.0f, 0.0f, ((float)vp.MaxZ) - ((float)vp.MinZ), 0.0f,
		((float)vp.X) + ((float)vp.Width)*0.5f, ((float)vp.Y) + ((float)vp.Height)*0.5f, ((float)vp.MinZ), 1.0f);

	D3DXVECTOR3 screencoord;

	if (position.x == 0.0f) position.x = 1.0e-7f;
	if (position.y == 0.0f) position.y = 1.0e-7f;
	if (position.z == 0.0f) position.z = 1.0e-7f;

	//	printf("Dbg position %1.9E %1.9E %1.9E matView 112233 %1.9E %1.9E %1.9E matProj %1.9E %1.9E %1.9E \n", position.x, position.y, position.z,
	//		matView._11, matView._22, matView._33, matProj._11, matProj._22, matProj._33);

	D3DXVec3TransformCoord(&transformed, &position, &(matView*matProj));

	// it looks like FP trap is found above not here. smth bad about transformed here
	// or, smth bad about the input to the line above.

	// DEBUG:
	//	printf("Dbg transformed %1.10E %1.10E %1.10E \n", transformed.x, transformed.y, transformed.z);
	//	printf("Screenmat diag %1.10E %1.10E %1.10E \n", screenmat._11, screenmat._22, screenmat._33);

	D3DXVec3TransformCoord(&screencoord, &transformed, &screenmat);

	rect.top = (int)screencoord.y - 15;
	rect.left = (int)screencoord.x ;

	if ((screencoord.x > vp.X) && (screencoord.x < vp.X + vp.Width)) {

		if (whichline == 0) {
			rect.top -= 8;
		}
		else {
			rect.top += 7;
		}

		rect.bottom = rect.top + 30;
		rect.right = rect.left + 500;

		Direct3D.g_pFontsmall->DrawText(NULL, text, strlen(text), &rect, DT_CENTER | DT_VCENTER, color);

		rect.top += 1;
		rect.bottom += 1;
		rect.left += 1;
		rect.right += 1;

		Direct3D.g_pFontsmall->DrawText(NULL, text, strlen(text), &rect, DT_LEFT | DT_VCENTER, 0xffffffff);

		rect.top -= 2;
		rect.bottom -= 2;
		Direct3D.g_pFontsmall->DrawText(NULL, text, strlen(text), &rect, DT_LEFT | DT_VCENTER, 0xffffffff);

		rect.left -= 2;
		rect.right -= 2;

		Direct3D.g_pFontsmall->DrawText(NULL, text, strlen(text), &rect, DT_LEFT | DT_VCENTER, 0xffffffff);

		rect.top += 2;
		rect.bottom += 2;
		Direct3D.g_pFontsmall->DrawText(NULL, text, strlen(text), &rect, DT_LEFT | DT_VCENTER, 0xffffffff);

		rect.top -= 3;
		rect.bottom -= 3;
		rect.left -= 1;
		rect.right -= 1;
		Direct3D.g_pFontsmall->DrawText(NULL, text, strlen(text), &rect, DT_LEFT | DT_VCENTER, 0xffffff55);
		rect.left += 4;
		rect.right += 4;
		Direct3D.g_pFontsmall->DrawText(NULL, text, strlen(text), &rect, DT_LEFT | DT_VCENTER, 0xffffff55);

		rect.top += 2;
		rect.bottom += 2;
		rect.left -= 2;
		rect.right -= 2;
		Direct3D.g_pFontsmall->DrawText(NULL, text, strlen(text), &rect, DT_LEFT | DT_VCENTER, 0xff000000);
		// try rendering black twice and hope it ends up on top.

	}
}

surfacegraph::~surfacegraph()
{
	for(int N = 0; N < NUMBER_VERTEX_ARRAYS; N++)
	{
		if ( VertexBuffer[N] != NULL)
			VertexBuffer[N]->Release();
		if ( IndexBuffer[N] != NULL)
			IndexBuffer[N]->Release();
	};
	
	
	delete mShadowMap;
}

#endif