#include "hip/hip_runtime.h"
#ifndef BASICSCPP
#define BASICSCPP

#include "mesh.h"
#include "globals.h"

#include "vector_tensor.cu"
//#include "vector_tensor.cpp"
// For manipulations of triangles and vertices.

smartlong GlobalVertexScratchList;
extern bool bDebugReorder;
extern __constant__ f64_tens2 Anticlockwise_d, Clockwise_d; // use this to do rotation.   

//real const minimum_pressure_SD_at_1e18_sq = minimum_pressure_SD_at_1e18*minimum_pressure_SD_at_1e18;
//real const min_variance_heat = min_SD_heat*min_SD_heat;

Tensor2 const Anticlockwise(cos(FULLANGLE),-sin(FULLANGLE),sin(FULLANGLE),cos(FULLANGLE));
Tensor2 const Clockwise(cos(FULLANGLE),sin(FULLANGLE),-sin(FULLANGLE),cos(FULLANGLE));
Tensor3 const Anticlockwise3 (cos(FULLANGLE),-sin(FULLANGLE), 0.0,
					sin(FULLANGLE),cos(FULLANGLE), 0.0,
					0.0, 0.0, 1.0);
Tensor3 const Clockwise3 (cos(FULLANGLE),sin(FULLANGLE), 0.0,
					-sin(FULLANGLE),cos(FULLANGLE), 0.0,
					0.0, 0.0, 1.0);
Tensor2 const HalfAnticlockwise (cos(HALFANGLE),-sin(HALFANGLE),sin(HALFANGLE),cos(HALFANGLE));
Tensor2 const HalfClockwise(cos(HALFANGLE),sin(HALFANGLE),-sin(HALFANGLE),cos(HALFANGLE));

real modelled_n;

bool bSpit = false;
bool bDebugcp = false;

void ConvexPolygon::CreateClockwiseImage(const ConvexPolygon & cpSrc) 
{
	numCoords = cpSrc.numCoords;
	for (int i = 0; i < numCoords; i++)
#ifdef __CUDA_ARCH__
		coord[i] = Clockwise_d*cpSrc.coord[i];
#else
		coord[i] = Clockwise*cpSrc.coord[i];
#endif
}
void ConvexPolygon::CreateAnticlockwiseImage(const ConvexPolygon & cpSrc) 
{
	numCoords = cpSrc.numCoords;
	for (int i = 0; i < numCoords; i++)
#ifdef __CUDA_ARCH__
		coord[i] = Anticlockwise_d*cpSrc.coord[i];	
#else
		coord[i] = Anticlockwise*cpSrc.coord[i];
#endif
}

fluid_nvT fluid_nvT::Clockwise() const
	{
		fluid_nvT result;
		memcpy(&(result.n),&(n),sizeof(real)*3);
		memcpy(&(result.nT),&(nT),sizeof(real)*3);
		result.nv[0] = Clockwise3*nv[0];
		result.nv[1] = Clockwise3*nv[1];
		result.nv[2] = Clockwise3*nv[2];
		return result;
	}
fluid_nvT fluid_nvT::Anticlockwise() const
	{
		fluid_nvT result;
		memcpy(&(result.n),&(n),sizeof(real)*3);
		memcpy(&(result.nT),&(nT),sizeof(real)*3);
		result.nv[0] = Anticlockwise3*nv[0];
		result.nv[1] = Anticlockwise3*nv[1];
		result.nv[2] = Anticlockwise3*nv[2];
		return result;
	}
void fluidnvT::Interpolate ( fluidnvT* pvv1, fluidnvT * pvv2,
							Vector2 & pos1, Vector2 & pos2, Vector2 & ourpos)
	{
		// want to take dist1/(dist1 + dist2)
		//  (dist1/(dist1+dist2)) = dist1/dist2 / (1 + dist1/dist2)
//		real ratio = sqrt( 
//			((pos1.x-ourpos.x)*(pos1.x-ourpos.x)+(pos1.y-ourpos.y)*(pos1.y-ourpos.y))/
//			((pos2.x-ourpos.x)*(pos2.x-ourpos.x)+(pos2.y-ourpos.y)*(pos2.y-ourpos.y)));
		// this is too dangerous - maybe ourpos == pos2
	
		real dist1 = sqrt((pos1.x-ourpos.x)*(pos1.x-ourpos.x)+(pos1.y-ourpos.y)*(pos1.y-ourpos.y));
		real dist2 = sqrt((pos2.x-ourpos.x)*(pos2.x-ourpos.x)+(pos2.y-ourpos.y)*(pos2.y-ourpos.y));
		real ppn = dist1/(dist1+dist2); 
		real minus = 1.0-ppn;
		n = ppn*pvv2->n + minus*pvv1->n;
		T = ppn*pvv2->T + minus*pvv1->T;
		v = ppn*pvv2->v + minus*pvv1->v;
	}

void GetInterpolationCoefficients( real beta[3],
							real x, real y,
							Vector2 pos0, Vector2 pos1, Vector2 pos2)
{
	// idea is to form a plane that passes through z0,z1,z2.

	// so firstly if we lie on a line between 0 and 1, we know what that is;
	// then we have some gradient in the direction normal to that which is determined by y2

	//relative = pos-pos0;
	//along01 = relative.dot(pos1-pos0)/(pos1-pos0).modulus(); 
	//// by being clever we should be able to avoid the square root since have z0 + (z1-z0)/(pos1-pos0).modulus()
	//perp.x = pos0.y-pos1.y;
	//perp.y = pos1.x-pos0.x;
	//away = relative.dot(perp)/perp.modulus();

	//pos2along01 = (pos2 - pos0).dot(pos1-pos0)/(pos1-pos0).modulus();
	//pos2away = (pos2-pos0).dot(perp)/perp.modulus();

	//real z_ = z0 + pos2along01*(z1-z0)/(pos1-pos0).modulus();
	//gradient_away = (z2-z_)/pos2away;

	//real z = z0 + along01*((z1-z0)/(pos1-pos0).modulus()) + away*gradient_away;
	//*pResult = z;


	// fast version:

	Vector2 pos(x,y);
	Vector2 perp;
	real ratio;//, coeff_on_z0, coeff_on_z1, coeff_on_z2;
	Vector2 relative = pos-pos0;
	Vector2 rel1 = pos1-pos0;
	Vector2 rel2 = pos2-pos0;
	real mod01sq = rel1.dot(rel1);
	real along01_over_mod01 = relative.dot(rel1)/mod01sq;
	real pos2along01_over_mod01 = rel2.dot(rel1)/mod01sq;
	//real z_expect = z0 + pos2along01_over_mod01*(z1-z0);
	//gradient_away = (z2-z_expect)*(perp.modulus()/((pos2-pos0).dot(perp)));
	//away_times_gradient_away = (z2-z_expect)*relative.dot(perp)/((pos2-pos0).dot(perp));
	//real z = z0 + along01_over_mod01*((z1-z0)) + away_times_gradient_away;

	// can we work out coefficients actually on z0,z1,z2 because then can do faster in 2D,3D. :
	
	perp.x = -rel1.y;
	perp.y = rel1.x;
	ratio = relative.dot(perp)/(rel2.dot(perp));
	
	//beta[0] = 1.0 - along01_over_mod01 - ratio + ratio*pos2along01_over_mod01;
	beta[1] =         along01_over_mod01             - ratio*pos2along01_over_mod01;
	beta[2] =                              ratio;
	beta[0] = 1.0 - beta[1] - beta[2];
	
	//*pResult = coeff_on_z0*z0 + coeff_on_z1*z1 + coeff_on_z2*z2;
}


void TriMesh::RecalculateCentroid(Vertex * pVertex)
	{
		// ASSUMES TRI CENTROIDS SET.
/*
		long tri_len, i;
		Triangle * pTri;
		ConvexPolygon cp;
		long izTri[128];
		Vector2 tri_cent;

		if ((pVertex->flags != INNERMOST) && (pVertex->flags != CONVEX_EDGE_VERTEX))
		{
			cp.Clear();
			tri_len = pVertex->GetTriIndexArray(izTri);
			for (i = 0; i < tri_len; i++)
			{
				pTri = T + izTri[i];
				tri_cent = pTri->GetContiguousCent_AssumingCentroidsSet(pVertex);
				cp.add(tri_cent);
			}
			pVertex->centroid = cp.CalculateBarycenter();			
		} else {
			pVertex->centroid = pVertex->pos; // at edge of memory
		};
		*/

	}
	
Vector3 Triangle::GetAAvg() const
{
	Vector3 A(0.0, 0.0, 0.0);
	return A;

	/*Vector3 A;
	if (u8domain_flag != CROSSING_INS)
	{
		if (periodic == 0)
		{	A = (cornerptr[0]->A + cornerptr[1]->A + cornerptr[2]->A)/3.0;
			return A;
		};
		int par[3];
		GetParity(par);
		A.x = 0.0; A.y = 0.0; A.z = 0.0;
		if (par[0] == 0){
			A += cornerptr[0]->A;
		} else {
			A += Anticlockwise3*cornerptr[0]->A;
		};
		if (par[1] == 0) {
			A += cornerptr[1]->A;
		} else {
			A += Anticlockwise3*cornerptr[1]->A;
		};
		if (par[2] == 0) {
			A += cornerptr[2]->A;
		} else {
			A += Anticlockwise3*cornerptr[2]->A;
		};
		A /= 3.0;
		return A;
	}
	// In the insulator crossing case the used position is shifted from
	// the centroid down to the insulator.
	
	real beta[3];
	GetInterpolationCoefficients(beta, cent.x, cent.y,
						cornerptr[0]->pos,
						cornerptr[1]->pos,
						cornerptr[2]->pos);
	if (periodic == 0) {
		A = beta[0]*cornerptr[0]->A + beta[1]*cornerptr[1]->A + beta[2]*cornerptr[2]->A;
		return A;
	};
	int par[3];
	GetParity(par);
	A.x = 0.0; A.y = 0.0; A.z = 0.0;
	if (par[0] == 0){
		A += beta[0]*cornerptr[0]->A;
	} else {
		A += beta[0]*(Anticlockwise3*cornerptr[0]->A);
	};
	if (par[1] == 0) {
		A += beta[1]*cornerptr[1]->A;
	} else {
		A += beta[1]*(Anticlockwise3*cornerptr[1]->A);
	};
	if (par[2] == 0) {
		A += beta[2]*cornerptr[2]->A;
	} else {
		A += beta[2]*(Anticlockwise3*cornerptr[2]->A);
	};
	return A;*/

}


/*macroscopic macroscopic::operator* (const real hh,const macroscopic &vars)
	{
		macroscopic cv;
		cv.mass = hh*vars.mass;
		cv.heat = hh*vars.heat;
		cv.mom = hh*vars.mom;
		return cv;
	}
*/
/*
bool inline AuxTriangle::has_vertex(AuxVertex * pVertex)
{
	return ((cornerptr[0] == pVertex) || (cornerptr[1] == pVertex) || (cornerptr[2] == pVertex));
};

AuxVertex::AuxVertex() {
		flags = 0;
		tri_len = 0;
		neigh_len = 0;
	};

	void AuxVertex::addtri(long iTri)
	{
		iTriangles[tri_len] = iTri;
		tri_len++;
		if (tri_len > MAXNEIGH) {
			printf("\n\ntri_len > MAXNEIGH. stop.\n\n");
			getch();
  			tri_len = tri_len;
		};
		if (tri_len > 8) 
		{
			tri_len = tri_len;
		}
	};

	void AuxVertex::remove_tri(long iTri)
	{
		long iWhich = 0;
		while ((iWhich < tri_len)
			&& (iTriangles[iWhich] != iTri)) iWhich++;
		if (iWhich == tri_len)
		{
			iWhich = iWhich;
		}

		memmove(iTriangles+iWhich,iTriangles+iWhich+1,sizeof(long)*(tri_len-iWhich-1));
		tri_len--;
		
	};

	void AuxVertex::add_neigh(long iNeigh)
	{
		if (neigh_len == MAXNEIGH){
			printf("Had to stop: too many neighs in Aux mesh.\n");
			getch();
			return;
		}
		iNeighbours[neigh_len] = iNeigh;
		neigh_len++;
	};

	int AuxVertex::add_neigh_unique(long iNeigh)
	{
		int i;
		for (i = 0; i < neigh_len; i++)
			if (iNeighbours[i] == iNeigh) return 0;
		if (neigh_len == MAXNEIGH){
			printf("Had to stop: too many neighs in Aux mesh.\n");
			getch();
			return 2;
		}
		iNeighbours[neigh_len] = iNeigh;
		neigh_len++;
		return 1;
	};

	//void coeff_add(long iVertex, real beta)
	//{
	//	coeff_extra.add(beta);
	//	coeff_self -= beta;
	//	index_extra.add(iVertex);

	//};

	void AuxVertex::PopulatePosition(Vector2 & result)
	{
		result.x = x; result.y = y;
	}

	AuxTriangle::AuxTriangle() {
		flags = DOMAIN_TRIANGLE;
		periodic = 0;
	}
	AuxTriangle::~AuxTriangle() {}

	void AuxTriangle::PopulatePositions(Vector2 & u0, Vector2 & u1, Vector2 & u2)
	{
		cornerptr[0]->PopulatePosition(u0);
		cornerptr[1]->PopulatePosition(u1);
		cornerptr[2]->PopulatePosition(u2);
	};

	int AuxTriangle::GetLeftmostIndex()
	{
		// Note: we could put an argument for returning the one with leftmost gradient x/y
		int c1 = 1;
		if (cornerptr[2]->pos.x/cornerptr[2]->pos.y < cornerptr[1]->pos.x/cornerptr[1]->pos.y)
			c1 = 2;
		if (cornerptr[0]->pos.y != 0.0) {
			if (cornerptr[0]->pos.x/cornerptr[0]->pos.y < cornerptr[c1]->x/cornerptr[c1]->y)
				c1 = 0;
		};
		return c1;
	}

	int AuxTriangle::GetRightmostIndex()
	{
		int c1 = 1;
		if (cornerptr[2]->pos.x/cornerptr[2]->pos.y > cornerptr[1]->pos.x/cornerptr[1]->pos.y)
			c1 = 2;
		if (cornerptr[0]->pos.y != 0.0) {
			if (cornerptr[0]->pos.x/cornerptr[0]->pos.y > cornerptr[c1]->x/cornerptr[c1]->y)
				c1 = 0;
		};
		return c1;
	}
*/


smartlong::smartlong()
	{
		ptr = NULL;
		len = 0;
		alloclen = 0;
	};

void smartlong::clear()
	{
		if (ptr != NULL) free(ptr);
		ptr = NULL;
		len = 0;
		alloclen = 0;
	};

void smartlong::remove_if_exists(long what)
	{
		if (len == 0) return;
		
		long * look = ptr;
		long * ptrlast = ptr+len-1;

		for (look = ptr; look <= ptrlast; ++look)
		{
			if (*look == what) {
				for (; look < ptrlast; ++look)
					*look = look[1];
				len--;
				return;
			}
		};
	}

void smartlong::remove(long what)
	{
		// DEBUG VERSION:
		//
		if (len == 0) return;
		
		//long * look = ptr;
		//while (*look != what) ++look;
		//long * ptrlast = ptr+len-1;
		//for (; look < ptrlast; ++look)
		//	*look = look[1];
		//len--;		
		
		long * look = ptr;
		long * ptrlast = ptr+len-1;

		while ((look <= ptrlast) && (*look != what)) ++look;
		if (look > ptrlast) 
		{
			printf("!!!");
			getch();
		};
		for (; look < ptrlast; ++look)
			*look = look[1];
		len--;		
	}

void smartlong::IncreaseDim()
	{
	
//	iCaret 14849 iTriCaret 55802 Figure of merit 4.86248937E+04 7.55109451E+04
//	smartlong memory alloc failed!!!alloclen = -32768 ALLOC = 8
//		Understandable that it would fail. alloclen is a short so it has overrun.
	
		ptr = (long *)realloc(ptr,sizeof(long)*(alloclen+ALLOC));
		if (ptr == 0) 
		{
			printf("smartlong memory alloc failed!!! alloclen = %d ALLOC = %d\n",
				alloclen, ALLOC);
			getch();
			len = len;
		};
		alloclen = alloclen+ALLOC;
	}

void smartlong::add(long what)
	{
		// make another function to only add unique....
		len++;
		if (len >= alloclen) IncreaseDim();
		
		ptr[len-1] = what;
	};

void smartlong::add_at_element(long what,long iInsert)
	{
		len++;
		if (len >= alloclen) IncreaseDim();
		
		memmove(ptr+iInsert+1,ptr+iInsert,sizeof(long)*(len-iInsert-1)); // new len ...

		ptr[iInsert] = what;
	}

void smartlong::copyfrom(smartlong & src)
	{
		clear();
		for (int i = 0; i < src.len; i++)
			add(src.ptr[i]);
	}

bool smartlong::contains(long what)
	{
		if (len == 0) return false;
		long * look = ptr;
		long * ptrlast = ptr+len-1;
		for (; look <= ptrlast; ++look)
			if (*look == what) return true;
		return false;
	};

long smartlong::FindIndex(long what)
	{
		long * look = ptr;
		long * ptrafter = ptr+len;
		while ( look < ptrafter )
		{
			if (*look == what) return look-ptr;
			++look;
		};
		return -1;
	}

void smartlong::add_unique(long what)
	{
		long * look = ptr;
		for (long k = 0; k < len; k++)
		{
			if (*look == what) return;
			++look;
		};
		// Still here => it was not already in the array.
		add(what);
	}

void smartlong::remove_element( long iWhich )
	{
		if (iWhich >= len) {
			iWhich = iWhich;
		}
		long * look = ptr+iWhich;
		memmove(look, look+1,sizeof(long)*(len-iWhich-1));
		// if len == 4: 0 1 2 3, delete element 2 -> copy 1 element.
		len--;		
		if (len <= 0) {
			printf("Pls don't use remove_element to delte all elemetns. \n");
			getch();
		};
	}

int smartlong::remove_elements( long iStart, long iHowmany)
	{
		int iReturn;
		if (iStart+iHowmany > len) {
			
			memmove(ptr,ptr+(iHowmany+iStart-len),sizeof(long)*(len-iHowmany));
			len-= iHowmany;
			
			iReturn = 0;
		} else {
			long * look = ptr+iStart;
			memmove(look,look+iHowmany,sizeof(long)*(len-iStart-iHowmany));
			len -= iHowmany;
			iReturn = iStart;
		};
		
		if (len <= 0) {
				printf("Pls don't use remove_elements to delte all elemetns. \n");
				getch();
		};
		return iReturn;
		// check this over again.		
	}

smartlong::~smartlong()
	{
		if (ptr != NULL) free(ptr);
	}


Triangle::Triangle()	
	{
		indicator = 0;
	}
	
	
	
	int Triangle::FindNeighbour(Triangle * pTri)
	{
		if (pTri == neighbours[0]) return 0;
		if (pTri == neighbours[1]) return 1;
		if (pTri == neighbours[2]) return 2;
		return -1;
	}
	
	void Triangle::IncrementPeriodic(void)
	{
		++periodic;
		if (periodic == 3) periodic = 0;
	}
	void Triangle::DecrementPeriodic(void)
	{
		--periodic;
		if (periodic < 0) periodic = 2;
	}
	
	void TriMesh::SetTriangleVertex(int iWhichCorner, Triangle * pTri, Vertex * pVertex)
	{
		pTri->cornerptr[iWhichCorner] = pVertex;
		pVertex->AddTriIndex(pTri-T);
	}


bool Triangle::ContainsPointInterior (Vertex * pVert)
{
	if (cornerptr[0] == pVert) return false;
	if (cornerptr[1] == pVert) return false;
	if (cornerptr[2] == pVert) return false;

	return ContainsPoint(pVert->pos.x,pVert->pos.y);
}

// Helper function:
void GetIntercept(const Vector2 & a1,const Vector2 & b1, const Vector2 & a2, const Vector2 & b2,
								Vector2 * pIntercept)
{
	// where does (a1 -> b1) cross (a2 -> b2) ?

	real t1 = ((a1.x-a2.x)*(b2.y-a2.y)-(b2.x-a2.x)*(a1.y-a2.y))/
			((a1.x-b1.x)*(b2.y-a2.y)-(b2.x-a2.x)*(a1.y-b1.y));

	pIntercept->x = a1.x + t1*(b1.x-a1.x);
	pIntercept->y = a1.y + t1*(b1.y-a1.y);
}

/*real GetPossiblyPeriodicDist(Vertex * pVert1, Vertex * pVert2)
{
	real dist1sq,dist2sq,dist3sq,mindistsq;
	Vector2 uL,uR;
	
	uL = Anticlockwise*pVert1->pos;
	uR = Clockwise*pVert1->pos;

	dist1sq = (pVert2->pos.x-uL.x)*(pVert2->pos.x-uL.x)+(pVert2->pos.y-uL.y)*(pVert2->pos.y-uL.y);
	dist2sq = (pVert2->pos.x-pVert1->pos.x)*(pVert2->pos.x-pVert1->pos.x)+(pVert2->pos.y-pVert1->pos.y)*(pVert2->pos.y-pVert1->pos.y);
	dist3sq = (pVert2->pos.x-uR.x)*(pVert2->pos.x-uR.x)+(pVert2->pos.y-uR.y)*(pVert2->pos.y-uR.y);
	
	mindistsq = min(dist1sq,min(dist2sq,dist3sq));
	return sqrt(mindistsq);
}*/ // use GetPossiblyPeriodicDist(pVert1->pos,pVert2->pos);

real CalculateAngle(real x, real y)
{
	static const real TWOPI = 2.0*PI;
	real angle = atan2(y,x);
	if (angle < 0.0) angle += TWOPI;

#ifdef DEBUG
	if (((x > 0.0) && (y > 0.0)) && ((angle > PI*0.5) || (angle < 0.0)))
	{
		x = x;
	};
	if (((x < 0.0) && (y > 0.0)) && ((angle > PI) || (angle < PI*0.5)))
	{
		 x = x;
	};
	if (((x < 0.0) && (y < 0.0)) && ((angle > PI*1.5) || (angle < PI)))
	{
		 x= x;
	};
	if (((x > 0.0) && (y < 0.0)) && ((angle > PI*2.0) || (angle < PI*1.5)))
	{
		 x= x;
	};
#endif
	return angle;
}

real GetPossiblyPeriodicDist(Vector2 & vec1, Vector2 & vec2)
{
	real dist1sq,dist2sq,dist3sq,mindistsq;
	Vector2 uL,uR;
	uL = Anticlockwise*vec1;
	uR = Clockwise*vec1;
	dist1sq = (vec2.x-uL.x)*(vec2.x-uL.x)+(vec2.y-uL.y)*(vec2.y-uL.y);
	dist2sq = (vec2.x-vec1.x)*(vec2.x-vec1.x)+(vec2.y-vec1.y)*(vec2.y-vec1.y);
	dist3sq = (vec2.x-uR.x)*(vec2.x-uR.x)+(vec2.y-uR.y)*(vec2.y-uR.y);
	
	mindistsq = min(dist1sq,min(dist2sq,dist3sq));
	return sqrt(mindistsq);
}

real GetPossiblyPeriodicDistSq(Vector2 & vec1, Vector2 & vec2)
{
	real dist1sq,dist2sq,dist3sq,mindistsq;
	Vector2 uL,uR;
	uL = Anticlockwise*vec1;
	uR = Clockwise*vec1;
	dist1sq = (vec2.x-uL.x)*(vec2.x-uL.x)+(vec2.y-uL.y)*(vec2.y-uL.y);
	dist2sq = (vec2.x-vec1.x)*(vec2.x-vec1.x)+(vec2.y-vec1.y)*(vec2.y-vec1.y);
	dist3sq = (vec2.x-uR.x)*(vec2.x-uR.x)+(vec2.y-uR.y)*(vec2.y-uR.y);
	
	mindistsq = min(dist1sq,min(dist2sq,dist3sq));
	return (mindistsq);
}
/*real GetPossiblyPeriodicDistSq(Vertex * pVert1, Vector2 & u)
{
	real dist1sq,dist2sq,dist3sq,mindistsq;
	Vector2 uL,uR;

	pVert1->periodic_image(uL,0,1);
	pVert1->periodic_image(uR,1,1);
	dist1sq = (u.x-uL.x)*(u.x-uL.x)+(u.y-uL.y)*(u.y-uL.y);
	dist2sq = (u.x-pVert1->pos.x)*(u.x-pVert1->pos.x)+(u.y-pVert1->pos.y)*(u.y-pVert1->pos.y);
	dist3sq = (u.x-uR.x)*(u.x-uR.x)+(u.y-uR.y)*(u.y-uR.y);
	
	mindistsq = min(dist1sq,min(dist2sq,dist3sq));
	return mindistsq;
}

real GetPossiblyPeriodicDistSq(real x1, real y1, real x2, real y2)
{
	real dist1sq,dist2sq,dist3sq,mindistsq;
	Vector2 uL,uR;

	Vector2 u1(x1,y1), u2(x2,y2);
	uL = Anticlockwise*u1;
	uR = Clockwise*u1;
	dist1sq = (uL.x-u2.x)*(uL.x-u2.x)+(u2.y-uL.y)*(u2.y-uL.y);
	dist2sq = (u1.x-u2.x)*(u1.x-u2.x)+(u1.y-u2.y)*(u1.y-u2.y);
	dist3sq = (u2.x-uR.x)*(u2.x-uR.x)+(u2.y-uR.y)*(u2.y-uR.y);
	
	mindistsq = min(dist1sq,min(dist2sq,dist3sq));
	return mindistsq;
}

real GetPossiblyPeriodicDistAcrossTriangle(Triangle * pTri,int which)
{
	int i1,i2;
	real linex,liney,modulus,dist1x,dist1y;
		// to avoid periodic woes, if it's periodic then we map to left(?) and then
		// call again for our temporary triangle
		// (Make sure any pointers internal to Triangle are reset before it goes out of scope!)

	if (pTri->periodic == 1)
	{
		// one point clockwise wrapped
		// unwrap...
		Triangle Tri2;
		Vertex Tempvert;
		Vector2 u;

		i1 = pTri->GetLeftmostIndex(); 
		for (int i = 0; i < 3; i++)
		{
			if (i == i1)
			{
				pTri->cornerptr[i1]->periodic_image(u,1,1);
				Tempvert.x = u.x;
				Tempvert.y = u.y;
				Tri2.cornerptr[i] = &Tempvert;
			} else {
				Tri2.cornerptr[i] = pTri->cornerptr[i];
			};
		};
		Tri2.periodic = 0;

		return GetPossiblyPeriodicDistAcrossTriangle(&Tri2,which);
	};
	if (pTri->periodic == 2)
	{
		// one point not clockwise wrapped
		Triangle Tri2;
		Vertex Tempvert;
		Vector2 u;
		
		i1 = pTri->GetRightmostIndex(); 
		for (int i = 0; i < 3; i++)
		{
			if (i == i1)
			{
				pTri->cornerptr[i1]->periodic_image(u,0,1);
				Tempvert.x = u.x;
				Tempvert.y = u.y;
				Tri2.cornerptr[i] = &Tempvert;
			} else {
				Tri2.cornerptr[i] = pTri->cornerptr[i];
			};
		};
		Tri2.periodic = 0;

		return GetPossiblyPeriodicDistAcrossTriangle(&Tri2,which);
	};


	//// distance across from cornerptr[which]

	//if ((pTri->flags == TRIFLAG_LOWWEDGE) || (pTri->flags == TRIFLAG_HIGHWEDGE))
	//{
	//	// Assume which == 0 or which == 1

	//	i1 = 1-which;
	//	
	//	linex = pTri->cornerptr[i1]->x;
	//	liney = pTri->cornerptr[i1]->y;
	//	
	//	modulus = sqrt(linex*linex+liney*liney);
	//	linex /= modulus;
	//	liney /= modulus;

	//	dist1x = pTri->cornerptr[which]->x-pTri->cornerptr[i1]->x;
	//	dist1y = pTri->cornerptr[which]->y-pTri->cornerptr[i1]->y;
	//		
	//	// project on to (liney,-linex)

	//	return fabs(dist1x*liney - dist1y*linex);
	//};

	// Triangle...
		
	i1 = which+1;
	i2 = which+2;
	if (i1 == 3) i1 = 0;
	if (i2 > 2) i2 -= 3;

	linex = pTri->cornerptr[i1]->x-pTri->cornerptr[i2]->x;
	liney = pTri->cornerptr[i1]->y-pTri->cornerptr[i2]->y;

	modulus = sqrt(linex*linex+liney*liney);
	linex /= modulus;
	liney /= modulus;

	// we want the distance to that line...

	dist1x = pTri->cornerptr[which]->x-pTri->cornerptr[i1]->x;
	dist1y = pTri->cornerptr[which]->y-pTri->cornerptr[i1]->y;
	
	// project on to (liney,-linex)

	return fabs(dist1x*liney - dist1y*linex);

}


real GetSqDistance_SetGlobalFlagNeedPeriodicImage(Vertex * pVertSrc, Vertex * pVert2)
{
	real distx = pVertSrc->x-pVert2->pos.x;
	real disty = pVertSrc->y-pVert2->pos.y;
	if (GlobalPeriodicSearch)
	{
		// in this case we check for Clockwise and anti-Clockwise rotations
		Vector2 u_anti;
		Vector2 u_clock;

		pVertSrc->periodic_image(u_anti,0,1); // Anticlockwise
		pVertSrc->periodic_image(u_clock,1,1);
	
		real distx_anti,disty_anti,distx_clock,disty_clock;
		distx_anti = u_anti.x-pVert2->pos.x;
		disty_anti = u_anti.y-pVert2->pos.y;
		distx_clock = u_clock.x-pVert2->pos.x;
		disty_clock = u_clock.y-pVert2->pos.y;

		real distsqanti = distx_anti*distx_anti+disty_anti*disty_anti;
		real distsq0 = distx*distx+disty*disty;
		real distsqclock = distx_clock*distx_clock+disty_clock*disty_clock;
		
		// If we find that Clockwise is nearest, set a flag on VertSrc
		// If we find that Anticlockwise is nearest, set a flag on VertSrc
			
		if (distsq0 < distsqanti)
		{
			if (distsq0 < distsqclock) {
				return distsq0;
			} else {
				//pVertSrc->flags |= VERTFLAGS_INFLUENCE_Anticlockwise;
				GlobalFlagNeedPeriodicImage = true;
				return distsqclock;
			};
		} else {
			if (distsqanti < distsqclock) {
				//pVertSrc->flags |= VERTFLAGS_INFLUENCE_CLOCKWISE;
				GlobalFlagNeedPeriodicImage = true;
				return distsqanti;
			} else { 
				//pVertSrc->flags |= VERTFLAGS_INFLUENCE_Anticlockwise; // this vertex was Clockwise rotated.
				GlobalFlagNeedPeriodicImage = true;
				return distsqclock;
			};
		};
	};
	return distx*distx+disty*disty;
}

/*real GetSqDistance_SetPerInfluenceFlagOnVertex_Full(Vertex * pVertSrc, Vertex * pVert2, real * pRetDistx, real * pRetDisty)
{
	real distx = pVertSrc->x-pVert2->pos.x;
	real disty = pVertSrc->y-pVert2->pos.y;
	if (GlobalPeriodicSearch)
	{
		// in this case we check for Clockwise and anti-Clockwise rotations
		Vector2 u_anti;
		Vector2 u_clock;

		pVertSrc->periodic_image(u_anti,0,1); // Anticlockwise
		pVertSrc->periodic_image(u_clock,1,1);
	
		real distx_anti,disty_anti,distx_clock,disty_clock;
		distx_anti = u_anti.x-pVert2->pos.x;
		disty_anti = u_anti.y-pVert2->pos.y;
		distx_clock = u_clock.x-pVert2->pos.x;
		disty_clock = u_clock.y-pVert2->pos.y;

		real distsqanti = distx_anti*distx_anti+disty_anti*disty_anti;
		real distsq0 = distx*distx+disty*disty;
		real distsqclock = distx_clock*distx_clock+disty_clock*disty_clock;
		
		// If we find that Clockwise is nearest, set a flag on VertSrc
		// If we find that Anticlockwise is nearest, set a flag on VertSrc
			
		if (distsq0 < distsqanti)
		{
			if (distsq0 < distsqclock) {
				*pRetDistx = distx;
				*pRetDisty = disty;
				return distsq0;
			} else {
				pVertSrc->flags |= VERTFLAGS_INFLUENCE_Anticlockwise;
				*pRetDistx = distx_clock;
				*pRetDisty = disty_clock;
				return distsqclock;
			};
		} else {
			if (distsqanti < distsqclock) {
				pVertSrc->flags |= VERTFLAGS_INFLUENCE_CLOCKWISE;
				*pRetDistx = distx_anti;
				*pRetDisty = disty_anti;
				return distsqanti;
			} else { 
				pVertSrc->flags |= VERTFLAGS_INFLUENCE_Anticlockwise; // this vertex was Clockwise rotated.
				*pRetDistx = distx_clock;
				*pRetDisty = disty_clock;
				return distsqclock;
			};
		};
	};
	*pRetDistx = distx;
	*pRetDisty = disty;
	return distx*distx+disty*disty;
}
*/

int sgn(real x)
{
	if (x > 0.0) return 1;
	return -1;
};

int GetNumberSharedVertices(Triangle & tri1, Triangle & tri2)
{
	int match;
	int matches = 0;
	for (int i = 0; i < 3; i++)
	{
		match = 0;
		for (int j = 0; j < 3; j++)
			if (tri1.cornerptr[i] == tri2.cornerptr[j])
				match = 1;
		if (match == 1) matches++;
	};
	return matches;
}

real Triangle::ReturnAngle(Vertex * pVertex)
{
	Vector2 v1,v2,u[3];
	real dotproduct_over_moduli,weight;
	static const real TWOPI = 2.0*PI;

	MapLeftIfNecessary(u[0],u[1],u[2]);

	if (pVertex == cornerptr[0]) {
		v1 = u[1]-u[0];
		v2 = u[2]-u[0];				
	} else {
		if (pVertex == cornerptr[1]) {
			v1 = u[0]-u[1];
			v2 = u[2]-u[1];
		} else {
			v1 = u[0]-u[2];
			v2 = u[1]-u[2];
		};
	};

	dotproduct_over_moduli = (v1.x*v2.x+v1.y*v2.y)/
				sqrt((v1.x*v1.x+v1.y*v1.y)*(v2.x*v2.x+v2.y*v2.y));

	weight = acos(dotproduct_over_moduli)/TWOPI;
	return weight;
}

Vector2 Triangle::RecalculateCentroid()
{
	Vector2 u[3];
	MapLeftIfNecessary(u[0], u[1], u[2]);
	cent = (u[0] + u[1] + u[2]) / 3.0;

	if (u8domain_flag == CROSSING_INS)
	{
		// Modify the centre to be the centre of the intersection of insulator

		if (SWITCH_TO_CENTRE_OF_INTERSECTION_WITH_INSULATOR_FOR_TRI_CENTROID_CPU)
		{
			GetCentreOfIntersectionWithInsulator(cent);
		} else {
			Vector2 cent2 = cent;
			cent2.project_to_radius(cent, DEVICE_RADIUS_INSULATOR_OUTER);
			if (bDebugReorder) printf("%d %d  :  %d %d\n",
				cent2.x, cent2.y, cent.x, cent.y);
		}
	}
	if (u8domain_flag == OUTER_FRILL) {
		Vector2 temp = 0.5*(u[0] + u[1]); // ? compare to GPU
	};
	if (u8domain_flag == INNER_FRILL) {
		Vector2 temp = 0.5*(u[0] + u[1]); // ? compare to GPU
		
	};
	return cent;
}

Vector2 Triangle::RecalculateCentroid(real InnermostFrillCentroidRadius,real OutermostFrillCentroidRadius)
{
	Vector2 u[3];
	MapLeftIfNecessary(u[0],u[1],u[2]);
	cent = (u[0]+u[1]+u[2])/3.0;

	if (u8domain_flag == CROSSING_INS)
	{
		if (SWITCH_TO_CENTRE_OF_INTERSECTION_WITH_INSULATOR_FOR_TRI_CENTROID_CPU)
		{
			// Modify the centre to be the centre of the intersection of insulator
			GetCentreOfIntersectionWithInsulator(cent);
		} else {
			Vector2 cent2 = cent;
			// Only project if otherwise inside insulator.
			if (cent2.dot(cent2) < DEVICE_RADIUS_INSULATOR_OUTER*DEVICE_RADIUS_INSULATOR_OUTER)
				cent2.project_to_radius(cent, DEVICE_RADIUS_INSULATOR_OUTER);
		};
	};
	if (u8domain_flag == OUTER_FRILL) {
		Vector2 temp = 0.5*(u[0]+u[1]); // ? compare to GPU
		temp.project_to_radius(cent, OutermostFrillCentroidRadius);
	};
	if (u8domain_flag == INNER_FRILL) {
		Vector2 temp = 0.5*(u[0]+u[1]); // ? compare to GPU
		temp.project_to_radius(cent, InnermostFrillCentroidRadius);
	};
	return cent;
}


Vector2 Triangle::GetContiguousCent_AssumingCentroidsSet(Vertex * pVertex)
{
	if (periodic == 0)	return cent;
	// It is assumed that pVertex is one of the corners.
	if (pVertex->pos.x < 0.0) return cent;
	return Clockwise*cent;
}

void TriMesh::Recalculate_TriCentroids_VertexCellAreas_And_Centroids()
{
	ConvexPolygon cp;
	Triangle * pTri;
	Vertex * pVertex;
	long iVertex, iTri;
	Vector2 u;
	int i;

	// 1. Reset triangle centroids.

	pTri = T;
	for (iTri = 0; iTri < numTriangles; iTri++)
	{
		pTri->RecalculateCentroid(this->InnermostFrillCentroidRadius,
			this->OutermostFrillCentroidRadius);
		++pTri; // this seems like it should still work if we have not wrapped any vertex that moved, even if tri no longer periodic in truth but some pts outside tranche
	};

	// 2. Reset vertex cell areas.

	long izTri[128];
	long tri_len;
	pVertex = X;
	for (iVertex = 0; iVertex < numVertices; iVertex++)
	{
		tri_len = pVertex->GetTriIndexArray(izTri);
		cp.Clear();

		if ((pVertex->flags == CONCAVE_EDGE_VERTEX) ||
			(pVertex->flags == CONVEX_EDGE_VERTEX) )
		{
			for (i = 0; i < tri_len; i++)
			{
				pTri = T + izTri[i];
				u = pTri->GetContiguousCent_AssumingCentroidsSet(pVertex);
				if (u.x*u.x+u.y*u.y < INNER_A_BOUNDARY*INNER_A_BOUNDARY)
					u.project_to_radius(u,INNER_A_BOUNDARY);
				if (u.x*u.x+u.y*u.y > DOMAIN_OUTER_RADIUS*DOMAIN_OUTER_RADIUS)
					u.project_to_radius(u,DOMAIN_OUTER_RADIUS);
				cp.add(u);				
			};
			
			/*
			// Project to a radius ...
			pTri = T + izTri[0];
			u = pTri->GetContiguousCent_AssumingCentroidsSet(pVertex);
			if (pVertex->flags == INNERMOST) {
				u.project_to_radius(u,INNER_A_BOUNDARY);
			} else {
				u.project_to_radius(u,DOMAIN_OUTER_RADIUS);
			};
			cp.add(u);
			
			// Outermost should project to DOMAIN_OUTER_RADIUS.
			// what if innermost should project also?
			// Innermost is on INNERMOST_A_BOUNDARY.
			
			for (i = 0; i < tri_len; i++)
			{
				pTri = T + izTri[i];
				cp.add(pTri->GetContiguousCent_AssumingCentroidsSet(pVertex));
			};
			
			u = pTri->GetContiguousCent_AssumingCentroidsSet(pVertex);
			if (pVertex->flags == INNERMOST) {
				u.project_to_radius(u,INNER_A_BOUNDARY);
			} else {
				u.project_to_radius(u,DOMAIN_OUTER_RADIUS);
			};
			cp.add(u);
			*/
		} else {
		
			for (i = 0; i < tri_len; i++)
			{
				pTri = T + izTri[i];
				cp.add(pTri->GetContiguousCent_AssumingCentroidsSet(pVertex));
			//	if (iVertex == 22537) printf("added %1.9E %1.9E \n", cp.coord[cp.numCoords - 1].x, cp.coord[cp.numCoords - 1].y);
			
			};
		};

		pVertex->AreaCell = cp.GetArea();


	//	pVertex->centroid = cp.CalculateBarycenter();
				
		//if (iVertex == 36685) {
		//	printf("vertex %d flag %d \n",iVertex,pVertex->flags);
		//	for (i = 0; i < cp.numCoords; i++)
		//		printf("%1.5E %1.5E ... %1.5E \n",cp.coord[i].x,cp.coord[i].y,
		//									cp.coord[i].modulus());
		//	printf("\n\n");
		//	
		//	for (i = 0; i < tri_len; i++)
		//	{
		//		pTri = T + izTri[i];
		//		u = pTri->GetContiguousCent_AssumingCentroidsSet(pVertex);
		//		printf("%1.5E %1.5E ... %1.5E \n",u.x,u.y,u.modulus());			
		//	};

		//	getch();
		//}
		//
		++pVertex;
	};
}


void Triangle::MapLeftIfNecessary(Vector2 & u0, Vector2 & u1, Vector2 & u2) const
{	
	PopulatePositions(u0,u1,u2);
	
	if (periodic == 1)
	{
		int o1 = GetLeftmostIndex(); 
		if (o1 != 0) u0 = Anticlockwise*u0;
		if (o1 != 1) u1 = Anticlockwise*u1;
		if (o1 != 2) u2 = Anticlockwise*u2;
		return;
	};
	if (periodic == 2)
	{
		int o1 = GetRightmostIndex();
		if (o1 == 0) u0 = Anticlockwise*u0;
		if (o1 == 1) u1 = Anticlockwise*u1;
		if (o1 == 2) u2 = Anticlockwise*u2;	
		return;
	};
}

real Triangle::GetDomainIntersectionAreaROC(Vector2 u[3],int iWhichMove,Vector2 ROC)
{
	// Call once for each moving corner to get total ROC area.
	// ROC is the rate of change of position u[iWhichMove] which is in the domain.
	bool bDomain[3];
	int iDomain, iWhich, iWhich1, iWhich2;
	Vector2 intercept1, intercept2, ROCintercept1, ROCintercept2,
		dArea_by_d_top, dArea_by_d1,dArea_by_d2;
	real shoelace;

	bDomain[0] = (cornerptr[0]->flags == DOMAIN_VERTEX)?1:0;
	bDomain[1] = (cornerptr[1]->flags == DOMAIN_VERTEX)?1:0;
	bDomain[2] = (cornerptr[2]->flags == DOMAIN_VERTEX)?1:0;

	iDomain = bDomain[0]+bDomain[1]+bDomain[2];

	if (iDomain == 1) {
		if (bDomain[iWhichMove] != 1) {
			printf("dodginesse\n");
			getch();
		} 

		iWhich = 0; while (bDomain[iWhich] == 0) iWhich++;
		iWhich1 = iWhich-1; if (iWhich1 == -1) iWhich1 = 2;
		iWhich2 = iWhich+1; if (iWhich2 == 3) iWhich2 = 0;

		GetInsulatorIntercept(&intercept1,u[iWhich1],u[iWhich]);
		GetInsulatorIntercept(&intercept2,u[iWhich2],u[iWhich]);

		Get_ROC_InsulatorIntercept(&ROCintercept1,u[iWhich1],u[iWhich],ROC);
		Get_ROC_InsulatorIntercept(&ROCintercept2,u[iWhich2],u[iWhich],ROC);

		// cp.GetArea contents:

		// for (i = 0; i < numCoords-1; i++)
		//		area += coord[i].x*coord[i+1].y - coord[i+1].x*coord[i].y;
		// area += coord[i].x*coord[0].y - coord[0].x*coord[i].y;
		// return fabs(area*0.5);

		// ROCArea = sum_i[top & intercepts] dArea/dx_i . dx_i/dt

		// establish which way round shoelace is positive:
		shoelace = u[iWhich].x*u[iWhich2].y - u[iWhich2].x*u[iWhich].y
					  + u[iWhich2].x*u[iWhich1].y - u[iWhich1].x*u[iWhich2].y
					  + u[iWhich1].x*u[iWhich].y - u[iWhich].x*u[iWhich1].y;
		real sign = 1.0;
		if (shoelace < 0.0) sign = -1.0;
		// area = 0.5*sign* that shoelace.
		
		dArea_by_d_top.x = 0.5*sign*(u[iWhich2].y - u[iWhich1].y);
		dArea_by_d_top.y = 0.5*sign*(u[iWhich1].x - u[iWhich2].x);
		dArea_by_d1.x = 0.5*sign*(u[iWhich].y-u[iWhich2].y);
		dArea_by_d1.y = 0.5*sign*(u[iWhich2].x-u[iWhich].x);
		dArea_by_d2.x = 0.5*sign*(u[iWhich1].y-u[iWhich].y);
		dArea_by_d2.y = 0.5*sign*(u[iWhich].x-u[iWhich1].x);

		real answer = dArea_by_d1.dot(ROCintercept1) + dArea_by_d2.dot(ROCintercept2)
						+ dArea_by_d_top.dot(ROC);
		return answer;
	};

	// We consider one corner moving at a time. 

	iWhich = 0; while (bDomain[iWhich] == 1) iWhich++;
	iWhich1 = 0; while (bDomain[iWhich1] == 0) iWhich1++;
	iWhich2 = iWhich1+1; while (bDomain[iWhich2] == 0) iWhich2++;

	GetInsulatorIntercept(&intercept1,u[iWhich1],u[iWhich]);
	GetInsulatorIntercept(&intercept2,u[iWhich2],u[iWhich]);

	if (iWhichMove == iWhich1) {

		Get_ROC_InsulatorIntercept(&ROCintercept1,u[iWhich],u[iWhich1],ROC);
		
		shoelace = u[iWhich1].x*intercept1.y - intercept1.x*u[iWhich1].y
			     + intercept1.x*intercept2.y - intercept2.x*intercept1.y
				 + intercept2.x*u[iWhich2].y - u[iWhich2].x*intercept2.y
				 + u[iWhich2].x*u[iWhich1].y - u[iWhich1].x*u[iWhich2].y;
		real sign = 1.0;
		if (shoelace < 0.0) sign = -1.0;

		dArea_by_d_top.x = 0.5*sign*(intercept1.y - u[iWhich2].y);
		dArea_by_d_top.y = 0.5*sign*(u[iWhich2].x - intercept1.x);
		dArea_by_d1.x = 0.5*sign*(intercept2.y - u[iWhich1].x);
		dArea_by_d1.y = 0.5*sign*(u[iWhich1].y - intercept2.y);

		real answer = dArea_by_d_top.dot(ROC) + dArea_by_d1.dot(ROCintercept1);
		return answer;
	} else {

		Get_ROC_InsulatorIntercept(&ROCintercept2,u[iWhich],u[iWhich2],ROC);

		shoelace = u[iWhich1].x*intercept1.y - intercept1.x*u[iWhich1].y
			     + intercept1.x*intercept2.y - intercept2.x*intercept1.y
				 + intercept2.x*u[iWhich2].y - u[iWhich2].x*intercept2.y
				 + u[iWhich2].x*u[iWhich1].y - u[iWhich1].x*u[iWhich2].y;
		real sign = 1.0;
		if (shoelace < 0.0) sign = -1.0;

		dArea_by_d_top.x = 0.5*sign*(u[iWhich1].y - intercept2.y);
		dArea_by_d_top.y = 0.5*sign*(intercept2.x - u[iWhich2].x);
		dArea_by_d2.x = 0.5*sign*(u[iWhich2].y - intercept1.y);
		dArea_by_d2.y = 0.5*sign*(intercept1.x - u[iWhich2].x);
		
		real answer = dArea_by_d_top.dot(ROC) + dArea_by_d2.dot(ROCintercept2);
		return answer;
	};
}
real Triangle::GetDomainIntersectionArea(bool bUseOwnCoords, Vector2 u[3]) const
{
	ConvexPolygon cp;
	int iDomain, iWhich, iWhich1, iWhich2;
	int bDomain[3];
	Vector2 intercept1, intercept2;

	if (u8domain_flag == OUT_OF_DOMAIN) return 0.0;
	if (u8domain_flag == DOMAIN_TRIANGLE) return this->GetArea();

	if (bUseOwnCoords) MapLeftIfNecessary(u[0],u[1],u[2]); // This gives for the original triangle.

	bDomain[0] = (cornerptr[0]->flags == DOMAIN_VERTEX)?1:0;
	bDomain[1] = (cornerptr[1]->flags == DOMAIN_VERTEX)?1:0;
	bDomain[2] = (cornerptr[2]->flags == DOMAIN_VERTEX)?1:0;

	iDomain = bDomain[0]+bDomain[1]+bDomain[2];

	if (iDomain == 1) {
		iWhich = 0; while (bDomain[iWhich] == 0) iWhich++;
		iWhich1 = 0; while (bDomain[iWhich1] == 1) iWhich1++;
		iWhich2 = iWhich1+1; while (bDomain[iWhich2] == 1) iWhich2++;

		GetInsulatorIntercept(&intercept1,u[iWhich1],u[iWhich]);
		GetInsulatorIntercept(&intercept2,u[iWhich2],u[iWhich]);

		cp.Clear();
		cp.add(intercept1);
		cp.add(u[iWhich]);
		cp.add(intercept2);

		return cp.GetArea();
	};
	if (iDomain != 2) {
		printf("Error in GetDomainIntersectionArea.\n");
		return 0.0;
	};

	iWhich = 0; while (bDomain[iWhich] == 1) iWhich++;
	iWhich1 = 0; while (bDomain[iWhich1] == 0) iWhich1++;
	iWhich2 = iWhich1+1; while (bDomain[iWhich2] == 0) iWhich2++;
	// iWhich1 shall go next to intercept1 in the sequence.

	GetInsulatorIntercept(&intercept1,u[iWhich1],u[iWhich]);
	GetInsulatorIntercept(&intercept2,u[iWhich2],u[iWhich]);

	cp.Clear();
	cp.add(intercept1);
	cp.add(u[iWhich1]);
	cp.add(u[iWhich2]);
	cp.add(intercept2);

	return cp.GetArea();
}

void Triangle::GuessPeriodic(void)
{
	real ratio0,ratio1,ratio2,gradient;
	
	ratio0 = cornerptr[0]->pos.x/cornerptr[0]->pos.y;
	ratio1 = cornerptr[1]->pos.x/cornerptr[1]->pos.y;
	ratio2 = cornerptr[2]->pos.x/cornerptr[2]->pos.y;
	gradient = GRADIENT_X_PER_Y/2.0;

	periodic = 0;

	if (ratio0 > gradient)
	{
		// number periodic is the number of others that are < -GRADIENT_X_PER_Y/3.0	
		if (ratio1 < -gradient)
			++periodic;
		if (ratio2 < -gradient)
			++periodic;
	} else {
		if (ratio1 > gradient)
		{
			if (ratio0 < -gradient)
				++periodic;
			if (ratio2 < -gradient)
				++periodic;
		} else {
			if (ratio2 > gradient)
			{
				if (ratio0 < -gradient)
					++periodic;
				if (ratio1 < -gradient)
					++periodic;
			};
		};
	};	
}

void Triangle::RecalculateEdgeNormalVectors(bool normalise)
{
	int iPrev,iNext;
	Vector2 u[3];

	MapLeftIfNecessary(u[0],u[1],u[2]);

	for (int i = 0; i < 3; i++)
	{
		iPrev = i-1; if (iPrev < 0) iPrev = 2;
		iNext = i+1; if (iNext > 2) iNext = 0;			
		edge_normal[i].x = u[iNext].y-u[iPrev].y;
		edge_normal[i].y = u[iPrev].x-u[iNext].x;
		if (edge_normal[i].dot(u[i]-u[iPrev]) > 0.0)
		{
			// facing the wrong way - should face away from u[i]
			edge_normal[i].x = -edge_normal[i].x;
			edge_normal[i].y = -edge_normal[i].y;
		};
		if (normalise) edge_normal[i].Normalise();
		// NOTE: if normalise == false then the length of edge_normal is the side length -- quite convenient
	};
	// Same code will work even if looking out of the domain.
}


// Better if we make this part of some prototypical base class.
// OR, AuxTriangles just are Triangles. Why not?
/*void AuxTriangle::GuessPeriodic(void)
{
	real ratio0,ratio1,ratio2,gradient;
		
	ratio0 = cornerptr[0]->pos.x/cornerptr[0]->pos.y;
	ratio1 = cornerptr[1]->pos.x/cornerptr[1]->pos.y;
	ratio2 = cornerptr[2]->pos.x/cornerptr[2]->pos.y;
	gradient = GRADIENT_X_PER_Y/2.0;

	periodic = 0;

	if (ratio0 > gradient)
	{
			// number periodic is the number of others that are < -GRADIENT_X_PER_Y/3.0	
		if (ratio1 < -gradient)
			++periodic;
		if (ratio2 < -gradient)
			++periodic;
	} else {
		if (ratio1 > gradient)
		{
			if (ratio0 < -gradient)
				++periodic;
			if (ratio2 < -gradient)
				++periodic;
		} else {
			if (ratio2 > gradient)
			{
				if (ratio0 < -gradient)
					++periodic;
				if (ratio1 < -gradient)
					++periodic;
			};
		};
	};
}


void AuxTriangle::RecalculateEdgeNormalVectors(bool normalise)
{
	// copy of function below
	// !
	int iPrev,iNext;
	Vector2 u[3];

	if (periodic == 0)
	{
		this->PopulatePositions(u[0],u[1],u[2]);
	} else {
		this->MapLeft(u[0],u[1],u[2]);
	};

	for (int i = 0; i < 3; i++)
	{
		iPrev = i-1; if (iPrev < 0) iPrev = 2;
		iNext = i+1; if (iNext > 2) iNext = 0;			
		edge_normal[i].x = u[iNext].y-u[iPrev].y;
		edge_normal[i].y = u[iPrev].x-u[iNext].x;
		if (edge_normal[i].dot(u[i]-u[iPrev]) > 0.0)
		{
			// facing the wrong way - should face away from u[i]
			edge_normal[i].x = -edge_normal[i].x;
			edge_normal[i].y = -edge_normal[i].y;
		};

		if (normalise) edge_normal[i].Normalise();
		// NOTE: if normalise == false then the length of edge_normal is the side length -- quite convenient
	};
	// Same code will work even if looking out of the domain.
}
*/



// unnecessary as far as I know:

/*void AuxTriangle::RecalculateEdgeNormalVectors(bool normalise)
{
	// in CUDA version we will only use edge_normal and get rid of transvec stuff
	
	int iPrev,iNext;
	Vector2 u[3];

	if (periodic == 0)
	{
		this->PopulatePositions(u[0],u[1],u[2]);
	} else {
		this->MapLeft(u[0],u[1],u[2]);
	};

	for (int i = 0; i < 3; i++)
	{
		iPrev = i-1; if (iPrev < 0) iPrev = 2;
		iNext = i+1; if (iNext > 2) iNext = 0;
			
		edge_normal[i].x = u[iNext].y-u[iPrev].y;
		edge_normal[i].y = u[iPrev].x-u[iNext].x;

		if (edge_normal[i].dot(u[i]-u[iPrev]) > 0.0)
		{
			// facing the wrong way - should face away from u[i]
			edge_normal[i].x = -edge_normal[i].x;
			edge_normal[i].y = -edge_normal[i].y;
		};

		if (normalise) edge_normal[i].Normalise();
		// NOTE: if normalise == false then the length of edge_normal is the side length -- quite convenient
	};
	// Same code will work even if looking out of the domain.
}

*/
/*real Triangle::GetShortArea()
{
	Vector2 u0,u1,u2;
	Vector2 u0dash, u1dash;
	real u0mod, u1mod;

	if (flags != 2)
	{
		printf("bad call.\n");
		getch();
	};

	// place u0dash, u1dash at projected coordinates NOTIONAL_DISTANCE further out.

	if (periodic == 0) {
		PopulatePositions(u0,u1,u2);
	} else {
		MapLeft(u0,u1,u2);
	};

	u0mod = u0.modulus();
	u1mod = u1.modulus();
	u0dash = ((u0mod+NOTIONAL_DISTANCE)/u0mod)*u0;
	u1dash = ((u0mod+NOTIONAL_DISTANCE)/u1mod)*u1;

	// shoelace:
	
	//return 0.5*fabs( u0.x*u1.y - u1.x*u0.y
	//						+ u1.x*u2.y - u2.x*u1.y
	//						+ u2.x*u0.y - u0.x*u2.y);

	return 0.5*fabs( u0.x*u0dash.y - u0dash.x*u0.y
							 + u0dash.x*u1dash.y - u1dash.x*u0dash.y
							 + u1dash.x*u1.y - u1.x*u1dash.y
							 + u1.x*u0.y - u0.x*u1.y);
};*/

/*real Triangle::GetNormalDistance(int opp)
{
/*	// Requires that transvec be already set correctly.
	real diffx,diffy;
	// This is only for the non-periodic case -- we need to allow for periodic
	real transmod = sqrt(transvecx[opp]*transvecx[opp]+transvecy[opp]*transvecy[opp]);
	real transhatx = transvecx[opp]/transmod;
	real transhaty = transvecy[opp]/transmod;


	// But how to know if transhat faces in or out??


	// we're not guaranteed this is same as transhat start.
	int which = opp-1;
	if (which < 0) which = 2;


	if (periodic == 0)
	{
		
		// difference dot with trans hat vector = normal distance
		diffx = cornerptr[opp]->x-cornerptr[which]->x;
		diffy = cornerptr[opp]->y-cornerptr[which]->y;

	} else {

		// note that transvec is calculated by moving all to the left.
		
		// difference dot with trans hat vector = normal distance
		Vector2 v_opp, v_which;		

		cornerptr[opp]->periodic_image(v_opp,0);
		cornerptr[which]->periodic_image(v_which,0);

		diffx = v_opp.x-v_which.x;
		diffy = v_opp.y-v_which.y;
	};

	return diffx*transhatx + diffy*transhaty;*/

	// Better to do differently:

		// Pythagoras: 
	
	// side1^2 = x^2 + y^2
	// side2^2 = (a-x)^2 + y^2
	// we want y, we do not know x, we know a & side1,side2.
	
	// side2^2 - side1^2 = a^2 - 2ax
	
	// x = (s2^2 - s1^2 - a^2)/(-2a)
	// y = sqrt(s1^2 - x^2)


	// PERIODIC CASE?
	// Normal distance may need to be given correctly!!!


	// ->proceed to populate 3 Vector2's based on flags, periodic and
	// then do the Pythagoras normal distance calculation.

/*
	Vector2 u1,u2,uO;

	if (flags == 0)
	{
	
		int c1 = opp-1;
		if (c1 < 0) c1 = 2;
		int c2 = opp+1;
		if (c2 > 2) c2 = 0;

		if (periodic == 0)
		{
			cornerptr[c1]->PopulatePosition(u1);
			cornerptr[c2]->PopulatePosition(u2);
			cornerptr[opp]->PopulatePosition(uO);
		} else {
			cornerptr[c1]->periodic_image(u1,0);
			cornerptr[c2]->periodic_image(u2,0);
			cornerptr[opp]->periodic_image(uO,0);
		};
	} else {
		if (flags == 1)
		{
			// low wedge

			// in this case, normal distance is construed relative to
			// the sides and top 

			// X Don't do -->>>  just assume periodic to make code shorter.
			
			if (periodic > 0)
			{
				
				if (opp == 0)
				{
					// side goes from 1 to ins

					cornerptr[1]->periodic_image(u1,0);
					cornerptr[1]->project_to_ins_periodic(u2,0);
					cornerptr[0]->periodic_image(uO,0);

				} else {
					if (opp == 1)
					{
						// side goes from 0 to ins

						cornerptr[0]->periodic_image(u1,0);
						cornerptr[0]->project_to_ins_periodic(u2,0);
						cornerptr[1]->periodic_image(uO,0);
					} else {

						// side goes from 0 to 1

						cornerptr[0]->periodic_image(u1,0);
						cornerptr[1]->periodic_image(u2,0);
						cornerptr[0]->project_to_ins_periodic(uO,0);
					};
				};
			} else {
				if (opp == 0)
				{
					// side goes from 1 to ins

					cornerptr[1]->PopulatePosition(u1);
					cornerptr[1]->project_to_ins(u2);
					cornerptr[0]->PopulatePosition(uO);

				} else {
					if (opp == 1)
					{
						// side goes from 0 to ins

						cornerptr[0]->PopulatePosition(u1);
						cornerptr[0]->project_to_ins(u2);
						cornerptr[1]->PopulatePosition(uO);
					} else {

						// side goes from 0 to 1

						cornerptr[0]->PopulatePosition(u1);
						cornerptr[1]->PopulatePosition(u2);
						cornerptr[0]->project_to_ins(uO);
					};
				};
			};
		} else {
			if (periodic > 0)
			{
				// high wedge
				if (opp == 0)
				{
					// side goes from 1 to ins

					cornerptr[1]->periodic_image(u1,0);
					cornerptr[1]->project_to_100cm_periodic(u2,0);
					cornerptr[0]->periodic_image(uO,0);

				
				} else {
					if (opp == 1)
					{
						// side goes from 0 to ins

						cornerptr[0]->periodic_image(u1,0);
						cornerptr[0]->project_to_100cm_periodic(u2,0);
						cornerptr[1]->periodic_image(uO,0);
					} else {

						// side goes from 0 to 1

						cornerptr[0]->periodic_image(u1,0);
						cornerptr[1]->periodic_image(u2,0);
						cornerptr[0]->project_to_100cm_periodic(uO,0);
					};
				};
			} else {
				// high wedge
				if (opp == 0)
				{
					// side goes from 1 to ins

					cornerptr[1]->PopulatePosition(u1);
					cornerptr[1]->project_to_100cm(u2);
					cornerptr[0]->PopulatePosition(uO);

				
				} else {
					if (opp == 1)
					{
						// side goes from 0 to ins

						cornerptr[0]->PopulatePosition(u1);
						cornerptr[0]->project_to_100cm(u2);
						cornerptr[1]->PopulatePosition(uO);
					} else {

						// side goes from 0 to 1

						cornerptr[0]->PopulatePosition(u1);
						cornerptr[1]->PopulatePosition(u2);
						cornerptr[0]->project_to_100cm(uO);
					};
				};
			};
		};
	};

	
	real dist1sq = (u1.x-uO.x)*(u1.x-uO.x)+(u1.y-uO.y)*(u1.y-uO.y);
	real dist2sq = (u2.x-uO.x)*(u2.x-uO.x)+(u2.y-uO.y)*(u2.y-uO.y);
	real distasq = (u1.x-u2.x)*(u1.x-u2.x)+(u1.y-u2.y)*(u1.y-u2.y);;
		
	real x = (dist2sq - dist1sq - distasq)/(-2.0*sqrt(distasq));
	real y = sqrt(dist1sq - x*x);
	return y;
	
}*/
	
/*real Vertex::CalculateVoronoiArea()
{
	// Voronoi area is found assuming that ... circumcenters were already calculated.
	// (!)
	// and stored as pTri->numerator_x,y
	Triangle * pTri;
	ConvexPolygon cp;
	Vector2 circumcenter, cc;
	real theta;
	int i,j,k;
	real angle[100];
	int index[100];
	Proto * tempptr[100];
	
	if (triangles.len >= 100)
	{
		printf("\ncannot do it - static array not big enough\n");
		getch();
	};

	// First got to sort the triangles:
	for (i = 0; i < triangles.len; i++)
	{
		pTri = (Triangle *)(triangles.ptr[i]);
		
		pTri->ReturnCentre(&cc,this); 
		// For now do this lazy and inefficient trig way:
		theta = CalculateAngle(cc.x-x,cc.y-y);		
		j = 0;
		while ((j < i) && (theta > angle[j])) j++; // if i == 1 then we can only move up to place 1, since we have 1 element already
		if (j < i) {
			// move the rest of them forward in the list:
			for (k = i; k > j ; k--)
			{
				index[k] = index[k-1];
				angle[k] = angle[k-1];
			};
		}
		angle[j] = theta;
		index[j] = i;
	};			
	for (i = 0; i < triangles.len; i++)
		tempptr[i] = triangles.ptr[index[i]];
	for (i = 0; i < triangles.len; i++)
		triangles.ptr[i] = tempptr[i];			


	for (i = 0; i < triangles.len; i++)
	{
		pTri = (Triangle *)(triangles.ptr[i]);
		circumcenter.x = pTri->numerator_x;
		circumcenter.y = pTri->numerator_y;

		if ((pTri->periodic > 0) && (x > 0.0))
		{
			// for a per tri, the circumcenter is found for left image.
			// if our point on the right, circumcenter will need to be mapped over.
			circumcenter = Clockwise*circumcenter;
		};
		cp.add(circumcenter);
	};

	return cp.GetArea();
}

*/
void Triangle::ReturnPositionOtherSharedVertex_conts_tranche(Triangle * pTri, Vertex * pVert, Vector2 * pResult)
{
	// First find the common vertex that is not pVert :
	int iShared;

	if (pVert == cornerptr[0])
	{
		if (   (pTri->cornerptr[0] == cornerptr[1])
			|| (pTri->cornerptr[1] == cornerptr[1])
			|| (pTri->cornerptr[2] == cornerptr[1]) )
		{
			//cornerptr[1] is it
			iShared = 1;
		} else {
			iShared = 2; 
			
			// DEBUG:
			if ((pTri->cornerptr[0] != cornerptr[2])
			&& (pTri->cornerptr[1] != cornerptr[2])
			&& (pTri->cornerptr[2] != cornerptr[2]) )
			{
				printf("!JDdewjiw!\n"); getch();
			};		
		};
	} else {
		if (pVert == cornerptr[1]) 
		{
			if ((pTri->cornerptr[0] == cornerptr[0])
			|| (pTri->cornerptr[1] == cornerptr[0])
			|| (pTri->cornerptr[2] == cornerptr[0]) )
			{
				iShared = 0;
			} else {
				iShared = 2; 

				// DEBUG:
				if (   (pTri->cornerptr[0] != cornerptr[2])
				&& (pTri->cornerptr[1] != cornerptr[2])
				&& (pTri->cornerptr[2] != cornerptr[2]) )
				{
					printf("!JDdewjiw!\n"); getch();
				};		
			};
		} else {
			// pVert == cornerptr[2]
			if (   (pTri->cornerptr[0] == cornerptr[0])
			|| (pTri->cornerptr[1] == cornerptr[0])
			|| (pTri->cornerptr[2] == cornerptr[0]) )
			{
				iShared = 0;
			} else {
				iShared = 1; 

				// DEBUG:
				if (   (pTri->cornerptr[0] != cornerptr[1])
				&& (pTri->cornerptr[1] != cornerptr[1])
				&& (pTri->cornerptr[2] != cornerptr[1]) )
				{
					printf("!JDdewjiw!\n"); getch();
				};		
			};
		};
	};

	// Populate position with same wrapping as pVert .
	// If the vertex is INS_VERT or HIGH_VERT -- as it may be -- then populate by projection.

	//if (cornerptr[iShared] == INS_VERT)
	//{
	//	pVert->project_to_ins(*pResult);
	//} else {
	//	if (cornerptr[iShared] == HIGH_VERT)
	//	{
	//		pVert->project_to_radius(*pResult, HIGH_WEDGE_OUTER_RADIUS);
	//	} else {
	if (periodic == 0)
	{
		// usual case:
		*pResult = cornerptr[iShared]->pos;
	} else {
		// periodic; not at inner or outer boundary
		*pResult = cornerptr[iShared]->pos;

		int iVert = 0;
		while (pVert != cornerptr[iVert]) iVert++;
		if (periodic == 1)
		{
			int o = GetLeftmostIndex();
			// o is the wrapped point
			if (o == iVert)
			{
				// want to wrap the other point anticlockwise:				
				*pResult = Anticlockwise*(*pResult);
			} else {
				if (o == iShared) {
					// unwrap the other point to clockwise:
					*pResult = Clockwise*(*pResult);					
				};
			};
		} else {
			int o = GetRightmostIndex();
			if (o == iVert)
			{
				// unwrap the other point clockwise:
				*pResult = Clockwise*(*pResult);	
			} else {
				if (o == iShared) 
				{
					// wrap it anticlockwise:
					*pResult = Anticlockwise*(*pResult);
				};
			};
		};
	};
	//	};
	//};
	//	

}

Vertex * Triangle::ReturnOtherSharedVertex(Triangle * pTri,Vertex * pVertex)
{
	if ((cornerptr[0] != pVertex) && (pTri->has_vertex(cornerptr[0]))) return cornerptr[0];
	if ((cornerptr[1] != pVertex) && (pTri->has_vertex(cornerptr[1]))) return cornerptr[1];
	return cornerptr[2];
}
/*
AuxVertex * AuxTriangle::ReturnUnsharedVertex(AuxTriangle * pTri2, int * pwhich)
{
	
	if (   (pTri2->cornerptr[0] == cornerptr[0])
			|| (pTri2->cornerptr[1] == cornerptr[0])
			|| (pTri2->cornerptr[2] == cornerptr[0]) )
	{
		// it's not vertices[0]
		if (   (pTri2->cornerptr[0] == cornerptr[1])
			|| (pTri2->cornerptr[1] == cornerptr[1])
			|| (pTri2->cornerptr[2] == cornerptr[1]) )
		{
			if (pwhich != 0) *pwhich = 2;
			return cornerptr[2];  // which might well be 0 
		} else {
			if (pwhich != 0) *pwhich = 1;
			return cornerptr[1];
		};
	} else {
		if (pwhich != 0) *pwhich = 0;
		return cornerptr[0];
	};
}
*/

Vertex * Triangle::ReturnUnsharedVertex(Triangle * pTri2, int * pwhich) // pwhich = 0
{
	// test each one in turn.

	//if (flags == 0)
	//{

	// in case of wedge, we do want to return 0 if it is not sharing either of the top ones.

	if (   (pTri2->cornerptr[0] == cornerptr[0])
			|| (pTri2->cornerptr[1] == cornerptr[0])
			|| (pTri2->cornerptr[2] == cornerptr[0]) )
	{
		// it's not vertices[0]

		if (   (pTri2->cornerptr[0] == cornerptr[1])
			|| (pTri2->cornerptr[1] == cornerptr[1])
			|| (pTri2->cornerptr[2] == cornerptr[1]) )
		{
			if (pwhich != 0) *pwhich = 2;
			return cornerptr[2];  // which might well be 0 
		} else {
			if (pwhich != 0) *pwhich = 1;
			return cornerptr[1];
		};
	} else {
		if (pwhich != 0) *pwhich = 0;
		return cornerptr[0];
	};
	//} else {
	//	// Only test vertex 0 and 1.

	//	if (   (pTri2->cornerptr[0] == cornerptr[0])
	//		|| (pTri2->cornerptr[1] == cornerptr[0])
	//		|| (pTri2->cornerptr[2] == cornerptr[0]) )
	//	{
	//		// it's not vertices[0]

	//		if (pwhich != 0) *pwhich = 1;
	//		return cornerptr[1];
	//		
	//	} else {
	//		
	//		if (pwhich != 0) *pwhich = 0;
	//		return cornerptr[0];
	//	};
	//};
}


//int Triangle::DecodeSign(int other)
//{
//	switch(other)
//	{
//	case 0:
//		return ((sign_other_dot_transvec & TRIFLAG_SIGN0) > 0)?1:-1;
//	case 1:
//		return ((sign_other_dot_transvec & TRIFLAG_SIGN1) > 0)?1:-1;
//	case 2:
//		return ((sign_other_dot_transvec & TRIFLAG_SIGN2) > 0)?1:-1;
//	}
//	return 0; // just to suppress warning
//}

// double-precision overload:


// NEW TESTAGAINSTEDGE FUNCTIONS:

/*
int Triangle::TestAgainstEdge(float x,float y, 
							int c1, // the "start" of the relevant edge
							  int other, // the point opposite the relevant edge
							  Triangle ** ppNeigh)
{
	// returns 1 in the event that (x,y) is outside the triangle.

	Vector2 u1;//, u2, uO;
	bool outside;
//	Vector2 edge;
	Vector2 transverse;
	//long Tindex;
	real x_dot_transverse;//,other_dot_transverse;
	
	if (periodic == 0)
	{
		cornerptr[c1]->PopulatePosition(u1);
	} else {
		// ensure c1 is mapped to left if need be:
		if (periodic == 1)
		{
			int iMapped = GetLeftmostIndex();
			if (iMapped == c1)
			{
				cornerptr[c1]->PopulatePosition(u1);
			} else {
				cornerptr[c1]->periodic_image(u1,0,1); 
			};
		} else {
			int iUnmapped = GetRightmostIndex();
			if (iUnmapped == c1)
			{
				cornerptr[c1]->periodic_image(u1,0,1); 
			} else {
				cornerptr[c1]->PopulatePosition(u1);
			};
		};
	};

	x_dot_transverse = (x-u1.x)*transvecx[other] + (y-u1.y)*transvecy[other];
		
	outside = (sgn(x_dot_transverse) == DecodeSign(other))?0:1;

	if (outside)
	{
		*ppNeigh = neighbours[other]; // neighbours is now always a valid value.

		return 1;
	};
	return 0;
}
	*/

int Triangle::TestAgainstEdge(real x,real y, 
							int c1, // the "start" of the relevant edge
							  int other, // the point opposite the relevant edge
							  Triangle ** ppNeigh)
{
	// returns 1 in the event that (x,y) is outside the triangle.

	Vector2 u1;//, u2, uO;
	bool outside;
	Vector2 transverse;
	real x_dot_transverse;
	
	u1 = cornerptr[c1]->pos;
	if (periodic == 0)
	{
	} else {
		// ensure c1 is mapped to left if need be:
		if (periodic == 1)
		{
			int iMapped = GetLeftmostIndex();
			if (iMapped != c1)
				u1 = Anticlockwise*u1;			
		} else {
			int iUnmapped = GetRightmostIndex();
			if (iUnmapped == c1)
				u1 = Anticlockwise*u1;
		};
	};

	// Seems this routine assumes that we can test against a left-mapped periodic triangle.

	x_dot_transverse = (x-u1.x)*edge_normal[other].x + (y-u1.y)*edge_normal[other].y;
		
	outside = (x_dot_transverse > 0.0)?1:0;	// edge_normal points outside

	if (outside)
	{
		*ppNeigh = neighbours[other]; // neighbours is now always a valid value.
		if ((neighbours[other]->u8domain_flag == OUTER_FRILL) ||
			(neighbours[other]->u8domain_flag == INNER_FRILL))
			*ppNeigh = this;
		return 1;
	};
	return 0;
}
/*
int Triangle::TestAgainstEdge(real x,real y, 
							int c1, // the "start" of the relevant edge
							  int other, // the point opposite the relevant edge
							  Triangle ** ppNeigh)
{
	// returns 1 in the event that (x,y) is outside the triangle.

		Vector2 u1;//, u2, uO;
	bool outside;
//	Vector2 edge;
	Vector2 transverse;
//	long Tindex;
	real x_dot_transverse;//,other_dot_transverse;
	
	if (periodic == 0)
	{
		cornerptr[c1]->PopulatePosition(u1);
	} else {
		// ensure c1 is mapped to left if need be:
		if (periodic == 1)
		{
			int iMapped = GetLeftmostIndex();
			if (iMapped == c1)
			{
				cornerptr[c1]->PopulatePosition(u1);
			} else {
				cornerptr[c1]->periodic_image(u1,0,1); 
			};
		} else {
			int iUnmapped = GetRightmostIndex();
			if (iUnmapped == c1)
			{
				cornerptr[c1]->periodic_image(u1,0,1); 
			} else {
				cornerptr[c1]->PopulatePosition(u1);
			};
		};
	};

	x_dot_transverse = (x-u1.x)*transvecx[other] + (y-u1.y)*transvecy[other];
		
	outside = (sgn(x_dot_transverse) == DecodeSign(other))?0:1;

	if (outside)
	{
		*ppNeigh = neighbours[other]; // neighbours is now always a valid value.

		return 1;
	};
	return 0;
}
*/

/*



int Triangle::TestAgainstEdge(float x,float y, 
							int c1, // the "start" of the relevant edge
							  int other, // the point opposite the relevant edge
							  Triangle ** ppNeigh)
{
	// returns 1 in the event that (x,y) is outside the triangle.

	Vector2 u1;//, u2, uO;
	bool outside;
//	Vector2 edge;
	Vector2 transverse;
	long Tindex;
	real x_dot_transverse,other_dot_transverse;
	
	
	if (periodic == 0)
	{
		cornerptr[c1]->PopulatePosition(u1);
		//pTri->cornerptr[c2]->PopulatePosition(u2);
		//pTri->cornerptr[other]->PopulatePosition(uO);

		// test whether we are on the inside side of edge [0]<->[1]:
		// %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
		
			// transverse:
//		transverse.x = (u1.y-u2.y);//-edge.y;
//		transverse.y = (u2.x-u1.x);//edge.x; 
//		x_dot_transverse = (x-u1.x)*transverse.x + (y-u1.y)*transverse.y;

		x_dot_transverse = (x-u1.x)*transvecx[other] + (y-u1.y)*transvecy[other];
		
		// now take the vector headed to the other point, to compare:
		//edge.x = uO.x-u1.x;
		//edge.y = uO.y-u1.y;

//		other_dot_transverse = //edge.x*transverse.x + edge.y*transverse.y;
//							(uO.x-u1.x)*transverse.x + (uO.y-u1.y)*transverse.y;

		// we're outside the triangle in the case that these are not the same sign.
		outside = (sgn(x_dot_transverse) == DecodeSign(other))?0:1;

		if (outside)
		{
			// This should work OK for wedges, I think. [2] should send us north.

			*ppNeigh = neighbours[other]; // neighbours is now always a valid value.

	//		Tindex = neighbours[other];
	//		if (Tindex >= 0)
	//		{
	//			*piNeigh = Tindex;
	//			return 1;
	//		} else {
				// if that is the only boundary it's outside then stick with this triangle...
	//			*piNeigh = -2; // this triangle not known
				// -1 indicates outside hopefully the upper boundary
	//			return 0;
	//		};
			return 1;
		};
		return 0;
	} else {
		int side = (x > 0.0f)?1:0;
		Vector2 u2,uO;
		int c2 = other-1; if (c2 == -1) c2 = 2;

		cornerptr[c1]->periodic_image(u1,side);
		cornerptr[c2]->periodic_image(u2,side);
		cornerptr[other]->periodic_image(uO,side);
			
		// test whether we are on the inside side of edge [0]<->[1]:
		// %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
		
			// the vector that runs along this edge:
			// this is why Vertex should have contained a Float2

			// transverse:
		transverse.x = (u1.y-u2.y);//-edge.y;
		transverse.y = (u2.x-u1.x);//edge.x; 
		x_dot_transverse = (x-u1.x)*transverse.x + (y-u1.y)*transverse.y;
		// now take the vector headed to the other point, to compare:
		//edge.x = uO.x-u1.x;
		//edge.y = uO.y-u1.y;
		other_dot_transverse = //edge.x*transverse.x + edge.y*transverse.y;
							(uO.x-u1.x)*transverse.x + (uO.y-u1.y)*transverse.y;
		// we're outside the triangle in the case that these are not the same sign.
		outside = (sgn(x_dot_transverse) == sgn(other_dot_transverse))?0:1;

		if (outside)
		{
			*ppNeigh = neighbours[other];
			return 1;

			////Tindex = pTri->neighbours[other];
			////if (Tindex >= 0)
			////{
			////	*piNeigh = Tindex;
			////	return 1;
			////} else {
			////	// if that is the only boundary it's outside then stick with this triangle...
			////	*piNeigh = -2; // this triangle not known
			////	// -1 indicates outside hopefully the upper boundary

			////	// Thing is, that means we're allowing this triangle's domain to be a spray
			////	// Oh well to that.

			////	return 0;
			////};
		};
		return 0;
	};
}

int Triangle::TestAgainstEdge(real x,real y, 
							int c1, // the "start" of the relevant edge
							  int other, // the point opposite the relevant edge
							  Triangle ** ppNeigh)
{
	// returns 1 in the event that (x,y) is outside the triangle.

	Vector2 u1;//, u2, uO;
	bool outside;
//	Vector2 edge;
	Vector2 transverse;
	long Tindex;
	real x_dot_transverse,other_dot_transverse;

	if (periodic == 0)
	{
		cornerptr[c1]->PopulatePosition(u1);
		
		x_dot_transverse = (x-u1.x)*transvecx[other] + (y-u1.y)*transvecy[other];
		
		//outside = (sgn(x_dot_transverse) == DecodeSign(other))?false:true;

	} else {
		// But should we even be doing this? 
		
		// Note that the recorded transverse sign is set up by using everything mapped to left side.
		// That should be fine?

//		int side = (x > 0.0)?1:0;
		Vector2 u2,uO;

		// In periodic case, we have set up transvec etc for the left-mapped cell.
		// Therefore we are willing to compare multiple periodic images of (x,y) to this cell.
		// If x > 0 however then we might as well just compare the rotated (x,y); if x < 0 just compare (x,y)

		cornerptr[c1]->periodic_image(u1,0);

		Vector2 u;
			
		if (x > 0)
		{
			Vertex temp;
			temp.x = x;
			temp.y = y;
			temp.periodic_image(u,0); 
		} else {
			u.x = x;
			u.y = y;
		};

		x_dot_transverse = (u.x-u1.x)*transvecx[other] + (u.y-u1.y)*transvecy[other];

//		int c2 = 0; while ((c2 == other) || (c2 == c1)) c2++;

//		cornerptr[c1]->periodic_image(u1,side);
//		cornerptr[c2]->periodic_image(u2,side);
		// Nothing to stop c2 from being 0 here .

		// The following brazen function call gives an error because cornerptr[2] might be ==0.
		//cornerptr[other]->periodic_image(uO,side);
		
		// test whether we are on the inside side of edge [0]<->[1]:
		// %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
		
			// transverse:
//		transverse.x = (u1.y-u2.y);//-edge.y;
//		transverse.y = (u2.x-u1.x);//edge.x; 
//		x_dot_transverse = (x-u1.x)*transverse.x + (y-u1.y)*transverse.y;
//		other_dot_transverse = //edge.x*transverse.x + edge.y*transverse.y;
//							(uO.x-u1.x)*transverse.x + (uO.y-u1.y)*transverse.y;
//		outside = (sgn(x_dot_transverse) == sgn(other_dot_transverse))?0:1;

	};

	
	outside = (sgn(x_dot_transverse) == DecodeSign(other))?false:true;

	if (outside)
	{
		*ppNeigh = neighbours[other]; // neighbours is now always a valid value.
		return 1;
	};
	return 0;
}


*/


Triangle * TriMesh::ReturnPointerToOtherSharedTriangle(
		Vertex * pVert,
		Vertex * pOther,
		Triangle * p_not_this_one, int iLevel)
{

	// Ask what tri contains pVert and pOther that is not p_not_this_one
	// Usually used for setting neighbours.

	long tri_len, izTri[128];
	Triangle *pTri,*Tarray;
	long iNot;

	if (iLevel == -1) {
		Tarray = T;
	} else {
		Tarray = AuxT[iLevel];
	};
	iNot = p_not_this_one-Tarray;

	tri_len = pVert->GetTriIndexArray(izTri);
	for (int i = 0; i < tri_len; i++)
	{
		if (izTri[i] != iNot)
		{
			pTri = Tarray + izTri[i];
			// does this triangle also contain otherpoint?
			if (   (pTri->cornerptr[0] == pOther)
				|| (pTri->cornerptr[1] == pOther)
				|| (pTri->cornerptr[2] == pOther) ) // note that since cornerptr[2] == 0 for wedge, this should not cause a problem.
				return pTri;
		}; 
	};
	
	// got here -> no triangle found

	return p_not_this_one;
	// Return itself instead.
}


/*
AuxTriangle * TriMesh::ReturnPointerToOtherSharedTriangle(
		AuxVertex * pVert,
		AuxVertex * pOther,
		AuxTriangle * p_not_this_one)
{

	// Ask what tri contains pVert and pOther that is not p_not_this_one
	// Usually used for setting neighbours.

	// Note that if pOther == 0 then the question is just,
	// what tri contains pVert and is a wedge -
	// in this case p_not_this_one should also be a wedge so that
	// the answer can be unique.
	
	// If pOther == 0 then of course, cornerptr[2] == 0 will test for wedgeness anyway.

//	AuxTriangle ** ptr = (Triangle **)pVert->triangles.ptr;

	AuxTriangle * ptr;
	for (int i = 0; i < pVert->tri_len; i++)
	{
		ptr = InnerT + pVert->iTriangles[i];
		if (ptr != p_not_this_one)
		{
			// does this triangle also contain otherpoint?
			if (   (ptr->cornerptr[0] == pOther)
				|| (ptr->cornerptr[1] == pOther)
				|| (ptr->cornerptr[2] == pOther) ) // note that since cornerptr[2] == 0 for wedge, this should not cause a problem.
				return ptr;
		}; 
	};
	
	return 0;
}*/
/*
AuxTriangle * TriMesh::ReturnPointerToOtherSharedTriangleAux(
		AuxVertex * pVert,
		AuxVertex * pOther,
		AuxTriangle * p_not_this_one,
		int iLevel)
{

	// Ask what tri contains pVert and pOther that is not p_not_this_one
	// Usually used for setting neighbours.

	AuxTriangle * ptr;
	for (int i = 0; i < pVert->tri_len; i++)
	{
		ptr = AuxT[iLevel] + pVert->iTriangles[i];
		if (ptr != p_not_this_one)
		{
			// does this triangle also contain otherpoint?
			if (   (ptr->cornerptr[0] == pOther)
				|| (ptr->cornerptr[1] == pOther)
				|| (ptr->cornerptr[2] == pOther) ) // note that since cornerptr[2] == 0 for wedge, this should not cause a problem.
				return ptr;
		}; 
	};
	
	return p_not_this_one; // default if another neighbour sharing the edge not found.
}
*/

/*Triangle * TriMesh::SearchCornerptr(long index0, long index1, long index2, Triangle * pTriSeed)
{
	// Is pTriSeed, it?
	int test[3];

	Triangle * pTri = pTriSeed;

	test[0] = pTri->cornerptr[0]-X;
	test[1] = pTri->cornerptr[1]-X;
	test[2] = pTri->cornerptr[2]-X;
	
	if (((test[0] == index0) || (test[1] == index0) || (test[2] == index0))
		&&
		((test[0] == index1) || (test[1] == index1) || (test[2] == index1))
		&&
		((test[0] == index2) || (test[1] == index2) || (test[2] == index2)))
		return pTri;
	
	// No? Then search all tris of X+index0

	Vertex * pVertex = X + index0;
	int i;
	for (i = 0; i < pVertex->triangles.len; i++)
	{
		pTri = (Triangle *)(pVertex->triangles.ptr[i]);

		test[0] = pTri->cornerptr[0]-X;
		test[1] = pTri->cornerptr[1]-X;
		test[2] = pTri->cornerptr[2]-X;
		
		if (((test[0] == index0) || (test[1] == index0) || (test[2] == index0))
			&&
			((test[0] == index1) || (test[1] == index1) || (test[2] == index1))
			&&
			((test[0] == index2) || (test[1] == index2) || (test[2] == index2)))
			return pTri;
	}

	return 0; // triangle with these 3 did not exist!
}


int AuxTriangle::TestAgainstEdge(real x,real y, 
							int c1, // the "start" of the relevant edge
							  int other, // the point opposite the relevant edge
							  AuxTriangle ** ppNeigh)
{
	// returns 1 in the event that (x,y) is outside the triangle.

	Vector2 u1;//, u2, uO;
	bool outside;
	Vector2 transverse;
	real x_dot_transverse;
	
	if (periodic == 0)
	{
		cornerptr[c1]->PopulatePosition(u1);
	} else {
		// ensure c1 is mapped to left if need be:
		if (periodic == 1)
		{
			int iMapped = GetLeftmostIndex();
			if (iMapped == c1)
			{
				cornerptr[c1]->PopulatePosition(u1);
			} else {
				cornerptr[c1]->periodic_image(u1,0,1); 
			};
		} else {
			int iUnmapped = GetRightmostIndex();
			if (iUnmapped == c1)
			{
				cornerptr[c1]->periodic_image(u1,0,1); 
			} else {
				cornerptr[c1]->PopulatePosition(u1);
			};
		};
	};

	x_dot_transverse = (x-u1.x)*edge_normal[other] .x+ (y-u1.y)*edge_normal[other].y;
	outside = (x_dot_transverse > 0.0)?1:0;	// edge_normal points outside
	//(sgn(x_dot_transverse) == DecodeSign(other))?0:1;

	if (outside)
	{
		*ppNeigh = neighbours[other]; // neighbours is now always a valid value.

		return 1;
	};
	return 0;
}

bool AuxTriangle::ContainsPoint(real x, real y)
{
	// Note that this is returning true in the case that it is 
	// only outside on the side where neighbours[i] == this.
	// That is a very liberal test for being inside.
	// We ought to probably put a limit on what can be considered azimuthally to belong to this one.	
	AuxTriangle * pNeigh;

	// always test for our triangle unmapped:
	int out = 0;
	if (TestAgainstEdge(x,y,
								0,       // edge corner
								 2,       // the opposite point
								 &pNeigh    // neighbour in this direction, if it's outside this way
								 ) && (neighbours[2] != this))
	{
		out = 1;
	} else {
		if (TestAgainstEdge(x,y, 1, 0, &pNeigh) && (neighbours[0] != this))
		{
			out = 1;
		} else {
			if (TestAgainstEdge(x,y, 0, 1, &pNeigh) && (neighbours[1] != this))
				out = 1;
		};
	};
	
	if (periodic == 0) return (1-out);
	if (out == 0) return (1-out); // found point already in left interpreted tri
	
	// if periodic > 0, we want to also test RH
	int out2 = 0;
	real destx,desty;
	// map point Anticlockwise to represent mapping triangle Clockwise:
	destx = Anticlockwise.xx*x + Anticlockwise.xy*y;
	desty = Anticlockwise.yx*x + Anticlockwise.yy*y;
	
	if ((TestAgainstEdge(destx,desty,0,2,&pNeigh)) && (neighbours[2] != this))
		return false;
	if ((TestAgainstEdge(destx,desty, 1, 0, &pNeigh)) && (neighbours[0] != this))
		return false;
	if ((TestAgainstEdge(destx,desty, 0, 1, &pNeigh)) && (neighbours[1] != this))
		return false;
	return true;		
	
	// Note that this is returning true in the case that it is 
	// only outside on the side where neighbours[i] == this.
	// That is a very liberal test for being inside.
	// We ought to probably put a limit on what can be considered azimuthally to belong to this one.	
}


*/

bool Triangle::ContainsPoint(real x, real y)
{
	Triangle * pNeigh;

	// always test for our triangle unmapped:
	int out = 0;
	if(TestAgainstEdge(x,y,
								0,       // edge corner
								 2,       // the opposite point
								 &pNeigh    // neighbour in this direction, if it's outside this way
								 ))
	{
		out = 1;
	} else {
		if(TestAgainstEdge(x,y, 1, 0, &pNeigh))
		{
			out = 1;
		} else {
			if(TestAgainstEdge(x,y, 0, 1, &pNeigh)) 
				out = 1;
		};
	};

	if (periodic == 0) return (1-out);
	if (out == 0) return (1-out); // found point already in left interpreted tri

	// if periodic > 0, we want to also test RH
	int out2 = 0;
	real destx,desty;
	// map point Anticlockwise to represent mapping triangle Clockwise:
	destx = Anticlockwise.xx*x + Anticlockwise.xy*y;
	desty = Anticlockwise.yx*x + Anticlockwise.yy*y;

	if(TestAgainstEdge(destx,desty,0,2,&pNeigh))
	{
		out2 = 1;
	} else {
		if(TestAgainstEdge(destx,desty, 1, 0, &pNeigh))
		{
			out2 = 1;
		} else {
			if(TestAgainstEdge(destx,desty, 0, 1, &pNeigh)) // do not send vertex 2 here in case it does not exist.
				out2 = 1;
		};
	};
	return (1-out2);
}

// same as above function basically but now periodic lives only on left
bool Triangle::TestAgainstEdges(real x,real y, Triangle ** ppNeigh)
{
	// If an edge triangle, we require it give preference to a neighbour other than itself. 
//	static real const FP_FUZZY_THRESH_LARGE = 1.0e-8;
	// ^^  more sophistication called for. !


	// Two things to concern with it seems:
	
	// If point is outside edge of memory, it should test positive that it is
	// NOT in this triangle... there are no holes in the domain though.
	// Return self in this case. ??
	// Then look at calls to this function to see how to handle that.
	// Maybe change function return type to int, return a flag for Out Of Memory Domain.

	// However we must FIRST test against the other edges. This rules out that
	// the point in question belongs to the memory domain at all.

	if ((periodic != 0) && (x > 0.0)) // in this case test anticlock image of x, which had to be within domain tranche to begin with.
	{
		real newx = Anticlockwise.xx*x+Anticlockwise.xy*y;
		real newy = Anticlockwise.yx*x+Anticlockwise.yy*y;
		x = newx; y = newy;

		// Prioritize looking left.
		if ((cornerptr[2]->pos.x > 0.0) && (cornerptr[1]->pos.x > 0.0))
			if (TestAgainstEdge(x,y, 1, 0, ppNeigh)) return 1;
		if ((cornerptr[0]->pos.x > 0.0) && (cornerptr[2]->pos.x > 0.0))
			if (TestAgainstEdge(x,y, 0, 1, ppNeigh)) return 1;
		if ((cornerptr[1]->pos.x > 0.0) && (cornerptr[0]->pos.x > 0.0))
			if (TestAgainstEdge(x,y, 0, 2, ppNeigh)) return 1;
	
		// Second favourite: stay within periodic.
		if ((cornerptr[2]->pos.x > 0.0) || (cornerptr[1]->pos.x > 0.0))
			if (TestAgainstEdge(x,y, 1, 0, ppNeigh)) return 1;
		if ((cornerptr[0]->pos.x > 0.0) || (cornerptr[2]->pos.x > 0.0))
			if (TestAgainstEdge(x,y, 0, 1, ppNeigh)) return 1;
		if ((cornerptr[1]->pos.x > 0.0) || (cornerptr[0]->pos.x > 0.0))
			if (TestAgainstEdge(x,y, 0, 2, ppNeigh)) return 1;
	
		// Got here: we'll have to exit to the left side of domain then.
		if (TestAgainstEdge(x,y, 1, 0, ppNeigh)) return 1;
		if (TestAgainstEdge(x,y, 0, 1, ppNeigh)) return 1;
		if (TestAgainstEdge(x,y, 0, 2, ppNeigh)) return 1;
	}; 
	// Idea: We need to prioritize NOT CROSSING to the x < 0 side if the target x > 0.

	// New attempt:

	if ((neighbours[0]->u8domain_flag == OUTER_FRILL) || (neighbours[0]->u8domain_flag == INNER_FRILL)
		|| (neighbours[0] == this))
	{
		// In this case:
		// prioritize neighbours[1] and [2].

		if(TestAgainstEdge(x,y, 0, 1, ppNeigh)) return 1;
		if(TestAgainstEdge(x,y,	0,        // edge corner
								2,        // the opposite point - ie which edge
								ppNeigh)) // neighbour in this direction, if it's outside this way
			return 1;

		if(TestAgainstEdge(x,y, 1, 0, ppNeigh)) return 1;
		return 0; // inside *this
	}
	if ((neighbours[1]->u8domain_flag == OUTER_FRILL) || (neighbours[1]->u8domain_flag == INNER_FRILL)
		|| (neighbours[1] == this))
	{
		if (TestAgainstEdge(x,y, 0, 2, ppNeigh)) return 1;
		if (TestAgainstEdge(x,y, 1, 0, ppNeigh)) return 1;
		if (TestAgainstEdge(x,y, 0, 1, ppNeigh)) return 1;
		return 0;
	}
	// Just changed the order of tests.

	/*
	if (neighbours[0] == this) {
		
		if(TestAgainstEdge(x,y, 0, 1, ppNeigh)) return 1;
		if(TestAgainstEdge(x,y,	0,       // edge corner
								 2,       // the opposite point - ie which edge
									 ppNeigh))    // neighbour in this direction, if it's outside this way
			return 1;

		if(TestAgainstEdge(x,y, 1, 0, ppNeigh)) return 1;
		return 0; // inside *this
	}
	if (neighbours[1] == this) {
		if (TestAgainstEdge(x,y, 0, 2, ppNeigh)) return 1;
		if (TestAgainstEdge(x,y, 1, 0, ppNeigh)) return 1;
		if (TestAgainstEdge(x,y, 0, 1, ppNeigh)) return 1;
		return 0;
	}
	*/
	
	if ((periodic != 0) && (x > 0.0))
	
	if (TestAgainstEdge(x,y, 1, 0, ppNeigh)) return 1;
	if (TestAgainstEdge(x,y, 0, 1, ppNeigh)) return 1;
	if (TestAgainstEdge(x,y, 0, 2, ppNeigh)) return 1;
	
	return 0;
	


	// .
	// .

	// If point is within insulator, we should return the correct triangle.
	// But how that is handled, in the case of placement?


	//if (u8EdgeFlag > 0)
	//{
	//	int c1,c2;
	//	// Find not base corner:
	//	int iBase = 0; while (cornerptr[iBase]->flags > 3) iBase++;

	//	c1 = iBase+1; if (c1 == 3) c1 = 0;
	//	c2 = c1+1; if (c2 == 3) c2 = 0;

	//	if(TestAgainstEdge(x,y,c1,c2,       // the opposite point
	//								 ppNeigh ))   // neighbour in this direction, if it's outside this way
	//		return 1;

	//	if (TestAgainstEdge(x,y,c2,c1,       // the opposite point
	//								 ppNeigh ))   // neighbour in this direction, if it's outside this way
	//		return 1;

	//	// finally test against inner edge .... but maybe wish to return 0
	//	if (TestAgainstEdge(x,y,c1,iBase,       // the opposite point
	//								 ppNeigh ))   // neighbour in this direction, if it's outside this way
	//	{
	//		// ask if it is azimuthally compatible. If it is, we can return 0.
	//		Vector2 u[3];
	//		PopulatePositions(u[0],u[1],u[2]);
	//		if (periodic) {
	//			if (x > 0.0) { MapRight(u[0],u[1],u[2]); } else {MapLeft(u[0],u[1],u[2]); };
	//		};
	//		real grad1 = u[c1].x/u[c1].y; real grad2 = u[c2].x/u[c2].y;
	//		real grad = x/y;

	//		// realise we can afford to be quite liberal here
	//		// It failed to test as outside either of the other two sides
	//		// The only options: we are nowhere near, or this really is it.
	//		// If we are nowhere near , the following comparator will be quite large, so:
	//		if ((grad-grad1)*(grad-grad2) <= FP_FUZZY_THRESH_LARGE) return 0;
	//		return 1; 
	//		// outside azimuthally and on edge of domain therefore returning 1 with *ppNeigh as itself.
	//	}
	//	return 0;

	//} else {
	//	if(TestAgainstEdge(x,y,
	//								0,       // edge corner
	//								 2,       // the opposite point
	//								 ppNeigh    // neighbour in this direction, if it's outside this way
	//								 ))
	//		return 1;
	//	if(TestAgainstEdge(x,y, 1, 0, ppNeigh)) return 1;
	//	if(TestAgainstEdge(x,y, 0, 1, ppNeigh)) return 1;
	//	return 0;
	//};
}			

// same as above function basically
/*bool Triangle::TestAgainstEdges(float x,float y, Triangle ** ppNeigh)
{
	int out = 0;
	if(TestAgainstEdge(x,y,
								0,       // edge corner
								 2,       // the opposite point
								 ppNeigh    // neighbour in this direction, if it's outside this way
								 ))
	{
		out = 1;
	} else {
		if(TestAgainstEdge(x,y, 1, 0, ppNeigh))
		{
			out = 1;
		} else {
		if(TestAgainstEdge(x,y, 0, 1, ppNeigh)) // do not send vertex 2 here in case it does not exist.
			out = 1;
		};
	};
	return out;	
}			
*/

// Now here we implement routines to calculate triangle intersections:


#define DYDX   0
#define DXDY   1

void GetIntersection(Vector2 * result,const Vector2 & x0,real gradient,int flagdydx, Vector2 & a, Vector2 & b)
{
	real x,y;
	// where is line a->b cut by the line that starts at start and has gradient gradient,

	// DEBUG:
	if (!(_finite(a.x) && _finite(a.y) && _finite(b.x) && _finite(b.y)))
	{
		a.x = a.x;
	}

	if (flagdydx == DYDX)
	{
		// on first line, 
		// x = x0.x + t
		// y = x0.y + t dy/dx
		// x - y/ dy/dx = x0.x - x0.y / dy/dx

		// y = dy/dx(x - x0.x) + x0.y 

		// on second line
		// x = a.x + t(b.x-a.x)
		// y = a.y + t(b.y-a.y)

		// (b.x-a.x)y - x(b.y-a.y) = (b.x-a.x)a.y - (b.y-a.y)a.x
		
		// For both to be true simultaneously?

		// (b.x-a.x)(dy/dx(x - x0.x) + x0.y ) - x(b.y-a.y) = (b.x-a.x)a.y - (b.y-a.y)a.x

		// ((b.x - a.x)dy/dx - (b.y-a.y) ) x =  (b.x-a.x)a.y - (b.y-a.y)a.x + (b.x-a.x)dy/dx x0.x -  (b.x-a.x)x0.y
		
		// looks too complicated?

		x = ((b.x-a.x)*(a.y + gradient*x0.x - x0.y) - (b.y-a.y)*a.x)/
			((b.x - a.x)*gradient - (b.y-a.y));
		y = x0.y + gradient*(x - x0.x);

		// DEBUG:

		// Test that (x,y) is actually a solution:


	} else {

		// x = dx/dy (y- x0.y) + x0.x

		// (b.x-a.x)y - x(b.y-a.y) = (b.x-a.x)a.y - (b.y-a.y)a.x

		// (b.x-a.x)y - (dx/dy (y- x0.y) + x0.x)(b.y-a.y) = (b.x-a.x)a.y - (b.y-a.y)a.x

		// (b.x-a.x - dx/dy (b.y-a.y)) y = (b.x-a.x)a.y - (b.y-a.y)a.x + (x0.x - dx/dy x0.y) (b.y-a.y)

		y = ((b.x-a.x)*a.y + (x0.x - gradient*x0.y - a.x)*(b.y-a.y))/
			(b.x-a.x - gradient * (b.y-a.y));
		x = gradient*(y-x0.y) + x0.x;

	};
	result->x = x;
	result->y = y;
}

void Triangle::CalculateCircumcenter(Vector2 & cc, real * pdistsq)
{
	Vector2 Bb,C,b,c,a;
	
	MapLeftIfNecessary(a,b,c);
	
	Bb = b-a;
	C = c-a;		
	real D = 2.0*(Bb.x*C.y-Bb.y*C.x);
	real modB = Bb.x*Bb.x+Bb.y*Bb.y;
	real modC = C.x*C.x+C.y*C.y;
	if (D == 0.0) {
		printf("\aCalculateCircumcenter a %1.12E %1.12E b %1.12E %1.12E c %1.12E %1.12E D=0\n",
			a.x, a.y, b.x, b.y, c.x, c.y);
		getch(); getch();
	}
	cc.x = (C.y*modB-Bb.y*modC)/D + a.x;
	cc.y = (Bb.x*modC-C.x*modB)/D + a.y;
	*pdistsq = (a.x-cc.x)*(a.x-cc.x)+(a.y-cc.y)*(a.y-cc.y); // why?
}


void GetInsulatorIntercept(Vector2 *result, const Vector2 & x1, const Vector2 & x2)
{
	// find where line x1->x2 crosses r = DEVICE_RADIUS_INSULATOR_OUTER

	// x = x1.x + t(x2.x-x1.x)
	// y = x1.y + t(x2.y-x1.y)
	// x^2+y^2 = c^2

	// (x1.x + t(x2.x-x1.x))^2 + (x1.y + t(x2.y-x1.y))^2 = c^2

	// or, y = x1.y + dy/dx (x-x1.x)

	// x^2 + (x1.y - dy/dx x1.x + dy/dx x)^2 = c^2


	// (x1.x + t(x2.x-x1.x))^2 + (x1.y + t(x2.y-x1.y))^2 = c^2

	// t^2 ( (x2.x-x1.x)^2 + (x2.y - x1.y)^2 ) + 2t (x1.x (x2.x-x1.x) + x1.y (x2.y-x1.y) )
	//    + x1.x^2 + x1.y^2 = c^2
	// t^2 + 2t ( -- ) / (-- ) = (c^2 - x1.x^2 - x1.y^2)/ (-- )
	
	real den = (x2.x-x1.x)*(x2.x-x1.x) + (x2.y - x1.y)*(x2.y - x1.y) ;
	real a = (x1.x * (x2.x-x1.x) + x1.y * (x2.y-x1.y) ) / den;

	// (t + a)^2 - a^2 = (  c^2 - x1.x^2 - x1.y^2  )/den
	
	real root = sqrt( (DEVICE_RADIUS_INSULATOR_OUTER*DEVICE_RADIUS_INSULATOR_OUTER
							- x1.x*x1.x - x1.y*x1.y)/den + a*a ) ;
	
	real t1 = root - a;
	real t2 = -root - a;
	
	// since this is a sufficient condition to satisfy the circle, this probably means that
	// the other solution is on the other side of the circle.
	
	// Which root is within x1, x2 ? Remember x2 would be t = 1.

	if (t1 > 1.0) 
	{
		if ((t2 < 0.0) || (t2 > 1.0))
		{	
			// This usually means one of the points actually is on the curve.

			real dist1 = min(fabs(t1-1.0),fabs(t1));
			real dist2 = min(fabs(t2-1.0),fabs(t2));

			if (dist1 < dist2)
			{
				// use t1				
				if (dist1 > 0.00000001)
				{
					printf("\n\nError.\n"); 
					getch();
				};				
				result->x = x1.x + t1*(x2.x-x1.x);
				result->y = x1.y + t1*(x2.y-x1.y);
			} else {
				// use t2				
				if (dist2 > 0.00000001)
				{
					printf("\n\nError.\n"); 
					printf("t1 = %1.10E , \nt2 = %1.10E , \nx1.x= %1.10E ,\nx1.y = %1.10E ,\nx2.x = %1.10E ,\nx2.y = %1.10E\n",
								t1,t2,x1.x,x1.y,x2.x,x2.y);
					getch();
				};				
				result->x = x1.x + t2*(x2.x-x1.x);
				result->y = x1.y + t2*(x2.y-x1.y);
			};
		} else {		
			// use t2:		
			result->x = x1.x + t2*(x2.x-x1.x);
			result->y = x1.y + t2*(x2.y-x1.y);
		};
	} else {
		if (t1 < -1.0e-13) 
		{	
			printf("\n\nError.KL\n"); 
			printf("t1 = %1.10E , \nt2 = %1.10E , \nx1.x= %1.10E ,\nx1.y = %1.10E ,\nx2.x = %1.10E ,\nx2.y = %1.10E\n",
				t1,t2,x1.x,x1.y,x2.x,x2.y);
			getch(); 
		};
		result->x = x1.x + t1*(x2.x-x1.x);
		result->y = x1.y + t1*(x2.y-x1.y);		
	};
#ifdef DEBUG
	if (result->x*result->x + result->y*result->y > 1.000001*DEVICE_RADIUS_INSULATOR_OUTER*DEVICE_RADIUS_INSULATOR_OUTER)
	{
		result = result;
	};
	if (result->y < 0.0)
	{
		result = result;
	};
#endif
}
//f64_vec2 GetInsulatorIntercept(f64_vec2 pos0, f64_vec2 pos1)
//{
//	f64_vec2 result;
//	GetInsulatorIntercept(&result,pos0,pos1);
//	return result;
//}
// Doesn't work, don't know why not.

void Get_ROC_InsulatorIntercept(Vector2 * pROCintercept1,
								Vector2 lower , Vector2 moving,Vector2 ROC)
{
	// A rough estimate might do.	
	// Moving away directly from lower does not change the intercept;
	// moving perpendicularly does.

	// Do empirically:
	Vector2 interceptplus, interceptminus;

	real length = (moving-lower).modulus();
	real ROClength = ROC.modulus();
	real ROCfactor = length*0.0001/ROClength;
	Vector2 ROCmove = ROC*ROCfactor;
	Vector2 plus = moving + ROCmove;
	Vector2 minus = moving - ROCmove;

	GetInsulatorIntercept(&interceptplus, lower, plus);
	GetInsulatorIntercept(&interceptminus, lower, minus);

	Vector2 derivative = (interceptplus-interceptminus)/(2.0*ROCfactor);
	*pROCintercept1 = derivative;
}

int Triangle::GetCentreOfIntersectionWithInsulator(Vector2 & cc)
{

	// where this triangle crosses r=3.44,
	// we want to return the middle of that arc.

	// 3 lines; should give 2 intercepts of 3.44
	// If not, failed.

	real azimuth01,azimuth12,azimuth02;
	real r0sq, r1sq, r2sq, Rsq;
	int number_of_intercepts;
	real angle;
	Vector2 u[3];
	Vector2 intercept;

	MapLeftIfNecessary(u[0],u[1],u[2]);

	r0sq = u[0].dot(u[0]);
	r1sq = u[1].dot(u[1]);
	r2sq = u[2].dot(u[2]);
	Rsq = DEVICE_RADIUS_INSULATOR_OUTER*DEVICE_RADIUS_INSULATOR_OUTER;

	number_of_intercepts = 0;
	azimuth01 = 0.0;
	azimuth12 = 0.0;
	azimuth02 = 0.0;

	// 0-1:

	if ((r0sq-Rsq)*(r1sq-Rsq) < 0.0)
	{
		number_of_intercepts++;

		GetInsulatorIntercept(&intercept,u[0],u[1]);
		azimuth01 = atan2(intercept.y,intercept.x);
	};
	// 1-2:

	if ((r2sq-Rsq)*(r1sq-Rsq) < 0.0)
	{
		number_of_intercepts++;

		GetInsulatorIntercept(&intercept,u[1],u[2]);
		azimuth12 = atan2(intercept.y,intercept.x);
	};
	// 0-2:

	if ((r2sq-Rsq)*(r0sq-Rsq) < 0.0)
	{
		number_of_intercepts++;

		GetInsulatorIntercept(&intercept,u[0],u[2]);
		azimuth02 = atan2(intercept.y,intercept.x);
	};

	if (number_of_intercepts != 2) 
	{
		printf("intercept fail\n"); getch();

		return 1;
	};

	angle = 0.5*(azimuth01+azimuth12+azimuth02);
	//if (angle < -HALFANGLE+PI*0.5) angle += FULLANGLE;
	//if (angle > HALFANGLE+PI*0.5) angle -= FULLANGLE; // not wanted
	
	// If this is periodic triangle then allow that angle is contiguous with tri, not part of canonical tranche

	cc.x = cos(angle)*DEVICE_RADIUS_INSULATOR_OUTER;
	cc.y = sin(angle)*DEVICE_RADIUS_INSULATOR_OUTER;

	// defend against errors:
	// Is cc within triangle?
	
	// This is a fairly unnecessary way of doing it.
	// Here's a better one:
	// take linear average then project. That requires sqrt not atan.

	return 0;
}

/*void AuxTriangle::CalculateCircumcenter(Vector2 & cc, real * pdistsq)
{
	Vector2 Bb,C,b,c,a;
	Vector2 basea,baseb;
	
	if (periodic > 0)
	{
		// map everything to left hand side.
		MapLeft(a,b,c);
	} else {
		PopulatePositions(a,b,c);
	};
	Bb = b-a;
	C = c-a;		
	real D = 2.0*(Bb.x*C.y-Bb.y*C.x);
	real modB = Bb.x*Bb.x+Bb.y*Bb.y;
	real modC = C.x*C.x+C.y*C.y;
	cc.x = (C.y*modB-Bb.y*modC)/D + a.x;
	cc.y = (Bb.x*modC-C.x*modB)/D + a.y;

	if (pdistsq != 0)
		*pdistsq = (a.x-cc.x)*(a.x-cc.x)+(a.y-cc.y)*(a.y-cc.y); 
}





*/

	void ConvexPolygon::SetTri(const Vector2 & x1,const Vector2 & x2, const Vector2 & x3)
	{
		numCoords = 3;
		coord[0] = x1;
		coord[1] = x2;
		coord[2] = x3;
	}
	ConvexPolygon::ConvexPolygon(const Vector2 & x1,const Vector2 & x2,const Vector2 & x3)
	{
		SetTri(x1, x2, x3);
	}

	ConvexPolygon::ConvexPolygon()
	{
		numCoords = 0;
	}

	void ConvexPolygon::Get_Bxy_From_Az(real Az_array[], real * pBx,real * pBy)
	{
		// Assume we have coords that are sorted anticlockwise

		int i, inext;
		real Bx = 0, By = 0;
		real Integral_x, Integral_y;
		for (i = 0; i < numCoords; i++)
		{
			inext = i+1; if (inext == numCoords) inext = 0;
			Integral_x = (Az_array[i] + Az_array[inext])
							*(coord[inext].x-coord[i].x);
			Integral_y = (Az_array[i] + Az_array[inext])
							*(coord[inext].y-coord[i].y);
			Bx += Integral_x;
			By += Integral_y;
		}
		
		real area = this->GetArea();
		*pBx = 0.5*Bx/area;
		*pBy = 0.5*By/area;
	}

	Vector3 ConvexPolygon::Get_curl2D_from_anticlockwise_array(Vector3 A[])
	{
		// Assuming we have coords that are sorted anticlockwise

		int i, inext;
		Vector3 B;
		memset(&B,0,sizeof(Vector3));
		
		real Integral_x, Integral_y, Integral_z;
		for (i = 0; i < numCoords; i++)
		{
			inext = i+1; if (inext == numCoords) inext = 0;
			Integral_x = (A[i].z + A[inext].z)
							*(coord[inext].x-coord[i].x); // [anticlockwise]--> -dAz/dy
			Integral_y = (A[i].z + A[inext].z)
							*(coord[inext].y-coord[i].y); // [anticlockwise]--> dAz/dx

			Integral_z = (A[i].y+A[inext].y)
							*(coord[inext].y-coord[i].y) // [anticlockwise] --> dAy/dx 
						+ (A[i].x+A[inext].x)
						    *(coord[inext].x-coord[i].x);  // --> -dAx/dy
						 			
			B.x += Integral_x;
			B.y += Integral_y;
			B.z += Integral_z;
		}
		
		real area = this->GetArea();
		B *= 0.5/area;
		B.z += BZ_CONSTANT;
		
		return B;
	}
	Vector2 ConvexPolygon::Get_Integral_grad_from_anticlockwise_array(real Te[])
	{
		Vector2 grad;
		int i, inext;
		memset(&grad,0,sizeof(Vector2));

		real Integral_x, Integral_y;
		for (i = 0; i < numCoords; i++)
		{
			inext = i+1; if (inext == numCoords) inext = 0;
			Integral_x = 0.5*(Te[i] + Te[inext])
							*(coord[inext].y-coord[i].y); // [anticlockwise]--> dTe/dx
			Integral_y = 0.5*(Te[i] + Te[inext])
							*(coord[i].x-coord[inext].x); // [anticlockwise]--> dTe/dy
			grad.x += Integral_x;
			grad.y += Integral_y;
		}
		// ---> Compare to GradTe formula on tris.

		return grad;
	}
	Vector2 ConvexPolygon::Get_grad_from_anticlockwise_array(real Te[])
	{
		Vector2 grad;
		grad = Get_Integral_grad_from_anticlockwise_array(Te);
		real area = this->GetArea();
		grad /= area;
		return grad;
	}
	Vector2 ConvexPolygon::CalculateBarycenter()
	{
		// Assume we have coords that are sorted anticlockwise or clockwise
		Vector2 u;
		int i, inext;
		real Integral_x, Integral_y, shoelace;
		Integral_x = 0.0;
		Integral_y = 0.0;
		shoelace = 0.0;
		real lace;
		for (i = 0; i < numCoords; i++)
		{
			inext = i+1; if (inext == numCoords) inext = 0;

			lace = (coord[i].x * coord[inext].y - coord[inext].x*coord[i].y);
			Integral_x += (coord[i].x + coord[inext].x)*lace;
			Integral_y += (coord[i].y + coord[inext].y)*lace;
			shoelace += lace;
		};

		u.x = THIRD*Integral_x/shoelace;
		u.y = THIRD*Integral_y/shoelace;
		return u;
	}



			// Now, we will make sure that we enter the points in anticlockwise
			// order. 

			// Make polygon method: return_Bxy_from_Az( set of anticlockwise values)
		



	// Note: cases that apply:
	// when we introduce the first corner, it may cut the LH and bottom sides of the "house".
	// It cannot cut off the bottom entirely, I don't think. 
	
	// The second one may cut the RH edge and either the bottom or the LH diagonal.

	// ^^^ 1. Get some confidence that this is true.

	// %%% then we can proceed to say? First look at LH then at RH. We want to end up with 
	// 1. Edge length in each of 5 directions: maintain this as we go?
	// 2. Voronoi ConvexPolygon so that we can take intersections with wedges.
	


	
/*
	bool VoronoiPolygon::ClipAgainstHalfplane_Update_SideIndexList(const Vector2 & r1, const Vector2 & r2, const Vector2 & r3, int flag_new)
	{
		// Similar to the standard function except here, we will maintain for each vertex a number that indicates, for the side to anticlockwise
		// what flag that side corresponded to.
		// When we clip a vertex, the new vertex on the right takes up the rightmost clipped vertex's index
		// The new vertex on the left takes up the flag that was passed.
		
		// We can then look back (in the caller) and see which sides still exist and how long they are.
		// Less clear what we can do about overlaps with other wedges ... maybe ignore carefully.

		// first le's cut n paste:


		bool intersect;
		bool above_is_inside;
		real compare;
		int first, last;
		bool setfirst;
		real gradient ;
		int flag, pullback, i, post_last, pre_first;
		Vector2 cross1, cross2;
		
		static const real EPS = 5.0e-14;

		// Now we have to be able to deal with degenerate cases.
		// =====================================

		// We assign each point a status: INSIDE the clip region, NEAR the clip boundary, or OUTSIDE the clip region.

		// When some consecutive points are found to be OUTSIDE the clip region, we also
		// remove any consecutive points that are NEAR the clip boundary.
		// This ensures hopefully that intersections are only taken towards points that are away from the clip boundary.

		// If no points figure as actually OUTSIDE (further outside than EPS) then clipping is skipped.

		// If no points figure as actually INSIDE (further inside than EPS) then we return no intersection.

#define INSIDE     0
#define OUTSIDE    1
#define NEARBY       2
		// There is no point distinguishing near inside and near outside as this will be unreliable due to rounding anyway.
		
		Vector2 direction = r2-r1;
		// r1 + alpha.direction is the line
		
		if (direction.x*direction.x > direction.y*direction.y) 
			// defend against case that it's basically vertical
		{
			flag = DYDX;
			// determine whether r3 is above line:
		
			gradient = direction.y/direction.x;
			above_is_inside = (r3.y > r1.y + (r3.x-r1.x)*gradient) ;
			// Hope we didn't get passed r3 that is on the line r1-r2 . If so it's an unfair call and we have to change the caller.
			
			// Now ask which of our existing coordinates is in/near/out:			
			intersect = false;
			setfirst = false;
			for (int i = 0; i < numCoords; i++)
			{
				compare = r1.y + (coord[i].x-r1.x)*gradient;
				
				//if (above)
				//{
				//	is_above[i] = (coord[i].y > r1.y + (coord[i].x-r1.x)*gradient - EPS);
				//} else {
				//	is_above[i] = (coord[i].y > r1.y + (coord[i].x-r1.x)*gradient + EPS);
				//};
				
				if (above_is_inside)
				{
					if (coord[i].y > compare + EPS) {
						status[i] = INSIDE;
						intersect = true;
					} else {
						if (coord[i].y < compare - EPS) {
							status[i] = OUTSIDE;
							first = i;
							setfirst = true;
						} else {
							status[i] = NEARBY;
						};
					};
				} else {
					if (coord[i].y > compare + EPS) {
						status[i] = OUTSIDE;
						first = i;
						setfirst = true;
					} else {
						if (coord[i].y < compare - EPS) {
							status[i] = INSIDE;
							intersect = true;
						} else {
							status[i] = NEARBY;
						};
					};
				};
			//		last = i; // ordinarily this means we stored the first and last scrappable vertex
					// but beware that it won't work if the scrapped section crossed 0
			};
			
		} else {
			// line was more vertical than horizontal so take gradient x per y

			flag = DXDY;

			gradient = direction.x/direction.y;
			above_is_inside = (r3.x > r1.x + (r3.y-r1.y)*gradient); // true if r3 is to right

			// Now ask which of our existing coordinates is above
			
			intersect = false;
			setfirst = false;
			for (int i = 0; i < numCoords; i++)
			{
				compare = r1.x + (coord[i].y-r1.y)*gradient;
				
				if (above_is_inside)
				{
					if (coord[i].x > compare + EPS) {
						status[i] = INSIDE;
						intersect = true;
					} else {
						if (coord[i].x < compare - EPS) {
							status[i] = OUTSIDE;
							first = i;
							setfirst = true;
						} else {
							status[i] = NEARBY;
						};
					};
				} else {
					if (coord[i].x > compare + EPS) {
						status[i] = OUTSIDE;
						first = i;
						setfirst = true;
					} else {
						if (coord[i].x < compare - EPS) {
							status[i] = INSIDE;
							intersect = true;
						} else {
							status[i] = NEARBY;
						};
					};
				};
			};

		};

		if (intersect == false)
		{
			printf("error - Voronoi cell eclipsed.\n");
			// no intersection of halfplane and existing polygon
			return false;		
		};
		if (setfirst == false) 
		{
			// no clipping applies
			return true;
		};

		// If we get here, some polygon vertices were (properly) clipped and some were (properly) not.

		// OK now scrap those that did not intersect, if any, and replace them with the points where the lines are intersected

		// Let's get the first point before our OUTSIDE subset that is not INSIDE.

		while (status[first] != INSIDE)
		{
			first--;
			if (first < 0) first = numCoords-1;
		};
		pre_first = first;
		first++;
		if (first == numCoords) first = 0; // the first clipped vertex
		
		GetIntersection(&cross1,r1,gradient,flag,coord[pre_first],coord[first]); 
		
		// Now move forward to last scrappable.

		last = first;
		while (status[last] != INSIDE)
		{
			last++;
			if (last == numCoords) last = 0;
		};
		post_last = last;
		last--;
		if (last < 0) last = numCoords-1; // the last clipped vertex
		
		GetIntersection(&cross2,r1,gradient,flag,coord[last],coord[post_last]); 

		// now repopulate the array:
		// cases:
		if (last >= first)
		{
			// easy cases:
			if (last == first)
			{
				// exactly one vertex is clipped.
				
				// debug check: Never scrap a circumcenter:
				if (last <= max_index_no_scrap) {
					printf("circumcenter scrapped - not cool\n");
					last = last;
				};


				//	*	// When we clip a vertex, the new vertex on the right takes up the rightmost clipped vertex's index
				//	*	// The new vertex on the left takes up the flag that was passed.

				// array gets longer - need to move elements outwards first
				for (i = numCoords-1; i >= last+1; i--)
				{
					coord[i+1] = coord[i];
					edge_flag[i+1] = edge_flag[i];
				}
				numCoords++;
				coord[first] = cross1;
				coord[first+1] = cross2; 			
				// The anticlockwise added point now indexes the rest of the existing anticlock side:
				edge_flag[first+1] = edge_flag[first];
				// The clockwise added point indexes our new side:
				edge_flag[first] = flag_new;

			} else {
				
				if (first <= max_index_no_scrap) {
					printf("circumcenter scrapped - not cool - first %d max_index_no_scrap %d \n",first,max_index_no_scrap);
					first = first;
					getch();
				};


				edge_flag[first+1] = edge_flag[last];
				edge_flag[first] = flag_new;

				// last > first so we may need to pull some elements backwards
				pullback = last-first-1; // last == first +1 => pullback == 0
				for (i = last+1; i < numCoords; i++)
				{
					coord[i-pullback] = coord[i];
					edge_flag[i-pullback] = edge_flag[i];
				};
				numCoords -= pullback;
				coord[first] = cross1;
				coord[first+1] = cross2;
			};
		} else {
			// scrappable subset crosses 0
			// post_last is the first element that is INSIDE
				
			// debug check: Never scrap a circumcenter:
			if (this->max_index_no_scrap >= 0)
			{
				printf("circumcenter scrapped - not cool - first %d last %d post_last %d \n", first, last, post_last);

				numCoords = numCoords;
				getch();
			};
			
			// Move elements back to 0; if post_last = 4, first = 6 then there are 2 such elements + 2 new ones
			
			i = first-post_last;
			edge_flag[i+1] = edge_flag[last]; // last should be last scrapped vertex, > 0
			for (i =0; i < first-post_last; i++)
			{
				coord[i] = coord[i+post_last];
				edge_flag[i] = edge_flag[i+post_last];
			};
			coord[i] = cross1;
			coord[i+1] = cross2;
			edge_flag[i] = flag_new;
			numCoords = i+2;
		};

		return true;
	}
*/

	real ConvexPolygon::GetSucceedingSideLength(int side)
	{
		int next = side+1;
		if (next == numCoords) next = 0;
		return sqrt(
			(coord[next].x-coord[side].x)*(coord[next].x-coord[side].x)+
			(coord[next].y-coord[side].y)*(coord[next].y-coord[side].y));
	}

	real ConvexPolygon::GetPrecedingSideLength(int side)
	{
		int prev = side-1;
		if (prev == -1) prev = numCoords-1;
		return sqrt(
			(coord[prev].x-coord[side].x)*(coord[prev].x-coord[side].x)+
			(coord[prev].y-coord[side].y)*(coord[prev].y-coord[side].y));
	}

	bool ConvexPolygon::IsConvex()
	{
		int inext, inext2, inext3;
		f64_vec2 v1, v2, direction;
		for (int i = 0; i < numCoords; i++)
		{
			inext = i + 1; if (inext == numCoords) inext = 0;
			inext2 = inext + 1; if (inext2 == numCoords) inext2 = 0;
			inext3 = inext2 + 1; if (inext3 == numCoords) inext3 = 0;
			v1 = coord[i];
			v2 = coord[inext2];
			direction.x = v2.y - v1.y;
			direction.y = v1.x - v2.y; // perpendicular
			if ((coord[inext] - v1).dot(direction)*((coord[inext3] - v1).dot(direction)) > 0.0) {
				// Same side, not convex
				return false;
			};
		};
		return true;
	}

	int ConvexPolygon::ClipAgainstHalfplane(const Vector2 & r1, const Vector2 & r2, const Vector2 & r3)
	{
		// 'The reason this way is not succeeding: basically we can create two equal points due to 
		// clipping a vertex that is on the boundary and replacing with 2'

		bool intersect;
		bool above_is_inside;
		real compare;
		int first, last;
		bool setfirst;
		real gradient ;
		int flag, pullback, i, post_last, pre_first;
		Vector2 cross1, cross2;
		
		static const real EPS = 5.0e-14;

		// Now we have to be able to deal with degenerate cases.
		// =====================================

		// We assign each point a status: INSIDE the clip region, NEAR the clip boundary, or OUTSIDE the clip region.

		// When some consecutive points are found to be OUTSIDE the clip region, we also
		// remove any consecutive points that are NEAR the clip boundary.
		// This ensures hopefully that intersections are only taken towards points that are away from the clip boundary.

		// If no points figure as actually OUTSIDE (further outside than EPS) then clipping is skipped.
		// If no points figure as actually INSIDE (further inside than EPS) then we return no intersection.

#define INSIDE     0
#define OUTSIDE    1
#define NEARBY       2
		// There is no point distinguishing near inside and near outside as this will be unreliable due to rounding anyway.
		
		Vector2 direction = r2-r1;
		// r1 + alpha.direction is the line
		
		if (direction.x*direction.x > direction.y*direction.y) 
			// defend against case that it's basically vertical
		{
			flag = DYDX;
			// determine whether r3 is above line:
		
			gradient = direction.y/direction.x;
			above_is_inside = (r3.y > r1.y + (r3.x-r1.x)*gradient) ;
			// Hope we didn't get passed r3 that is on the line r1-r2 . If so it's an unfair call and we have to change the caller.
			
			// Now ask which of our existing coordinates is in/near/out:			
			intersect = false;
			setfirst = false;
			for (int i = 0; i < numCoords; i++)
			{
				compare = r1.y + (coord[i].x-r1.x)*gradient;
				
				//if (above)
				//{
				//	is_above[i] = (coord[i].y > r1.y + (coord[i].x-r1.x)*gradient - EPS);
				//} else {
				//	is_above[i] = (coord[i].y > r1.y + (coord[i].x-r1.x)*gradient + EPS);
				//};
				
				if (above_is_inside)
				{
					if (coord[i].y > compare + EPS) {
						status[i] = INSIDE;
						intersect = true;
					} else {
						if (coord[i].y < compare - EPS) {
							status[i] = OUTSIDE;
							first = i;
							setfirst = true;
						} else {
							status[i] = NEARBY;
						};
					};
				} else {
					if (coord[i].y > compare + EPS) {
						status[i] = OUTSIDE;
						first = i;
						setfirst = true;
					} else {
						if (coord[i].y < compare - EPS) {
							status[i] = INSIDE;
							intersect = true;
						} else {
							status[i] = NEARBY;
						};
					};
				};
			//		last = i; // ordinarily this means we stored the first and last scrappable vertex
					// but beware that it won't work if the scrapped section crossed 0
			};
			
		} else {
			// line was more vertical than horizontal so take gradient x per y

			flag = DXDY;

			gradient = direction.x/direction.y;
			above_is_inside = (r3.x > r1.x + (r3.y-r1.y)*gradient); // true if r3 is to right

			// Now ask which of our existing coordinates is above
			
			intersect = false;
			setfirst = false;
			for (int i = 0; i < numCoords; i++)
			{
				compare = r1.x + (coord[i].y-r1.y)*gradient;
				
				if (above_is_inside)
				{
					if (bDebugcp) printf("qwe{} coord[i].x %1.14E compare %1.14E compare+EPS %1.14E dir %1.14E %1.14E gradient %1.14E y %1.14E %1.14E\n",
						coord[i].x, compare, compare + EPS,
						direction.x, direction.y, gradient, coord[i].y, r1.y);
					if (coord[i].x > compare + EPS) {
						status[i] = INSIDE;
						intersect = true;
						if (bDebugcp) printf(".");
					} else {
						if (coord[i].x < compare - EPS) {
							status[i] = OUTSIDE;
							first = i;
							setfirst = true;
							if (bDebugcp) printf("~");
						} else {
							status[i] = NEARBY;
							if (bDebugcp) printf("+");
						};
					};
				} else {
					if (coord[i].x > compare + EPS) {
						status[i] = OUTSIDE;
						first = i;
						setfirst = true;
					} else {
						if (coord[i].x < compare - EPS) {
							status[i] = INSIDE;
							intersect = true;
						} else {
							status[i] = NEARBY;
						};
					};
				};
			};
		};

		if (intersect == false) return 0;		// nothing INSIDE
		if (setfirst == false) return 1;			// no change to ConvexPolygon object

		// If all inside then we do not printf status list.


		// If we get here, some polygon vertices were (properly) clipped and some were (properly) not.
		// OK now scrap those that did not intersect, if any, and replace them with the points where the lines are intersected

		// Let's get the first point before our OUTSIDE subset that is not INSIDE.

		if (bDebugcp) {

			printf("Status list : ");
			for (int sd = 0; sd < numCoords; sd++)
				printf("%d", status[sd]);
			printf("\n");
		}

		while (status[first] != INSIDE)
		{
			first--;
			if (first < 0) first = numCoords-1;
		};
		pre_first = first;
		first++;
		if (first == numCoords) first = 0;
		
		GetIntersection(&cross1,r1,gradient,flag,coord[pre_first],coord[first]); 

		// Now move forward to last scrappable.

		last = first;
		while (status[last] != INSIDE)
		{
			last++;
			if (last == numCoords) last = 0;
		};
		post_last = last;
		last--;
		if (last < 0) last = numCoords-1;
		int store_post_last = post_last;

		// post_last is one that should say 'inside' and so did pre_first.

		// Are there more 'outside' between post_last and pre_first?

		// Check for two-part intersection with half-plane, and if so return error:
		while ((status[post_last] != OUTSIDE) && (post_last != pre_first)) {
			post_last++; if (post_last == numCoords) post_last = 0;
		};
		if (post_last != pre_first) return 2;
		post_last = store_post_last;
		// Note that in the case of double intersection, no clipping was performed.


		if (!_finite(coord[last].y)) {
			printf("bad ness. %1.8E %1.8E %1.8E  %1.8E %1.8E %1.8E \n",r1.x,r1.y,r2.x,r2.y,r3.x,r3.y);
			getch();			
		};
		
		GetIntersection(&cross2,r1,gradient,flag,coord[last],coord[post_last]); // this was passed a.y == #INF

		if (bDebugcp) 
			printf("first: %d last: %d post_last %d\n", first, last, post_last);

		// now repopulate the array:
		// cases:
		if (last >= first)
		{
			// easy cases:
			if (last == first)
			{
				// array gets longer - need to move elements outwards first
				for (i = numCoords-1; i >= last+1; i--)
					coord[i+1] = coord[i];
				numCoords++;
				coord[first] = cross1;
				coord[first+1] = cross2; 			
			} else {
				// last > first so we may need to pull some elements backwards
				pullback = last-first-1; // last == first +1 => pullback == 0
				for (i = last+1; i < numCoords; i++)
					coord[i-pullback] = coord[i];
				numCoords -= pullback;
				coord[first] = cross1;
				coord[first+1] = cross2;
			};
		} else {
			// scrappable subset crosses 0
			// post_last is the first element that is INSIDE
			
			// Move elements back to 0; if post_last = 4, first = 6 then there are 2 such elements + 2 new ones
			
				
			for (i = 0; i < first - post_last; i++) {
				coord[i] = coord[i + post_last];
				if (bDebugcp) printf("XXXX i %d coord %1.10E %1.10E\n", i, coord[i].x, coord[i].y);
			};
			coord[i] = cross1;
			if (bDebugcp) printf("i %d = cross1 %1.10E %1.10E\n", i, cross1.x, cross1.y);
			coord[i+1] = cross2;
			if (bDebugcp) printf("i+1 %d = cross2 %1.10E %1.10E\n", i+1, cross2.x, cross2.y);
			numCoords = i+2;
		};
		
		if (bDebugcp) {
			printf("Clipped :\n");
			for (i = 0; i < numCoords; i++)
				printf("%1.10E %1.10E -- ", coord[i].x, coord[i].y);
			printf("\n-------------------\n");
		};

		return 1;
		

		//// we know it goes from first to last
		//// except in the case that first == 0 for which we have to go again

		//if (first > 0)
		//{
		//	// get the coordinates where it crosses
		//	
		//	GetIntersection(&cross1,r1,gradient,flag,coord[first-1],coord[first]); 
		//	if (last < numCoords-1)
		//	{
		//		GetIntersection(&cross2,r1,gradient,flag,coord[last],coord[last+1]);
		//	} else {
		//		GetIntersection(&cross2,r1,gradient,flag,coord[last],coord[0]);
		//	};
		//	if (last == first)
		//	{
		//		// in this case we need to bop points forward, there are more coords in total
		//		for (i = numCoords-1; i >= last+1; i--)
		//			coord[i+1] = coord[i];
		//		numCoords++;
		//		coord[first] = cross1;
		//		coord[first+1] = cross2; 					
		//	} else {
		//		// in this case we may need to pull points backward in the array
		//		pullback = last-first-1; // last-first == 1 => 0
		//		if (pullback > 0)
		//			for (i = last+1; i < numCoords; i++)
		//				coord[i-pullback] = coord[i];
		//		coord[first] = cross1;
		//		coord[first+1] = cross2;
		//		numCoords -= pullback;
		//	};
		//	
		//} else {
		//	// have to handle this special case that point 0 was not in: seek again to find the ends of the interval of scrappable vertices
		//	// work backwards from the end
		//	first = numCoords;
		//	while (is_above[first-1] != above) first--;
		//	if (first == numCoords) first = 0;
		//	last = 0;
		//	while (is_above[last+1] != above) last++;
		//	if (first == 0)
		//	{
		//		GetIntersection(&cross1,r1,gradient,flag,coord[numCoords-1],coord[0]); 
		//		GetIntersection(&cross2,r1,gradient,flag,coord[last],coord[last+1]); 
		//		
		//		// Now do exactly as above.
		//		// COPY-PASTE:
		//		if (last == first)
		//		{
		//			// in this case we need to bop points forward, there are more coords in total
		//			for (i = numCoords-1; i >= last+1; i--)
		//				coord[i+1] = coord[i];
		//			numCoords++;
		//			coord[first] = cross1;
		//			coord[first+1] = cross2; 					
		//		} else {
		//			// in this case we may need to pull points backward in the array
		//			pullback = last-first-1; // last-first = 1 -> 0

		//			if (pullback > 0)
		//				for (i = last+1; i < numCoords; i++)
		//					coord[i-pullback] = coord[i];
		//			coord[first] = cross1;
		//			coord[first+1] = cross2;
		//			numCoords -= pullback;
		//		};
		//	} else {
		//		
		//		// all those from first onwards are considered destroyed...
		//		GetIntersection(&cross1,r1,gradient,flag,coord[first-1],coord[first]); 
		//		GetIntersection(&cross2,r1,gradient,flag,coord[last],coord[last+1]); 
		//		
		//		coord[0] = cross2; 
		//		// remove up until last
		//		pullback = last; // if last == 0 then we don't need to move anything
		//		// if last == 1 then we move element 2 to element 1
		//		for (i = last+1; i < numCoords; i++)
		//			coord[i-pullback] = coord[i];
		//		coord[first-pullback] = cross1;
		//		numCoords = first-pullback+1;
		//	};
		//};
		//
		//return true;

	}


	void ConvexPolygon::CopyFrom(ConvexPolygon & cp)
	{
		numCoords = cp.numCoords;
		for (int i = 0; i < numCoords; i++)
			coord[i] = cp.coord[i];
	}

	void ConvexPolygon::GetCentre(Vector2 & centre)
	{
		centre.x = 0.0;
		centre.y = 0.0;
		for (int i = 0; i < numCoords; i++)
		{
			centre.x += coord[i].x;
			centre.y += coord[i].y;
		}
		centre.x /= (real)numCoords;
		centre.y /= (real)numCoords;
	}

	real ConvexPolygon::FindTriangleIntersectionArea(Vector2 & r1, Vector2 & r2, Vector2 & r3)
	{
		ConvexPolygon cp;
		cp.CopyFrom(*this);

		if (!cp.ClipAgainstHalfplane(r1,r2,r3)) return 0.0;
		if (!cp.ClipAgainstHalfplane(r1,r3,r2)) return 0.0;
		if (!cp.ClipAgainstHalfplane(r2,r3,r1)) return 0.0;
		return cp.GetArea();
	}

	bool ConvexPolygon::GetIntersectionWithTriangle(ConvexPolygon * pPoly,Vector2 & r1, Vector2 & r2, Vector2 & r3)
	{
		pPoly->CopyFrom(*this);
		if (bDebugcp) {
			printf("r1 %1.10E %1.10E r2 %1.10E %1.10E r3 %1.10E %1.10E \n",
				r1.x, r1.y, r2.x, r2.y, r3.x, r3.y);
		};

		if (bDebugcp) {
			// output pPoly
			for (int f = 0; f < pPoly->numCoords; f++)
				printf("After none %d %1.12E %1.12E \n", f, pPoly->coord[f].x, pPoly->coord[f].y);
		};

		int retval = pPoly->ClipAgainstHalfplane(r1, r2, r3);
		if (retval == 0) return false;
		if (retval == 2) {

			// Try a different sequence of halfplanes! Desperate!

			retval = pPoly->ClipAgainstHalfplane(r1, r3, r2);
			if (retval == 0) return false;
			if (retval == 2) // oh dear
			{
				// Try the other one first:
				retval = pPoly->ClipAgainstHalfplane(r2, r3, r1);
				if (retval == 0) return false;
				if (retval == 2) return false; // give up
				retval = pPoly->ClipAgainstHalfplane(r1, r3, r2);
				if (retval == 0) return false;
				if (retval == 2) return false; // give up

			} else {

				retval = pPoly->ClipAgainstHalfplane(r2, r3, r1);
				if (retval == 0) return false;
				if (retval == 2) return false; // give up
			};

			int retval = pPoly->ClipAgainstHalfplane(r1, r2, r3);
			if (retval == 0) return false;
			if (retval == 2) return false; // give up because there were still two intersections.
			
			return true;
		} else {
			
			if (bDebugcp) {
				// output pPoly
				for (int f = 0; f < pPoly->numCoords; f++)
					printf("After 1 %d %1.12E %1.12E \n", f, pPoly->coord[f].x, pPoly->coord[f].y);
			};

			retval = pPoly->ClipAgainstHalfplane(r1, r3, r2);
			if (retval == 0) return false;
			if (retval == 2) {

				retval = pPoly->ClipAgainstHalfplane(r2, r3, r1);
				if (retval == 0) return false;
				if (retval == 2) return false; // give up
				retval = pPoly->ClipAgainstHalfplane(r1, r3, r2);
				if (retval == 0) return false;
				if (retval == 2) return false; // give up
			} else {

				if (bDebugcp) {
					// output pPoly
					for (int f = 0; f < pPoly->numCoords; f++)
						printf("After 2 %d %1.12E %1.12E \n", f, pPoly->coord[f].x, pPoly->coord[f].y);
				};

				retval = pPoly->ClipAgainstHalfplane(r2, r3, r1);
				if (retval == 0) return false;
				if (retval == 2) return false; // give up

				if (bDebugcp) {
					// output pPoly
					for (int f = 0; f < pPoly->numCoords; f++)
						printf("After 3 %d %1.12E %1.12E \n", f, pPoly->coord[f].x, pPoly->coord[f].y);
				};
			};
		};
		return true;
	}
	
	bool ConvexPolygon::GetIntersectionWithPolygon(ConvexPolygon * pPoly, // target
												ConvexPolygon * pClip)// clip this against that.
	{
		// IF CALLING THIS WITHOUT CONVEX POLYGONS, BEWARE AND RECONFIGURE TO SPLIT UP NONCONVEX INTO TRIANGLES.

		int i, inext, inext2;

		pPoly->CopyFrom(*this);
		
		// convex polygon: if we take any edge then we should be fine to supply any other point as being on the "in" side of that edge..

		// pClip, the one we clip against, needs a centroid so we can determine what's inside.

		f64_vec2 centclip(0.0, 0.0);
		for (i = 0; i < pClip->numCoords; i++)
		{
			centclip += pClip->coord[i];
		}
		centclip.x /= (real)pClip->numCoords;
		centclip.y /= (real)pClip->numCoords;

		for (i = 0; i < pClip->numCoords; i++)
		{
			inext = i+1; if (inext == pClip->numCoords) inext = 0;
			inext2 = inext+1; if (inext2 == pClip->numCoords) inext2 = 0;

			if (bSpit) {
				printf("pPoly: %d : ", pPoly->numCoords);
				for (int ii = 0; ii < pPoly->numCoords; ii++)
					printf("%1.8E %1.8E | ", pPoly->coord[ii].x, pPoly->coord[ii].y);
				printf("\n--------------------\n"
					"%1.8E %1.8E ; %1.8E %1.8E ; %1.8E %1.8E \n",
					pClip->coord[i].x, pClip->coord[i].y,
					pClip->coord[inext].x, pClip->coord[inext].y,
					centclip.x, centclip.y);
			};
			
			if (bDebugcp) printf("i = %d : ClipAgainstHalfplane %1.10E %1.10E | %1.10E %1.10E | %1.10E %1.10E\n",
				i, pClip->coord[i].x, pClip->coord[i].y, pClip->coord[inext].x, pClip->coord[inext].y, centclip.x, centclip.y);
			
			int ret = pPoly->ClipAgainstHalfplane(pClip->coord[i], pClip->coord[inext], centclip);
			if ((ret == 0) || (ret == 2)) {
				if (bSpit) printf("return false.\n");
				return false;
			};
		};		
		return true;
	}

/*
	real ConvexPolygon::IntegratePlane(Vector2 & r1, Vector2 & r2, Vector2 & r3,
																	real y1, real y2, real y3)
	{
		// Procedure: 
		// evaluate planar variable at all corners
		// chop up this into triangles
		// assume average attained by plane on each triangle
		// take sum of multiplying average by area of triangle
		real y[CP_MAX];


		// make tri-aligned coordinates:

		Vector2 x12 = r2 - r1;
		real dist12 = x12.modulus();
		x12.x /= dist12;
		x12.y /= dist12;
		Vector2 x12perp;
		x12perp.x = x12.y;
		x12perp.y = -x12.x;
		
		dbyd12 = (y2-y1)/dist12;
		Vector2 x13 = r3-r1;
		real x13_12 = x13.x*x12.x + x13.y*x12.y;
		real x13_perp = x13.x*x12perp.x + x13.y*x12perp.y;
		//Vector2 position = r1 + x13_12*x12;
		real ypos = y1 + dbyd12*x13_12;
		real dbydperp = (y3-ypos)/x13_perp;
		Vector2 relpos;
		// Now plane is
		// y1 + dbyd12*((x-r1) dot x12) + dbydperp*((x-r1) dot x12perp)
		
		// evaluate planar variable at all corners
		for (int i = 0; i < numCoords; i++)
		{
			relpos = coord[i] - r1;
			y[i] = y1 + dbyd12*(relpos.x*x12.x + relpos.y*x12.y) + dbydperp*(relpos.x*x12perp.x+relpos.y*x12perp.y);
		};

		// chop up this into triangles
		// assume average attained by plane on each triangle
		// take sum of multiplying average by area of triangle

		// pick point 0 and make tris
		// we know the points should always be ordered
		real average;
		ConvexPolygon cpTri;
		real sum = 0.0;
		for (int i = 2; i < numCoords; i++)
		{
			average = (y[0] + y[i-1] + y[i])*THIRD;
			cpTri.clear();
			cpTri.add(coord[0]);
			cpTri.add(coord[i-1]);
			cpTri.add(coord[i]);
			area = cpTri.GetArea();
			sum += average*area;
		};
		// case of 3 coords: 0,1,2
		// case of 4 coords: 0,1,2  0,2,3

		return sum;
	}*/
	void ConvexPolygon::IntegrateMass(Vector2 & r1, Vector2 & r2, Vector2 & r3,
									real yvals1, real yvals2, real yvals3, real * pResult)
	{
		real y[CP_MAX];

		Vector2 relpos;
		Vector2 x12perp,x12;
		real dist12;
		real ypos, dbyd12, dbydperp;
		
		// make tri-aligned coordinates:
		x12 = r2 - r1;
		dist12 = x12.modulus();
		x12.x /= dist12;
		x12.y /= dist12;
		x12perp.x = x12.y;
		x12perp.y = -x12.x;
		
		Vector2 x13 = r3-r1;
		// dot products to give lengths:
		real x13_12 = x13.x*x12.x + x13.y*x12.y;
		real x13_perp = x13.x*x12perp.x + x13.y*x12perp.y;
		
		dbyd12 = (yvals2-yvals1)/dist12;
		ypos = yvals1 + dbyd12*x13_12;
		dbydperp = (yvals3-ypos)/x13_perp;

		for (int i = 0; i < numCoords; i++)
		{
			relpos = coord[i] - r1;
			y[i] = yvals1 + dbyd12*(relpos.x*x12.x + relpos.y*x12.y) + dbydperp*(relpos.x*x12perp.x+relpos.y*x12perp.y);
		};

		*pResult = 0.0;
		// pick point 0 and make tris
		// we know the points should always be ordered
		real average,area;
		ConvexPolygon cpTri;
		for (int i = 2; i < numCoords; i++)
		{
			cpTri.Clear();
			cpTri.add(coord[0]);
			cpTri.add(coord[i-1]);
			cpTri.add(coord[i]);
			area = cpTri.GetArea();
			average = (y[0] + y[i-1] + y[i])*THIRD;			
			*pResult += average*area;
		};
	}
						
	void ConvexPolygon::Integrate_Planes(Vector2 & r1, Vector2 & r2, Vector2 & r3,
										real yvals1[],
										real yvals2[],
										real yvals3[],	
										real results[],
										long N_planes)
	{
		// Procedure: 
		// evaluate planar variable at all corners
		// chop up this into triangles
		// assume average attained by plane on each triangle
		// take sum of multiplying average by area of triangle


		// So what are we assuming here? That the polygon to integrate over is
		// a subset of the triangle??

		real y[CP_MAX][15]; // max 15 planes
		Vector2 relpos;
		Vector2 x12perp,x12;
		real dist12;
		real ypos, dbyd12[15], dbydperp[15];
		
		if (this->numCoords >= CP_MAX) printf("Warning! numCoords %d >= CP_MAX\n\a", numCoords);

		// make tri-aligned coordinates:
		x12 = r2 - r1;
		dist12 = x12.modulus();

	//	printf("dist12 %1.8E  \n", dist12);
		x12.x /= dist12;
		x12.y /= dist12;
		x12perp.x = x12.y;
		x12perp.y = -x12.x;
		
		// So x12 is a unit vector

		Vector2 x13 = r3-r1;
		// dot products to give lengths:
		real x13_12 = x13.x*x12.x + x13.y*x12.y;
		real x13_perp = x13.x*x12perp.x + x13.y*x12perp.y;
		
		// So x13_12 is projection of x13 in direction 12

	//	printf("dist12 %1.8E x13_perp %1.8E \n", dist12, x13_perp);

		for (int j = 0; j < N_planes; j++)
		{
			dbyd12[j] = (yvals2[j]-yvals1[j])/dist12;
			ypos = yvals1[j] + dbyd12[j]*x13_12; // value at point along 12 as far as x13 projected
			dbydperp[j] = (yvals3[j]-ypos)/x13_perp;
		};
		// Now plane is
		// y1 + dbyd12*((x-r1) dot x12) + dbydperp*((x-r1) dot x12perp)
		 
		// evaluate planar variable at all corners
		for (int i = 0; i < numCoords; i++)
		{
			relpos = coord[i] - r1;
			for (int j = 0; j < N_planes; j++)
				y[i][j] = yvals1[j] + dbyd12[j]*(relpos.x*x12.x + relpos.y*x12.y) + dbydperp[j]*(relpos.x*x12perp.x+relpos.y*x12perp.y);
		};
		
		// chop up this into triangles
		// assume average attained by plane on each triangle
		// take sum of multiplying average by area of triangle

		for (int j = 0; j < N_planes; j++)
			results[j] = 0.0;
		// pick point 0 and make tris
		// we know the points should always be ordered
		real average,area;
		ConvexPolygon cpTri;
		for (int i = 2; i < numCoords; i++)
		{
			cpTri.Clear();
			cpTri.add(coord[0]);
			cpTri.add(coord[i-1]);
			cpTri.add(coord[i]);
			area = cpTri.GetArea();
		//	printf("cpTri area %1.8E i %d\n", area, i);

			// 012 023 034 045

			for (int j = 0; j < N_planes; j++)
			{
				average = (y[0][j] + y[i-1][j] + y[i][j])*THIRD;			

	//			printf("j %d avg %1.8E ", j, average);
				results[j] += average*area;
			};
		};
	//	printf("jjj");
		// case of 3 coords: 0,1,2
		// case of 4 coords: 0,1,2  0,2,3
	}




	real ConvexPolygon::FindQuadrilateralIntersectionArea(Vector2 & r1, Vector2 & r2, Vector2 & r3, Vector2 & r4)
	{
		ConvexPolygon cp;
		cp.CopyFrom(*this);

		if (!cp.ClipAgainstHalfplane(r1,r2,r3)) return 0.0;
		if (!cp.ClipAgainstHalfplane(r2,r3,r4)) return 0.0;
		if (!cp.ClipAgainstHalfplane(r3,r4,r1)) return 0.0;
		if (!cp.ClipAgainstHalfplane(r1,r4,r2)) return 0.0;
		return cp.GetArea();
	}
	real ConvexPolygon::GetArea()
	{
		// shoelace formula as we should use elsewhere also.
		if (numCoords == 0) return 0.0;
		real area = 0.0;
		int i;
		for (i = 0; i < numCoords-1; i++)
		{
			area += coord[i].x*coord[i+1].y - coord[i+1].x*coord[i].y;
		};
		area += coord[i].x*coord[0].y - coord[0].x*coord[i].y;
		if (area < 0.0)
			return -area*0.5;
		return area*0.5;
	}

real CalculateTriangleIntersectionArea(Vector2 & x1, Vector2 & x2, Vector2 & x3,
													          Vector2 & r1, Vector2 & r2, Vector2 & r3)
{
	// Get stack overflow and it appears here ?!!

	// Sometimes this way fails.

	// Spurious point gets added when we clip against a plane that meets the boundary.

	// This invalidates the shoelace area formula which relies on convexity.


	//bool boolIntersection_exists;
	ConvexPolygon cp (x1,x2,x3);

	// Clip against half plane created by r1 to r2 in the direction of r3

	if (!cp.ClipAgainstHalfplane(r1,r2, r3)) // returns true if intersection existed
		return 0.0;

	if (!cp.ClipAgainstHalfplane(r2,r3,r1))
		return 0.0;

	if (!cp.ClipAgainstHalfplane(r1,r3,r2))
		return 0.0;

	return cp.GetArea(); // shoelace formula as we should use for triangle also.

 	// The way sketched out below may be faster so leave it in and try it after.
}


int Triangle::GetCornerIndex(Vertex * pVertex)
{
	if (cornerptr[0] == pVertex) return 0;
	if (cornerptr[1] == pVertex) return 1;
	return 2;
}



real Triangle::ReturnNormalDist(Vertex * pOppVert)
{
	Vector2 u[3];
	real dist;
	MapLeftIfNecessary(u[0],u[1],u[2]);
	
	if (pOppVert == cornerptr[0])
	{
		dist = edge_normal[0].dot(u[1]-u[0]);
		return dist;
	};
	if (pOppVert == cornerptr[1])
	{
		dist = edge_normal[1].dot(u[0]-u[1]);
		return dist;
	};
	dist = edge_normal[2].dot(u[0]-u[2]);
	return dist;
}

Vector2 CreateOutwardNormal(real x1, real y1,
					real x2, real y2,
					real x, real y)
{
	// (x,y) is on the "inside"
	Vector2 normal;

	normal.x = y2-y1;
	normal.y = x1-x2;
	if (normal.x*(x-x1)+normal.y*(y-y1) > 0.0)
	{
		// case: normal points towards (x,y) from (x1,y1)
		normal.x = -normal.x;
		normal.y = -normal.y;
	}

	return normal;
}

Vertex * TriMesh::Search_for_iVolley_equals (Vertex * pSeed,int value)
{
	if (pSeed->iVolley == value) return pSeed; // should not happen though
	pSeed->iIndicator = 1; // searched
	smartlong searched;
	searched.add(pSeed-X);

	Vertex * pNeigh,*pVertex,*pReturn;
	int iNeigh, i;
	// work outwards: look at neighbours
	// set indicator for search? Need smth like this.

	// Idea: Repeatedly take element from searched, scroll down it;
	// if neighbours are not already searched, search them and add to
	// bottom of the list.		

	long neigh_len;
	long izNeighs[128];

	long iSearchCaret = 0;
	int not_found = true;
	do
	{
		pVertex = X + searched.ptr[iSearchCaret];

		neigh_len = pVertex->GetNeighIndexArray(izNeighs);

		for (i = 0; i < neigh_len; i++)
		{
			pNeigh = X + izNeighs[i];
			if (pNeigh->iIndicator == 0) {
				if (pNeigh->iVolley == value) 
				{
					not_found = false;
					pReturn = pNeigh;
					break; // where this takes us out to, not sure, but hopefully doesn't matter
				}
				pNeigh->iIndicator = 1;
				searched.add(pNeigh-X);
			}
		}
		iSearchCaret++;
	} while (not_found);

	// Need to restore iIndicator == 0 at the end of a search....
	for (i = 0; i < searched.len; i++)
		(X + searched.ptr[i])->iIndicator = 0;
	
	return pReturn;
}


/*AuxVertex * TriMesh::Search_for_iVolley_equals (AuxVertex * pSeed,int value, int iLevel)
{
	if (pSeed->iVolley == value) return pSeed; // should not happen though
	pSeed->iIndicator = 1; // searched
	smartlong searched;
	searched.add(pSeed-AuxX[iLevel]);

	AuxVertex * pNeigh,*pAux,*pReturn;
	int iNeigh, i;
	// work outwards: look at neighbours
	// set indicator for search? Need smth like this.

	// Idea: Repeatedly take element from searched, scroll down it;
	// if neighbours are not already searched, search them and add to
	// bottom of the list.		

	long iSearchCaret = 0;
	int not_found = true;
	do
	{
		pAux = AuxX[iLevel] + searched.ptr[iSearchCaret];
		for (i = 0; i < pAux->neigh_len; i++)
		{
			pNeigh = AuxX[iLevel] + pAux->iNeighbours[i];
			if (pNeigh->iIndicator == 0) {
				if (pNeigh->iVolley == value) 
				{
					not_found = false;
					pReturn = pNeigh;
					break; // where this takes us out to, not sure, but hopefully doesn't matter
				}
				pNeigh->iIndicator = 1;
				searched.add(pNeigh-AuxX[iLevel]);
			}
		}
		iSearchCaret++;
	} while (not_found);

	// Need to restore iIndicator == 0 at the end of a search....
	for (i = 0; i < searched.len; i++)
		(AuxX[iLevel] + searched.ptr[i])->iIndicator = 0;
	
	return pReturn;
}*/
/*
void AuxTriangle::Set(AuxVertex * p1, AuxVertex * p2, AuxVertex * p3, long iTri)
{
	cornerptr[0] = p1;
	cornerptr[1] = p2;
	cornerptr[2] = p3;
	
	// Also calc circumcentre because we may be about to use it:
	Vector2 Bb,C,b,c,a;
	Vector2 basea,baseb;
	
	if (periodic > 0)
	{
		// map everything to left hand side.
		MapLeft(a,b,c);
	} else {
		PopulatePositions(a,b,c);
	};
	Bb = b-a;
	C = c-a;		
	real D = 2.0*(Bb.x*C.y-Bb.y*C.x);
	real modB = Bb.x*Bb.x+Bb.y*Bb.y;
	real modC = C.x*C.x+C.y*C.y;
	cc.x = (C.y*modB-Bb.y*modC)/D + a.x;
	cc.y = (Bb.x*modC-C.x*modB)/D + a.y;

	p1->addtri(iTri);
	p2->addtri(iTri);
	p3->addtri(iTri);
	
}

void AuxTriangle::Reset(AuxVertex * p1, AuxVertex * p2, AuxVertex * p3, long iTri)
{
	// First delete from existing vertex lists
	cornerptr[0]->remove_tri(iTri);
	cornerptr[1]->remove_tri(iTri);
	cornerptr[2]->remove_tri(iTri);

	Set(p1,p2,p3,iTri);	
}

AuxTriangle * TriMesh::GetAuxTriangleContaining(AuxVertex * pAux1,
							   AuxVertex * pAux2,
							   int iLevel)
{
	int i;
	AuxTriangle * pITri;
	for (i = 0; i < pAux1->tri_len; i++)
	{
		pITri = AuxT[iLevel]+pAux1->iTriangles[i];
		if ((pITri->cornerptr[0] == pAux2) ||
			(pITri->cornerptr[1] == pAux2) ||
			(pITri->cornerptr[2] == pAux2))
		{
			return pITri;
		}
	}
	return 0;
}

bool AuxTriangle::TestDelaunay(AuxVertex * pAux)
{
	real qdistsq = (pAux->x-cc.x)*(pAux->x-cc.x)+(pAux->y-cc.y)*(pAux->y-cc.y);
	real pdistsq = (cornerptr[0]->pos.x-cc.x)*(cornerptr[0]->pos.x-cc.x)
				+ (cornerptr[0]->pos.y-cc.y)*(cornerptr[0]->pos.y-cc.y);
	return (qdistsq < pdistsq);
	// return 1 if q is within circumcircle
}


/*
real Triangle::CalculateIntersectionArea(Vector2 & x1, Vector2 & x2, Vector2 & x3,
										                          Vector2 & r1, Vector2 & r2, Vector2 & r3)
{
	// Note that this routine works strictly on the actual given coordinates
	// So make sure any periodic mapping is done beforehand.

	// First let's establish a series of points that split up columns, starting with the leftmost point we have got.

	// a. Put each triangle into left-to-right order
	Vector2 temp,temp2;

	if (x2.x > x1.x )
	{
		if (x3.x > x2.x) 
		{
			// nothing to do
		} else {
			if (x3.x < x1.x)
			{
				// order is x3, x1, x2
				temp.x = x1.x;
				temp.y = x1.y;
				x1.x = x3.x;
				x1.y = x3.y;
				x3.x = x2.x;
				x3.y = x2.y;
				x2.x = temp.x;
				x2.y = temp.y;
			} else {
				// order is x1, x3, x2
				temp.x = x2.x;
				temp.y = x2.y;
				x2.x = x3.x;
				x2.y = x3.y;
				x3.x = temp.x;
				x3.y = temp.y;
			};
		};
	} else {
		if (x3.x > x1.x)
		{
			// order is x2 x1 x3
			temp.x = x1.x;
			temp.y = x1.y;
			x1.x = x2.x;
			x1.y = x2.y;
			x2.x = temp.x;
			x2.y = temp.y;
		} else {
			if (x3.x > x2.x)
			{
				// order is x2 x3 x1
				temp.x = x1.x;
				temp.y = x1.y;
				x1.x = x2.x;
				x1.y = x2.y;
				x2.x = x3.x;
				x2.y = x3.y;
				x3.x = temp.x;
				x3.y = temp.y;
			} else {
				// order is x3 x2 x1
				temp.x = x1.x;
				temp.y = x1.y;
				x1.x = x3.x;
				x1.y = x3.y;
				x3.x = temp.x;
				x3.y = temp.y;
			};
		};
	};
	
	// b. Where is the leftmost point? 
	// No point starting until both triangles are started.

	// Write this out in a longwinded way unless and until we think of a clever way.

	real area, gradient_r12, gradient_r23, gradient_x12, gradient_x23;

	area = 0.0;

	gradient_r12 = (r2.y-r1.y)/(r2.x-r1.x);
	gradient_r13 = (r3.y-r1.y)/(r3.x-r1.x);
	gradient_x12 = (x2.y-x1.y)/(x2.x-x1.x);
	gradient_x13 = (x3.y-x1.y)/(x3.x-x1.x);

	if (x1.x < r1.x)
	{
		// start at r1.x
		if (x2.x < r2.x)
		{
			// x1 r1 x2
			// going from r1.x to x2.x
			area += ColumnIntersection(
									// column x-values start and finish:
									r1.x,x2.x,
									// y-values:
									r1.y,			// left top for r
									r1.y,			// left bot for r
									r1.y + gradient_r12*(x2.x-r1.x),//(r2.y-r1.y)*(x2.x-r1.x)/(r2.x-r1.x),   // right top for r
									r1.y + gradient_r13*(x2.x-r1.x),//(r3.y-r1.y)*(x2.x-r1.x)/(r3.x-r1.x),   // right bot for r
									x1.y + gradient_x12*(r1.x-x1.x),//(x2.y-x1.y)*(r1.x-x1.x)/(x2.x-x1.x), // left top for x
									x1.y + gradient_x13*(r1.x-x1.x),//(x3.y-x1.y)*(r1.x-x1.x)/(x3.x-x1.x), // left bot for x
									x2.y,    // right top for x
									x1.y + gradient_x13*(x2.x-x1.x)//(x3.y-x1.y)*(x2.x-x1.x)/(x3.x-x1.x) // right bot for x
									);		
			if (r2.x < x3.x)
			{
				// going from x2.x to r2.x

				area += ColumnIntersection(
									// column x-values start and finish:
									x2.x,r2.x,
									// y-values:
									r1.y+gradient_r12*(x2.x-r1.x),//(r2.y-r1.y)*(x2.x-r1.x)/(r2.x-r1.x),			// left top for r
									r1.y+gradient_r13*(x2.x-r1.x),//(r3.y-r1.y)*(x2.x-r1.x)/(r3.x-r1.x),			// left bot for r
									r2.y,   // right top for r
									r1.y + gradient_r13*(r2.x-r1.x),//(r3.y-r1.y)*(r2.x-r1.x)/(r3.x-r1.x),   // right bot for r
									x2.y, // left top for x
									x1.y + gradient_x13*(x2.x-x1.x),//(x3.y-x1.y)*(x2.x-x1.x)/(x3.x-x1.x), // left bot for x
									x2.y + gradient_x23*(r2.x-x2.x),//(x3.y-x2.y)*(r2.x-x2.x)/(x3.x-x2.x),    // right top for x
									x1.y + gradient_x13*(r2.x-x1.x)//(x3.y-x1.y)*(r2.x-x1.x)/(x3.x-x1.x) // right bot for x
									);		

				if (x3.x < r3.x)
				{
					// the final one goes from r2.x to x3.x

					area += ColumnIntersection(
										r2.x,x3.x,
										// y-values:
										r2.y,							// left via r2
										r1.y+gradient_r13*(r2.x-r1.x),//(r3.y-r1.y)*(r2.x-r1.x)/(r3.x-r1.x), // left, r1->r3
										r2.y + gradient_r23*(x3.x-r2.x),//(r3.y-r2.y)*(x3.x-r2.x)/(r3.x-r2.x), // right via r2,
										r1.y + gradient_r13*(x3.x-r1.x),//(r3.y-r1.y)*(x3.x-r1.x)/(r3.x-r1.x), // right, r1->r3,
										
										x2.y + gradient_x23*(r2.x-x2.x),//(x3.y-x2.y)*(r2.x-x2.x)/(x3.x-x2.x),
										x1.y + gradient_x13*(r2.x-x1.x),//(x3.y-x1.y)*(r2.x-x1.x)/(x3.x-x1.x),
										x3.y,
										x3.y
										);
				} else {
					// the final one goes from r2.x to r3.x

					area += ColumnIntersection(
										r2.x,r3.x,
										// y-values:
										r2.y,
										r1.y+gradient_13*(r2.x-r1.x),//(r3.y-r1.y)*(r2.x-r1.x)/(r3.x-r1.x), // left, r1->r3
										r3.y,
										r3.y,
										x2.y + gradient_x23*(r2.x-x2.x),//(x3.y-x2.y)*(r2.x-x2.x)/(x3.x-x2.x),
										x1.y + gradient_x13*(r2.x-x1.x),//(x3.y-x1.y)*(r2.x-x1.x)/(x3.x-x1.x),
										x2.y + gradient_x23*(r3.x-x2.x),//(x3.y-x2.y)*(r3.x-x2.x)/(x3.x-x2.x),
										x1.y + gradient_x13*(r3.x-x1.x)//(x3.y-x1.y)*(r3.x-x1.x)/(x3.x-x1.x)
										);										
				};
			} else {
				// the final one from x2.x to x3.x
				// x1 r1 x2 x3
				// some efficiency savings to be made if we calculate gradient_s before calling these functions.
				area += ColumnIntersection(
								x2.x,x3.x,
								r1.y+gradient_r12*(x2.x-r1.x),
								r1.y+gradient_r13*(x2.x-r1.x),
								r1.y+gradient_r12*(x3.x-r1.x),
								r1.y+gradient_r13*(x3.x-r1.x),
								x2.y,
								x1.y+gradient_x13*(x2.x-x1.x),
								x3.y,
								x3.y);
			};
		} else {
			// x1 r1 r2 
			// going from r1.x to r2.x

			area += ColumnIntersection(
									// column x-values start and finish:
									r1.x,r2.x,
									// y-values:
									r1.y,			// left bot for r
									r1.y,			// left top for r
									r2.y,			 // right top for r
									r1.y + gradient_r13*(r2.x-r1.x),   // right bot for r
									
									x1.y + gradient_x12*(r1.x-x1.x), // left top for x
									x1.y + gradient_x13*(r1.x-x1.x), // left bot for x
									x1.y + gradient_x12*(r2.x-x1.x), // left top for x
									x1.y + gradient_x13*(r2.x-x1.x), // left bot for x
									);
			if (x2.x < r3.x)
			{
				// x1 r1 r2 x2

				// going from r2.x to x2.x

				area += ColumnIntersection(
									r2.x,x2.x,
									// y-values:
									r2.y,
									r1.y+gradient_r13*(r2.x-r1.x),
									r2.y+gradient_r23*(x2.x-r2.x),
									r1.y+gradient_r13*(x2.x-r1.x),
									
									x1.y+gradient_x12*(r2.x-x1.x),
									x1.y+gradient_x13*(r2.x-x1.x),
									x2.y,
									x1.y+gradient_x13*(x2.x-x1.x)
									);
									


				if (r3.x < x3.x)
				{
					// the final one from x2.x to r3.x
					
					// x1 r1 r2 x2 r3

					area += ColumnIntersection(
									x2.x,r3.x,
									
									r2.y + gradient_r23*(x2.x-r2.x),
									r1.y + gradient_r13*(x2.x-r1.x),
									r3.y,
									r3.y,

									x2.y,
									x1.y+gradient_x13*(x2.x-x1.x),
									x2.y+gradient_x23*(r3.x-x2.x),
									x1.y+gradient_x13*(r3.x-x1.x)
									);
				} else {
					// the final one from x2.x to x3.x
					
					// x1 r1 r2 x2 x3
					area += ColumnIntersection(
									x2.x,x3.x,

									r2.y + gradient_r23*(x2.x-r2.x),
									r1.y + gradient_r13*(x2.x-r1.x),
									r2.y + gradient_r23*(x3.x-r2.x),
									r1.y + gradient_r13*(x3.x-r1.x),

									x2.y,
									x1.y + gradient_x13*(x2.x-x1.x),
									x3.y,
									x3.y);
									
				};
			} else {
				// the final one from r2.x to r3.x

				// x1 r1 r2 r3
				area += ColumnIntersection(
					r2.x,r3.x,

					r2.y,
					r1.y + gradient_r13*(r2.x-r1.x),
					r3.y,
					r3.y,

					x1.y + gradient_r12*(r2.x-x1.x),
					x1.y + gradient_r13*(r2.x-x1.x),
					x1.y + gradient_r12*(r3.x-x1.x),
					x1.y + gradient_r13*(r3.x-x1.x)
					);
			};
		};
	} else {
		// r1  x1
		if (x2.x < r2.x)
		{
			// going from x1.x to x2.x

		} else {
			// r1 x1 r2 
			// going from x1.x to r2.x

			




		}
	};

	return area;
}

real ColumnIntersection( real x1, real x2,
						real y_a_1_left,
						real y_a_2_left,
						real y_a_1_right,
						real y_a_2_right,

						real y_b_1_left,
						real y_b_2_left,
						real y_b_1_right,
						real y_b_2_right)
{
	// Is it possible for x1->x2->x3 and x1->x3 to cross in this region?
	
	// take average to determine which of 1,2 is higher for a
	real b_bvg,a_avg, y_a_top_left,y_a_top_right,y_a_bot_left,y_a_bot_right,
		y_b_top_left,y_b_top_right,y_b_bot_left,y_b_bot_right;

	real width = x2-x1;
	a_avg1 = y_a_1_left + y_a_1_right;
	a_avg2 = y_a_2_left + y_a_2_right;		// can avoid this by keeping global flag for whether gradient12 > gradient13 for first triangle
	if (a_avg2 > a_avg1)
	{
		y_a_top_left = y_a_2_left; y_a_top_right = y_a_2_right;
		y_a_bot_left = y_a_1_left; y_a_bot_right = y_a_1_right;
	} else {
		y_a_top_left = y_a_1_left; y_a_top_right = y_a_1_right;
		y_a_bot_left = y_a_2_left; y_a_bot_right = y_a_2_right;
	};

	b_bvg1 = y_b_1_left + y_b_1_right;
	b_bvg2 = y_b_2_left + y_b_2_right;
	if (b_bvg2 > b_bvg1)
	{
		y_b_top_left = y_b_2_left; y_b_top_right = y_b_2_right;
		y_b_bot_left = y_b_1_left; y_b_bot_right = y_b_1_right;
	} else {
		y_b_top_left = y_b_1_left; y_b_top_right = y_b_1_right;
		y_b_bot_left = y_b_2_left; y_b_bot_right = y_b_2_right;
	};

	// Now consider some cases
	
	// We are going to kind of clip a against the two half planes given by b

	// Distinguish 3 possible locations for y_b_top_left rel to a, do cases for where y_b_top right is,
	// then go from there with y_b_bot

	// But do this in an order so that non-intersections can be detected quickly.

	total = 0.0;

		a_bot_gradient = (y_a_bot_right-y_a_bot_left)/width;
		a_top_gradient = (y_a_top_right-y_a_top_left)/width;
		b_bot_gradient = (y_b_bot_right-y_b_bot_left)/width;
		b_top_gradient = (y_b_top_right-y_b_top_left)/width;


	if (y_b_top_left < y_a_bot_left)
	{
		if (y_b_top_right < y_a_bot_right) return 0.0; // CASE I

		if (y_b_top_right < y_a_top_right) {
			// CASE II: b top cuts a bot upwards
			
			total = triangle area()
			x = (b

			if (y_b_bot_right > y_a_bot_right) 
			{
				// remove a triangle
				total -= triangle area()

			};
			return total;

		} else {
			// CASE III: b top cuts both lines of a from below
			
			if (
		};
	} else {
		if (y_b_top_left > y_a_top_left)
		{
			// b top started above a
			if (y_b_top_right > y_a_top_right)
			{
				// CASE IV: b top had no effect on a
			} else {
				if (y_b_top_right > y_a_bot_right)
				{
					// CASE V: b top cuts a top downwards

				} else {
					// CASE VI: b top cuts both lines of a, downwards

				};
			};
		} else {
			// b_top started in between
			
			if (y_b_top_right > y_a_top_right)
			{
				// CASE VII: b top starts inside, goes up and outside
			} else {
				if (y_b_top_right > y_a_bot_right)
				{
					// CASE VIII: b top starts inside, stays inside;

				} else {
					// CASE XI: b top starts inside, goes below; only a triangle is then relevant

				};
			};
		};
	};


	if (y_b_bot_left > y_a_top_left)
	{
		// First test for no intersection:
		if (y_b_bot_right > y_a_top_right) return 0.0;
				
		// therefore, at some point b_bot crosses a_top.
		// until that point, we have no intersection.
		
		// difference of gradients (b bot' - a top') * x + original difference (b bot - a top) = 0
		// x = - original difference / difference of gradients

		a_bot_gradient = (y_a_bot_right-y_a_bot_left)/width;
		a_top_gradient = (y_a_top_right-y_a_top_left)/width;
		b_bot_gradient = (y_b_bot_right-y_b_bot_left)/width;
		b_top_gradient = (y_b_top_right-y_b_top_left)/width;
		
		x = (y_a_top_left-y_b_bot_left)/(b_bot_gradient - a_top_gradient);
		
		// Now from this point onwards what can happen?


		x = (y_b_bot_left-y_a_top_left)/
			

	} else {
		if ((y_b_top_left < y_a_bot_left) && (y_b_top_right < y_a_bot_right)) return 0.0;

	};

	
	// OK so there is some intersection
	
		
}
*/



#endif
