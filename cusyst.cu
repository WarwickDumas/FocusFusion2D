 
#include "cuda_struct.h"
#pragma once
#ifndef CUSYSTCU
#define CUSYSTCU

extern real evaltime;
extern long GlobalStepsCounter;
extern bool GlobalSuppressSuccessVerbosity;
extern f64 * p_graphdata1_host, *p_graphdata2_host, *p_graphdata3_host, *p_graphdata4_host, *p_graphdata5_host, *p_graphdata6_host;
extern f64 * p_Tgraph_host[9];
extern f64 * p_accelgraph_host[12];
extern f64 * p_Ohmsgraph_host[20];
extern f64 * p_arelz_graph_host[12];
extern f64 * p_temphost5;

__host__ bool Call(hipError_t cudaStatus, char str[])
{
	if (cudaStatus == hipSuccess) {
		if (strncmp(str,"hipMemcpy",8) != 0)
			if (GlobalSuppressSuccessVerbosity == false) printf("\tSuccess: %s ||| \n",str);
		return false;
	} else {
		printf("Error: %s\nReturned %d : %s\n",
			str, cudaStatus, hipGetErrorString(cudaStatus));
		printf("press o\n");
		while (getch() != 'o');
		PerformCUDA_Revoke();
		exit(2030);
	};
	return true;
} 
   

cuSyst::cuSyst(){
	bInvoked = false;
	bInvokedHost = false;
}

int cuSyst::Invoke()
{
	 Nverts = NUMVERTICES;
	 Ntris = NUMTRIANGLES; // FFxtubes.h
	 Nminor = Nverts + Ntris;

	if (bInvoked == false) {

		if (
			   (!CallMAC(hipMalloc((void**)&p_info, Nminor * sizeof(structural))))

			&& (!CallMAC(hipMalloc((void**)&p_izTri_vert, Nverts*MAXNEIGH_d * sizeof(long))))
			&& (!CallMAC(hipMalloc((void**)&p_izNeigh_vert, Nverts*MAXNEIGH_d * sizeof(long))))
			&& (!CallMAC(hipMalloc((void**)&p_szPBCtri_vert, Nverts*MAXNEIGH_d * sizeof(char))))
			&& (!CallMAC(hipMalloc((void**)&p_szPBCneigh_vert, Nverts*MAXNEIGH_d * sizeof(char))))
			
			&& (!CallMAC(hipMalloc((void**)&p_izNeigh_TriMinor, Ntris*6 * sizeof(long))))
			&& (!CallMAC(hipMalloc((void**)&p_szPBC_triminor, Ntris * 6 * sizeof(char))))
			&& (!CallMAC(hipMalloc((void**)&p_tri_corner_index, Ntris * sizeof(LONG3))))
			&& (!CallMAC(hipMalloc((void**)&p_tri_periodic_corner_flags, Ntris * sizeof(CHAR4))))
			&& (!CallMAC(hipMalloc((void**)&p_tri_neigh_index, Ntris * sizeof(LONG3))))
			&& (!CallMAC(hipMalloc((void**)&p_tri_periodic_neigh_flags, Ntris * sizeof(CHAR4))))
			&& (!CallMAC(hipMalloc((void**)&p_who_am_I_to_corner, Ntris * sizeof(LONG3))))

			&& (!CallMAC(hipMalloc((void**)&p_n_major, Nverts * sizeof(nvals))))
			&& (!CallMAC(hipMalloc((void**)&p_n_minor, Nminor * sizeof(nvals))))
			&& (!CallMAC(hipMalloc((void**)&p_T_minor, Nminor * sizeof(T3))))

			&& (!CallMAC(hipMalloc((void**)&p_AAdot, Nminor * sizeof(AAdot))))
			
			&& (!CallMAC(hipMalloc((void**)&p_v_n, Nminor * sizeof(f64_vec3))))
			&& (!CallMAC(hipMalloc((void**)&p_vie, Nminor * sizeof(v4))))
			&& (!CallMAC(hipMalloc((void**)&p_B, Nminor * sizeof(f64_vec3))))

			&& (!CallMAC(hipMalloc((void**)&p_Lap_Az, Nminor * sizeof(f64))))
			&& (!CallMAC(hipMalloc((void**)&p_v_overall_minor, Nminor * sizeof(f64_vec2))))
			&& (!CallMAC(hipMalloc((void**)&p_n_upwind_minor, Nminor * sizeof(nvals))))
						
			&& (!CallMAC(hipMalloc((void**)&p_AreaMinor, Nminor * sizeof(f64))))
			&& (!CallMAC(hipMalloc((void**)&p_AreaMajor, Nverts * sizeof(f64))))
			&& (!CallMAC(hipMalloc((void**)&p_cc, Nminor*sizeof(f64_vec2))))
			
			&& (!CallMAC(hipMalloc((void**)&p_iVolley, Nverts * sizeof(char))))

			)
		{
			bInvoked = true;
			//Zero();
			printf("Dimensioned for MAXNEIGH_d = %d\n", MAXNEIGH_d);
			return 0;
		}
		else {
			printf("There was an error in dimensioning Systdata object.\n");
			getch();	getch();
			return 1;
		};
	}
	else {
		if (Nverts != NUMVERTICES) { printf("cuSyst Error - Nverts %d != N %d\n", Nverts, NUMVERTICES); getch(); }
		return 2;
	};
}
int cuSyst::InvokeHost()
{
	Nverts = NUMVERTICES;
	Ntris = NUMTRIANGLES;
	Nminor = Nverts + Ntris;
	p_info = ( structural * )malloc(Nminor* sizeof(structural));
		
	p_izTri_vert = ( long *)malloc(Nverts*MAXNEIGH_d * sizeof(long));
	p_izNeigh_vert = (long * )malloc(Nverts*MAXNEIGH_d * sizeof(long));
	p_szPBCtri_vert = (char * )malloc(Nverts*MAXNEIGH_d * sizeof(char));
	p_szPBCneigh_vert = (char *)malloc(Nverts*MAXNEIGH_d * sizeof(char));

	p_izNeigh_TriMinor = (long * )malloc(Ntris * 6 * sizeof(long));
	p_szPBC_triminor = (char * )malloc(Ntris * 6 * sizeof(char));
	p_tri_corner_index = ( LONG3 *)malloc(Ntris * sizeof(LONG3));
	p_tri_periodic_corner_flags = (CHAR4 *)malloc(Ntris * sizeof(CHAR4));
	p_tri_neigh_index = (LONG3 *)malloc(Ntris * sizeof(LONG3));
	p_tri_periodic_neigh_flags = (CHAR4 *)malloc(Ntris * sizeof(CHAR4));
	p_who_am_I_to_corner = (LONG3 * )malloc(Ntris * sizeof(LONG3));

	p_n_major = (nvals * )malloc(Nverts * sizeof(nvals));
	p_n_minor = (nvals * )malloc(Nminor * sizeof(nvals));
	p_T_minor = (T3 * )malloc(Nminor * sizeof(T3));

	p_AAdot = ( AAdot *)malloc(Nminor * sizeof(AAdot));

	p_v_n = ( f64_vec3 *)malloc(Nminor * sizeof(f64_vec3));
	p_vie = (v4 * )malloc(Nminor * sizeof(v4));
	p_B = ( f64_vec3 *)malloc(Nminor * sizeof(f64_vec3));

	p_Lap_Az = (f64 * )malloc(Nminor * sizeof(f64));
	p_v_overall_minor = (f64_vec2 *)malloc(Nminor * sizeof(f64_vec2));
	p_n_upwind_minor = (nvals *)malloc(Nminor * sizeof(nvals));

	p_AreaMinor = (f64 * )malloc(Nminor * sizeof(f64));
	p_AreaMajor = (f64 * )malloc(Nverts * sizeof(f64));

	p_cc = (f64_vec2 *)malloc(Nminor * sizeof(f64));
	
	p_iVolley = (char *)malloc(Nverts * sizeof(char));

	
	if (p_cc == 0) {
		printf("failed to invokeHost the cusyst.\n");
		getch();
		return 1;
	}
	else {
		bInvokedHost = true;
		return 0;
	};
}
cuSyst::~cuSyst(){
	if (bInvoked)
	{

		hipFree(p_info);
		hipFree(p_izTri_vert);
		hipFree(p_izNeigh_vert);
		hipFree(p_szPBCtri_vert);
		hipFree(p_szPBCneigh_vert);
		hipFree(p_izNeigh_TriMinor);
		hipFree(p_szPBC_triminor);
		hipFree(p_tri_corner_index);
		hipFree(p_tri_periodic_corner_flags);
		hipFree(p_tri_neigh_index);
		hipFree(p_tri_periodic_neigh_flags);
		hipFree(p_who_am_I_to_corner);
		hipFree(p_n_major);
		hipFree(p_n_minor);
		hipFree(p_n_upwind_minor);
		hipFree(p_T_minor);
		hipFree(p_AAdot);
		hipFree(p_v_n);
		hipFree(p_vie);
		hipFree(p_B);
		hipFree(p_Lap_Az);
		hipFree(p_v_overall_minor);
		hipFree(p_AreaMinor);
		hipFree(p_AreaMajor);
		hipFree(p_cc);
		hipFree(p_iVolley);

	}
	if (bInvokedHost) {

free(p_info);
free(p_izTri_vert);
free(p_izNeigh_vert);
free(p_szPBCtri_vert);
free(p_szPBCneigh_vert);
free(p_izNeigh_TriMinor);
free(p_szPBC_triminor);
free(p_tri_corner_index);
free(p_tri_periodic_corner_flags);
free(p_tri_neigh_index);
free(p_tri_periodic_neigh_flags);
free(p_who_am_I_to_corner);
free(p_n_major);
free(p_n_minor);
free(p_n_upwind_minor);
free(p_T_minor);
free(p_AAdot);
free(p_v_n);
free(p_vie);
free(p_B);
free(p_Lap_Az);
free(p_v_overall_minor);
free(p_AreaMinor);
free(p_AreaMajor);
free(p_cc);
free(p_iVolley);
	};
}

void cuSyst::SaveGraphs(const char filename[])
{
	FILE * fp = fopen(filename, "wb");
	if (fp == 0) { printf("open %s failed\n\n", filename); getch();  return; }
	else { printf("opened file %s ..", filename); }

	long filevers = 1;
	fwrite(&filevers, sizeof(long), 1, fp);
	fwrite(&Nverts, sizeof(long), 1, fp);
	fwrite(&Ntris, sizeof(long), 1, fp);

	fwrite(&GlobalStepsCounter, sizeof(long), 1, fp);
	fwrite(&evaltime, sizeof(f64), 1, fp);

	fwrite(p_info, sizeof(structural), NMINOR, fp);

	fwrite(p_izTri_vert, sizeof(long), Nverts*MAXNEIGH_d, fp);

	fwrite(p_izNeigh_vert, sizeof(long), Nverts*MAXNEIGH_d, fp);
	fwrite(p_szPBCtri_vert, sizeof(char), Nverts*MAXNEIGH_d, fp);
	fwrite(p_szPBCneigh_vert, sizeof(char), Nverts*MAXNEIGH_d, fp);

	fwrite(p_izNeigh_TriMinor, sizeof(long), Ntris * 6, fp);
	fwrite(p_szPBC_triminor, sizeof(char), Ntris * 6, fp);
	fwrite(p_tri_corner_index, sizeof(LONG3), Ntris, fp);
	fwrite(p_tri_periodic_corner_flags, sizeof(CHAR4), Ntris, fp);
	fwrite(p_tri_neigh_index, sizeof(LONG3), Ntris, fp);
	fwrite(p_tri_periodic_neigh_flags, sizeof(CHAR4), Ntris, fp);
	fwrite(p_who_am_I_to_corner, sizeof(LONG3), Ntris, fp);

	fwrite(p_iVolley, sizeof(char), Nverts, fp); // Not changed yet in load.

	fwrite(p_n_major, sizeof(nvals), Nverts, fp);
	fwrite(p_T_minor + BEGINNING_OF_CENTRAL, sizeof(T3), Nverts, fp);
	fwrite(p_AAdot + BEGINNING_OF_CENTRAL, sizeof(AAdot), Nverts, fp);
	fwrite(p_v_n + BEGINNING_OF_CENTRAL, sizeof(f64_vec3), Nverts, fp);
	fwrite(p_vie + BEGINNING_OF_CENTRAL, sizeof(v4), Nverts, fp);
	fwrite(p_B + BEGINNING_OF_CENTRAL, sizeof(f64_vec3), Nverts, fp);
	fwrite(p_AreaMajor, sizeof(f64), Nverts, fp);
	// Now save the graphing data that we use ...
	fwrite(p_graphdata1_host + BEGINNING_OF_CENTRAL, sizeof(f64),Nverts, fp);
	fwrite(p_graphdata2_host + BEGINNING_OF_CENTRAL, sizeof(f64),Nverts, fp);
	fwrite(p_graphdata3_host + BEGINNING_OF_CENTRAL, sizeof(f64),Nverts, fp);
	fwrite(p_graphdata4_host + BEGINNING_OF_CENTRAL, sizeof(f64),Nverts, fp);
	fwrite(p_graphdata5_host + BEGINNING_OF_CENTRAL, sizeof(f64),Nverts, fp);
	fwrite(p_graphdata6_host + BEGINNING_OF_CENTRAL, sizeof(f64),Nverts, fp);
	int i;
	for (i = 0; i < 9; i++)
		fwrite(p_Tgraph_host[i], sizeof(f64),NUMVERTICES, fp);
	for (i = 0; i < 12; i++)
		fwrite(p_accelgraph_host[i], sizeof(f64),NUMVERTICES, fp);
	//for (i = 0; i < 20; i++)
	//	fwrite(p_Ohmsgraph_host[i] = (f64 *)malloc(NUMVERTICES * sizeof(f64));
	// skip ohmsgraph...
	for (i = 0; i < 12; i++)
		fwrite(p_arelz_graph_host[i], sizeof(f64),NUMVERTICES, fp);

	// We only really wanted to save 1D graphs of these, so maybe we should have stuck to that!

	fwrite(p_temphost5 + BEGINNING_OF_CENTRAL, sizeof(f64),NUMVERTICES, fp); // Lap Az

	// so that makes 33 extra doubles per vertex so far. We have to expect the total size will be the same.


	//	fwrite(p_Lap_Az, Nminor * sizeof(f64));
	//	fwrite(p_v_overall_minor, Nminor * sizeof(f64_vec2));
	//	fwrite(p_n_upwind_minor, Nminor * sizeof(nvals));

	

	fclose(fp);
	printf("File save done.\n");
}
void cuSyst::Save(const char filename[])
{
	FILE * fp = fopen(filename, "wb");
	if (fp == 0) { printf("open %s failed\n\n", filename); getch();  return; }
	else { printf("opened file %s ..", filename); }

	long filevers = 1;
	fwrite(&filevers, sizeof(long),1,fp);
	fwrite(&Nverts, sizeof(long),1,fp);
	fwrite(&Ntris, sizeof(long),1,fp);

	fwrite(&GlobalStepsCounter, sizeof(long), 1, fp);
	fwrite(&evaltime, sizeof(f64), 1, fp);

	fwrite(p_info, sizeof(structural),NMINOR, fp);

	fwrite(p_izTri_vert, sizeof(long), Nverts*MAXNEIGH_d, fp);

	fwrite(p_izNeigh_vert, sizeof(long),Nverts*MAXNEIGH_d, fp);
	fwrite(p_szPBCtri_vert, sizeof(char),Nverts*MAXNEIGH_d, fp);
	fwrite(p_szPBCneigh_vert, sizeof(char),Nverts*MAXNEIGH_d, fp);

	fwrite(p_izNeigh_TriMinor, sizeof(long), Ntris * 6 , fp);
	fwrite(p_szPBC_triminor, sizeof(char), Ntris * 6, fp);
	fwrite(p_tri_corner_index, sizeof(LONG3), Ntris , fp);
	fwrite(p_tri_periodic_corner_flags, sizeof(CHAR4), Ntris, fp);
	fwrite(p_tri_neigh_index, sizeof(LONG3), Ntris , fp);
	fwrite(p_tri_periodic_neigh_flags, sizeof(CHAR4), Ntris , fp);
	fwrite(p_who_am_I_to_corner, sizeof(LONG3), Ntris, fp);

	fwrite(p_iVolley, sizeof(char), Nverts, fp); // Not changed yet in load.

	fwrite(p_n_major, sizeof(nvals), Nverts, fp);
	fwrite(p_n_minor, sizeof(nvals), Nminor,fp);
	fwrite(p_T_minor, sizeof(T3), Nminor,fp);

	fwrite(p_AAdot, sizeof(AAdot), Nminor,fp);

	fwrite(p_v_n, sizeof(f64_vec3), Nminor , fp);
	fwrite(p_vie, sizeof(v4), Nminor , fp);
	fwrite(p_B, sizeof(f64_vec3), Nminor , fp);

//	fwrite(p_Lap_Az, Nminor * sizeof(f64));
//	fwrite(p_v_overall_minor, Nminor * sizeof(f64_vec2));
//	fwrite(p_n_upwind_minor, Nminor * sizeof(nvals));

	fwrite(p_AreaMinor, sizeof(f64), Nminor, fp);
	fwrite(p_AreaMajor, sizeof(f64), Nverts, fp);
	
	fclose(fp);
	printf("File save done.\n");
}

void cuSyst::Load(const char filename[])
{
	FILE * fp = fopen(filename, "rb");
	if (fp == 0) { printf("open %s failed\n\n", filename); getch();  return; } 
	else { printf("opened file %s ..", filename); }
	rewind(fp);
	long Nverttest, Ntritest, filevers;
	fread(&filevers, sizeof(long), 1, fp);
	fread(&Nverttest, sizeof(long), 1, fp);
	fread(&Ntritest, sizeof(long), 1, fp);

	if ( (filevers != 1) || (Nverttest != Nverts) || (Ntritest != Ntris) ) {
		printf("filevers %d Nverts Ntris %d %d File: %d %d \n\n", filevers, Nverts, Ntris, Nverttest, Ntritest);
		return;
	}
	
	fread(&GlobalStepsCounter, sizeof(long), 1, fp);
	fread(&evaltime, sizeof(f64), 1, fp);

	fread(p_info, sizeof(structural), NMINOR, fp);

	fread(p_izTri_vert, sizeof(long), Nverts*MAXNEIGH_d, fp);
	fread(p_izNeigh_vert, sizeof(long), Nverts*MAXNEIGH_d, fp);
	fread(p_szPBCtri_vert, sizeof(char), Nverts*MAXNEIGH_d, fp);
	fread(p_szPBCneigh_vert, sizeof(char), Nverts*MAXNEIGH_d, fp);

	fread(p_izNeigh_TriMinor, sizeof(long), Ntris * 6, fp);
	fread(p_szPBC_triminor, sizeof(char), Ntris * 6, fp);
	fread(p_tri_corner_index, sizeof(LONG3), Ntris, fp);
	fread(p_tri_periodic_corner_flags, sizeof(CHAR4), Ntris, fp);
	fread(p_tri_neigh_index, sizeof(LONG3), Ntris, fp);
	fread(p_tri_periodic_neigh_flags, sizeof(CHAR4), Ntris, fp);
	fread(p_who_am_I_to_corner, sizeof(LONG3), Ntris, fp);

	// p_iVolley
	fread(p_iVolley, sizeof(char), Nverts, fp);

	fread(p_n_major, sizeof(nvals), Nverts, fp);
	fread(p_n_minor, sizeof(nvals), NMINOR, fp);
	fread(p_T_minor, sizeof(T3), NMINOR, fp);
	fread(p_AAdot, sizeof(AAdot), NMINOR, fp);
	fread(p_v_n, sizeof(f64_vec3), NMINOR, fp);
	fread(p_vie, sizeof(v4), NMINOR, fp);
	fread(p_B, sizeof(f64_vec3), NMINOR, fp);

	//	fread(p_Lap_Az, Nminor * sizeof(f64));
	//	fread(p_v_overall_minor, Nminor * sizeof(f64_vec2));
	//	fread(p_n_upwind_minor, Nminor * sizeof(nvals));

	fread(p_AreaMinor, sizeof(f64), NMINOR, fp);
	fread(p_AreaMajor, sizeof(f64), Nverts, fp);

	fclose(fp);
	printf("File read done.\n");
}


void cuSyst::SendToHost(cuSyst & Xhost)
{
	// We are going to need a host-allocated cuSyst in order to
	// do the populating basically.
	if ((!CallMAC(hipMemcpy(Xhost.p_info, p_info, Nminor * sizeof(structural), hipMemcpyDeviceToHost)))

		&& (!CallMAC(hipMemcpy(Xhost.p_izTri_vert, p_izTri_vert, Nverts*MAXNEIGH_d * sizeof(long), hipMemcpyDeviceToHost)))
		&& (!CallMAC(hipMemcpy(Xhost.p_izNeigh_vert, p_izNeigh_vert, Nverts*MAXNEIGH_d * sizeof(long), hipMemcpyDeviceToHost)))
		&& (!CallMAC(hipMemcpy(Xhost.p_szPBCtri_vert, p_szPBCtri_vert, Nverts*MAXNEIGH_d * sizeof(char), hipMemcpyDeviceToHost)))
		&& (!CallMAC(hipMemcpy(Xhost.p_szPBCneigh_vert, p_szPBCneigh_vert, Nverts*MAXNEIGH_d * sizeof(char), hipMemcpyDeviceToHost)))

		&& (!CallMAC(hipMemcpy(Xhost.p_izNeigh_TriMinor, p_izNeigh_TriMinor, Ntris * 6 * sizeof(long), hipMemcpyDeviceToHost)))
		&& (!CallMAC(hipMemcpy(Xhost.p_szPBC_triminor, p_szPBC_triminor, Ntris * 6 * sizeof(char), hipMemcpyDeviceToHost)))
		
		&& (!CallMAC(hipMemcpy(Xhost.p_tri_corner_index, p_tri_corner_index, Ntris * sizeof(LONG3), hipMemcpyDeviceToHost)))
		&& (!CallMAC(hipMemcpy(Xhost.p_tri_periodic_corner_flags, p_tri_periodic_corner_flags, Ntris * sizeof(CHAR4), hipMemcpyDeviceToHost)))
		&& (!CallMAC(hipMemcpy(Xhost.p_tri_neigh_index, p_tri_neigh_index, Ntris * sizeof(LONG3), hipMemcpyDeviceToHost)))
		&& (!CallMAC(hipMemcpy(Xhost.p_tri_periodic_neigh_flags, p_tri_periodic_neigh_flags, Ntris * sizeof(CHAR4), hipMemcpyDeviceToHost)))

		&& (!CallMAC(hipMemcpy(Xhost.p_who_am_I_to_corner, p_who_am_I_to_corner, Ntris * sizeof(LONG3), hipMemcpyDeviceToHost)))

		&& (!CallMAC(hipMemcpy(Xhost.p_iVolley, p_iVolley, Nverts * sizeof(char), hipMemcpyDeviceToHost)))

		&& (!CallMAC(hipMemcpy(Xhost.p_n_major, p_n_major, Nverts * sizeof(nvals), hipMemcpyDeviceToHost)))
		&& (!CallMAC(hipMemcpy(Xhost.p_n_minor, p_n_minor, Nminor * sizeof(nvals), hipMemcpyDeviceToHost)))
		&& (!CallMAC(hipMemcpy(Xhost.p_T_minor, p_T_minor, Nminor * sizeof(T3), hipMemcpyDeviceToHost)))

		&& (!CallMAC(hipMemcpy(Xhost.p_AAdot, p_AAdot, Nminor * sizeof(AAdot), hipMemcpyDeviceToHost)))

		&& (!CallMAC(hipMemcpy(Xhost.p_v_n, p_v_n, Nminor * sizeof(f64_vec3), hipMemcpyDeviceToHost)))
		&& (!CallMAC(hipMemcpy(Xhost.p_vie, p_vie, Nminor * sizeof(v4), hipMemcpyDeviceToHost)))
		&& (!CallMAC(hipMemcpy(Xhost.p_B, p_B, Nminor * sizeof(f64_vec3), hipMemcpyDeviceToHost)))

		&& (!CallMAC(hipMemcpy(Xhost.p_Lap_Az, p_Lap_Az, Nminor * sizeof(f64), hipMemcpyDeviceToHost)))
		&& (!CallMAC(hipMemcpy(Xhost.p_v_overall_minor, p_v_overall_minor, Nminor * sizeof(f64_vec2), hipMemcpyDeviceToHost)))

		&& (!CallMAC(hipMemcpy(Xhost.p_AreaMinor, p_AreaMinor, Nminor * sizeof(f64), hipMemcpyDeviceToHost)))
		&& (!CallMAC(hipMemcpy(Xhost.p_AreaMajor, p_AreaMajor, Nverts * sizeof(f64), hipMemcpyDeviceToHost)))
		)
	{
		// success - do nothing
	}
	else {
		printf("hipMemcpy error");
		getch();
	}
	Call(hipDeviceSynchronize(), "hipDeviceSynchronize cuSyst::SendToHost");

}
void cuSyst::SendToDevice(cuSyst & Xdevice)
{
	//printf("Value sending to device [92250 * 6 + 1]: %d", p_szPBC_triminor[92250 * 6 + 1]);
	//getch();

	// We are going to need a host-allocated cuSyst in order to
	// do the populating basically.
	if (
		   (!CallMAC(hipMemcpy(Xdevice.p_info, p_info, Nminor * sizeof(structural), hipMemcpyHostToDevice)))

		&& (!CallMAC(hipMemcpy(Xdevice.p_izTri_vert, p_izTri_vert, Nverts*MAXNEIGH_d * sizeof(long), hipMemcpyHostToDevice)))
		&& (!CallMAC(hipMemcpy(Xdevice.p_izNeigh_vert, p_izNeigh_vert, Nverts*MAXNEIGH_d * sizeof(long), hipMemcpyHostToDevice)))
		&& (!CallMAC(hipMemcpy(Xdevice.p_szPBCtri_vert, p_szPBCtri_vert, Nverts*MAXNEIGH_d * sizeof(char), hipMemcpyHostToDevice)))
		&& (!CallMAC(hipMemcpy(Xdevice.p_szPBCneigh_vert, p_szPBCneigh_vert, Nverts*MAXNEIGH_d * sizeof(char), hipMemcpyHostToDevice)))

		&& (!CallMAC(hipMemcpy(Xdevice.p_izNeigh_TriMinor, p_izNeigh_TriMinor, Ntris * 6 * sizeof(long), hipMemcpyHostToDevice)))
		&& (!CallMAC(hipMemcpy(Xdevice.p_szPBC_triminor, p_szPBC_triminor, Ntris * 6 * sizeof(char), hipMemcpyHostToDevice)))
		&& (!CallMAC(hipMemcpy(Xdevice.p_tri_corner_index, p_tri_corner_index, Ntris * sizeof(LONG3), hipMemcpyHostToDevice)))
		&& (!CallMAC(hipMemcpy(Xdevice.p_tri_periodic_corner_flags, p_tri_periodic_corner_flags, Ntris * sizeof(CHAR4), hipMemcpyHostToDevice)))
		&& (!CallMAC(hipMemcpy(Xdevice.p_tri_neigh_index, p_tri_neigh_index, Ntris * sizeof(LONG3), hipMemcpyHostToDevice)))
		&& (!CallMAC(hipMemcpy(Xdevice.p_tri_periodic_neigh_flags, p_tri_periodic_neigh_flags, Ntris * sizeof(CHAR4), hipMemcpyHostToDevice)))
		&& (!CallMAC(hipMemcpy(Xdevice.p_who_am_I_to_corner, p_who_am_I_to_corner, Ntris * sizeof(LONG3), hipMemcpyHostToDevice)))

		&& (!CallMAC(hipMemcpy(Xdevice.p_iVolley, p_iVolley, Nverts * sizeof(char), hipMemcpyHostToDevice)))

		&& (!CallMAC(hipMemcpy(Xdevice.p_n_major, p_n_major, Nverts * sizeof(nvals), hipMemcpyHostToDevice)))
		&& (!CallMAC(hipMemcpy(Xdevice.p_n_minor, p_n_minor, Nminor * sizeof(nvals), hipMemcpyHostToDevice)))
		&& (!CallMAC(hipMemcpy(Xdevice.p_T_minor, p_T_minor, Nminor * sizeof(T3), hipMemcpyHostToDevice)))

		&& (!CallMAC(hipMemcpy(Xdevice.p_AAdot, p_AAdot, Nminor * sizeof(AAdot), hipMemcpyHostToDevice)))

		&& (!CallMAC(hipMemcpy(Xdevice.p_v_n, p_v_n, Nminor * sizeof(f64_vec3), hipMemcpyHostToDevice)))
		&& (!CallMAC(hipMemcpy(Xdevice.p_vie, p_vie, Nminor * sizeof(v4), hipMemcpyHostToDevice)))
		&& (!CallMAC(hipMemcpy(Xdevice.p_B, p_B, Nminor * sizeof(f64_vec3), hipMemcpyHostToDevice)))

		&& (!CallMAC(hipMemcpy(Xdevice.p_Lap_Az, p_Lap_Az, Nminor * sizeof(f64), hipMemcpyHostToDevice)))
		&& (!CallMAC(hipMemcpy(Xdevice.p_v_overall_minor, p_v_overall_minor, Nminor * sizeof(f64_vec2), hipMemcpyHostToDevice)))

		&& (!CallMAC(hipMemcpy(Xdevice.p_AreaMinor, p_AreaMinor, Nminor * sizeof(f64), hipMemcpyHostToDevice)))
		&& (!CallMAC(hipMemcpy(Xdevice.p_AreaMajor, p_AreaMajor, Nverts * sizeof(f64), hipMemcpyHostToDevice)))

		)
	{

	}
	else {
		printf("SendToDevice error"); getch();
	}
	Call(hipDeviceSynchronize(), "hipDeviceSynchronize cuSyst::SendToHost");
}


void cuSyst::ReportDifferencesHost(cuSyst &X2)
{
	long iMinor, iVertex, iTri;

	printf("\nDIFFERENCES:\n");
	for (iMinor = 0; iMinor < Nminor; iMinor++)
	{
		if (X2.p_info[iMinor].flag != p_info[iMinor].flag) printf("%d flag %d %d \n",iMinor, X2.p_info[iMinor].flag, p_info[iMinor].flag);
		if (X2.p_info[iMinor].pos.x != p_info[iMinor].pos.x) printf("%d pos %1.9E %1.9E \n", iMinor, X2.p_info[iMinor].pos.x, p_info[iMinor].pos.x);
	}
	for (iVertex = 0; iVertex < Nverts; iVertex++)
	{
		if (memcmp(X2.p_izTri_vert + iVertex*MAXNEIGH_d, p_izTri_vert + iVertex*MAXNEIGH_d, MAXNEIGH_d * sizeof(long)) != 0) {
			printf("vertex %d izTri_vert \n", iVertex);
		}
		if (memcmp(X2.p_izNeigh_vert + iVertex*MAXNEIGH_d, p_izNeigh_vert + iVertex*MAXNEIGH_d, MAXNEIGH_d * sizeof(long)) != 0) {
			printf("vertex %d izNeigh_vert \n", iVertex);
		}
		if (memcmp(X2.p_szPBCtri_vert + iVertex*MAXNEIGH_d, p_szPBCtri_vert + iVertex*MAXNEIGH_d, MAXNEIGH_d * sizeof(char)) != 0) {
			printf("vertex %d szPBCtri_vert \n", iVertex);
		}
		if (memcmp(X2.p_szPBCneigh_vert + iVertex*MAXNEIGH_d, p_szPBCneigh_vert + iVertex*MAXNEIGH_d, MAXNEIGH_d * sizeof(char)) != 0) {
			printf("vertex %d szPBCneigh_vert \n", iVertex);
		}
	};

	for (iTri = 0; iTri < Ntris; iTri++)
	{
		if (memcmp(X2.p_izNeigh_TriMinor + iTri*6, p_izNeigh_TriMinor + iTri*6, 6 * sizeof(long)) != 0) {
			printf("tri %d izNeigh_TriMinor \n", iTri);
		}
		if (memcmp(X2.p_szPBC_triminor + iTri * 6, p_szPBC_triminor + iTri * 6, 6 * sizeof(char)) != 0) {
			printf("tri %d szPBC_triminor \n", iTri);
		}
		if (memcmp(X2.p_tri_corner_index + iTri, p_tri_corner_index + iTri, sizeof(LONG3)) != 0) {
			printf("tri %d p_tri_corner_index %d %d %d | %d %d %d \n", iTri,
				(X2.p_tri_corner_index + iTri)->i1, (X2.p_tri_corner_index + iTri)->i2, (X2.p_tri_corner_index + iTri)->i3,
				(p_tri_corner_index + iTri)->i1, (p_tri_corner_index + iTri)->i2, (p_tri_corner_index + iTri)->i3);
		}
		if (memcmp(X2.p_tri_periodic_corner_flags + iTri, p_tri_periodic_corner_flags + iTri, sizeof(CHAR4)) != 0) {
			printf("tri %d p_tri_periodic_corner_flags \n", iTri);
		}
		if (memcmp(X2.p_tri_neigh_index + iTri, p_tri_neigh_index + iTri, sizeof(LONG3)) != 0) {
			printf("tri %d p_tri_neigh_index \n", iTri);
		}
		if (memcmp(X2.p_tri_periodic_neigh_flags + iTri, p_tri_periodic_neigh_flags + iTri, sizeof(CHAR4)) != 0) {
			printf("tri %d p_tri_periodic_neigh_flags \n", iTri);
		}
		if (memcmp(X2.p_who_am_I_to_corner + iTri, p_who_am_I_to_corner + iTri, sizeof(LONG3)) != 0) {
			printf("tri %d p_who_am_I_to_corner \n", iTri);
		}
	};
	for (iVertex = 0; iVertex < Nverts; iVertex++)
	{
		if (memcmp(X2.p_n_major + iVertex, p_n_major + iVertex, sizeof(nvals)) != 0) {
			printf("n_major %d %1.10E %1.10E \n", iVertex, (X2.p_n_major + iVertex)->n, (p_n_major + iVertex)->n);
		}
	};
	for (iMinor = 0; iMinor < Nminor; iMinor++)
	{
		if (memcmp(X2.p_n_minor + iMinor, p_n_minor + iMinor, sizeof(nvals)) != 0) {
			printf("n_minor %d %1.10E %1.10E \n", iMinor, (X2.p_n_minor + iMinor)->n, (p_n_minor + iMinor)->n);
		} // hmm
		if (memcmp(X2.p_T_minor + iMinor, p_T_minor + iMinor, sizeof(T3)) != 0) {
			printf("T_minor %d %1.10E %1.10E \n", iMinor, (X2.p_T_minor + iMinor)->Te, (p_T_minor + iMinor)->Te);
		} 
		if (memcmp(X2.p_AAdot + iMinor, p_AAdot + iMinor, sizeof(AAdot)) != 0) {
			printf("AAdot %d %1.10E %1.10E \n", iMinor, (X2.p_AAdot + iMinor)->Azdot, (p_AAdot + iMinor)->Azdot);
		}
		if (memcmp(X2.p_v_n + iMinor, p_v_n + iMinor, sizeof(f64_vec3)) != 0) {
			printf("v_n %d %1.10E %1.10E \n", iMinor, (X2.p_v_n + iMinor)->x, (p_v_n + iMinor)->x);
		}
		if (memcmp(X2.p_vie + iMinor, p_vie + iMinor, sizeof(v4)) != 0) {
			printf("vie %d %1.10E %1.10E \n", iMinor, (X2.p_vie + iMinor)->vez, (p_vie + iMinor)->vez);
		}
		if (memcmp(X2.p_B + iMinor, p_B + iMinor, sizeof(f64_vec3)) != 0) {
			printf("B %d %1.10E %1.10E \n", iMinor, (X2.p_B + iMinor)->x, (p_B + iMinor)->x);
		}
		if (memcmp(X2.p_AreaMinor + iMinor, p_AreaMinor + iMinor, sizeof(f64)) != 0) {
			printf("AreaMinor %d %1.10E %1.10E \n", iMinor, *(X2.p_AreaMinor + iMinor), *(p_AreaMinor + iMinor));
		}
	}
	printf("Difference detection done! \n\n");

}

void cuSyst::Output(const char * filename)
{
	FILE * fp = fopen(filename, "w");
	if (fp != 0) {

		long i;
		for (i = 0; i < NUMVERTICES; i++)
		{
			fprintf(fp, "izTri %d : %d %d %d %d %d %d \n",
				i, p_izTri_vert[MAXNEIGH*i + 0],
				p_izTri_vert[MAXNEIGH*i + 1],
				p_izTri_vert[MAXNEIGH*i + 2],
				p_izTri_vert[MAXNEIGH*i + 3],
				p_izTri_vert[MAXNEIGH*i + 4],
				p_izTri_vert[MAXNEIGH*i + 5]);
		}
		for (i = 0; i < NMINOR; i++)
		{
			fprintf(fp, "%d pos %1.14E %1.14E T %1.14E %1.14E %1.14E n %1.14E vxy %1.14E %1.14E vez %1.14E\n",
				i, p_info[i].pos.x, p_info[i].pos.y, p_T_minor[i].Tn, p_T_minor[i].Ti, p_T_minor[i].Te,
				p_n_minor[i].n, p_vie[i].vxy.x, p_vie[i].vxy.y, p_vie[i].vez );
		}
		fclose(fp);
	}
	else {
		printf("file error: cannot open %s ..\n", filename);
	}
}
void cuSyst::PopulateFromTriMesh(TriMesh * pX)
{
	// AsSUMES THIS cuSyst has been allocated on the host.
	// USES pTri->cent

	// Variables on host are called TriMinorNeighLists and TriMinorPBCLists
	memcpy(p_izNeigh_TriMinor, pX->TriMinorNeighLists, Ntris * 6 * sizeof(long)); // pointless that we duplicate it but nvm
	memcpy(p_szPBC_triminor, pX->TriMinorPBCLists, Ntris * 6 * sizeof(char));

	if ((Nverts != pX->numVertices) ||
		(Ntris != pX->numTriangles))
	{
		printf("ERROR (nVerts %d != pX->numVertices %d) || (nTris != pX->numTriangles)\n",
			Nverts, pX->numVertices);
		getch();
		return;
	}

	plasma_data data;
	long iMinor;
	for (iMinor = 0; iMinor < NMINOR; iMinor++)
	{
		memcpy(&data, &(pX->pData[iMinor]), sizeof(plasma_data));
		p_n_minor[iMinor].n = data.n;
		p_n_minor[iMinor].n_n = data.n_n;
		if (iMinor >= BEGINNING_OF_CENTRAL) {
			p_n_major[iMinor - BEGINNING_OF_CENTRAL].n = data.n;
			p_n_major[iMinor - BEGINNING_OF_CENTRAL].n_n = data.n_n;

		}
		p_T_minor[iMinor].Tn = data.Tn;
		p_T_minor[iMinor].Ti = data.Ti;
		p_T_minor[iMinor].Te = data.Te;
		p_AAdot[iMinor].Az = data.Az;
		p_AAdot[iMinor].Azdot = data.Azdot;
		p_v_n[iMinor] = data.v_n;
		p_vie[iMinor].vxy = data.vxy;
		p_vie[iMinor].vez = data.vez;
		p_vie[iMinor].viz = data.viz;

		//if (iMinor == 25964 - BEGINNING_OF_CENTRAL) {
		//	printf("iMinor %d p_vie[iMinor].vez %1.10E viz %1.10E\n", iMinor, p_vie[iMinor].vez, p_vie[iMinor].viz);
		////	getch();
		//}

		p_B[iMinor] = data.B;
		p_AreaMinor[iMinor] = pX->AreaMinorArray[iMinor];
	}
	
	pX->SetupMajorPBCTriArrays();
	// AreaMajor??? pVertex->AreaCell?
	Vertex * pVertex;
	pVertex = pX->X;
	long izTri[MAXNEIGH],izNeigh[MAXNEIGH];
	char szPBCtri[MAXNEIGH], szPBCneigh[MAXNEIGH];
	short tri_len, neigh_len;
	long iVertex;
	short i;
	structural info;
	for (iVertex = 0; iVertex < Nverts; iVertex++)
	{
		
		tri_len = pVertex->GetTriIndexArray(izTri);
		info.neigh_len = tri_len;
		memset(izTri+tri_len, 0, sizeof(long)*(MAXNEIGH-tri_len));
		memcpy(p_izTri_vert + iVertex*MAXNEIGH, izTri, sizeof(long)*MAXNEIGH);

		neigh_len = pVertex->GetNeighIndexArray(izNeigh);
		memset(izNeigh + neigh_len, 0, sizeof(long)*(MAXNEIGH - neigh_len));
		memcpy(p_izNeigh_vert + iVertex*MAXNEIGH,izNeigh, sizeof(long)*MAXNEIGH);
		
		// For INNERMOST, tri_len != neigh_len. 5 tris inc frills, 4 neighs.
		
		// PB lists:
		memset(szPBCtri + tri_len, 0, sizeof(char)*(MAXNEIGH - tri_len));
		memcpy(szPBCtri, pX->MajorTriPBC[iVertex], sizeof(char)*tri_len);
		memcpy(p_szPBCtri_vert + iVertex*MAXNEIGH, szPBCtri, sizeof(char)*MAXNEIGH);
		
		memset(szPBCneigh, 0, sizeof(char)*MAXNEIGH);
		for (i = 0; i < neigh_len; i++)
		{
			if ((pX->T + izTri[i])->periodic == 0) {
				// do nothing: neighbour must be contiguous
				// tris >= neighs
			} else {
				if (((pX->X + izNeigh[i])->pos.x > 0.0) && (pVertex->pos.x < 0.0))
					szPBCneigh[i] = ROTATE_ME_ANTICLOCKWISE;
				if (((pX->X + izNeigh[i])->pos.x < 0.0) && (pVertex->pos.x > 0.0))
					szPBCneigh[i] = ROTATE_ME_CLOCKWISE;
			};
		}
		memcpy(p_szPBCneigh_vert + iVertex*MAXNEIGH, szPBCneigh, sizeof(char)*MAXNEIGH);
		info.flag = pVertex->flags;
		info.pos = pVertex->pos;
		p_info[iVertex + BEGINNING_OF_CENTRAL] = info;

		p_iVolley[iVertex] = (char)(pVertex->iVolley);
		++pVertex;
	};
	

	long iTri; 
	// Triangle structural?
	Triangle * pTri = pX->T;
	for (iTri = 0; iTri < Ntris; iTri++)
	{
		LONG3 tri_corner_index;
		CHAR4 tri_periodic_corner_flags;
		LONG3 who_am_I_to_corner;
		LONG3 tri_neigh_index;
		CHAR4 tri_periodic_neigh_flags;

		tri_corner_index.i1 = pTri->cornerptr[0] - pX->X;
		tri_corner_index.i2 = pTri->cornerptr[1] - pX->X;
		tri_corner_index.i3 = pTri->cornerptr[2] - pX->X;
		p_tri_corner_index[iTri] = tri_corner_index;
		tri_neigh_index.i1 = pTri->neighbours[0] - pX->T;
		tri_neigh_index.i2 = pTri->neighbours[1] - pX->T;
		tri_neigh_index.i3 = pTri->neighbours[2] - pX->T;
		p_tri_neigh_index[iTri] = tri_neigh_index;

		tri_len = pTri->cornerptr[0]->GetTriIndexArray(izTri);
		for (i = 0; i < tri_len; i++)
		{
			if (izTri[i] == iTri) who_am_I_to_corner.i1 = i;
		}
		tri_len = pTri->cornerptr[1]->GetTriIndexArray(izTri);
		for (i = 0; i < tri_len; i++)
		{
			if (izTri[i] == iTri) who_am_I_to_corner.i2 = i;
		}
		tri_len = pTri->cornerptr[2]->GetTriIndexArray(izTri);
		for (i = 0; i < tri_len; i++)
		{
			if (izTri[i] == iTri) who_am_I_to_corner.i3 = i;
		}
		p_who_am_I_to_corner[iTri] = who_am_I_to_corner;
		
		memset(&tri_periodic_corner_flags, 0, sizeof(CHAR4));
		tri_periodic_corner_flags.flag = pTri->u8domain_flag;
		if (pTri->periodic != 0) {
			if (pTri->cornerptr[0]->pos.x > 0.0) tri_periodic_corner_flags.per0 = ROTATE_ME_ANTICLOCKWISE;
			if (pTri->cornerptr[1]->pos.x > 0.0) tri_periodic_corner_flags.per1 = ROTATE_ME_ANTICLOCKWISE;
			if (pTri->cornerptr[2]->pos.x > 0.0) tri_periodic_corner_flags.per2 = ROTATE_ME_ANTICLOCKWISE;
		}
		p_tri_periodic_corner_flags[iTri] = tri_periodic_corner_flags;
				
		memset(&tri_periodic_neigh_flags, 0, sizeof(CHAR4));
		tri_periodic_neigh_flags.flag = pTri->u8domain_flag;
		if ((pTri->periodic == 0) && (pTri->cent.x > 0.0)) {
			if (pTri->neighbours[0]->periodic != 0) 
				tri_periodic_neigh_flags.per0 = ROTATE_ME_CLOCKWISE;
			if (pTri->neighbours[1]->periodic != 0)
				tri_periodic_neigh_flags.per1 = ROTATE_ME_CLOCKWISE;
			if (pTri->neighbours[2]->periodic != 0)
				tri_periodic_neigh_flags.per2 = ROTATE_ME_CLOCKWISE;
		} else {
			// if we are NOT periodic but on left, neighs are not rotated rel to us.
			// If we ARE periodic but neigh is not and neigh cent > 0.0 then it is rotated.
			if (pTri->periodic != 0) {
				if ((pTri->neighbours[0]->periodic == 0) && (pTri->neighbours[0]->cent.x > 0.0))
					tri_periodic_neigh_flags.per0 = ROTATE_ME_ANTICLOCKWISE;
				if ((pTri->neighbours[1]->periodic == 0) && (pTri->neighbours[1]->cent.x > 0.0))
					tri_periodic_neigh_flags.per1 = ROTATE_ME_ANTICLOCKWISE;
				if ((pTri->neighbours[2]->periodic == 0) && (pTri->neighbours[2]->cent.x > 0.0))
					tri_periodic_neigh_flags.per2 = ROTATE_ME_ANTICLOCKWISE;
			}
		}
		p_tri_periodic_neigh_flags[iTri] = tri_periodic_neigh_flags;
		info.pos = pTri->cent;
		info.flag = pTri->u8domain_flag;
		info.neigh_len = 6;
		p_info[iTri] = info;
		++pTri;
	};
	
}

void cuSyst::CopyStructuralDetailsFrom(cuSyst & src) // this assume both live on device
{
	// info contains flag .... do we know that?
	hipMemcpy(p_info, src.p_info, sizeof(structural)*NMINOR, hipMemcpyDeviceToDevice);
	hipMemcpy(p_izTri_vert, src.p_izTri_vert, sizeof(long)*MAXNEIGH*Nverts, hipMemcpyDeviceToDevice);
	hipMemcpy(p_izNeigh_vert, src.p_izNeigh_vert, sizeof(long)*MAXNEIGH*Nverts, hipMemcpyDeviceToDevice);
	hipMemcpy(p_szPBCtri_vert, src.p_szPBCtri_vert, sizeof(char)*MAXNEIGH*Nverts, hipMemcpyDeviceToDevice);
	hipMemcpy(p_szPBCneigh_vert, src.p_szPBCneigh_vert, sizeof(char)*MAXNEIGH*Nverts, hipMemcpyDeviceToDevice);
	hipMemcpy(p_izNeigh_TriMinor, src.p_izNeigh_TriMinor, sizeof(long)*6*Ntris, hipMemcpyDeviceToDevice);
	hipMemcpy(p_szPBC_triminor, src.p_szPBC_triminor, sizeof(char)*6*Ntris, hipMemcpyDeviceToDevice);
	hipMemcpy(p_tri_corner_index, src.p_tri_corner_index, sizeof(LONG3) * Ntris, hipMemcpyDeviceToDevice);
	hipMemcpy(p_tri_periodic_corner_flags, src.p_tri_periodic_corner_flags, sizeof(CHAR4) * Ntris, hipMemcpyDeviceToDevice);
	hipMemcpy(p_tri_neigh_index, src.p_tri_neigh_index, sizeof(LONG3) * Ntris, hipMemcpyDeviceToDevice);
	hipMemcpy(p_tri_periodic_neigh_flags, src.p_tri_periodic_neigh_flags, sizeof(CHAR4) * Ntris, hipMemcpyDeviceToDevice);
	hipMemcpy(p_who_am_I_to_corner, src.p_who_am_I_to_corner, sizeof(LONG3) * Ntris, hipMemcpyDeviceToDevice);
	hipMemcpy(p_iVolley, src.p_iVolley, sizeof(char)*Nverts, hipMemcpyDeviceToDevice);
	// find another way would be better. Just a waste of memory and processing having duplicate info, creates unnecessary risks.
}

void cuSyst::PopulateTriMesh(TriMesh * pX)
{
	// AsSUMES THIS cuSyst has been allocated on the host.

	long izTri[MAXNEIGH], izNeigh[MAXNEIGH];
	char szPBCtri[MAXNEIGH], szPBCneigh[MAXNEIGH];
	short tri_len, neigh_len;

	plasma_data data;
	long iMinor;
	for (iMinor = 0; iMinor < NMINOR; iMinor++)
	{
		data.n = p_n_minor[iMinor].n ;
		data.n_n = p_n_minor[iMinor].n_n ;
		if (iMinor >= BEGINNING_OF_CENTRAL) {
			data.n = p_n_major[iMinor - BEGINNING_OF_CENTRAL].n ;
			data.n_n = p_n_major[iMinor - BEGINNING_OF_CENTRAL].n_n ;
		} 
		data.Tn = p_T_minor[iMinor].Tn;
		data.Ti = p_T_minor[iMinor].Ti ;
		data.Te = p_T_minor[iMinor].Te ;
		data.Az = p_AAdot[iMinor].Az ;
		data.Azdot = p_AAdot[iMinor].Azdot ;
		data.v_n = p_v_n[iMinor] ;
		data.vxy = p_vie[iMinor].vxy;
		data.vez = p_vie[iMinor].vez;
		data.viz = p_vie[iMinor].viz ;
		data.B = p_B[iMinor] ;
		
		data.pos = p_info[iMinor].pos;

		memcpy(&(pX->pData[iMinor]), &data, sizeof(plasma_data));
		pX->AreaMinorArray[iMinor] = p_AreaMinor[iMinor];
	};

	// UNTIL we go the whole hog and change graphs to cuSyst.

	structural info;
	long iTri, iVertex;

	Vertex * pVertex = pX->X;
	for (iVertex = 0; iVertex < Nverts; iVertex++)
	{
		info = p_info[iVertex + BEGINNING_OF_CENTRAL];
		pVertex->pos = info.pos;
		pVertex->flags = info.flag;

		//tri_len = pVertex->GetTriIndexArray(izTri);
		//info.neigh_len = tri_len;
		//memset(izTri + tri_len, 0, sizeof(long)*(MAXNEIGH - tri_len));
		//memcpy(p_izTri_vert + iVertex*MAXNEIGH, izTri, sizeof(long)*MAXNEIGH);

		memcpy(izTri, p_izTri_vert + iVertex*MAXNEIGH, sizeof(long)*MAXNEIGH);
		pVertex->SetTriIndexArray(izTri, info.neigh_len); // FOR SOME REASON WE PUT == tri_len when we x-ferred????
		//neigh_len = pVertex->GetNeighIndexArray(izNeigh); 
		//memset(izNeigh + neigh_len, 0, sizeof(long)*(MAXNEIGH - neigh_len));
		//memcpy(p_izNeigh_vert + iVertex*MAXNEIGH, izNeigh, sizeof(long)*MAXNEIGH);

		memcpy(izNeigh, p_izNeigh_vert + iVertex*MAXNEIGH, sizeof(long)*MAXNEIGH);
		pVertex->SetNeighIndexArray(izNeigh, info.neigh_len + (((info.flag == INNERMOST) || (info.flag == OUTERMOST))? -1:0));
		// For INNERMOST, tri_len != neigh_len. 5 tris inc frills, 4 neighs.
		
		// PB lists:
		//memcpy(szPBCtri, pX->MajorTriPBC[iVertex], sizeof(char)*tri_len);
		memcpy(szPBCtri, p_szPBCtri_vert + iVertex*MAXNEIGH, sizeof(char)*MAXNEIGH);
		memcpy(pX->MajorTriPBC[iVertex], szPBCtri, sizeof(char)*MAXNEIGH);
		pVertex->iVolley = p_iVolley[iVertex];

		++pVertex;
	}
	
	printf(".....");

	// Triangle structural?
	Triangle * pTri = pX->T;
	for (iTri = 0; iTri < Ntris; iTri++)
	{
		LONG3 tri_corner_index;
		CHAR4 tri_periodic_corner_flags;
		LONG3 who_am_I_to_corner;
		LONG3 tri_neigh_index;
		CHAR4 tri_periodic_neigh_flags;

		tri_corner_index = p_tri_corner_index[iTri];
		pTri->cornerptr[0] = pX->X + tri_corner_index.i1;
		pTri->cornerptr[1] = pX->X + tri_corner_index.i2;
		pTri->cornerptr[2] = pX->X + tri_corner_index.i3;

		tri_neigh_index = p_tri_neigh_index[iTri];
		pTri->neighbours[0] = tri_neigh_index.i1 + pX->T;
		pTri->neighbours[1] = tri_neigh_index.i2 + pX->T;
		pTri->neighbours[2] = tri_neigh_index.i3 + pX->T;
		
		tri_periodic_corner_flags = p_tri_periodic_corner_flags[iTri];
		pTri->periodic = ((tri_periodic_corner_flags.per0 == ROTATE_ME_ANTICLOCKWISE) ? 1 : 0)
			+ ((tri_periodic_corner_flags.per1 == ROTATE_ME_ANTICLOCKWISE) ? 1 : 0)
			+ ((tri_periodic_corner_flags.per2 == ROTATE_ME_ANTICLOCKWISE) ? 1 : 0);
		if (pTri->periodic > 0) pTri->periodic = 3 - pTri->periodic;
		// CPU periodic is how many need to be clockwise rotated.

		info = p_info[iTri];
		pTri->cent = info.pos;
		pTri->u8domain_flag = info.flag;
		++pTri;
	}
	
}
                            
#endif
