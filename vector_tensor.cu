#include "hip/hip_runtime.h"
#ifndef VECTOR_TENSOR_H
#define VECTOR_TENSOR_H

#include "FFxtubes.h"
#include <conio.h>
#include <stdio.h>

// will want to do #include type.h
// for #define real, qd_or_d

#define real double
#define __HIPCC__

int const MAX_TRIS_PER_VERTEX = 12;  
#ifdef __HIPCC__

#define QUALIFIERS __host__ __device__ __forceinline__ 
#define QUALS __host__ __device__ inline
// __host__ == cpu
// __global__ == kernel
// __device__ == call from kernel

#else

#define QUALIFIERS inline
#define QUALS   inline

#endif

struct Vector2
{
    double x, y;  
	
	QUALIFIERS Vector2 (){}

	QUALIFIERS Vector2 (double newx,double newy)
	{
		x = newx; y = newy;
	}

	real QUALIFIERS dot(const Vector2 &v) const
	{
		return x*v.x+y*v.y;
	}


	// NVCC will not put up with forward declaration of friend.
	//friend Vector2 operator* (const real h,const Vector2 &v);
	//friend Vector2 operator* (const Vector2 &v,const real h);
	
	// Have to try naked definition in cpp file of prefix multiply instead.
	// But that's no good for MSVS : the definition/declaration will get 
	// parsed 0 or several times. Better declare just outside class instead.
 
	Vector2 QUALIFIERS operator -() {
		return Vector2(-x,-y); 
	}
	
	Vector2 QUALIFIERS operator -(const Vector2 &v) const
	{
		Vector2 result;
		result.x = x - v.x;
		result.y = y - v.y;
		return result;
	}
	Vector2 QUALIFIERS operator +(const Vector2 &v) const
	{
		Vector2 result;
		result.x = x + v.x;
		result.y = y + v.y;
		return result;
	}

	Vector2 QUALIFIERS operator* (const real hh) const
	{
		Vector2 result;
		result.x = hh*x; result.y = hh*y;
		return result;
	}	

	Vector2 QUALIFIERS operator /(const real h) const
	{
		Vector2 result;
		result.x = x/h;
		result.y = y/h;
		return result;
	}
	void QUALIFIERS operator += (const Vector2 &v) 
	{
		x += v.x;
		y += v.y;
	}
	void QUALIFIERS operator -= (const Vector2 &v)
	{
		x -= v.x;
		y -= v.y;
		// Don't think we should try to use return *this in NVCC. 
	}
	void QUALIFIERS operator *= (const real alpha)
	{
		x *= alpha;
		y *= alpha;
	}
	void QUALIFIERS operator /= (const real alpha)
	{
		x /= alpha;
		y /= alpha;
	}
	real QUALIFIERS modulus() const
	{
		return sqrt(x*x+y*y);
	}

	void QUALIFIERS Normalise()
	{
		real r = modulus();
		x /= r; y /= r;
	}
	
	void QUALIFIERS project_to_ins(Vector2 & result) const
	{
		// If it never gets called, hopefully NVCC won't care what is in that.
		real factor = DEVICE_RADIUS_INSULATOR_OUTER/sqrt(x*x+y*y);
		result.x = x*factor; result.y = y*factor;
	}

	void QUALIFIERS project_to_radius
				(Vector2 & result, real radius)
	{
		real factor = radius/sqrt(x*x+y*y);
		result.x = x*factor; result.y = y*factor;
	}
};

Vector2 QUALS operator* (const real hh,const Vector2 &v)
{
	return Vector2(hh*v.x,hh*v.y);
}

// NOTE: For MSVC, declaring members inline means that 
// they have to be defined in the header - and
// that seems to be okay.
// So we probably should do that with everything here.

struct Vector3
{
	real x,y,z;

	QUALIFIERS Vector3() {};
	QUALIFIERS Vector3(real a, real b, real c)
	{
		x = a; y = b; z = c;
	}

	Vector3 QUALIFIERS operator- () const
	{
		return Vector3 (-x,-y,-z);
	}

	Vector3 QUALIFIERS cross(const Vector3 & v) const
	{
		return Vector3(y*v.z-z*v.y,
			           z*v.x-x*v.z,
					   x*v.y-y*v.x);
	}
	
	Vector3 QUALIFIERS operator +(const Vector3 &v) const
	{
		return Vector3(x+v.x,y+v.y,z+v.z);
	}

	Vector3 QUALIFIERS operator -(const Vector3 &v) const
	{
		return Vector3(x-v.x,y-v.y,z-v.z);
	}

	
	Vector3 QUALIFIERS operator* (const real hh) const
	{
		return Vector3(hh*x,hh*y,hh*z);
	}
	Vector3 QUALIFIERS operator/ (const real hh) const 
	{
		return Vector3(x/hh,y/hh,z/hh);
	}
	
	void QUALIFIERS operator +=(const Vector3 &v) 
	{
		x += v.x; y += v.y; z += v.z;
	}
	void QUALIFIERS operator -=(const Vector3 &v) 
	{
		x -= v.x; y -= v.y; z -= v.z;
	}
	void QUALIFIERS operator *=(const real xx) 
	{
		x *= xx; y *= xx; z *= xx;
	}
	void QUALIFIERS operator /=(const real xx) 
	{
		x /= xx; y /= xx; z /= xx;
	}

	bool QUALIFIERS operator != (const Vector3 &v) const 
	{
		return ((v.x != x) || (v.y != y) || (v.z != z));
	}

	real QUALIFIERS dotxy(const Vector3 &v) const
	{
		return x*v.x+y*v.y;
	}
	real QUALIFIERS dotxy(const Vector2 &v) const
	{
		return x*v.x+y*v.y;
	}
	real QUALIFIERS dot(const Vector2 &v) const
	{
		return x*v.x+y*v.y;
	}
	real QUALIFIERS dot(const Vector3 &v) const
	{
		return x*v.x+y*v.y+z*v.z;
	}

	Vector2 QUALIFIERS xypart() const
	{
		Vector2 u;
		u.x = x; 
		u.y = y;
		return u;
	}

	real QUALIFIERS modulusxy()
	{
		return sqrt(x*x+y*y);
	}
	
	real QUALIFIERS modulus()
	{
		return sqrt(x*x+y*y+z*z);
	}

	void Reflect_radially(Vector2 & centre);
	void ZeroRadially(Vector2 & centre);
};

Vector3 QUALS operator* (const real hh,const Vector3 &v)
{
	return Vector3(hh*v.x,hh*v.y,hh*v.z);
}


real QUALS dotxy(const Vector2 & v1, const Vector3 & v2)
{
	return v1.x*v2.x+v1.y*v2.y;
}
real QUALS dotxy(const Vector3 & v1, const Vector2 & v2)
{
	return v1.x*v2.x+v1.y*v2.y;
}

struct Tensor2
{
	real xx, xy, yx, yy;
	QUALIFIERS Tensor2() {}
	QUALIFIERS Tensor2(real x_x, real x_y, real y_x, real y_y)
	{
		xx = x_x; xy = x_y; yx = y_x; yy = y_y;
	}
	QUALIFIERS ~Tensor2() {}
	QUALIFIERS Tensor2 operator +(const Tensor2 &X) const
	{
		return Tensor2(
			xx + X.xx,
			xy + X.xy,
			yx + X.yx,
			yy + X.yy);
	}

	QUALIFIERS Tensor2 operator *(const Tensor2 &X) const
	{
		Tensor2 result;
		// did a test: X is the one on the right.
		result.xx = xx*X.xx + xy*X.yx;
		result.xy = xx*X.xy + xy*X.yy;
		result.yx = yx*X.xx + yy*X.yx;
		result.yy = yx*X.xy + yy*X.yy;
		return result;
	}

	QUALIFIERS Tensor2 operator *(const real hh) const
	{
		return Tensor2 (hh*xx,hh*xy,hh*yx, hh*yy);
	}

	QUALIFIERS Tensor2 operator -(const Tensor2 &X) const
	{
		return Tensor2(
			xx - X.xx, xy-X.xy, yx-X.yx, yy-X.yy
			);
	}
	
	QUALIFIERS void Inverse(Tensor2 & result) const
	{
		real overdet = 1.0/(xx*yy-xy*yx);
		result.xx = yy*overdet;
		result.xy = -xy*overdet;
		result.yx = -yx*overdet;
		result.yy = xx*overdet;
	};

	QUALIFIERS Vector2 operator *(const Vector2 &v) const
	{
		return Vector2(xx*v.x+xy*v.y,yx*v.x+yy*v.y);
	}
	
	QUALIFIERS void operator +=(const Tensor2 &X)
	{
		xx += X.xx; xy += X.xy;
		yx += X.yx; yy += X.yy;
	}

	QUALIFIERS void operator *=(const real hh) 
	{
		xx *= hh; xy *= hh;
		yx *= hh; yy *= hh;
	}
};
QUALIFIERS Tensor2 operator *(const real hh, const Tensor2 &X) 
{
	return Tensor2(hh*X.xx,hh*X.xy,hh*X.yx,hh*X.yy);
}

extern Tensor2 ID2x2;
extern Tensor2 zero2x2;

struct Tensor3
{
	real xx,xy,xz,yx,yy,yz,zx,zy,zz;
	QUALIFIERS Tensor3() {};

	QUALIFIERS Tensor3(real x_x, real x_y, real x_z, 
		          real y_x, real y_y, real y_z,
				  real z_x, real z_y, real z_z) 
	{
		xx = x_x; xy = x_y; xz = x_z; 
		yx = y_x; yy = y_y; yz = y_z; 
		zx = z_x; zy = z_y; zz = z_z;
	}

	QUALIFIERS void MakeCross (const Vector3 om)
	{
		xx = 0.0;
		xy = -om.z;
		xz = om.y;
		yx = om.z;
		yy = 0.0;
		yz = -om.x;
		zx = -om.y;
		zy = om.x;
		zz = 0.0;
	}
	
	QUALIFIERS Tensor3 Inverse()
	{
		Tensor3 result;
		real det =	  xx*(yy*zz-yz*zy)
					+ xy*(zx*yz-yx*zz)
					+ xz*(yx*zy-yy*zx);
		
		// Fill in matrix of minor determinants; 
		// transposed with applied cofactors (signs)
		
		result.xx = yy*zz-yz*zy;
		result.yx = zx*yz-yx*zz; 
		result.zx = yx*zy-yy*zx;
		result.xy = zy*xz-xy*zz;
		result.yy = xx*zz-xz*zx;
		result.zy = zx*xy-xx*zy;
		result.xz = xy*yz-xz*yy;
		result.yz = yx*xz-xx*yz;
		result.zz = xx*yy-yx*xy;

		if (det != 0.0) {
			result = result / det;
		} else {
			printf("\n\nMATRIX INVERSE FAILED. Det==0\n\n\n");
			memset(&result, 0, sizeof(Tensor3));
			result.xx = 1.0; result.yy = 1.0; result.zz = 1.0;
		}
		return result; // inline so return object doesn't matter
	};

	QUALIFIERS void Inverse(Tensor3 & result)
	{
		real det = (xx*(yy*zz - yz*zy)
			+ xy*(zx*yz - yx*zz)
			+ xz*(yx*zy - yy*zx));

		if (det == 0.0) {
			printf("\n\nMATRIX INVERSE FAILED II. Det == 0\n\n\n");
			return;
		}
		real over =	1.0/det;
		
		// Fill in matrix of minor determinants; 
		// transposed with applied cofactors (signs)
		
		result.xx = (yy*zz-yz*zy)*over;
		result.yx = (zx*yz-yx*zz)*over; 
		result.zx = (yx*zy-yy*zx)*over;
		result.xy = (zy*xz-xy*zz)*over;
		result.yy = (xx*zz-xz*zx)*over;
		result.zy = (zx*xy-xx*zy)*over;
		result.xz = (xy*yz-xz*yy)*over;
		result.yz = (yx*xz-xx*yz)*over;
		result.zz = (xx*yy-yx*xy)*over;

		//return result; // inline so return object doesn't matter
	};

QUALIFIERS Tensor2 xy2x2part () const
	{
		Tensor2 res;
		res.xx = xx;
		res.xy = xy;
		res.yx = yx;
		res.yy = yy;
		return res;
	}

QUALIFIERS Tensor3 operator- () const
	{
		Tensor3 res;
		res.xx = -xx; res.xy = -xy; res.xz = -xz;
		res.yx = -yx; res.yy = -yy; res.yz = -yz;
		res.zx = -zx; res.zy = -zy; res.zz = -zz;
		return res;
	}

QUALIFIERS Vector3 operator* (const Vector3 &v) const
	{
		Vector3 res;
		res.x = xx*v.x + xy*v.y + xz*v.z;
		res.y = yx*v.x + yy*v.y + yz*v.z;
		res.z = zx*v.x + zy*v.y + zz*v.z;
		return res;
	}
	
	
QUALIFIERS Tensor3 operator* (const real hh) const
	{ 
		return Tensor3(
			hh*xx, hh*xy, hh*xz,
			hh*yx, hh*yy, hh*yz,
			hh*zx, hh*zy, hh*zz);
	};
		
QUALIFIERS Tensor3 operator/ (const real r) const
	{
		Tensor3 result;
		// did a test: X is the one on the right.
		result.xx = xx/r;
		result.xy = xy/r;
		result.xz = xz/r;
		result.yx = yx/r;
		result.yy = yy/r;
		result.yz = yz/r;
		result.zx = zx/r;
		result.zy = zy/r;
		result.zz = zz/r;
		return result;
	}

QUALIFIERS Tensor3 operator +(const Tensor3 &v) const
	{
		Tensor3 result;
		result.xx = xx + v.xx;
		result.xy = xy + v.xy;
		result.xz = xz + v.xz;
		result.yx = yx + v.yx;
		result.yy = yy + v.yy;
		result.yz = yz + v.yz;
		result.zx = zx + v.zx;
		result.zy = zy + v.zy;
		result.zz = zz + v.zz;
		return result;
	}

QUALIFIERS Tensor3 operator -(const Tensor3 &v) const
	{
		Tensor3 result;
		result.xx = xx - v.xx;
		result.xy = xy - v.xy;
		result.xz = xz - v.xz;
		result.yx = yx - v.yx;
		result.yy = yy - v.yy;
		result.yz = yz - v.yz;
		result.zx = zx - v.zx;
		result.zy = zy - v.zy;
		result.zz = zz - v.zz;
		return result;
	}
	
QUALIFIERS Tensor3 operator *(const Tensor3 &X) const
	{
		Tensor3 result;
		result.xx = xx*X.xx + xy*X.yx + xz*X.zx;
		result.xy = xx*X.xy + xy*X.yy + xz*X.zy;
		result.xz = xx*X.xz + xy*X.yz + xz*X.zz;
		result.yx = yx*X.xx + yy*X.yx + yz*X.zx;
		result.yy = yx*X.xy + yy*X.yy + yz*X.zy;
		result.yz = yx*X.xz + yy*X.yz + yz*X.zz;
		result.zx = zx*X.xx + zy*X.yx + zz*X.zx;
		result.zy = zx*X.xy + zy*X.yy + zz*X.zy;
		result.zz = zx*X.xz + zy*X.yz + zz*X.zz;
		return result;
	}
	
QUALIFIERS Tensor3 operator +=(const Tensor3 &X) 
	{
		xx += X.xx;
		xy += X.xy;
		xz += X.xz;
		yx += X.yx;
		yy += X.yy;
		yz += X.yz;
		zx += X.zx;
		zy += X.zy;
		zz += X.zz;
		return *this;
	}
	QUALIFIERS Tensor3 operator -=(const Tensor3 &X) 
	{
		xx -= X.xx;
		xy -= X.xy;
		xz -= X.xz;
		yx -= X.yx;
		yy -= X.yy;
		yz -= X.yz;
		zx -= X.zx;
		zy -= X.zy;
		zz -= X.zz;
		return *this;
	}
	
	void Make3DRotationAboutAxis(Vector3 w, real t);
	void spitout(void);
};
QUALIFIERS Tensor3 operator* (const real hh,const Tensor3 &X)
	{		
		Tensor3 result;
		result.xx = hh*X.xx;
		result.xy = hh*X.xy;
		result.xz = hh*X.xz;
		result.yx = hh*X.yx;
		result.yy = hh*X.yy;
		result.yz = hh*X.yz;
		result.zx = hh*X.zx;
		result.zy = hh*X.zy;
		result.zz = hh*X.zz;
		return result;
	}

// Not clear to me : do we want the following for NVCC to be here?
// It actually makes sense to keep "matrix" here!

struct Matrix3
{
	real a[3][3];

	QUALIFIERS void Inverse(Matrix3 & result)
	{
		// find+replace on the above

		real det =	  a[0][0]*(a[1][1]*a[2][2]-a[1][2]*a[2][1])
					+ a[0][1]*(a[2][0]*a[1][2]-a[1][0]*a[2][2])
					+ a[0][2]*(a[1][0]*a[2][1]-a[1][1]*a[2][0]);

		// Fill in matrix of minor determinants; 
		// transposed with applied cofactors (signs)
	
		result.a[0][0] = a[1][1]*a[2][2]-a[1][2]*a[2][1];
		result.a[1][0] = a[2][0]*a[1][2]-a[1][0]*a[2][2]; 
		result.a[2][0] = a[1][0]*a[2][1]-a[1][1]*a[2][0];
		result.a[0][1] = a[2][1]*a[0][2]-a[0][1]*a[2][2];
		result.a[1][1] = a[0][0]*a[2][2]-a[0][2]*a[2][0];
		result.a[2][1] = a[2][0]*a[0][1]-a[0][0]*a[2][1];
		result.a[0][2] = a[0][1]*a[1][2]-a[0][2]*a[1][1];
		result.a[1][2] = a[1][0]*a[0][2]-a[0][0]*a[1][2];
		result.a[2][2] = a[0][0]*a[1][1]-a[1][0]*a[0][1];

		//real * ptr = (real *)(result.a);
		for (int i = 0; i < 3; i++)
		for (int j = 0; j < 3; j++)
		{
			result.a[i][j] /= det; // 99% sure static array elems are contiguous but hey.
		}
	
	};

	QUALIFIERS void multiply(real RHS[3], real output[3])
	{
		output[0] = a[0][0]*RHS[0] + a[0][1]*RHS[1] + a[0][2]*RHS[2];
		output[1] = a[1][0]*RHS[0] + a[1][1]*RHS[1] + a[1][2]*RHS[2];
		output[2] = a[2][0]*RHS[0] + a[2][1]*RHS[1] + a[2][2]*RHS[2];
	};
	
};

extern Tensor3 ID3x3;
extern Tensor3 zero3x3;

struct f64_tens3mag {
	real bx, by, bz, Px, Py, Pz, Hx, Hy, Hz;
};
struct f64_vec3mag {
	real b, P, H;
};
Vector3 QUALS Make3(const Vector2 & v, const real scalar)
{
	Vector3 result;
	result.x = v.x;
	result.y = v.y;
	result.z = scalar;
	return result;
};

// Never used? :
struct Symmetric3
{
	real xx,yy,zz,xy,xz,yz;
	QUALIFIERS Symmetric3() {};
	QUALIFIERS Symmetric3(real x_x, real x_y, real y_y, real x_z, real y_z, real z_z) ;
	
	Vector3 QUALIFIERS operator* (const Vector3 &v) const;
};


// Never used? :
struct Symmetric2
{
	real xx,yy,xy;
};


#define f64 real
#define f64_vec2 Vector2
#define f64_vec3 Vector3
#define f64_tens2 Tensor2
#define f64_tens3 Tensor3
//#define u32 unsigned long


//struct vertinfo
//{
//	long flag;
//	long numTris;
//	f64_vec2 pos; 
//	long iTriIndex[MAX_TRIS_PER_VERTEX]; // 10 x 8
//};
////
//struct structural
//{
//	u32 u32corner[3];                    
//	u32 u32neigh[3];                     // 8x3
//	int iDomain_flag, iPeriodic;       
//	f64_vec2 edge_normal[3];             // 8x6
//	f64_vec2 gradT;                        
//	f64 weight[3]; // weights used for averaging at corners. :/ from CPU
//	f64_vec2 pos; // centroid
//	f64 area;                            // 8x5+8x3
//	// size ~ 24 + 8 + 48 + 40 + 24 
//	// 144 bytes or so
//	f64_vec2 coeff[3]; // for each triangle, the coefficient by which nT_cell creates pressure at vertex.
//	
//	// Demoralised from having to do vertex aggregation so let's bung this in here.
//	// In the next version we can get rid of it.
//};

struct species_f64
{
	f64 neut, ion, elec;
};

struct species_vec2
{
	f64_vec2 neut, ion, elec;
};

struct species_vec3
{
	f64_vec3 neut, ion, elec;
};

struct f64_vec4
{
	f64 x[4];
};
#endif


#undef QUALIFIERS