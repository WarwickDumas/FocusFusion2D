#include "hip/hip_runtime.h"
#define real double
#define f64 double
 
#define HISTORY										4
     
#include <stdlib.h>
#include <stdio.h>
#include "lapacke.h"
  
// we must find out what causes graphics crash during SPECIES_ION

/* Auxiliary routines prototypes */
extern void print_matrix(char* desc, lapack_int m, lapack_int n, double* a, lapack_int lda);
extern void print_int_vector(char* desc, lapack_int n, lapack_int* a);
 
extern void Go_visit_the_other_file();
extern void Setup_residual_array();
      
#include "headers.h"
#include "hip/hip_runtime.h"
#include ""

#include <time.h>
#include <stdio.h>
#include <windows.h>
#include "resource.h"
#include "flags.h"
#include "FFxtubes.h"
//#include "cppconst.h"
#include "cuda_struct.h"
#include "constant.h"
#include "d3d.h"    
#include <d3dx9.h> 
#include <dxerr.h>
#include <commdlg.h>    // probably used by avi_utils
#include "surfacegraph_tri.h"
//#include "avi_utils.cpp"     // for making .avi
#include "kernel.h"

#include <mfapi.h>
#include <mfidl.h>
#include <Mfreadwrite.h>
#include <mferror.h>
#include <iostream>
#include <shlwapi.h>
#include <combaseapi.h>

#pragma comment(lib, "mfreadwrite")
#pragma comment(lib, "mfplat")
#pragma comment(lib, "mf")
#pragma comment(lib, "mfuuid")

template <class T> void SafeRelease(T **ppT)
{
	if (*ppT)
	{
		(*ppT)->Release();
		*ppT = NULL;
	}
}


//=======================================================
// Declarations of functions:
   
void RefreshGraphs(TriMesh & X, const int iGraphsFlag);
LRESULT CALLBACK	WndProc(HWND, UINT, WPARAM, LPARAM);
INT_PTR CALLBACK	About(HWND, UINT, WPARAM, LPARAM);
INT_PTR CALLBACK	SetupBox(HWND, UINT, WPARAM, LPARAM);
extern f64 GetEzShape__(f64 r);
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
extern void Zap_the_back();

extern f64 * temp_array_host;
extern OhmsCoeffs * p_OhmsCoeffs_host;
extern f64 * p_graphdata1_host,* p_graphdata2_host,* p_graphdata3_host, *p_graphdata4_host, *p_graphdata5_host, *p_graphdata6_host;
extern f64 * p_Tgraph_host[9];
extern f64 * p_accelgraph_host[12];
extern f64 * p_Ohmsgraph_host[20];

extern f64 * p_arelz_graph_host[12];

// Global variables:
// =================
//extern f64_vec3 * p_B_host;
extern f64 EzStrength_;
extern cuSyst cuSyst1, cuSyst2, cuSyst3;
extern D3D Direct3D;
extern f64 * p_temphost1, *p_temphost2,
*p_temphost3, *p_temphost4, *p_temphost5, *p_temphost6;
 
extern __device__ f64 * p_LapCoeffself;
extern __device__ f64 * p_temp1;
extern __device__ long * p_longtemp;
extern __device__ f64 * p_Az, *p_LapAz;


float xzscale;

bool bCullNone = false;
bool bGlobalsave = false;
int GlobalSwitchBox = 0;
int iGlobalScratch;

real GlobalHeightScale;

int GlobalSpeciesToGraph = SPECIES_ION;
int GlobalWhichLabels = 0;
bool GlobalRenderLabels = false;
int GlobalColoursPlanView = 0;
bool GlobalBothSystemsInUse;

bool GlobalCutaway = true;

unsigned int cw; // control word for floating point hardware exception hiding

TriMesh * pX, *pXnew;
TriMesh X1, X2, X3, X4;
cuSyst cuSyst_host, cuSyst_host2;

D3DXVECTOR3 GlobalEye, GlobalLookat, GlobalPlanEye, GlobalPlanEye2, GlobalPlanLookat,
GlobalPlanLookat2, GlobalEye2, GlobalLookat2;

D3DXVECTOR3 newEye;
D3DXVECTOR3 newLookat;

IDirect3DSurface9* p_backbuffer_surface;

long steps_remaining, GlobalStepsCounter, steps_remaining_CPU;
real evaltime, h;

extern real GlobalIzElasticity;
FILE * massfile, *maxfile;

// Global Variables:
HINSTANCE hInst;   // current instance
				   // window vars:
HWND hWnd, hwndGraphics;
WNDCLASSEX wcex;
TCHAR szTitle[1024];					// The title bar text
TCHAR szWindowClass[1024];			// the main window class name

char Functionalfilename[1024];
int GlobalGraphSetting[8]; 
surfacegraph Graph[8]; // why was it 5? // 5th one can be whole thing.

float Historic_max[512][HISTORY]; // if max is falling, use historic maximum for graph.
float Historic_min[512][HISTORY];
int Historic_powermax[512];
int Historic_powermin[512]; // just store previous value only.

bool flaglist[NMINOR];

bool boolGlobalHistory, GlobalboolDisplayMeshWireframe;

// avi file -oriented variables
int const NUMAVI = 9;
//HAVI hAvi[NUMAVI + 1]; // does it work without OHMSLAW? //  OHMSLAW,
int const GraphFlags[NUMAVI] = { SPECIES_ION, OVERALL, JZAZBXYEZ, ONE_D, IONIZEGRAPH,
				DTGRAPH, ACCELGRAPHS, OHMS2, ARELZ};

WCHAR szmp4[NUMAVI][128] = { L"Elec",L"Total",L"JzAzBxy",L"Test", 
L"Ionize", L"dT", L"Accel",	L"Ohms", L"arelz"};

//AVICOMPRESSOPTIONS opts;
int counter;
HBITMAP surfbit, dib;
HDC surfdc, dibdc;
LPVOID lpvBits;
BITMAPINFO bitmapinfo;

DWORD dwBits[VIDEO_HEIGHT*VIDEO_WIDTH];

f64 graphdata[20][10000]; 
f64 graph_r[10000];
int numgraphs = 4;
int num_graph_data_points = 10000;
f64 maximum[20];
f64 truemax[20]; 

extern TriMesh * pTriMesh;


char * report_time(int action)
{
	/* action = 0: reset ; action = 1: report */
	/* requires timebuffy to be defined as char[255] globally */

	static char timebuffer[255];
	static clock_t start;
	double timenow;
	long ops;

	if (action == 0)
	{
		start = clock();
	}
	else
	{
		timenow = ((double)(clock() - start) / (double)CLOCKS_PER_SEC);
		ops = (long)(clock() - start);
		/* create a null-terminated string */
		sprintf(timebuffer, "%6.4f sec.", timenow);
	};
	return &(timebuffer[0]);
};

f64 GetTriangleArea(f64_vec2 pos0, f64_vec2 pos1, f64_vec2 pos2)
{
	f64 area = 0.5*((pos0.x + pos1.x)*(pos1.y - pos0.y) + (pos1.x + pos2.x)*(pos2.y - pos1.y)
		+ (pos0.x + pos2.x)*(pos0.y - pos2.y));
	return fabs(area);
}


// Format constants
//const UINT32 VIDEO_WIDTH = 640;
//const UINT32 VIDEO_HEIGHT = 480;
const UINT32 VIDEO_FPS = 5;
const UINT64 VIDEO_FRAME_DURATION = 10 * 1000 * 1000 / VIDEO_FPS; // ?
const UINT32 VIDEO_BIT_RATE = 1048768;
const UINT32 VIDEO_PELS = VIDEO_WIDTH * VIDEO_HEIGHT;
const UINT32 VIDEO_FRAME_COUNT = 5;
// with 50 frames per nanosecond and 30 nanoseconds in file, it's 1500
// But to begin let's say 5?
const GUID   VIDEO_INPUT_FORMAT = MFVideoFormat_RGB24;

HRESULT InitializeSinkWriter(
	IMFSinkWriter **ppWriter, 
	DWORD *pStreamIndex, 
	LPCWSTR szFilename)
{
	*ppWriter = NULL;
	*pStreamIndex = NULL;

	IMFSinkWriter   *pSinkWriter = NULL;
	IMFMediaType    *pMediaTypeOut = NULL;
	IMFMediaType    *pMediaTypeIn = NULL;
	DWORD           streamIndex;
	
	HRESULT hr = MFCreateSinkWriterFromURL(szFilename, NULL, NULL, &pSinkWriter);

	// Set the output media type.
	if (SUCCEEDED(hr)) 
		hr = MFCreateMediaType(&pMediaTypeOut);	
	if (SUCCEEDED(hr))
		hr = pMediaTypeOut->SetGUID(MF_MT_MAJOR_TYPE, MFMediaType_Video);
	if (SUCCEEDED(hr)) 
		hr = pMediaTypeOut->SetGUID(MF_MT_SUBTYPE, MFVideoFormat_H264);
	// whereas webcam capture sample says WMMEDIASUBTYPE_I420
	
	if (SUCCEEDED(hr)) 
		hr = pMediaTypeOut->SetUINT32(MF_MT_AVG_BITRATE, VIDEO_BIT_RATE);
	if (SUCCEEDED(hr)) 
		hr = pMediaTypeOut->SetUINT32(MF_MT_INTERLACE_MODE, MFVideoInterlace_Progressive);
	if (SUCCEEDED(hr)) 
		hr = MFSetAttributeSize(pMediaTypeOut, MF_MT_FRAME_SIZE, VIDEO_WIDTH, VIDEO_HEIGHT);
	if (SUCCEEDED(hr)) 
		hr = MFSetAttributeRatio(pMediaTypeOut, MF_MT_FRAME_RATE, VIDEO_FPS, 1);
	if (SUCCEEDED(hr)) 
		hr = MFSetAttributeRatio(pMediaTypeOut, MF_MT_PIXEL_ASPECT_RATIO, 1, 1);
	if (SUCCEEDED(hr)) 
		hr = pSinkWriter->AddStream(pMediaTypeOut, &streamIndex);
	
	// Set the input media type.
	if (SUCCEEDED(hr)) 
		hr = MFCreateMediaType(&pMediaTypeIn);	
	if (SUCCEEDED(hr)) 
		hr = pMediaTypeIn->SetGUID(MF_MT_MAJOR_TYPE, MFMediaType_Video);	
	if (SUCCEEDED(hr)) 
		hr = pMediaTypeIn->SetGUID(MF_MT_SUBTYPE, VIDEO_INPUT_FORMAT);
	if (SUCCEEDED(hr)) 
		hr = pMediaTypeIn->SetUINT32(MF_MT_INTERLACE_MODE, MFVideoInterlace_Progressive); 
		
	// should that be 0 ? 

	if (SUCCEEDED(hr)) 
		hr = MFSetAttributeSize(pMediaTypeIn, MF_MT_FRAME_SIZE, VIDEO_WIDTH, VIDEO_HEIGHT);
	if (SUCCEEDED(hr)) 
		hr = MFSetAttributeRatio(pMediaTypeIn, MF_MT_FRAME_RATE, VIDEO_FPS, 1);
	if (SUCCEEDED(hr)) 
		hr = MFSetAttributeRatio(pMediaTypeIn, MF_MT_PIXEL_ASPECT_RATIO, 1, 1);
	if (SUCCEEDED(hr)) 
		hr = pSinkWriter->SetInputMediaType(streamIndex, pMediaTypeIn, NULL);
	
	// Tell the sink writer to start accepting data.
	if (SUCCEEDED(hr)) {
		hr = pSinkWriter->BeginWriting();
	}

	// Return the pointer to the caller.
	if (SUCCEEDED(hr)) {
		*ppWriter = pSinkWriter;
		(*ppWriter)->AddRef();
		*pStreamIndex = streamIndex;
	}

	SafeRelease(&pSinkWriter);
	SafeRelease(&pMediaTypeOut);
	SafeRelease(&pMediaTypeIn);
	return hr;
}

HRESULT WriteFrame(
	IMFSinkWriter *pWriter,
	DWORD streamIndex,
	const LONGLONG& rtStart        // Time stamp.
)
{
	IMFSample *pSample = NULL;
	IMFMediaBuffer *pBuffer = NULL;

	// SHOULD THIS BE 3 * ?

	const LONG cbWidth = 3 * VIDEO_WIDTH; // 4 bytes --- why?

	// so cbWidth is width in bytes

	const DWORD cbBuffer = cbWidth * VIDEO_HEIGHT;
	BYTE *pData = NULL;
	// Create a new memory buffer.
	HRESULT hr = MFCreateMemoryBuffer(cbBuffer, &pBuffer);
	// Lock the buffer and copy the video frame to the buffer.
	if (SUCCEEDED(hr))
		hr = pBuffer->Lock(&pData, NULL, NULL);
	
	if (SUCCEEDED(hr))
		hr = MFCopyImage(
			pData,                      // Destination buffer.
			cbWidth,                    // Destination stride.
			(BYTE *)lpvBits,//(BYTE*)videoFrameBuffer,    // First row in source image.
			cbWidth,                    // Source stride.
			cbWidth,                    // Image width in bytes.
			//I added x 3
			VIDEO_HEIGHT                // Image height in pixels.
		);
	
	if (pBuffer) pBuffer->Unlock();
	
	// Set the data length of the buffer.
	if (SUCCEEDED(hr))
		hr = pBuffer->SetCurrentLength(cbBuffer);

	// Create a media sample and add the buffer to the sample.
	if (SUCCEEDED(hr))
		hr = MFCreateSample(&pSample);
	if (SUCCEEDED(hr))
		hr = pSample->AddBuffer(pBuffer);
	// Set the time stamp and the duration.
	if (SUCCEEDED(hr))
		hr = pSample->SetSampleTime(rtStart);
	if (SUCCEEDED(hr))
		hr = pSample->SetSampleDuration(VIDEO_FRAME_DURATION);

	// Send the sample to the Sink Writer.
	if (SUCCEEDED(hr))
		hr = pWriter->WriteSample(streamIndex, pSample);

	SafeRelease(&pSample);
	SafeRelease(&pBuffer);
	return hr;
}


void TriMesh::CalculateTotalGraphingData()
{
	/*long iVertex;
	Vertex * pVertex = X;
	for (iVertex = 0; iVertex < numVertices; iVertex++)
	{
	if ((pVertex->flags == DOMAIN_VERTEX) || (pVertex->flags == OUTERMOST))
	{
	pVertex->n = (pVertex->Neut.mass + pVertex->Ion.mass) / pVertex->AreaCell;
	pVertex->v = (m_n*pVertex->Neut.mom + m_ion * pVertex->Ion.mom + m_e * pVertex->Elec.mom) /
	(m_n*pVertex->Neut.mass + m_ion * pVertex->Ion.mass + m_e * pVertex->Elec.mass);
	pVertex->T = (pVertex->Neut.heat + pVertex->Ion.heat + pVertex->Elec.heat) /
	(pVertex->Neut.mass + pVertex->Ion.mass + pVertex->Elec.mass);
	pVertex->Temp.x = pVertex->Ion.mass / (pVertex->Neut.mass + pVertex->Ion.mass);
	};
	++pVertex;
	}*/
}

void TriMesh::Setup_J()
{
	/*long iVertex;
	Vertex * pVertex = X;
	for (iVertex = 0; iVertex < numVertices; iVertex++)
	{
	if ((pVertex->flags == DOMAIN_VERTEX) || (pVertex->flags == OUTERMOST))
	{
	pVertex->Temp = q * (pVertex->Ion.mom - pVertex->Elec.mom) / pVertex->AreaCell;
	}
	else {
	memset(&(pVertex->Temp), 0, sizeof(Vector3));
	}
	++pVertex;
	}*/
}

void surfacegraph::DrawSurface(const char * szname,
	const int heightflag,
	const real * var_ptr_0,
	const int colourflag,
	const real * var_ptr_c,
	const bool bDisplayInner,
	const int code, // graph code, to pass to called routines - sometimes useful
	const TriMesh * pX // for passing to SetDataWithColour and Render
					   // and for working out offsets
)
{
	// replaced CreateSurfaceGraphs.
	// I think this is about the right balance.

	char buff[256];
	real * temprealptr = (real *)(pX->pData);
	long offset = var_ptr_0 - temprealptr;
	long offset_c = var_ptr_c - temprealptr;

	// Does shader always go with colour type?? yes I think.
	switch (colourflag) {
	case VELOCITY_COLOUR:
		this->mhTech = mFX->GetTechniqueByName("VelociTech");
		break;
	case SEGUE_COLOUR:
		this->mhTech = mFX->GetTechniqueByName("SegueTech");
		break;
	case CURRENT_COLOUR:
		this->mhTech = mFX->GetTechniqueByName("XYZTech");
		break;
	case AZSEGUE_COLOUR:
		mhTech = mFX->GetTechniqueByName("AzSegueTech");
		break;
	case IONISE_COLOUR:
		mhTech = mFX->GetTechniqueByName("IoniseTech");
		break;
	case PPN_COLOUR:
		mhTech = mFX->GetTechniqueByName("ProportionTech"); // 1 = blue
		break;
	};

	// Usual settings:
	//if (GlobalGraphSetting[i] != GRAPH_NONE) {

	this->boolDisplayShadow = true;
	this->boolDisplayMainMesh = true;
	this->boolDisplayMeshWireframe = GlobalboolDisplayMeshWireframe;
	this->boolClearZBufferBeforeWireframe = false;
	// Or try setting true and CULL_CCW to see if this stops it showing "the back of the wireframe"
	this->SetEyeAndLookat(GlobalEye, GlobalLookat);
	this->boolDisplayScales = true;
	this->boolDisplayInnerMesh = bDisplayInner;

	// work out whether to display key button:
	if (((colourflag == FLAG_VELOCITY_COLOUR) || (colourflag == FLAG_CURRENT_COLOUR))
		&& (bDisplayInner == 0))
	{
		this->boolDisplayKeyButton = true;
	}
	else {
		this->boolDisplayKeyButton = false;
	};
	//int const FLAG_COLOUR_MESH = 0;
	//int const FLAG_SEGUE_COLOUR = 1;
	//int const FLAG_VELOCITY_COLOUR = 2;
	//int const FLAG_CURRENT_COLOUR = 3;
	//int const FLAG_AZSEGUE_COLOUR = 4;
	//int const FLAG_IONISE_COLOUR = 5;

	this->SetDataWithColour(*pX,
		colourflag, heightflag, // apparently it's that way round
		offset, offset_c,
		code);

	printf("DrawSurface code %d : calling Render:\n", code);
	if (this->bDisplayTimestamp) {
		sprintf(buff, "%6.2f ns", evaltime*1.0e9);
		this->Render(szname, false, pX, buff);
	}
	else {
		this->Render(szname, false, pX);
	};

}

void Draw1Dgraph(int iWhichGraph, int flag)
{
	float const MAXX = 11.0f;
	float const MAXY = 6.0f;
	float const YADJUST = -2.8f;

	char graphname[4][128] = { "Azdot","Azdotdot","Lap Az","-4pi/c Jz" };
	char Tgraphname[9][128] = { "conduction","ionization","viscosity","frictional","interspecies","dTe/dt total","compressive" ,
		"DnT","undefined" };
	char accelgraphname[9][128] = { "dvy/dt total", "v x B", "pressure", "neutral soak","viscosity", "ionization", "advection","grad_y Az" };
	char Ohmsgraphname[20][128] = { "elastic effective fric coeff", "ionization effective fric coeff",
	"thermal pressure y", "electromotive aez-aiz", "thermal force aiz-aez", "v-response T_zy", "v-response T_zz",
		"T_zy * thermal pressure y", "T_zz * electromotive", "T_zz * thermal force", "Predicted vez-viz",
	"Difference: prediction-vrelzk","vrelzk progress",
	"viscous aez-aiz","Predicted Jz","Conductivity sigma_zy","Conductivity sigma_zz",
	"sigma_zz * -electromotive", "Difference: Jz prediction-Jz","$$$" };

	char arelzgraphname[12][128] = { "arelz", "MAR_ion contribution", "MAR_elec contribution",
	    "Ez_ext effect", "dAz/dt effect","v x B effect", "thermal force effect", "friction to neutrals",
		"friction_ei", "sum of effects", "difference (error)"};


	char buffer[256];
	float x, y, z;
	float zeroplane = 0.0f;
	D3DXMATRIXA16 matWorld;
	vertex1 linedata[10000];
	vertex1 linedata2[12];
	int iGraph;

	D3DCOLOR colourlist[20];
	char namelist[20][256];
	bool bAlternating[20];
	int numgraphs;
	memset(bAlternating, 0, sizeof(bool) * 20);
	if (flag == ONE_D) {
		numgraphs = 4;
		for (int i = 0; i < numgraphs; i++)
			sprintf(namelist[i],"%s   : graph max&min = +- %1.3E", 
				graphname[i], maximum[i]);		
		colourlist[0] = 0xff000000;
		colourlist[1] = 0xff0022ff;
		colourlist[2] = 0xffff0055;
		colourlist[3] = 0xff22ff00;
	};
	if (flag == DTGRAPH) {
		numgraphs = 8;
		for (int i = 0; i < numgraphs; i++)
			sprintf(namelist[i], "%s  : graph max&min = +- %1.3E",
				Tgraphname[i], maximum[i]);
		colourlist[0] = 0xffffaa00; // conduction: orange red
		colourlist[1] = 0xff0000ff; // ionization: royal blue
		colourlist[2] = 0xff009999; // viscosity: aqua
		colourlist[3] = 0xffd500ff; // resistive: heliotrope
		colourlist[4] = 0xff00ff00; // soak: green
		colourlist[5] = 0xff000000; // total
		colourlist[6] = 0xff906545; // compressive: brown
	};
	if (flag == ACCELGRAPHS)
	{
		numgraphs = 8;
		for (int i = 0; i < numgraphs; i++)
			sprintf(namelist[i], "%s   : graph max&min = +- %1.3E",
				accelgraphname[i], maximum[i]);
		colourlist[0] = 0xff000000; // total: 
		colourlist[1] = 0xffd500ff; // vxB: heliotrope
		colourlist[2] = 0xffff2200; // pressure: red
		colourlist[3] = 0xff00ff33; // soak:   use green
		colourlist[4] = 0xff009999; // viscosity: aqua
		colourlist[5] = 0xff0000ff; // ionization : royal blue
		colourlist[6] = 0xff906545; // advection :  brown
		colourlist[7] = 0xffeecd00; // grady_Az : olive?
	}

	if (flag == ARELZ)
	{
		numgraphs = 11;
		for (int i = 0; i < numgraphs; i++)
			sprintf(namelist[i], "%s   : graph max&min = +- %1.3E",
				arelzgraphname[i], maximum[i]);
		colourlist[0] = 0xff000000; // total: 
		colourlist[1] = 0xff009999; // ion visc : aqua
		colourlist[2] = 0xffeecd00; // elec visc : olive
		colourlist[3] = 0xffdada66; // electromotive
		colourlist[4] = 0xff4400ff; // inductive electromotive: indigo
		colourlist[5] = 0xffd500ff; // vxB: heliotrope
		colourlist[6] = 0xffff7700; // "thermal force effect"
		colourlist[7] = 0xff00ff33; // neutral soak :green
		colourlist[8] = 0xff00aa00; // dkgreen e-i friction
		colourlist[9] = 0xffff2299; // pink : sum
		colourlist[10] = 0xff000011; // navy 
		bAlternating[10] = true;		
	}

	if (flag == OHMS2) {
		numgraphs = 11;
		for (int i = 0; i < numgraphs; i++)
			sprintf(namelist[i], "%s :grmax+- %1.3E own|max| %1.3E",
				Ohmsgraphname[i], maximum[i], truemax[i]);
		
		colourlist[0] = 0xffcc0033; // elastic fric coeff: maroon
		colourlist[1] = 0xff00aa00; // dkgreen ionization fric coeff
		colourlist[2] = 0xffff0000; // pressure: red
		colourlist[3] = 0xffda00ff; // electromotive: violet
		colourlist[4] = 0xffff7700; // thermal force: orange
		colourlist[5] = 0xff00aadd; bAlternating[5] = true; // Tzy
		colourlist[6] = 0xff0000ff; // Tzz
		colourlist[7] = 0xffff55aa; bAlternating[7] = true;
		colourlist[8] = 0xffda00ff; bAlternating[8] = true;
		colourlist[9] = 0xffffaa00; bAlternating[9] = true;
		colourlist[10] = 0xff000000;
	};

	if (flag == OHMSLAW) {
		numgraphs = 9;
		for (int i = 0; i < numgraphs; i++)
			sprintf(namelist[i], "%s :grmax+- %1.3E own|max| %1.3E",
				Ohmsgraphname[i+10], maximum[i+10], truemax[i+10]);
		colourlist[0] = 0xff000000;
		colourlist[1] = 0xffff3333; // red: difference
		colourlist[2] = 0xffaadd00; // lime yellow: progress
		colourlist[3] = 0xff009999; // viscosity: aqua
		colourlist[4] = 0xffd500ff; // prediction Jz
		colourlist[5] = 0xff00bb33; // green conductivity
		colourlist[6] = 0xff0022ff; // blue conductivity
		colourlist[7] = 0xffda00ff; bAlternating[7] = true; // sigma_zz Ez
		colourlist[8] = 0xff666666; // difference of prediction
	}

	f64 rmax = GRAPH1D_MAXR;
	if (flag == ONE_D) rmax = DOMAIN_OUTER_RADIUS;
	f64 rmin = DEVICE_RADIUS_INSULATOR_OUTER - 0.01;
	if (flag == ONE_D) rmin = INNER_A_BOUNDARY;


	Graph[iWhichGraph].SetEyeAndLookat(newEye, newLookat); // sets matView not matProj
	printf("Eye %f %f %f\n", newEye.x, newEye.y, newEye.z);
	Direct3D.pd3dDevice->SetViewport(&(Graph[iWhichGraph].vp));

	D3DXMatrixIdentity(&matWorld);
	//D3DXMatrixIdentity(&Graph[6].matProj); // ???????????????
	Direct3D.pd3dDevice->SetTransform(D3DTS_WORLD, &matWorld);
	Direct3D.pd3dDevice->SetTransform(D3DTS_VIEW, &(Graph[iWhichGraph].matView));
	Direct3D.pd3dDevice->SetTransform(D3DTS_PROJECTION, &(Graph[iWhichGraph].matProj));

	Direct3D.pd3dDevice->Clear(0, NULL, D3DCLEAR_TARGET | D3DCLEAR_ZBUFFER,
		D3DCOLOR_XRGB(250, 255, 250), 1.0f, 0);

	if (SUCCEEDED(Direct3D.pd3dDevice->BeginScene()))
	{
		Direct3D.pd3dDevice->SetFVF(point_fvf);

		real theta = -HALFANGLE;
		real r = 3.44;

		linedata[0].x = -MAXX;
		linedata[0].z = 3.44*xzscale;
		linedata[0].y = YADJUST;
		linedata[0].colour = 0xff888888; // grey

		linedata[1].x = -linedata[0].x;
		linedata[1].y = YADJUST;
		linedata[1].z = linedata[0].z;
		linedata[1].colour = linedata[0].colour;

		Direct3D.pd3dDevice->DrawPrimitiveUP(D3DPT_LINESTRIP, 1, linedata, sizeof(vertex1));

			//Graph[6].RenderLabel2(buffer,  // text
			//	MAXX*0.66f + 1.2f*(float)iGraph,
			//	0.0f,
		for (iGraph = 0; iGraph < numgraphs; iGraph++)
		{
			if (iGraph < 6) {
				linedata[0].x = -MAXX;
				linedata[0].z = 3.44*xzscale;
				linedata[0].y = MAXY + 4.0f - 0.9f*(float)iGraph;				
			} else {
				linedata[0].x = 0.8f;
				linedata[0].z = 3.44*xzscale;
				linedata[0].y = MAXY + 4.0f - 0.9f*(float)(iGraph-6);
			}
			linedata[1].x = linedata[0].x + 0.5f;
			linedata[1].y = linedata[0].y;
			linedata[1].z = linedata[0].z;
			linedata[2].x = linedata[0].x + 1.0f;
			linedata[2].y = linedata[0].y;
			linedata[2].z = linedata[0].z;
			linedata[0].colour = colourlist[iGraph];
			linedata[1].colour = linedata[0].colour;
			linedata[2].colour = linedata[0].colour;
			if (bAlternating[iGraph]) linedata[1].colour = 0xffffffff;

			Direct3D.pd3dDevice->DrawPrimitiveUP(D3DPT_LINESTRIP, 2, linedata, sizeof(vertex1));
			
			Graph[iWhichGraph].RenderLabel2(namelist[iGraph], linedata[2].x + 0.1f, linedata[1].y - 0.3f, linedata[1].z, 0,0xff000000, true);

			
			int asdf;
			
			if (flag != OHMSLAW) {
				for (asdf = 0; asdf < num_graph_data_points; asdf++)
				{
					linedata[asdf].x = (float)(MAXX - 2.0*MAXX*((graph_r[asdf] - rmin) /
						(rmax - rmin)));

					// map 0 to 0.0f, maximum[iGraph] to MAXY and -maximum[iGraph] to MINY
					// Decide on graph scales maximum[] in preceding bit of code
					linedata[asdf].y = YADJUST + (float)(MAXY*graphdata[iGraph][asdf] / maximum[iGraph]);
					linedata[asdf].z = 3.44f*xzscale;
					linedata[asdf].colour = colourlist[iGraph];
					if ((bAlternating[iGraph]) && (asdf % 3 == 1)) linedata[asdf].colour = 0xffffffff;
				};
				Direct3D.pd3dDevice->DrawPrimitiveUP(D3DPT_LINESTRIP, num_graph_data_points - 1, linedata, sizeof(vertex1));
			} else {
				for (asdf = 0; asdf < num_graph_data_points; asdf++)
				{
					linedata[asdf].x = (float)(MAXX - 2.0*MAXX*((graph_r[asdf] - rmin) /
						(rmax - rmin)));
					// map 0 to 0.0f, maximum[iGraph] to MAXY and -maximum[iGraph] to MINY
					// Decide on graph scales maximum[] in preceding bit of code
					linedata[asdf].y = YADJUST + (float)(MAXY*graphdata[iGraph+10][asdf] / maximum[iGraph+10]);
					linedata[asdf].z = 3.44f*xzscale;
					linedata[asdf].colour = colourlist[iGraph];
					if ((bAlternating[iGraph]) && (asdf % 3 == 1)) linedata[asdf].colour = 0xffffffff;
				};
				Direct3D.pd3dDevice->DrawPrimitiveUP(D3DPT_LINESTRIP, num_graph_data_points - 1, linedata, sizeof(vertex1));

			};
			//sprintf(buffer, "%2.2E", maximum[iGraph]);
			//Graph[6].RenderLabel2(buffer,  // text
			//	MAXX*0.66f + 1.2f*(float)iGraph,
			//	MAXY,
			//	linedata[0].z, 0, linedata[0].colour);
			//sprintf(buffer
			//	linedata[0].z, 0, linedata[0].colour);
			//sprintf(buffer, "-%2.2E", maximum[iGraph]);
			//Graph[6].RenderLabel2(buffer,  // text
			//	MAXX*0.66f + 1.2f*(float)iGraph,
			//	-MAXY,
			//	linedata[0].z, 0, linedata[0].colour);
		};
		
		// Vertical lines:
		for (int i = 0; i < 9; i++)
		{
			x = 0.16*(-r*xzscale + 2.0*r*xzscale*(((real)i) / 8.0));
			z = 3.44*xzscale;// (float)(cos(HALFANGLE)*DEVICE_RADIUS_INSULATOR_OUTER)*xzscale;

			linedata[0].x = x; linedata[0].z = z;
			linedata[1].x = x; linedata[1].z = z;
			linedata[0].colour = 0xff220011;
			linedata[1].colour = 0xff220011;
			linedata[0].y = -6.8f + YADJUST;// GRAPHIC_MIN_Y - 1.0f;  
			linedata[1].y = YADJUST + (((i == 0) || (i == 8)) ? 6.0f : 0.0f);// GRAPHIC_MAX_Y + 2.5f;

			Direct3D.pd3dDevice->DrawPrimitiveUP(D3DPT_LINESTRIP, 1, linedata, sizeof(vertex1));

			sprintf(buffer, "%5.2f", rmin + (1.0 - ((real)i) / 8.0)*(rmax - rmin));
			Graph[iWhichGraph].RenderLabel2(buffer,  // text
				linedata[0].x,
				YADJUST - 7.6f,
				linedata[0].z, 0);

		};
		//DXChk(mFX->SetValue(mhEyePos, &Eye, sizeof(D3DXVECTOR3)));

		linedata[0].x = -0.16*r*xzscale;
		linedata[0].y = YADJUST;
		linedata[0].z = 3.44*xzscale;
		linedata[0].colour = 0xff000000; // 

		linedata[1].x = 0.16*r*xzscale;
		linedata[1].y = YADJUST;
		linedata[1].z = linedata[0].z;
		linedata[1].colour = linedata[0].colour;

		Direct3D.pd3dDevice->DrawPrimitiveUP(D3DPT_LINESTRIP, 1, linedata, sizeof(vertex1));

		Direct3D.pd3dDevice->EndScene();
	}
	else {
		printf("BeginScene failed!\n\n");
		getch();
	}
}

void Create1DGraphingData(TriMesh * pX, bool bTdata = false, bool bAcceldata = false,
	bool bOhmsData = false, bool b_arelz_data = false)
{
	// Takes p_temphost3,4,5,6 and turns them into graphdata[iGraph=0,1,2,3][]

	Vertex * pVertex, * pVert2;
	f64_vec2 pos, pos0, pos1, pos2;
	f64 dist0, dist1, dist2, wt0, wt1, wt2, wttotal, y0, y1, y2;
	int iGraph, asdf, iWhich, iCorner, tri_len, i;
	bool has_more, has_less, has_grad;
	Triangle * pTri;
	long izTri[MAXNEIGH];

	long VertexIndexArray[10000];

	num_graph_data_points = pX->GetVertsRightOfCutawayLine_Sorted(VertexIndexArray, graph_r, true);
	
	printf("Xebeques furious\n Number of points %d\n", num_graph_data_points);

	memset(maximum, 0, sizeof(f64) * 20);

	// Method used in Render routine looks quite reasonable: find tri that crosses cutaway,
	// use some kind of interp on tri. But we need to use values from p_temphost array not a graph position.

	for (asdf = 0; asdf < num_graph_data_points; asdf++)
	{
	//	if (asdf % 10 == 0) printf("<");
	//	printf("%d ; ", VertexIndexArray[asdf]);
		pVertex = pX->X + VertexIndexArray[asdf];

		// We want the tri directly to the left of it, through which (-1,0) passes.
		// 1.Get these vertex indices
		// which tri contains a point which is further and a point less far?

		real rr = pVertex->pos.x*pVertex->pos.x + pVertex->pos.y*pVertex->pos.y;
		iWhich = -1;
		tri_len = pVertex->GetTriIndexArray(izTri);
		for (i = 0; i < tri_len; i++)
		{
			pTri = pX->T + izTri[i];
			has_more = false; has_less = false; has_grad = false;
			for (iCorner = 0; iCorner < 3; iCorner++)
			{
				pVert2 = pTri->cornerptr[iCorner];
				if (pVert2 != pVertex)
				{
					if (pVert2->pos.x*pVert2->pos.x + pVert2->pos.y*pVert2->pos.y > rr)
					{
						has_more = true;
					}
					else {
						has_less = true;
					};
				};
				if (pVert2->pos.x / pVert2->pos.y < pVertex->pos.x / pVertex->pos.y)
					has_grad = true;
			};

			if (has_more && has_less && has_grad)
			{
				iWhich = i;
			}
		};

		if (iWhich == -1) {// give up, do nothing} 
			printf("gave up. %d \n", VertexIndexArray[asdf]);
			graphdata[0][asdf] = 0.0;
			graphdata[1][asdf] = 0.0;
			graphdata[2][asdf] = 0.0;
			graphdata[3][asdf] = 0.0;
		} else {
			pTri = pX->T + izTri[iWhich];
			while ((pTri->u8domain_flag != DOMAIN_TRIANGLE) && (iWhich >= 0)) {
				pTri = pX->T + izTri[iWhich];
				iWhich--;
			};
			iWhich++;

			// we are needing to adjust graph_r and interp graphdata

			pos.y = pVertex->pos.y;
			pos.x = pVertex->pos.y*CUTAWAYANGLE; // can leave graph_r undisturbed

			pos0 = pTri->cornerptr[0]->pos;
			pos1 = pTri->cornerptr[1]->pos;
			pos2 = pTri->cornerptr[2]->pos;

			// if one sits at the CUTAWAYANGLE then we can get dist == 0.

			dist0 = sqrt((pos0 - pos).dot(pos0 - pos));
			dist1 = sqrt((pos1 - pos).dot(pos1 - pos));
			dist2 = sqrt((pos2 - pos).dot(pos2 - pos));
			
			if (dist0 == 0.0) {
				wt0 = 1.0; wt1 = 0.0; wt2 = 0.0;
			}
			else {
				if (dist1 == 0.0) {
					wt0 = 0.0; wt1 = 1.0; wt2 = 0.0;
				}
				else {
					if (dist2 == 0.0) {
						wt0 = 0.0; wt1 = 0.0; wt2 = 1.0;
					} else {

						wt0 = 1.0f / dist0;
						wt1 = 1.0f / dist1;
						wt2 = 1.0f / dist2;
						wttotal = wt0 + wt1 + wt2;
						wt0 /= wttotal;
						wt1 /= wttotal;
						wt2 /= wttotal;
						// Not a great way it has to be said.
					}
				}
			}


			if ((bTdata == false) && (bAcceldata == false) && (bOhmsData == false)
				&& (b_arelz_data == false)) {
				y0 = p_temphost3[(pTri->cornerptr[0] - pX->X) + BEGINNING_OF_CENTRAL];
				y1 = p_temphost3[(pTri->cornerptr[1] - pX->X) + BEGINNING_OF_CENTRAL];
				y2 = p_temphost3[(pTri->cornerptr[2] - pX->X) + BEGINNING_OF_CENTRAL];
				graphdata[0][asdf] = wt0*y0 + wt1*y1 + wt2*y2;
				if (fabs(graphdata[0][asdf]) > maximum[0]) maximum[0] = fabs(graphdata[0][asdf]);

				if (numgraphs > 1) {
					y0 = p_temphost4[(pTri->cornerptr[0] - pX->X) + BEGINNING_OF_CENTRAL];
					y1 = p_temphost4[(pTri->cornerptr[1] - pX->X) + BEGINNING_OF_CENTRAL];
					y2 = p_temphost4[(pTri->cornerptr[2] - pX->X) + BEGINNING_OF_CENTRAL];
					graphdata[1][asdf] = wt0*y0 + wt1*y1 + wt2*y2;
					if (fabs(graphdata[1][asdf]) > maximum[1]) maximum[1] = fabs(graphdata[1][asdf]);
				};

				if (numgraphs > 2) {
					y0 = p_temphost5[(pTri->cornerptr[0] - pX->X) + BEGINNING_OF_CENTRAL];
					y1 = p_temphost5[(pTri->cornerptr[1] - pX->X) + BEGINNING_OF_CENTRAL];
					y2 = p_temphost5[(pTri->cornerptr[2] - pX->X) + BEGINNING_OF_CENTRAL];
					graphdata[2][asdf] = wt0*y0 + wt1*y1 + wt2*y2;
					if (fabs(graphdata[2][asdf]) > maximum[2]) maximum[2] = fabs(graphdata[2][asdf]);
				};
				if (numgraphs > 3) {
					y0 = p_temphost6[(pTri->cornerptr[0] - pX->X) + BEGINNING_OF_CENTRAL];
					y1 = p_temphost6[(pTri->cornerptr[1] - pX->X) + BEGINNING_OF_CENTRAL];
					y2 = p_temphost6[(pTri->cornerptr[2] - pX->X) + BEGINNING_OF_CENTRAL];
					graphdata[3][asdf] = wt0*y0 + wt1*y1 + wt2*y2;
					if (fabs(graphdata[3][asdf]) > maximum[3]) maximum[3] = fabs(graphdata[3][asdf]);
				}
			} else {
				// go through from 0 = conduction to 5 = dTe/dt itself
				// we have missed out compressive...

				if (bTdata) {
					for (int j = 0; j < 8; j++)
					{
						y0 = p_Tgraph_host[j][(pTri->cornerptr[0] - pX->X)];
						y1 = p_Tgraph_host[j][(pTri->cornerptr[1] - pX->X)];
						y2 = p_Tgraph_host[j][(pTri->cornerptr[2] - pX->X)];
						graphdata[j][asdf] = wt0*y0 + wt1*y1 + wt2*y2;
						if (fabs(graphdata[j][asdf]) > maximum[j]) maximum[j] = fabs(graphdata[j][asdf]);
					}
				} else {
					if (bAcceldata) {
						int j;
						j = 1; // total
						y0 = p_accelgraph_host[j][(pTri->cornerptr[0] - pX->X)];
						y1 = p_accelgraph_host[j][(pTri->cornerptr[1] - pX->X)];
						y2 = p_accelgraph_host[j][(pTri->cornerptr[2] - pX->X)];
						graphdata[0][asdf] = wt0*y0 + wt1*y1 + wt2*y2;
						
						j = 3; // vxB
						y0 = p_accelgraph_host[j][(pTri->cornerptr[0] - pX->X)];
						y1 = p_accelgraph_host[j][(pTri->cornerptr[1] - pX->X)];
						y2 = p_accelgraph_host[j][(pTri->cornerptr[2] - pX->X)];
						graphdata[1][asdf] = wt0*y0 + wt1*y1 + wt2*y2;
						j = 5; // pressure
						y0 = p_accelgraph_host[j][(pTri->cornerptr[0] - pX->X)];
						y1 = p_accelgraph_host[j][(pTri->cornerptr[1] - pX->X)];
						y2 = p_accelgraph_host[j][(pTri->cornerptr[2] - pX->X)];
						graphdata[2][asdf] = wt0*y0 + wt1*y1 + wt2*y2;
						j = 6; // neutral soak
						y0 = p_accelgraph_host[j][(pTri->cornerptr[0] - pX->X)];
						y1 = p_accelgraph_host[j][(pTri->cornerptr[1] - pX->X)];
						y2 = p_accelgraph_host[j][(pTri->cornerptr[2] - pX->X)];
						graphdata[3][asdf] = wt0*y0 + wt1*y1 + wt2*y2;

						j = 8; // viscosity
						y0 = p_accelgraph_host[j][(pTri->cornerptr[0] - pX->X)];
						y1 = p_accelgraph_host[j][(pTri->cornerptr[1] - pX->X)];
						y2 = p_accelgraph_host[j][(pTri->cornerptr[2] - pX->X)];
						graphdata[4][asdf] = wt0*y0 + wt1*y1 + wt2*y2;

						j = 9; // ionization
						y0 = p_accelgraph_host[j][(pTri->cornerptr[0] - pX->X)];
						y1 = p_accelgraph_host[j][(pTri->cornerptr[1] - pX->X)];
						y2 = p_accelgraph_host[j][(pTri->cornerptr[2] - pX->X)];
						graphdata[5][asdf] = wt0*y0 + wt1*y1 + wt2*y2;

						j = 10; // advection
						y0 = p_accelgraph_host[j][(pTri->cornerptr[0] - pX->X)];
						y1 = p_accelgraph_host[j][(pTri->cornerptr[1] - pX->X)];
						y2 = p_accelgraph_host[j][(pTri->cornerptr[2] - pX->X)];
						graphdata[6][asdf] = wt0*y0 + wt1*y1 + wt2*y2;

						// works if comment here

					//	printf("%d ", asdf);
						for (int j = 0; j < 7; j++)
						{ 
							//		printf("%d", j);
							if (fabs(graphdata[j][asdf]) > maximum[0]) {
								maximum[0] = fabs(graphdata[j][asdf]);
								//			printf("maximum %1.9E\n", maximum[0]);
							}
						}

						// does it work if comment here? no

						j = 11; // grad_y Az
						y0 = p_accelgraph_host[j][(pTri->cornerptr[0] - pX->X)];
						y1 = p_accelgraph_host[j][(pTri->cornerptr[1] - pX->X)];
						y2 = p_accelgraph_host[j][(pTri->cornerptr[2] - pX->X)];
						graphdata[7][asdf] = wt0*y0 + wt1*y1 + wt2*y2;
						if (fabs(graphdata[7][asdf]) > maximum[7]) maximum[7] = fabs(graphdata[7][asdf]);
					} else {
						if (bOhmsData) {
							int j;
							for (j = 0; j < 19; j++) {
								y0 = p_Ohmsgraph_host[j][(pTri->cornerptr[0] - pX->X)];
								y1 = p_Ohmsgraph_host[j][(pTri->cornerptr[1] - pX->X)];
								y2 = p_Ohmsgraph_host[j][(pTri->cornerptr[2] - pX->X)];
								graphdata[j][asdf] = wt0*y0 + wt1*y1 + wt2*y2;
								if ((pos.y < 4.6) && (pos.y > 3.44) && (fabs(graphdata[j][asdf]) > maximum[j])) maximum[j] = fabs(graphdata[j][asdf]);
							};
						} else {
							int j;
							for (j = 0; j < 12; j++) {
								y0 = p_arelz_graph_host[j][(pTri->cornerptr[0] - pX->X)];
								y1 = p_arelz_graph_host[j][(pTri->cornerptr[1] - pX->X)];
								y2 = p_arelz_graph_host[j][(pTri->cornerptr[2] - pX->X)];
								graphdata[j][asdf] = wt0*y0 + wt1*y1 + wt2*y2;
								if ((pos.y < 4.8) && (pos.y > 3.44) && (fabs(graphdata[j][asdf]) > maximum[j])) maximum[j] = fabs(graphdata[j][asdf]);
							};
						};
					};
				};
			};
		}; // found triangle		
	}; // asdf	
	if ((bTdata == false) && (bAcceldata == false) && (bOhmsData == false) && (b_arelz_data == false)) {		
		maximum[3] = max(maximum[3], maximum[2]);
		maximum[2] = maximum[3];
	} else {
		// for dT graphs, let maximum be overall
		if (bTdata) {
			for (int j = 1; j <= 6; j++)
				maximum[j] = max(maximum[j], maximum[j - 1]);
			for (int j = 5; j >= 0; j--)
				maximum[j] = maximum[j + 1];
		} else {
			if (bAcceldata) {
				for (int j = 1; j < 7; j++)
					maximum[j] = maximum[0];
			} else {
				if (bOhmsData) {
					memcpy(truemax, maximum, sizeof(f64) * 20);

					// use max 0 and 1 combined:
					f64 temp = max(maximum[0], maximum[1]);
					maximum[0] = temp;
					maximum[1] = temp;
					temp = max(max(maximum[2], maximum[3]), max(maximum[4], maximum[13]));
					maximum[2] = temp;
					maximum[3] = temp;
					maximum[4] = temp; // thermal force
					maximum[13] = temp; // viscous
					temp = max(maximum[5], maximum[6]);
					maximum[5] = temp;
					maximum[6] = temp;
					temp = max(max(maximum[7], maximum[8]), max(maximum[9], maximum[10]));
					maximum[7] = temp;
					maximum[8] = temp;
					maximum[9] = temp;
					maximum[10] = temp;
					//	temp = max(maximum[11], maximum[12]); // difference, progress
					//	maximum[11] = temp;
					//	maximum[12] = temp;
					temp = max(maximum[14], maximum[17]);
					maximum[14] = temp;
					maximum[17] = temp;
					temp = max(maximum[15], maximum[16]);
					maximum[15] = temp;
					maximum[16] = temp;
				} else {
					// All same scale except for "difference" = element 10
					// ... and the arelz itself?
					int j;
					f64 temp = maximum[1];
					for (j = 2; j < 9; j++)
						temp = max(temp, maximum[j]);
					for (j = 1; j < 9; j++)
						maximum[j] = temp;
					temp = max(maximum[0], maximum[9]);
					maximum[0] = temp; 
					maximum[9] = temp; // actual vs sum

				}
			}
		}
	}
}


void RefreshGraphs(TriMesh & X, // only not const because of such as Reset_vertex_nvT
	const int iGraphsFlag)
{
	D3DXMATRIXA16 matWorld;
	Vertex * pVertex;
	long iVertex;
	plasma_data * pdata;
	int offset_v, offset_T;
	char buff[256];
	sprintf(buff, "%5.2f ns", evaltime*1.0e9);
	f64 overc;
	char buffer[256];
	overc = 1.0 / c_;
	float x, y, z;
	float zeroplane = 0.0f;
	int i;
	int iGraph;
	

	float const MAXX = 11.0f;
	float const MAXY = 6.0f;
	long iMinor;

	switch (iGraphsFlag) {
		
	case ONE_D:

		// We are going to have to think about using LineTo the way it is done in RenderGraphs
		// let's start by rendering in the x-y plane and we can let the present camera look at it
		printf("\n\nGot to here: ONE_D\n\n");
		
		// Create data:
		Create1DGraphingData(&X);
		
		Draw1Dgraph(6, ONE_D);


		pVertex = X.X;
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			pdata->temp.x = p_temphost3[iVertex+BEGINNING_OF_CENTRAL];
			++pVertex;
			++pdata;
		}
		Graph[4].DrawSurface("Azdot",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
			true,
			GRAPH_AZDOT, &X);

		pVertex = X.X;
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			pdata->temp.x = p_temphost4[iVertex + BEGINNING_OF_CENTRAL];
			++pVertex;
			++pdata;
		}
		Graph[1].DrawSurface("Azdotdot",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
			true,
			GRAPH_AZDOT, &X);

		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			pdata->temp.x = p_temphost5[iVertex + BEGINNING_OF_CENTRAL];
			++pdata;
		}
		Graph[3].DrawSurface("Lap Az",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
			true,
			GRAPH_LAPAZ, &X);

		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			if ((pVertex->flags == DOMAIN_VERTEX) || (pVertex->flags == OUTERMOST))
			{
				pdata->temp.x = p_temphost6[iVertex + BEGINNING_OF_CENTRAL];
			}
			else {
				pdata->temp.x = 0.0;
			}
			++pdata;
		}
		Graph[5].DrawSurface("Jz",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
			true,
			GRAPH_JZ, &X);
		
		break;

	case AZSOLVERGRAPHS:

		pdata = X.pData;
		// Bear in mind: iMinor won't actually get displayed
		for (iMinor = 0; iMinor < NMINOR; iMinor++)
		{
			pdata->temp.x = p_temphost1[iMinor]; // epsilon
			pdata->Azdot = p_temphost2[iMinor]; // Azdot0
			pdata->temp.y = p_temphost3[iMinor]; // gamma
			pdata->Az = p_temphost4[iMinor]; // Az			
			++pdata;
		}
		Graph[0].DrawSurface("epsilon",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
			true,
			GRAPH_EPSILON, &X);

		Graph[2].DrawSurface("Azdot0",
			DATA_HEIGHT, (real *)(&(X.pData[0].Azdot)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].Azdot)),
			true,
			GRAPH_AZDOT, &X);
		Graph[3].DrawSurface("regressorn",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.y)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.y)),
			true,
			GRAPH_OPTI, &X);

		Graph[4].DrawSurface("Az",
			DATA_HEIGHT, (real *)(&(X.pData[0].Az)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].Az)),
			true,
			GRAPH_AZ, &X);
		pdata = X.pData;
		for (iMinor = 0; iMinor < NMINOR; iMinor++)
		{
			pdata->temp.x = p_temphost5[iMinor]; // epsilon
			pdata->temp.y = p_temphost6[iMinor]; // Azdot0
			++pdata;
		}
		Graph[1].DrawSurface("regressori",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
			true,
			GRAPH_LAPAZ, &X);
		Graph[5].DrawSurface("Jacobi",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.y)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.y)),
			true,
			GRAPH_REGRESSOR, &X);

		break;
	case DTGRAPH:

		// We are going to have to think about using LineTo the way it is done in RenderGraphs
		// let's start by rendering in the x-y plane and we can let the present camera look at it
		printf("\n\nRefreshGraphs: DTGRAPHS\n\n");

		// Create data:
		Create1DGraphingData(&X, true);

		Draw1Dgraph(6, DTGRAPH);

		pVertex = X.X;
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			pdata->temp.x = p_Tgraph_host[5][iVertex];
			++pVertex;
			++pdata;
		}
		Graph[4].DrawSurface("dTe/dt",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			SEGUE_COLOUR, (real *)(&(X.pData[0].Te)),
			false,
			GRAPH_DTE, &X);

		pVertex = X.X;
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			pdata->temp.x = p_Tgraph_host[7][iVertex];
			++pVertex;
			++pdata;
		}
		Graph[1].DrawSurface("d/dt nTe",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
			false,
			GRAPH_DNT, &X);

		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			pdata->temp.x = p_Tgraph_host[5][iVertex];
			++pdata;
		}
		Graph[3].DrawSurface("n",
			DATA_HEIGHT, (real *)(&(X.pData[0].n)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
			false,
			GRAPH_ION_N, &X);

		Graph[5].DrawSurface("Te",
			DATA_HEIGHT, (real *)(&(X.pData[0].Te)),
			SEGUE_COLOUR, (real *)(&(X.pData[0].Te)),
			false,
			GRAPH_ELEC_T, &X);

		break;
	case OHMS2:

			// We are going to have to think about using LineTo the way it is done in RenderGraphs
			// let's start by rendering in the x-y plane and we can let the present camera look at it
			printf("\n\nRefreshGraphs: OHMS2\n\n");

			// Create data:
			Create1DGraphingData(&X, false, false, true);

			Draw1Dgraph(6, OHMS2);
			Draw1Dgraph(7, OHMSLAW);

			pVertex = X.X;
			pdata = X.pData + BEGINNING_OF_CENTRAL;
			for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
			{
				pdata->temp.x = p_Ohmsgraph_host[14][iVertex];
				++pVertex;
				++pdata;
			}
			Graph[4].DrawSurface("Jz prediction",
				DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
				SEGUE_COLOUR, (real *)(&(X.pData[0].Te)),
				false,
				GRAPH_JZ, &X);

			pVertex = X.X;
			pdata = X.pData + BEGINNING_OF_CENTRAL;
			for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
			{
				pdata->temp.x = p_Ohmsgraph_host[17][iVertex];
				++pVertex;
				++pdata;
			}
			Graph[5].DrawSurface("electromotive-only prediction",
				DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
				SEGUE_COLOUR, (real *)(&(X.pData[0].Te)),
				false,
				GRAPH_VE0Z, &X);

			break;

	case ACCELGRAPHS:

		// We are going to have to think about using LineTo the way it is done in RenderGraphs
		// let's start by rendering in the x-y plane and we can let the present camera look at it
		printf("\n\nRefreshGraphs: ACCELGRAPHS\n\n");

		// Create data:
		Create1DGraphingData(&X, false, true);
		
		Draw1Dgraph(6, ACCELGRAPHS);

		pVertex = X.X;
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			pdata->temp.x = p_accelgraph_host[0][iVertex];
			pdata->temp.y = p_accelgraph_host[1][iVertex];
			++pVertex;
			++pdata;
		}
		Graph[4].DrawSurface("dvxy/dt",
			VELOCITY_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			VELOCITY_COLOUR, (real *)(&(X.pData[0].temp.x)),
			false,
			GRAPH_AXY, &X);

		pVertex = X.X;
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			pdata->temp.x = p_accelgraph_host[2][iVertex];
			pdata->temp.y = p_accelgraph_host[3][iVertex];
			++pVertex;
			++pdata;
		}
		Graph[1].DrawSurface("axy : v x B",
			VELOCITY_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			VELOCITY_COLOUR, (real *)(&(X.pData[0].temp.x)),
			false,
			GRAPH_AXY2, &X);

		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			pdata->temp.x = p_accelgraph_host[4][iVertex];
			pdata->temp.y = p_accelgraph_host[5][iVertex];
			++pdata;
		}
		Graph[3].DrawSurface("axy : pressure",
			VELOCITY_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			VELOCITY_COLOUR, (real *)(&(X.pData[0].temp.x)),
			false,
			GRAPH_AXY3, &X);

		Graph[5].DrawSurface("vxy",
			VELOCITY_HEIGHT, (real *)(&(X.pData[0].vxy)),
			VELOCITY_COLOUR, (real *)(&(X.pData[0].vxy)),
			false,
			GRAPH_ION_V, &X);
		
		break;

	case ARELZ:

		// We are going to have to think about using LineTo the way it is done in RenderGraphs
		// let's start by rendering in the x-y plane and we can let the present camera look at it
		printf("\n\nRefreshGraphs: ARELZ\n\n");

		// Create data:
		Create1DGraphingData(&X, false, false, false, true);

		Draw1Dgraph(6, ARELZ);

		// Graphs:
		// .. arelz
		// .. electromotive
		// .. v x B
		// .. error

		pVertex = X.X;
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			pdata->temp.x = p_arelz_graph_host[0][iVertex];
			pdata->temp.y = p_arelz_graph_host[0][iVertex];
			++pVertex;
			++pdata;
		}
		Graph[4].DrawSurface("arelz",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
			false,
			GRAPH_ARELZ, &X);

		pVertex = X.X;
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			pdata->temp.x = p_arelz_graph_host[3][iVertex] +
				p_arelz_graph_host[4][iVertex];

			++pVertex;
			++pdata;
		}
		Graph[1].DrawSurface("-e/m Ez_total",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
			false,
			GRAPH_ELECTROMOTIVE, &X);

		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			pdata->temp.x = p_arelz_graph_host[5][iVertex];
			++pdata;
		}
		Graph[3].DrawSurface("arelz : v x B",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
			false,
			GRAPH_VXBARELZ, &X);

		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			pdata->temp.x = p_arelz_graph_host[10][iVertex];
			if (pdata->temp.x > 1.0e13) {
				printf("%d %1.9E | ", iVertex, pdata->temp.x);
			}
			++pdata;
		}
		Graph[5].DrawSurface("error in sum",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
			false,
			GRAPH_ERROR, &X);


	//	Cannot explain why maximum on graph is 1e13 not 1e5 as reported on 1D graph.


		break;
		/*
		case JXY_RHO_EXY_GRADPHI_AXYDOTOC_AXY:

		X.Setup_J(); // the others can already exist.

		Graph[4].bDisplayTimestamp = true;

		pVertex = pX->X;
		for (iVertex = 0; iVertex < pX->numVertices; iVertex++)
		{
		if (pVertex->flags == DOMAIN_VERTEX) {
		pVertex->temp2.x = q * (pVertex->Ion.mass - pVertex->Elec.mass) / pVertex->AreaCell;
		}
		else {
		pVertex->temp2.x = 0.0;
		};
		pVertex->Adot /= c;
		++pVertex;
		}

		Graph[0].DrawSurface("Exy[statV/cm]",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].E)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].E)),
		true,
		GRAPH_EXY, &X);

		Graph[1].DrawSurface("Adotxy/c[statV/cm]",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].Adot.x)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].Adot.x)),
		true,
		GRAPH_ADOTXY, &X);

		Graph[2].DrawSurface("Jxy",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].Temp.x)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].Temp.x)),
		false, // no inner mesh display.
		GRAPH_JXY, &X);

		Graph[3].DrawSurface("phidot[statV/s]",
		DATA_HEIGHT, (real *)(&(X.X[0].phidot)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].phidot)),
		true,
		GRAPH_PHIDOT, &X);

		Graph[4].DrawSurface("rho",
		DATA_HEIGHT, (real *)(&(X.X[0].temp2.x)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].temp2.x)),
		false, // no inner mesh display.
		GRAPH_RHO, &X);

		Graph[5].DrawSurface("phi[statV]",
		DATA_HEIGHT, (real *)(&(X.X[0].phi)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].phi)),
		true,
		GRAPH_PHI, &X);

		pVertex = pX->X;
		for (iVertex = 0; iVertex < pX->numVertices; iVertex++)
		{
		pVertex->Adot *= c;
		++pVertex;
		}
		break;
		case JZ_AZ_BXY_EZ_ADOTZOC_NVZ:

		X.Reset_vertex_nvT(SPECIES_ELEC);

		X.Setup_J(); // the others can already exist.

		pVertex = pX->X;
		for (iVertex = 0; iVertex < pX->numVertices; iVertex++)
		{
		pVertex->Adot /= c;
		++pVertex;
		}
		Graph[4].bDisplayTimestamp = true;

		Graph[0].DrawSurface("Ez[statV/cm]",
		DATA_HEIGHT, (real *)(&(X.X[0].E.z)),
		FLAG_SEGUE_COLOUR, (real *)(&(X.X[0].E.z)),
		false, // ??
		GRAPH_EZ, &X);

		Graph[1].DrawSurface("Az",
		DATA_HEIGHT, (real *)(&(X.X[0].A.z)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].A.z)),
		true,
		GRAPH_AZ, &X);
		Graph[2].DrawSurface("Jz",
		DATA_HEIGHT, (real *)(&(X.X[0].Temp.z)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].Temp.z)),
		false, // no inner mesh display.
		GRAPH_JZ, &X);
		Graph[3].DrawSurface("Bxy[Gs]",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].B)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].B)),
		true, // no inner mesh display: ??
		GRAPH_BXY, &X);

		Graph[4].DrawSurface("Adotz/c [statV/cm]",
		DATA_HEIGHT, (real *)(&(X.X[0].Adot.z)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].Adot.z)),
		true,
		GRAPH_AZ, &X);

		Graph[5].colourmax = Graph[2].colourmax;
		Graph[5].DrawSurface("Elec n",
		DATA_HEIGHT, (real *)(&(X.X[0].n)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].Temp.z)),
		false, // no inner mesh display
		GRAPH_ELEC_N, &X);

		pVertex = pX->X;
		for (iVertex = 0; iVertex < pX->numVertices; iVertex++)
		{
		pVertex->Adot *= c;
		++pVertex;
		}

		break;
		case SPECIES_ELECTRON2:

		X.Reset_vertex_nvT(SPECIES_ELEC);

		Graph[0].DrawSurface("Elec n [/cc]",
		DATA_HEIGHT, (real *)(&(X.X[0].n)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].v)),
		false, // no inner mesh display
		GRAPH_ELEC_N, &X);

		Graph[1].DrawSurface("v_e_xy[cm/s]",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].v)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].v)),
		false, // no inner mesh display
		GRAPH_ELEC_V, &X);

		Graph[3].DrawSurface("v_e_z[cm/s]",
		DATA_HEIGHT, (real *)(&(X.X[0].v.z)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].v.z)),
		false, // no inner mesh display.
		GRAPH_VEZ, &X);

		pVertex = pX->X;
		for (iVertex = 0; iVertex < pX->numVertices; iVertex++)
		{
		if (pVertex->flags == DOMAIN_VERTEX) {
		pVertex->temp2.x = (pVertex->Ion.mass - pVertex->Elec.mass) / pVertex->AreaCell;
		}
		else {
		pVertex->temp2.x = 0.0;
		};
		++pVertex;
		}
		Graph[2].bDisplayTimestamp = false;
		Graph[2].DrawSurface("n_i-n_e",
		DATA_HEIGHT, (real *)(&(X.X[0].temp2.x)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].temp2.x)),
		false, // no inner mesh display.
		GRAPH_NINE, &X);

		Graph[5].TickRescaling = 1.0 / kB;
		Graph[5].DrawSurface("Elec T [eV]",
		DATA_HEIGHT, (real *)(&(X.X[0].T)),
		SEGUE_COLOUR, (real *)(&(X.X[0].T)),
		false, // no inner mesh display
		GRAPH_ELEC_T, &X);
		Graph[5].TickRescaling = 1.0;

		offset_v = (real *)(&(X.X[0].v)) - (real *)(&(X.X[0]));
		offset_T = (real *)(&(X.X[0].T)) - (real *)(&(X.X[0]));

		Graph[4].SetEyePlan(GlobalPlanEye);
		Graph[4].boolDisplayMeshWireframe = true;
		Graph[4].boolClearZBufferBeforeWireframe = true;
		Graph[4].boolDisplayMainMesh = true;
		Graph[4].boolDisplayInnerMesh = false;
		Graph[4].boolDisplayScales = false;
		Graph[4].boolDisplayShadow = false;
		Graph[4].mhTech = Graph[4].mFX->GetTechniqueByName("VelociTech");
		Graph[4].colourmax = Graph[0].colourmax; // match colours
		Graph[4].SetDataWithColour(X, FLAG_VELOCITY_COLOUR, FLAG_FLAT_MESH, offset_v, offset_v,
		GRAPH_FLAT_WIRE_MESH);
		Graph[4].Render(buff, GlobalRenderLabels, &X);

		break;
		*/

case IONIZEGRAPH:
	printf("\n\nRefreshGraphs: IONIZEGRAPHS\n\n");

	// When we come to speed up graphs, make it so we can
	// just pass an array of f64. !!!!
	// Investigate graphs half an hour: what's up with the rest?

	// Move table, start running.
	// Can we bring back cutaway any how? 
	// Wanted acceleration graphs. 
	// Want to do a big run. 

	pVertex = X.X;
	pdata = X.pData + BEGINNING_OF_CENTRAL;
	for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
	{
		pdata->temp.x = p_graphdata1_host[iVertex];
		pdata->temp.y = p_graphdata2_host[iVertex]; // dn/dt /n
		++pVertex;
		++pdata;
	}
	Graph[0].DrawSurface("dn/dt",
		DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
		SEGUE_COLOUR, (real *)(&(X.pData[0].Te)),
		false,
		GRAPH_DNDT, &X);

	Graph[1].DrawSurface("dn/dt / n",
		DATA_HEIGHT, (real *)(&(X.pData[0].temp.y)),
		SEGUE_COLOUR, (real *)(&(X.pData[0].Te)),
		false,
		GRAPH_DNDT_OVER_n, &X);
	

	pVertex = X.X;
	pdata = X.pData + BEGINNING_OF_CENTRAL;
	for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
	{
		pdata->temp.x = p_graphdata3_host[iVertex]; // log10 n
		++pVertex;
		++pdata;
	}
	Graph[3].DrawSurface("log10(n)",
		DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
		AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
		false,
		GRAPH_LOG10N, &X);

	pVertex = X.X;
	pdata = X.pData + BEGINNING_OF_CENTRAL;
	for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
	{
		pdata->temp.x = p_graphdata4_host[iVertex]; // dTe/dt
		pdata->temp.y = p_graphdata6_host[iVertex]; // n/nn
		++pVertex;
		++pdata;
	} 
	Graph[2].DrawSurface("dTe/dt[ionization]",
		DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
		SEGUE_COLOUR, (real *)(&(X.pData[0].Te)),
		false,
		GRAPH_DTEDT, &X);

	Graph[4].DrawSurface("n_e / n_total",
		DATA_HEIGHT, (real *)(&(X.pData[0].temp.y)),
		IONISE_COLOUR, (real *)(&(X.pData[0].temp.y)),
		false,
		GRAPH_FRACTION, &X);

	pVertex = X.X;
	pdata = X.pData + BEGINNING_OF_CENTRAL;
	for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
	{
		pdata->temp.x = p_graphdata5_host[iVertex]; // dvez/dt
		//if (pdata->vez != 0.0f) {
		//	pdata->temp.y = pdata->temp.x / (pdata->vez);
		//} else {
		//	pdata->temp.y = 0.0;
		//}
		++pVertex;
		++pdata;
	}
	printf("got to here 1");
	Graph[5].DrawSurface("accel ez[ionization]",
		DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
		AZSEGUE_COLOUR, (real *)(&(X.pData[0].vez)),
		false,
		GRAPH_AEZ1, &X);
	// Do we need another shader? Or can we reset limits?
	// see what scale is like.
	printf("got to here 2");

	break;




case OVERALL:
	printf("\n\nRefreshGraphs: OVERALL\n\n");

		pVertex = X.X;
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			if ((pVertex->flags == DOMAIN_VERTEX) || (pVertex->flags == OUTERMOST))
			{
				pdata->temp.x = pdata->n + pdata->n_n;
				pdata->temp.y = pdata->n / (1.0 + pdata->temp.x);
			} else {
				pdata->temp.x = 0.0;
				pdata->temp.y = 0.0;
			}
			++pVertex;
			++pdata;
		}
		
		Graph[0].DrawSurface("n_n + n_ion",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			IONISE_COLOUR, (real *)(&(X.pData[0].temp.y)),
			false,
			GRAPH_TOTAL_N, &X);
		
		pVertex = X.X;
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			if ((pVertex->flags == DOMAIN_VERTEX) || (pVertex->flags == OUTERMOST))
			{
				pdata->temp.x = (m_neutral_*pdata->n_n*pdata->v_n.x
					+ (m_ion_ + m_e_) * pdata->n*pdata->vxy.x) /
					(m_neutral_*pdata->n_n + (m_ion_ + m_e_)*pdata->n);
				pdata->temp.y = (m_neutral_*pdata->n_n*pdata->v_n.y
					+ (m_ion_ + m_e_) * pdata->n*pdata->vxy.y) /
					(m_neutral_*pdata->n_n + (m_ion_ + m_e_)*pdata->n);
			} else {
				pdata->temp.x = 0.0; pdata->temp.y = 0.0;
			}
			++pVertex;
			++pdata;
		}
		Graph[1].DrawSurface("sum[n_s v_s m_s]/sum[n_s m_s]",
			VELOCITY_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			VELOCITY_COLOUR, (real *)(&(X.pData[0].temp.x)),
			false, // no inner mesh display
			GRAPH_TOTAL_V, &X);

		pVertex = X.X;
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			if ((pVertex->flags == DOMAIN_VERTEX) || (pVertex->flags == OUTERMOST))
			{
				pdata->temp.x = (pdata->n_n*pdata->Tn
					+ pdata->n*(pdata->Ti + pdata->Te)) /
					(pdata->n_n + pdata->n + pdata->n);
			} else {
				pdata->temp.x = 0.0; pdata->temp.y = 0.0;
			}
			++pVertex;
			++pdata;
		}
		Graph[3].TickRescaling = 1.0 / kB_;
		Graph[3].DrawSurface("sum[n_s T_s]/sum[n_s]",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			SEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
			false,
			GRAPH_TOTAL_T, &X);
		Graph[3].TickRescaling = 1.0;


		Graph[2].DrawSurface("Neutral n",
			DATA_HEIGHT, (real *)(&(X.pData[0].n_n)),
			VELOCITY_COLOUR, (real *)(&(X.pData[0].v_n)),
			false, // no inner mesh display
			GRAPH_NEUT_N, &X);
		Graph[4].DrawSurface("Neutral v",
			VELOCITY_HEIGHT, (real *)(&(X.pData[0].v_n)),
			VELOCITY_COLOUR, (real *)(&(X.pData[0].v_n)),
			false, // no inner mesh display
			GRAPH_NEUT_V, &X);


		Graph[5].TickRescaling = 1.0 / kB_;
		Graph[5].DrawSurface("Neutral T",
			DATA_HEIGHT, (real *)(&(X.pData[0].Tn)),
			SEGUE_COLOUR, (real *)(&(X.pData[0].Tn)),
			false, // no inner mesh display
			GRAPH_NEUT_T, &X);
		Graph[5].TickRescaling = 1.0;
		
		break;
	case SPECIES_ION:
		printf("\n\nRefreshGraphs: SPECIES_ION\n\n");

		Graph[3].TickRescaling = 1.0 / kB_;
		Graph[3].DrawSurface("Ion T",
			DATA_HEIGHT, (real *)(&(X.pData[0].Ti)),
			SEGUE_COLOUR, (real *)(&(X.pData[0].Ti)),
			false, // no inner mesh display
			GRAPH_ION_T, &X);
		Graph[3].TickRescaling = 1.0;

		// labels only appear on first 1 called.

		Graph[0].DrawSurface("Ion n",
			DATA_HEIGHT, (real *)(&(X.pData[0].n)),
			VELOCITY_COLOUR, (real *)(&(X.pData[0].vxy)),
			false, // no inner mesh display
			GRAPH_ION_N, &X);
		Graph[1].DrawSurface("Ion v",
			VELOCITY_HEIGHT, (real *)(&(X.pData[0].vxy)),
			VELOCITY_COLOUR, (real *)(&(X.pData[0].vxy)),
			false, // no inner mesh display
			GRAPH_ION_V, &X);

		// These are same so double up with elec.

		Graph[5].TickRescaling = 1.0 / kB_;
		Graph[5].DrawSurface("Elec T",
			DATA_HEIGHT, (real *)(&(X.pData[0].Te)),
			SEGUE_COLOUR, (real *)(&(X.pData[0].Te)),
			false, // no inner mesh display
			GRAPH_ELEC_T, &X);
		Graph[5].TickRescaling = 1.0;

		break;

/*	case SPECIES_ELEC:

		Graph[0].DrawSurface("Elec n",
			DATA_HEIGHT, (real *)(&(X.pData[0].n)),
			VELOCITY_COLOUR, (real *)(&(X.pData[0].vxy)),
			false, // no inner mesh display
			GRAPH_ELEC_T, &X);
		// colours == 0.0 ... because v = 0
		// First........... let's understand why surface normals come out unpredictable.
		// Then............ let's go and see what it does with y values (in Render and .fx)

		Graph[1].DrawSurface("Elec v",
			VELOCITY_HEIGHT, (real *)(&(X.pData[0].vxy)),
			VELOCITY_COLOUR, (real *)(&(X.pData[0].vxy)),
			false, // no inner mesh display
			GRAPH_ELEC_V, &X);
		break;

		// In other cases, (and even for the above),
		// here is a good place to call the 
		// setup routines for temp variables.
		*/

case OHMSLAW:
	printf("\n\nRefreshGraphs: OHMSLAW\n\n");

		// 0. q/ m_e nu_sum 
		// 1. qn / m_e nu_sum
		// 2. nu_sum
		// 3. prediction of Jz from uniform Ez
		// 4. prediction of Jz from actual Ez
		// 5. Actual Jz
		
		// Let temphost1 = nu_en + nu_ei_effective
		// Let temphost2 = nu_en/temphost1


	// Cannot explain why, that comes out black and this doesn't.
	// Oh because colourmax has been set to 1 or not?

	// Yet the following crashes it. Bizarre? Maybe dividing by 0?
	
	overc = 1.0 / c_;
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		pVertex = X.X;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			if (pVertex->flags == DOMAIN_VERTEX) {
				pdata->temp.x = q_ / (m_e_ * (1.0 + p_temphost1[iVertex + BEGINNING_OF_CENTRAL]));
				pdata->temp.y = p_temphost2[iVertex + BEGINNING_OF_CENTRAL]; // colour
			} else {
				pdata->temp.x = 0.0;
				pdata->temp.y = 0.0;
			}
			++pdata;
			++pVertex;
		};
		Graph[0].DrawSurface("q over m nu_effective",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			PPN_COLOUR, (real *)(&(X.pData[0].temp.y)),
			false, // no inner mesh display.
			GRAPH_VRESPONSEOHMS, &X);

		
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		pVertex = X.X;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			if (pVertex->flags == DOMAIN_VERTEX) {
				pdata->temp.x = q_*X.pData[iVertex + BEGINNING_OF_CENTRAL].n /
					(m_e_ * (1.0 + p_temphost1[iVertex + BEGINNING_OF_CENTRAL]));
				pdata->temp.y = p_temphost2[iVertex + BEGINNING_OF_CENTRAL]; // colour
			} else {
				pdata->temp.x = 0.0;
				pdata->temp.y = 0.0;
			};
			++pdata;
			++pVertex;
		};

		Graph[1].DrawSurface("qn / m nu_effective",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			PPN_COLOUR, (real *)(&(X.pData[0].temp.y)),
			false, // no inner mesh display.
			GRAPH_CONDUCTIVITYOHMS, &X);
			

		pdata = X.pData + BEGINNING_OF_CENTRAL;
		pVertex = X.X;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			if (pVertex->flags == DOMAIN_VERTEX) {
				pdata->temp.x = p_temphost1[iVertex + BEGINNING_OF_CENTRAL];
				pdata->temp.y = p_temphost2[iVertex + BEGINNING_OF_CENTRAL]; // colour
			};
			++pVertex;
			++pdata;
		};
		Graph[2].DrawSurface("nu_effective (blue=neut dominates)",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			PPN_COLOUR, (real *)(&(X.pData[0].temp.y)),
			false, // no inner mesh display.
			GRAPH_NU_EFFECTIVE, &X);
			
		
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		pVertex = X.X;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			if (pVertex->flags == DOMAIN_VERTEX) {
				pdata->temp.x = EzStrength_*q_*q_*X.pData[iVertex + BEGINNING_OF_CENTRAL].n /
					(m_e_ * (1.0 + p_temphost1[iVertex + BEGINNING_OF_CENTRAL]));
			};
			++pdata;
		};
		Graph[3].DrawSurface("predict Jz (uniform Ez)",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),			
			false, // no inner mesh display.
			GRAPH_JZ, &X);
					
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		pVertex = X.X;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			if (pVertex->flags == DOMAIN_VERTEX) {
				pdata->temp.x = (EzStrength_
					- X.pData[iVertex + BEGINNING_OF_CENTRAL].Azdot*overc
					)*q_*q_*X.pData[iVertex + BEGINNING_OF_CENTRAL].n /
					(m_e_ * (1.0 + p_temphost1[iVertex + BEGINNING_OF_CENTRAL]));
			};
			++pdata;
		};
		Graph[4].DrawSurface("predict Jz (Ez)",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
			false, // no inner mesh display.
			GRAPH_JZ, &X);
			
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		pVertex = X.X;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			if (pVertex->flags == DOMAIN_VERTEX) {
				pdata->temp.x = q_*X.pData[iVertex + BEGINNING_OF_CENTRAL].n*
					(X.pData[iVertex + BEGINNING_OF_CENTRAL].viz - X.pData[iVertex + BEGINNING_OF_CENTRAL].vez);
			};
			++pdata;
		};
		Graph[5].DrawSurface("actual Jz",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
			false, // no inner mesh display.
			GRAPH_JZ, &X);

		break;

	case JZAZBXYEZ:
		printf("\n\nRefreshGraphs: JZAZBXYEZ\n\n");

		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			pdata->temp.x = q_ * pdata->n*(pdata->viz - pdata->vez);
			++pdata;
		};
		Graph[3].DrawSurface("Jz",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
			false, // no inner mesh display.
			GRAPH_JZ, &X);

		// create graph data for Ez : add Ez_strength*Ezshape to -Azdot/c
		overc = 1.0 / c_;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			X.pData[iVertex + BEGINNING_OF_CENTRAL].temp.y =
				-X.pData[iVertex + BEGINNING_OF_CENTRAL].Azdot*overc
				+ GetEzShape__(X.pData[iVertex + BEGINNING_OF_CENTRAL].pos.modulus())*EzStrength_;
		} 
		Graph[2].DrawSurface("Ez",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.y)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)), // use Jz's colour
			false, 
			GRAPH_EZ, &X);

		Graph[0].DrawSurface("Az",
			DATA_HEIGHT, (real *)(&(X.pData[0].Az)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].Az)),
			true, GRAPH_AZ, &X);

//		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
//		{
//			X.pData[iVertex + BEGINNING_OF_CENTRAL].temp.x = p_B_host[iVertex + BEGINNING_OF_CENTRAL].x;
//			X.pData[iVertex + BEGINNING_OF_CENTRAL].temp.y = p_B_host[iVertex + BEGINNING_OF_CENTRAL].y;
//		}
		Graph[1].DrawSurface("Bxy",
		VELOCITY_HEIGHT, (real *)(&(X.pData[0].B.x)),
		VELOCITY_COLOUR, (real *)(&(X.pData[0].B.x)),
		false,
		GRAPH_BXY, &X);

		Graph[5].DrawSurface("vez",
			DATA_HEIGHT, (real *)(&(X.pData[0].vez)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)), // colour is for Jz?
			false, GRAPH_VEZ, &X);


		pVertex = X.X;
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			pdata->temp.x = -p_temphost3[iVertex + BEGINNING_OF_CENTRAL]/c_;
			++pVertex;
			++pdata;
		}
		Graph[4].DrawSurface("-Azdot/c",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
			true,
			GRAPH_AZDOT, &X);
	//	pdata = X.pData + BEGINNING_OF_CENTRAL;
	//	for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
	//	{
	//		pdata->temp.x = temp_array_host[iVertex + BEGINNING_OF_CENTRAL];
	//		++pdata;
	//	};
	//	Graph[4].DrawSurface("Lap Az",
	//		DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
	//		AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
	//		true, GRAPH_LAPAZ, &X);
		break;

	case VIZVEZJZAZDOT:
		printf("\n\nRefreshGraphs: VIZVEZJZAZDOT\n\n");

		// Set Jz:
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			pdata->temp.x = q_ * pdata->n*(pdata->viz - pdata->vez);
			++pdata;
		};

		Graph[0].DrawSurface("viz",
			DATA_HEIGHT, (real *)(&(X.pData[0].viz)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
			false, GRAPH_VIZ, &X);

		Graph[1].DrawSurface("vez",
			DATA_HEIGHT, (real *)(&(X.pData[0].vez)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
			false, GRAPH_VEZ, &X);

		Graph[2].DrawSurface("Azdot",
			DATA_HEIGHT, (real *)(&(X.pData[0].Azdot)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].Azdot)),
			true, GRAPH_AZDOT, &X);

		Graph[3].DrawSurface("Jz",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
			false, GRAPH_JZ, &X);

		break;
		/*
	case NEWSTUFF:

		// Too bad substep is not stated. We should divide by substep to give anything meaningful
		// in these graphs.

		// Let temphost3 = vez0
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
		pdata->temp.x = p_temphost3[iVertex + BEGINNING_OF_CENTRAL];
		++pdata;
		};
		Graph[0].DrawSurface("vez0 : vez = vez0 + sigma Ez",
		DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
		AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
		false, // no inner mesh display.
		GRAPH_VEZ0, &X);

		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
		pdata->temp.x = p_OhmsCoeffs_host[iVertex + BEGINNING_OF_CENTRAL].sigma_e_zz;
		++pdata;
		};
		Graph[1].DrawSurface("sigma : vez = vez0 + sigma Ez",
		DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
		AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
		false, // no inner mesh display.
		GRAPH_RESPONSE, &X);

		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
		pdata->temp.x = q_*X.pData[iVertex + BEGINNING_OF_CENTRAL].n*
		(p_OhmsCoeffs_host[iVertex + BEGINNING_OF_CENTRAL].sigma_i_zz
		- p_OhmsCoeffs_host[iVertex + BEGINNING_OF_CENTRAL].sigma_e_zz);

		// Will show something not very useful ---- in a brief instant there
		// isn't much time for second-order (frictional) effects.
		++pdata;
		};
		Graph[2].DrawSurface("Ez=0 v addition: vez0-vez",
		DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
		AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
		false, // no inner mesh display.
		GRAPH_DECEL, &X);
		// Too bad substep is not stated. We should divide by substep to give anything meaningful
		// in these graphs.

		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
		pdata->temp.x = q_*X.pData[iVertex + BEGINNING_OF_CENTRAL].n*
		(p_OhmsCoeffs_host[iVertex + BEGINNING_OF_CENTRAL].sigma_i_zz
		- p_OhmsCoeffs_host[iVertex + BEGINNING_OF_CENTRAL].sigma_e_zz);

		// Will show something not very useful ---- in a brief instant there
		// isn't much time for second-order (frictional) effects.
		++pdata;
		};
		Graph[3].DrawSurface("dynamic conductivity q n sigma : vez = vez0 + sigma Ez",
		DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
		AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
		false, // no inner mesh display.
		GRAPH_DYNCONDUCTIVITY, &X);

		// create graph data for Ez : add Ez_strength*Ezshape to -Azdot/c
		overc = 1.0 / c_;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
		X.pData[iVertex + BEGINNING_OF_CENTRAL].temp.y =
		-X.pData[iVertex + BEGINNING_OF_CENTRAL].Azdot*overc
		+ GetEzShape__(X.pData[iVertex + BEGINNING_OF_CENTRAL].pos.modulus())*EzStrength_;
		}
		Graph[4].DrawSurface("Ez",
		DATA_HEIGHT, (real *)(&(X.pData[0].temp.y)),
		AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)), // use Jz's colour
		false,
		GRAPH_EZ, &X);

		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
		pdata->temp.x = q_ * pdata->n*(pdata->viz - pdata->vez);
		++pdata;
		};
		Graph[5].DrawSurface("Jz",
		DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
		AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
		false, // no inner mesh display.
		GRAPH_JZ, &X);

		break;*/
	case LAPAZ_AZ:
		
		printf("\n\nRefreshGraphs: LAPAZ_AZ\n\n");
		// Assume temp.x contains Lap Az
		Graph[0].DrawSurface("Lap Az",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
			true, GRAPH_LAPAZ, &X);
		Graph[1].DrawSurface("Az",
			DATA_HEIGHT, (real *)(&(X.pData[0].Az)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].Az)),
			true, GRAPH_AZ, &X);
		Graph[2].DrawSurface("Azdot",
			DATA_HEIGHT, (real *)(&(X.pData[0].Azdot)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].Azdot)),
			true, GRAPH_AZDOT, &X);
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			pdata->temp.y = q_ * pdata->n*(pdata->viz - pdata->vez);
			++pdata;
		};
		Graph[3].DrawSurface("Jz",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.y)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.y)),
			false, GRAPH_JZ, &X);

		break;
	case EXYCOMPONENTS:
		/*
		X.Setup_J(); // the others can already exist.

		Graph[0].DrawSurface("Adotxy",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].Adot.x)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].Adot.x)),
		true,
		GRAPH_ADOTXY, &X);
		Graph[1].DrawSurface("Grad phi",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].GradTe)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].GradTe)),
		true, // no inner mesh display: ??
		GRAPH_GRADPHI, &X);
		Graph[2].DrawSurface("Exy",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].E)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].E)),
		true,
		GRAPH_EXY, &X);
		Graph[3].DrawSurface("Jxy",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].Temp.x)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].Temp.x)),
		false, // no inner mesh display.
		GRAPH_JXY, &X);

		*/
		// Set GradTe to grad phi
		break;
	case JXYAXYBZEXY:
		/*
		X.Setup_J(); // the others can already exist.

		Graph[0].DrawSurface("Axy",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].A.x)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].A.x)),
		true,
		GRAPH_AXY, &X);
		Graph[1].DrawSurface("Bz",
		DATA_HEIGHT, (real *)(&(X.X[0].B.z)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].B.z)),
		true, // no inner mesh display: ??
		GRAPH_BZ, &X);
		Graph[2].DrawSurface("Exy",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].E)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].E)),
		true,
		GRAPH_EXY, &X);
		Graph[3].DrawSurface("Jxy",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].Temp.x)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].Temp.x)),
		false, // no inner mesh display.
		GRAPH_JXY, &X);
		*/
		break;
	case EXY_RHO_PHI_PHIDOT:
		/*
		// For this one do n_i-n_e
		pVertex = pX->X;
		for (iVertex = 0; iVertex < pX->numVertices; iVertex++)
		{
		if (pVertex->flags == DOMAIN_VERTEX) {
		pVertex->temp2.x = (pVertex->Ion.mass - pVertex->Elec.mass) / pVertex->AreaCell;
		}
		else {
		pVertex->temp2.x = 0.0;
		};
		++pVertex;
		}

		Graph[0].DrawSurface("phi",
		DATA_HEIGHT, (real *)(&(X.X[0].phi)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].phi)),
		true,
		GRAPH_PHI, &X);
		Graph[1].DrawSurface("phidot",
		DATA_HEIGHT, (real *)(&(X.X[0].phidot)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].phidot)),
		true,
		GRAPH_PHIDOT, &X);
		Graph[2].DrawSurface("Exy",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].E)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].E)),
		true,
		GRAPH_EXY, &X);
		Graph[3].DrawSurface("n_i-n_e",
		DATA_HEIGHT, (real *)(&(X.X[0].temp2.x)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].temp2.x)),
		false, // no inner mesh display.
		GRAPH_NINE, &X);
		*/
		break;
	case EXY_RHO_PHI_JXY:
		// create rho on pVertex->temp2.x ... 
		/*
		pVertex = pX->X;
		for (iVertex = 0; iVertex < pX->numVertices; iVertex++)
		{
		if (pVertex->flags == DOMAIN_VERTEX) {
		pVertex->temp2.x = q * (pVertex->Ion.mass - pVertex->Elec.mass) / pVertex->AreaCell;
		}
		else {
		pVertex->temp2.x = 0.0;
		};
		++pVertex;
		}

		X.Setup_J();

		Graph[0].DrawSurface("phi",
		DATA_HEIGHT, (real *)(&(X.X[0].phi)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].phi)),
		false,
		GRAPH_PHI, &X);
		Graph[1].DrawSurface("Jxy",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].Temp)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].Temp)),
		false, // no inner mesh display: ??
		GRAPH_JXY, &X);
		Graph[2].DrawSurface("Exy",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].E)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].E)),
		false,
		GRAPH_EXY, &X);
		Graph[3].DrawSurface("rho",
		DATA_HEIGHT, (real *)(&(X.X[0].temp2.x)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].temp2.x)),
		false, // no inner mesh display.
		GRAPH_RHO, &X);
		*/
		break;

	case EXY_RHO_BZ_JXY:
		/*
		// create rho on pVertex->temp2.x ...
		pVertex = pX->X;
		for (iVertex = 0; iVertex < pX->numVertices; iVertex++)
		{
		if (pVertex->flags == DOMAIN_VERTEX) {
		pVertex->temp2.x = q * (pVertex->Ion.mass - pVertex->Elec.mass) / pVertex->AreaCell;
		}
		else {
		pVertex->temp2.x = 0.0;
		};
		++pVertex;
		}

		X.Setup_J();

		Graph[0].DrawSurface("Bz",
		DATA_HEIGHT, (real *)(&(X.X[0].B.z)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].B.z)),
		true, // no inner mesh display: ??
		GRAPH_BZ, &X);
		Graph[1].DrawSurface("Jxy",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].Temp)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].Temp)),
		false, // no inner mesh display: ??
		GRAPH_JXY, &X);
		Graph[2].DrawSurface("Exy",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].E)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].E)),
		false,
		GRAPH_EXY, &X);
		Graph[3].DrawSurface("rho",
		DATA_HEIGHT, (real *)(&(X.X[0].temp2.x)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].temp2.x)),
		false, // no inner mesh display.
		GRAPH_RHO, &X);
		*/
		break;

	//case SIGMA_E_J:
		/*
		X.Setup_J(); // the others can already exist.

		Graph[0].DrawSurface("sigma_e_zz",
		DATA_HEIGHT, (real *)(&(X.X[0].sigma_e.zz)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].sigma_e.zz)),
		true,
		GRAPH_SIGMA_E, &X);
		//Graph[1].DrawSurface("v_e_0.z",
		//	DATA_HEIGHT,(real *)(&(X.X[0].v_e_0.z)),
		//	AZSEGUE_COLOUR,(real *)(&(X.X[0].v_e_0.z)),
		//false, // no inner mesh display: ??
		// GRAPH_VE0Z, &X);
		Graph[1].DrawSurface("nsigma",
		DATA_HEIGHT, (real *)(&(X.X[0].xdotdot.x)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].xdotdot.x)),
		true, GRAPH_SIGMATEMP, &X);
		Graph[2].DrawSurface("Ez",
		DATA_HEIGHT, (real *)(&(X.X[0].E.z)),
		FLAG_AZSEGUE_COLOUR, (real *)(&(X.X[0].E.z)), // how to make SEGUE_COLOUR work?
		false, // ??
		GRAPH_EZ, &X);
		Graph[3].DrawSurface("Jz",
		DATA_HEIGHT, (real *)(&(X.X[0].Temp.z)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].Temp.z)),
		false, // no inner mesh display.
		GRAPH_JZ, &X);
		*/
	//	break;


	case TOTAL:
		
		// In this case we have to create data,
		// as we go.
		
		// Best put it here so we can see where
		// data is being populated.

		/*long iVertex;
		Vertex * pVertex = X;
		for (iVertex = 0; iVertex < numVertices; iVertex++)
		{
		if ((pVertex->flags == DOMAIN_VERTEX) || (pVertex->flags == OUTERMOST))
		{
		pVertex->n = (pVertex->Neut.mass + pVertex->Ion.mass) / pVertex->AreaCell;
		pVertex->v = (m_n*pVertex->Neut.mom + m_ion * pVertex->Ion.mom + m_e * pVertex->Elec.mom) /
		(m_n*pVertex->Neut.mass + m_ion * pVertex->Ion.mass + m_e * pVertex->Elec.mass);
		pVertex->T = (pVertex->Neut.heat + pVertex->Ion.heat + pVertex->Elec.heat) /
		(pVertex->Neut.mass + pVertex->Ion.mass + pVertex->Elec.mass);
		pVertex->Temp.x = pVertex->Ion.mass / (pVertex->Neut.mass + pVertex->Ion.mass);
		};
		++pVertex;
		}*/
		//X.CalculateTotalGraphingData();

		printf("\n\nRefreshGraphs: TOTAL\n\n");
		// ought to change this to use variables n,v,T !
		pVertex = X.X;
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			if ((pVertex->flags == DOMAIN_VERTEX) || (pVertex->flags == OUTERMOST))
			{
				pdata->temp.x = pdata->n + pdata->n_n;
				pdata->temp.y = pdata->n / pdata->temp.x;
			}
			++pVertex;
			++pdata;
		}
		Graph[0].DrawSurface("n_n + n_ion",
		DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
		IONISE_COLOUR, (real *)(&(X.pData[0].temp.y)),
		false,
		GRAPH_TOTAL_N, &X);


		pVertex = X.X;
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			if ((pVertex->flags == DOMAIN_VERTEX) || (pVertex->flags == OUTERMOST))
			{
				pdata->temp.x = (m_neutral_*pdata->n_n*pdata->v_n.x
					+ (m_ion_ + m_e_) * pdata->n*pdata->vxy.x) /
					(m_neutral_*pdata->n_n + (m_ion_ + m_e_)*pdata->n);
				pdata->temp.y = (m_neutral_*pdata->n_n*pdata->v_n.y
					+ (m_ion_ + m_e_) * pdata->n*pdata->vxy.y) /
					(m_neutral_*pdata->n_n + (m_ion_ + m_e_)*pdata->n);
			}
			++pVertex;
			++pdata;
		}
		Graph[1].DrawSurface("sum[n_s v_s m_s]/sum[n_s m_s]",
		VELOCITY_HEIGHT, (real *)(&(X.pData[0].temp.x)),
		VELOCITY_COLOUR, (real *)(&(X.pData[0].temp.x)),
		false, // no inner mesh display
		GRAPH_TOTAL_V, &X);
		
		
		//Graph[2].DrawSurface("n_n+n_ion",
		//DATA_HEIGHT, (real *)(&(X.X[0].n)),
		//VELOCITY_COLOUR, (real *)(&(X.X[0].v)),
		//false,
		//GRAPH_TOTAL_N_II, &X);   // ok what we did here? we thought we'd colour with velocity .. but we haven't given ourselves room for 3 temp vars so drop this for now.
		
		
		pVertex = X.X;
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			if ((pVertex->flags == DOMAIN_VERTEX) || (pVertex->flags == OUTERMOST))
			{
				pdata->temp.x = (pdata->n_n*pdata->Tn
							+  pdata->n*(pdata->Ti + pdata->Te)) /
								(pdata->n_n + pdata->n + pdata->n);
			}
			++pVertex;
			++pdata;
		}
		Graph[3].TickRescaling = 1.0 / kB_;
		Graph[3].DrawSurface("sum[n_s T_s]/sum[n_s]",
		DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
		SEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
		false,
		GRAPH_TOTAL_T, &X);
		Graph[3].TickRescaling = 1.0;
		break;
		
	};

	// Graph 2 and 4, in case of species graphs:

	switch (iGraphsFlag) {
	//case SPECIES_NEUTRAL:
	case SPECIES_ION:
	//case SPECIES_ELEC:
	//case TOTAL:

		int offset_v, offset_T;
		offset_v = (real *)(&(X.pData[0].vxy)) - (real *)(&(X.pData[0]));
		offset_T = (real *)(&(X.pData[0].Te)) - (real *)(&(X.pData[0]));

		Graph[2].SetEyePlan(GlobalPlanEye);
		Graph[2].boolDisplayMeshWireframe = true;
		Graph[2].boolClearZBufferBeforeWireframe = true;
		Graph[2].boolDisplayMainMesh = true;
		Graph[2].boolDisplayInnerMesh = false;
		Graph[2].boolDisplayScales = false;

		if (GlobalColoursPlanView == 0)
		{
			// nothing
			Graph[2].mhTech = Graph[2].mFX->GetTechniqueByName("MeshTech");
			Graph[2].SetDataWithColour(X, FLAG_COLOUR_MESH, FLAG_FLAT_MESH, 0, 0,
				GRAPH_FLAT_WIRE_MESH);
			Graph[2].Render(buff, GlobalRenderLabels, &X);

		} else {
			// Tell SDWC not to mess with colourmax if it's a flat mesh.

			if (GlobalColoursPlanView == 1)
			{
				// velocity
				Graph[2].mhTech = Graph[2].mFX->GetTechniqueByName("VelociTech");
				Graph[2].colourmax = Graph[0].colourmax; // match colours

				Graph[2].SetDataWithColour(X, FLAG_VELOCITY_COLOUR, FLAG_FLAT_MESH, offset_v, offset_v,
					GRAPH_FLAT_WIRE_MESH);
				Graph[2].Render(buff, GlobalRenderLabels, &X);
			};
			////else {
			////	// temperature
			////	Graph[2].mhTech = Graph[2].mFX->GetTechniqueByName("SegueTech");
			////	// SegueVS should take maximum as a parameter;
			////	// at least for colours we should prefer an absolute scale for T
			////	// Is it ever used for anything else? Not so far? eps?

			////	Graph[2].SetDataWithColour(X, FLAG_SEGUE_COLOUR, FLAG_FLAT_MESH, offset_T, offset_T,
			////		GRAPH_FLAT_WIRE_MESH);
			////	Graph[2].Render(buff, GlobalRenderLabels, &X);
			////};
		};

		// =================================================================================

		printf("\ngot to here; graph [4]:\n\n");

		Graph[4].boolDisplayKeyButton = false; // it's temperature
		Graph[4].SetEyePlan(GlobalPlanEye);
		Graph[4].boolDisplayMeshWireframe = true;
		Graph[4].boolClearZBufferBeforeWireframe = true;
		Graph[4].boolDisplayMainMesh = true;
		Graph[4].boolDisplayInnerMesh = false;
		Graph[4].boolDisplayScales = false;

		Graph[4].mhTech = Graph[4].mFX->GetTechniqueByName("SegueTech");
		
		Graph[4].SetDataWithColour(X, FLAG_SEGUE_COLOUR, FLAG_FLAT_MESH, offset_T, offset_T,
					GRAPH_FLAT_WIRE_MESH);
		Graph[4].Render(buff, GlobalRenderLabels, &X);
				
		break;
	}
	printf("End of Refreshgraphs\n");

}


IMFSinkWriter *pSinkWriter[NUMAVI];
DWORD izStream[NUMAVI];

HRESULT hresult;

LONGLONG rtStart = 0;

int main()
{
   
	printf("hello\n");


	HINSTANCE hInstance = GetModuleHandle(NULL);
	HWND hwndConsole = GetConsoleWindow();
	WCHAR szInitialFilenameAvi[512];
	MSG msg;
	HDC hdc;
	//	HACCEL hAccelTable;
	real x, y, temp;
	int i, j;
	float a1, a2, a3, a4; 
	//HWND hwndConsole;
	FILE * fp;
	extern char Functionalfilename[1024];
	
	int nDevices, iWhich;
	hipDeviceProp_t prop;
	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++) {
		
		hipGetDeviceProperties(&prop, i);
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", prop.name);
		printf("  Memory Clock Rate (KHz): %d\n",
			prop.memoryClockRate);
		printf("  Memory Bus Width (bits): %d\n",
			prop.memoryBusWidth);
		printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
			2.0*prop.memoryClockRate*(prop.memoryBusWidth / 8) / 1.0e6);

		if (prop.memoryBusWidth == 384) iWhich = i;
	}
	printf("Picked %d \n", iWhich);
	getch(); 

	hipSetDevice(iWhich); // K40?
	hipDeviceReset();

	size_t uFree, uTotal;
	hipMemGetInfo(&uFree, &uTotal);
	printf("Memory on device: uFree %zd uTotal %zd\n", uFree, uTotal);

	HRESULT hr = CoInitializeEx(NULL, COINIT_APARTMENTTHREADED);
	if (!SUCCEEDED(hr)) {
		printf("CoInitializeEx failed. press p\n");
		while (getch() != 'p');
		exit(23233);
	}
	hr = MFStartup(MF_VERSION);
	if (!SUCCEEDED(hr)) {
		printf("MFStartup failed. press p\n");
		while (getch() != 'p');
		exit(23234);
	}


	h = TIMESTEP;
	evaltime = 0.0; // gets updated before advance

	memset(Historic_powermax, 0, 200 * sizeof(int));
	memset(Historic_powermin, 0, 200 * sizeof(int));

	ZeroMemory(Historic_max, 512 * HISTORY * sizeof(float));
	ZeroMemory(Historic_min, 512 * HISTORY * sizeof(float));
	GlobalStepsCounter = 0; steps_remaining = 0; steps_remaining_CPU = 0;

	SetConsoleTitle("2D 1/16 annulus DPF simulation");
	Sleep(40);
	//hwndConsole = FindWindow(NULL, "2D 1/16 annulus DPF simulation");
	MoveWindow(hwndConsole, 0, 0, SCREEN_WIDTH - VIDEO_WIDTH - 10, SCREEN_HEIGHT - 30, TRUE);

	report_time(0);

	int filetag = 0;
	do { 
		filetag++;
		sprintf(Functionalfilename, FUNCTIONALFILE_START "%03d.txt", filetag);
	} while ((_access(Functionalfilename, 0)) != -1);

	printf("\n\nopening %s \n", Functionalfilename);
	fp = fopen(Functionalfilename, "w");
	if (fp == 0) {
		printf("error with %s \n", Functionalfilename);
		getch();
	}
	else {
		printf("opened %s \n", Functionalfilename);
	}; 
	fprintf(fp, "GSC evaltime Area neut.N ion.N elec.N neut.r ion.r elec.r SDneut.r SDion.r SDelec.r "
		" neut.vr neut.vth neut.vz  ion.vr ion.vth ion.vz elec.vr elec.vth elec.vz neut.heat ion.heat elec.heat neut.T ion.T elec.T "
		" neut.mnvv/3 ion.mnvv/3 elec.mnvv/3 elec.force(vxB)r within3.6 elec.Bth EE BB Heatings and dT changes - see code \n");
	fclose(fp);
	 
	X1.Initialise(1); // Set evaltime first
	X2.Initialise(2);
	X3.Initialise(3);
	printf("Got to here 1\n");	    
	{
		X4.Initialise(4);
		printf("Got to here 2\n");
		X4.CreateTilingAndResequence2(&X1);
		X4.CreateTilingAndResequence2(&X2);
		X4.CreateTilingAndResequence2(&X3);
		printf("Got to here 3\n");
		// 
		// Dropping it for now so we can pursue solving equations first.
		//  
	}
	X1.Recalculate_TriCentroids_VertexCellAreas_And_Centroids();
	X1.EnsureAnticlockwiseTriangleCornerSequences_SetupTriMinorNeighboursLists();
	X1.SetupMajorPBCTriArrays();
	X2.Recalculate_TriCentroids_VertexCellAreas_And_Centroids();
	X2.EnsureAnticlockwiseTriangleCornerSequences_SetupTriMinorNeighboursLists();
	X2.SetupMajorPBCTriArrays();
	X3.Recalculate_TriCentroids_VertexCellAreas_And_Centroids();
	X3.EnsureAnticlockwiseTriangleCornerSequences_SetupTriMinorNeighboursLists();
	X3.SetupMajorPBCTriArrays();
	printf("Got to here 4\n");
	X1.InitialPopulate();
	X2.InitialPopulate();
	X3.InitialPopulate();

	X1.Create4Volleys();
	X2.Create4Volleys();
	X3.Create4Volleys();

	pTriMesh = &X1;

	pX = &X1;
	pXnew = &X2;
	
	GlobalBothSystemsInUse = 0;

	printf(report_time(1));
	printf("\n");
	report_time(0);
	
	// Window setup
	LoadString(hInstance, IDS_APP_TITLE, szTitle, 1024);
	LoadString(hInstance, IDC_F2DVALS, szWindowClass, 1024);
	wcex.cbSize = sizeof(WNDCLASSEX);
	wcex.style = CS_HREDRAW | CS_VREDRAW;
	wcex.lpfnWndProc = WndProc;
	wcex.cbClsExtra = 0;
	wcex.cbWndExtra = 0;
	wcex.hInstance = hInstance;
	wcex.hIcon = LoadIcon(hInstance, MAKEINTRESOURCE(IDI_F2DVALS));
	wcex.hCursor = LoadCursor(NULL, IDC_ARROW);
	wcex.hbrBackground = (HBRUSH)(COLOR_WINDOW + 1);
	wcex.lpszMenuName = MAKEINTRESOURCE(IDR_MENU1);
	wcex.lpszClassName = szWindowClass;
	wcex.hIconSm = LoadIcon(wcex.hInstance, MAKEINTRESOURCE(IDI_SMALL));
	if (RegisterClassEx(&wcex) == 0) {
		char buff[128];
		MessageBox(NULL, "RegisterClassEx failed", itoa(GetLastError(), buff, 10), MB_OK);
	};

	printf("SCREEN_WIDTH %d VIDEO_WIDTH %d VIDEO_HEIGHT %d \n",
		SCREEN_WIDTH, VIDEO_WIDTH, VIDEO_HEIGHT);

	hWnd = CreateWindowEx(NULL, szWindowClass, szTitle, WS_BORDER | WS_POPUP,
		SCREEN_WIDTH - VIDEO_WIDTH - 5, 0, VIDEO_WIDTH + 5, VIDEO_HEIGHT + 20, NULL, NULL, hInstance, NULL);
	if (!hWnd) {
		DWORD dword = GetLastError();
		char buff[128];
		MessageBox(NULL, "CreateWindowEx failed", itoa(dword, buff, 10), MB_OK);
		return dword;
	}
	// This is sending a message to WndProc before any of the following happens.

	ShowWindow(hWnd, SW_SHOWNORMAL);
	UpdateWindow(hWnd);

	hwndGraphics = hWnd;

	xzscale = 2.0 / 0.1; // very zoomed in. Now what?

	DXChk(Direct3D.Initialise(hWnd, hInstance, VIDEO_WIDTH, VIDEO_HEIGHT));

	// With Field Of View = PI/4 used this:
	/*
	GlobalEye.x = 0.0f;
	GlobalEye.y = 12.4f;  //7.2f;
	GlobalEye.z = -18.0f + 2.5*xzscale;//DEVICE_RADIUS_INSULATOR_OUTER*xzscale;//-17.8f+

	GlobalLookat.x = 0.4f;
	GlobalLookat.y = 3.0f;
	GlobalLookat.z = DEVICE_RADIUS_INITIAL_FILAMENT_CENTRE * xzscale;

	GlobalPlanEye.x = 0.0f;
	GlobalPlanEye.y = 35.0f;
	GlobalPlanEye.z = (3.44 + 4.1)*0.5*xzscale;

	GlobalPlanEye2.x = -0.1f;
	GlobalPlanEye2.y = 19.5f;
	GlobalPlanEye2.z = 2.8*xzscale;

	GlobalPlanLookat.x = GlobalPlanEye.x;
	GlobalPlanLookat.y = 0.0f;
	GlobalPlanLookat.z = GlobalPlanEye.z + 0.0001;

	GlobalPlanLookat2.x = GlobalPlanEye2.x;
	GlobalPlanLookat2.y = 0.0f;
	GlobalPlanLookat2.z = GlobalPlanEye2.z + 0.0001;*/

	GlobalEye.x = -10.4f;
	GlobalEye.y = 16.4f;  //7.2f;
	GlobalEye.z = 44.0f;

	GlobalLookat.x = 1.20f;
	GlobalLookat.y = 3.0f;
	GlobalLookat.z = 72.2f;

	GlobalPlanEye.x = 2.9f;
	GlobalPlanEye.y = 17.97f;
	GlobalPlanEye.z = 71.95f;

	GlobalPlanEye2.x = -0.1f;
	GlobalPlanEye2.y = 19.5f;
	GlobalPlanEye2.z = 2.8*xzscale;
	 
	GlobalPlanLookat.x = GlobalPlanEye.x;
	GlobalPlanLookat.y = 0.0f;
	GlobalPlanLookat.z = GlobalPlanEye.z + 0.0001;

	GlobalPlanLookat2.x = GlobalPlanEye2.x;
	GlobalPlanLookat2.y = 0.0f;
	GlobalPlanLookat2.z = GlobalPlanEye2.z + 0.0001;
	
	newEye.x = 0.0f;
	newEye.y = 0.1f;
	newEye.z = 40.0f;
	newLookat.x = 0.0f;
	newLookat.y = 0.0f;
	newLookat.z = 72.0f;

						 // Add vectors in parallel.
	hipError_t cudaStatus;

	if (DXChk(Graph[0].InitialiseWithoutBuffers(0, 0, GRAPH_WIDTH, GRAPH_HEIGHT, GlobalEye, GlobalLookat)) +
		DXChk(Graph[0].InitialiseBuffers(X1))
		)
	{
		PostQuitMessage(200);
	};
	if (DXChk(Graph[1].InitialiseWithoutBuffers(0, GRAPH_HEIGHT, GRAPH_WIDTH, GRAPH_HEIGHT, GlobalEye, GlobalLookat)) +
		DXChk(Graph[1].InitialiseBuffers(X1))
		)
	{
		PostQuitMessage(201);
	};
	if (DXChk(Graph[2].InitialiseWithoutBuffers(GRAPH_WIDTH, 0, GRAPH_WIDTH, GRAPH_HEIGHT, GlobalPlanEye, GlobalPlanLookat)) +
		DXChk(Graph[2].InitialiseBuffers(X1))
		)
	{
		PostQuitMessage(202);
	};
	if (DXChk(Graph[3].InitialiseWithoutBuffers(GRAPH_WIDTH, GRAPH_HEIGHT, GRAPH_WIDTH, GRAPH_HEIGHT, GlobalEye, GlobalLookat)) +
		DXChk(Graph[3].InitialiseBuffers(X1))
		)
	{
		PostQuitMessage(203);
	};
	   
	if (NUMGRAPHS > 4) {

		if (DXChk(Graph[4].InitialiseWithoutBuffers(GRAPH_WIDTH * 2, 0, GRAPH_WIDTH, GRAPH_HEIGHT, GlobalPlanEye, GlobalPlanLookat)) +
			DXChk(Graph[4].InitialiseBuffers(X1))
			)
		{
			PostQuitMessage(204);
		};

		if (DXChk(Graph[5].InitialiseWithoutBuffers(GRAPH_WIDTH * 2, GRAPH_HEIGHT, GRAPH_WIDTH, GRAPH_HEIGHT, GlobalEye, GlobalLookat)) +
			DXChk(Graph[5].InitialiseBuffers(X1))
			)
		{
			PostQuitMessage(204);
		};
		if (DXChk(Graph[6].InitialiseWithoutBuffers(0, 0, GRAPH_WIDTH*2, GRAPH_HEIGHT, newEye, GlobalLookat, true)) +
			DXChk(Graph[6].InitialiseBuffers(X1))
			)
		{
			PostQuitMessage(204);
		};
		if (DXChk(Graph[7].InitialiseWithoutBuffers(0, GRAPH_HEIGHT, GRAPH_WIDTH * 2, GRAPH_HEIGHT, newEye, GlobalLookat, true)) +
			DXChk(Graph[7].InitialiseBuffers(X1))
			)
		{
			PostQuitMessage(204);
		};
	};

	Graph[0].bDisplayTimestamp = false;
	Graph[1].bDisplayTimestamp = false;
	Graph[2].bDisplayTimestamp = false;
	Graph[3].bDisplayTimestamp = false;
	Graph[4].bDisplayTimestamp = true;
	Graph[5].bDisplayTimestamp = false;
	Graph[6].bDisplayTimestamp = true;
	Graph[7].bDisplayTimestamp = false;

	Direct3D.pd3dDevice->GetBackBuffer(0, 0, D3DBACKBUFFER_TYPE_MONO, &p_backbuffer_surface);

	if (DXChk(p_backbuffer_surface->GetDC(&surfdc), 1000))
		MessageBox(NULL, "GetDC failed", "oh dear", MB_OK);

	surfbit = CreateCompatibleBitmap(surfdc, VIDEO_WIDTH, VIDEO_HEIGHT); // EXTRAHEIGHT = 90
	SelectObject(surfdc, surfbit);
	dibdc = CreateCompatibleDC(surfdc);

	long VideoWidth = VIDEO_WIDTH;
	long VideoHeight = VIDEO_HEIGHT;

	// pasted here just to set up format:
	bitmapinfo.bmiHeader.biSize = sizeof(BITMAPINFO);
	bitmapinfo.bmiHeader.biWidth = VideoWidth;
	bitmapinfo.bmiHeader.biHeight = VideoHeight;
	bitmapinfo.bmiHeader.biPlanes = 1;
	bitmapinfo.bmiHeader.biBitCount = 24;
	bitmapinfo.bmiHeader.biCompression = BI_RGB; // uncompressed  
	bitmapinfo.bmiHeader.biSizeImage = bitmapinfo.bmiHeader.biHeight;
	bitmapinfo.bmiHeader.biXPelsPerMeter = 3000;
	bitmapinfo.bmiHeader.biYPelsPerMeter = 3000;
	bitmapinfo.bmiHeader.biClrUsed = 0;
	bitmapinfo.bmiHeader.biClrImportant = 0;
	bitmapinfo.bmiColors->rgbBlue = 0;
	bitmapinfo.bmiColors->rgbRed = 0;
	bitmapinfo.bmiColors->rgbGreen = 0;
	bitmapinfo.bmiColors->rgbReserved = 0;
	// dimension DIB and set up pointer to bits
	dib = CreateDIBSection(dibdc, &bitmapinfo, DIB_RGB_COLORS, &lpvBits, NULL, 0);
	SelectObject(dibdc, dib);

	BitBlt(dibdc, 0, 0, VIDEO_WIDTH, VIDEO_HEIGHT, surfdc, 0, 0, SRCCOPY);
	
	rtStart = 0; // timeslice : where to place frames into mp4 files.

	for (i = 0; i < NUMAVI; i++)
	{
		swprintf(szInitialFilenameAvi, L"%s%s_%s", FOLDER, szmp4[i], INITIALMP4);
	
		pSinkWriter[i] = NULL;

		hr = InitializeSinkWriter(&(pSinkWriter[i]), &(izStream[i]), szInitialFilenameAvi);

		if (!SUCCEEDED(hr)) {
			printf("Failed to create mp4 file %d %ls \n", i, szmp4[i]);
		}

		// hAvi[i] = CreateAvi(szInitialFilenameAvi, AVIFRAMEPERIOD, NULL);
		//if (hAvi[i] == 0) {
		//	printf("Failed to create avi file %d", i);
		//	getch(); getch(); getch();
		//}
	};
	
	printf("got to here: Initialized SinkWriters \n");
	getch();

	// 1000/25 = 40
	//ZeroMemory(&opts, sizeof(opts));
	//opts.fccHandler = mmioFOURCC('D', 'I', 'B', ' ');//('d','i','v','x');
	//opts.dwFlags = 8;

	//for (i = 0; i < NUMAVI; i++)
	//{
	//	hresult = SetAviVideoCompression(hAvi[i], dib, &opts, false, hWnd); // always run this for every avi file but can
	//															  // call with false as long as we know opts contains valid information. 
	//	if (hresult != 0) {
	//		printf("error: i = %d, hresult = %d", i, (long)hresult);
	//		getch(); getch(); getch();
	//	};
	//};

	counter = 0;
	//ReleaseDC(hWnd,surfdc);
	p_backbuffer_surface->ReleaseDC(surfdc);
	GlobalCutaway = true; // dies if true
	
	RefreshGraphs(*pX, GlobalSpeciesToGraph);
	
	Direct3D.pd3dDevice->Present(NULL, NULL, NULL, NULL);


	// Main message loop:
	memset(&msg, 0, sizeof(MSG));
	while (msg.message != WM_QUIT)
	{
		if (PeekMessage(&msg, NULL, 0U, 0U, PM_REMOVE))
		{
			TranslateMessage(&msg);
			DispatchMessage(&msg);
		} else {
			Direct3D.pd3dDevice->Present(NULL, NULL, NULL, NULL);
		};
	};

	UnregisterClass(szWindowClass, wcex.hInstance);
	
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

/* Auxiliary routine: printing a matrix */
void print_matrix(char* desc, lapack_int m, lapack_int n, double* a, lapack_int lda) {
	lapack_int i, j;
	printf("\n %s\n", desc);
	for (i = 0; i < m; i++) {
		for (j = 0; j < n; j++) printf(" %2.5E", a[i*lda + j]);
		printf("\n");
	}
}
 
/* Auxiliary routine: printing a vector of integers */
void print_int_vector(char* desc, lapack_int n, lapack_int* a) {
	lapack_int j;
	printf("\n %s\n", desc);
	for (j = 0; j < n; j++) printf(" %6i", a[j]);
	printf("\n");
}

LRESULT CALLBACK WndProc(HWND hWnd, UINT message, WPARAM wParam, LPARAM lParam)
{

	f64 lowest_vez;
	long iLow, iMinor;
	Triangle * pTri;
	Vertex * pVertex;
	
	long izTri[128];
	 
	static bool bInvoked_cuSyst = false;
	static long GSCCPU = 0;
	int iAntiskips;
	int wmId, wmEvent;
	int i, j, ctr;
	PAINTSTRUCT ps;
	HDC hdc;
	real time_back_for_Adot;
	FILE * file, *fp;
	int maxeerr, count, iMin;
	WCHAR buf1000[1024];
	char buf1001[1024];
	int attempts;
	real store_h;
	char ch, o;
	int failed;
	RECT rect;
	real TotalArea, TotalCharge;
	long iVertex;
	real mass_avg, mass_SD, mass_min, mass_max;
	OPENFILENAME ofn;       // common dialog box structure
	char szFile[260];       // buffer for file name
	char szFilter[1000]; // buffer for file filter
	char szfilter[256];
	char buffer[256];

	TriMesh * temp;

	static const real XCENTRE2 = DEVICE_RADIUS_INITIAL_FILAMENT_CENTRE * sin(PI / 32.0);
	static const real XCENTRE1 = -XCENTRE2;
	static const real YCENTRE = DEVICE_RADIUS_INITIAL_FILAMENT_CENTRE * cos(PI / 32.0);
	 
	switch (message)
	{
	case WM_CREATE: 

		// Don't ever try doing initialisation here;
		// That should be done manually from the menus.		

		break;
		 
	case WM_COMMAND:
		wmId = LOWORD(wParam);
		wmEvent = HIWORD(wParam);

		printf("\nWM_COMMAND: wmId %d\n\n", wmId);
	
		// Ensure that display menu items are consecutive IDs.
		// Parse the menu selections:
		switch (wmId)
		{
			
		case ID_DISPLAY_ONE_D:
			// printf("\a\n");
			// Don't know why resource.h is not working;
			// Maybe some #define overwrites it with 40024.
			//wmId += 50007 - 40024;
			GlobalSpeciesToGraph = ONE_D;
			printf("\nGlobalSpeciesToGraph = %d \n", GlobalSpeciesToGraph);
			RefreshGraphs(*pX, GlobalSpeciesToGraph);
			Direct3D.pd3dDevice->Present(NULL, NULL, NULL, NULL);
			break;
		// int const GraphFlags[NUMAVI] = { SPECIES_ION, OVERALL, JZAZBXYEZ, OHMSLAW, ONE_D, IONIZEGRAPH };
		case ID_DISPLAY_ION:
			GlobalSpeciesToGraph = SPECIES_ION;
			printf("\nGlobalSpeciesToGraph = %d \n", GlobalSpeciesToGraph);
			RefreshGraphs(*pX, GlobalSpeciesToGraph);
			Direct3D.pd3dDevice->Present(NULL, NULL, NULL, NULL);
			break;

		case ID_DISPLAY_TOTAL:
			GlobalSpeciesToGraph = OVERALL;
			printf("\nGlobalSpeciesToGraph = %d \n", GlobalSpeciesToGraph);
			RefreshGraphs(*pX, GlobalSpeciesToGraph);
			Direct3D.pd3dDevice->Present(NULL, NULL, NULL, NULL);
			break;

		case ID_DISPLAY_JZAZBXYEZ:
			GlobalSpeciesToGraph = JZAZBXYEZ;
			printf("\nGlobalSpeciesToGraph = %d \n", GlobalSpeciesToGraph);
			RefreshGraphs(*pX, GlobalSpeciesToGraph);
			Direct3D.pd3dDevice->Present(NULL, NULL, NULL, NULL);
			break;
		case ID_DISPLAY_IONIZEGRAPHS:
			GlobalSpeciesToGraph = IONIZEGRAPH;
			printf("\nGlobalSpeciesToGraph = %d \n", GlobalSpeciesToGraph);
			RefreshGraphs(*pX, GlobalSpeciesToGraph);
			Direct3D.pd3dDevice->Present(NULL, NULL, NULL, NULL);
			break;
		case ID_DISPLAY_OHMS:
			GlobalSpeciesToGraph = OHMSLAW;
			printf("\nGlobalSpeciesToGraph = %d \n", GlobalSpeciesToGraph);
			RefreshGraphs(*pX, GlobalSpeciesToGraph);
			Direct3D.pd3dDevice->Present(NULL, NULL, NULL, NULL);
			break;
		case ID_DISPLAY_DTGRAPH:

			GlobalSpeciesToGraph = DTGRAPH;
			printf("\nGlobalSpeciesToGraph = %d \n", GlobalSpeciesToGraph);
			RefreshGraphs(*pX, GlobalSpeciesToGraph);
			Direct3D.pd3dDevice->Present(NULL, NULL, NULL, NULL);

			break;
		case ID_DISPLAY_ACCELGRAPH:

			GlobalSpeciesToGraph = ACCELGRAPHS;
			printf("\nGlobalSpeciesToGraph = %d \n", GlobalSpeciesToGraph);
			RefreshGraphs(*pX, GlobalSpeciesToGraph);
			Direct3D.pd3dDevice->Present(NULL, NULL, NULL, NULL);

			break;

		case ID_DISPLAY_TENSOROHMS:

			GlobalSpeciesToGraph = OHMS2;
			printf("\nGlobalSpeciesToGraph = %d \n", GlobalSpeciesToGraph);
			RefreshGraphs(*pX, GlobalSpeciesToGraph);
			Direct3D.pd3dDevice->Present(NULL, NULL, NULL, NULL);

			break;

		case ID_DISPLAY_ACCELRELZ:
			GlobalSpeciesToGraph = ARELZ;
			printf("\nGlobalSpeciesToGraph = %d \n", GlobalSpeciesToGraph);
			RefreshGraphs(*pX, GlobalSpeciesToGraph);
			Direct3D.pd3dDevice->Present(NULL, NULL, NULL, NULL);

			break;
		case ID_DISPLAY_SIGMAEJ:

			i = wmId - ID_DISPLAY_NEUT;
			GlobalSpeciesToGraph = i;
			printf("\nGlobalSpeciesToGraph = %d \n", GlobalSpeciesToGraph);
			RefreshGraphs(*pX, GlobalSpeciesToGraph);
			Direct3D.pd3dDevice->Present(NULL, NULL, NULL, NULL);
			break;

		case ID_HELP_ABOUT:
			DialogBox(hInst, MAKEINTRESOURCE(IDD_ABOUTBOX), hWnd, About);
			break;
		case ID_FILE_EXIT:
			DestroyWindow(hWnd);
			break;

		case ID_FILE_SAVECAMERA:
			ZeroMemory(&ofn, sizeof(ofn));
			ofn.lStructSize = sizeof(ofn);
			ofn.hwndOwner = hWnd;
			ofn.lpstrFile = szFile;
			// Set lpstrFile[0] to '\0' so that GetOpenFileName does not 
			// use the contents of szFile to initialize itself.
			ofn.lpstrFile[0] = '\0';
			ofn.nMaxFile = sizeof(szFile);
			memcpy(szfilter, "All\0*.*\0cam\0*.CAM\0\0", 19); // strcpy stops at first null !!
			ofn.lpstrFilter = szfilter; //"All\0*.*\0Dat\0*.DAT\0\0";	// summat weird about that example code
			ofn.nFilterIndex = 1;
			ofn.lpstrFileTitle = NULL;
			ofn.nMaxFileTitle = 0;
			ofn.lpstrInitialDir = NULL;
			ofn.Flags = OFN_OVERWRITEPROMPT;
			ofn.lpstrTitle = NULL;

			if (GetSaveFileName(&ofn) == TRUE)
			{
				printf("\nsaving camera...");
				fp = fopen(ofn.lpstrFile, "wt");
				if (fp == 0) {
					printf("save failed.\n");
				}
				else {
					fprintf(fp, "%f %f %f ", GlobalPlanEye.x, GlobalPlanEye.y, GlobalPlanEye.z);
					fprintf(fp, "%f %f %f ", GlobalLookat.x, GlobalLookat.y, GlobalLookat.z);
					fprintf(fp, "%f %f %f ", GlobalEye.x, GlobalEye.y, GlobalEye.z);
					fprintf(fp, "%f %f %f ", GlobalPlanLookat.x, GlobalPlanLookat.y, GlobalPlanLookat.z);
					fclose(fp);
					printf("done\n");
				};
			}
			else {
				printf("there was an issue\n");
			};
			break;
		case ID_FILE_LOADCAMERA:
			// Initialize OPENFILENAME
			ZeroMemory(&ofn, sizeof(ofn));
			ofn.lStructSize = sizeof(ofn);
			ofn.hwndOwner = hWnd;
			ofn.lpstrFile = szFile;
			// Set lpstrFile[0] to '\0' so that GetOpenFileName does not 
			// use the contents of szFile to initialize itself.
			ofn.lpstrFile[0] = '\0';
			ofn.nMaxFile = sizeof(szFile);
			memcpy(szfilter, "All\0*.*\0*.cam\0*.Cam\0\0", 21); // strcpy stops at first null !!
			ofn.lpstrFilter = szfilter; //"All\0*.*\0*.Dat\0*.DAT\0\0";	// summat weird about that example code
			ofn.nFilterIndex = 1;
			ofn.lpstrFileTitle = NULL;
			ofn.nMaxFileTitle = 0;
			ofn.lpstrInitialDir = NULL;
			ofn.Flags = OFN_PATHMUSTEXIST | OFN_FILEMUSTEXIST;
			// Display the Open dialog box. 
			if (GetOpenFileName(&ofn) == TRUE)
			{
				printf("\nloading camera...");
				fp = fopen(ofn.lpstrFile, "rt");
				if (fp == 0) {
					printf("failed.\n");
				}
				else {
					rewind(fp);
					fscanf(fp, "%f %f %f ", &(GlobalPlanEye.x), &(GlobalPlanEye.y), &(GlobalPlanEye.z));
					fscanf(fp, "%f %f %f ", &(GlobalLookat.x), &(GlobalLookat.y), &(GlobalLookat.z));
					fscanf(fp, "%f %f %f ", &(GlobalEye.x), &(GlobalEye.y), &(GlobalEye.z));
					fscanf(fp, "%f %f %f ", &(GlobalPlanLookat.x), &(GlobalPlanLookat.y), &(GlobalPlanLookat.z));
					fclose(fp);
				};
				RefreshGraphs(*pX, GlobalSpeciesToGraph); // sends data to graphs AND renders them
				Direct3D.pd3dDevice->Present(NULL, NULL, NULL, NULL);

			}
			else {
				printf("file error camera\n");
			};
			break; 
		case ID_FILE_LOADGPU:

			// Initialize OPENFILENAME:
			ZeroMemory(&ofn, sizeof(ofn));
			ofn.lStructSize = sizeof(ofn);
			ofn.hwndOwner = hWnd;
			ofn.lpstrFile = szFile;
			//
			// Set lpstrFile[0] to '\0' so that GetOpenFileName does not 
			// use the contents of szFile to initialize itself.
			ofn.lpstrFile[0] = '\0';
			ofn.nMaxFile = sizeof(szFile);
			//strcpy(szFilter, "All\0*.*\0Dat\0*.DAT\0\0");
			memcpy(szfilter, "All\0*.*\0Dat\0*.DAT\0\0", 19); // strcpy stops at first null !!
			ofn.lpstrFilter = szfilter; //"All\0*.*\0Dat\0*.DAT\0\0";	// summat weird about that example code
			ofn.nFilterIndex = 1;
			ofn.lpstrFileTitle = NULL;
			ofn.nMaxFileTitle = 0;
			ofn.lpstrInitialDir = NULL;
			ofn.Flags = OFN_PATHMUSTEXIST | OFN_FILEMUSTEXIST;

			// Display the Open dialog box. 
			if (GetOpenFileName(&ofn) == TRUE)
			{
				if (bInvoked_cuSyst == false) {
					bInvoked_cuSyst = true;

					pX->EnsureAnticlockwiseTriangleCornerSequences_SetupTriMinorNeighboursLists();
					pX->Average_n_T_to_tris_and_calc_centroids_and_minorpos();

					pX->Create4Volleys(); // THIS SHOULD NOT ALWAYS BE HERE !!
					printf("Called Create4Volleys! This should be removed in favour of loaded iVolley.\n");

					cuSyst_host.InvokeHost();
					cuSyst_host.PopulateFromTriMesh(pX);
					cuSyst_host2.InvokeHost();
					cuSyst_host2.PopulateFromTriMesh(pX);
					// transfer information.

					PerformCUDA_Invoke_Populate(
						&cuSyst_host,
						NUMVERTICES,
						pX->InnermostFrillCentroidRadius,
						pX->OutermostFrillCentroidRadius,
						pX->numStartZCurrentTriangles,
						pX->numEndZCurrentTriangles);
				};

				cuSyst_host.Load(ofn.lpstrFile);
			};
			printf("Populate *pX\n");
			cuSyst_host.PopulateTriMesh(pX);
			printf("send to device\n");
			cuSyst_host.SendToDevice(cuSyst1);
			printf("done\n");

			// Debug: redelaun on load:
			pX->RefreshVertexNeighboursOfVerticesOrdered();
	//		pX->Redelaunerize(true, true);

			// This isn't actually helpful?

			// pX->RefreshVertexNeighboursOfVerticesOrdered();
			// pX->X[89450-BEGINNING_OF_CENTRAL].GetTriIndexArray(izTri);
//			printf("89450 : %d %d %d %d %d %d \n",
//				izTri[0], izTri[1], izTri[2], izTri[3], izTri[4], izTri[5]);
//
			pX->EnsureAnticlockwiseTriangleCornerSequences_SetupTriMinorNeighboursLists();			 
			//	pX->Average_n_T_to_tris_and_calc_centroids_and_minorpos(); // Obviates some of our flip calcs to replace tri n,T 
			// not sure if needed .. just for calc centroid .. they do soon get wiped out anyway.
			cuSyst_host.PopulateFromTriMesh(pX);
			cuSyst_host.SendToDevice(cuSyst1); // check this is right
			cuSyst2.CopyStructuralDetailsFrom(cuSyst1);
			cuSyst3.CopyStructuralDetailsFrom(cuSyst1);
				// Let's assume these always carry through during GPU runs.
				// It certainly does not work as it stands if you don't populate them all the same, put it that way!!
			printf("sent back re-delaunerized system\n");

			break;
		case ID_FILE_SAVEBINARY:
			// Initialize OPENFILENAME
			ZeroMemory(&ofn, sizeof(ofn));
			ofn.lStructSize = sizeof(ofn);
			ofn.hwndOwner = hWnd;
			ofn.lpstrFile = szFile;
			// Set lpstrFile[0] to '\0' so that GetOpenFileName does not 
			// use the contents of szFile to initialize itself.
			ofn.lpstrFile[0] = '\0';
			ofn.nMaxFile = sizeof(szFile);
			memcpy(szfilter, "All\0*.*\0*.dat\0*.Dat\0\0", 21); // strcpy stops at first null !!
			ofn.lpstrFilter = szfilter; //"All\0*.*\0Dat\0*.DAT\0\0";	// summat weird about that example code
			ofn.nFilterIndex = 1;
			ofn.lpstrFileTitle = NULL;
			ofn.nMaxFileTitle = 0;
			ofn.lpstrInitialDir = NULL;
			ofn.Flags = OFN_OVERWRITEPROMPT;
			ofn.lpstrTitle = NULL;
			// Display the Open dialog box. 
			if (GetSaveFileName(&ofn) == TRUE)
			{
				printf("\nsaving system...");
				pX->Save(ofn.lpstrFile);
				printf("done\n");
			}
			else {
				printf("there was an issue\n");
			};
			break;
		case ID_FILE_SAVETEXT:
			// Initialize OPENFILENAME
			ZeroMemory(&ofn, sizeof(ofn));
			ofn.lStructSize = sizeof(ofn);
			ofn.hwndOwner = hWnd;
			ofn.lpstrFile = szFile;
			//
			// Set lpstrFile[0] to '\0' so that GetOpenFileName does not 
			// use the contents of szFile to initialize itself.
			ofn.lpstrFile[0] = '\0';
			ofn.nMaxFile = sizeof(szFile);
			//strcpy(szFilter,"All\0*.*\0Text\0*.TXT\0");
			memcpy(szfilter, "All\0*.*\0Dat\0*.DAT\0\0", 19); // strcpy stops at first null !!
			ofn.lpstrFilter = szfilter; //"All\0*.*\0Dat\0*.DAT\0\0";	// summat weird about that example code
			ofn.nFilterIndex = 1;
			ofn.lpstrFileTitle = NULL;
			ofn.nMaxFileTitle = 0;
			ofn.lpstrInitialDir = NULL;
			ofn.Flags = OFN_OVERWRITEPROMPT;
			ofn.lpstrTitle = NULL;
			// Display the Open dialog box. 
			if (GetSaveFileName(&ofn) == TRUE)
			{
				printf("\nsaving system...");
				pX->SaveText(ofn.lpstrFile);
				printf("done\n");
			}
			else {
				printf("there was an issue\n");
			};
			break;
		case ID_FILE_LOAD:

			// Initialize OPENFILENAME:
			ZeroMemory(&ofn, sizeof(ofn));
			ofn.lStructSize = sizeof(ofn);
			ofn.hwndOwner = hWnd;
			ofn.lpstrFile = szFile;
			//
			// Set lpstrFile[0] to '\0' so that GetOpenFileName does not 
			// use the contents of szFile to initialize itself.
			ofn.lpstrFile[0] = '\0';
			ofn.nMaxFile = sizeof(szFile);
			//strcpy(szFilter, "All\0*.*\0Dat\0*.DAT\0\0");
			memcpy(szfilter, "All\0*.*\0Dat\0*.DAT\0\0", 19); // strcpy stops at first null !!
			ofn.lpstrFilter = szfilter; //"All\0*.*\0Dat\0*.DAT\0\0";	// summat weird about that example code
			ofn.nFilterIndex = 1;
			ofn.lpstrFileTitle = NULL;
			ofn.nMaxFileTitle = 0;
			ofn.lpstrInitialDir = NULL;
			ofn.Flags = OFN_PATHMUSTEXIST | OFN_FILEMUSTEXIST;

			// Display the Open dialog box. 
			if (GetOpenFileName(&ofn) == TRUE)
			{
				pX->Load(ofn.lpstrFile);
				printf("\ndoing nothing...");
			};
		break;

		case ID_RUN_SIMULATIONSTEPS:

			GlobalSwitchBox = 0;
			DialogBox(hInst, MAKEINTRESOURCE(IDD_DIALOG1), hWnd, SetupBox);
			// that will not return with steps_remaining unset.
			 
			if (steps_remaining > 0)
				SetTimer(hWnd, 1, 1, NULL); // 1 millisecond delay

			break;

		case ID_RUN_SIMULATIONSTEPS_CPU:

			GlobalSwitchBox = 0;
			steps_remaining_CPU = 1;
			// that will not return with steps_remaining unset.

			if (steps_remaining_CPU > 0)
				SetTimer(hWnd, 2, 1, NULL); // 1 millisecond delay

			break;

		case ID_INITIALISE_ZAPTHEBACK:

			Zap_the_back();
			printf("done");

			RefreshGraphs(*pX, GlobalSpeciesToGraph); // sends data to graphs AND renders them
			Direct3D.pd3dDevice->Present(NULL, NULL, NULL, NULL);


			break;

		case ID_RUN_STOP:

			steps_remaining = 0;
			steps_remaining_CPU = 0;
			break;
		case ID_INITIALISE_IONISATIONSTEPS:
			break;

		default:
			return DefWindowProc(hWnd, message, wParam, lParam);
		}
		break;

	case WM_TIMER:
		
		KillTimer(hWnd, wParam);
		report_time(0);
		if (wParam == 1)
		{
			if (bInvoked_cuSyst == false) {
				bInvoked_cuSyst = true;

				pX->EnsureAnticlockwiseTriangleCornerSequences_SetupTriMinorNeighboursLists();
				pX->Average_n_T_to_tris_and_calc_centroids_and_minorpos();
//
//				printf("tri 340: %d %d %d \n%1.14E %1.14E \n%1.14E %1.14E \n%1.14E %1.14E\n",
//					pX->T[340].cornerptr[0] - pX->X, pX->T[340].cornerptr[1] - pX->X, pX->T[340].cornerptr[2] - pX->X,
//					pX->T[340].cornerptr[0]->pos.x, pX->T[340].cornerptr[0]->pos.y,
//					pX->T[340].cornerptr[1]->pos.x, pX->T[340].cornerptr[1]->pos.y,
//					pX->T[340].cornerptr[2]->pos.x, pX->T[340].cornerptr[2]->pos.y);
//				printf("tri 340 periodic %d \n", pX->T[340].periodic);
//				getch(); 
				 
				cuSyst_host.InvokeHost();
				cuSyst_host.PopulateFromTriMesh(pX);
				cuSyst_host2.InvokeHost();
				cuSyst_host2.PopulateFromTriMesh(pX);
				 
				//		cuSyst_host.Output("n0.txt");

				PerformCUDA_Invoke_Populate(
					&cuSyst_host,
					NUMVERTICES,
					pX->InnermostFrillCentroidRadius,
					pX->OutermostFrillCentroidRadius,
					pX->numStartZCurrentTriangles,
					pX->numEndZCurrentTriangles);
			}

			// Run 1 step:
			printf("evaltime %1.9E\n", evaltime);

			//	PerformCUDA_RunStepsAndReturnSystem_Debug(&cuSyst_host, &cuSyst_host2, pX, &X3, pXnew);

			PerformCUDA_RunStepsAndReturnSystem(&cuSyst_host);

			//	printf("Stamp GPU over CPU y/n:");
			//	do {
			//		o = getch();
			//	} while ((o != 'y') && (o != 'n'));
			//	printf("%c\n\n", o);
			//	if (o == 'y') 

				// Auto-save system:
			if (GlobalStepsCounter % DATA_SAVE_FREQUENCY == 0)
			{
				sprintf(szFile, "auto%d.dat", GlobalStepsCounter);
				// SAVE cuSyst:
				cuSyst_host.Save(szFile);
			}

			// even number of steps should lead us back to pX having it
			steps_remaining--;
			GlobalStepsCounter++;

			printf("Done steps: %d   ||   Remaining this run: %d\n\n", GlobalStepsCounter, steps_remaining);

			if ((GlobalStepsCounter % GRAPHICS_FREQUENCY == 0) ||
				(GlobalStepsCounter % REDELAUN_FREQUENCY == 0) ||
				(steps_remaining == 0))
			{
				cuSyst_host.PopulateTriMesh(pX); // vertex n is populated into the minor array available on CPU
				printf("pulled back to host\n");
			}
		}
		else {
			pX->Advance(pXnew, &X3);
			temp = pX;
			pX = pXnew;
			pXnew = temp;

			steps_remaining_CPU--;
			GSCCPU++;
			printf("Done steps CPU: %d   ||   Remaining this run: %d\n\n", GSCCPU, steps_remaining_CPU);
			  
			sprintf(buf1001, "autosaveCPU%d.dat", GSCCPU);
			pX->Save(buf1001);
			printf("saved as %s\n", buf1001);
		};
		printf("%s\n", report_time(1));
		  
		if (GlobalStepsCounter % GRAPHICS_FREQUENCY == 0)
		{
			// make video frames:
			for (i = 0; i < NUMAVI; i++)
			{
				printf("i = %d \n", i);
				RefreshGraphs(*pX, GraphFlags[i]); // sends data to graphs AND renders them
															   //	::PlanViewGraphs1(*pX);
				printf(".DISHMOPS.\n");
				Direct3D.pd3dDevice->Present(NULL, NULL, NULL, NULL);
				printf("got to here 7\n");

				if (DXChk(p_backbuffer_surface->GetDC(&surfdc), 100))
					MessageBox(NULL, "GetDC failed", "oh dear", MB_OK);
				//SelectObject(surfdc,surfbit);
				BitBlt(dibdc, 0, 0, VIDEO_WIDTH, VIDEO_HEIGHT, surfdc, 0, 0, SRCCOPY);
				p_backbuffer_surface->ReleaseDC(surfdc);

				//GetDIBits(dibdc, dib, 0, VIDEO_HEIGHT, dwBits, &bmi, 0);
				// just use lpvBits
				
				// getting hAvi[i] == 0 for the last one.
				// But on debug? No such thing? Same.

				printf("Adding frame to %d : \n", i);
				hresult = WriteFrame(pSinkWriter[i], izStream[i], rtStart);

				//hresult = AddAviFrame(hAvi[i], dib);
				if (hresult != 0) printf("\n******************************************************* \n"
					"hresult = %d\n********************************************** \n", hresult);
				
			};
			rtStart += VIDEO_FRAME_DURATION;

			// sprintf(szFile, "System_%d", GlobalStepsCounter);
			// pX->SaveText(szFile);
		};

		if (GlobalStepsCounter % (AVI_FILE_PINCHOFF_FREQUENCY * GRAPHICS_FREQUENCY) == 0)
		{
			for (i = 0; i < NUMAVI; i++)
			{
				// now have to pinch out avi file and make a new one
				pSinkWriter[i]->Finalize();
				// CloseAvi(hAvi[i]);

				swprintf(buf1000, L"%s%s_%d.mp4", FOLDER, szmp4[i], GlobalStepsCounter);
				//hAvi[i] = CreateAvi(buf1000, AVIFRAMEPERIOD, NULL);
				SafeRelease(&pSinkWriter[i]);
				pSinkWriter[i] = NULL;
				rtStart = 0;
				if (!SUCCEEDED(InitializeSinkWriter(&(pSinkWriter[i]), &(izStream[i]), szmp4[i])))
				{
					printf("Failed to create %ls \n", buf1000);
					getch();
				}

				//hresult = SetAviVideoCompression(hAvi[i], dib, &opts, false, hWnd);
				//if (hresult != 0) printf("\n******************************************************* \n"
				//	"SetAviVideoCompression: hresult = %d\n********************************************** \n", hresult);

			};
		};
		 
		RefreshGraphs(*pX,GlobalSpeciesToGraph); // sends data to graphs AND renders them
		Direct3D.pd3dDevice->Present( NULL, NULL, NULL, NULL );
		 
		if (GlobalStepsCounter % REDELAUN_FREQUENCY == 0)
		{
			Setup_residual_array(); // We have not specifically checked that cuSyst1 is the
			// most up-to-date, but it doesn't matter really.

			pX->RefreshVertexNeighboursOfVerticesOrdered();
			long iFlips = pX->Redelaunerize(true, true);
			// Send back to GPU:
			pX->EnsureAnticlockwiseTriangleCornerSequences_SetupTriMinorNeighboursLists();

		//	Appears in lots of places so hard to believe data is not updated.
		//	There is no wrapping on GPU?... or is there?
		//	Need to debug inside routine and find out what it is doing for these two triangles.


			//	pX->Average_n_T_to_tris_and_calc_centroids_and_minorpos(); // Obviates some of our flip calcs to replace tri n,T 
			// not sure if needed .. just for calc centroid .. they do soon get wiped out anyway.
			
			cuSyst_host.PopulateFromTriMesh(pX);// 1. Does it update lists? --- some had to be updated on CPU first.
			 
			// Seems to copy structural information as well as data. n is copied from n_minor on CPU.
			
			//cuSyst1.SendToHost(cuSyst_host2);
			//cuSyst_host.ReportDifferencesHost(cuSyst_host2);
			cuSyst_host.SendToDevice(cuSyst1); 
			cuSyst2.CopyStructuralDetailsFrom(cuSyst1);
			cuSyst3.CopyStructuralDetailsFrom(cuSyst1);
			// Let's assume these always carry through during GPU runs.
			// It certainly does not work as it stands if you don't populate them all the same, put it that way!!

			// We don't actually know which system is pointed to by pX1 that is the initial system
			// for the steps --- so just copy it over all of them
			cuSyst_host.SendToDevice(cuSyst2);
			cuSyst_host.SendToDevice(cuSyst3); 
			// There almost certainly is a better way. But this is unimportant for now.

			printf("sent back re-delaunerized system\n");
			 
			// Now reset A values more carefully in the sent-back system:
			 
			if (iFlips == 0) {
				printf(" NO DELAUNAY FLIPS");
				for (int sj = 0; sj < 10; sj++) printf("-\n");
			}
			if (iFlips > 0) {

				Go_visit_the_other_file();
				
			};
			
		};
		
		if (steps_remaining > 0) {
			SetTimer(hWnd, 1, DELAY_MILLISECS, NULL);
			printf("Waiting %d milliseconds to allow user input.\n", DELAY_MILLISECS);
		};
		if (steps_remaining_CPU > 0) {
			SetTimer(hWnd, 2, DELAY_MILLISECS, NULL);
			printf("Waiting %d milliseconds to allow user input.\n", DELAY_MILLISECS);
		};

		/*
		if (wParam == 1) {
			sprintf(buf1000, "autosaveGPU%d.dat", GlobalStepsCounter);
		} else {
			sprintf(buf1000, "autosaveCPU%d.dat", GSCCPU);
		}
		pX->Save(buf1000);
		printf("saved as %s\n", buf1000);
		
		lowest_vez = 0.0;
		iLow = 0;
		pTri = pX->T;
		for (iMinor = 0; iMinor < BEGINNING_OF_CENTRAL; iMinor++)
		{
			if ((pTri->u8domain_flag == DOMAIN_TRIANGLE) && (pX->pData[iMinor].vez < lowest_vez)) {
				lowest_vez = pX->pData[iMinor].vez;
				iLow = iMinor;
			}
			++pTri;
		}
		printf("Tris: lowest_vez %1.14E iLow %d \n", lowest_vez, iLow);
		iLow = 0;
		lowest_vez = 0.0;
		pVertex = pX->X;
		for (; iMinor < NMINOR; iMinor++)
		{
			if ((pVertex->flags == DOMAIN_VERTEX) && (pX->pData[iMinor].vez < lowest_vez)) {
				lowest_vez = pX->pData[iMinor].vez;
				iLow = iMinor;
			}
			++pVertex;
		}
		printf("Vertices: lowest_vez %1.14E iLow %d \n\n", lowest_vez, iLow);


		printf("save ascii?");
		do {
			o = getch();
		} while ((o != 'y') && (o != 'n'));
		printf("%c\n", o);
		if (o == 'y') {
			sprintf(buf1000, "SaveGPUtext1_trackedAA");
			pX->SaveText(buf1000);
			printf("Ascii file saved %s.\n",buf1000);
		}
		*/

		printf("steps_remaining GPU: %d  CPU: %d\n",steps_remaining, steps_remaining_CPU);
		
		
		break;

	case WM_KEYDOWN:

		switch (wParam)
		{
		case 'W':
			GlobalEye.z += 1.0f;
			printf("GlobalEye %f %f %f  \n",
				GlobalEye.x, GlobalEye.y, GlobalEye.z);
			break;
		case 'S':
			GlobalEye.z -= 1.0f;
			printf("GlobalEye %f %f %f  \n",
				GlobalEye.x, GlobalEye.y, GlobalEye.z);
			break;
		case 'A':
			GlobalEye.x -= 0.8f;
			printf("GlobalEye %f %f %f  \n",
				GlobalEye.x, GlobalEye.y, GlobalEye.z);
			break;
		case 'D':
			GlobalEye.x += 0.8f;
			printf("GlobalEye %f %f %f  \n",
				GlobalEye.x, GlobalEye.y, GlobalEye.z);
			break;
		case 'E':
			GlobalEye.y += 0.8f;
			printf("GlobalEye %f %f %f  \n",
				GlobalEye.x, GlobalEye.y, GlobalEye.z);
			break;
		case 'C':
			GlobalEye.y -= 0.8f;
			printf("GlobalEye %f %f %f  \n",
				GlobalEye.x, GlobalEye.y, GlobalEye.z);
			break;

		case 'V':
			GlobalLookat.z -= 0.4f;
			printf("GlobalEye %f %f %f  GlobalLookat %f %f %f\n",
				GlobalEye.x, GlobalEye.y, GlobalEye.z, GlobalLookat.x, GlobalLookat.y, GlobalLookat.z);
			break;
		case 'R':
			GlobalLookat.z += 0.4f;
			printf("GlobalEye %f %f %f  GlobalLookat %f %f %f\n",
				GlobalEye.x, GlobalEye.y, GlobalEye.z, GlobalLookat.x, GlobalLookat.y, GlobalLookat.z);
			break;
		case 'F':
			GlobalLookat.x -= 0.4f;
			printf("GlobalEye %f %f %f  GlobalLookat %f %f %f\n",
				GlobalEye.x, GlobalEye.y, GlobalEye.z, GlobalLookat.x, GlobalLookat.y, GlobalLookat.z);
			break;
		case 'G':
			GlobalLookat.x += 0.4f;
			printf("GlobalEye %f %f %f  GlobalLookat %f %f %f\n",
				GlobalEye.x, GlobalEye.y, GlobalEye.z, GlobalLookat.x, GlobalLookat.y, GlobalLookat.z);
			break;
		case 'T':
			GlobalLookat.y += 0.4f;
			printf("GlobalLookat %f %f %f\n",
				GlobalLookat.x, GlobalLookat.y, GlobalLookat.z);
			break;
		case 'B':
			GlobalLookat.y -= 0.4f;
			printf("GlobalLookat %f %f %f\n",
				GlobalLookat.x, GlobalLookat.y, GlobalLookat.z);
			break;
		case '+':
			GlobalCutaway = !GlobalCutaway;
			break;
		case 'Y':
		case '<':
			GlobalEye.x = -10.4; GlobalEye.y = 16.4; GlobalEye.z = 44.0;
			GlobalLookat.x = -3.6; GlobalLookat.y = 3.0; GlobalLookat.z = 72.2;
			printf("GlobalEye %f %f %f  GlobalLookat %f %f %f\n",
				GlobalEye.x, GlobalEye.y, GlobalEye.z, GlobalLookat.x, GlobalLookat.y, GlobalLookat.z);

			GlobalPlanEye.x = 7.1; GlobalPlanEye.y = 11.5; GlobalPlanEye.z = 71.35;
			printf("GlobalPlanEye %f %f %f\n",
				GlobalPlanEye.x, GlobalPlanEye.y, GlobalPlanEye.z);

			break;
		case '_':
		case '-':
		case '>':
			GlobalPlanEye.x = 7.0; GlobalPlanEye.y = 14.0; GlobalPlanEye.z = 71.0;
			printf("GlobalPlanEye %f %f %f\n",
				GlobalPlanEye.x, GlobalPlanEye.y, GlobalPlanEye.z);
			break;

		case 'U':
			GlobalPlanEye.z += 0.6f;
			printf("GlobalPlanEye %f %f %f\n",
				GlobalPlanEye.x, GlobalPlanEye.y, GlobalPlanEye.z);
			break;
		case 'J':
			GlobalPlanEye.z -= 0.6f;
			printf("GlobalPlanEye %f %f %f\n",
				GlobalPlanEye.x, GlobalPlanEye.y, GlobalPlanEye.z);
			break;
		case 'H':
			GlobalPlanEye.x -= 0.6f;
			printf("GlobalPlanEye %f %f %f\n",
				GlobalPlanEye.x, GlobalPlanEye.y, GlobalPlanEye.z);
			break;
		case 'K':
			GlobalPlanEye.x += 0.6f;
			printf("GlobalPlanEye %f %f %f\n",
				GlobalPlanEye.x, GlobalPlanEye.y, GlobalPlanEye.z);
			break;
		case 'I':
			GlobalPlanEye.y *= 1.25f;
			printf("GlobalPlanEye %f %f %f\n",
				GlobalPlanEye.x, GlobalPlanEye.y, GlobalPlanEye.z);
			break;
		case 'M':
			GlobalPlanEye.y *= 0.8f;
			printf("GlobalPlanEye %f %f %f\n",
				GlobalPlanEye.x, GlobalPlanEye.y, GlobalPlanEye.z);
			break;
		case 'N':
			GlobalboolDisplayMeshWireframe = !GlobalboolDisplayMeshWireframe;
			//Graph1.boolDisplayMeshWireframe = (!(Graph1.boolDisplayMeshWireframe));
			break;
		case '9':
			GlobalRenderLabels = false;
			break;
		case '5':
			GlobalRenderLabels = true;
			GlobalWhichLabels = 0;// iTri
			break;
		case '8':
			GlobalRenderLabels = true;
			GlobalWhichLabels = 1;//T
			break;
		case '7':
			GlobalRenderLabels = true;
			GlobalWhichLabels = 2;//v
			break;
		case '6':
			GlobalRenderLabels = true;
			GlobalWhichLabels = 3;	//n
			break;
		case '1':
			GlobalColoursPlanView = 1;//v
			break;
		case '4':
			GlobalColoursPlanView = 0;//nothing
			break;
		case '2':
			GlobalColoursPlanView = 2;//T
			break;
		case '0':
			steps_remaining = 0;
			break;
			
		case 'Q':
			newEye.z += 5.0f;
			printf("newEye.z %1.9E\n", newEye.z);
			break;
		case 'P':
			newEye.z -= 5.0f;
			printf("newEye.z %1.9E\n", newEye.z);
			break;
		case 'X':
			newEye.y += 5.0f;			
			printf("newEye.y %1.9E\n", newEye.y);
			break;
		case 'Z':
			newEye.y -= 5.0f;
			printf("newEye.y %1.9E\n", newEye.y);
			break;
		case 'O':
			newLookat.z -= 3.0f;
			printf("newLookat.z %1.9E\n", newLookat.z);
			break;
		case ';':
		case ':':
			newLookat.z += 3.0f;
			printf("newLookat.z %1.9E\n", newLookat.z);
			break;

		default:
			return DefWindowProc(hWnd, message, wParam, lParam);

		};

		//PlanViewGraphs1(*pX);

		RefreshGraphs(*pX, GlobalSpeciesToGraph); // sends data to graphs AND renders them
		Direct3D.pd3dDevice->Present(NULL, NULL, NULL, NULL);

		break;
	case WM_PAINT:
		// Not sure, do we want to do this?
		//	RefreshGraphs(*pX,); // sends data to graphs AND renders them
		GetUpdateRect(hWnd, &rect, FALSE);
		if (Direct3D.pd3dDevice != NULL)
			Direct3D.pd3dDevice->Present(&rect, &rect, NULL, NULL);

		ValidateRect(hWnd, NULL);
		break;
	case WM_DESTROY:
		DeleteObject(dib);
		DeleteDC(dibdc);
		for (i = 0; i < NUMAVI; i++)
		{
			pSinkWriter[i]->Finalize();
			SafeRelease(&(pSinkWriter[i]));
		}
		// CloseAvi(hAvi[i]);

		//  _controlfp_s(0, cw, _MCW_EM); // Line A
		PerformCUDA_Revoke();

		MFShutdown();
		CoUninitialize();

		PostQuitMessage(0);
		break;
	default:
		return DefWindowProc(hWnd, message, wParam, lParam);
	}
	return 0;
}

// Message handler for about box.
INT_PTR CALLBACK About(HWND hDlg, UINT message, WPARAM wParam, LPARAM lParam)
{
	UNREFERENCED_PARAMETER(lParam);
	switch (message)
	{
	case WM_INITDIALOG:
		return (INT_PTR)TRUE;

	case WM_COMMAND:
		if (LOWORD(wParam) == IDOK || LOWORD(wParam) == IDCANCEL)
		{
			EndDialog(hDlg, LOWORD(wParam));
			return (INT_PTR)TRUE;
		}
		break;
	}
	return (INT_PTR)FALSE;
}


INT_PTR CALLBACK SetupBox(HWND hDlg, UINT message, WPARAM wParam, LPARAM lParam)
{
	UNREFERENCED_PARAMETER(lParam);
	char buffer[2048];
	char string[1024];
	real newh;

	switch (message)
	{
	case WM_INITDIALOG:
		sprintf(buffer, "New h? (present = %1.10E)", h);
		if (GlobalSwitchBox)
			SetDlgItemText(hDlg, IDC_STATIC, buffer);
		return (INT_PTR)TRUE;

	case WM_COMMAND:
		if (LOWORD(wParam) == IDOK)
		{
			// try to read data from edit control:
			GetDlgItemText(hDlg, IDC_EDIT1, buffer, 2048);
			if (GlobalSwitchBox == 0)
			{
				// 
				steps_remaining = atoi(buffer);
				if (steps_remaining >= 0)
				{
					EndDialog(hDlg, LOWORD(wParam));
				}
				else {
					MessageBox(NULL, "incorrect value", "Enter a nonnegative integer.", MB_OK);
				};
			}
			else {
				newh = atof(buffer);
				if (newh > 0.0)
				{
					EndDialog(hDlg, LOWORD(wParam));
					sprintf(string, "h = %1.10E\n", newh);
					h = newh;
					MessageBox(NULL, string, "New value of h", MB_OK);
				}
				else {
					MessageBox(NULL, "no good", "Negative h entered", MB_OK);
				};
			};
			return (INT_PTR)TRUE;
		}
		break;
	}
	return (INT_PTR)FALSE;
}

