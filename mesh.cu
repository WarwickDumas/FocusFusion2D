#include "hip/hip_runtime.h"

#define TRIS_TO_TRIS             0
#define TRI_AND_WEDGE_TO_WEDGES  1
#define WEDGES_TO_WEDGE_AND_TRI  2

#include "mesh.h"
#include "globals.h"

extern long steps_remaining;

#define CHOSEN 28516  

int GlobalWedgeSwitch; 
smartlong GlobalAffectedTriIndexList;
bool bGlobalAffectedTriArray = false;
extern bool flaglist[NMINOR];

	TriMesh::TriMesh()
	{
		
#ifdef DYNAMIC_VERTS
		X = NULL;
		Xdomain = NULL;
		T = NULL;
	//	InnerX = NULL;
	//	InnerT = NULL;
#endif

		numVertices = 0;
		numDomainVertices = 0;
	//	numInnerVertices = 0;
	//	numInnerTriangles = 0;

		numTriangles = 0;
		numTrianglesAllocated = 0;
		numRows = 0;

		numInnerVertices = 0;

		ZeroMemory(AuxX,sizeof(Vertex *)*NUM_COARSE_LEVELS);
	//	ZeroMemory(AuxT,sizeof(AuxVertex *)*NUM_COARSE_LEVELS);

		//numVertsLow = 0;
		//numVertsOuter = 0;
	//	insulator_verts = 0;
	//	outer_verts = 0;
	};

	TriMesh::~TriMesh()
	{
	//	if (X != NULL) delete [] X;
	//	X = NULL;
	//	if (T != NULL) delete [] T;
	//	T = NULL;
		
	//if (InnerX != NULL) delete [] InnerX;
		//if (InnerT != NULL) delete [] InnerT;
		
		//numInnerVertices = 0;
		//numInnerTriangles = 0;
		numVertices = 0;
		numTriangles = 0;
		numTrianglesAllocated = 0;
		
		Disconnected.clear();
		TriangleHeap.clear();
		ScratchSearchTris.clear();
	};


real inline distsq(Vector2 const u1, Vector2 const u2)
{
	return (u1.x-u2.x)*(u1.x-u2.x)+(u1.y-u2.y)*(u1.y-u2.y);
};

real GetCos(const Vector2 & u1, const Vector2 & centre, const Vector2 & u2)
{
	Vector2 diff1 = u1-centre;
	Vector2 diff2 = u2-centre;

	// a.b = |a| |b| cos theta

	real a_dot_b = diff1.x*diff2.x+diff1.y*diff2.y;
	real costheta = a_dot_b/sqrt((diff1.x*diff1.x+diff1.y*diff1.y)*(diff2.x*diff2.x+diff2.y*diff2.y));
	return costheta;
};

real GetCos(const Vector2 & v1, const Vector2 & v2)
{
	real costheta = v1.dot(v2)/sqrt(
		(v1.x*v1.x+v1.y*v1.y)*(v2.x*v2.x+v2.y*v2.y));
	// for this version, send the 2 difference vectors.
	return costheta;
}

Vector2 Vertex::PopulateContiguousPosition__Guesswork(Vertex * pVertex)
	{
		Vector2 u = pos;
		if ((pVertex->pos.x/pVertex->pos.y < -0.5*GRADIENT_X_PER_Y)
			&& ( u.x/u.y > 0.5*GRADIENT_X_PER_Y))
			u = Anticlockwise*u; 	
		if ((pVertex->pos.x/pVertex->pos.y > 0.5*GRADIENT_X_PER_Y)
			&& ( u.x/u.y < -0.5*GRADIENT_X_PER_Y))
			u = Clockwise*u;
		return u;
	}





/*
void Triangle::Set(Vertex * pV1, Vertex * pV2, Vertex * pV3)
{
	// set the cornerptr to these values and tell vertices to add this triangle also.

	// Set Triangle::flags based on Vertex::flags
	// and Triangle::periodic based on Vertex::iScratch which records relative wrapping apparently.

	cornerptr[0] = pV1;
	cornerptr[1] = pV2;
	cornerptr[2] = pV3;
	pV1->triangles.add(this);
	pV2->triangles.add(this);
	pV3->triangles.add(this);
	
	periodic = cornerptr[0]->iScratch + cornerptr[1]->iScratch + cornerptr[2]->iScratch;
	while (periodic >= 3) periodic -= 3;
	while (periodic < 0) periodic += 3;

}*/	// Do not have this: with integer arrays need to use TriMesh object.


char Triangle::InferRelativeWrapping(Vertex * pVert, Vertex * pVertDisco)
{
	if (periodic == 0) return 0;

	int c1;

	if (periodic == 1)
	{
		c1 = GetLeftmostIndex();
		if (cornerptr[c1] == pVert) {
			// pVert is wrapped relative to the others, Clockwise
			return 1;
		};
		if (cornerptr[c1] == pVertDisco) {
			return -1;
		};

		return 0; // the other vertex was wrapped only
	};
	
	// got here => periodic == 2
	
	c1 = GetRightmostIndex();
	if (cornerptr[c1] == pVert) {
		// pVert is wrapped Anticlockwise
		return -1;
	};
	if (cornerptr[c1] == pVertDisco) return 1;
	return 0;

}


int TriMesh::SeekVertexInsideTriangle(Vertex * v1,
									  Vertex * v2,
									  Vertex * v3,          // up to 4 points to check for
									  Vertex * v4,           // in order of preference;
									  Triangle * pSeedTri,    // Seed triangle to begin radar
									  Vertex ** ppReturnVert,  // address for returning guilty vertex
									  Triangle ** ppReturnTri) // address for returning triangle that contains
{
	// non-directed radar outwards from pSeedTri to find if a triangle contains
	// one of the listed vertices;
	// return 0,1,2,3 for which one it was; 
	// fill in *ppReturnVert with that vertex address.
	
	// return -1 if no vertex in tri interior found

	// Note that if something is a vertex of a triangle it cannot be considered to be within that triangle.
	
	Vertex * pVert[4];
	int i;
	Triangle *pTri;

	pVert[0] = v1;
	pVert[1] = v2;
	pVert[2] = v3;
	pVert[3] = v4;

	for (i = 0; i < 4; i++)
	if (pSeedTri->ContainsPointInterior(pVert[i]))
	{
		*ppReturnVert = pVert[i];
		*ppReturnTri = pSeedTri;
		return i;
	};

	ScratchSearchTris.clear();
	ScratchSearchTris.add(pSeedTri-T);
	long checkcaret = 1; // the next unchecked element
	// add neighbours to list:
	ScratchSearchTris.add_unique(pSeedTri->neighbours[0]-T);
	ScratchSearchTris.add_unique(pSeedTri->neighbours[1]-T);
	ScratchSearchTris.add_unique(pSeedTri->neighbours[2]-T);

	do
	{
		// for each unchecked triangle: check it; if it's not a hit, add all its neighbours (unique add) to list.
		// make sure we did not run out of neighbours:

		if (checkcaret >= ScratchSearchTris.len) {
			// 3rror
			printf("Did not find enough triangle neighbours in radar search! \n");
			getch();
		};

		pTri = T+ScratchSearchTris.ptr[checkcaret];
		
		for (i = 0; i < 4; i++)
		if (pTri->ContainsPointInterior(pVert[i]))
		{
			*ppReturnVert = pVert[i];
			*ppReturnTri = pTri;
			return i;
		};
		
		// if we're still here, it did not contain one of them. Add all its neighbours to look in:
		ScratchSearchTris.add_unique(pTri->neighbours[0]-T);
		ScratchSearchTris.add_unique(pTri->neighbours[1]-T);
		ScratchSearchTris.add_unique(pTri->neighbours[2]-T);		
		checkcaret++;
	} while (checkcaret < 2000); // arbitrary cutoff

	return -1;	
}

int TriMesh::DestroyOverlaps(int max_attempts)
{
	// Let's just search all triangles and see if we can find shared edges.
	
	// Can assume neighbours[0] is the triangle that shares edge [vertex1, vertex2]
	
	long iTri;
//	int j;
	Triangle * pTri;//, * pTri2;
	//Vertex * pvert, *q2;
	//Vector2 u;
	//bool test;
	long actions = 0;
	int attempts = 0;
	//int q2_cornerind;
	long iNeigh = 0; // not used?
//	Triangle *pNeigh; // not useful

	// FIRST DISCONNECT VERTICES UNTIL NO MORE OVERLAPS EXIST.

	TriangleHeap.clear();
	Disconnected.clear();

	do
	{
		actions = 0;
		pTri = T;// &(T[0]);
		for (iTri = 0; iTri < numTriangles; iTri++)
		{
			// Note that inside ExamineNeighbour, we do a test for if pTri has already been scrapped.
	
			// Look at each neighbour to make a pair and see if edge is shared.
			// If it is, disconnect some vertex and reconnect.

			if (pTri->neighbours[0] != pTri) // do not do comparison at the edge of the domain!
				actions += ExamineNeighbourAndDisconnectIfNeeded(pTri,0,1);
			if (pTri->neighbours[1] != pTri)
				actions += ExamineNeighbourAndDisconnectIfNeeded(pTri,1,0);
			if (pTri->neighbours[2] != pTri)			
				actions += ExamineNeighbourAndDisconnectIfNeeded(pTri,2,0);
			 
			++pTri;
		};
		//printf("TriMesh::DestroyOverlaps Actions taken: %d \n",actions);
		++attempts;
		
	} while ((actions > 0) && (attempts < max_attempts));


	printf("TriMesh::DestroyOverlaps disconnected: %d \n",Disconnected.len);
	//if (Disconnected.len > 0) getch();

	if (attempts == max_attempts)
	{
		printf("****************************\n");
		printf("Max attempts to destroy overlaps by disconnecting points exceeded.\n");
		getch();
	};

	if (TriangleHeap.len != 2*Disconnected.len)
	{ 
		printf("****************************\n");
		printf("Wrong amount of triangles and disco points accumulated:\n %d tris, %d points\n",
			TriangleHeap.len,Disconnected.len);
		getch();
	};

	// Now we have a heap of disconnected points and unused triangles.

	// NOW RECONNECT POINTS IN SAFETY

	while (Disconnected.len > 0)
	{
		// Reconnect 
		ReconnectLastPointInDiscoArray(); // this will also remove it from array and remove tri's from their array
	};
	
	// Replenish vertex neighbour array in untangled mesh:
	// have checked that it does not need to be intermediately kept refreshed above.
	if (attempts > 1) this->RefreshVertexNeighboursOfVerticesOrdered();
	
	return attempts-1;
	
}


	
int TriMesh::ExamineNeighbourAndDisconnectIfNeeded(Triangle * pTri, int opp, int c1)
{
	// c1 is on the edge we check
	// opp is opposite the edge we check
	// If pTri is a wedge, opp will not ==2.

	Triangle * pTri2, *pTriContain;
	Vertex * q2;
	int q2_cornerind; // not used for anything
	Vertex * pVertUse, * pVertRogue;
	Vector2 u,unsh2;
	Triangle *pNeigh; // not used for anything
	bool test;
	int which;
	int iMapped,iUnmapped;
	
	if (TriangleHeap.contains(pTri-T)) // can end up calling this function for scrapped triangle -- do nothing
		return 0;
	
	pTri2 = pTri->neighbours[opp];
	q2 = pTri2->ReturnUnsharedVertex(pTri,&q2_cornerind);
	
	// If they're both wedges, q2_cornerind will be 0 or 1
	
	if (pTri2->periodic == 0)
	{
		if (pTri->periodic == 0)
		{
			test = pTri->TestAgainstEdge(
						 q2->pos.x,q2->pos.y, 
						 c1,          // edge corner
						 opp,         // the opposite point
						 &pNeigh      // not used
						 );
		} else {
			// pTri is periodic but pTri2 is not.
			// This means the shared edge is inside;
			
			// pTri->periodic == 1 means shared edge on right so map
			// q2 to compare with pTri on the left because we map pTri fully to the left.
			
			// pTri->periodic == 2 means q2 is already on the left and we make sure
			// we map pTri->c1 to the left if it's needing to be mapped.

			if (pTri->periodic == 1)
			{
				unsh2 = Anticlockwise*q2->pos;
				test = pTri->TestAgainstEdge(
						 unsh2.x,unsh2.y, 
						 c1,          // edge corner
						 opp,         // the opposite point
						 &pNeigh      // not used
						 );
			} else {
				// We edited to ensure that c1 point does get mapped to left.
				test = pTri->TestAgainstEdge(
						 q2->pos.x,q2->pos.y, 
						 c1,          // edge corner
						 opp,         // the opposite point
						 &pNeigh      // not used
						 );
			};
		};

	} else {

		// pTri2->periodic > 0

		if (pTri->periodic == 0)
		{

			// pTri2 is periodic but pTri is not.
			// This means shared edge is inside.
			// pTri2->periodic == 1 => that one is q2 and so it must be mapped
			// to the RIGHT to be with pTri
			// pTri2->periodic == 2 => q2 is on the right but pTri is on the left
			if (pTri2->periodic == 1)
			{
				unsh2 = Clockwise*q2->pos;						
			} else {
				unsh2 = Anticlockwise*q2->pos;
			};
			test = pTri->TestAgainstEdge(
					 unsh2.x,unsh2.y, 
					 c1,          // edge corner
					 opp,         // the opposite point
					 &pNeigh      // not used
					 );
		} else {
			// since pTri is periodic, comparison is done on the left
			// is q2 on the left?
			if (pTri2->periodic == 1)
			{
				iMapped = pTri2->GetLeftmostIndex();
				if (iMapped == q2_cornerind) {
					// already mapped
					test = pTri->TestAgainstEdge(
						 q2->pos.x,q2->pos.y, 
						 c1,          // edge corner
						 opp,         // the opposite point
						 &pNeigh      // not used
						 );
				} else {
					unsh2 = Anticlockwise*q2->pos;

					test = pTri->TestAgainstEdge(
						 unsh2.x,unsh2.y, 
						 c1,          // edge corner
						 opp,         // the opposite point
						 &pNeigh      // not used
						 );	
				};
			} else {
				iUnmapped = pTri2->GetRightmostIndex();
				if (iUnmapped == q2_cornerind) {
					unsh2 = Anticlockwise*q2->pos;

					test = pTri->TestAgainstEdge(
						 unsh2.x,unsh2.y, 
						 c1,          // edge corner
						 opp,         // the opposite point
						 &pNeigh      // not used
						 );	
				} else {
					test = pTri->TestAgainstEdge(
						 q2->pos.x,q2->pos.y, 
						 c1,          // edge corner
						 opp,         // the opposite point
						 &pNeigh      // not used
						 );
				};
			};					
		};
	};
	// still within tri vs tri

	if (!test)
	{
		int c2 = 0;
		while ((c2 == c1) || (c2 == opp)) c2++;
		which = SeekVertexInsideTriangle(
			pTri->cornerptr[opp],
			q2,
			pTri->cornerptr[c1],   // up to 4 points to check for
			pTri->cornerptr[c2],	// in order of preference;
			pTri,				// Seed triangle to begin radar
			&pVertRogue,      // address for returning guilty vertex
			&pTriContain);

		if (which == -1) // found no vertex within a triangle. Strange?
		{
			printf("\nOverlap %d %d No vert within tri\n",pTri-T,pTri2-T);
			
			// kill the further one

			real dist1sq = distsq(pTri->cornerptr[opp]->pos,pTri->cornerptr[c1]->pos)
				          + distsq(pTri->cornerptr[opp]->pos,pTri->cornerptr[c2]->pos);

			real dist2sq = distsq(q2->pos,pTri->cornerptr[c1]->pos) + distsq(q2->pos,pTri->cornerptr[c2]->pos);
			
			if (dist1sq > dist2sq)
			{
				FullDisconnect(pTri->cornerptr[opp],pTri2);
			} else {
				FullDisconnect(q2,pTri);
			};
			
			return 1;
		};

		if ((pVertRogue == pTri->cornerptr[opp]) ||
			(pVertRogue == q2))
		{
			// an unshared point --
			// choose the closer of shared points for pVertUse
			real dist1sq = distsq(pVertRogue->pos,pTri->cornerptr[c1]->pos);
			real dist2sq = distsq(pVertRogue->pos,pTri->cornerptr[c2]->pos);
			if (dist1sq < dist2sq)
			{
				pVertUse = pTri->cornerptr[c1];
			} else {
				pVertUse = pTri->cornerptr[c2];
			};
		} else {
			// a shared point
			// choose the other shared point (for now at least)
			if (pVertRogue == pTri->cornerptr[c1])
			{
				pVertUse = pTri->cornerptr[c2];
			} else {
				pVertUse = pTri->cornerptr[c1];
			};
		};
		if (pVertRogue == q2)
		{
			FullDisconnect(pVertRogue,pTriContain);				
		} else {
			FullDisconnect(pVertRogue,pTriContain);
		};
		
		return 1; // action taken
	} else {
		return 0;
	};

}


bool TriMesh::FindOtherNeigh(Vertex * pVertex1, Vertex * pVertex2, Vertex * pVertNot, Vertex ** ppOtherNeigh)
{
	// found = FindOtherNeigh(pCaret, pNext1, pVertDisco, &pOtherNeigh);

	// pVertex1,2 should have a shared triangle with pVertNot on one side and another
	// vertex on the other, which we are to find.
	// If it is not this case, return false.
	bool thistri;
	int iCorner, i;
	Triangle * pTri;
	long tri_len, izTri[128];

	tri_len = pVertex1->GetTriIndexArray(izTri);

	for (i = 0; i < tri_len; i++)
	{
		pTri = T + izTri[i];
		thistri = false;
		for (iCorner = 0; iCorner < 3; iCorner++)
			if (pTri->cornerptr[iCorner] == pVertex2) thistri = true;
		for (iCorner = 0; iCorner < 3; iCorner++)
			if (pTri->cornerptr[iCorner] == pVertNot) thistri = false;
		if (thistri == true) {
			iCorner = 0;
			while ((pTri->cornerptr[iCorner] == pVertex1) || (pTri->cornerptr[iCorner] == pVertex2)) 
				iCorner++;
			*ppOtherNeigh = pTri->cornerptr[iCorner];
			return true;
		};
	};
	return false;
}

int TriMesh::FullDisconnect(Vertex *pVertDisco, Triangle *pTriContain)
{
	long iRet;
	long iDisconnected = 0;
	do
	{
		iRet = Disconnect(pVertDisco, pTriContain);
		if (iRet >= 0) pVertDisco = X + iRet; 
		// if Disconnect returns a value >=0  then it is the index of 
		// another vertex that needs disconnecting.
		iDisconnected++;
	} while (iRet > 0);
	return iDisconnected;
}

int TriMesh::Disconnect(Vertex *pVertDisco, Triangle *pTriContain)
{
	// pTriContain is the triangle that we will leave in its list when it's disconnected.
	// ================================================================

	int returnvalue = -1;

	if(pVertDisco->pos.x != pVertDisco->pos.x)
	{
		printf("\n\nIndeterminate value for x. Time to stop\n");
		steps_remaining = 0;
		getch();
		return -2;
	}
	//if (pVertDisco->flags == 3) {
	//	// should never happen
	//	printf("tried to disconnect insulator vertex %d \n",pVertDisco-X);
	//	getch();
	//}
	// For flags == 4, we want to worry about changing flags of triangles and vertices.


	// We no longer accept that in general, we can have more than two triangles 
	// that use one edge (pVertDisco,pVertUse). Consequently we do not worry about that.
	// If that happens, it indicates somewhere we allowed the "two tri per edge" rule
	// to be violated, and we should not be allowing that.

	// Therefore we have exactly 2 triangles with both pVertDisco and pVertUse; 
	// we remove these, and the rest we reconfigure to substitute pVertUse for pVertDisco
	// ^^^^^ --- X this is not correct. We go around the circle making new triangles in a spiral.***

	// Note that we can never have 2 points with only 3 neighs adjoining as long as
	// we insist every edge has 0 or 2 triangles.
	// Therefore we can avoid creating orphans by disconnecting all 3-neighbour points to begin with.

	// If we get rid of pVertUse then we switch to another vertex that pVertUse had in common with pVertDisco.
	// How to choose which one? Bear in mind, there must always be 2 - we do not allow complete overlap tri's
	// to come into being. We choose the other corner of pDestroy.

	// Meanwhile, this function is called when one of the 4 points for a shared edge belongs to
	// a triangle itself -- is that for the best? We did decide on that and it seemed to generally find one.
	// Otherwise kill all four?
	// For otherwise it's not clear which to disconnect, as it may not be one of the two unsharing points.
	
	// In case that pVertUse == 0, it's still the case that pVertDisco should belong to exactly 2 wedges.

	Vertex *pNext, *pVert, *pCaret;
	bool Used[1024];  
	Triangle * pNextTri;
	bool VertUseIsClockwise = 0;
	bool VertUseIsAnticlockwise = 0;
	int i,c1;
	Triangle *pTri, *pTriUse;
	smartlong hitlist;
	long iCaret;		
	Vector2 uCaret, uNext2, uNext1, uOther, perp;
	int jj;
	smartlong workingverts;
	smartlong nextverts;
	int tri_index;
//	long unviolated;
	static int passes = 0;
	int foundyet = 0;

	smartlong vert_ind;

	long izTrisDisco[128];
	long tri_len_disco = pVertDisco->GetTriIndexArray(izTrisDisco);

	//smartvp & tris = pVertDisco->triangles;
	//long * izTrisDisco; // pVertDisco->izTri;
	// Hoping to manipulate it with this pointer -- careful.

	int debug_extras = 0;
	// Spit out information used to draw graphs for debugging

	char filename[1024];
	FILE * file;

	// old comment:

	// We USED TO wish to replace pVertDisco with pVertUse in all triangles that have it in.
	// If the triangle already contains pVertUse it is scrapped and added to the 
	// TriangleHeap array.
	// Update: triangles list for neighbouring vertices
	//         triangle::cornerptr obviously
	//         triangle data such as periodic, flags etc
	//         triangle::neighbour lists
	// 
	// Finally we add pVertDisco to the Disconnected array and keep a note that
	// gives some idea of what triangle it used to inhabit (for reconnecting);
	
	// Preliminary 1:
	// Try to establish that neighbouring vertices form 1 circle as a graph.
	// _______________________________________________________
	
	if (tri_len_disco >= 1024) {printf("\n\ntris.len>1024 - something is about to break down\n");getch();};
	for (i = 0; i < tri_len_disco; i++)
		Used[i] = 0;
	int old_tri_len = tri_len_disco; // CAREFUL
		
	// vert_ind is the array of neighbours that are in order that they are linked by triangles.
	// We cannot do anything as easy as just taking angles here. We take the mesh structure as the guide for the circle.
	// Add to vert_ind the two corners from izTrisDisco[0] that are not == pVertDisco:
	
	if (pVertDisco->flags < 4) // Not edge
	{
		
		int first = 1;
		pTri = T + izTrisDisco[0];
		for (jj = 0; jj < 3; jj++)
		{
			if (pTri->cornerptr[jj] != pVertDisco)
			{
				if (first) {
					vert_ind.add(pTri->cornerptr[jj]-X);
					// Set periodic -1,0,+1 wrapping relative to pVertDisco ; is used by Triangle::Set to set triangle periodic flag
					pTri->cornerptr[jj]->iScratch = pTri->InferRelativeWrapping(pTri->cornerptr[jj],pVertDisco); 
					
					first = 0;
				} else {
					pNext = pTri->cornerptr[jj]; // we'll add this one when we find it in another tri.				
				};
			};
		};
		Used[0] = 1;
					
		for (int ii = 0; ii < old_tri_len-1; ii++)
		{
			// look for a match for pNext amongst the remaining triangles of pVertDisco

			int found = 0;
			for (i = 0; i < old_tri_len; i++)
			{
				if (Used[i] == 0)
				{
					// does this tri contain the point pNext?
					pTri = T + izTrisDisco[i];
					if ((pTri->cornerptr[0] == pNext) ||
						(pTri->cornerptr[1] == pNext) ||
						(pTri->cornerptr[2] == pNext) )
					{
						found = 1;
						pNextTri = pTri;
						Used[i] = 1;
					};
				};
			};
			if (found == 0)
			{
				// no pNext ??!?!!
				// => not 1 circular seq of neigh verts. Investigate further
				printf("Oh dear - non-circular sequence of neighbour vertices.\n");
				getch();
			};
			
			// Now found a match for this vertex; add it to the sequence
			vert_ind.add(pNext-X);
			pNext->iScratch = pNextTri->InferRelativeWrapping(pNext,pVertDisco);
			
			// Determine next thing to search for:
			jj = 0;
			while ((jj < 3) && 
				((pNextTri->cornerptr[jj] == pVertDisco) || (pNextTri->cornerptr[jj] == pNext)))
				jj++;
			// Note that we treat 0 as a valid member of the vertex list, that should appear at most once.
			if (jj == 3) { printf("\n3rror!\njj == 3\n"); getch(); }; // check that this is tri is remotely sensible.
			
			pNext = pNextTri->cornerptr[jj]; 
		};
		
		pTri = T + izTrisDisco[0]; // purpose?

		if (
			(pNext != X + vert_ind.ptr[0]) && 
			((pNext != 0) || (vert_ind.ptr[0] != -1)))
		{
			printf("\n3rror!\nCircle of neigh verts did not connect.\n"); getch();
		};
		// we already added that one at the start of the list.

		// Note this means vert_ind.len == tris.len. Verify that it is so:
		if (vert_ind.len != old_tri_len) {
			printf("vert_ind.len != old_tri_len\n"); getch();
		};
	} else {
		// EDGE VERTEX - CIRCLE WILL NOT CONNECT, AS WE INTEND THINGS.
		
		printf("Trying to call Disconnect for an edge vertex.\n");
		getch();

		int foundbase = 0;
		int iFirsttri = 0;
		do {
			if (iFirsttri == old_tri_len) {
				printf("failed to find even 1 base vertex by another.\n"); getch();
				return -23939;
			};
			
			pTri = T + izTrisDisco[iFirsttri]; 
			for (jj = 0; jj < 3; jj++)
			{
				if (pTri->cornerptr[jj] != pVertDisco)
				{
					if (pTri->cornerptr[jj]->flags >= 4) {
						vert_ind.add(pTri->cornerptr[jj]-X);
						// Set periodic -1,0,+1 wrapping relative to pVertDisco ; is used by Triangle::Set to set triangle periodic flag
						pTri->cornerptr[jj]->iScratch = pTri->InferRelativeWrapping(pTri->cornerptr[jj],pVertDisco); 
						foundbase = 1;
					} else {
						pNext = pTri->cornerptr[jj]; // we'll add this one when we find it in another tri.				
					};
				};
			};
			iFirsttri++;
			
		} while (foundbase = 0);
		iFirsttri--;
		Used[iFirsttri] = 1;
					
		for (int ii = 0; ii < old_tri_len-1; ii++) // ii index not used.
		{
			// look repeatedly for a match for pNext amongst the remaining triangles of pVertDisco
			int found = 0;
			for (i = 0; i < old_tri_len; i++)
			{
				if (Used[i] == 0)
				{
					// does this tri contain the point pNext?
					pTri = T+izTrisDisco[i];
					if ((pTri->cornerptr[0] == pNext) ||
						(pTri->cornerptr[1] == pNext) ||
						(pTri->cornerptr[2] == pNext) )
					{
						found = 1;
						pNextTri = pTri;
						Used[i] = 1;
					};
				};
			};

			if (found == 0)
			{
				// no pNext ??!?!! => not 1 circular seq of neigh verts. Investigate further
				printf("Oh dear - non-circular sequence of neighbour vertices (edge vert).\n");
				getch();
			};
			
			// Now found a match for this vertex; add it to the sequence
			vert_ind.add(pNext-X);
			pNext->iScratch = pNextTri->InferRelativeWrapping(pNext,pVertDisco);
			
			// Determine next thing to search for:
			jj = 0;
			while ((jj < 3) && 
				((pNextTri->cornerptr[jj] == pVertDisco) || (pNextTri->cornerptr[jj] == pNext)))
				jj++;
			// Note that we treat 0 as a valid member of the vertex list, that should appear at most once.
			if (jj == 3) { printf("\n3rror!\njj == 3\n"); getch(); }; // check that this is tri is remotely sensible.
			
			pNext = pNextTri->cornerptr[jj]; 
		};
		
		if (vert_ind.len != old_tri_len+1) {
			printf("vert_ind.len != tris.len+1\n"); getch();
		};
	};
	
	// No guarantee that we are in a position to run the standard "anticlockwise sort" routine, things could be in a complete tangle.
	// That is why we had to do the above.
	
	//=============================================================================

	// We used to delete neighs with 3 neighbours. 
	// The point was to create a convex set of remaining neighbours. 
	// But it just doesn't prevent having a non-convex set of neighbours. Having 3-ers probably does almost guarantee the set is not convex,
	// unless other things are also wrong. But that is all we can say.

	//=============================================================================

	// We might as well use the first tris.len-2 triangles for filling the space.
	// tris.len-1 in case of an edge vertex.

	// &&&&&&&&&&&&&&& &&&&&&&&&&&&&&& &&&&&&&&&&&&&&&
	// To scrap triangles, we just want to remove them from Vertex::Triangle lists:
	for (int ii = 0; ii < vert_ind.len; ii++)
	{
		pVert = X + vert_ind.ptr[ii];
		if (pVert != pVertDisco) // do not touch tri list of disconnection point !!!!
		{
			// remove all pVertDisco's triangles from pVert if they are there:
			for (i = 0; i < old_tri_len; i++)
				pVert->RemoveTriIndexIfExists(izTrisDisco[i]);
		};
	};  
	
	// &&&&&&&&&&&&&&& &&&&&&&&&&&&&&& &&&&&&&&&&&&&&&
	// Make triangles to fill the space.

	long iStart;
	long iEnd;
	long iPrevEnd,iNext1,iNext2;
	Vertex * pNext1, * pNext2;
	
	// copy verts_ind over workingverts:
	workingverts.copyfrom(vert_ind);
	tri_index = 0;
	
	// we will always make a triangle from iCaret to iCaret+1, iCaret+2
	// then move iCaret to iCaret+2, until we can do no more triangles;
	// adding the 0th,2nd,4th,etc points to the next circle.
	
	// Should be guaranteed that each edge has only 2 sharing. If we check that it's convex to cut out the intermediate point,
	// well I don't know but I think we should stick to just that! Disconnect more points if it still has a problem!
	// But maybe if we are careful here then we end up not creating "a new overlap" ?

	// Meh. Come back and try adding back that test later if we have problems. For now we just test whether we are cutting 
	// the intermediate point inside. 
	
	// nextverts is the array of uncovered points that will be used to then fill in the next, inner concentric set of tri's.

	Triangle * pTriTest;
	int bad;
	int tris_added;
	int always_onward;
	int num_edge;
	Vertex * pVertex, *pOtherNeigh;
	long iNext;
	bool found;
	int cautious = 1;
	long tri_len, izTri[128];

	bool bRanSet = 0;

	iStart = 0;
	while (workingverts.len >= 3)
	{
		nextverts.clear();
	
		// We do not want the LAST triangle to be faced with a triple of insulator points. === How to avoid? ===				
		// The way it can come about is if the previous triangle linked the first and last base points.
		// Such triangles are bad! We want to block off a base point not the last domain point -- so always
		// start our circle before the end of the base points, if they exist.
	
		num_edge = 0;
		for (i = 0; i < workingverts.len; i++)
		{
			pVertex = X+workingverts.ptr[i];
			if (pVertex->flags >= 4) num_edge++;
		};
		if (num_edge > 2) {
			// pedal backwards until the *next* vertex is an the edge
			do
			{
				iStart--;
				if (iStart < 0) iStart = workingverts.len-1;
				iNext = iStart+1; if (iNext == workingverts.len) iNext = 0;
				pVertex = X+workingverts.ptr[iNext];
			} while (pVertex->flags < 4);
		};
		

		iCaret = iStart;
		
		// Do we always add anything that iCaret passes through? :
		//	nextverts.add(workingverts.ptr[iCaret]);
		// No - because we might find iStart is covered over by final triangle;
		// Instead add points at end of triangle placement, knowing it is not covered over.
		
		iNext1 = iCaret+1;if (iNext1 >= workingverts.len) iNext1 = 0;
		iNext2 = iNext1+1;if (iNext2 >= workingverts.len) iNext2 = 0;
		
		tris_added = 0; // for checking we do not get infinite loop
		
		// inner loop: add triangles around edge of a circle
		//===================================================
		do 
		{

			// test this triangle: is it one that we want to use?
			pCaret = X+workingverts.ptr[iCaret];
			pNext1 = X+workingverts.ptr[iNext1];
			pNext2 = X+workingverts.ptr[iNext2];
			
			if (workingverts.len > 3)
			{	
				// We have to decide whether (pCaret,pNext1,pNext2) is a valid triangle to place.
				// Now because pVertDisco may have been moved somewhere daft,
				// it is no good testing whether pNext1 is same side of pCaret--pNext2 as pVertDisco.
				
				bad = 0;
				// Disallow connecting 3 edge vertices:
				if ((pCaret->flags >= 4) && (pNext1->flags >= 4) && (pNext2->flags >= 4)) 
				{
					bad = 1;
				} else {
					// OK so we have 3 sides we have to test.
					// pCaret-pNext1 exists ; one of its neighs was pVertDisco; we do not know yet which
					// side the other one was on. We propose a new neighbour pNext2.

					found = FindOtherNeigh(pCaret,pNext1, pVertDisco, &pOtherNeigh);
					if (found == 0) {
						// This will happen routinely, because pCaret and pNext1 were probably
						// not together in the initial list of vertices.

						printf("error -- had a side with only one triangle. Shouldna happened?");
						found = found;
						bad = 1;
					};
					
					if (pOtherNeigh == pNext2) {
						// concave shape then probably.
						bad = 1;
					} else {
						// pOtherNeigh same side as pNext2 ?
						// how to make sure we use contiguous image?
						// Make all contiguous using pNext1->has_periodic. pNext1 has all the others as neighbours.
						uCaret=pCaret->pos;
						uNext2=pNext2->pos;
						uNext1=pNext1->pos;
						uOther=pOtherNeigh->pos;
						if (pNext1->has_periodic) {
							if (uCaret.x > 0.0) uCaret = Anticlockwise*uCaret;
							if (uNext2.x > 0.0) uNext2 = Anticlockwise*uNext2;
							if (uNext1.x > 0.0) uNext1 = Anticlockwise*uNext1;
							if (uOther.x > 0.0) uOther = Anticlockwise*uOther;
						};
						perp.x = uNext1.y-uCaret.y; perp.y = uCaret.x-uNext1.x;
						if ((uNext2-uNext1).dot(perp) * (uOther-uNext1).dot(perp) > 0.0){
							if (cautious) {
								bad = 1;
							} else {
								returnvalue = pOtherNeigh-X;// disconnect that index next.
							};
						};
					};

					// pNext1-pNext2 exists. We propose a new neighbour pCaret.
					found = FindOtherNeigh(pNext1,pNext2, pVertDisco, &pOtherNeigh);
					if (found == 0) {
						printf("error -- had a side with only one triangle. Shouldna happened?");
						found = found;
						bad = 1;
					};
					if (pOtherNeigh==pCaret) {
						bad = 1;
					} else {
						// Again contiguous image is based on pNext1.
						uCaret=pCaret->pos;
						uNext2=pNext2->pos;
						uNext1=pNext1->pos;
						uOther=pOtherNeigh->pos;
						if (pNext1->has_periodic) {
							if (uCaret.x > 0.0) uCaret = Anticlockwise*uCaret;
							if (uNext2.x > 0.0) uNext2 = Anticlockwise*uNext2;
							if (uNext1.x > 0.0) uNext1 = Anticlockwise*uNext1;
							if (uOther.x > 0.0) uOther = Anticlockwise*uOther;
						};
						perp.x = uNext1.y-uNext2.y; perp.y = uNext2.x-uNext1.x;
						if ((uCaret-uNext1).dot(perp) * (uOther-uNext1).dot(perp) > 0.0){
							if (cautious) {
								bad = 1;
							} else {
								returnvalue = pOtherNeigh-X; // disconnect that index next.
							};
						};
					};
					
					// pCaret-pNext2 may not already exist.  
					// If it does then we'd be creating a 3x shared edge because it's guaranteed that two triangles
					// have pCaret<->pNext2 as an edge, and neither of them was one that we are deleting.
					// (Most of the time if it does exist, it just means that there is a concave part to the shape here.)
					// Creating 3x shared edge is not a problem that will be fixable with more mesh maintenance so we do not do that under any circumstances.

					tri_len = pCaret->GetTriIndexArray(izTri);
					for (int iiii = 0; iiii < tri_len; iiii++)
					{
						pTriTest = T+izTri[iiii];
						for (int jj = 0; jj < 3; jj++)
							if (pTriTest->cornerptr[jj] == pNext2)
								bad = 1;
					};

				}; // whether a base triple
			} else {
				// exactly 3 points - ie only triangle in final circle
				bad = 0; // have no choice to accept 3 points
				
				// Note that since these points were not linked in original circle they certainly can already be
				// connected elsewhere, even though in this circle they are adjacent ('linked')
				// If that happens, it could be that a different choice of triangulation might have avoided the problem
				// We are making a choice in using nearby neighbours and starting from a particular point.
				// Maybe we do need to reinstitute deleting any 3-neighbour neighbours to begin with.

				if ((pCaret->flags >= 4) && (pNext1->flags >= 4) && (pNext2->flags >= 4)) 
				{
					printf("Disconnect failed - left with a triple of edge points left over.\n");
					bad = 1;
					iStart = iStart;
				};
			};
			
			always_onward = 0;			
			if (iCaret == iStart)
			{
				// place iEnd at the place we should stop testing triangles
				if (bad == 0)
				{
					iEnd = iStart; // stop when we reach the point before, which would reach forward to one no longer existent, or at this point.
					nextverts.add(workingverts.ptr[iStart]); // make sure though that we do not add it twice
				} else {
					// make sure we do test for tri at (prevstart, (start), start+1), given modulo of 2
					iEnd = iStart+1; if (iEnd >= workingverts.len) iEnd = 0;
					always_onward = 1; // make sure it doesn't stop before it gets past the beginning.
					// If we did not put a triangle at start, iPrevEnd will be iStart and so we stop when we get back to iStart or beyond.
				};				
				iPrevEnd = iEnd-1; if (iPrevEnd < 0) iPrevEnd = workingverts.len-1; 
				// We will stop if we reach iCaret == iPrevEnd or iEnd.

				// Every point should be either added to nextverts or killed out of the set.
			};

			if (bad == 0)
			{
				pTriUse = T+izTrisDisco[tri_index];
				//pTriUse = (Triangle *)(tris.ptr[tri_index]);
				tri_index++;
				SetTriangle(pTriUse,pCaret,pNext1,pNext2); // will use iScratch to set pTriUse->periodic and vertex flags to set pTriUse->flags

				// Trouble with this:
				// we expect triangle arrays to be anticlockwise sorted.
				// * *** *** *** *** *** *** *** *** *** *** *** *** *** **
				bRanSet = 1; // to remind to fix later

				pCaret = pNext2;
				iCaret = iNext2;
				iNext1+=2;if (iNext1 >= workingverts.len) iNext1 -= workingverts.len;
				iNext2+=2;if (iNext2 >= workingverts.len) iNext2 -= workingverts.len;

				tris_added++;
			} else {
				// Advance 1.
				
				pCaret = pNext1;
				iCaret = iNext1;
				iNext1 = iNext2;
				pNext1 = pNext2;
				iNext2++;
				if (iNext2 >= workingverts.len) iNext2 -= workingverts.len;			
				// what on earth the point of advancing only 2 out of 3 pointers.
			};

			// iCaret is now the next point that is not covered over by a triangle ;
			// add the iCaret vertex, before we test it for a forward-looking triangle next time:
			if (iCaret == iStart) {
				nextverts.add_unique(workingverts.ptr[iCaret]); // we may have 2 triangles meeting at iStart; add it only once
			} else {
				if (iCaret != iEnd) 
					nextverts.add(workingverts.ptr[iCaret]); // if we just moved iEnd to iStart, we do add it the first time.
			};

		} while (((iCaret != iPrevEnd) && (iCaret != iEnd)) || (always_onward == 1));
		// We have tested triangles starting from iPrevEnd, or, have a triangle at iEnd
		
		if (tris_added == 0) {
			// failed circle
			if (cautious) {
				printf("warning -- disconnect creating more shared edges.  ");
				cautious = 0;
				// go around again and set returnvalue to index of vertex we might wish to disconnect.
			} else {
				printf("Disconnect failed. No way to connect decent circle. iVertDisco %d \n",pVertDisco-X);
				getch();
				iCaret = iCaret;
			};
		} else {
			// succeeded circle
			workingverts.copyfrom(nextverts);					
			// can't see a point in moving iStart unless it is > workingverts.len now. oh well
			iStart++;
			if (iStart >= workingverts.len) iStart = 0;
		};
	};

	if (pVertDisco->flags < 4)
	{
		if (tri_index != vert_ind.len-2) {
			printf("tri_index error %d %d \n",tri_index,vert_ind.len);
			getch();
		};
	} else {
		if (tri_index != vert_ind.len-1) {
			printf("tri_index error edge %d %d \n",tri_index,vert_ind.len);
			getch();
		};
	};
	
	// Now go through and set transvec
	for (i = 0; i < tri_index; i++)
	{
		pTriUse = T+izTrisDisco[i];
		// Important to do periodic before RecalculateUnnormalisedEdgeNormalVectors.
		// Flags and periodic were set in the Triangle::Set routine.
		pTriUse->RecalculateEdgeNormalVectors(false);
		GlobalAffectedTriIndexList.add(pTriUse-T); // remember list of those that altered(?)
	};
	
	// Now reset Triangle::neighbours for all tri's of the affected vertices.
	
	for (i = 0; i < vert_ind.len; i++)
	{
		pVert = X + vert_ind.ptr[i];
		// each of your triangles needs to reset its neighbours:
		
		tri_len = pVert->GetTriIndexArray(izTri);
		for (int ii = 0; ii < tri_len; ii++)
		{
			pTri = T+izTri[ii];
			ResetTriangleNeighbours(pTri);
		};
	};

	//
	// If an edge vertex being disconnected, will have 1 triangle left over not 2.
	// 

	// Last two triangles get put on scrapheap
	//=========================================

	// Now punch a hole in the remaining 2 triangles to mark them as scrapped, and
	// list our point as disconnected:
	
	//pTriUse = (Triangle *)(tris.ptr[tri_index]);
	pTriUse = T+izTrisDisco[tri_index];
	pTriUse->cornerptr[0] = 0;
	TriangleHeap.add(pTriUse-T);
	GlobalAffectedTriIndexList.add(pTriUse-T); // remember list of those that altered(?)
	tri_index++;
	pTriUse = T + izTrisDisco[tri_index];
	// Scrapped the idea of shuffling triangles past numTriangles. Have fixed total number.
	pTriUse->cornerptr[0] = 0;
	TriangleHeap.add(pTriUse-T);
	GlobalAffectedTriIndexList.add(pTriUse-T); // remember list of those that altered(?)

	long index = (long)(pVertDisco-X); 
	Disconnected.add(index);
	// Note we did not change flag on pVertDisco -- it can remember it is a base point if it gets reconnected in situ.
	
	// store some information in pVertDisco: what triangle should be used to look for it?
	//pVertDisco->triangles.clear();
	//pVertDisco->triangles.add(pTriContain);
	pVertDisco->ClearTris();
	pVertDisco->AddTriIndex(pTriContain-T);

	// What about its neighbours? Will they be cleared on reconnect?
	// Exmaine it.
	// WHAT ABOUT THE OPINION OF NEIGHBOURS ON WHETHER THIS ONE IS A NEIGHBOUR.
	// PROBLEMOS.
	
	workingverts.clear();
	nextverts.clear(); // just in case going out of scope left a memory leak.

	printf("Done disconnect %d \n",index);

#ifndef RELEASE
	//
	//sprintf(filename,"endII%d.txt",passes);
	//file = fopen(filename,"w");
	//spit_out_point_triangles(pVertUse,file);	
	//spit_out_point_triangles(pVertDefault,file);
	//fclose(file);
	//
	//DebugTestForLinkedScrapTris();
	//DebugTestForVertexInOnlyTwoTriangles();
	//DebugTestWrongNumberTrisPerEdge();
	//DebugTestNumberOfWedgeRings();
#endif

	passes++;
	if (bRanSet) {
		printf("unordered tri lists! Need to fix program! vertex neighs were unset also\n"
			"we run RVNOVO but intermediately what happened?"
			"and does RVNOVO know to look for disconnected vertex?");
		getch();
		RefreshVertexNeighboursOfVerticesOrdered();
	}

	return returnvalue;
}



void TriMesh::SetTriangle(Triangle * pTri, Vertex * pV1, Vertex * pV2, Vertex * pV3)
{
	// set the cornerptr to these values and tell vertices to add this triangle also.

	// Set Triangle::flags based on Vertex::flags
	// and Triangle::periodic based on Vertex::iScratch which records relative wrapping apparently.

	pTri->cornerptr[0] = pV1;
	pTri->cornerptr[1] = pV2;
	pTri->cornerptr[2] = pV3;
	pV1->AddTriIndex(pTri-T);
	pV2->AddTriIndex(pTri-T);
	pV3->AddTriIndex(pTri-T);
	
	pTri->periodic = pTri->cornerptr[0]->iScratch
		+ pTri->cornerptr[1]->iScratch + pTri->cornerptr[2]->iScratch;
	while (pTri->periodic >= 3) pTri->periodic -= 3;
	while (pTri->periodic < 0) pTri->periodic += 3;

}

Triangle * TriMesh::SetAuxTri(int iLevel, long iVertex1, long iVertex2, long iVertex3)
{
	Triangle * pTri = AuxT[iLevel] + numAuxTriangles[iLevel];
	long index = numAuxTriangles[iLevel];
	pTri->cornerptr[0] = AuxX[iLevel]+iVertex1;
	pTri->cornerptr[1] = AuxX[iLevel]+iVertex2;
	pTri->cornerptr[2] = AuxX[iLevel]+iVertex3;

	// Now what?
	pTri->cornerptr[0]->AddTriIndex(index);
	pTri->cornerptr[1]->AddTriIndex(index);
	pTri->cornerptr[2]->AddTriIndex(index);

	numAuxTriangles[iLevel]++;
	if (numAuxTriangles[iLevel] >= numTrianglesAuxAllocated[iLevel])
	{
		printf("got to end of allocated aux tris!"); getch();
	};
	return pTri;
}

void TriMesh::ReconnectLastPointInDiscoArray()
{
	// We want to reconnect the vertex indexed by Disconnected.ptr[0]
	// We use two triangles from TriangleHeap.
	// The vertex includes one thing in its triangles array, that gives us the place to seek its location.

	// To update: Triangle::cornerptr (obviously)
	//            Vertex::triangles
	//            Triangle::transvec,periodic,flag
	//            Triangle::neighbours

	real grad,gradleft,gradmid,gradright;
	int i,ii,iWhich;
	int additional, valueone, valuetwo;
	long lendisc = Disconnected.len;
	Vertex * pVert = X + Disconnected.ptr[lendisc-1];
	Vertex * pVertexTemp, *pVertPrev;

	long tri_len, izTri[128];
	tri_len = pVert->GetTriIndexArray(izTri);
	long izTriTemp[128];

	Triangle * pSeedTri = T+izTri[0];
	// stored record of where to look for it
	
	Triangle * pTri, *pTri2, *pTriCopy, *pTriNeigh;
	int BaseFlag;
	real grad1, grad2;
	int found,c1,c2,iApex,use;
	Vector2 u[3];

	if ((pSeedTri->u8domain_flag == INNER_FRILL) || 
		(pSeedTri->u8domain_flag == OUTER_FRILL))
	{
		pSeedTri = pSeedTri->neighbours[0];
	}

	// Get hold of two spare triangles:
	long lentris = TriangleHeap.len;
	Triangle * pTri_extra1 = T+TriangleHeap.ptr[lentris-1];
	Triangle * pTri_extra2 = T+TriangleHeap.ptr[lentris-2];
	if (lentris < 2) {
		printf("fatal error, lentris < 2\n"); getch(); return;
	}

	if (pVert->flags < 3) { // MAGIC NUMBERS 
	
		// Check something:
		if (pSeedTri->cornerptr[0] == 0)
		{
			// The nearby triangle got scrapped unfortunately, so restart from triangle 0:
			pSeedTri = T;
			while (pSeedTri->cornerptr[0] == 0)
				pSeedTri++;
		};
		// Following function should work fine because overlaps have been eliminated before we call Reconnect:
		pTri = ReturnPointerToTriangleContainingPoint(
					pSeedTri, pVert->pos.x, pVert->pos.y);
	
		// pTri now is what contains pVert.

		// We subdivide it into 3 triangles and pVert becomes the new cornerptr[2]

		// For periodic we see if pTri was periodic.
		// If it is, we have to decide which points pVert is on same side as.
		if (pTri->periodic > 0)
		{
			if (pTri->periodic == 1)
			{
				// one point is to Clockwise. Which one?
				int c1 = pTri->GetLeftmostIndex();
				int c3 = pTri->GetRightmostIndex();
				c2 = 0; while ((c2 == c1) || (c2 == c3)) c2++;
				//int c2 = pTri->GetxMidIndex();
				// Now we want to know where pVert lies in this.
				
				// Right of leftmost unwrapped is one case;
				// Left of wrapped point is another;
				// Otherwise something has failed.

				grad = pVert->pos.x/pVert->pos.y;
				gradleft = pTri->cornerptr[c1]->pos.x/pTri->cornerptr[c1]->pos.y;
				gradmid = pTri->cornerptr[c2]->pos.x/pTri->cornerptr[c2]->pos.y;

				if (grad > gradmid)
				{
					// In this case 2 triangles have periodic == 1
					// and 1 triangle, formed without c1, has periodic == 0				
					additional = 0;
				} else {
					if (grad < gradleft)
					{
						// In this case 2 triangles have periodic == 2
						// and 1 triangle, formed without c1, has periodic == 1					
						additional = 1;
					} else {
						printf("\n\nReconnection point not in its triangle. (periodic issue)\n");
						getch();
					};
				};
				if (c1 == 0)
				{
					pTri_extra1->periodic = 1 + additional;
					pTri_extra2->periodic = additional;
					pTri->periodic = 1 + additional;
				};
				if (c1 == 1)
				{
					pTri_extra1->periodic = additional;
					pTri_extra2->periodic = 1 + additional;
					pTri->periodic = 1 + additional;
				};
				if (c1 == 2)
				{
					pTri_extra1->periodic = 1 + additional;
					pTri_extra2->periodic = 1 + additional;
					pTri->periodic = additional;
				};
			} else {
				// one point is to Anticlockwise. Which one?
				int c1 = pTri->GetRightmostIndex();
				int c3 = pTri->GetLeftmostIndex();
				c2 = 0; while ((c2 == c1) || (c2 == c3)) c2++;

				grad = pVert->pos.x/pVert->pos.y;
				gradright = pTri->cornerptr[c1]->pos.x/pTri->cornerptr[c1]->pos.y;
				gradmid = pTri->cornerptr[c2]->pos.x/pTri->cornerptr[c2]->pos.y;

				// Now we want to know where pVert lies in this.
				
				// Left of rightmost unwrapped is one case;
				// Right of wrapped point is another;
				// Otherwise something has failed.
				if (grad < gradmid)
				{
					// In this case 2 triangles have periodic == 2
					// and 1 triangle, formed without c1, has periodic == 0
					valuetwo = 2;
					valueone = 0;
				} else {
					if (grad > gradright)
					{
						// In this case 2 triangles have periodic == 1
						// and 1 triangle, formed without c1, has periodic == 2
						valuetwo = 1;
						valueone = 2;
					} else {
						printf("(anticlock) Reconnection point not in its triangle.\n");
						getch();
					};
				};				
				if (c1 == 0)
				{
					pTri_extra1->periodic = valuetwo;
					pTri_extra2->periodic = valueone;
					pTri->periodic = valuetwo;
				};
				if (c1 == 1)
				{
					pTri_extra1->periodic = valueone;
					pTri_extra2->periodic = valuetwo;
					pTri->periodic = valuetwo;
				};
				if (c1 == 2)
				{
					pTri_extra1->periodic = valuetwo;
					pTri_extra2->periodic = valuetwo;
					pTri->periodic = valueone;
				};
			};
		} else {
			// pTri not periodic at all :
			pTri_extra1->periodic = 0;
			pTri_extra2->periodic = 0;
		};

		// Update Triangle::flags ..
		// u8domain_flag
		if (pTri->u8domain_flag == DOMAIN_TRIANGLE) {
			pTri_extra1->u8domain_flag = DOMAIN_TRIANGLE;
			pTri_extra2->u8domain_flag = DOMAIN_TRIANGLE;
		} else {
			if (pTri->u8domain_flag == OUT_OF_DOMAIN) {
				pTri_extra1->u8domain_flag = OUT_OF_DOMAIN;
				pTri_extra2->u8domain_flag = OUT_OF_DOMAIN;
			} else {
				// CROSSING_INS
				
				// New pTri will go between cornerptr 0 and 1 and the new vertex
				// pTri_extra1 will have cornerptr 0 and 2
				// pTri_extra2 will have cornerptr 1 and 2
				pTri_extra1->u8domain_flag = CROSSING_INS;
				if ((pTri->cornerptr[0]->flags == DOMAIN_VERTEX) &&
					(pTri->cornerptr[2]->flags == DOMAIN_VERTEX))
					pTri_extra1->u8domain_flag = DOMAIN_TRIANGLE;
				pTri_extra2->u8domain_flag = CROSSING_INS;
				if ((pTri->cornerptr[1]->flags == DOMAIN_VERTEX) &&
					(pTri->cornerptr[2]->flags == DOMAIN_VERTEX))
					pTri_extra2->u8domain_flag = DOMAIN_TRIANGLE;
				pTri->u8domain_flag = CROSSING_INS;
				if ((pTri->cornerptr[0]->flags == DOMAIN_VERTEX) &&
					(pTri->cornerptr[1]->flags == DOMAIN_VERTEX))
					pTri->u8domain_flag = DOMAIN_TRIANGLE;
			};
		};


		pTri->cornerptr[2]->RemoveTriIndexIfExists(pTri-T);
		pTri->cornerptr[2]->AddTriIndex(pTri_extra1-T);
		pTri->cornerptr[2]->AddTriIndex(pTri_extra2-T);

		pTri->cornerptr[0]->AddTriIndex(pTri_extra1-T);
		pTri->cornerptr[1]->AddTriIndex(pTri_extra2-T);

		// Now pVert:
		pVert->ClearTris();
		pVert->AddTriIndex(pTri-T);
		pVert->AddTriIndex(pTri_extra1-T);
		pVert->AddTriIndex(pTri_extra2-T);

		// Now (finally) change verts:

		pTri_extra1->cornerptr[0] = pTri->cornerptr[0];
		pTri_extra1->cornerptr[1] = pVert;
		pTri_extra1->cornerptr[2] = pTri->cornerptr[2]; 

		pTri_extra2->cornerptr[0] = pVert;
		pTri_extra2->cornerptr[1] = pTri->cornerptr[1];
		pTri_extra2->cornerptr[2] = pTri->cornerptr[2]; 

		pTri->cornerptr[2] = pVert;

		// Triangle::edge_normal :

		pTri->RecalculateEdgeNormalVectors(false);
		pTri_extra1->RecalculateEdgeNormalVectors(false);
		pTri_extra2->RecalculateEdgeNormalVectors(false);

		// Finally Triangle::neighbours (requires Vertex::triangles) :
		// First reset neighbours of our 3 triangles:
		ResetTriangleNeighbours(pTri);
		ResetTriangleNeighbours(pTri_extra1);
		ResetTriangleNeighbours(pTri_extra2);	

		// Now look at each of those neighbours and reset THEIR neighbour lists also:

		if (pVert->GetTriIndexArray(izTriTemp) != 3)
		{
			printf("shucks! TriMesh::ReconnectLastPointInDiscoArray  tris.len != 3 !\n");
			getch();
		};

	} else {

		// %%%%%%%%%%%%%%%%%%%%%%%%
		// reconnecting edge vertex
		// %%%%%%%%%%%%%%%%%%%%%%%%

		// Chances are this NEVER happens.
		// Edges of memory should remain inviolate.
		// But be careful for tri-based!!

		// Actually it seems I do not know what behaviour I want in this case.
		// We can identify azimuthally which triangle is split up by introducing
		// this edge vertex. Then what? We are supposed to use two triangles
		// by putting it in. 
		// It probably created 2 extra triangles when disconnected!

		// comment out whole thing for now
		printf("Trying to reconnect edge vertex but code has been commented out.\n");
		getch();


		////if (pVert->flags == 3) {BaseFlag = 6;} else {BaseFlag = 24;};
		//grad = pVert->x/pVert->y;
		//// Find base triangle where it is azimuthally within corners

		//if ( (pSeedTri->flags == pVert->flags) && (pSeedTri->cornerptr[0] != 0) )
		//{
		//	pTri = pSeedTri;
		//} else {
		//	pTri = T;
		//	while (pTri->flags != pVert->flags) pTri++;
		//};

		//pVertPrev = pTri->cornerptr[0];
		//if (pTri->cornerptr[0]->flags != pVert->flags) pVertPrev = pTri->cornerptr[1];
		//
		//found = 0;
		//do {
		//	//1. get corners c1,c2 on edge:
		//	if (pTri->cornerptr[0]->flags != pVert->flags) {
		//		c1 = 1; c2 = 2; iApex = 0;
		//	};
		//	if (pTri->cornerptr[1]->flags != pVert->flags) {
		//		c1 = 0; c2 = 2; iApex = 1;
		//	};
		//	if (pTri->cornerptr[2]->flags != pVert->flags) {
		//		c1 = 0; c2 = 1; iApex = 2;
		//	};
		//	if (pTri->periodic == 0) {
		//		grad1= pTri->cornerptr[c1]->x/pTri->cornerptr[c1]->y;
		//		grad2 = pTri->cornerptr[c2]->x/pTri->cornerptr[c2]->y;				
		//	} else {
		//		if (pVert->x > 0.0) {
		//			pTri->MapRight(u[0],u[1],u[2]);
		//		} else {
		//			pTri->MapLeft(u[0],u[1],u[2]);
		//		};
		//		grad1 = u[c1].x/u[c1].y;
		//		grad2 = u[c2].x/u[c2].y;
		//	};
		//	if ((grad-grad1)*(grad-grad2) <= 0.0) // quick fix .... is == enough?
		//	{
		//		found = 1;
		//	} else {
		//		if (pTri->cornerptr[c1] == pVertPrev)
		//		{	use = c2;	} else { use = c1 ;};
		//		pVertPrev = pTri->cornerptr[use];
		//		// Now find next base triangle: belongs to new pVertPrev but is not this tri
		//		iWhich = -1;
		//		for (i = 0; i < pVertPrev->triangles.len; i++)
		//		{
		//			pTri2 = (Triangle *)(pVertPrev->triangles.ptr[i]);
		//			if ((pTri2 != pTri) && (pTri2->flags == pTri->flags))
		//				iWhich = i;
		//		};				
		//		if (iWhich == -1) {	printf("error iWhich == -1\n"); getch();	};
		//		pTri = (Triangle *)(pVertPrev->triangles.ptr[iWhich]);
		//	};
		//}	while (found == 0);

		//// Now we seek to place pVert on the edge of pTri

		//// Subdivide it into 2 triangles. pTri_extra1 becomes one;
		//// pTri_extra2 will be consigned to spare triangles.

		//// seek numEdgeVerts++ to see same code elsewhere.
		//pTriCopy = T+numTriangles-1;
		//memcpy(pTri_extra2,pTriCopy,sizeof(Triangle));
		//
		//// 2. all the other triangles and vertices that looked at pTriCopy have to now be looking at its new location.
		//for (i = 0; i < 3; i++)
		//{
		//	pTriNeigh = pTri->neighbours[i];
		//	for (int ii = 0; ii < 3; ii++)
		//	{
		//		if (pTriNeigh->neighbours[ii] == pTriCopy) pTriNeigh->neighbours[ii] = pTri_extra2;
		//	};
		//};
		//for (i = 0; i < 3; i++)
		//{
		//	pVertexTemp = pTriCopy->cornerptr[i]; // pTri or pTriCopy doesn't matter - copied over cornerptr already.
		//	for (int ii = 0; ii < pVertexTemp->triangles.len; ii++)
		//	{
		//		pTri2 = (Triangle *)(pVertexTemp->triangles.ptr[ii]);
		//		if (pTri2 == pTriCopy) pVertexTemp->triangles.ptr[ii] = pTri_extra2;
		//	};
		//};
		//
		//// assign spare triangle index and change counts:
		////pVert->iTriSpare = numTriangles-1;		
		//// INDEX NEVER USED FOR ANYTHING - JUST STACK
		//this->numTriangles--;
		//this->numEdgeVerts++;
		//

		//// To rewrite so that we don't have spares but secret tris underneath, would be
		//// possible but I'm not sure there would be a point.
		//
		//if (pTri->periodic > 0) {
		//	if (pVert->x > 0.0) { // unwrapped
		//		if (pTri->cornerptr[iApex]->x > 0.0) // unwrapped
		//		{
		//			pTri->periodic = 0;
		//			pTri_extra1->periodic = 1;
		//		} else {
		//			pTri->periodic = 1;
		//			pTri_extra1->periodic = 2;
		//		};
		//	} else {
		//		if (pTri->cornerptr[iApex]->x > 0.0) // unwrapped
		//		{
		//			pTri->periodic = 1;
		//			pTri_extra1->periodic = 2;
		//		} else {
		//			pTri->periodic = 2;
		//			pTri_extra1->periodic = 0;
		//		};
		//	};
		//} else {
		//	// pTri was not periodic:
		//	pTri_extra1->periodic = 0;
		//};
		// 
		//pTri_extra1->flags = pVert->flags;
		//pTri->flags = pVert->flags; 
		//// no others change since their cornerptrs do not change.

		//// Deal with Vertex::triangle
		//// CHANGE c2 TO NEW VERTEX
		//pTri->cornerptr[c2]->triangles.remove(pTri);
		//pTri->cornerptr[c2]->triangles.add(pTri_extra1);
		//
		//pTri->cornerptr[iApex]->triangles.add(pTri_extra1);
		//
		//pVert->triangles.clear();
		//pVert->triangles.add(pTri);
		//pVert->triangles.add(pTri_extra1);
		//
		//// Now Triangle::cornerptr

		//pTri_extra1->cornerptr[0] = pTri->cornerptr[iApex];
		//pTri_extra1->cornerptr[1] = pVert;
		//pTri_extra1->cornerptr[2] = pTri->cornerptr[c2]; 

		//pTri->cornerptr[c2] = pVert;

		//// Triangle::edge_normal :

		//pTri->RecalculateEdgeNormalVectors(false);
		//pTri_extra1->RecalculateEdgeNormalVectors(false);
		//
		//// Finally Triangle::neighbours (requires Vertex::triangles) :
		//ResetTriangleNeighbours(pTri);
		//ResetTriangleNeighbours(pTri_extra1);
		//
		//// Now look at each of those neighbours and reset THEIR neighbour lists also:

		//if (pVert->triangles.len != 2)
		//{
		//	printf("shucks! TriMesh::ReconnectLastPointInDiscoArray  tris.len !=2 !\n");
		//	getch();
		//};
	};

	
	tri_len = pVert->GetTriIndexArray(izTri);
	for (i = 0; i < tri_len; i++) // tris.len should equal 3 !!!!!!
	{
		pTri = T+izTri[i];
					
		ResetTriangleNeighbours(pTri->neighbours[0]);
		ResetTriangleNeighbours(pTri->neighbours[1]);
		ResetTriangleNeighbours(pTri->neighbours[2]);
	};
	
	// Remove triangles from scrapheap:
	// --------------------------------------
	Disconnected.remove(Disconnected.ptr[lendisc-1]);
	TriangleHeap.remove(pTri_extra1-T);
	TriangleHeap.remove(pTri_extra2-T);

	printf("reconnection tests");
	//DebugTestForVertexInOnlyTwoTriangles();
	DebugTestWrongNumberTrisPerEdge();

	// So then it IS missing any 
	// neighbours -> clear
	// neighbours -> add

	// Is it in RVNOVO?

	// In OUR version, we need vertex neighbours to be maintained.

	// ** That applies to Delaunay flips also.**

	// ??
}


bool TriMesh::DebugTestForOverlaps()
{
	Triangle * pTri,* pNeigh;
	Vertex * pOpp;
	long iTri;
	int iNeigh, iWhich, iprev;
	Vector2 u_1[3], u_2[3], diff_ours, diff_ther;

	pTri = T;
	for (iTri = 0; iTri < numTriangles; iTri++)
	{
		for (iNeigh = 0; iNeigh<3; iNeigh++)
		{
			pNeigh = pTri->neighbours[iNeigh];
			if (pNeigh != pTri) {
				
				pTri->MapLeftIfNecessary(u_1[0],u_1[1],u_1[2]);
				pNeigh->MapLeftIfNecessary(u_2[0],u_2[1],u_2[2]);
						
				if ((pTri->periodic == 0) && (pNeigh->periodic > 0)
					&& (pTri->cornerptr[0]->pos.x > 0.0) )
				{
					u_2[0] = Clockwise*u_2[0];
					u_2[1] = Clockwise*u_2[1];
					u_2[2] = Clockwise*u_2[2];
				};
				if ((pNeigh->periodic == 0) && (pTri->periodic > 0)
				   && (pNeigh->cornerptr[0]->pos.x > 0.0))
				{
					u_2[0] = Anticlockwise*u_2[0];
					u_2[1] = Anticlockwise*u_2[1];
					u_2[2] = Anticlockwise*u_2[2];
				};

				pOpp = pNeigh->ReturnUnsharedVertex(pTri, &iWhich);
				// difference from a shared vertex:
				iprev = iNeigh-1; if (iprev == -1) iprev = 2;
				diff_ther = u_2[iWhich]-u_1[iprev];
				diff_ours = u_1[iNeigh]-u_1[iprev];
			
				if ( (diff_ours.dot(pTri->edge_normal[iNeigh]))
					*(diff_ther.dot(pTri->edge_normal[iNeigh])) > 0.0)
				{
					// same sign; error
					printf("overlap created by skips; should not be possible.\niTri %d ",iTri);
					iTri = iTri;
					getch();
					return true;
				};
			};

		};
	};
	return false;
}


long TriMesh::Flips(long Trilist[], short num)
{
	long iTri2;
	Triangle * pTri2, * pTri;
	Vertex * pVertq;
	Vector2 cc;
	real pdistsq;
	real qdistx,qdisty;//,pdistx,pdisty;
	long flips;
	int side;
	Vertex tempV;
	Vector2 nearest;
	int c1,c2;
	int iprev, inext;
	Vertex * pVertex1, * pVertex2, *pUnsh;
	ShardData shard_data1, shard_data2;
	Vector2 corner1,corner2,centre,projected, diff1, diff2;
	bool to_flip,perflag;
	Triangle newtri;

	static real const REL_TOLERANCE = 1.0e-11; 
	// 5e-13 does not seem to avoid back-and-forward flips due to rounding.

	long flip_tri_to_tri = 0;
	long highflip_tri_to_wedge = 0;
	long highflip_wedge_to_tri = 0;
	long lowflip_tri_to_wedge = 0;
	long lowflip_wedge_to_tri = 0;

	static real const COS60 = 0.5;
	long iTri;
	// whereas cos 90 = 0
	int iTriCaret;

	memset(flaglist, 0, sizeof(bool)*NMINOR);
	
	//printf("in flips routine.. line 1881  ");

	long totalflips = 0;	
	do
	{
		flips = 0;
		pTri = T;
		for (iTriCaret = 0; iTriCaret < num; ++iTriCaret)
		{
			// calculate circumcenter first....
			pTri = T + Trilist[iTriCaret];
		//	printf("@ %d ", Trilist[iTriCaret]);
			pTri->CalculateCircumcenter(cc, &pdistsq);
					
			for (int iNeigh = 0; iNeigh < 3; iNeigh++)
			{
				pTri2 = pTri->neighbours[iNeigh];
				// Is it in the list?
		//		printf("pTri %d : %d %d %d  : pTri2 %d : %d %d %d\n"
		//			"%1.8E %1.8E %1.8E %1.8E %1.8E %1.8E | %1.8E %1.8E %1.8E %1.8E %1.8E %1.8E\n",
		//			pTri - T, pTri->cornerptr[0] - X, pTri->cornerptr[1] - X, pTri->cornerptr[2] - X,
		//			pTri2 - T, pTri2->cornerptr[0] - X, pTri2->cornerptr[1] - X, pTri2->cornerptr[2] - X,
		//			pTri->cornerptr[0]->pos.x, pTri->cornerptr[0]->pos.y, pTri->cornerptr[1]->pos.x, pTri->cornerptr[1]->pos.y,
		//			pTri->cornerptr[2]->pos.x, pTri->cornerptr[2]->pos.y, pTri2->cornerptr[0]->pos.x, pTri2->cornerptr[0]->pos.y,
		//			pTri2->cornerptr[1]->pos.x, pTri2->cornerptr[1]->pos.y, pTri2->cornerptr[2]->pos.x, pTri2->cornerptr[2]->pos.y
//				);

				bool bother = false;
				for (int ii = 0; ii < num; ii++)
					if (Trilist[ii] == pTri2-T) bother = true;
						
			//	printf("bother = %d\n", (bother ? 1 : 0));

				if (bother) {
					if (	(pTri2 != pTri) 
						// Edit 04/04/19: change the following line to && to make life easier.
						// Flipping with tris within ins shouldn't be strictly necessary since we do not have
						// points inside but close to the insulator. It does stop us from being Delaunay if we do not do this though.
						&& (
								((pTri2->u8domain_flag == DOMAIN_TRIANGLE) || (pTri2->u8domain_flag == CROSSING_INS)) && 
								((pTri->u8domain_flag == DOMAIN_TRIANGLE) || (pTri->u8domain_flag == CROSSING_INS)) )
						&& (pTri2->cornerptr[0]->flags != OUTERMOST)
						&& (pTri2->cornerptr[1]->flags != OUTERMOST)
						&& (pTri2->cornerptr[2]->flags != OUTERMOST)
						)
					{
			//			printf("got here 1\n");

						pVertq = pTri2->ReturnUnsharedVertex(pTri);
							// we compare two triangles							
						real qdistsq = GetPossiblyPeriodicDistSq(pVertq->pos,cc); // less elegant but should still work.
						if (qdistsq < pdistsq-pdistsq*REL_TOLERANCE)
						{
							++flips;
							++flip_tri_to_tri;
							
			//				printf("pTri %d : %d %d %d --vs-- pTri2 %d : %d %d %d pVertq %d pdistsq %1.9E qdistsq %1.9E\n",
			//					pTri - T, pTri->cornerptr[0] - X, pTri->cornerptr[1] - X, pTri->cornerptr[2] - X,
			//					pTri2 - T, pTri2->cornerptr[0] - X, pTri2->cornerptr[1] - X, pTri2->cornerptr[2] - X,
			//					pVertq-X, pdistsq, qdistsq);
							
							Flip(pTri, pTri2, -1);		

						//	printf("DONE : pTri %d : %d %d %d  pTri2 %d : %d %d %d\n",
						//		pTri - T, pTri->cornerptr[0] - X, pTri->cornerptr[1] - X, pTri->cornerptr[2] - X,
						//		pTri2 - T, pTri2->cornerptr[0] - X, pTri2->cornerptr[1] - X, pTri2->cornerptr[2] - X);

					//		DebugTestWrongNumberTrisPerEdge();
							
							iNeigh = 4; // Skip out of loop.
						};
					}; // whether an edge worth looking through, vs, going off the domain
				}; // is it in the list
			}; // next neighbour			
		};
		// printf("Flips: %d \n", flips);	
		// Vertex::flags does not need to be changed as a result of this routine.

		totalflips += flips;
	} while ((flips > 0));

	//printf("ended Redelaunerize. totalflips: %d \n",totalflips);	
	// Checked that
	// 1. vertex neighs do not need to be maintained during Delaunay routines.
	// 2. Flip does actually maintain lists, but not order of tris or vertices.
	// Therefore, we must maintain afterwards.

//	DebugTestWrongNumberTrisPerEdge();

	return totalflips;
}


long TriMesh::Redelaunerize(bool exhaustion, bool bReplace)
{
	long iTri2;
	Triangle * pTri2, * pTri;
	Vertex * pVertq;
	Vector2 cc;
	real pdistsq;
	real qdistx,qdisty;//,pdistx,pdisty;
	long flips;
	int side;
	Vertex tempV;
	Vector2 nearest;
	int c1,c2;
	int iprev, inext;
	Vertex * pVertex1, * pVertex2, *pUnsh;
	ShardData shard_data1, shard_data2;
	Vector2 corner1,corner2,centre,projected, diff1, diff2;
	bool to_flip,perflag;
	Triangle newtri;

	static real const REL_TOLERANCE = 1.0e-11; 
	// 5e-13 does not seem to avoid back-and-forward flips due to rounding.

	long flip_tri_to_tri = 0;
	long highflip_tri_to_wedge = 0;
	long highflip_wedge_to_tri = 0;
	long lowflip_tri_to_wedge = 0;
	long lowflip_wedge_to_tri = 0;

	static real const COS60 = 0.5;
	long iTri;
	// whereas cos 90 = 0

	printf("start of redelaunerize");
	DebugTestWrongNumberTrisPerEdge();
	printf("got to here\n");

	memset(flaglist, 0, sizeof(bool)*NMINOR);

	long totalflips = 0;
	// if exhaustion == true, carry on to exhaustion; otherwise do 1 pass.
	do
	{
		flips = 0;
		pTri = T;
		for (iTri = 0; iTri < numTriangles; ++iTri)
		{
			// Do not play at outer edge of memory: (fluid replace is not designed to work there)

			if ((pTri->cornerptr[0]->flags == OUTERMOST) ||
				(pTri->cornerptr[1]->flags == OUTERMOST) ||
				(pTri->cornerptr[2]->flags == OUTERMOST) ||
				(pTri->u8domain_flag == OUTER_FRILL) ||
				(pTri->u8domain_flag == INNER_FRILL)
				) 
			{
				// do nothing
			} else {
				// calculate circumcenter first....
				pTri->CalculateCircumcenter(cc, &pdistsq);
					
				for (int iNeigh = 0; iNeigh < 3; iNeigh++)
				{
					
					pTri2 = pTri->neighbours[iNeigh];
					
					if (	(pTri2 != pTri) 
						// Edit 04/04/19: change the following line to && to make life easier.
						// Flipping with tris within ins shouldn't be strictly necessary since we do not have
						// points inside but close to the insulator. It does stop us from being Delaunay if we do not do this though.
						&& (
								((pTri2->u8domain_flag == DOMAIN_TRIANGLE) || (pTri2->u8domain_flag == CROSSING_INS)) && 
								((pTri->u8domain_flag == DOMAIN_TRIANGLE) || (pTri->u8domain_flag == CROSSING_INS)) )
						&& (pTri2->cornerptr[0]->flags != OUTERMOST)
						&& (pTri2->cornerptr[1]->flags != OUTERMOST)
						&& (pTri2->cornerptr[2]->flags != OUTERMOST)
						)
					{
						pVertq = pTri2->ReturnUnsharedVertex(pTri);
							// we compare two triangles
							
						// need to consider another case:
						// one of the neighbours is across PBC
						// in which case we should map q to same side as p

						real qdistsq = GetPossiblyPeriodicDistSq(pVertq->pos,cc); // less elegant but should still work.

						if (qdistsq < pdistsq-pdistsq*REL_TOLERANCE)
						{
							++flips;
							++flip_tri_to_tri;
							if ((bReplace)) { //&& ((pTri->u8domain_flag == DOMAIN_TRIANGLE) || (pTri2->u8domain_flag == DOMAIN_TRIANGLE))) {
								// need to do even if it's just AZ

								plasma_data data_tri1, data_tri2, data_vert;

								memcpy(&data_tri1, &(pData[pTri - T]), sizeof(plasma_data));
								memcpy(&data_tri2, &(pData[pTri2 - T]), sizeof(plasma_data));
								
								Flip(pTri, pTri2, -1);
								
								pVertex1 = pTri->ReturnUnsharedVertex(pTri2);
								pVertex2 = pTri2->ReturnUnsharedVertex(pTri);

								// simple average FOR NOW:
								
								// EXCEPTIONAL CASE 1: vertex is below insulator in which case just average tri data.
								// EXCEPTIONAL CASE 2: flip involved tris below ins: ruled this out above.
								// EXCEPTIONAL CASE 3: created 2 CROSSING_INS tris: fine.

								plasma_data temp;
								memcpy(&data_vert, &(pData[pVertex1 - X + BEGINNING_OF_CENTRAL]), sizeof(plasma_data));
								temp.Az = THIRD*(data_vert.Az + data_tri1.Az + data_tri2.Az);
								temp.Azdot = THIRD*(data_vert.Azdot + data_tri1.Azdot + data_tri2.Azdot);
								temp.B = THIRD*(data_vert.B + data_tri1.B + data_tri2.B);
								if (pVertex1->flags != DOMAIN_VERTEX) {
									temp.n = 0.5*(data_tri1.n + data_tri2.n);
									temp.n_n = 0.5*(data_tri1.n_n + data_tri2.n_n);
									temp.Tn = 0.5*(data_tri1.Tn + data_tri2.Tn);
									temp.Ti = 0.5*( data_tri1.Ti + data_tri2.Ti);
									temp.Te = 0.5*(data_tri1.Te + data_tri2.Te);
									temp.v_n = 0.5*( data_tri1.v_n + data_tri2.v_n);
									temp.vxy = 0.5*( data_tri1.vxy + data_tri2.vxy);
									temp.vez = 0.5*( data_tri1.vez + data_tri2.vez);
									temp.viz = 0.5*( data_tri1.viz + data_tri2.viz);
								} else {
									temp.n = THIRD*(data_vert.n + data_tri1.n + data_tri2.n);
									temp.n_n = THIRD*(data_vert.n_n + data_tri1.n_n + data_tri2.n_n);
									temp.Tn = THIRD*(data_vert.Tn + data_tri1.Tn + data_tri2.Tn);
									temp.Ti = THIRD*(data_vert.Ti + data_tri1.Ti + data_tri2.Ti);
									temp.Te = THIRD*(data_vert.Te + data_tri1.Te + data_tri2.Te);
									temp.v_n = THIRD*(data_vert.v_n + data_tri1.v_n + data_tri2.v_n);
									temp.vxy = THIRD*(data_vert.vxy + data_tri1.vxy + data_tri2.vxy);
									temp.vez = THIRD*(data_vert.vez + data_tri1.vez + data_tri2.vez);
									temp.viz = THIRD*(data_vert.viz + data_tri1.viz + data_tri2.viz);
								}								
								Vector2 pos0, pos1, pos2;
								pTri2->MapLeftIfNecessary(pos0, pos1, pos2); // check if we think this'll work.
								temp.pos = THIRD*(pos0 + pos1 + pos2);
								//temp.pos = THIRD*(
								//	pData[(pTri->cornerptr[0] - X) + BEGINNING_OF_CENTRAL].pos +
								//	pData[(pTri->cornerptr[1] - X) + BEGINNING_OF_CENTRAL].pos +
								//	pData[(pTri->cornerptr[2] - X) + BEGINNING_OF_CENTRAL].pos);
								//
								
								// Note we did not rotate vxy or v_n_xy. -- fail

								memcpy(&(pData[pTri - T]), &temp, sizeof(plasma_data)); // so must populate every member

								memcpy(&data_vert, &(pData[pVertex2 - X + BEGINNING_OF_CENTRAL]), sizeof(plasma_data));
								temp.Az = THIRD*(data_vert.Az + data_tri1.Az + data_tri2.Az);
								temp.Azdot = THIRD*(data_vert.Azdot + data_tri1.Azdot + data_tri2.Azdot);
								temp.B = THIRD*(data_vert.B + data_tri1.B + data_tri2.B);

								if (pVertex2->flags != DOMAIN_VERTEX) {
									temp.n = 0.5*(data_tri1.n + data_tri2.n);
									temp.n_n = 0.5*(data_tri1.n_n + data_tri2.n_n);
									temp.Tn = 0.5*(data_tri1.Tn + data_tri2.Tn);
									temp.Ti = 0.5*(data_tri1.Ti + data_tri2.Ti);
									temp.Te = 0.5*(data_tri1.Te + data_tri2.Te);
									temp.v_n = 0.5*(data_tri1.v_n + data_tri2.v_n);
									temp.vxy = 0.5*(data_tri1.vxy + data_tri2.vxy);
									temp.vez = 0.5*(data_tri1.vez + data_tri2.vez);
									temp.viz = 0.5*(data_tri1.viz + data_tri2.viz);
								} else {
									temp.n = THIRD*(data_vert.n + data_tri1.n + data_tri2.n);
									temp.n_n = THIRD*(data_vert.n_n + data_tri1.n_n + data_tri2.n_n);
									temp.Tn = THIRD*(data_vert.Tn + data_tri1.Tn + data_tri2.Tn);
									temp.Ti = THIRD*(data_vert.Ti + data_tri1.Ti + data_tri2.Ti);
									temp.Te = THIRD*(data_vert.Te + data_tri1.Te + data_tri2.Te);
									temp.v_n = THIRD*(data_vert.v_n + data_tri1.v_n + data_tri2.v_n);
									temp.vxy = THIRD*(data_vert.vxy + data_tri1.vxy + data_tri2.vxy);
									temp.vez = THIRD*(data_vert.vez + data_tri1.vez + data_tri2.vez);
									temp.viz = THIRD*(data_vert.viz + data_tri1.viz + data_tri2.viz);
								}

							//	temp.pos = THIRD*(
							//		pData[(pTri2->cornerptr[0] - X) + BEGINNING_OF_CENTRAL].pos +
									//			pData[(pTri2->cornerptr[1] - X) + BEGINNING_OF_CENTRAL].pos +
									//			pData[(pTri2->cornerptr[2] - X) + BEGINNING_OF_CENTRAL].pos);

											// We could need to bring positions to same periodic orientation as triangle.
							
								pTri2->MapLeftIfNecessary(pos0, pos1, pos2); // check if we think this'll work.
								temp.pos = THIRD*(pos0 + pos1 + pos2);
								// pos012 shuold also allow us to conclude corners of tri minor cell.
								/*
								// Constructing Lap Az:
								f64 LapAz = 0.0;
								if (pTri->neighbours[0] != pTri2) {
									lap = 0;
									neighpos = pData[pNeigh - T].pos;
									// adjust for periodic:


								} else {
									neighpos = pData[pNeigh - T].pos;

									coeff_12 = ;
								}
								LapAz /= area;
								coeff_12 /= area;

								// Now solve: Lap Az = -4pi/c Jz

								f64 det = coeff_11*coeff_22 - coeff_12*coeff_21;
								inv_11 = coeff_22 / det;
								inv_12 = -coeff_12 / det;
								inv_21 = -coeff_21 / det;
								inv_22 = coeff_11 / det;

								temp1.Az = inv_11*(-LapAz1 - FOURPIOVERC_*Jz1) + inv_12*(-LapAz2 - FOURPIOVERC_*Jz2);
								temp2.Az = inv_21*(-LapAz1 - FOURPIOVERC_*Jz1) + inv_22*(-LapAz2 - FOURPIOVERC_*Jz2);
								*/

								memcpy(&(pData[pTri2 - T]), &temp, sizeof(plasma_data)); // so must populate every member

								printf("FLIP: %d %d %d %d :Az %1.8E %1.8E | %1.8E %1.8E \n"
									"viz %1.8E %1.8E | %1.8E %1.8E \n", pVertex1 - X, pVertex2-X, pTri - T, pTri2 - T,
									data_tri1.Az, data_tri2.Az, pData[pTri - T].Az, pData[pTri2 - T].Az,
									data_tri1.viz, data_tri2.viz, pData[pTri - T].viz, pData[pTri2 - T].viz);
								 
								flaglist[pTri2 - T] = true;
								flaglist[pTri - T] = true;

								/*
								
								
								// pTri2 is neighbours[iNeigh]
								// so pTri->cornerptr[iNeigh] is unshared.
								
								// Get shared vertices:
								inext = iNeigh+1; if (inext == 3) inext = 0;
								iprev = inext+1; if (iprev == 3) iprev = 0;
								pVertex1 = pTri->cornerptr[inext];
								pVertex2 = pTri->cornerptr[iprev];
								// These will be the giving vertcells. IF domain.
								if (pVertex1->flags == DOMAIN_VERTEX)
									CreateShards(pVertex1,shard_data1);  // seems to be assuming an interior vertex.
								if (pVertex2->flags == DOMAIN_VERTEX)
									CreateShards(pVertex2,shard_data2);
								pUnsh = pTri->cornerptr[iNeigh];
								
								// store indices (in shard array) of
								// corners where cell of pVertex1 meets pUnsh,pVertq...
								
								// That means finding common tri that is not
								// one of the flipping ones.
								
								// There could be confusion though if only 4 tris on vertex
								// - maybe it's better not to try to be clever but to just
								// intersect every shard with both destinations.
								
								Flip(pTri, pTri2, -1);
							
								// take from pVertex1 whatever we send to pUnsh:
								if ((pUnsh->flags == DOMAIN_VERTEX) && (pVertex1->flags == DOMAIN_VERTEX))
									GiveAndTake(shard_data1,pUnsh,pVertex1);
								if ((pVertq->flags == DOMAIN_VERTEX) && (pVertex1->flags == DOMAIN_VERTEX))
									GiveAndTake(shard_data1,pVertq,pVertex1);
								if ((pUnsh->flags == DOMAIN_VERTEX) && (pVertex2->flags == DOMAIN_VERTEX))
									GiveAndTake(shard_data2,pUnsh,pVertex2);
								if ((pVertq->flags == DOMAIN_VERTEX) && (pVertex2->flags == DOMAIN_VERTEX))
									GiveAndTake(shard_data2,pVertq,pVertex2);

									*/
							} else {
								Flip(pTri, pTri2, -1);
							};							
							iNeigh = 4; // Skip out of loop.
						};
					}; // whether an edge worth looking through, vs, going off the domain
				}; // next neighbour
			};		
		};

		printf("Flips: %d \n", flips);
	// Vertex::flags does not need to be changed as a result of this routine.
//#ifndef RELEASE
		if (flips > 0) DebugTestWrongNumberTrisPerEdge();
//#endif
		totalflips += flips;
	} while ((exhaustion == true) && (flips > 0));

	printf("ended Redelaunerize. totalflips: %d \n",totalflips);
	
	// Checked that
	// 1. vertex neighs do not need to be maintained during Delaunay routines.
	// 2. Flip does actually maintain lists, but not order of tris or vertices.
	// Therefore, we maintain afterwards:

	pTri = T;
	for (iTri = 0; iTri < numTriangles; iTri++)
	{
		pTri->RecalculateCentroid(this->InnermostFrillCentroidRadius, this->OutermostFrillCentroidRadius);
		++pTri; // this seems like it should still work if we have not wrapped any vertex that moved, even if tri no longer periodic in truth but some pts outside tranche
	};

	if (totalflips > 0) {
		this->RefreshVertexNeighboursOfVerticesOrdered();
	//	printf("press z"); // debug
	//	while (getch() != 'z');
	};

	return totalflips;
}

void TriMesh::GiveAndTake(ShardData & shard_data, Vertex * pVDest,Vertex * pVSrc)
{
	// ASSUME TRI CENTROIDS SET

	// intersect shards with pVDest, take away results from pVDest.
	// Should not get zero.
	// No motion going on when this is called, to cause heating/cooling - only flips.

	// Can assume shard_data has same rotation as pVSrc -- so iff we have to rotate
	// to intersect with pVDest, then rotate back when we subtract.

	int i,inext;
	bool bIntersected;
	ConvexPolygon cptri,cptri2, cpIntersection, cpDest;
	long izTri[128], tri_len;
	Triangle * pTri;
	fluid_nvT vals0,vals1,vals2;
	fluid_NvT integrals;
	/*

	// First create ConvexPolygon for pVDest. We have to streamline this...
	tri_len = pVDest->GetTriIndexArray(izTri);
	cpDest.Clear();
	for (i = 0; i < tri_len; i++)
	{
		pTri = T + izTri[i];
		cpDest.add(pTri->GetContiguousCent_AssumingCentroidsSet(pVDest));
	};

	for (i = 0; i < shard_data.len; i++)
	{
		inext = i+1; if (inext == shard_data.len) inext = 0;
		cptri.Clear();
		cptri.add(shard_data.central);
		cptri.add(shard_data.cp.coord[i]);
		cptri.add(shard_data.cp.coord[inext]);

		bIntersected = cpDest.GetIntersectionWithTriangle(
			&cpIntersection, cptri.coord[0],cptri.coord[1],cptri.coord[2]);
		
		if (bIntersected) {
			
			cpIntersection.Integrate_Planes(
							cptri.coord[0],cptri.coord[1],cptri.coord[2],
							(real *)(&(shard_data.cdata)), // pass object as array of reals
							(real *)(&(shard_data.fluidnvT[i])),
							(real *)(&(shard_data.fluidnvT[inext])),
							(real *)(&integrals),// output - this is of form {N,NT,Nv} x 3.
							15);
			// Give, and take:
			pVDest->Neut.mass += integrals.N[0];
			pVSrc->Neut.mass -= integrals.N[0];
			pVDest->Neut.heat += integrals.NT[0];
			pVSrc->Neut.heat -= integrals.NT[0];
			pVDest->Neut.mom += integrals.Nv[0];
			pVSrc->Neut.mom -= integrals.Nv[0];
			pVDest->Ion.mass += integrals.N[1];
			pVSrc->Ion.mass -= integrals.N[1];
			pVDest->Ion.heat += integrals.NT[1];
			pVSrc->Ion.heat -= integrals.NT[1];
			pVDest->Ion.mom += integrals.Nv[1];
			pVSrc->Ion.mom -= integrals.Nv[1];
			pVDest->Elec.mass += integrals.N[2];
			pVSrc->Elec.mass -= integrals.N[2];
			pVDest->Elec.heat += integrals.NT[2];
			pVSrc->Elec.heat -= integrals.NT[2];
			pVDest->Elec.mom += integrals.Nv[2];
			pVSrc->Elec.mom -= integrals.Nv[2];

		} else {

			// Let's be careful: we could be heading either way to another tranche.

			if ((pVDest->pos.x > 0.0) && (shard_data.central.x < 0.0))
			{
				// Make sure whether rotating shard_data to right might hit it:
				cptri2.CreateClockwiseImage(cptri);

				bIntersected = cpDest.GetIntersectionWithTriangle(
					&cpIntersection, cptri2.coord[0],
					cptri2.coord[1],
					cptri2.coord[2]);
				if (bIntersected) {

					vals0 = shard_data.cdata.Clockwise();
					vals1 = shard_data.fluidnvT[i].Clockwise();
					vals2 = shard_data.fluidnvT[inext].Clockwise();

					cpIntersection.Integrate_Planes(
							cptri2.coord[0],
							cptri2.coord[1],
							cptri2.coord[2],
							(real *)(&vals0),
							(real *)(&vals1),
							(real *)(&vals2),
							(real *)(&integrals),// output
							15);
					// Give, and take:
					pVDest->Neut.mass += integrals.N[0];
					pVSrc->Neut.mass -= integrals.N[0];
					pVDest->Neut.heat += integrals.NT[0];
					pVSrc->Neut.heat -= integrals.NT[0];
					pVDest->Neut.mom += integrals.Nv[0];
					pVSrc->Neut.mom -= Anticlockwise3*integrals.Nv[0];
					pVDest->Ion.mass += integrals.N[1];
					pVSrc->Ion.mass -= integrals.N[1];
					pVDest->Ion.heat += integrals.NT[1];
					pVSrc->Ion.heat -= integrals.NT[1];
					pVDest->Ion.mom += integrals.Nv[1];
					pVSrc->Ion.mom -= Anticlockwise3*integrals.Nv[1];
					pVDest->Elec.mass += integrals.N[2];
					pVSrc->Elec.mass -= integrals.N[2];
					pVDest->Elec.heat += integrals.NT[2];
					pVSrc->Elec.heat -= integrals.NT[2];
					pVDest->Elec.mom += integrals.Nv[2];
					pVSrc->Elec.mom -= Anticlockwise3*integrals.Nv[2];
					
				};
				
			};
			if ((pVDest->pos.x < 0.0) && (shard_data.central.x > 0.0))
			{
				// Try rotating shard_data anticlockwise
				cptri2.CreateAnticlockwiseImage(cptri);
				
				bIntersected = cpDest.GetIntersectionWithTriangle(
					&cpIntersection, cptri2.coord[0],
					cptri2.coord[1],
					cptri2.coord[2]);
				if (bIntersected) {

					vals0 = shard_data.cdata.Anticlockwise();
					vals1 = shard_data.fluidnvT[i].Anticlockwise();
					vals2 = shard_data.fluidnvT[inext].Anticlockwise();

					cpIntersection.Integrate_Planes(
							cptri2.coord[0],cptri2.coord[1],cptri2.coord[2],
							(real *)(&vals0),
							(real *)(&vals1),
							(real *)(&vals2),
							(real *)(&integrals),// output
							15);

					// Give, and take:
					pVDest->Neut.mass += integrals.N[0];
					pVSrc->Neut.mass -= integrals.N[0];
					pVDest->Neut.heat += integrals.NT[0];
					pVSrc->Neut.heat -= integrals.NT[0];
					pVDest->Neut.mom += integrals.Nv[0];
					pVSrc->Neut.mom -= Clockwise3*integrals.Nv[0];
					pVDest->Ion.mass += integrals.N[1];
					pVSrc->Ion.mass -= integrals.N[1];
					pVDest->Ion.heat += integrals.NT[1];
					pVSrc->Ion.heat -= integrals.NT[1];
					pVDest->Ion.mom += integrals.Nv[1];
					pVSrc->Ion.mom -= Clockwise3*integrals.Nv[1];
					pVDest->Elec.mass += integrals.N[2];
					pVSrc->Elec.mass -= integrals.N[2];
					pVDest->Elec.heat += integrals.NT[2];
					pVSrc->Elec.heat -= integrals.NT[2];
					pVDest->Elec.mom += integrals.Nv[2];
					pVSrc->Elec.mom -= Clockwise3*integrals.Nv[2];
				};
			};
		};		
	};*/
}

void TriMesh::CreateShards(Vertex * pVertex, ShardData & shard_data)
{
	long izTri[128];
	Vector2 u[3];
	ShardData temp;
	fluid_nvT fluids[3];
	int tri_len, i;
	Triangle * pTri;
	int parity[3];
	real beta[3];
	bool found;
	real coeffremain;

	// ASSUMES TRI CENTROIDS POPULATED
	// DOES NOT ASSUME VERTEX CENTROIDS POPULATED
	// ASSUMES VERTEX AREACELL POPULATED

	// Maybe we should chalk up what things to actually MAINTAIN.

	// ****&&&&****&&&&****&&&&****&&&&****&&&&****&&&&****&&&&****&&&&****

	// Vertex centroid should be maintained, and removed from this routine.

	/*
	
	shard_data.cp.Clear();

	// 1A. Add tri centroids to shard_data.cp
	tri_len = pVertex->GetTriIndexArray(izTri);
	shard_data.len = tri_len;

	for (i = 0; i < tri_len; i++)
	{
		pTri = T + izTri[i];

		if ((pTri->periodic) && (pVertex->pos.x > 0.0))
		{
			shard_data.cp.add(Clockwise*pTri->cent);
		} else {
			shard_data.cp.add(pTri->cent);
		};
		
		if (pTri->u8domain_flag == DOMAIN_TRIANGLE) {
			// set these first...
			// To set desired values at pTri's centroid (WHICH IS ASSUMED POPULATED)
			// need to ensure vertex centroids are populated:
			for (int iCorner = 0; iCorner < 3; iCorner++)
				RecalculateCentroid(pTri->cornerptr[iCorner]);

			u[0] = pTri->cornerptr[0]->centroid;
			u[1] = pTri->cornerptr[1]->centroid;
			u[2] = pTri->cornerptr[2]->centroid;
			if (pTri->periodic == 0) {
			} else {
				pTri->GetParity(parity); // 1 = clockwise side
				if (parity[0]) u[0] = Anticlockwise*u[0];
				if (parity[1]) u[1] = Anticlockwise*u[1];
				if (parity[2]) u[2] = Anticlockwise*u[2];			
			};		

			for (int iCorner = 0; iCorner < 3; iCorner++)
			{
				fluids[iCorner].n[0] = pTri->cornerptr[iCorner]->Neut.mass/pTri->cornerptr[iCorner]->AreaCell;
				fluids[iCorner].nv[0] = pTri->cornerptr[iCorner]->Neut.mom/pTri->cornerptr[iCorner]->AreaCell;
				fluids[iCorner].nT[0] = pTri->cornerptr[iCorner]->Neut.heat/pTri->cornerptr[iCorner]->AreaCell;
				fluids[iCorner].n[1] = pTri->cornerptr[iCorner]->Ion.mass/pTri->cornerptr[iCorner]->AreaCell;
				fluids[iCorner].nv[1] = pTri->cornerptr[iCorner]->Ion.mom/pTri->cornerptr[iCorner]->AreaCell;
				fluids[iCorner].nT[1] = pTri->cornerptr[iCorner]->Ion.heat/pTri->cornerptr[iCorner]->AreaCell;
				fluids[iCorner].n[2] = pTri->cornerptr[iCorner]->Elec.mass/pTri->cornerptr[iCorner]->AreaCell;
				fluids[iCorner].nv[2] = pTri->cornerptr[iCorner]->Elec.mom/pTri->cornerptr[iCorner]->AreaCell;
				fluids[iCorner].nT[2] = pTri->cornerptr[iCorner]->Elec.heat/pTri->cornerptr[iCorner]->AreaCell;
			};
			
			if (pTri->periodic == 0) {
				// do nothing to it.
			} else {
				// Have to rotate some momentum vectors to supply desired value contig to pTri,
				// as well as rotating positions u.				
				pTri->GetParity(parity); // 1 = clockwise side				
				for (int iCorner = 0; iCorner < 3; iCorner++)
				{
					if (parity[iCorner]) {
						fluids[iCorner].nv[0] = Anticlockwise3*fluids[iCorner].nv[0];
						fluids[iCorner].nv[1] = Anticlockwise3*fluids[iCorner].nv[1];
						fluids[iCorner].nv[2] = Anticlockwise3*fluids[iCorner].nv[2];
					};
				};
			};		
			GetInterpolationCoefficients(beta,pTri->cent.x,pTri->cent.y,u[0],u[1],u[2]);
			temp.fluidnvT[i].Interpolate(beta,fluids);
			
			// This made it contiguous for pTri.
			
		} else {
			// Not domain tri: do nothing; go again below:
		}
	};

	// 1B. Create desired values n,nT,nv at each CROSSING_INS tri centroid:
	long index[128];
	Triangle * pDomainTri;
	int numIntermed = 0;
	int iWhich, iNeigh, ii;
	real dist1, dist2, wt1, wt2, wtsum;
	int index_edge1 = -1, index_edge2 = -1;
	for (i = 0; i < tri_len; i++)
	{
		pTri = T + izTri[i];

		// To set desired values at pTri's centroid (WHICH IS ASSUMED POPULATED)
		// need to ensure vertex centroids are populated:
		if (pTri->u8domain_flag != DOMAIN_TRIANGLE) {
			if (pTri->u8domain_flag == CROSSING_INS) {

				// does it have a neighbour in the domain: if so, find it in the existing list of values.
				// It will be either i+1 (anticlock from here) or i-1 (clockwise from here).
				
				// Go again and set any that are left over; make a list of them here.
				
				if ((pTri->neighbours[0]->u8domain_flag != DOMAIN_TRIANGLE)
					&&
					(pTri->neighbours[1]->u8domain_flag != DOMAIN_TRIANGLE)
					&&
					(pTri->neighbours[2]->u8domain_flag != DOMAIN_TRIANGLE))
				{
					index[numIntermed] = i;
					memset(&(temp.fluidnvT[i]),0,sizeof(fluid_nvT)); // avoid crash later
					numIntermed++;
				} else {
					// keep note of index:
					if (index_edge1 == -1) {
						index_edge1 = i;
					} else {
						index_edge2 = i;
					};

					// Find domain neighbour in our list:
	
					iWhich = 0;
					while (pTri->neighbours[iWhich]->u8domain_flag != DOMAIN_TRIANGLE) iWhich++;
					iNeigh = pTri->neighbours[iWhich]-T;
					ii = 0; 
					while ((izTri[ii] != iNeigh) && (ii < tri_len)) ii++;
					if (ii == tri_len) {
						printf("error348231\n");
						getch();
					};
					
					// Now use two shared corners and the centre of this domain neighbour to create a plane;
					// the plane infers us the desirable value of n, nT, nv at our pTri centroid.
					// We can use Interpolate of course.

					pDomainTri = T + iNeigh;
					
					// First do exactly as before ... some of the data may be nonsense but it will be overwritten.
					u[0] = pTri->cornerptr[0]->centroid;
					u[1] = pTri->cornerptr[1]->centroid;
					u[2] = pTri->cornerptr[2]->centroid;
					if (pTri->periodic == 0) {
					} else {
						pTri->GetParity(parity); // 1 = clockwise side
						if (parity[0]) u[0] = Anticlockwise*u[0];
						if (parity[1]) u[1] = Anticlockwise*u[1];
						if (parity[2]) u[2] = Anticlockwise*u[2];			
					};
					for (int iCorner = 0; iCorner < 3; iCorner++)
					{
						fluids[iCorner].n[0] = pTri->cornerptr[iCorner]->Neut.mass/pTri->cornerptr[iCorner]->AreaCell;
						fluids[iCorner].nv[0] = pTri->cornerptr[iCorner]->Neut.mom/pTri->cornerptr[iCorner]->AreaCell;
						fluids[iCorner].nT[0] = pTri->cornerptr[iCorner]->Neut.heat/pTri->cornerptr[iCorner]->AreaCell;
						fluids[iCorner].n[1] = pTri->cornerptr[iCorner]->Ion.mass/pTri->cornerptr[iCorner]->AreaCell;
						fluids[iCorner].nv[1] = pTri->cornerptr[iCorner]->Ion.mom/pTri->cornerptr[iCorner]->AreaCell;
						fluids[iCorner].nT[1] = pTri->cornerptr[iCorner]->Ion.heat/pTri->cornerptr[iCorner]->AreaCell;
						fluids[iCorner].n[2] = pTri->cornerptr[iCorner]->Elec.mass/pTri->cornerptr[iCorner]->AreaCell;
						fluids[iCorner].nv[2] = pTri->cornerptr[iCorner]->Elec.mom/pTri->cornerptr[iCorner]->AreaCell;
						fluids[iCorner].nT[2] = pTri->cornerptr[iCorner]->Elec.heat/pTri->cornerptr[iCorner]->AreaCell;
					};					
					if (pTri->periodic != 0) 
					{
						// Have to rotate some momentum vectors to supply desired value contig to pTri,
						// as well as rotating positions u.				
						pTri->GetParity(parity); // 1 = clockwise side				
						for (int iCorner = 0; iCorner < 3; iCorner++)
						{
							if (parity[iCorner]) {
								fluids[iCorner].nv[0] = Anticlockwise3*fluids[iCorner].nv[0];
								fluids[iCorner].nv[1] = Anticlockwise3*fluids[iCorner].nv[1];
								fluids[iCorner].nv[2] = Anticlockwise3*fluids[iCorner].nv[2];
							};
						};
					};		

					// Now overwrite the bottom corner of this tri with the centre of the tri above it.
					u[iWhich] = pDomainTri->cent;
					fluids[iWhich] = temp.fluidnvT[ii];
					// These should already be contiguous for pTri, right? Actually might not be.
					if ( ((pDomainTri->periodic != 0) && (pTri->periodic == 0))
						||
						 ((pDomainTri->periodic == 0) && (pTri->periodic != 0)) )
					{
						printf("error ... not coded for this.\n");
						getch();
					}					
					GetInterpolationCoefficients(beta,pTri->cent.x,pTri->cent.y,u[0],u[1],u[2]);
					temp.fluidnvT[i].Interpolate(beta,fluids);										
				};				
			} else {
				printf("error: pTri->u8domain_flag = %d \n",pTri->u8domain_flag);
				getch();
			};
		};
	};

	for (i = 0; i < tri_len; i++)
	{
		pTri = T + izTri[i];
		// BETTER TO DO THIS LAST. MAKE EACH TRI'S fluidnvT contiguous TO ITS OWN TRI FIRST.
		// NOW MAKE IT CONTIGUOUS TO pVertex
		if ((pTri->periodic) && (pVertex->pos.x > 0.0))
		{
			temp.fluidnvT[i].nv[0] = Clockwise3*temp.fluidnvT[i].nv[0];
			temp.fluidnvT[i].nv[1] = Clockwise3*temp.fluidnvT[i].nv[1];
			temp.fluidnvT[i].nv[2] = Clockwise3*temp.fluidnvT[i].nv[2];
		};
	};
	
	// Now all are contiguous to pVertex, finally:
	if (numIntermed > 0) {
		// Fill in intermediate values along the insulator by just interpolating left to right between
		// the guard triangles index_edge1, index_edge2 :

		for (i = 0; i < numIntermed; i++)
		{
			iWhich = index[i];
			pTri = T + izTri[iWhich];
			dist1 = GetPossiblyPeriodicDist(shard_data.cp.coord[i],shard_data.cp.coord[index_edge1]);
			dist2 = GetPossiblyPeriodicDist(shard_data.cp.coord[i],shard_data.cp.coord[index_edge2]);
			wt1 = 1.0/dist1;
			wt2 = 1.0/dist2;
			wtsum = wt1+wt2;
			beta[0] = wt1/wtsum;
			beta[1] = wt2/wtsum;
			beta[2] = 0.0;
			fluids[0] = temp.fluidnvT[index_edge1];
			fluids[1] = temp.fluidnvT[index_edge2];
			fluids[2] = temp.fluidnvT[index_edge1]; // doesn't matter what
			temp.fluidnvT[i].Interpolate(beta,fluids);
		};
	};
	// Hopefully that's it.

	
	// ===
	// Is minmod even beneficial much? Why is it better than setting the desired edge values
	// and then just setting the centre, generally.
	// Think it's beneficial if there is a cliff...

	// MINMOD IS GOOD.

	// This has to be done for each of 15 surfaces (??)
	// Pretty much.

	//shard_data.Minmod(temp);

	real y[128], result[128];
	for (i = 0; i < tri_len; i++)
		y[i] = temp.fluidnvT[i].n[0];// temp.fluidnvT are the desired values.
	shard_data.cp.minmod(result,y,pVertex->Neut.mass,pVertex->pos);
	for (i = 0; i < tri_len; i++)
		shard_data.fluidnvT[i].n[0] = result[i];
	
	for (i = 0; i < tri_len; i++)
		y[i] = temp.fluidnvT[i].nT[0];
	shard_data.cp.minmod(result,y,pVertex->Neut.heat,pVertex->pos);
	for (i = 0; i < tri_len; i++)
		shard_data.fluidnvT[i].nT[0] = result[i];
	
	for (i = 0; i < tri_len; i++)
		y[i] = temp.fluidnvT[i].nv[0].x;
	shard_data.cp.minmod(result,y,pVertex->Neut.mom.x,pVertex->pos);
	for (i = 0; i < tri_len; i++)
		shard_data.fluidnvT[i].nv[0].x = result[i];
	for (i = 0; i < tri_len; i++)
		y[i] = temp.fluidnvT[i].nv[0].y;
	shard_data.cp.minmod(result,y,pVertex->Neut.mom.y,pVertex->pos);
	for (i = 0; i < tri_len; i++)
		shard_data.fluidnvT[i].nv[0].y = result[i];
	for (i = 0; i < tri_len; i++)
		y[i] = temp.fluidnvT[i].nv[0].z;
	shard_data.cp.minmod(result,y,pVertex->Neut.mom.z,pVertex->pos);
	for (i = 0; i < tri_len; i++)
		shard_data.fluidnvT[i].nv[0].z = result[i];

		for (i = 0; i < tri_len; i++)
		y[i] = temp.fluidnvT[i].n[1];// temp.fluidnvT are the desired values.
	shard_data.cp.minmod(result,y,pVertex->Ion.mass,pVertex->pos);
	for (i = 0; i < tri_len; i++)
		shard_data.fluidnvT[i].n[1] = result[i];
	
	for (i = 0; i < tri_len; i++)
		y[i] = temp.fluidnvT[i].nT[1];
	shard_data.cp.minmod(result,y,pVertex->Ion.heat,pVertex->pos);
	for (i = 0; i < tri_len; i++)
		shard_data.fluidnvT[i].nT[1] = result[i];
	
	for (i = 0; i < tri_len; i++)
		y[i] = temp.fluidnvT[i].nv[1].x;
	shard_data.cp.minmod(result,y,pVertex->Ion.mom.x,pVertex->pos);
	for (i = 0; i < tri_len; i++)
		shard_data.fluidnvT[i].nv[1].x = result[i];
	for (i = 0; i < tri_len; i++)
		y[i] = temp.fluidnvT[i].nv[1].y;
	shard_data.cp.minmod(result,y,pVertex->Ion.mom.y,pVertex->pos);
	for (i = 0; i < tri_len; i++)
		shard_data.fluidnvT[i].nv[1].y = result[i];
	for (i = 0; i < tri_len; i++)
		y[i] = temp.fluidnvT[i].nv[1].z;
	shard_data.cp.minmod(result,y,pVertex->Ion.mom.z,pVertex->pos);
	for (i = 0; i < tri_len; i++)
		shard_data.fluidnvT[i].nv[1].z = result[i];

	for (i = 0; i < tri_len; i++)
		y[i] = temp.fluidnvT[i].n[2];// temp.fluidnvT are the desired values.
	shard_data.cp.minmod(result,y,pVertex->Elec.mass,pVertex->pos);
	for (i = 0; i < tri_len; i++)
		shard_data.fluidnvT[i].n[2] = result[i];
	for (i = 0; i < tri_len; i++)
		y[i] = temp.fluidnvT[i].nT[2];
	shard_data.cp.minmod(result,y,pVertex->Elec.heat,pVertex->pos);
	for (i = 0; i < tri_len; i++)
		shard_data.fluidnvT[i].nT[2] = result[i];
	
	for (i = 0; i < tri_len; i++)
		y[i] = temp.fluidnvT[i].nv[2].x;
	shard_data.cp.minmod(result,y,pVertex->Elec.mom.x,pVertex->pos);
	for (i = 0; i < tri_len; i++)
		shard_data.fluidnvT[i].nv[2].x = result[i];
	for (i = 0; i < tri_len; i++)
		y[i] = temp.fluidnvT[i].nv[2].y;
	shard_data.cp.minmod(result,y,pVertex->Elec.mom.y,pVertex->pos);
	for (i = 0; i < tri_len; i++)
		shard_data.fluidnvT[i].nv[2].y = result[i];
	for (i = 0; i < tri_len; i++)
		y[i] = temp.fluidnvT[i].nv[2].z;
	shard_data.cp.minmod(result,y,pVertex->Elec.mom.z,pVertex->pos);
	for (i = 0; i < tri_len; i++)
		shard_data.fluidnvT[i].nv[2].z = result[i];

	// Decided I don't like the way round that data is being stored...
	
	// Write minmod routine; change around afterwards.
	*/
}



real ConvexPolygon::minmod(real n[], // output array
					  real ndesire[], // array of desired values 
						real N,      // total mass N
					  Vector2 central // central position
) // returns centre n
{

	// 2. Decide whether we can attain these values and get N,NT,Nv by setting
	// n,nT,nv at vertex to be between highest and lowest corner desired value.
	
	// If not, do not move a lower corner than n_avg, even lower, to attain a lower mass
	// than we can get by putting the centre on a par with the lowest.
	// instead, attain the desired low values then see how much mass then there is available to
	// push somewhat up to the high corners, in sequence.

	// Note: if n_avg < all desired corners, that is a sign we have to default to say 
	// constant-in-cell.
	real coeff[128];
	bool fixed[128];
	ConvexPolygon cptri;
	real tri_area, coeffcent, N0, n_C,n_acceptable,N_attained, coeffremain;
	real low_n,high_n;
	int i,inext;
	bool found;
		

	low_n = ndesire[0];
	high_n = ndesire[0];
	i = 0;
	while (i < numCoords) {
		if (low_n > ndesire[i]) low_n = ndesire[i];
		if (high_n < ndesire[i]) high_n = ndesire[i];
		++i;
	};
	
	real area = GetArea();
	real n_avg = N/area;
	if ((n_avg > high_n) || (n_avg < low_n)){
		// above/below all of them: minmod says give up and set constant
		for (i = 0;i < numCoords; i++)
			n[i] = n_avg;
		return n_avg; // n_C 
	};
	// Perhaps not best behaviour -- rather we should treat each as an elastic link and min sum of squares.
	
	// Now see if we can set n_avg to a value that achieves ndesire and N.
	// work up a coefficient on n_C as well as what ndesire is giving us.

	// We assign to each corner a coefficient to make life easier.
	memset(coeff,0,sizeof(real)*128);
	coeffcent = 0.0;
	N0 = 0.0;
	for (i = 0; i < numCoords; i++)
	{
		inext = i+1; if (inext == numCoords) inext = 0;
		
		cptri.Clear();
		cptri.add(coord[i]);
		cptri.add(coord[inext]);
		cptri.add(central);
		tri_area = cptri.GetArea();
		
		N0 += tri_area*THIRD*(ndesire[i]+ndesire[inext]);
		coeff[i] += tri_area*THIRD;
		coeff[inext] += tri_area*THIRD;
		coeffcent += tri_area*THIRD;
	};
	
	real n_C_need = (N-N0)/coeffcent;
	if ((n_C_need > low_n) && (n_C_need < high_n))
	{
		// accept:
		for (i = 0;i < numCoords; i++)
			n[i] = ndesire[i];
		return n_C_need;  // hopefully this is frequently the case.
	};
	
	memset(fixed,0,sizeof(bool)*128);
			
	if (n_C_need < low_n) {
		// the mass is low. So for those less than n_avg let's fix
		// them in place, and fix n_C = low_n.
		// Then we'll see how high we can go.		
		n_C = low_n;
		n_acceptable = (N - coeffcent*n_C )/(area - THIRD*area);
		// area-THIRD*area = sum of other coeffs, and of course
		// coeffcent = THIRD*area
		// n_acceptable > N/area since N=area*n_avg > area*low_n.
		
		// We accept things that are less than this 'max average', and
		// let that increase the threshold; go again until
		// the time we do not find any new lower items ;				
		do {
			found = 0;	
			coeffremain = 0.0;
			N_attained = coeffcent*low_n;
			for (i = 0; i < numCoords; i++)
			{
				if (fixed[i] == 0) {
					if (ndesire[i] < n_acceptable) {
						// yes, use ndesire[i] ...
						fixed[i] = true;
						n[i] = ndesire[i];
						N_attained += n[i]*coeff[i];
						found = true;
					} else {
						coeffremain += coeff[i];
					};
				} else {
					N_attained += n[i]*coeff[i];
				};
			};
			// It can happen that eventually ALL are found
			// to be < n_acceptable due to FP error.
			// On next pass found will be false.
			if ((found != 0) && (coeffremain > 0.0)) {
				n_acceptable = (N - N_attained)/coeffremain;
				// The value to which we have to set the remaining
				// n values.
			};						
		} while (found != 0);
		// Now we should set the remaining values to n_acceptable
		// which is less than ndesire[i] in all those cases.
		for (i = 0; i < numCoords; i++)
		{
			if (fixed[i] == 0) n[i] = n_acceptable;
		};
		return n_C;
		
	} else {
		n_C = high_n;
		n_acceptable = (N - coeffcent*n_C)/(area - THIRD*area);
		do {
			found = 0;	
			coeffremain = 0.0;
			N_attained = coeffcent*high_n;
			for (i = 0; i < numCoords; i++)
			{
				if (fixed[i] == 0) {
					if (ndesire[i] > n_acceptable) {
						// yes, use ndesire[i] ...
						fixed[i] = true;
						n[i] = ndesire[i];
						N_attained += n[i]*coeff[i];
						found = true;
					} else {
						coeffremain += coeff[i];
					};
				} else {
					N_attained += n[i]*coeff[i];
				};
			};
			if ((found!= 0) && (coeffremain > 0.0)) {
				n_acceptable = (N - N_attained)/coeffremain;
			};						
		} while (found != 0);
		
		for (i = 0; i < numCoords; i++)
		{
			if (fixed[i] == 0) n[i] = n_acceptable;
		};
		return n_C;
	};
}

/*void TriMesh::Flip(AuxTriangle *pTri1, AuxTriangle * pTri2, int iLevel)
{

	int iClockwise, iAnticlockwise;
	AuxVertex * pVertex_unshared1, * pVertex_unshared2;
	int which, other_index_1, other_index_2;

	pVertex_unshared1 = pTri1->ReturnUnsharedVertex(pTri2, &which);
	pVertex_unshared2 = pTri2->ReturnUnsharedVertex(pTri1);
		
	if (which == 0) {
		other_index_1 = 1;
		other_index_2 = 2;
	} else {
		if (which == 1) {
			other_index_1 = 0; 
			other_index_2 = 2;
		} else {
			other_index_1 = 0;
			other_index_2 = 1;
		};
	};


	// Use iIndicator then wipe it clean.


	// First we set up periodic "scratch" data on the 4 vertices:
	if (pTri1->periodic == 0)
	{
		if (pTri2->periodic == 0)
		{
			pVertex_unshared1->iIndicator = 0;
			pVertex_unshared2->iIndicator = 0;
			pTri1->cornerptr[other_index_2]->iIndicator = 0;
			pTri1->cornerptr[other_index_1]->iIndicator = 0;
		} else {
			if (pTri2->periodic == 1)
			{
				// Should be the unshared point across PBC
				iClockwise = pTri2->GetLeftmostIndex();
				if (pTri2->cornerptr[iClockwise] != pVertex_unshared2)
				{
					// not possible
					printf("\n\n\nerror periodic :1: \n\n"); getch();
					pTri2 = pTri2;
				};
				pVertex_unshared1->iIndicator = 0;
				pVertex_unshared2->iIndicator = 1;
				pTri1->cornerptr[other_index_2]->iIndicator = 0;
				pTri1->cornerptr[other_index_1]->iIndicator = 0;
			} else {
				
				// pTri2->periodic == 2

				// Should be the two shared points across PBC
				iAnticlockwise = pTri2->GetRightmostIndex();
				if (pTri2->cornerptr[iAnticlockwise] != pVertex_unshared2)
				{
					// not possible
					printf("\n\n\nerror periodic\n\n"); getch();
					pTri2 = pTri2;
				};
				pVertex_unshared1->iIndicator = 1;
				pVertex_unshared2->iIndicator = 0;
				pTri1->cornerptr[other_index_2]->iIndicator = 1;
				pTri1->cornerptr[other_index_1]->iIndicator = 1;

			};				
		};
	} else {
		// use tri 1 to define the wrapping initially
		if (pTri1->periodic == 1)
		{
			// which one is across PBC?
			iClockwise = pTri1->GetLeftmostIndex();
			pTri1->cornerptr[0]->iIndicator = (iClockwise == 0)?1:0;
			pTri1->cornerptr[1]->iIndicator = (iClockwise == 1)?1:0;
			pTri1->cornerptr[2]->iIndicator = (iClockwise == 2)?1:0;

			if (pTri1->cornerptr[iClockwise] == pVertex_unshared1)
			{
				// the two shared points are not wrapped
				if (pTri2->periodic == 0) {
					pVertex_unshared2->iIndicator = 0;
				} else {
					if (pTri2->periodic == 1) {
						pVertex_unshared2->iIndicator = 1;
					} else {
						printf("\n\n\nerror periodic J\n\n"); getch();
						pTri2 = pTri2;
					};
				};
			} else {
				// one of the two shared points is wrapped
				if (pTri2->periodic == 0) { 
					printf("\n\n\nerror periodic KK\n\n"); getch();
					pTri2 = pTri2;
				};
				if (pTri2->periodic == 1) {
					// only the shared point was mapped
					pVertex_unshared2->iIndicator = 0;
				} else {
					pVertex_unshared2->iIndicator = 1;
				};
			};
		} else {
			// pTri1->periodic == 2
			iAnticlockwise = pTri1->GetRightmostIndex();
			pTri1->cornerptr[0]->iIndicator = (iAnticlockwise == 0)?0:1;
			pTri1->cornerptr[1]->iIndicator = (iAnticlockwise == 1)?0:1;
			pTri1->cornerptr[2]->iIndicator = (iAnticlockwise == 2)?0:1;
			
			if (pTri1->cornerptr[iAnticlockwise] == pVertex_unshared1)
			{
				// the two shared points are wrapped
				if (pTri2->periodic == 0) {
					pVertex_unshared2->iIndicator = 1;
				} else {
					if (pTri2->periodic == 1) {
						printf("\n\n\nerror periodic kljkl\n\n"); getch();
						pTri2 = pTri2;
					} else {
						pVertex_unshared2->iIndicator = 0;
					};
				};
			} else {
				// one of the two shared points is wrapped relative to the other
				if (pTri2->periodic == 0) {
					printf("\n\n\nerror periodic 223\n\n"); getch();
					pTri2 = pTri2;
				} else {
					if (pTri2->periodic == 1) {
						pVertex_unshared2->iIndicator = 0;
					} else {
						pVertex_unshared2->iIndicator = 1;
					};
				};					
			};
		};
	};


	// Now we need to tell the vertices affected about the triangles that overlap them:
	
	// First do Vertex::triangles
	// ==========================

	// For pVertex_unshared1, we now connect to the triangle of our pair that was not there before
	// For pVertex_unshared2, likewise
	
	pVertex_unshared1->addtri(pTri2-AuxT[iLevel]);
	pVertex_unshared2->addtri(pTri1-AuxT[iLevel]);

	// For old pTri1->vertices[other_index_2] we delete iTri1
	// For old pTri1->vertices[other_index_1] we delete iTri2
	
	pTri1->cornerptr[other_index_2]->remove_tri(pTri1-AuxT[iLevel]);  
	pTri1->cornerptr[other_index_1]->remove_tri(pTri2-AuxT[iLevel]);
	
	// Main assignment: Triangle::vertices,cornerptr
	// =============================================
	
	pTri2->cornerptr[0] = pVertex_unshared1;
	pTri2->cornerptr[1] = pVertex_unshared2;
	pTri2->cornerptr[2] = pTri1->cornerptr[other_index_2];
	
	pTri1->cornerptr[2] = pTri1->cornerptr[other_index_1];
	pTri1->cornerptr[0] = pVertex_unshared1;
	pTri1->cornerptr[1] = pVertex_unshared2;
	
	
	pTri1->periodic = pTri1->cornerptr[0]->iIndicator + 
					  pTri1->cornerptr[1]->iIndicator + 
					  pTri1->cornerptr[2]->iIndicator;
	while (pTri1->periodic >= 3) pTri1->periodic -= 3;
	pTri2->periodic = pTri2->cornerptr[0]->iIndicator + 
					  pTri2->cornerptr[1]->iIndicator + 
					  pTri2->cornerptr[2]->iIndicator;
	while (pTri2->periodic >= 3) pTri2->periodic -= 3;
	pTri1->cornerptr[0]->iIndicator = 0;
	pTri1->cornerptr[1]->iIndicator = 0;
	pTri1->cornerptr[2]->iIndicator = 0;
	pTri2->cornerptr[0]->iIndicator = 0;
	pTri2->cornerptr[1]->iIndicator = 0;
	pTri2->cornerptr[2]->iIndicator = 0;

	// Triangle::neighbours
	// =====================
	
	// six triangles affected in triangle-triangle case.
	// first get hold of them.
	smartvp neighs;
	AuxTriangle * pTri;
	
	neighs.add(pTri1->neighbours[0]);
	neighs.add(pTri1->neighbours[1]);
	neighs.add(pTri1->neighbours[2]);
	neighs.add(pTri2->neighbours[0]);
	neighs.add(pTri2->neighbours[1]);
	neighs.add(pTri2->neighbours[2]); 
	
	for (int i = 0; i < 6; i++)
	{
		pTri = (AuxTriangle *)neighs.ptr[i];				
		pTri->neighbours[2] = ReturnPointerToOtherSharedTriangleAux(pTri->cornerptr[0],pTri->cornerptr[1],pTri,iLevel);
		pTri->neighbours[0] = ReturnPointerToOtherSharedTriangleAux(pTri->cornerptr[1],pTri->cornerptr[2],pTri,iLevel);
		pTri->neighbours[1] = ReturnPointerToOtherSharedTriangleAux(pTri->cornerptr[0],pTri->cornerptr[2],pTri,iLevel);	
	};
	
}
*/


void TriMesh::Flip(Triangle * pTri1, Triangle * pTri2, int iLevel)
{
	// We alter the triangles:
	
	// Of the points they share, allocate one to each in the new setup; the remaining 2 points are now shared:
	// ===============================
	int which, other_index_1,other_index_2;
	int iClockwise,iAnticlockwise;
	int num_ins, num_out;
	int num_in_domain, BaseFlag;
	Triangle *pTri;
	Vertex * pVertex_unshared1, * pVertex_unshared2;

	if (iLevel == -1) {
		if (bGlobalAffectedTriArray) {
			GlobalAffectedTriIndexList.add(pTri1 - T);
			GlobalAffectedTriIndexList.add(pTri2 - T);
		};

	//	printf("iTri1 %d iTri2 %d \n",pTri1-T,pTri2-T);
	};

	pVertex_unshared1 = pTri1->ReturnUnsharedVertex(pTri2, &which);
	pVertex_unshared2 = pTri2->ReturnUnsharedVertex(pTri1);
	
	// New tri 1: pVertex_unshared1, pVertex_unshared2, another tri1 point
	// New tri 2: pVertex_unshared1, pVertex_unshared2, remaining tri1 point
	
	if (which == 0) {
		other_index_1 = 1;
		other_index_2 = 2;
	} else {
		if (which == 1) {
			other_index_1 = 0; 
			other_index_2 = 2;
		} else {
			other_index_1 = 0;
			other_index_2 = 1;
		};
	};

	// First we set up periodic "scratch" data on the 4 vertices:
	if (pTri1->periodic == 0)
	{
		if (pTri2->periodic == 0)
		{
			pVertex_unshared1->iScratch = 0;
			pVertex_unshared2->iScratch = 0;
			pTri1->cornerptr[other_index_2]->iScratch = 0;
			pTri1->cornerptr[other_index_1]->iScratch = 0;
		} else {
			if (pTri2->periodic == 1)
			{
				// Should be the unshared point across PBC
				iClockwise = pTri2->GetLeftmostIndex();
				if (pTri2->cornerptr[iClockwise] != pVertex_unshared2)
				{
					// not possible
					printf("\n\n\nerror periodic\n\n"); getch();
					pTri2 = pTri2;
				};
				pVertex_unshared1->iScratch = 0;
				pVertex_unshared2->iScratch = 1;
				pTri1->cornerptr[other_index_2]->iScratch = 0;
				pTri1->cornerptr[other_index_1]->iScratch = 0;
			} else {				
				// pTri2->periodic == 2

				// Should be the two shared points across PBC
				iAnticlockwise = pTri2->GetRightmostIndex();
				if (pTri2->cornerptr[iAnticlockwise] != pVertex_unshared2)
				{
					// not possible
					printf("\n\n\nerror periodic\n\n"); getch();
					pTri2 = pTri2;
				};
				pVertex_unshared1->iScratch = 1;
				pVertex_unshared2->iScratch = 0;
				pTri1->cornerptr[other_index_2]->iScratch = 1;
				pTri1->cornerptr[other_index_1]->iScratch = 1;
			};				
		};
	} else {
		// use tri 1 to define the wrapping initially
		if (pTri1->periodic == 1)
		{
			// which one is across PBC?
			iClockwise = pTri1->GetLeftmostIndex();
			pTri1->cornerptr[0]->iScratch = (iClockwise == 0)?1:0;
			pTri1->cornerptr[1]->iScratch = (iClockwise == 1)?1:0;
			pTri1->cornerptr[2]->iScratch = (iClockwise == 2)?1:0;

			if (pTri1->cornerptr[iClockwise] == pVertex_unshared1)
			{
				// the two shared points are not wrapped
				if (pTri2->periodic == 0) {
					pVertex_unshared2->iScratch = 0;
				} else {
					if (pTri2->periodic == 1) {
						pVertex_unshared2->iScratch = 1;
					} else {
						printf("\n\n\nerror periodic\n\n"); getch();
						pTri2 = pTri2;
					};
				};
			} else {
				// one of the two shared points is wrapped
				if (pTri2->periodic == 0) { 
					printf("\n\n\nerror periodic\n\n"); getch();
					pTri2 = pTri2;
				};
				if (pTri2->periodic == 1) {
					// only the shared point was mapped
					pVertex_unshared2->iScratch = 0;
				} else {
					pVertex_unshared2->iScratch = 1;
				};
			};
		} else {
			// pTri1->periodic == 2
			iAnticlockwise = pTri1->GetRightmostIndex();
			pTri1->cornerptr[0]->iScratch = (iAnticlockwise == 0)?0:1;
			pTri1->cornerptr[1]->iScratch = (iAnticlockwise == 1)?0:1;
			pTri1->cornerptr[2]->iScratch = (iAnticlockwise == 2)?0:1;
			
			if (pTri1->cornerptr[iAnticlockwise] == pVertex_unshared1)
			{
				// the two shared points are wrapped
				if (pTri2->periodic == 0) {
					pVertex_unshared2->iScratch = 1;
				} else {
					if (pTri2->periodic == 1) {
						printf("\n\n\nerror periodic\n\n"); getch();
						pTri2 = pTri2;
					} else {
						pVertex_unshared2->iScratch = 0;
					};
				};
			} else {
				// one of the two shared points is wrapped relative to the other
				if (pTri2->periodic == 0) {
					printf("\n\n\nerror periodic\n\n"); getch();

					//  We get here.
					

					pTri2 = pTri2;
				} else {
					if (pTri2->periodic == 1) {
						pVertex_unshared2->iScratch = 0;
					} else {
						pVertex_unshared2->iScratch = 1;
					};
				};					
			};
		};
	};


	// Now we need to tell the vertices affected about the triangles that overlap them:
	
	// First do Vertex::triangles
	// ==========================

	// For pVertex_unshared1, we now connect to the triangle of our pair that was not there before
	// For pVertex_unshared2, likewise
	
	if (iLevel == -1) {

		pVertex_unshared1->AddTriIndex(pTri2-T);
		pVertex_unshared2->AddTriIndex(pTri1-T);

		// For old pTri1->vertices[other_index_2] we delete iTri1
		// For old pTri1->vertices[other_index_1] we delete iTri2
	
		pTri1->cornerptr[other_index_2]->RemoveTriIndexIfExists(pTri1-T);  
		pTri1->cornerptr[other_index_1]->RemoveTriIndexIfExists(pTri2-T);
	
		// Vertices also want to know what vertices are nearby:
		// Vertex::neighbours
		// ===============================

		// Why this was commented? Perhaps it spoils something after ???
		// Is it right?

		// The shared ones are no longer mutually connected
		pTri1->cornerptr[other_index_2]->RemoveNeighIndexIfExists(pTri1->cornerptr[other_index_1]-X);
		pTri1->cornerptr[other_index_1]->RemoveNeighIndexIfExists(pTri1->cornerptr[other_index_2]-X);
		// The unshared ones now are connected
		pVertex_unshared1->AddNeighbourIndex(pVertex_unshared2-X);
		pVertex_unshared2->AddNeighbourIndex(pVertex_unshared1-X);
	} else {

		pVertex_unshared1->AddTriIndex(pTri2-AuxT[iLevel]);
		pVertex_unshared2->AddTriIndex(pTri1-AuxT[iLevel]);
		pTri1->cornerptr[other_index_2]->RemoveTriIndexIfExists(pTri1-AuxT[iLevel]);  
		pTri1->cornerptr[other_index_1]->RemoveTriIndexIfExists(pTri2-AuxT[iLevel]);
		
		// We do not play with neighbour lists for auxiliary: these document where coefficients apply.
	};

	// We have to use remove_if_exists because this func is sometimes called when
	// neighbour lists are NOT populated.
	// However, if it is called with them populated, we'd prefer to maintain them
	// -- although the ORDER OF THE VERTEX LIST IS NOT MAINTAINED HERE.
	// And neither is the ORDER OF THE VERTEX TRIANGLE LIST.


	// Main assignment: Triangle::vertices,cornerptr
	// =============================================
	
	//pTri2->vertices[0] = pVertex_unshared1->index;
	pTri2->cornerptr[0] = pVertex_unshared1;
	//pTri2->vertices[1] = pVertex_unshared2->index;
	pTri2->cornerptr[1] = pVertex_unshared2;
	//pTri2->vertices[2] = pTri1->vertices[other_index_2];
	pTri2->cornerptr[2] = pTri1->cornerptr[other_index_2];
	
	//pTri1->vertices[2] = pTri1->vertices[other_index_1];
	pTri1->cornerptr[2] = pTri1->cornerptr[other_index_1];
	//pTri1->vertices[0] = pVertex_unshared1->index;
	pTri1->cornerptr[0] = pVertex_unshared1;
	//pTri1->vertices[1] = pVertex_unshared2->index;
	pTri1->cornerptr[1] = pVertex_unshared2;

	pTri1->periodic = pTri1->cornerptr[0]->iScratch + 
					  pTri1->cornerptr[1]->iScratch + 
					  pTri1->cornerptr[2]->iScratch;
	while (pTri1->periodic >= 3) pTri1->periodic -= 3;
	pTri2->periodic = pTri2->cornerptr[0]->iScratch + 
					  pTri2->cornerptr[1]->iScratch + 
					  pTri2->cornerptr[2]->iScratch;
	while (pTri2->periodic >= 3) pTri2->periodic -= 3;
	
	if (iLevel == -1) {
		// Note that the periodic flag is used in the following:
		pTri1->RecalculateEdgeNormalVectors(false);
		pTri2->RecalculateEdgeNormalVectors(false);
	 // these are not used during Redelaunerize so for aux we can set them after.

		// Set new Triangle::flags:
		// We treat Vertex::flags as valid.

		if ((pTri1->u8domain_flag == 0) && (pTri2->u8domain_flag == 0))
		{
			// do nothing

			// This is the only case that ever actually applies for tri data.
			// For VERTBASED we affect the whole mesh.

		} else {
			if ((pTri1->u8domain_flag == OUT_OF_DOMAIN) && (pTri2->u8domain_flag == OUT_OF_DOMAIN))
			{
				// do nothing
			}  else {

				// This fixed 180116 :
				num_in_domain =	((pTri1->cornerptr[0]->flags == 0)?1:0) + 
								((pTri1->cornerptr[1]->flags == 0)?1:0) + 
								((pTri1->cornerptr[2]->flags == 0)?1:0); // 0 == plasma domain
				// Not counting edge vertices here -- 
				// unlikely that u8domain_flag is differing if near those.
				pTri1->u8domain_flag = CROSSING_INS;
				if (num_in_domain == 3) pTri1->u8domain_flag = DOMAIN_TRIANGLE;
				if (num_in_domain == 0) pTri1->u8domain_flag = OUT_OF_DOMAIN;
				num_in_domain =	((pTri2->cornerptr[0]->flags == 0)?1:0) + 
								((pTri2->cornerptr[1]->flags == 0)?1:0) + 
								((pTri2->cornerptr[2]->flags == 0)?1:0);
				// Not counting edge vertices here -- 
				// unlikely that u8domain_flag is not making it obvious if near those.
				pTri2->u8domain_flag = CROSSING_INS;
				if (num_in_domain == 3) pTri2->u8domain_flag = DOMAIN_TRIANGLE;
				if (num_in_domain == 0) pTri2->u8domain_flag = OUT_OF_DOMAIN;
		
			};
		};
	};

	// Triangle::neighbours: only handles neighbours of neighbours!! But perhaps these included.

	Triangle * neighs[6];

	neighs[0] = pTri1->neighbours[0];
	neighs[1] = pTri1->neighbours[1];
	neighs[2] = pTri1->neighbours[2];
	neighs[3] = pTri2->neighbours[0];
	neighs[4] = pTri2->neighbours[1];
	neighs[5] = pTri2->neighbours[2];
	
	for (int i = 0; i < 6; i++)
	{
		pTri = neighs[i];
		pTri->neighbours[2] = ReturnPointerToOtherSharedTriangle(pTri->cornerptr[0],pTri->cornerptr[1],pTri,iLevel);
		pTri->neighbours[0] = ReturnPointerToOtherSharedTriangle(pTri->cornerptr[1],pTri->cornerptr[2],pTri,iLevel);
		pTri->neighbours[1] = ReturnPointerToOtherSharedTriangle(pTri->cornerptr[0],pTri->cornerptr[2],pTri,iLevel);
	};

}


void TriMesh::DebugTestWrongNumberTrisPerEdge(void)
{
	Vertex * pVert, *pVert2;
	Triangle * pTri;
	int count,count2;
	smartlong neighs;
	long tri_len, izTri[128];

	for (long i = 0; i < numVertices; i++)
	{
		if (!(Disconnected.contains(i)))
		{
			pVert = X + i;
			// for each thing contained in neighbouring triangles...
			
			neighs.clear();

			tri_len = pVert->GetTriIndexArray(izTri);
			for (int ii = 0; ii < tri_len; ii++)
			{
				pTri = T + izTri[ii];

				for (int jj = 0; jj < 3; jj++)
					if (pTri->cornerptr[jj] != pVert)
						neighs.add_unique(pTri->cornerptr[jj]-X);
			};

			for (int iii = 0; iii < neighs.len; iii++)
			{
				pVert2 = X+neighs.ptr[iii];//MakePtr(neighs,iii);
				// count how many tris have pVert2:

				count = 0;
				for (int ii = 0; ii < tri_len; ii++)
				{
					pTri = T+izTri[ii];
					int jj = 0;
					while ((jj < 3) && (pTri->cornerptr[jj] != pVert2)) jj++;
					if (jj < 3) count++; 
				};
				if ((count != 2) && (count != 0))
				{
					if (count == 1) {
						// could be this edge is looking off the edge of the domain.
						if ((pVert->flags == pVert2->flags) && (pVert->flags != DOMAIN_VERTEX))
						{
							// give it a pass
						} else {

							printf("Value %d appears in %d tris at vertex %d. flags %d %d \n",neighs.ptr[iii],count,pVert-X,pVert->flags,pVert2->flags);
							count = count;
							getch();
						};
					} else {
						printf("Value %d appears in %d tris at vertex %d.\n",neighs.ptr[iii],count,pVert-X);

						count = count;
						getch();
					};
					count = count;
				};
			};
			
			
			// count how many tri's have a 0
			// should be exactly 2 or 0
			// same should hold for every other value.
			/*count = 0;
			count2 = 0;
			for (int ii = 0; ii < pVert->triangles.len; ii++)
			{
				pTri = (Triangle *)pVert->triangles.ptr[ii];
				if (pTri->cornerptr[2] == INS_VERT) count++;
				if (pTri->cornerptr[2] == HIGH_VERT) count2++;
			}
			if ((count != 0) && (count != 2))
			{
				printf("detected %d low wedges at one vertex.\n",count);
				getch();
				count = count;
			};
			if ((count2 != 0) && (count2 != 2))
			{
				printf("detected %d high wedges at one vertex.\n",count);
				getch();
				count = count;
			};
			*/
		};
	};
}
 

void TriMesh::ShiftVertexPositionsEquanimity()
{
	// Now we want to move each domain interior vertex
	// towards the centre of its polygon.
	// Only move if it is more than 25% of the sq root
	// of polygon area distant from the centre.
	// Do not move neighbours at the same time.

	Vertex * pVertex, *pNeigh, *pNeigh2;
	long iVertex, i, inext;
	ConvexPolygon cp;
	Vector2 u, u2, sum, avg, direction, cand, oldpos;
	real distsq, area, r, to_move, newdist, maxratio, dist, maxdist;
	bool bNo;
	long izNeighs[128];
	long neigh_len;

	real maxrat[3] = {0.38,0.3,0.24};

	for (int iLoop = 0; iLoop < 3; iLoop++)
	{
		maxratio = maxrat[iLoop];
		// make the comb finer over 3 loop repeats.

		pVertex = X;
		for (iVertex = 0; iVertex < numVertices; iVertex++)
		{
			pVertex->iIndicator = 0;
			++pVertex;
		};

		pVertex = X;
		for (iVertex = 0; iVertex < numVertices; iVertex++)
		{
			if (pVertex->flags == DOMAIN_VERTEX)
			{
				// check if any neighbours modified already:
				bNo = false;

				neigh_len = pVertex->GetNeighIndexArray(izNeighs);
				for (i = 0; i < neigh_len; i++)
				{
					pNeigh = X + izNeighs[i];
					if (pNeigh->iIndicator == 1) bNo = true;
				}

				if (bNo == false) {

					cp.Clear();
					sum.x = 0.0; sum.y = 0.0;
					for (i = 0; i < neigh_len; i++)
					{
						pNeigh = X + izNeighs[i];
						u.x = pNeigh->pos.x; 
						u.y = pNeigh->pos.y;
						if ((pVertex->pos.x/pVertex->pos.y < -0.5*GRADIENT_X_PER_Y) && (pNeigh->pos.x/pNeigh->pos.y > 0.5*GRADIENT_X_PER_Y))
							u = Anticlockwise*u;

						if ((pVertex->pos.x/pVertex->pos.y > 0.5*GRADIENT_X_PER_Y) && (pNeigh->pos.x/pNeigh->pos.y < -0.5*GRADIENT_X_PER_Y))
							u = Clockwise*u;
					
						cp.add(u);
						sum += u;
					};
					avg = sum/(real)neigh_len;
					distsq = (avg.x-pVertex->pos.x)*(avg.x-pVertex->pos.x)
						+ (avg.y-pVertex->pos.y)*(avg.y-pVertex->pos.y);
					
					area = cp.GetArea();

					// criterion: if dist > 0.2*sqrt(area)

					if (distsq > maxratio*maxratio*area) {
						dist = sqrt(distsq);
						maxdist = 0.18*sqrt(area);
						newdist = maxdist*dist/(dist + maxdist);
						// -> maxdist as dist -> infinity
						// -> dist as dist << maxdist
						
						to_move = dist-newdist;
						direction.x = avg.x-pVertex->pos.x;
						direction.y = avg.y-pVertex->pos.y;
						r = direction.modulus();
						direction.x /= r;
						direction.y /= r;
												
						cand.x = pVertex->pos.x + direction.x*to_move;
						cand.y = pVertex->pos.y + direction.y*to_move;
												
						// avoid wrapping:
						if ((cand.x/cand.y < -GRADIENT_X_PER_Y) || 
							(cand.x/cand.y > GRADIENT_X_PER_Y))
						{
							// do nothing
						} else {

							oldpos.x = pVertex->pos.x;
							oldpos.y = pVertex->pos.y;
							pVertex->pos.x = cand.x;
							pVertex->pos.y = cand.y;
							pVertex->iIndicator = 1;
									
							for (int j = 0; j < neigh_len; j++)
							{
								inext = j+1; if (inext == neigh_len) inext = 0;
								pNeigh = X + izNeighs[j];
								if (pNeigh->iIndicator == 1) bNo = true;
								pNeigh2 = X + izNeighs[inext];

								u.x = pNeigh->pos.x; 
								u.y = pNeigh->pos.y;
								if ((pVertex->pos.x/pVertex->pos.y < -0.5*GRADIENT_X_PER_Y) && (pNeigh->pos.x/pNeigh->pos.y > 0.5*GRADIENT_X_PER_Y))
									u = Anticlockwise*u;
								if ((pVertex->pos.x/pVertex->pos.y > 0.5*GRADIENT_X_PER_Y) && (pNeigh->pos.x/pNeigh->pos.y < -0.5*GRADIENT_X_PER_Y))
									u = Clockwise*u;
						
								u2.x = pNeigh->pos.x; 
								u2.y = pNeigh->pos.y;
								if ((pVertex->pos.x/pVertex->pos.y < -0.5*GRADIENT_X_PER_Y) && (pNeigh2->pos.x/pNeigh2->pos.y > 0.5*GRADIENT_X_PER_Y))
									u2 = Anticlockwise*u2;
								if ((pVertex->pos.x/pVertex->pos.y > 0.5*GRADIENT_X_PER_Y) && (pNeigh2->pos.x/pNeigh2->pos.y < -0.5*GRADIENT_X_PER_Y))
									u2 = Clockwise*u2;
							
								// now rotate u-u2 90 degrees

								direction.x = u2.y-u.y;
								direction.y = u.x-u2.x;

								// is dot product with vector to new pos same sign as to old pos?

								if (
									(direction.x*(pVertex->pos.x-u.x) + direction.y*(pVertex->pos.y-u.y))*(direction.dot(oldpos-u)) < 0.0    )
								{
									printf("\n\nalert");
									getch();
									j = j;
								};
							}; // next j

						};
					};

				}; // whether neighbour modified
			};
			++pVertex;
		};
	};
	
	// clean up indicator
	pVertex = X;
	for (iVertex = 0; iVertex < numVertices; iVertex++)
	{
		pVertex->iIndicator = 0;
		++pVertex;
	};

	this->DebugTestWrongNumberTrisPerEdge();
}